#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, double * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  double * __tilevar_2__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double * __tilevar_3__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double * __tilevar_4__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double * __tilevar_5__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);

  double t2_0=0.0f, t2_1=0.0f, t3_0=0.0f, t3_1=0.0f, t4_0=0.0f, t4_1=0.0f, t5_0=0.0f, t5_1=0.0f;
  double b2_0=0.0f, b2_1=0.0f, b3_0=0.0f, b3_1=0.0f, b4_0=0.0f, b4_1=0.0f, b5_0=0.0f, b5_1=0.0f;
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X-16);
  int __iter_y__ = FORMA_MAX((int)(blockIdx.y)*FORMA_BLOCKDIM_Y-8, 0);

  //Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){ 
    b2_0 = input[__iter_3__+M*__iter_y__];
    __tilevar_2__[__iter_3__-__iter_0__] = input[__iter_3__+M*(__iter_y__+1)];
    t2_0 = input[__iter_3__+M*(__iter_y__+2)];
    t2_1 = input[__iter_3__+M*(__iter_y__+3)];
  }
  // Initial computation
  for (int __iter_1__ = __iter_y__+2; __iter_1__ < FORMA_MIN(N-2,__iter_y__+14); __iter_1__++) {
    if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
	b2_1 = b2_0;
      	b2_0 = __tilevar_2__[__iter_3__-__iter_0__]; 
      	__tilevar_2__[__iter_3__-__iter_0__] = t2_0;
      	t2_0 = t2_1;
	t2_1 = input[__iter_3__+M*(__iter_1__+2)];
    }
    __syncthreads ();
    if (__iter_3__ >= FORMA_MAX((__iter_0__+2),2) &  __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-3)) ){
        double __temp_a__ = b2_1;
        double __temp_2__ = b2_0;
        double __temp_b__ = (7 * __temp_a__ + 5 * __temp_2__);
        double __temp_c__ = (__tilevar_2__[__iter_3__-2-__iter_0__]);
        double __temp_d__ = (__temp_b__ + 9 * __temp_c__);
        double __temp_5__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (__temp_d__ + 12 * __temp_5__);
        double __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 15 * __temp_9__);
        double __temp_13__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        double __temp_e__ =  (__tilevar_2__[__iter_3__+2-__iter_0__]);
        double __temp_f__ = (__temp_14__ + 9 * __temp_e__);
        double __temp_17__ = t2_0;
        double __temp_18__ = (__temp_f__ + 5 * __temp_17__);
        double __temp_g__ = t2_1;
        double __temp_h__ = (__temp_18__ + 7 * __temp_g__);
        double __temp_19__ = (__temp_h__ / 118);
	b3_1 = b3_0;
	b3_0 = __tilevar_3__[__iter_3__-__iter_0__];   
        __tilevar_3__[__iter_3__-__iter_0__] = t3_0;
	t3_0 = t3_1;
	t3_1 = __temp_19__;
    }
    __syncthreads();
    if (__iter_3__ >= FORMA_MAX((__iter_0__+4),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-3))) {
        double __temp_a__ = b3_1;
        double __temp_2__ = b3_0;
        double __temp_b__ = (7 * __temp_a__ + 5 * __temp_2__);
        double __temp_c__ = (__tilevar_3__[__iter_3__-2-__iter_0__]);
        double __temp_d__ = (__temp_b__ + 9 * __temp_c__);
        double __temp_5__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (__temp_d__ + 12 * __temp_5__);
        double __temp_9__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 15 * __temp_9__);
        double __temp_13__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        double __temp_e__ =  (__tilevar_3__[__iter_3__+2-__iter_0__]);
        double __temp_f__ = (__temp_14__ + 9 * __temp_e__);
        double __temp_17__ = t3_0;
        double __temp_18__ = (__temp_f__ + 5 * __temp_17__);
        double __temp_g__ = t3_1;
        double __temp_h__ = (__temp_18__ + 7 * __temp_g__);
        double __temp_19__ = (__temp_h__ / 118);
	b4_1 = b4_0;
	b4_0 = __tilevar_4__[__iter_3__-__iter_0__];   
        __tilevar_4__[__iter_3__-__iter_0__] = t4_0;
	t4_0 = t4_1;
	t4_1 = __temp_19__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+6),2) &  __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(M-3))) {
        double __temp_a__ = b4_1;
        double __temp_2__ = b4_0;
        double __temp_b__ = (7 * __temp_a__ + 5 * __temp_2__);
        double __temp_c__ = (__tilevar_4__[__iter_3__-2-__iter_0__]);
        double __temp_d__ = (__temp_b__ + 9 * __temp_c__);
        double __temp_5__ = (__tilevar_4__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (__temp_d__ + 12 * __temp_5__);
        double __temp_9__ = (__tilevar_4__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 15 * __temp_9__);
        double __temp_13__ = (__tilevar_4__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        double __temp_e__ =  (__tilevar_4__[__iter_3__+2-__iter_0__]);
        double __temp_f__ = (__temp_14__ + 9 * __temp_e__);
        double __temp_17__ = t4_0;
        double __temp_18__ = (__temp_f__ + 5 * __temp_17__);
        double __temp_g__ = t4_1;
        double __temp_h__ = (__temp_18__ + 7 * __temp_g__);
        double __temp_19__ = (__temp_h__ / 118);
	b5_1 = b5_0;
	b5_0 = __tilevar_5__[__iter_3__-__iter_0__];   
        __tilevar_5__[__iter_3__-__iter_0__] = t5_0;
	t5_0 = t5_1;
	t5_1 = __temp_19__;
    }
  }
  // Rest of the computation
  for (int __iter_1__ = FORMA_MAX(2,__iter_y__+14); __iter_1__ < FORMA_MIN(N-2,__iter_y__+FORMA_BLOCKDIM_Y+14); __iter_1__++) {
    if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
	b2_1 = b2_0;
      	b2_0 = __tilevar_2__[__iter_3__-__iter_0__]; 
      	__tilevar_2__[__iter_3__-__iter_0__] = t2_0;
      	t2_0 = t2_1;
	t2_1 = input[__iter_3__+M*(__iter_1__+2)];
    }
    __syncthreads ();
    if (__iter_3__ >= FORMA_MAX((__iter_0__+2),2) &  __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-3)) ){
        double __temp_a__ = b2_1;
        double __temp_2__ = b2_0;
        double __temp_b__ = (7 * __temp_a__ + 5 * __temp_2__);
        double __temp_c__ = (__tilevar_2__[__iter_3__-2-__iter_0__]);
        double __temp_d__ = (__temp_b__ + 9 * __temp_c__);
        double __temp_5__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (__temp_d__ + 12 * __temp_5__);
        double __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 15 * __temp_9__);
        double __temp_13__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        double __temp_e__ =  (__tilevar_2__[__iter_3__+2-__iter_0__]);
        double __temp_f__ = (__temp_14__ + 9 * __temp_e__);
        double __temp_17__ = t2_0;
        double __temp_18__ = (__temp_f__ + 5 * __temp_17__);
        double __temp_g__ = t2_1;
        double __temp_h__ = (__temp_18__ + 7 * __temp_g__);
        double __temp_19__ = (__temp_h__ / 118);
	b3_1 = b3_0;
	b3_0 = __tilevar_3__[__iter_3__-__iter_0__];   
        __tilevar_3__[__iter_3__-__iter_0__] = t3_0;
	t3_0 = t3_1;
	t3_1 = __temp_19__;
    }
    __syncthreads();
    if (__iter_3__ >= FORMA_MAX((__iter_0__+4),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-3))) {
        double __temp_a__ = b3_1;
        double __temp_2__ = b3_0;
        double __temp_b__ = (7 * __temp_a__ + 5 * __temp_2__);
        double __temp_c__ = (__tilevar_3__[__iter_3__-2-__iter_0__]);
        double __temp_d__ = (__temp_b__ + 9 * __temp_c__);
        double __temp_5__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (__temp_d__ + 12 * __temp_5__);
        double __temp_9__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 15 * __temp_9__);
        double __temp_13__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        double __temp_e__ =  (__tilevar_3__[__iter_3__+2-__iter_0__]);
        double __temp_f__ = (__temp_14__ + 9 * __temp_e__);
        double __temp_17__ = t3_0;
        double __temp_18__ = (__temp_f__ + 5 * __temp_17__);
        double __temp_g__ = t3_1;
        double __temp_h__ = (__temp_18__ + 7 * __temp_g__);
        double __temp_19__ = (__temp_h__ / 118);
	b4_1 = b4_0;
	b4_0 = __tilevar_4__[__iter_3__-__iter_0__];   
        __tilevar_4__[__iter_3__-__iter_0__] = t4_0;
	t4_0 = t4_1;
	t4_1 = __temp_19__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+6),2) &  __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(M-3))) {
        double __temp_a__ = b4_1;
        double __temp_2__ = b4_0;
        double __temp_b__ = (7 * __temp_a__ + 5 * __temp_2__);
        double __temp_c__ = (__tilevar_4__[__iter_3__-2-__iter_0__]);
        double __temp_d__ = (__temp_b__ + 9 * __temp_c__);
        double __temp_5__ = (__tilevar_4__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (__temp_d__ + 12 * __temp_5__);
        double __temp_9__ = (__tilevar_4__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 15 * __temp_9__);
        double __temp_13__ = (__tilevar_4__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        double __temp_e__ =  (__tilevar_4__[__iter_3__+2-__iter_0__]);
        double __temp_f__ = (__temp_14__ + 9 * __temp_e__);
        double __temp_17__ = t4_0;
        double __temp_18__ = (__temp_f__ + 5 * __temp_17__);
        double __temp_g__ = t4_1;
        double __temp_h__ = (__temp_18__ + 7 * __temp_g__);
        double __temp_19__ = (__temp_h__ / 118);
	b5_1 = b5_0;
	b5_0 = __tilevar_5__[__iter_3__-__iter_0__];   
        __tilevar_5__[__iter_3__-__iter_0__] = t5_0;
	t5_0 = t5_1;
	t5_1 = __temp_19__;
    }
    __syncthreads();
    if (__iter_3__ >= FORMA_MAX((__iter_0__+8),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-9),(M-3))){
	double __temp_a__ = b5_1;
        double __temp_2__ = b5_0;
        double __temp_b__ = (7 * __temp_a__ + 5 * __temp_2__);
        double __temp_c__ = (__tilevar_5__[__iter_3__-2-__iter_0__]);
        double __temp_d__ = (__temp_b__ + 9 * __temp_c__);
        double __temp_5__ = (__tilevar_5__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (__temp_d__ + 12 * __temp_5__);
        double __temp_9__ = (__tilevar_5__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 15 * __temp_9__);
        double __temp_13__ = (__tilevar_5__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        double __temp_e__ =  (__tilevar_5__[__iter_3__+2-__iter_0__]);
        double __temp_f__ = (__temp_14__ + 9 * __temp_e__);
        double __temp_17__ = t5_0;
        double __temp_18__ = (__temp_f__ + 5 * __temp_17__);
        double __temp_g__ = t5_1;
        double __temp_h__ = (__temp_18__ + 7 * __temp_g__);
        double __temp_19__ = (__temp_h__ / 118);
        __var_1__[__iter_3__+M*FORMA_MAX(__iter_1__-6,0)] = __temp_19__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(double)*(4*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(double * h_input, int N, int M, double * __var_0__){

/* Host allocation Begin */
  double * input;
  hipMalloc(&input,sizeof(double)*(N*M));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(double)*(N*M), memcpy_kind_h_input);
  }
  double * __var_1__;
  hipMalloc(&__var_1__,sizeof(double)*(N*M));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = M;
  int __size_1___kernel___forma_kernel__0__ = N;
  int __block_0___kernel___forma_kernel__0__ = 128;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-16);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__, __size_1___kernel___forma_kernel__0__/64);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __size_1___kernel___forma_kernel__0__/64, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(double)*(N*M), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

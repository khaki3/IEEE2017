#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"
#include <nvml.h>
#include <assert.h>


#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
	template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
	int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
	input[loc] = value;
}


	template<typename T>
void initialize_array(T* d_input, int size, T value)
{
	dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
	dim3 init_block(FORMA_MAX_BLOCKDIM_0);
	__kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(struct double4 * __restrict__ input, double dx, double dy, double dz, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, struct double4 * __restrict__ __var_1__){
	int __FORMA_SHARED_MEM_OFFSET__ = 0;
	struct double4* __tilevar_0__ = (struct double4*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
	__FORMA_SHARED_MEM_OFFSET__ += sizeof(struct double4)*((FORMA_BLOCKDIM_Z*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));

	double b_x = 0.0f, b_y = 0.0f;

	int __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X-2);
	int __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y-2);
	int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
	int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;

#pragma unroll 4 
	for (int __iter_2__ = 0; __iter_2__ < L-1; __iter_2__++) {
		if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),M-2) ){
			if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),N-2) ){
				double __temp_0__ = (input[__iter_5__+N*(__iter_4__+M*(__iter_2__+1))].y - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].y);
				double __temp_1__ = (dz * __temp_0__);
				double __temp_2__ = (input[__iter_5__+N*(__iter_4__+1+M*__iter_2__)].z - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].z);
				double __temp_3__ = (dy * __temp_2__);
				double __temp_4__ = (__temp_1__ - __temp_3__);
				double __temp_5__ = (input[__iter_5__+N*(__iter_4__+M*(__iter_2__+1))].x - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].x);
				double __temp_6__ = (dz * __temp_5__);
				double __temp_7__ = (input[__iter_5__+1+N*(__iter_4__+M*__iter_2__)].z - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].z);
				double __temp_8__ = (dx * __temp_7__);
				double __temp_9__ = (__temp_6__ - __temp_8__);
				double __temp_10__ = (input[__iter_5__+N*(__iter_4__+1+M*__iter_2__)].x - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].x);
				double __temp_11__ = (dy * __temp_10__);
				double __temp_12__ = (input[__iter_5__+1+N*(__iter_4__+M*__iter_2__)].y - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].y);
				double __temp_13__ = (dx * __temp_12__);
				double __temp_14__ = (__temp_11__ - __temp_13__);
				double __temp_15__ = (__temp_4__ + input[__iter_5__+N*(__iter_4__+M*__iter_2__)].x);
				double __temp_16__ = (__temp_9__ + input[__iter_5__+N*(__iter_4__+M*__iter_2__)].y);
				double __temp_17__ = (__temp_14__ + input[__iter_5__+N*(__iter_4__+M*__iter_2__)].z);
				b_x = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].x;
				b_y = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y;
				__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].x = __temp_15__;
				__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y = __temp_16__;
				__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].z = __temp_17__;
				__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].w = 0.0f;
			}
		}
		__syncthreads();
		if(__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-1))){
			if(__iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-1))){
				double __temp_36__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y - b_y);
				double __temp_37__ = (dz * __temp_36__);
				double __temp_38__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].z - __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)].z);
				double __temp_39__ = (dy * __temp_38__);
				double __temp_40__ = (__temp_37__ - __temp_39__);
				double __temp_41__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].x - b_x);
				double __temp_42__ = (dz * __temp_41__);
				double __temp_43__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].z - __tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].z);
				double __temp_44__ = (dx * __temp_43__);
				double __temp_45__ = (__temp_42__ - __temp_44__);
				double __temp_46__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].x - __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)].x);
				double __temp_47__ = (dy * __temp_46__);
				double __temp_48__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y - __tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y);
				double __temp_49__ = (dx * __temp_48__);
				double __temp_50__ = (__temp_47__ - __temp_49__);
				double __temp_51__ = (__temp_40__ + __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].x);
				double __temp_52__ = (__temp_45__ + __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y);
				double __temp_53__ = (__temp_50__ + __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].z);
				__var_1__[__iter_5__+N*(__iter_4__+M*__iter_2__)].x = __temp_51__;
				__var_1__[__iter_5__+N*(__iter_4__+M*__iter_2__)].y = __temp_52__;
				__var_1__[__iter_5__+N*(__iter_4__+M*__iter_2__)].z = __temp_53__;
				__var_1__[__iter_5__+N*(__iter_4__+M*__iter_2__)].w = 0.0f;
			}
		}
		__syncthreads ();
	}
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
	int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int SMemSize = 0;
	SMemSize += sizeof(struct double4)*((FORMA_BLOCKDIM_Z*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
	return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void curl(struct double4 * h_input, double dx, double dy, double dz, int L, int M, int N, struct double4 * __var_0__){

	/* Host allocation Begin */
	struct double4 * input;
	hipMalloc(&input,sizeof(struct double4)*(L*M*N));
	Check_CUDA_Error("Allocation Error!! : input\n");
	hipPointerAttribute_t ptrAttrib_h_input;
	hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
	if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
		if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
			memcpy_kind_h_input = hipMemcpyDeviceToDevice;
	hipGetLastError();
	if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
		hipMemcpy(input,h_input,sizeof(struct double4)*(L*M*N), memcpy_kind_h_input);
	}
	struct double4 * __var_1__;
	hipMalloc(&__var_1__,sizeof(struct double4)*(L*M*N));
	Check_CUDA_Error("Allocation Error!! : __var_1__\n");
	/*Host Allocation End */
	/* Kernel Launch Begin */
	int __FORMA_MAX_SHARED_MEM__;
	hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
	hipEvent_t _forma_timer_start_,_forma_timer_stop_;
	hipEventCreate(&_forma_timer_start_);
	hipEventCreate(&_forma_timer_stop_);
	hipEventRecord(_forma_timer_start_,0);
#endif
	int __size_0___kernel___forma_kernel__0__ = N;
	int __size_1___kernel___forma_kernel__0__ = M;
	int __block_0___kernel___forma_kernel__0__ = 32;
	int __block_1___kernel___forma_kernel__0__ = 16;
	int __block_2___kernel___forma_kernel__0__ = 1;
	dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
	int __SMemSize___kernel___forma_kernel__0__ = 0;
	__SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
	int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-2);
	int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y-2);
	int __grid_2___kernel___forma_kernel__0__ = 1;
	dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<500; x++) {
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, dx, dy, dz, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();

	Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

	hipPointerAttribute_t ptrAttrib___var_0__;
	hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
	if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
		if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
			memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
	hipGetLastError();
	hipMemcpy(__var_0__,__var_1__, sizeof(struct double4)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
	hipEventRecord(_forma_timer_stop_,0);
	hipEventSynchronize(_forma_timer_stop_);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
	printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
	hipEventDestroy(_forma_timer_start_);
	hipEventDestroy(_forma_timer_stop_);
#endif
	/*Kernel Launch End */
	/* Host Free Begin */
	hipFree(input);
	hipFree(__var_1__);
}
/*Host Free End*/

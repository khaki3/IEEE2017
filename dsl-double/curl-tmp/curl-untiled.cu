#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(struct double4 * __restrict__ input, double dx, double dy, double dz, int L, int M, int N, struct double4 * __restrict__ __var_2__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 0;
  if(__iter_0__ <= (N-2)){
    int __iter_1__;
    __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 0;
    if(__iter_1__ <= (M-2)){
      int __iter_2__;
      __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 0;
      if(__iter_2__ <= (L-2)){
        double __temp_0__;
        __temp_0__ = (input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__+(1)))].y - input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].y);
        double __temp_1__;
        __temp_1__ = (dz * __temp_0__);
        double __temp_2__;
        __temp_2__ = (input[__iter_0__+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__))].z - input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].z);
        double __temp_3__;
        __temp_3__ = (dy * __temp_2__);
        double __temp_4__;
        __temp_4__ = (__temp_1__ - __temp_3__);
        double __temp_5__;
        __temp_5__ = (input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__+(1)))].x - input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].x);
        double __temp_6__;
        __temp_6__ = (dz * __temp_5__);
        double __temp_7__;
        __temp_7__ = (input[__iter_0__+(1)+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].z - input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].z);
        double __temp_8__;
        __temp_8__ = (dx * __temp_7__);
        double __temp_9__;
        __temp_9__ = (__temp_6__ - __temp_8__);
        double __temp_10__;
        __temp_10__ = (input[__iter_0__+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__))].x - input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].x);
        double __temp_11__;
        __temp_11__ = (dy * __temp_10__);
        double __temp_12__;
        __temp_12__ = (input[__iter_0__+(1)+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].y - input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].y);
        double __temp_13__;
        __temp_13__ = (dx * __temp_12__);
        double __temp_14__;
        __temp_14__ = (__temp_11__ - __temp_13__);
        double __temp_15__;
        __temp_15__ = (__temp_4__ + input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].x);
        double __temp_16__;
        __temp_16__ = (__temp_9__ + input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].y);
        double __temp_17__;
        __temp_17__ = (__temp_14__ + input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].z);
        __var_2__[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].x = __temp_15__;
        __var_2__[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].y = __temp_16__;
        __var_2__[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].z = __temp_17__;
        __var_2__[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))].w = 0.000000f;
      }
    }
  }
}
__global__ void __kernel___forma_kernel__1__(struct double4 * __restrict__ __var_2__, double dx, double dy, double dz, int L, int M, int N, struct double4 * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_3__;
  __iter_3__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_3__ <= (N-1)){
    int __iter_4__;
    __iter_4__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_4__ <= (M-1)){
      int __iter_5__;
      __iter_5__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_5__ <= (L-1)){
        double __temp_18__;
        __temp_18__ = (__var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].y - __var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__+(-1)))].y);
        double __temp_19__;
        __temp_19__ = (dz * __temp_18__);
        double __temp_20__;
        __temp_20__ = (__var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].z - __var_2__[__iter_3__+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__))].z);
        double __temp_21__;
        __temp_21__ = (dy * __temp_20__);
        double __temp_22__;
        __temp_22__ = (__temp_19__ - __temp_21__);
        double __temp_23__;
        __temp_23__ = (__var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].x - __var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__+(-1)))].x);
        double __temp_24__;
        __temp_24__ = (dz * __temp_23__);
        double __temp_25__;
        __temp_25__ = (__var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].z - __var_2__[__iter_3__+(-1)+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].z);
        double __temp_26__;
        __temp_26__ = (dx * __temp_25__);
        double __temp_27__;
        __temp_27__ = (__temp_24__ - __temp_26__);
        double __temp_28__;
        __temp_28__ = (__var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].x - __var_2__[__iter_3__+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__))].x);
        double __temp_29__;
        __temp_29__ = (dy * __temp_28__);
        double __temp_30__;
        __temp_30__ = (__var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].y - __var_2__[__iter_3__+(-1)+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].y);
        double __temp_31__;
        __temp_31__ = (dx * __temp_30__);
        double __temp_32__;
        __temp_32__ = (__temp_29__ - __temp_31__);
        double __temp_33__;
        __temp_33__ = (__temp_22__ + __var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].x);
        double __temp_34__;
        __temp_34__ = (__temp_27__ + __var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].y);
        double __temp_35__;
        __temp_35__ = (__temp_32__ + __var_2__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].z);
        __var_1__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].x = __temp_33__;
        __var_1__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].y = __temp_34__;
        __var_1__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].z = __temp_35__;
        __var_1__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))].w = 0.000000f;
      }
    }
  }
}
/*Device code End */
/* Host Code Begin */
extern "C" void curl(struct double4 * h_input, double dx, double dy, double dz, int L, int M, int N, struct double4 * __var_0__){

/* Host allocation Begin */
  struct double4 * input;
  hipMalloc(&input,sizeof(struct double4)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(struct double4)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
  }
  struct double4 * __var_1__;
  hipMalloc(&__var_1__,sizeof(struct double4)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  struct double4 * __var_2__;
  hipMalloc(&__var_2__,sizeof(struct double4)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((N-2) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((M-2) - 0 ) + 1;
  int __size_2___kernel___forma_kernel__0__ = ((L-2) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 16;
  int __block_1___kernel___forma_kernel__0__ = 4;
  int __block_2___kernel___forma_kernel__0__ = 4;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, dx, dy, dz, L, M, N, __var_2__);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, dx, dy, dz, L, M, N, __var_1__);


  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(struct double4)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

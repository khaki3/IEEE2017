#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-6))*(FORMA_BLOCKDIM_X-(-6))));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 1;
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 1;
  int __iter_2__;
  __iter_2__ = (FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)+(-1)) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= (FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2))+1),(N-2))+1),(N-2))+1) ){
    int __iter_3__;
    __iter_3__ = (FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)+(-1)) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= (FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2))+1),(M-2))+1),(M-2))+1) ){
      int __temp_0__;
      __temp_0__ = __iter_2__;
      int __temp_1__;
      __temp_1__ = __iter_3__;
      __tilevar_3__[__iter_3__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__iter_2__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] = input[__temp_1__+(M-0)*(__temp_0__)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2))+1),(N-2))+1),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2))+1),(M-2))+1),(M-2)) ){
      int __temp_2__;
      __temp_2__ = __iter_4__+(-1);
      int __temp_3__;
      __temp_3__ = __iter_5__+(-1);
      float __temp_4__;
      __temp_4__ = (7 * __tilevar_3__[__temp_3__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_2__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      int __temp_5__;
      __temp_5__ = __iter_4__+(-1);
      int __temp_6__;
      __temp_6__ = __iter_5__;
      float __temp_7__;
      __temp_7__ = (5 * __tilevar_3__[__temp_6__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_5__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_8__;
      __temp_8__ = (__temp_4__ + __temp_7__);
      int __temp_9__;
      __temp_9__ = __iter_4__+(-1);
      int __temp_10__;
      __temp_10__ = __iter_5__+(1);
      float __temp_11__;
      __temp_11__ = (9 * __tilevar_3__[__temp_10__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_9__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_12__;
      __temp_12__ = (__temp_8__ + __temp_11__);
      int __temp_13__;
      __temp_13__ = __iter_4__;
      int __temp_14__;
      __temp_14__ = __iter_5__+(-1);
      float __temp_15__;
      __temp_15__ = (12 * __tilevar_3__[__temp_14__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_13__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_16__;
      __temp_16__ = (__temp_12__ + __temp_15__);
      int __temp_17__;
      __temp_17__ = __iter_4__;
      int __temp_18__;
      __temp_18__ = __iter_5__;
      float __temp_19__;
      __temp_19__ = (15 * __tilevar_3__[__temp_18__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_17__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_20__;
      __temp_20__ = (__temp_16__ + __temp_19__);
      int __temp_21__;
      __temp_21__ = __iter_4__;
      int __temp_22__;
      __temp_22__ = __iter_5__+(1);
      float __temp_23__;
      __temp_23__ = (12 * __tilevar_3__[__temp_22__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_21__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_24__;
      __temp_24__ = (__temp_20__ + __temp_23__);
      int __temp_25__;
      __temp_25__ = __iter_4__+(1);
      int __temp_26__;
      __temp_26__ = __iter_5__+(-1);
      float __temp_27__;
      __temp_27__ = (9 * __tilevar_3__[__temp_26__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_25__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_28__;
      __temp_28__ = (__temp_24__ + __temp_27__);
      int __temp_29__;
      __temp_29__ = __iter_4__+(1);
      int __temp_30__;
      __temp_30__ = __iter_5__;
      float __temp_31__;
      __temp_31__ = (5 * __tilevar_3__[__temp_30__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_29__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_32__;
      __temp_32__ = (__temp_28__ + __temp_31__);
      int __temp_33__;
      __temp_33__ = __iter_4__+(1);
      int __temp_34__;
      __temp_34__ = __iter_5__+(1);
      float __temp_35__;
      __temp_35__ = (7 * __tilevar_3__[__temp_34__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_33__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_36__;
      __temp_36__ = (__temp_32__ + __temp_35__);
      float __temp_37__;
      __temp_37__ = (__temp_36__ / 118);
      __tilevar_2__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] = __temp_37__;
    }
  }
  __syncthreads();
  int __iter_6__;
  __iter_6__ = FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2))+1),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2))+1),(M-2)) ){
      int __temp_38__;
      __temp_38__ = __iter_6__+(-1);
      int __temp_39__;
      __temp_39__ = __iter_7__+(-1);
      float __temp_40__;
      __temp_40__ = (7 * __tilevar_2__[__temp_39__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_38__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      int __temp_41__;
      __temp_41__ = __iter_6__+(-1);
      int __temp_42__;
      __temp_42__ = __iter_7__;
      float __temp_43__;
      __temp_43__ = (5 * __tilevar_2__[__temp_42__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_41__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_44__;
      __temp_44__ = (__temp_40__ + __temp_43__);
      int __temp_45__;
      __temp_45__ = __iter_6__+(-1);
      int __temp_46__;
      __temp_46__ = __iter_7__+(1);
      float __temp_47__;
      __temp_47__ = (9 * __tilevar_2__[__temp_46__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_45__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_48__;
      __temp_48__ = (__temp_44__ + __temp_47__);
      int __temp_49__;
      __temp_49__ = __iter_6__;
      int __temp_50__;
      __temp_50__ = __iter_7__+(-1);
      float __temp_51__;
      __temp_51__ = (12 * __tilevar_2__[__temp_50__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_49__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_52__;
      __temp_52__ = (__temp_48__ + __temp_51__);
      int __temp_53__;
      __temp_53__ = __iter_6__;
      int __temp_54__;
      __temp_54__ = __iter_7__;
      float __temp_55__;
      __temp_55__ = (15 * __tilevar_2__[__temp_54__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_53__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_56__;
      __temp_56__ = (__temp_52__ + __temp_55__);
      int __temp_57__;
      __temp_57__ = __iter_6__;
      int __temp_58__;
      __temp_58__ = __iter_7__+(1);
      float __temp_59__;
      __temp_59__ = (12 * __tilevar_2__[__temp_58__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_57__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_60__;
      __temp_60__ = (__temp_56__ + __temp_59__);
      int __temp_61__;
      __temp_61__ = __iter_6__+(1);
      int __temp_62__;
      __temp_62__ = __iter_7__+(-1);
      float __temp_63__;
      __temp_63__ = (9 * __tilevar_2__[__temp_62__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_61__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_64__;
      __temp_64__ = (__temp_60__ + __temp_63__);
      int __temp_65__;
      __temp_65__ = __iter_6__+(1);
      int __temp_66__;
      __temp_66__ = __iter_7__;
      float __temp_67__;
      __temp_67__ = (5 * __tilevar_2__[__temp_66__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_65__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_68__;
      __temp_68__ = (__temp_64__ + __temp_67__);
      int __temp_69__;
      __temp_69__ = __iter_6__+(1);
      int __temp_70__;
      __temp_70__ = __iter_7__+(1);
      float __temp_71__;
      __temp_71__ = (7 * __tilevar_2__[__temp_70__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_69__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_72__;
      __temp_72__ = (__temp_68__ + __temp_71__);
      float __temp_73__;
      __temp_73__ = (__temp_72__ / 118);
      __tilevar_1__[__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] = __temp_73__;
    }
  }
  __syncthreads();
  int __iter_8__;
  __iter_8__ = FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1) + (int)(threadIdx.y) ; 
  if( __iter_8__ <= FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2)) ){
    int __iter_9__;
    __iter_9__ = FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1) + (int)(threadIdx.x) ; 
    if( __iter_9__ <= FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2)) ){
      float __temp_74__;
      __temp_74__ = (7 * __tilevar_1__[__iter_9__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_75__;
      __temp_75__ = (5 * __tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_76__;
      __temp_76__ = (__temp_74__ + __temp_75__);
      float __temp_77__;
      __temp_77__ = (9 * __tilevar_1__[__iter_9__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_78__;
      __temp_78__ = (__temp_76__ + __temp_77__);
      float __temp_79__;
      __temp_79__ = (12 * __tilevar_1__[__iter_9__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_80__;
      __temp_80__ = (__temp_78__ + __temp_79__);
      float __temp_81__;
      __temp_81__ = (15 * __tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_82__;
      __temp_82__ = (__temp_80__ + __temp_81__);
      float __temp_83__;
      __temp_83__ = (12 * __tilevar_1__[__iter_9__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_84__;
      __temp_84__ = (__temp_82__ + __temp_83__);
      float __temp_85__;
      __temp_85__ = (9 * __tilevar_1__[__iter_9__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_86__;
      __temp_86__ = (__temp_84__ + __temp_85__);
      float __temp_87__;
      __temp_87__ = (5 * __tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_88__;
      __temp_88__ = (__temp_86__ + __temp_87__);
      float __temp_89__;
      __temp_89__ = (7 * __tilevar_1__[__iter_9__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_90__;
      __temp_90__ = (__temp_88__ + __temp_89__);
      float __temp_91__;
      __temp_91__ = (__temp_90__ / 118);
      __tilevar_0__[__iter_9__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_8__+(0-(FORMA_MAX(__iter_1__,1)-1)))] = __temp_91__;
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      float __temp_92__;
      __temp_92__ = (7 * __tilevar_0__[__iter_11__+(-1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(-1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_93__;
      __temp_93__ = (5 * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(-1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_94__;
      __temp_94__ = (__temp_92__ + __temp_93__);
      float __temp_95__;
      __temp_95__ = (9 * __tilevar_0__[__iter_11__+(1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(-1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_96__;
      __temp_96__ = (__temp_94__ + __temp_95__);
      float __temp_97__;
      __temp_97__ = (12 * __tilevar_0__[__iter_11__+(-1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_98__;
      __temp_98__ = (__temp_96__ + __temp_97__);
      float __temp_99__;
      __temp_99__ = (15 * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_100__;
      __temp_100__ = (__temp_98__ + __temp_99__);
      float __temp_101__;
      __temp_101__ = (12 * __tilevar_0__[__iter_11__+(1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_102__;
      __temp_102__ = (__temp_100__ + __temp_101__);
      float __temp_103__;
      __temp_103__ = (9 * __tilevar_0__[__iter_11__+(-1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_104__;
      __temp_104__ = (__temp_102__ + __temp_103__);
      float __temp_105__;
      __temp_105__ = (5 * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_106__;
      __temp_106__ = (__temp_104__ + __temp_105__);
      float __temp_107__;
      __temp_107__ = (7 * __tilevar_0__[__iter_11__+(1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_108__;
      __temp_108__ = (__temp_106__ + __temp_107__);
      float __temp_109__;
      __temp_109__ = (__temp_108__ / 118);
      __var_1__[__iter_11__+(M-0)*(__iter_10__)] = __temp_109__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-6))*(FORMA_BLOCKDIM_X-(-6))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,9))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  __block_0___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.x-(4-(-4));
  __block_1___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.y-(4-(-4));
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

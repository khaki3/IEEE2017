#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, double * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  double * __tilevar_0__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*FORMA_BLOCKDIM_X;
  double * __tilevar_1__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*FORMA_BLOCKDIM_X;
  double * __tilevar_2__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*FORMA_BLOCKDIM_X;
  double * __tilevar_3__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*FORMA_BLOCKDIM_X;

  double t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f, out = 0.0f;
  double b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X-8);
  int __iter_y__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y));

  // Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
    __tilevar_1__[__iter_3__-__iter_0__] = 0.0f;
    __tilevar_2__[__iter_3__-__iter_0__] = 0.0f;
    __tilevar_3__[__iter_3__-__iter_0__] = 0.0f;
  }
  // Initial loop
  for (int __iter_1__ = FORMA_MAX(0,__iter_y__-4); __iter_1__ <= __iter_y__+3; __iter_1__++) {
    if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
      __tilevar_0__[__iter_3__-__iter_0__] = input[__iter_3__+M*(__iter_1__)];
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))) {
	// Bottom
        double __temp_2__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_5__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        double __temp_9__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 9 * __temp_9__) / 118;
	t2 += __temp_10__;
	// Mid
        double __temp_13__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_17__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_18__ = (12 * __temp_13__ + 15 * __temp_17__);
        double __temp_21__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_22__ = (__temp_18__ + 12 * __temp_21__) / 118;
	b2 += __temp_22__;
	// Top
        double __temp_25__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_29__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_30__ = (9 * __temp_25__ + 5 * __temp_29__);
        double __temp_33__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 7 * __temp_33__) / 118;
        __tilevar_1__[__iter_3__-__iter_0__] += __temp_34__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))) {
	// Bottom
        double __temp_2__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_5__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        double __temp_9__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 9 * __temp_9__) / 118;
	t3 += __temp_10__;
	// Mid
        double __temp_13__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_17__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_18__ = (12 * __temp_13__ + 15 * __temp_17__);
        double __temp_21__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_22__ = (__temp_18__ + 12 * __temp_21__) / 118;
	b3 += __temp_22__;
	// Top
        double __temp_25__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_29__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_30__ = (9 * __temp_25__ + 5 * __temp_29__);
        double __temp_33__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 7 * __temp_33__) / 118;
        __tilevar_2__[__iter_3__-__iter_0__] += __temp_34__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))) {
	// Bottom
        double __temp_2__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        double __temp_5__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        double __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        double __temp_9__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 9 * __temp_9__) / 118;
	t4 += __temp_10__;
	// Mid
        double __temp_13__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        double __temp_17__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        double __temp_18__ = (12 * __temp_13__ + 15 * __temp_17__);
        double __temp_21__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        double __temp_22__ = (__temp_18__ + 12 * __temp_21__) / 118;
	b4 += __temp_22__;
	// Top
        double __temp_25__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        double __temp_29__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        double __temp_30__ = (9 * __temp_25__ + 5 * __temp_29__);
        double __temp_33__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 7 * __temp_33__) / 118;
        __tilevar_3__[__iter_3__-__iter_0__] += __temp_34__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2))) {
	// Bottom
        double __temp_2__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        double __temp_5__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        double __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        double __temp_9__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 9 * __temp_9__) / 118;
	t5 += __temp_10__;
	// Mid
        double __temp_13__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        double __temp_17__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        double __temp_18__ = (12 * __temp_13__ + 15 * __temp_17__);
        double __temp_21__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        double __temp_22__ = (__temp_18__ + 12 * __temp_21__) / 118;
	b5 += __temp_22__;
	// Top
        double __temp_25__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        double __temp_29__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        double __temp_30__ = (9 * __temp_25__ + 5 * __temp_29__);
        double __temp_33__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 7 * __temp_33__) / 118;
	out += __temp_34__;
    }
    __syncthreads ();
    // Now rotate
    __tilevar_1__[__iter_3__-__iter_0__] = b2;
    b2 = t2;
    t2 = 0.0f;
    __tilevar_2__[__iter_3__-__iter_0__] = b3;
    b3 = t3;
    t3 = 0.0f;
    __tilevar_3__[__iter_3__-__iter_0__] = b4;
    b4 = t4;
    t4 = 0.0f;
    out= b5;
    b5 = t5;
    t5 = 0.0f;
  }
  // Rest of the computation
  __syncthreads ();
  for (int __iter_1__ = __iter_y__+4; __iter_1__ <= FORMA_MIN(N-1,__iter_y__+FORMA_BLOCKDIM_Y+3); __iter_1__++) {
    if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
      __tilevar_0__[__iter_3__-__iter_0__] = input[__iter_3__+M*(__iter_1__)];
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))) {
	// Bottom
        double __temp_2__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_5__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        double __temp_9__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 9 * __temp_9__) / 118;
	t2 += __temp_10__;
	// Mid
        double __temp_13__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_17__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_18__ = (12 * __temp_13__ + 15 * __temp_17__);
        double __temp_21__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_22__ = (__temp_18__ + 12 * __temp_21__) / 118;
	b2 += __temp_22__;
	// Top
        double __temp_25__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_29__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_30__ = (9 * __temp_25__ + 5 * __temp_29__);
        double __temp_33__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 7 * __temp_33__) / 118;
        __tilevar_1__[__iter_3__-__iter_0__] += __temp_34__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))) {
	// Bottom
        double __temp_2__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_5__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        double __temp_9__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 9 * __temp_9__) / 118;
	t3 += __temp_10__;
	// Mid
        double __temp_13__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_17__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_18__ = (12 * __temp_13__ + 15 * __temp_17__);
        double __temp_21__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_22__ = (__temp_18__ + 12 * __temp_21__) / 118;
	b3 += __temp_22__;
	// Top
        double __temp_25__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_29__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_30__ = (9 * __temp_25__ + 5 * __temp_29__);
        double __temp_33__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 7 * __temp_33__) / 118;
        __tilevar_2__[__iter_3__-__iter_0__] += __temp_34__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))) {
	// Bottom
        double __temp_2__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        double __temp_5__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        double __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        double __temp_9__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 9 * __temp_9__) / 118;
	t4 += __temp_10__;
	// Mid
        double __temp_13__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        double __temp_17__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        double __temp_18__ = (12 * __temp_13__ + 15 * __temp_17__);
        double __temp_21__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        double __temp_22__ = (__temp_18__ + 12 * __temp_21__) / 118;
	b4 += __temp_22__;
	// Top
        double __temp_25__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        double __temp_29__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        double __temp_30__ = (9 * __temp_25__ + 5 * __temp_29__);
        double __temp_33__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 7 * __temp_33__) / 118;
        __tilevar_3__[__iter_3__-__iter_0__] += __temp_34__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2))) {
	// Bottom
        double __temp_2__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        double __temp_5__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        double __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        double __temp_9__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 9 * __temp_9__) / 118;
	t5 += __temp_10__;
	// Mid
        double __temp_13__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        double __temp_17__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        double __temp_18__ = (12 * __temp_13__ + 15 * __temp_17__);
        double __temp_21__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        double __temp_22__ = (__temp_18__ + 12 * __temp_21__) / 118;
	b5 += __temp_22__;
	// Top
        double __temp_25__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        double __temp_29__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        double __temp_30__ = (9 * __temp_25__ + 5 * __temp_29__);
        double __temp_33__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 7 * __temp_33__) / 118;
	out += __temp_34__;
        __var_1__[__iter_3__+M*FORMA_MAX(__iter_1__-4,0)] = out;
    }
    __syncthreads ();
    // Now rotate
    __tilevar_1__[__iter_3__-__iter_0__] = b2;
    b2 = t2;
    t2 = 0.0f;
    __tilevar_2__[__iter_3__-__iter_0__] = b3;
    b3 = t3;
    t3 = 0.0f;
    __tilevar_3__[__iter_3__-__iter_0__] = b4;
    b4 = t4;
    t4 = 0.0f;
    out= b5;
    b5 = t5;
    t5 = 0.0f;
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(double)*(4*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(double * h_input, int N, int M, double * __var_0__){

/* Host allocation Begin */
  double * input;
  hipMalloc(&input,sizeof(double)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(double)*((N)*(M)), memcpy_kind_h_input);
  }
  double * __var_1__;
  hipMalloc(&__var_1__,sizeof(double)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = M;
  int __size_1___kernel___forma_kernel__0__ = N;

  int __block_0___kernel___forma_kernel__0__ = 128;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-8);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__size_0___kernel___forma_kernel__0__/32);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __size_0___kernel___forma_kernel__0__/32, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(double)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

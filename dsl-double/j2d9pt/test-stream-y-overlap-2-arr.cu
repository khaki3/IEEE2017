#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define mod(x,y) ( (x) & (y-1))

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

void Check_CUDA_Error(const char* message);
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  __shared__ float __tilevar_0__[3][128];
  __shared__ float __tilevar_1__[3][128];
  int __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X-4);

  // Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
    __tilevar_0__[1][__iter_3__-__iter_0__] = input[__iter_3__+M*0];
    __tilevar_0__[2][__iter_3__-__iter_0__] = input[__iter_3__+M*1];
  }
  // Rest of the computation
  for (int __iter_1__ = 1; __iter_1__ < N-1; __iter_1__++) {
    if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
      __tilevar_0__[0][__iter_3__-__iter_0__] = __tilevar_0__[1][__iter_3__-__iter_0__];
      __tilevar_0__[1][__iter_3__-__iter_0__] = __tilevar_0__[2][__iter_3__-__iter_0__];
      __tilevar_0__[2][__iter_3__-__iter_0__] = input[__iter_3__+M*(__iter_1__+1)];
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      float __temp_2__ = (__tilevar_0__[0][__iter_3__-1-__iter_0__]);
      float __temp_5__ = (__tilevar_0__[0][__iter_3__-__iter_0__]);
      float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
      float __temp_9__ = (__tilevar_0__[0][__iter_3__+1-__iter_0__]);
      float __temp_10__ = (__temp_6__ + 9 * __temp_9__);
      float __temp_13__ = (__tilevar_0__[1][__iter_3__-1-__iter_0__]);
      float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
      float __temp_17__ = (__tilevar_0__[1][__iter_3__-__iter_0__]);
      float __temp_18__ = (__temp_14__ + 15 * __temp_17__);
      float __temp_21__ = (__tilevar_0__[1][__iter_3__+1-__iter_0__]);
      float __temp_22__ = (__temp_18__ + 12 * __temp_21__);
      float __temp_25__ = (__tilevar_0__[2][__iter_3__-1-__iter_0__]);
      float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
      float __temp_29__ = (__tilevar_0__[2][__iter_3__-__iter_0__]);
      float __temp_30__ = (__temp_26__ + 5 * __temp_29__);
      float __temp_33__ = (__tilevar_0__[2][__iter_3__+1-__iter_0__]);
      float __temp_34__ = (__temp_30__ + 7 * __temp_33__);
      float __temp_35__ = (__temp_34__ / 118);
      __tilevar_1__[0][__iter_3__-__iter_0__] = __tilevar_1__[1][__iter_3__-__iter_0__];
      __tilevar_1__[1][__iter_3__-__iter_0__] = __tilevar_1__[2][__iter_3__-__iter_0__];
      __tilevar_1__[2][__iter_3__-__iter_0__] = __temp_35__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      float __temp_2__ = (__tilevar_1__[0][__iter_3__-1-__iter_0__]);
      float __temp_5__ = (__tilevar_1__[0][__iter_3__-__iter_0__]);
      float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
      float __temp_9__ = (__tilevar_1__[0][__iter_3__+1-__iter_0__]);
      float __temp_10__ = (__temp_6__ + 9 * __temp_9__);
      float __temp_13__ = (__tilevar_1__[1][__iter_3__-1-__iter_0__]);
      float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
      float __temp_17__ = (__tilevar_1__[1][__iter_3__-__iter_0__]);
      float __temp_18__ = (__temp_14__ + 15 * __temp_17__);
      float __temp_21__ = (__tilevar_1__[1][__iter_3__+1-__iter_0__]);
      float __temp_22__ = (__temp_18__ + 12 * __temp_21__);
      float __temp_25__ = (__tilevar_1__[2][__iter_3__-1-__iter_0__]);
      float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
      float __temp_29__ = (__tilevar_1__[2][__iter_3__-__iter_0__]);
      float __temp_30__ = (__temp_26__ + 5 * __temp_29__);
      float __temp_33__ = (__tilevar_1__[2][__iter_3__+1-__iter_0__]);
      float __temp_34__ = (__temp_30__ + 7 * __temp_33__);
      float __temp_35__ = (__temp_34__ / 118);
      __var_1__[__iter_3__+(M)*FORMA_MAX(__iter_1__-1,0)] = __temp_35__;
    }
  }
} 

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N)*(M)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 128;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-4);
  int __grid_1___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_2__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__>>> (__var_2__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

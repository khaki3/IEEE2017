#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(double * input, int N, int M, double * __var_4__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_0__ <= (M-2)){
    int __iter_1__;
    __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_1__ <= (N-2)){
      double __temp_0__;
      __temp_0__ = (7 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(-1))]);
      double __temp_1__;
      __temp_1__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(-1))]);
      double __temp_2__;
      __temp_2__ = (__temp_0__ + __temp_1__);
      double __temp_3__;
      __temp_3__ = (9 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(-1))]);
      double __temp_4__;
      __temp_4__ = (__temp_2__ + __temp_3__);
      double __temp_5__;
      __temp_5__ = (12 * input[__iter_0__+(-1)+(M-0)*(__iter_1__)]);
      double __temp_6__;
      __temp_6__ = (__temp_4__ + __temp_5__);
      double __temp_7__;
      __temp_7__ = (15 * input[__iter_0__+(M-0)*(__iter_1__)]);
      double __temp_8__;
      __temp_8__ = (__temp_6__ + __temp_7__);
      double __temp_9__;
      __temp_9__ = (12 * input[__iter_0__+(1)+(M-0)*(__iter_1__)]);
      double __temp_10__;
      __temp_10__ = (__temp_8__ + __temp_9__);
      double __temp_11__;
      __temp_11__ = (9 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(1))]);
      double __temp_12__;
      __temp_12__ = (__temp_10__ + __temp_11__);
      double __temp_13__;
      __temp_13__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(1))]);
      double __temp_14__;
      __temp_14__ = (__temp_12__ + __temp_13__);
      double __temp_15__;
      __temp_15__ = (7 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(1))]);
      double __temp_16__;
      __temp_16__ = (__temp_14__ + __temp_15__);
      double __temp_17__;
      __temp_17__ = (__temp_16__ / 118);
      __var_4__[__iter_0__+(M-0)*(__iter_1__)] = __temp_17__;
    }
  }
}
__global__ void __kernel___forma_kernel__1__(double * __var_4__, int N, int M, double * __var_3__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_2__ <= (M-2)){
    int __iter_3__;
    __iter_3__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_3__ <= (N-2)){
      double __temp_18__;
      __temp_18__ = (7 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(-1))]);
      double __temp_19__;
      __temp_19__ = (5 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(-1))]);
      double __temp_20__;
      __temp_20__ = (__temp_18__ + __temp_19__);
      double __temp_21__;
      __temp_21__ = (9 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(-1))]);
      double __temp_22__;
      __temp_22__ = (__temp_20__ + __temp_21__);
      double __temp_23__;
      __temp_23__ = (12 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__)]);
      double __temp_24__;
      __temp_24__ = (__temp_22__ + __temp_23__);
      double __temp_25__;
      __temp_25__ = (15 * __var_4__[__iter_2__+(M-0)*(__iter_3__)]);
      double __temp_26__;
      __temp_26__ = (__temp_24__ + __temp_25__);
      double __temp_27__;
      __temp_27__ = (12 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__)]);
      double __temp_28__;
      __temp_28__ = (__temp_26__ + __temp_27__);
      double __temp_29__;
      __temp_29__ = (9 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(1))]);
      double __temp_30__;
      __temp_30__ = (__temp_28__ + __temp_29__);
      double __temp_31__;
      __temp_31__ = (5 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(1))]);
      double __temp_32__;
      __temp_32__ = (__temp_30__ + __temp_31__);
      double __temp_33__;
      __temp_33__ = (7 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(1))]);
      double __temp_34__;
      __temp_34__ = (__temp_32__ + __temp_33__);
      double __temp_35__;
      __temp_35__ = (__temp_34__ / 118);
      __var_3__[__iter_2__+(M-0)*(__iter_3__)] = __temp_35__;
    }
  }
}
__global__ void __kernel___forma_kernel__2__(double * __var_3__, int N, int M, double * __var_2__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_4__;
  __iter_4__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_4__ <= (M-2)){
    int __iter_5__;
    __iter_5__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_5__ <= (N-2)){
      double __temp_36__;
      __temp_36__ = (7 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(-1))]);
      double __temp_37__;
      __temp_37__ = (5 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(-1))]);
      double __temp_38__;
      __temp_38__ = (__temp_36__ + __temp_37__);
      double __temp_39__;
      __temp_39__ = (9 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(-1))]);
      double __temp_40__;
      __temp_40__ = (__temp_38__ + __temp_39__);
      double __temp_41__;
      __temp_41__ = (12 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__)]);
      double __temp_42__;
      __temp_42__ = (__temp_40__ + __temp_41__);
      double __temp_43__;
      __temp_43__ = (15 * __var_3__[__iter_4__+(M-0)*(__iter_5__)]);
      double __temp_44__;
      __temp_44__ = (__temp_42__ + __temp_43__);
      double __temp_45__;
      __temp_45__ = (12 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__)]);
      double __temp_46__;
      __temp_46__ = (__temp_44__ + __temp_45__);
      double __temp_47__;
      __temp_47__ = (9 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(1))]);
      double __temp_48__;
      __temp_48__ = (__temp_46__ + __temp_47__);
      double __temp_49__;
      __temp_49__ = (5 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(1))]);
      double __temp_50__;
      __temp_50__ = (__temp_48__ + __temp_49__);
      double __temp_51__;
      __temp_51__ = (7 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(1))]);
      double __temp_52__;
      __temp_52__ = (__temp_50__ + __temp_51__);
      double __temp_53__;
      __temp_53__ = (__temp_52__ / 118);
      __var_2__[__iter_4__+(M-0)*(__iter_5__)] = __temp_53__;
    }
  }
}
__global__ void __kernel___forma_kernel__3__(double * __var_2__, int N, int M, double * __var_1__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_6__;
  __iter_6__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_6__ <= (M-2)){
    int __iter_7__;
    __iter_7__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_7__ <= (N-2)){
      double __temp_54__;
      __temp_54__ = (7 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(-1))]);
      double __temp_55__;
      __temp_55__ = (5 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(-1))]);
      double __temp_56__;
      __temp_56__ = (__temp_54__ + __temp_55__);
      double __temp_57__;
      __temp_57__ = (9 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(-1))]);
      double __temp_58__;
      __temp_58__ = (__temp_56__ + __temp_57__);
      double __temp_59__;
      __temp_59__ = (12 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__)]);
      double __temp_60__;
      __temp_60__ = (__temp_58__ + __temp_59__);
      double __temp_61__;
      __temp_61__ = (15 * __var_2__[__iter_6__+(M-0)*(__iter_7__)]);
      double __temp_62__;
      __temp_62__ = (__temp_60__ + __temp_61__);
      double __temp_63__;
      __temp_63__ = (12 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__)]);
      double __temp_64__;
      __temp_64__ = (__temp_62__ + __temp_63__);
      double __temp_65__;
      __temp_65__ = (9 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(1))]);
      double __temp_66__;
      __temp_66__ = (__temp_64__ + __temp_65__);
      double __temp_67__;
      __temp_67__ = (5 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(1))]);
      double __temp_68__;
      __temp_68__ = (__temp_66__ + __temp_67__);
      double __temp_69__;
      __temp_69__ = (7 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(1))]);
      double __temp_70__;
      __temp_70__ = (__temp_68__ + __temp_69__);
      double __temp_71__;
      __temp_71__ = (__temp_70__ / 118);
      __var_1__[__iter_6__+(M-0)*(__iter_7__)] = __temp_71__;
    }
  }
}
/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(double * h_input, int N, int M, double * __var_0__){

/* Host allocation Begin */
  double * input;
  hipMalloc(&input,sizeof(double)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(double)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  double * __var_1__;
  hipMalloc(&__var_1__,sizeof(double)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  double * __var_2__;
  hipMalloc(&__var_2__,sizeof(double)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
  double * __var_3__;
  hipMalloc(&__var_3__,sizeof(double)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_3__\n");
  double * __var_4__;
  hipMalloc(&__var_4__,sizeof(double)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_4__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 32;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_4__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_4__, N, M, __var_3__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_3__, N, M, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, N, M, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");


  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(double)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
  hipFree(__var_3__);
  hipFree(__var_4__);
}
/*Host Free End*/

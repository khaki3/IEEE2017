#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define GAPX (22) 
#define EXTENT (5)
#define mod(x,y) ( (x) & (y-1))

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float * __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+16)*FORMA_BLOCKDIM_X);
  float * __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+16)*FORMA_BLOCKDIM_X);
  int rowy = FORMA_BLOCKDIM_Y+16;

  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X + GAPX);
  for (int __iter_1__ = 0; __iter_1__ <= N-1; __iter_1__ += FORMA_BLOCKDIM_Y) {
    int __iter_2__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
    int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
    if(__iter_2__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
      __tilevar_0__[__iter_3__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_2__,rowy)] = input[__iter_3__+M*__iter_2__];
    }
    __syncthreads();
    int __iter_4__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        float __temp_2__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__-1,rowy)]);
        float __temp_5__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__-1,rowy)]);
        float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        float __temp_9__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__-1,rowy)]);
        float __temp_10__ = (__temp_6__ + 9 * __temp_9__);
        float __temp_13__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        float __temp_17__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_18__ = (__temp_14__ + 15 * __temp_17__);
        float __temp_21__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_22__ = (__temp_18__ + 12 * __temp_21__);
        float __temp_25__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)]);
        float __temp_30__ = (__temp_26__ + 5 * __temp_29__);
        float __temp_33__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)]);
        float __temp_34__ = (__temp_30__ + 7 * __temp_33__);
        float __temp_35__ = (__temp_34__ / 118);
        __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)] = __temp_35__;
      }
    }
    int __iter_6__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        if (__iter_7__ < (FORMA_MAX((__iter_0__+1),1)+2) | __iter_7__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-2)) {
	  __copy_arr_0__[__iter_6__+(M)*(__iter_7__)] = __tilevar_1__[__iter_7__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_6__,rowy)];
        }
      }
    }
    __syncthreads();
    int __iter_10__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
      int __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
	float __temp_2__ = (__tilevar_1__[__iter_11__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__-1,rowy)]);
        float __temp_5__ = (__tilevar_1__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__-1,rowy)]);
        float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        float __temp_9__ = (__tilevar_1__[__iter_11__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__-1,rowy)]);
        float __temp_10__ = (__temp_6__ + 9 * __temp_9__);
        float __temp_13__ = (__tilevar_1__[__iter_11__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        float __temp_17__ = (__tilevar_1__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_18__ = (__temp_14__ + 15 * __temp_17__);
        float __temp_21__ = (__tilevar_1__[__iter_11__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_22__ = (__temp_18__ + 12 * __temp_21__);
        float __temp_25__ = (__tilevar_1__[__iter_11__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_1__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_30__ = (__temp_26__ + 5 * __temp_29__);
        float __temp_33__ = (__tilevar_1__[__iter_11__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_34__ = (__temp_30__ + 7 * __temp_33__);
        float __temp_35__ = (__temp_34__ / 118);
        __tilevar_0__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)] = __temp_35__;
      }
    }
    int __iter_12__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
      int __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        if (__iter_13__ < (FORMA_MAX((__iter_0__+2),1)+2) | __iter_13__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-2)) {
	  __copy_arr_1__[__iter_12__+(M)*(__iter_13__)] = __tilevar_0__[__iter_13__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_12__,rowy)]; 
        }
      }
    }
    __syncthreads();
    int __iter_16__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
      int __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        float __temp_2__ = (__tilevar_0__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__-1,rowy)]);
        float __temp_5__ = (__tilevar_0__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__-1,rowy)]);
        float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        float __temp_9__ = (__tilevar_0__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__-1,rowy)]);
        float __temp_10__ = (__temp_6__ + 9 * __temp_9__);
        float __temp_13__ = (__tilevar_0__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        float __temp_17__ = (__tilevar_0__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_18__ = (__temp_14__ + 15 * __temp_17__);
        float __temp_21__ = (__tilevar_0__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_22__ = (__temp_18__ + 12 * __temp_21__);
        float __temp_25__ = (__tilevar_0__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_0__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_30__ = (__temp_26__ + 5 * __temp_29__);
        float __temp_33__ = (__tilevar_0__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_34__ = (__temp_30__ + 7 * __temp_33__);
        float __temp_35__ = (__temp_34__ / 118);
        __tilevar_1__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)] = __temp_35__;
      }
    }
    int __iter_18__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
      int __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        if (__iter_19__ < (FORMA_MAX((__iter_0__+3),1)+2) | __iter_19__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-2)) {
	  __copy_arr_2__[__iter_18__+(M)*(__iter_19__)] = __tilevar_1__[__iter_19__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_18__,rowy)];
        }
      }
    }
    __syncthreads();
    int __iter_22__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
    if( __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) ){
      int __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
        float __temp_2__ = (__tilevar_1__[__iter_23__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__-1,rowy)]);
        float __temp_5__ = (__tilevar_1__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__-1,rowy)]);
        float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        float __temp_9__ = (__tilevar_1__[__iter_23__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__-1,rowy)]);
        float __temp_10__ = (__temp_6__ + 9 * __temp_9__);
        float __temp_13__ = (__tilevar_1__[__iter_23__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        float __temp_17__ = (__tilevar_1__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_18__ = (__temp_14__ + 15 * __temp_17__);
        float __temp_21__ = (__tilevar_1__[__iter_23__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_22__ = (__temp_18__ + 12 * __temp_21__);
        float __temp_25__ = (__tilevar_1__[__iter_23__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_1__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_30__ = (__temp_26__ + 5 * __temp_29__);
        float __temp_33__ = (__tilevar_1__[__iter_23__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_34__ = (__temp_30__ + 7 * __temp_33__);
        float __temp_35__ = (__temp_34__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__)] = __temp_35__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(2*(FORMA_BLOCKDIM_Y+16)*FORMA_BLOCKDIM_X);
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float * __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+16)*FORMA_BLOCKDIM_X);
  float * __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+16)*FORMA_BLOCKDIM_X);
  int rowy = FORMA_BLOCKDIM_Y+16;
 
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X + GAPX) + (int)FORMA_BLOCKDIM_X;
  for (int __iter_1__ = 0; __iter_1__ <= N-1; __iter_1__ += FORMA_BLOCKDIM_Y) {
    int __iter_2__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
    int __iter_3__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
    if( __iter_2__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1))){
      __tilevar_0__[__iter_3__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_2__,rowy)] = input[__iter_3__+(M)*(__iter_2__)];
    }
    __syncthreads();
    int __iter_4__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_5__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
        float __temp_2__ = (__tilevar_0__[__iter_5__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__-1,rowy)]);
        float __temp_5__ = (__tilevar_0__[__iter_5__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__-1,rowy)]);
        float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        float __temp_9__ = (__tilevar_0__[__iter_5__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__-1,rowy)]);
        float __temp_10__ = (__temp_6__ + 9 * __temp_9__);
        float __temp_13__ = (__tilevar_0__[__iter_5__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        float __temp_17__ = (__tilevar_0__[__iter_5__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_18__ = (__temp_14__ + 15 * __temp_17__);
        float __temp_21__ = (__tilevar_0__[__iter_5__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_22__ = (__temp_18__ + 12 * __temp_21__);
        float __temp_25__ = (__tilevar_0__[__iter_5__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_0__[__iter_5__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)]);
        float __temp_30__ = (__temp_26__ + 5 * __temp_29__);
        float __temp_33__ = (__tilevar_0__[__iter_5__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)]);
        float __temp_34__ = (__temp_30__ + 7 * __temp_33__);
        float __temp_35__ = (__temp_34__ / 118);
        __tilevar_1__[__iter_5__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)] = __temp_35__;
      }
    }
    int __iter_6__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ;
    int __iter_7__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ;
    if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      if (__iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) & (__iter_7__ < FORMA_MAX((__iter_0__-1),1) | __iter_7__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)))) {
        __tilevar_1__[__iter_7__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_6__,rowy)] = __copy_arr_0__[__iter_6__+(M)*(__iter_7__)];
      }
    }
    __syncthreads();
    int __iter_10__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
      int __iter_11__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
        float __temp_2__ = (__tilevar_1__[__iter_11__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__-1,rowy)]);
        float __temp_5__ = (__tilevar_1__[__iter_11__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__-1,rowy)]);
        float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        float __temp_9__ = (__tilevar_1__[__iter_11__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__-1,rowy)]);
        float __temp_10__ = (__temp_6__ + 9 * __temp_9__);
        float __temp_13__ = (__tilevar_1__[__iter_11__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        float __temp_17__ = (__tilevar_1__[__iter_11__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_18__ = (__temp_14__ + 15 * __temp_17__);
        float __temp_21__ = (__tilevar_1__[__iter_11__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_22__ = (__temp_18__ + 12 * __temp_21__);
        float __temp_25__ = (__tilevar_1__[__iter_11__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_1__[__iter_11__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_30__ = (__temp_26__ + 5 * __temp_29__);
        float __temp_33__ = (__tilevar_1__[__iter_11__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_34__ = (__temp_30__ + 7 * __temp_33__);
        float __temp_35__ = (__temp_34__ / 118);
        __tilevar_0__[__iter_11__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)] = __temp_35__;
      }
    }
    int __iter_12__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ;
    int __iter_13__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ;
    if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
      if (__iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) & (__iter_13__ < FORMA_MAX((__iter_0__-2),1) | __iter_13__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)))) {
        __tilevar_0__[__iter_13__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_12__,rowy)] = __copy_arr_1__[__iter_12__+(M)*(__iter_13__)];
      }
    }
    __syncthreads();
    int __iter_16__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
      int __iter_17__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
        float __temp_2__ = (__tilevar_0__[__iter_17__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__-1,rowy)]);
        float __temp_5__ = (__tilevar_0__[__iter_17__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__-1,rowy)]);
        float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        float __temp_9__ = (__tilevar_0__[__iter_17__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__-1,rowy)]);
        float __temp_10__ = (__temp_6__ + 9 * __temp_9__);
        float __temp_13__ = (__tilevar_0__[__iter_17__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        float __temp_17__ = (__tilevar_0__[__iter_17__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_18__ = (__temp_14__ + 15 * __temp_17__);
        float __temp_21__ = (__tilevar_0__[__iter_17__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_22__ = (__temp_18__ + 12 * __temp_21__);
        float __temp_25__ = (__tilevar_0__[__iter_17__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_0__[__iter_17__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_30__ = (__temp_26__ + 5 * __temp_29__);
        float __temp_33__ = (__tilevar_0__[__iter_17__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_34__ = (__temp_30__ + 7 * __temp_33__);
        float __temp_35__ = (__temp_34__ / 118);
        __tilevar_1__[__iter_17__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)] = __temp_35__;
      }
    }
    int  __iter_18__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    int __iter_19__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ;
    if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
      if (__iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(M-2)) & (__iter_19__ < FORMA_MAX((__iter_0__-3),1) | __iter_19__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)))) {
        __tilevar_1__[__iter_19__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_18__,rowy)] = __copy_arr_2__[__iter_18__+(M)*(__iter_19__)];
      }
    }
    __syncthreads();
    int __iter_22__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
    if( __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) ){
      int __iter_23__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
        float __temp_2__ = (__tilevar_1__[__iter_23__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__-1,rowy)]);
        float __temp_5__ = (__tilevar_1__[__iter_23__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__-1,rowy)]);
        float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        float __temp_9__ = (__tilevar_1__[__iter_23__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__-1,rowy)]);
        float __temp_10__ = (__temp_6__ + 9 * __temp_9__);
        float __temp_13__ = (__tilevar_1__[__iter_23__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        float __temp_17__ = (__tilevar_1__[__iter_23__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_18__ = (__temp_14__ + 15 * __temp_17__);
        float __temp_21__ = (__tilevar_1__[__iter_23__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_22__ = (__temp_18__ + 12 * __temp_21__);
        float __temp_25__ = (__tilevar_1__[__iter_23__-1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_1__[__iter_23__+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_30__ = (__temp_26__ + 5 * __temp_29__);
        float __temp_33__ = (__tilevar_1__[__iter_23__+1+EXTENT-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_34__ = (__temp_30__ + 7 * __temp_33__);
        float __temp_35__ = (__temp_34__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__)] = __temp_35__;
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N)*(M)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 16;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"
#include <nvml.h>
#include <assert.h>


#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
/* X, Y, Z */
__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, double * __restrict__ __var_1__){
	int __FORMA_SHARED_MEM_OFFSET__ = 0;
	double* __tilevar_0__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
	__FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
	double* __tilevar_1__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
	__FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);

	int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)-4);
	int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)-4);
	double t2=0.0f, t3=0.0f;
	double m2=0.0f, m3=0.0f;
	double out2=0.0f, out3=0.0f;

	// Initialize the values
	int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
	int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
	// Rest of the computation
	for (int __iter_2__ = 0; __iter_2__ <= L-1; __iter_2__++) {
		if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1))){
			__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = input[__iter_5__+N*(__iter_4__+M*(__iter_2__))]; 
		}
		__syncthreads ();
		if(__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))){
			// Bottom
			double c0 = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double b0 = -0.166f * c0;
			double a0 = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]; 
			double a1 = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)];
			double a2 = __tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double a3 = __tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double b1 = -0.0833f * (a0 + a1 + a2 + a3);
			t2 += (b0 + b1);
			// Mid
			double b2 = 2.666f * c0; 
			double a4 = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)];
			double a5 = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)];
			double a6 = __tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double a7 = __tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double b3 = -0.166f * (a4 + a5 + a6 + a7);
			double a8 = __tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]; 
			double a9 = __tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)];
			double a10 = __tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)];
			double a11 = __tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)];
			double b4 = -0.0833f * (a8 + a9 + a10 + a11);
			m2 += (b2 + b3 + b4);
			// Top
			double b5 = -0.166f * c0;
			double a12 = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)];
			double a13 = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)];
			double a14 = __tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double a15 = __tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double b6 = -0.0833f * (a12 + a13 + a14 + a15);
			out2 += (b5 + b6); 
			__tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = out2;
		}
		__syncthreads ();
		if(__iter_4__ >= FORMA_MAX((__iter_1__+2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+2),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))){
			// Bottom
			double c0 = __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double b0 = -0.166f * c0;
			double a0 = __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]; 
			double a1 = __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)];
			double a2 = __tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double a3 = __tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double b1 = -0.0833f * (a0 + a1 + a2 + a3);
			t3 += (b0 + b1);
			// Mid
			double b2 = 2.666f * c0; 
			double a4 = __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)];
			double a5 = __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)];
			double a6 = __tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double a7 = __tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double b3 = -0.166f * (a4 + a5 + a6 + a7);
			double a8 = __tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]; 
			double a9 = __tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)];
			double a10 = __tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)];
			double a11 = __tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)];
			double b4 = -0.0833f * (a8 + a9 + a10 + a11);
			m3 += (b2 + b3 + b4);
			// Top
			double b5 = -0.166f * c0;
			double a12 = __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)];
			double a13 = __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)];
			double a14 = __tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double a15 = __tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double b6 = -0.0833f * (a12 + a13 + a14 + a15);
			out3 += (b5 + b6); 
			__var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-2,0))] = out3;
		}
		__syncthreads ();
		// Now rotate 
		out2 = m2; m2 = t2; t2 = 0.0f;
		out3 = m3; m3 = t3; t3 = 0.0f;
	}
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int SMemSize = 0;
	SMemSize += sizeof(double)*(2*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
	return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void host_code (double * h_input, double * __var_0__, int L, int M, int N) {

	/* Host allocation Begin */
	double * input;
	hipMalloc(&input,sizeof(double)*(L*M*N));
	Check_CUDA_Error("Allocation Error!! : input\n");
	hipPointerAttribute_t ptrAttrib_h_input;
	hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
	if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
		if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
			memcpy_kind_h_input = hipMemcpyDeviceToDevice;
	hipGetLastError();
	if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
		hipMemcpy(input,h_input,sizeof(double)*(L*M*N), memcpy_kind_h_input);
	}

	double * __var_1__;
	hipMalloc(&__var_1__,sizeof(double)*(L*M*N));
	Check_CUDA_Error("Allocation Error!! : __var_1__\n");
	double * __var_2__;
	hipMalloc(&__var_2__,sizeof(double)*(L*M*N));
	Check_CUDA_Error("Allocation Error!! : __var_2__\n");
	/*Host Allocation End */
	/* Kernel Launch Begin */
	int __FORMA_MAX_SHARED_MEM__;
	hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
	hipEvent_t _forma_timer_start_,_forma_timer_stop_;
	hipEventCreate(&_forma_timer_start_);
	hipEventCreate(&_forma_timer_stop_);
	hipEventRecord(_forma_timer_start_,0);
#endif
	int __size_0___kernel___forma_kernel__0__ = N;
	int __size_1___kernel___forma_kernel__0__ = M;
	int __block_0___kernel___forma_kernel__0__ = 32;
	int __block_1___kernel___forma_kernel__0__ = 16;
	int __block_2___kernel___forma_kernel__0__ = 1;
	dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
	int __SMemSize___kernel___forma_kernel__0__ = 0;
	__SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
	int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-4);
	int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y-4);
	int __grid_2___kernel___forma_kernel__0__ = 1;
	dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
	dim3 unrollConfig (__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<500; x++) {

		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_2__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();

	hipPointerAttribute_t ptrAttrib___var_0__;
	hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
	if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
		if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
			memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
	hipGetLastError();
	hipMemcpy(__var_0__,__var_1__, sizeof(double)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
	hipEventRecord(_forma_timer_stop_,0);
	hipEventSynchronize(_forma_timer_stop_);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
	printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
	hipEventDestroy(_forma_timer_start_);
	hipEventDestroy(_forma_timer_stop_);
#endif
	/*Kernel Launch End */
	/* Host Free Begin */
	hipFree(input);
	hipFree(__var_1__);
	hipFree(__var_2__);
}
/*Host Free End*/

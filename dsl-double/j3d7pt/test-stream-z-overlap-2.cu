#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
/* X, Y, Z */
__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, double * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  double* __tilevar_2__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  double* __tilevar_3__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);

  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)-4);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)-4);
  double t2=0.0f, t3=0.0f;
  double b2=0.0f, b3=0.0f;

  // Initialize the values
  int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
  int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
  if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1))) {
      __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = input[__iter_5__+N*(__iter_4__+M*(0))];
      t2 = input[__iter_5__+N*(__iter_4__+M*(1))];  
  }
  // Rest of the computation
  for (int __iter_2__ = 1; __iter_2__ < L-1; __iter_2__++) {
    if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1))){
        b2 = __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t2; 
        t2 = input[__iter_5__+N*(__iter_4__+M*(__iter_2__+1))]; 
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))) {
        double __temp_a3__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_a7__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_a8__ = (0.161f * __temp_a3__ + 0.162f * __temp_a7__);
        double __temp_a12__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_a13__ = (__temp_a8__ + 0.163f * __temp_a12__);
        double __temp_a17__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_a18__ = (__temp_a13__ + 0.164f * __temp_a17__);
        double __temp_a23__ = (__temp_a18__ + 0.165f * t2);
        double __temp_a28__ = (__temp_a23__ + 0.166f * b2);
        double __temp_a32__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_a33__ = (__temp_a28__ - 1.670f * __temp_a32__);
	b3 = __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]; 
        __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t3;
        t3 = __temp_a33__;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+2),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))) {
        double __temp_a50__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_a54__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_a55__ = (0.161f * __temp_a50__ + 0.162f * __temp_a54__);
        double __temp_a59__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_a60__ = (__temp_a55__ + 0.163f * __temp_a59__);
        double __temp_a64__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_a65__ = (__temp_a60__ + 0.164f * __temp_a64__);
        double __temp_a70__ = (__temp_a65__ + 0.165f * t3);
        double __temp_a75__ = (__temp_a70__ + 0.166f * b3);
        double __temp_a79__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_a80__ = (__temp_a75__ - 1.670f * __temp_a79__);
        __var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-1,0))] = __temp_a80__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(double)*(2*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d7pt(double * h_input, int L, int M, int N, double * __var_0__){

/* Host allocation Begin */
  double * input;
  hipMalloc(&input,sizeof(double)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(double)*(L*M*N), memcpy_kind_h_input);
  }

  double * __var_1__;
  hipMalloc(&__var_1__,sizeof(double)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  double * __var_2__;
  hipMalloc(&__var_2__,sizeof(double)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = N;
  int __size_1___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 16;
  int __block_2___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-4);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y-4);
  int __grid_2___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 unrollConfig (__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(double)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

#define GAPX (22)
#define GAPY (6)
#define GAPZ (2)
#define EXTENT (5)

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
/* X, Y, Z */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY);
  int __iter_2__ = (int)(blockIdx.z)*((int)(FORMA_BLOCKDIM_Z)+GAPZ);

  int __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1)) ; __iter_3__+=(int)(blockDim.z) ){
    int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
    for(; __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)); __iter_4__+=(int)(blockDim.y)) {
      int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_3__-__iter_2__))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_6__+=(int)(blockDim.z) ){
    int __iter_7__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ;
    for(; __iter_7__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)); __iter_7__+=(int)(blockDim.y)) {
      int __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+1-__iter_2__))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-1-__iter_2__))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*( __iter_6__-__iter_2__))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))] = __temp_a33__;
      }
    }
  }
  __syncthreads ();
  int __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__+1),1)+2) || __iter_9__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))-2) || __iter_10__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_10__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2) || __iter_11__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_11__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))-2)) {
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))] = __tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_10__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_9__-__iter_2__))];
        }
      }
    }
  }
  int __iter_15__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ;
    for(; __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)); __iter_16__+=(int)(blockDim.y)) {
      int __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+1-__iter_2__))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-1-__iter_2__))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))] = __temp_a80__;
      }
    }
  }
  __syncthreads ();
  int __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__+2),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))-2) || __iter_19__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))-2) || __iter_20__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_20__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))-2)) {
          __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))] = __tilevar_4__[__iter_20__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_19__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_18__-__iter_2__))];
        }
      }
    }
  }
  int __iter_24__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ;
    for(; __iter_25__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)); __iter_25__+=(int)(blockDim.y)) {
      int __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+1-__iter_2__))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-1-__iter_2__))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))] = __temp_a106__;
      }
    }
  }
  __syncthreads ();
  int __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__+3),1)+2) || __iter_27__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))-2) || __iter_28__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_28__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))-2) || __iter_29__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_29__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))-2)) {
          __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))] = __tilevar_5__[__iter_29__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_28__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_27__-__iter_2__))];
        }
      }
    }
  }
  int __iter_33__ = FORMA_MAX((__iter_2__+4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__ = FORMA_MAX((__iter_1__+4),1) + (int)(threadIdx.y) ;
    for (; __iter_34__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2)); __iter_34__+=(int)(blockDim.y)) {
      int __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+1-__iter_2__))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-1-__iter_2__))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  return SMemSize;
}

/* X+GAP, Y, Z */
__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX) + (int)(FORMA_BLOCKDIM_X);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY);
  int __iter_2__ = (int)(blockIdx.z)*((int)(FORMA_BLOCKDIM_Z)+GAPZ);

  int __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1)) ; __iter_3__+=(int)(blockDim.z) ){
    int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
     for(; __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)); __iter_4__+=(int)(blockDim.y)) {
      int __iter_5__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-1)) ){
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_3__-__iter_2__))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_6__+=(int)(blockDim.z) ){
    int __iter_7__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ;
    for(; __iter_7__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)); __iter_7__+=(int)(blockDim.y)) {
      int __iter_8__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+1-__iter_2__))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-1-__iter_2__))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*( __iter_6__-__iter_2__))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))] = __temp_a33__;
      }
    }
  }
  __syncthreads();
  int __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__+1),1)+2) || __iter_9__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))-2) || __iter_10__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_10__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2)) {
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))] = __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_10__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_9__-__iter_2__))];
        }
      }
    }
  }
  __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ;
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ;
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ;
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        if (__iter_11__ < FORMA_MAX((__iter_0__-1),1) || __iter_11__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2))) {
         __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_10__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_9__-__iter_2__))] = __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_15__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ;
    for (;__iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)); __iter_16__+=(int)(blockDim.y)) {
      int __iter_17__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+1-__iter_2__))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-1-__iter_2__))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))] = __temp_a80__;
      }
    }
  }
  __syncthreads();
  int __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__+2),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))-2) || __iter_19__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))-2)) {
          __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))] = __tilevar_4__[__iter_20__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_19__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_18__-__iter_2__))];
        }
      }
    }
  }
  __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
        if (__iter_20__ < FORMA_MAX((__iter_0__-2),1) || __iter_20__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2))) {
          __tilevar_4__[__iter_20__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_19__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_18__-__iter_2__))] = __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ;
    for(; __iter_25__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)); __iter_25__+=(int)(blockDim.y)) {
      int __iter_26__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+1-__iter_2__))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-1-__iter_2__))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))] = __temp_a106__;
      }
    }
  }
  __syncthreads();
  int __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__+3),1)+2) || __iter_27__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))-2) || __iter_28__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_28__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))-2)) {
          __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))] = __tilevar_5__[__iter_29__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_28__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_27__-__iter_2__))];
        }
      }
    }
  }
  __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(N-2)) ){
        if (__iter_29__ < FORMA_MAX((__iter_0__-3),1) || __iter_29__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2))) {
          __tilevar_5__[__iter_29__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_28__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_27__-__iter_2__))] = __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__ = FORMA_MAX((__iter_2__+4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__ = FORMA_MAX((__iter_1__+4),1) + (int)(threadIdx.y) ;
    for(; __iter_34__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2)); __iter_34__+=(int)(blockDim.y)) {
      int __iter_35__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
	// __iter_34__
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+1-__iter_2__))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-1-__iter_2__))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
      }
    }
  }
}

/* X, Y+GAP, Z */
__global__ void __kernel___forma_kernel__2__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY) + (int)(FORMA_BLOCKDIM_Y);
  int __iter_2__ = (int)(blockIdx.z)*((int)(FORMA_BLOCKDIM_Z)+GAPZ);

  int __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1)) ; __iter_3__+=(int)(blockDim.z) ){
    int __iter_4__ = FORMA_MAX(__iter_1__-2,0) + (int)(threadIdx.y) ;
    for(; __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1)); __iter_4__+=(int)(blockDim.y)) {
      int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__-__iter_2__))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_6__+=(int)(blockDim.z) ){
    int __iter_7__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ;
    for(; __iter_7__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)); __iter_7__+=(int)(blockDim.y)) {
      int __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1-__iter_2__))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1-__iter_2__))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__-__iter_2__))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))] = __temp_a33__;
      }
    }
  }
  __syncthreads();
  int __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__+1),1)+2) || __iter_9__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))-2) || __iter_11__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_11__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))-2)) {
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))] = __tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_10__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_9__-__iter_2__))];
        }
      }
    }
  }
  __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        if (__iter_10__ < FORMA_MAX((__iter_1__-1),1) || __iter_10__ > FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2))) {
          __tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_10__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_9__-__iter_2__))] = __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_15__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ;
    for(; __iter_16__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2)); __iter_16__+=(int)(blockDim.y)) {
      int __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1-__iter_2__))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1-__iter_2__))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))] = __temp_a80__;
      }
    }
  }
  __syncthreads();
  int __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__+2),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))-2) || __iter_20__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_20__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))-2)) {
          __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))] = __tilevar_4__[__iter_20__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_19__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_18__-__iter_2__))];
        }
      }
    }
  }
  __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        if (__iter_19__ < FORMA_MAX((__iter_1__-2),1) || __iter_19__ > FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) {
          __tilevar_4__[__iter_20__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_19__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_18__-__iter_2__))] = __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ;
    for (; __iter_25__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)); __iter_25__+=(int)(blockDim.y)) {
      int __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1-__iter_2__))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1-__iter_2__))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))] = __temp_a106__;
      }
    }
  }
  __syncthreads();
  int __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__+3),1)+2) || __iter_27__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))-2) || __iter_29__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_29__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))-2)) {
          __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))] = __tilevar_5__[__iter_29__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_28__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_27__-__iter_2__))];
        }
      }
    }
  }
  __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__-5),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        if (__iter_28__ < FORMA_MAX((__iter_1__-3),1) || __iter_28__ > FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) {
          __tilevar_5__[__iter_29__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_28__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_27__-__iter_2__))] = __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__ = FORMA_MAX((__iter_2__+4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ;
    for(; __iter_34__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)); __iter_34__+=(int)(blockDim.y)) {
      int __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1-__iter_2__))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1-__iter_2__))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
      }
    }
  }
}

/* X, Y, Z+GAP */
__global__ void __kernel___forma_kernel__3__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY);
  int __iter_2__ = (int)(blockIdx.z)*((int)(FORMA_BLOCKDIM_Z)+GAPZ) + FORMA_BLOCKDIM_Z;

  int __iter_3__ = FORMA_MAX(__iter_2__-2,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-1)) ; __iter_3__+=(int)(blockDim.z) ){
    int __iter_4__ = FORMA_MAX(__iter_1__,0) + 4*(int)(threadIdx.y) ;
    if(__iter_4__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1))) {
      int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+1+M*(__iter_3__))];
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+2+M*(__iter_3__))];
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+(3)+M*(__iter_3__))];
      }
    }
    else if(__iter_4__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1))) {
      int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+1+M*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2)) ; __iter_6__+=(int)(blockDim.z) ){
    int __iter_7__ = FORMA_MAX((__iter_1__+1),1) + 4*(int)(threadIdx.y) ;
    if(__iter_7__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      int __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
	// __iter_7__ 
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_a33__;
	// __iter_7__ + 1 
        float __temp_b3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b8__ = (0.161000f * __temp_b3__ + 0.162000f * __temp_b7__);
        float __temp_b12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b13__ = (__temp_b8__ + 0.163000f * __temp_b12__);
        float __temp_b17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b18__ = (__temp_b13__ + 0.164000f * __temp_b17__);
        float __temp_b22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_b23__ = (__temp_b18__ + 0.165000f * __temp_b22__);
        float __temp_b27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_b28__ = (__temp_b23__ + 0.166000f * __temp_b27__);
        float __temp_b32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b33__ = (__temp_b28__ - 1.670000f * __temp_b32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_b33__;
	// __iter_7__ + 2 
        float __temp_c3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c8__ = (0.161000f * __temp_c3__ + 0.162000f * __temp_c7__);
        float __temp_c12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c13__ = (__temp_c8__ + 0.163000f * __temp_c12__);
        float __temp_c17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c18__ = (__temp_c13__ + 0.164000f * __temp_c17__);
        float __temp_c22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_c23__ = (__temp_c18__ + 0.165000f * __temp_c22__);
        float __temp_c27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_c28__ = (__temp_c23__ + 0.166000f * __temp_c27__);
        float __temp_c32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c33__ = (__temp_c28__ - 1.670000f * __temp_c32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_c33__;
	// __iter_7__ + 3 
        float __temp_d3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d8__ = (0.161000f * __temp_d3__ + 0.162000f * __temp_d7__);
        float __temp_d12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d13__ = (__temp_d8__ + 0.163000f * __temp_d12__);
        float __temp_d17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d18__ = (__temp_d13__ + 0.164000f * __temp_d17__);
        float __temp_d22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_d23__ = (__temp_d18__ + 0.165000f * __temp_d22__);
        float __temp_d27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_d28__ = (__temp_d23__ + 0.166000f * __temp_d27__);
        float __temp_d32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d33__ = (__temp_d28__ - 1.670000f * __temp_d32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_d33__;
      }
    }
    else if(__iter_7__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      int __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
	// __iter_7__ 
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_a33__;
	// __iter_7__ + 1 
        float __temp_b3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b8__ = (0.161000f * __temp_b3__ + 0.162000f * __temp_b7__);
        float __temp_b12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b13__ = (__temp_b8__ + 0.163000f * __temp_b12__);
        float __temp_b17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b18__ = (__temp_b13__ + 0.164000f * __temp_b17__);
        float __temp_b22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_b23__ = (__temp_b18__ + 0.165000f * __temp_b22__);
        float __temp_b27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_b28__ = (__temp_b23__ + 0.166000f * __temp_b27__);
        float __temp_b32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b33__ = (__temp_b28__ - 1.670000f * __temp_b32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_b33__;
	// __iter_7__ + 2 
        float __temp_c3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c8__ = (0.161000f * __temp_c3__ + 0.162000f * __temp_c7__);
        float __temp_c12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c13__ = (__temp_c8__ + 0.163000f * __temp_c12__);
        float __temp_c17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c18__ = (__temp_c13__ + 0.164000f * __temp_c17__);
        float __temp_c22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_c23__ = (__temp_c18__ + 0.165000f * __temp_c22__);
        float __temp_c27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_c28__ = (__temp_c23__ + 0.166000f * __temp_c27__);
        float __temp_c32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c33__ = (__temp_c28__ - 1.670000f * __temp_c32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_c33__;
      }
    }
    else if(__iter_7__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      int __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
	// __iter_7__ 
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_a33__;
	// __iter_7__ + 1 
        float __temp_b3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b8__ = (0.161000f * __temp_b3__ + 0.162000f * __temp_b7__);
        float __temp_b12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b13__ = (__temp_b8__ + 0.163000f * __temp_b12__);
        float __temp_b17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b18__ = (__temp_b13__ + 0.164000f * __temp_b17__);
        float __temp_b22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_b23__ = (__temp_b18__ + 0.165000f * __temp_b22__);
        float __temp_b27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_b28__ = (__temp_b23__ + 0.166000f * __temp_b27__);
        float __temp_b32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b33__ = (__temp_b28__ - 1.670000f * __temp_b32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_b33__;
      }
    }
  }
  __syncthreads();
  int __iter_9__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        if (__iter_10__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_10__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2) || __iter_11__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_11__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))-2)) {
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))] = __tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_10__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_9__+(EXTENT-__iter_2__)))];
        }
      }
    }
  }
  __iter_9__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__-1),1)) || __iter_9__ > (FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2)))) {
 	  __tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_10__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_9__+(EXTENT-__iter_2__)))] = __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_15__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__ = FORMA_MAX((__iter_1__+2),1) + 4*(int)(threadIdx.y) ; 
    if(__iter_16__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      int __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
	// __iter_16__
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_a80__;
	// __iter_16__ + 1
        float __temp_b50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b55__ = (0.161000f * __temp_b50__ + 0.162000f * __temp_b54__);
        float __temp_b59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b60__ = (__temp_b55__ + 0.163000f * __temp_b59__);
        float __temp_b64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b65__ = (__temp_b60__ + 0.164000f * __temp_b64__);
        float __temp_b69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_b70__ = (__temp_b65__ + 0.165000f * __temp_b69__);
        float __temp_b74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_b75__ = (__temp_b70__ + 0.166000f * __temp_b74__);
        float __temp_b79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b80__ = (__temp_b75__ - 1.670000f * __temp_b79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_b80__;
	// __iter_16__ + 2
        float __temp_c50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c55__ = (0.161000f * __temp_c50__ + 0.162000f * __temp_c54__);
        float __temp_c59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c60__ = (__temp_c55__ + 0.163000f * __temp_c59__);
        float __temp_c64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c65__ = (__temp_c60__ + 0.164000f * __temp_c64__);
        float __temp_c69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_c70__ = (__temp_c65__ + 0.165000f * __temp_c69__);
        float __temp_c74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_c75__ = (__temp_c70__ + 0.166000f * __temp_c74__);
        float __temp_c79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c80__ = (__temp_c75__ - 1.670000f * __temp_c79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_c80__;
	// __iter_16__ + 3
        float __temp_d50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d55__ = (0.161000f * __temp_d50__ + 0.162000f * __temp_d54__);
        float __temp_d59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d60__ = (__temp_d55__ + 0.163000f * __temp_d59__);
        float __temp_d64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d65__ = (__temp_d60__ + 0.164000f * __temp_d64__);
        float __temp_d69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_d70__ = (__temp_d65__ + 0.165000f * __temp_d69__);
        float __temp_d74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_d75__ = (__temp_d70__ + 0.166000f * __temp_d74__);
        float __temp_d79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d80__ = (__temp_d75__ - 1.670000f * __temp_d79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_d80__;
      }
    }
    else if(__iter_16__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      int __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
	// __iter_16__
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_a80__;
	// __iter_16__ + 1
        float __temp_b50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b55__ = (0.161000f * __temp_b50__ + 0.162000f * __temp_b54__);
        float __temp_b59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b60__ = (__temp_b55__ + 0.163000f * __temp_b59__);
        float __temp_b64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b65__ = (__temp_b60__ + 0.164000f * __temp_b64__);
        float __temp_b69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_b70__ = (__temp_b65__ + 0.165000f * __temp_b69__);
        float __temp_b74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_b75__ = (__temp_b70__ + 0.166000f * __temp_b74__);
        float __temp_b79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b80__ = (__temp_b75__ - 1.670000f * __temp_b79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_b80__;
	// __iter_16__ + 2
        float __temp_c50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c55__ = (0.161000f * __temp_c50__ + 0.162000f * __temp_c54__);
        float __temp_c59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c60__ = (__temp_c55__ + 0.163000f * __temp_c59__);
        float __temp_c64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c65__ = (__temp_c60__ + 0.164000f * __temp_c64__);
        float __temp_c69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_c70__ = (__temp_c65__ + 0.165000f * __temp_c69__);
        float __temp_c74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_c75__ = (__temp_c70__ + 0.166000f * __temp_c74__);
        float __temp_c79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c80__ = (__temp_c75__ - 1.670000f * __temp_c79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_c80__;
      }
    }
    else if(__iter_16__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      int __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
	// __iter_16__
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_a80__;
	// __iter_16__ + 1
        float __temp_b50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b55__ = (0.161000f * __temp_b50__ + 0.162000f * __temp_b54__);
        float __temp_b59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b60__ = (__temp_b55__ + 0.163000f * __temp_b59__);
        float __temp_b64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b65__ = (__temp_b60__ + 0.164000f * __temp_b64__);
        float __temp_b69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_b70__ = (__temp_b65__ + 0.165000f * __temp_b69__);
        float __temp_b74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_b75__ = (__temp_b70__ + 0.166000f * __temp_b74__);
        float __temp_b79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b80__ = (__temp_b75__ - 1.670000f * __temp_b79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_b80__;
      }
    }
  }
  __syncthreads();
  int __iter_18__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        if (__iter_19__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))-2) || __iter_20__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_20__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))-2)) {
          __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))] = __tilevar_4__[__iter_20__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_19__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_18__+(EXTENT-__iter_2__)))];
        }
      }
    }
  }
  __iter_18__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+GAPZ+4)-1),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__-2),1)) || __iter_18__ > (FORMA_MIN(((__iter_2__+2+GAPZ)-1),(L-2)))) {
	  __tilevar_4__[__iter_20__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_19__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_18__+(EXTENT-__iter_2__)))] = __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__ = FORMA_MAX((__iter_1__+3),1) + 4*(int)(threadIdx.y) ;
    if(__iter_25__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      int __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
	// __iter_25__ 
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_a106__;
	// __iter_25__ + 1 
        float __temp_b94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b96__ = (0.161000f * __temp_b94__ + 0.162000f * __temp_b95__);
        float __temp_b97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b98__ = (__temp_b96__ + 0.163000f * __temp_b97__);
        float __temp_b99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b100__ = (__temp_b98__ + 0.164000f * __temp_b99__);
        float __temp_b101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_b102__ = (__temp_b100__ + 0.165000f * __temp_b101__);
        float __temp_b103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_b104__ = (__temp_b102__ + 0.166000f * __temp_b103__);
        float __temp_b105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b106__ = (__temp_b104__ - 1.670000f * __temp_b105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_b106__;
	// __iter_25__ + 2
        float __temp_c94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c96__ = (0.161000f * __temp_c94__ + 0.162000f * __temp_c95__);
        float __temp_c97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c98__ = (__temp_c96__ + 0.163000f * __temp_c97__);
        float __temp_c99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c100__ = (__temp_c98__ + 0.164000f * __temp_c99__);
        float __temp_c101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_c102__ = (__temp_c100__ + 0.165000f * __temp_c101__);
        float __temp_c103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_c104__ = (__temp_c102__ + 0.166000f * __temp_c103__);
        float __temp_c105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c106__ = (__temp_c104__ - 1.670000f * __temp_c105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_c106__;
	// __iter_25__ + 3 
        float __temp_d94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d96__ = (0.161000f * __temp_d94__ + 0.162000f * __temp_d95__);
        float __temp_d97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d98__ = (__temp_d96__ + 0.163000f * __temp_d97__);
        float __temp_d99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d100__ = (__temp_d98__ + 0.164000f * __temp_d99__);
        float __temp_d101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_d102__ = (__temp_d100__ + 0.165000f * __temp_d101__);
        float __temp_d103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_d104__ = (__temp_d102__ + 0.166000f * __temp_d103__);
        float __temp_d105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d106__ = (__temp_d104__ - 1.670000f * __temp_d105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_d106__;
      }
    }
    else if(__iter_25__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      int __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
	// __iter_25__ 
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_a106__;
	// __iter_25__ + 1 
        float __temp_b94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b96__ = (0.161000f * __temp_b94__ + 0.162000f * __temp_b95__);
        float __temp_b97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b98__ = (__temp_b96__ + 0.163000f * __temp_b97__);
        float __temp_b99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b100__ = (__temp_b98__ + 0.164000f * __temp_b99__);
        float __temp_b101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_b102__ = (__temp_b100__ + 0.165000f * __temp_b101__);
        float __temp_b103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_b104__ = (__temp_b102__ + 0.166000f * __temp_b103__);
        float __temp_b105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b106__ = (__temp_b104__ - 1.670000f * __temp_b105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_b106__;
	// __iter_25__ + 2
        float __temp_c94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c96__ = (0.161000f * __temp_c94__ + 0.162000f * __temp_c95__);
        float __temp_c97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c98__ = (__temp_c96__ + 0.163000f * __temp_c97__);
        float __temp_c99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c100__ = (__temp_c98__ + 0.164000f * __temp_c99__);
        float __temp_c101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_c102__ = (__temp_c100__ + 0.165000f * __temp_c101__);
        float __temp_c103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_c104__ = (__temp_c102__ + 0.166000f * __temp_c103__);
        float __temp_c105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c106__ = (__temp_c104__ - 1.670000f * __temp_c105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_c106__;
      }
    }
    else if(__iter_25__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      int __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
	// __iter_25__ 
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_a106__;
	// __iter_25__ + 1 
        float __temp_b94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b96__ = (0.161000f * __temp_b94__ + 0.162000f * __temp_b95__);
        float __temp_b97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b98__ = (__temp_b96__ + 0.163000f * __temp_b97__);
        float __temp_b99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b100__ = (__temp_b98__ + 0.164000f * __temp_b99__);
        float __temp_b101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_b102__ = (__temp_b100__ + 0.165000f * __temp_b101__);
        float __temp_b103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_b104__ = (__temp_b102__ + 0.166000f * __temp_b103__);
        float __temp_b105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b106__ = (__temp_b104__ - 1.670000f * __temp_b105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_b106__;
      }
    }
  }
  __syncthreads();
  int __iter_27__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        if (__iter_28__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_28__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))-2) || __iter_29__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_29__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))-2)) {
          __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))] = __tilevar_5__[__iter_29__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_28__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_27__+(EXTENT-__iter_2__)))];
        }
      }
    }
  }
  __iter_27__ = FORMA_MAX((__iter_2__-5),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+GAPZ+5)-1),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__-3),1)) || __iter_27__ > (FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)))) {
	  __tilevar_5__[__iter_29__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_28__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_27__+(EXTENT-__iter_2__)))] = __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+GAPZ+4)-1),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__ = FORMA_MAX((__iter_1__+4),1) + 4*(int)(threadIdx.y) ;
    if(__iter_34__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2))) {
      int __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
	// __iter_34__
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
	// __iter_34__ + 1
        float __temp_b120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b122__ = (0.161000f * __temp_b120__ + 0.162000f * __temp_b121__);
        float __temp_b123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b124__ = (__temp_b122__ + 0.163000f * __temp_b123__);
        float __temp_b125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b126__ = (__temp_b124__ + 0.164000f * __temp_b125__);
        float __temp_b127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_b128__ = (__temp_b126__ + 0.165000f * __temp_b127__);
        float __temp_b129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_b130__ = (__temp_b128__ + 0.166000f * __temp_b129__);
        float __temp_b131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b132__ = (__temp_b130__ - 1.670000f * __temp_b131__);
        __var_1__[__iter_35__+N*(__iter_34__+1+M*(__iter_33__))] = __temp_b132__;
	// __iter_34__
        float __temp_c120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c122__ = (0.161000f * __temp_c120__ + 0.162000f * __temp_c121__);
        float __temp_c123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c124__ = (__temp_c122__ + 0.163000f * __temp_c123__);
        float __temp_c125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c126__ = (__temp_c124__ + 0.164000f * __temp_c125__);
        float __temp_c127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_c128__ = (__temp_c126__ + 0.165000f * __temp_c127__);
        float __temp_c129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_c130__ = (__temp_c128__ + 0.166000f * __temp_c129__);
        float __temp_c131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c132__ = (__temp_c130__ - 1.670000f * __temp_c131__);
        __var_1__[__iter_35__+N*(__iter_34__+2+M*(__iter_33__))] = __temp_c132__;
	// __iter_34__
        float __temp_d120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d122__ = (0.161000f * __temp_d120__ + 0.162000f * __temp_d121__);
        float __temp_d123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d124__ = (__temp_d122__ + 0.163000f * __temp_d123__);
        float __temp_d125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d126__ = (__temp_d124__ + 0.164000f * __temp_d125__);
        float __temp_d127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_d128__ = (__temp_d126__ + 0.165000f * __temp_d127__);
        float __temp_d129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_d130__ = (__temp_d128__ + 0.166000f * __temp_d129__);
        float __temp_d131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d132__ = (__temp_d130__ - 1.670000f * __temp_d131__);
        __var_1__[__iter_35__+N*(__iter_34__+(3)+M*(__iter_33__))] = __temp_d132__;
      }
    }
    else if(__iter_34__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2))) {
      int __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
	// __iter_34__
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
	// __iter_34__ + 1
        float __temp_b120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b122__ = (0.161000f * __temp_b120__ + 0.162000f * __temp_b121__);
        float __temp_b123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b124__ = (__temp_b122__ + 0.163000f * __temp_b123__);
        float __temp_b125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b126__ = (__temp_b124__ + 0.164000f * __temp_b125__);
        float __temp_b127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_b128__ = (__temp_b126__ + 0.165000f * __temp_b127__);
        float __temp_b129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_b130__ = (__temp_b128__ + 0.166000f * __temp_b129__);
        float __temp_b131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b132__ = (__temp_b130__ - 1.670000f * __temp_b131__);
        __var_1__[__iter_35__+N*(__iter_34__+1+M*(__iter_33__))] = __temp_b132__;
	// __iter_34__
        float __temp_c120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c122__ = (0.161000f * __temp_c120__ + 0.162000f * __temp_c121__);
        float __temp_c123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c124__ = (__temp_c122__ + 0.163000f * __temp_c123__);
        float __temp_c125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c126__ = (__temp_c124__ + 0.164000f * __temp_c125__);
        float __temp_c127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_c128__ = (__temp_c126__ + 0.165000f * __temp_c127__);
        float __temp_c129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_c130__ = (__temp_c128__ + 0.166000f * __temp_c129__);
        float __temp_c131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c132__ = (__temp_c130__ - 1.670000f * __temp_c131__);
        __var_1__[__iter_35__+N*(__iter_34__+2+M*(__iter_33__))] = __temp_c132__;
      }
    }
    else if(__iter_34__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2))) {
      int __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
	// __iter_34__
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
	// __iter_34__ + 1
        float __temp_b120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b122__ = (0.161000f * __temp_b120__ + 0.162000f * __temp_b121__);
        float __temp_b123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b124__ = (__temp_b122__ + 0.163000f * __temp_b123__);
        float __temp_b125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b126__ = (__temp_b124__ + 0.164000f * __temp_b125__);
        float __temp_b127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_b128__ = (__temp_b126__ + 0.165000f * __temp_b127__);
        float __temp_b129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_b130__ = (__temp_b128__ + 0.166000f * __temp_b129__);
        float __temp_b131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b132__ = (__temp_b130__ - 1.670000f * __temp_b131__);
        __var_1__[__iter_35__+N*(__iter_34__+1+M*(__iter_33__))] = __temp_b132__;
      }
    }
  }
}

/* X+GAP, Y+GAP, Z */
__global__ void __kernel___forma_kernel__4__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX) + (int)(FORMA_BLOCKDIM_X);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY) + (int)(FORMA_BLOCKDIM_Y);
  int __iter_2__ = (int)(blockIdx.z)*((int)(FORMA_BLOCKDIM_Z)+GAPZ);

  int __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
  for(; __iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1)) ; __iter_3__+=(int)(blockDim.z) ){
    int __iter_4__ = FORMA_MAX(__iter_1__-2,0) + (int)(threadIdx.y) ;
    for(; __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1)); __iter_4__+=(int)(blockDim.y)) {
      int __iter_5__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-1)) ){
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__-__iter_2__))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_6__+=(int)(blockDim.z) ){
    int __iter_7__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    for(; __iter_7__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)); __iter_7__+=(int)(blockDim.y)) {
      int __iter_8__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1-__iter_2__))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1-__iter_2__))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__-__iter_2__))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))] = __temp_a33__;
      }
    }
  }
  __syncthreads();
  int __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__+1),1)+2) || __iter_9__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))-2)) {
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))] = __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_10__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_9__-__iter_2__))];
        }
      }
    }
  }
  __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ;
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ;
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ;
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        if (__iter_10__ < FORMA_MAX((__iter_1__-1),1) || __iter_10__ > FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)) || __iter_11__ < FORMA_MAX((__iter_0__-1),1) || __iter_11__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2))) {
         __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_10__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_9__-__iter_2__))] = __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))];
        }
      }
    }
  }
 __syncthreads();
  int __iter_15__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ;
    for(; __iter_16__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2)); __iter_16__+=(int)(blockDim.y)) {
      int __iter_17__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1-__iter_2__))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1-__iter_2__))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-__iter_2__))] = __temp_a80__;
      }
    }
  }
  __syncthreads();
  int __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__+2),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))-2)) {
          __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))] = __tilevar_4__[__iter_20__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_19__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_18__-__iter_2__))];
        }
      }
    }
  }
  __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
        if (__iter_19__ < (FORMA_MAX((__iter_1__-2),1)) || __iter_19__ > (FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) || __iter_20__ < FORMA_MAX((__iter_0__-2),1) || __iter_20__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2))) {
          __tilevar_4__[__iter_20__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_19__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_18__-__iter_2__))] = __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    for(; __iter_25__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)); __iter_25__+=(int)(blockDim.y)) {
      int __iter_26__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1-__iter_2__))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1-__iter_2__))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-__iter_2__))] = __temp_a106__;
      }
    }
  }
  __syncthreads();
  int __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__+3),1)+2) || __iter_27__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))-2)) {
          __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))] = __tilevar_5__[__iter_29__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_28__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_27__-__iter_2__))];
        }
      }
    }
  }
  __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__-5),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(N-2)) ){
        if (__iter_28__ < (FORMA_MAX((__iter_1__-3),1)) || __iter_28__ > (FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) || __iter_29__ < FORMA_MAX((__iter_0__-3),1) || __iter_29__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2))) {
          __tilevar_5__[__iter_29__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_28__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_27__-__iter_2__))] = __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__ = FORMA_MAX((__iter_2__+4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
    for(; __iter_34__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)); __iter_34__+=(int)(blockDim.y)) {
      int __iter_35__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1-__iter_2__))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1-__iter_2__))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-__iter_2__))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
      }
    }
  }
}

/* X, Y+GAP, Z+GAP */
__global__ void __kernel___forma_kernel__5__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY) + (int)(FORMA_BLOCKDIM_Y);
  int __iter_2__ = (int)(blockIdx.z)*((int)(FORMA_BLOCKDIM_Z)+GAPZ) + (int)(FORMA_BLOCKDIM_Z);

  int __iter_3__ = FORMA_MAX(__iter_2__-2,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-1)) ; __iter_3__+=(int)(blockDim.z) ){
    int __iter_4__ = FORMA_MAX(__iter_1__-2,0) + 4*(int)(threadIdx.y) ;
    if(__iter_4__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1))) {
      int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+1+M*(__iter_3__))];
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+2+M*(__iter_3__))];
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+(3)+M*(__iter_3__))];
      }
    }
    else if(__iter_4__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1))) {
      int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+1+M*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2)) ; __iter_6__+=(int)(blockDim.z) ){
    int __iter_7__ = FORMA_MAX((__iter_1__-1),1) + 4*(int)(threadIdx.y) ; 
    if(__iter_7__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2))) {
      int __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
	// __iter_7__
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_a33__;
	// __iter_7__ + 1
        float __temp_b3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b8__ = (0.161000f * __temp_b3__ + 0.162000f * __temp_b7__);
        float __temp_b12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b13__ = (__temp_b8__ + 0.163000f * __temp_b12__);
        float __temp_b17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b18__ = (__temp_b13__ + 0.164000f * __temp_b17__);
        float __temp_b22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_b23__ = (__temp_b18__ + 0.165000f * __temp_b22__);
        float __temp_b27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_b28__ = (__temp_b23__ + 0.166000f * __temp_b27__);
        float __temp_b32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b33__ = (__temp_b28__ - 1.670000f * __temp_b32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_b33__;
	// __iter_7__ + 2
        float __temp_c3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c8__ = (0.161000f * __temp_c3__ + 0.162000f * __temp_c7__);
        float __temp_c12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c13__ = (__temp_c8__ + 0.163000f * __temp_c12__);
        float __temp_c17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c18__ = (__temp_c13__ + 0.164000f * __temp_c17__);
        float __temp_c22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_c23__ = (__temp_c18__ + 0.165000f * __temp_c22__);
        float __temp_c27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_c28__ = (__temp_c23__ + 0.166000f * __temp_c27__);
        float __temp_c32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c33__ = (__temp_c28__ - 1.670000f * __temp_c32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_c33__;
	// __iter_7__ + 3
        float __temp_d3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d8__ = (0.161000f * __temp_d3__ + 0.162000f * __temp_d7__);
        float __temp_d12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d13__ = (__temp_d8__ + 0.163000f * __temp_d12__);
        float __temp_d17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d18__ = (__temp_d13__ + 0.164000f * __temp_d17__);
        float __temp_d22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_d23__ = (__temp_d18__ + 0.165000f * __temp_d22__);
        float __temp_d27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_d28__ = (__temp_d23__ + 0.166000f * __temp_d27__);
        float __temp_d32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d33__ = (__temp_d28__ - 1.670000f * __temp_d32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_d33__;
      }
    }
    else if(__iter_7__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2))) {
      int __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
	// __iter_7__
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_a33__;
	// __iter_7__ + 1
        float __temp_b3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b8__ = (0.161000f * __temp_b3__ + 0.162000f * __temp_b7__);
        float __temp_b12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b13__ = (__temp_b8__ + 0.163000f * __temp_b12__);
        float __temp_b17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b18__ = (__temp_b13__ + 0.164000f * __temp_b17__);
        float __temp_b22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_b23__ = (__temp_b18__ + 0.165000f * __temp_b22__);
        float __temp_b27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_b28__ = (__temp_b23__ + 0.166000f * __temp_b27__);
        float __temp_b32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b33__ = (__temp_b28__ - 1.670000f * __temp_b32__);
        __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_b33__;
      }
    }
  }
  __syncthreads();
  int __iter_9__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        if (__iter_11__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_11__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))-2)) {
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))] = __tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_10__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_9__+(EXTENT-__iter_2__)))];
        }
      }
    }
  }
  __iter_9__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__-1),1)) || __iter_9__ > (FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2))) ||__iter_10__ < FORMA_MAX((__iter_1__-1),1) || __iter_10__ > FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2))) {
          __tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_10__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_9__+(EXTENT-__iter_2__)))] = __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_15__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__ = FORMA_MAX((__iter_1__-2),1) + 4*(int)(threadIdx.y) ; 
    if(__iter_16__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) {
      int __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
	// __iter_16__
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_a80__;
	// __iter_16__ + 1
        float __temp_b50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b55__ = (0.161000f * __temp_b50__ + 0.162000f * __temp_b54__);
        float __temp_b59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b60__ = (__temp_b55__ + 0.163000f * __temp_b59__);
        float __temp_b64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b65__ = (__temp_b60__ + 0.164000f * __temp_b64__);
        float __temp_b69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_b70__ = (__temp_b65__ + 0.165000f * __temp_b69__);
        float __temp_b74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_b75__ = (__temp_b70__ + 0.166000f * __temp_b74__);
        float __temp_b79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b80__ = (__temp_b75__ - 1.670000f * __temp_b79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_b80__;
	// __iter_16__ + 2 
        float __temp_c50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c55__ = (0.161000f * __temp_c50__ + 0.162000f * __temp_c54__);
        float __temp_c59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c60__ = (__temp_c55__ + 0.163000f * __temp_c59__);
        float __temp_c64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c65__ = (__temp_c60__ + 0.164000f * __temp_c64__);
        float __temp_c69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_c70__ = (__temp_c65__ + 0.165000f * __temp_c69__);
        float __temp_c74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_c75__ = (__temp_c70__ + 0.166000f * __temp_c74__);
        float __temp_c79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c80__ = (__temp_c75__ - 1.670000f * __temp_c79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_c80__;
	// __iter_16__ + 3
        float __temp_d50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d55__ = (0.161000f * __temp_d50__ + 0.162000f * __temp_d54__);
        float __temp_d59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d60__ = (__temp_d55__ + 0.163000f * __temp_d59__);
        float __temp_d64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d65__ = (__temp_d60__ + 0.164000f * __temp_d64__);
        float __temp_d69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_d70__ = (__temp_d65__ + 0.165000f * __temp_d69__);
        float __temp_d74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_d75__ = (__temp_d70__ + 0.166000f * __temp_d74__);
        float __temp_d79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d80__ = (__temp_d75__ - 1.670000f * __temp_d79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_d80__;
      }
    }
    else if(__iter_16__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) {
      int __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
	// __iter_16__
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_a80__;
	// __iter_16__ + 1
        float __temp_b50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b55__ = (0.161000f * __temp_b50__ + 0.162000f * __temp_b54__);
        float __temp_b59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b60__ = (__temp_b55__ + 0.163000f * __temp_b59__);
        float __temp_b64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b65__ = (__temp_b60__ + 0.164000f * __temp_b64__);
        float __temp_b69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_b70__ = (__temp_b65__ + 0.165000f * __temp_b69__);
        float __temp_b74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_b75__ = (__temp_b70__ + 0.166000f * __temp_b74__);
        float __temp_b79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b80__ = (__temp_b75__ - 1.670000f * __temp_b79__);
        __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_b80__;
      }
    }
  }
  __syncthreads();
  int __iter_18__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        if (__iter_20__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_20__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))-2)) {
          __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))] = __tilevar_4__[__iter_20__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_19__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_18__+(EXTENT-__iter_2__)))];
        }
      }
    }
  }
  __iter_18__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+GAPZ+4)-1),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__-2),1)) || __iter_18__ > (FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-2))) || __iter_19__ < FORMA_MAX((__iter_1__-2),1) || __iter_19__ > FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) {
          __tilevar_4__[__iter_20__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_19__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_18__+(EXTENT-__iter_2__)))] = __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__ = FORMA_MAX((__iter_1__-3),1) + 4*(int)(threadIdx.y) ; 
    if (__iter_25__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) {
      int __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
	// __iter_25__
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_a106__;
	// __iter_25__ + 1
        float __temp_b94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b96__ = (0.161000f * __temp_b94__ + 0.162000f * __temp_b95__);
        float __temp_b97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b98__ = (__temp_b96__ + 0.163000f * __temp_b97__);
        float __temp_b99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b100__ = (__temp_b98__ + 0.164000f * __temp_b99__);
        float __temp_b101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_b102__ = (__temp_b100__ + 0.165000f * __temp_b101__);
        float __temp_b103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_b104__ = (__temp_b102__ + 0.166000f * __temp_b103__);
        float __temp_b105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b106__ = (__temp_b104__ - 1.670000f * __temp_b105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_b106__;
	// __iter_25__ + 2
        float __temp_c94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c96__ = (0.161000f * __temp_c94__ + 0.162000f * __temp_c95__);
        float __temp_c97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c98__ = (__temp_c96__ + 0.163000f * __temp_c97__);
        float __temp_c99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c100__ = (__temp_c98__ + 0.164000f * __temp_c99__);
        float __temp_c101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_c102__ = (__temp_c100__ + 0.165000f * __temp_c101__);
        float __temp_c103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_c104__ = (__temp_c102__ + 0.166000f * __temp_c103__);
        float __temp_c105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c106__ = (__temp_c104__ - 1.670000f * __temp_c105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_c106__;
	// __iter_25__ + 3
        float __temp_d94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d96__ = (0.161000f * __temp_d94__ + 0.162000f * __temp_d95__);
        float __temp_d97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d98__ = (__temp_d96__ + 0.163000f * __temp_d97__);
        float __temp_d99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d100__ = (__temp_d98__ + 0.164000f * __temp_d99__);
        float __temp_d101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_d102__ = (__temp_d100__ + 0.165000f * __temp_d101__);
        float __temp_d103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_d104__ = (__temp_d102__ + 0.166000f * __temp_d103__);
        float __temp_d105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d106__ = (__temp_d104__ - 1.670000f * __temp_d105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_d106__;
      }
    }
    else if (__iter_25__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) {
      int __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
	// __iter_25__
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_a106__;
	// __iter_25__ + 1
        float __temp_b94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b96__ = (0.161000f * __temp_b94__ + 0.162000f * __temp_b95__);
        float __temp_b97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b98__ = (__temp_b96__ + 0.163000f * __temp_b97__);
        float __temp_b99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b100__ = (__temp_b98__ + 0.164000f * __temp_b99__);
        float __temp_b101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_b102__ = (__temp_b100__ + 0.165000f * __temp_b101__);
        float __temp_b103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_b104__ = (__temp_b102__ + 0.166000f * __temp_b103__);
        float __temp_b105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b106__ = (__temp_b104__ - 1.670000f * __temp_b105__);
        __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_b106__;
      }
    }
  }
  __syncthreads();
  int __iter_27__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        if (__iter_29__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_29__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))-2)) {
          __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))] = __tilevar_5__[__iter_29__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_28__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_27__+(EXTENT-__iter_2__)))];
        }
      }
    }
  }
  __iter_27__ = FORMA_MAX((__iter_2__-5),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+GAPZ+5)-1),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__-5),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__-3),1)) || __iter_27__ > (FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2))) || __iter_28__ < FORMA_MAX((__iter_1__-3),1) || __iter_28__ > FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) {
          __tilevar_5__[__iter_29__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_28__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_27__+(EXTENT-__iter_2__)))] = __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+GAPZ+4)-1),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__ = FORMA_MAX((__iter_1__-4),1) + 4*(int)(threadIdx.y) ; 
    if(__iter_34__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2))) {
      int __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
        // __iter_34__
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
	// __iter_34__ + 1
        float __temp_b120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b122__ = (0.161000f * __temp_b120__ + 0.162000f * __temp_b121__);
        float __temp_b123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b124__ = (__temp_b122__ + 0.163000f * __temp_b123__);
        float __temp_b125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b126__ = (__temp_b124__ + 0.164000f * __temp_b125__);
        float __temp_b127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_b128__ = (__temp_b126__ + 0.165000f * __temp_b127__);
        float __temp_b129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_b130__ = (__temp_b128__ + 0.166000f * __temp_b129__);
        float __temp_b131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b132__ = (__temp_b130__ - 1.670000f * __temp_b131__);
        __var_1__[__iter_35__+N*(__iter_34__+1+M*(__iter_33__))] = __temp_b132__;
	// __iter_34__ + 2
        float __temp_c120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c122__ = (0.161000f * __temp_c120__ + 0.162000f * __temp_c121__);
        float __temp_c123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c124__ = (__temp_c122__ + 0.163000f * __temp_c123__);
        float __temp_c125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c126__ = (__temp_c124__ + 0.164000f * __temp_c125__);
        float __temp_c127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_c128__ = (__temp_c126__ + 0.165000f * __temp_c127__);
        float __temp_c129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_c130__ = (__temp_c128__ + 0.166000f * __temp_c129__);
        float __temp_c131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+2+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c132__ = (__temp_c130__ - 1.670000f * __temp_c131__);
        __var_1__[__iter_35__+N*(__iter_34__+2+M*(__iter_33__))] = __temp_c132__;
	// __iter_34__ + 3
        float __temp_d120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d122__ = (0.161000f * __temp_d120__ + 0.162000f * __temp_d121__);
        float __temp_d123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d124__ = (__temp_d122__ + 0.163000f * __temp_d123__);
        float __temp_d125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d126__ = (__temp_d124__ + 0.164000f * __temp_d125__);
        float __temp_d127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_d128__ = (__temp_d126__ + 0.165000f * __temp_d127__);
        float __temp_d129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_d130__ = (__temp_d128__ + 0.166000f * __temp_d129__);
        float __temp_d131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(3)+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d132__ = (__temp_d130__ - 1.670000f * __temp_d131__);
        __var_1__[__iter_35__+N*(__iter_34__+(3)+M*(__iter_33__))] = __temp_d132__;
      }
    }
    else if(__iter_34__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2))) {
      int __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
        // __iter_34__
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
	// __iter_34__ + 1
        float __temp_b120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b122__ = (0.161000f * __temp_b120__ + 0.162000f * __temp_b121__);
        float __temp_b123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b124__ = (__temp_b122__ + 0.163000f * __temp_b123__);
        float __temp_b125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b126__ = (__temp_b124__ + 0.164000f * __temp_b125__);
        float __temp_b127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_b128__ = (__temp_b126__ + 0.165000f * __temp_b127__);
        float __temp_b129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_b130__ = (__temp_b128__ + 0.166000f * __temp_b129__);
        float __temp_b131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b132__ = (__temp_b130__ - 1.670000f * __temp_b131__);
        __var_1__[__iter_35__+N*(__iter_34__+1+M*(__iter_33__))] = __temp_b132__;
      }
    }
  }
}

/* X+GAP, Y, Z+GAP */
__global__ void __kernel___forma_kernel__6__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX) + FORMA_BLOCKDIM_X;
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY);
  int __iter_2__ = (int)(blockIdx.z)*((int)(FORMA_BLOCKDIM_Z)+GAPZ) + FORMA_BLOCKDIM_Z;

  int __iter_3__ = FORMA_MAX(__iter_2__-2,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-1)) ; __iter_3__+=(int)(blockDim.z) ){
    int __iter_4__ = FORMA_MAX(__iter_1__,0) + 4*(int)(threadIdx.y) ;
    if(__iter_4__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1))) {
      int __iter_5__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-1)) ){
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+1+M*(__iter_3__))];
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+2+M*(__iter_3__))];
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+(3)+M*(__iter_3__))];
      }
    }
    else if(__iter_4__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1))) {
      int __iter_5__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-1)) ){
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+1+M*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2)) ; __iter_6__+=(int)(blockDim.z) ){
    int __iter_7__ = FORMA_MAX((__iter_1__+1),1) + 4*(int)(threadIdx.y) ;
    if(__iter_7__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      int __iter_8__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
	// __iter_7__ 
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_a33__;
	// __iter_7__ + 1 
        float __temp_b3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b8__ = (0.161000f * __temp_b3__ + 0.162000f * __temp_b7__);
        float __temp_b12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b13__ = (__temp_b8__ + 0.163000f * __temp_b12__);
        float __temp_b17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b18__ = (__temp_b13__ + 0.164000f * __temp_b17__);
        float __temp_b22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_b23__ = (__temp_b18__ + 0.165000f * __temp_b22__);
        float __temp_b27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_b28__ = (__temp_b23__ + 0.166000f * __temp_b27__);
        float __temp_b32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b33__ = (__temp_b28__ - 1.670000f * __temp_b32__);
        __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_b33__;
	// __iter_7__ + 2 
        float __temp_c3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c8__ = (0.161000f * __temp_c3__ + 0.162000f * __temp_c7__);
        float __temp_c12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c13__ = (__temp_c8__ + 0.163000f * __temp_c12__);
        float __temp_c17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c18__ = (__temp_c13__ + 0.164000f * __temp_c17__);
        float __temp_c22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_c23__ = (__temp_c18__ + 0.165000f * __temp_c22__);
        float __temp_c27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_c28__ = (__temp_c23__ + 0.166000f * __temp_c27__);
        float __temp_c32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_c33__ = (__temp_c28__ - 1.670000f * __temp_c32__);
        __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_c33__;
	// __iter_7__ + 3 
        float __temp_d3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d8__ = (0.161000f * __temp_d3__ + 0.162000f * __temp_d7__);
        float __temp_d12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d13__ = (__temp_d8__ + 0.163000f * __temp_d12__);
        float __temp_d17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d18__ = (__temp_d13__ + 0.164000f * __temp_d17__);
        float __temp_d22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_d23__ = (__temp_d18__ + 0.165000f * __temp_d22__);
        float __temp_d27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_d28__ = (__temp_d23__ + 0.166000f * __temp_d27__);
        float __temp_d32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_d33__ = (__temp_d28__ - 1.670000f * __temp_d32__);
        __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_d33__;
      }
    }
    else if(__iter_7__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      int __iter_8__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
	// __iter_7__ 
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_a33__;
	// __iter_7__ + 1 
        float __temp_b3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b8__ = (0.161000f * __temp_b3__ + 0.162000f * __temp_b7__);
        float __temp_b12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b13__ = (__temp_b8__ + 0.163000f * __temp_b12__);
        float __temp_b17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b18__ = (__temp_b13__ + 0.164000f * __temp_b17__);
        float __temp_b22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_b23__ = (__temp_b18__ + 0.165000f * __temp_b22__);
        float __temp_b27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_b28__ = (__temp_b23__ + 0.166000f * __temp_b27__);
        float __temp_b32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_b33__ = (__temp_b28__ - 1.670000f * __temp_b32__);
        __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_b33__;
      }
    }
  }
  __syncthreads();
  int __iter_9__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
        if (__iter_10__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_10__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2)) {
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))] = __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_10__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_9__+(EXTENT-__iter_2__)))];
        }
      }
    }
  }
  __iter_9__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__-1),1)) || __iter_9__ > (FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2))) || __iter_11__ < (FORMA_MAX((__iter_0__-1),1)) || __iter_11__ > (FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)))) {
 	  __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_10__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_9__+(EXTENT-__iter_2__)))] = __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_15__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__ = FORMA_MAX((__iter_1__+2),1) + 4*(int)(threadIdx.y) ;
    if (__iter_16__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      int __iter_17__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
	// __iter_16__ 
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_a80__;
	// __iter_16__ + 1 
        float __temp_b50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b55__ = (0.161000f * __temp_b50__ + 0.162000f * __temp_b54__);
        float __temp_b59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b60__ = (__temp_b55__ + 0.163000f * __temp_b59__);
        float __temp_b64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b65__ = (__temp_b60__ + 0.164000f * __temp_b64__);
        float __temp_b69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_b70__ = (__temp_b65__ + 0.165000f * __temp_b69__);
        float __temp_b74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_b75__ = (__temp_b70__ + 0.166000f * __temp_b74__);
        float __temp_b79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b80__ = (__temp_b75__ - 1.670000f * __temp_b79__);
        __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_b80__;
	// __iter_16__ + 2
        float __temp_c50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c55__ = (0.161000f * __temp_c50__ + 0.162000f * __temp_c54__);
        float __temp_c59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c60__ = (__temp_c55__ + 0.163000f * __temp_c59__);
        float __temp_c64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c65__ = (__temp_c60__ + 0.164000f * __temp_c64__);
        float __temp_c69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_c70__ = (__temp_c65__ + 0.165000f * __temp_c69__);
        float __temp_c74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_c75__ = (__temp_c70__ + 0.166000f * __temp_c74__);
        float __temp_c79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_c80__ = (__temp_c75__ - 1.670000f * __temp_c79__);
        __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_c80__;
	// __iter_16__ + 3
        float __temp_d50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d55__ = (0.161000f * __temp_d50__ + 0.162000f * __temp_d54__);
        float __temp_d59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d60__ = (__temp_d55__ + 0.163000f * __temp_d59__);
        float __temp_d64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d65__ = (__temp_d60__ + 0.164000f * __temp_d64__);
        float __temp_d69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_d70__ = (__temp_d65__ + 0.165000f * __temp_d69__);
        float __temp_d74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_d75__ = (__temp_d70__ + 0.166000f * __temp_d74__);
        float __temp_d79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_d80__ = (__temp_d75__ - 1.670000f * __temp_d79__);
        __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_d80__;
      }
    }
    else if (__iter_16__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      int __iter_17__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
	// __iter_16__ 
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_a80__;
	// __iter_16__ + 1 
        float __temp_b50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b55__ = (0.161000f * __temp_b50__ + 0.162000f * __temp_b54__);
        float __temp_b59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b60__ = (__temp_b55__ + 0.163000f * __temp_b59__);
        float __temp_b64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b65__ = (__temp_b60__ + 0.164000f * __temp_b64__);
        float __temp_b69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_b70__ = (__temp_b65__ + 0.165000f * __temp_b69__);
        float __temp_b74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_b75__ = (__temp_b70__ + 0.166000f * __temp_b74__);
        float __temp_b79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_b80__ = (__temp_b75__ - 1.670000f * __temp_b79__);
        __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_b80__;
      }
    }
  }
  __syncthreads();
  int __iter_18__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
        if (__iter_19__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))-2)) {
          __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))] = __tilevar_4__[__iter_20__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_19__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_18__+(EXTENT-__iter_2__)))];
        }
      }
    }
  }
  __iter_18__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+GAPZ+4)-1),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__-2),1)) || __iter_18__ > (FORMA_MIN(((__iter_2__+2+GAPZ)-1),(L-2))) || __iter_20__ < (FORMA_MAX((__iter_0__-2),1)) || __iter_20__ > (FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)))) {
	  __tilevar_4__[__iter_20__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_19__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_18__+(EXTENT-__iter_2__)))] = __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__ = FORMA_MAX((__iter_1__+3),1) + 4*(int)(threadIdx.y) ; 
    if(__iter_25__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      int __iter_26__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
	// __iter_25__
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_a106__;
	// __iter_25__ + 1
        float __temp_b94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b96__ = (0.161000f * __temp_b94__ + 0.162000f * __temp_b95__);
        float __temp_b97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b98__ = (__temp_b96__ + 0.163000f * __temp_b97__);
        float __temp_b99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b100__ = (__temp_b98__ + 0.164000f * __temp_b99__);
        float __temp_b101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_b102__ = (__temp_b100__ + 0.165000f * __temp_b101__);
        float __temp_b103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_b104__ = (__temp_b102__ + 0.166000f * __temp_b103__);
        float __temp_b105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b106__ = (__temp_b104__ - 1.670000f * __temp_b105__);
        __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_b106__;
	// __iter_25__ + 2
        float __temp_c94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c96__ = (0.161000f * __temp_c94__ + 0.162000f * __temp_c95__);
        float __temp_c97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c98__ = (__temp_c96__ + 0.163000f * __temp_c97__);
        float __temp_c99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c100__ = (__temp_c98__ + 0.164000f * __temp_c99__);
        float __temp_c101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_c102__ = (__temp_c100__ + 0.165000f * __temp_c101__);
        float __temp_c103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_c104__ = (__temp_c102__ + 0.166000f * __temp_c103__);
        float __temp_c105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_c106__ = (__temp_c104__ - 1.670000f * __temp_c105__);
        __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_c106__;
	// __iter_25__ + 3
        float __temp_d94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d96__ = (0.161000f * __temp_d94__ + 0.162000f * __temp_d95__);
        float __temp_d97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d98__ = (__temp_d96__ + 0.163000f * __temp_d97__);
        float __temp_d99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d100__ = (__temp_d98__ + 0.164000f * __temp_d99__);
        float __temp_d101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_d102__ = (__temp_d100__ + 0.165000f * __temp_d101__);
        float __temp_d103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_d104__ = (__temp_d102__ + 0.166000f * __temp_d103__);
        float __temp_d105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_d106__ = (__temp_d104__ - 1.670000f * __temp_d105__);
        __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_d106__;
      }
    }
    else if(__iter_25__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      int __iter_26__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
	// __iter_25__
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_a106__;
	// __iter_25__ + 1
        float __temp_b94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b96__ = (0.161000f * __temp_b94__ + 0.162000f * __temp_b95__);
        float __temp_b97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b98__ = (__temp_b96__ + 0.163000f * __temp_b97__);
        float __temp_b99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b100__ = (__temp_b98__ + 0.164000f * __temp_b99__);
        float __temp_b101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_b102__ = (__temp_b100__ + 0.165000f * __temp_b101__);
        float __temp_b103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_b104__ = (__temp_b102__ + 0.166000f * __temp_b103__);
        float __temp_b105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_b106__ = (__temp_b104__ - 1.670000f * __temp_b105__);
        __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_b106__;
      }
    }
  }
  __syncthreads();
  int __iter_27__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        if (__iter_28__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_28__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))-2)) {
          __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))] = __tilevar_5__[__iter_29__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_28__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_27__+(EXTENT-__iter_2__)))];
        }
      }
    }
  }
  __iter_27__ = FORMA_MAX((__iter_2__-5),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+GAPZ+5)-1),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__-3),1)) || __iter_27__ > (FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2))) || __iter_29__ < (FORMA_MAX((__iter_0__-3),1)) || __iter_29__ > (FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)))) {
	  __tilevar_5__[__iter_29__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_28__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_27__+(EXTENT-__iter_2__)))] = __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+GAPZ+4)-1),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__ = FORMA_MAX((__iter_1__+4),1) + 4*(int)(threadIdx.y) ; 
    if(__iter_34__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2))) {
      int __iter_35__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
	// __iter_34__ 
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
	// __iter_34__ + 1 
        float __temp_b120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b122__ = (0.161000f * __temp_b120__ + 0.162000f * __temp_b121__);
        float __temp_b123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b124__ = (__temp_b122__ + 0.163000f * __temp_b123__);
        float __temp_b125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b126__ = (__temp_b124__ + 0.164000f * __temp_b125__);
        float __temp_b127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_b128__ = (__temp_b126__ + 0.165000f * __temp_b127__);
        float __temp_b129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_b130__ = (__temp_b128__ + 0.166000f * __temp_b129__);
        float __temp_b131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b132__ = (__temp_b130__ - 1.670000f * __temp_b131__);
        __var_1__[__iter_35__+N*(__iter_34__+1+M*(__iter_33__))] = __temp_b132__;
	// __iter_34__ + 2 
        float __temp_c120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c122__ = (0.161000f * __temp_c120__ + 0.162000f * __temp_c121__);
        float __temp_c123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c124__ = (__temp_c122__ + 0.163000f * __temp_c123__);
        float __temp_c125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c126__ = (__temp_c124__ + 0.164000f * __temp_c125__);
        float __temp_c127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_c128__ = (__temp_c126__ + 0.165000f * __temp_c127__);
        float __temp_c129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_c130__ = (__temp_c128__ + 0.166000f * __temp_c129__);
        float __temp_c131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(2-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_c132__ = (__temp_c130__ - 1.670000f * __temp_c131__);
        __var_1__[__iter_35__+N*(__iter_34__+2+M*(__iter_33__))] = __temp_c132__;
	// __iter_34__ + 3 
        float __temp_d120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d122__ = (0.161000f * __temp_d120__ + 0.162000f * __temp_d121__);
        float __temp_d123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d124__ = (__temp_d122__ + 0.163000f * __temp_d123__);
        float __temp_d125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d126__ = (__temp_d124__ + 0.164000f * __temp_d125__);
        float __temp_d127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_d128__ = (__temp_d126__ + 0.165000f * __temp_d127__);
        float __temp_d129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_d130__ = (__temp_d128__ + 0.166000f * __temp_d129__);
        float __temp_d131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(3-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_d132__ = (__temp_d130__ - 1.670000f * __temp_d131__);
        __var_1__[__iter_35__+N*(__iter_34__+(3)+M*(__iter_33__))] = __temp_d132__;
      }
    }
    else if(__iter_34__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2))) {
      int __iter_35__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
	// __iter_34__ 
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
	// __iter_34__ + 1 
        float __temp_b120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b122__ = (0.161000f * __temp_b120__ + 0.162000f * __temp_b121__);
        float __temp_b123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b124__ = (__temp_b122__ + 0.163000f * __temp_b123__);
        float __temp_b125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b126__ = (__temp_b124__ + 0.164000f * __temp_b125__);
        float __temp_b127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_b128__ = (__temp_b126__ + 0.165000f * __temp_b127__);
        float __temp_b129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_b130__ = (__temp_b128__ + 0.166000f * __temp_b129__);
        float __temp_b131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(1-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_b132__ = (__temp_b130__ - 1.670000f * __temp_b131__);
        __var_1__[__iter_35__+N*(__iter_34__+1+M*(__iter_33__))] = __temp_b132__;
      }
    }
  }
}

/* X+GAP, Y+GAP, Z+GAP */
__global__ void __kernel___forma_kernel__7__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX) + (int)FORMA_BLOCKDIM_X;
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY) + (int)FORMA_BLOCKDIM_Y;
  int __iter_2__ = (int)(blockIdx.z)*((int)(FORMA_BLOCKDIM_Z)+GAPZ) + (int)FORMA_BLOCKDIM_Z;

  int __iter_3__ = FORMA_MAX(__iter_2__-2,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-1)) ; __iter_3__+=(int)(blockDim.z) ){
    int __iter_4__ = FORMA_MAX(__iter_1__-2,0) + (int)(threadIdx.y) ;
    for(; __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1)); __iter_4__+=(int)(blockDim.y)) {
      int __iter_5__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-1)) ){
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_3__+(EXTENT-__iter_2__)))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2)) ; __iter_6__+=(int)(blockDim.z) ){
    int __iter_7__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ;
    for (; __iter_7__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)); __iter_7__+=(int)(blockDim.y)) {
      int __iter_8__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
        float __temp_a3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a8__ = (0.161000f * __temp_a3__ + 0.162000f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a13__ = (__temp_a8__ + 0.163000f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a18__ = (__temp_a13__ + 0.164000f * __temp_a17__);
        float __temp_a22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+1+(EXTENT-__iter_2__)))]);
        float __temp_a23__ = (__temp_a18__ + 0.165000f * __temp_a22__);
        float __temp_a27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__-1+(EXTENT-__iter_2__)))]);
        float __temp_a28__ = (__temp_a23__ + 0.166000f * __temp_a27__);
        float __temp_a32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*( __iter_6__+(EXTENT-__iter_2__)))]);
        float __temp_a33__ = (__temp_a28__ - 1.670000f * __temp_a32__);
        __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_6__+(EXTENT-__iter_2__)))] = __temp_a33__;
      }
    }
  }
  __syncthreads();
  int __iter_9__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    for(; __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
      int __iter_11__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__-1),1)) || __iter_9__ > (FORMA_MIN(((__iter_2__+GAPZ+1)-1),(L-2))) || __iter_10__ < (FORMA_MAX((__iter_1__-1),1)) || __iter_10__ > (FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2))) || __iter_11__ < (FORMA_MAX((__iter_0__-1),1)) || __iter_11__ > (FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)))) {
 	  __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_10__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_9__+(EXTENT-__iter_2__)))] = __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_15__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    for(; __iter_16__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2)); __iter_16__+=(int)(blockDim.y)) {
      int __iter_17__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
        float __temp_a50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a55__ = (0.161000f * __temp_a50__ + 0.162000f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a60__ = (__temp_a55__ + 0.163000f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a65__ = (__temp_a60__ + 0.164000f * __temp_a64__);
        float __temp_a69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+1+(EXTENT-__iter_2__)))]);
        float __temp_a70__ = (__temp_a65__ + 0.165000f * __temp_a69__);
        float __temp_a74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__-1+(EXTENT-__iter_2__)))]);
        float __temp_a75__ = (__temp_a70__ + 0.166000f * __temp_a74__);
        float __temp_a79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))]);
        float __temp_a80__ = (__temp_a75__ - 1.670000f * __temp_a79__);
        __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_15__+(EXTENT-__iter_2__)))] = __temp_a80__;
      }
    }
  }
  __syncthreads();
  int __iter_18__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+GAPZ+4)-1),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
    for(; __iter_19__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
      int __iter_20__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__-2),1)) || __iter_18__ > (FORMA_MIN(((__iter_2__+GAPZ+2)-1),(L-2))) || __iter_19__ < (FORMA_MAX((__iter_1__-2),1)) || __iter_19__ > (FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) || __iter_20__ < (FORMA_MAX((__iter_0__-2),1)) || __iter_20__ > (FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)))) {
	  __tilevar_4__[__iter_20__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_19__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_18__+(EXTENT-__iter_2__)))] = __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ;
    for(; __iter_25__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)); __iter_25__+=(int)(blockDim.y)) {
      int __iter_26__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        float __temp_a94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a96__ = (0.161000f * __temp_a94__ + 0.162000f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a98__ = (__temp_a96__ + 0.163000f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a100__ = (__temp_a98__ + 0.164000f * __temp_a99__);
        float __temp_a101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+1+(EXTENT-__iter_2__)))]);
        float __temp_a102__ = (__temp_a100__ + 0.165000f * __temp_a101__);
        float __temp_a103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__-1+(EXTENT-__iter_2__)))]);
        float __temp_a104__ = (__temp_a102__ + 0.166000f * __temp_a103__);
        float __temp_a105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))]);
        float __temp_a106__ = (__temp_a104__ - 1.670000f * __temp_a105__);
        __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_24__+(EXTENT-__iter_2__)))] = __temp_a106__;
      }
    }
  }
  __syncthreads();
  int __iter_27__ = FORMA_MAX((__iter_2__-5),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+GAPZ+5)-1),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__-5),1) + (int)(threadIdx.y) ; 
    for(; __iter_28__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
      int __iter_29__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__-3),1)) || __iter_27__ > (FORMA_MIN(((__iter_2__+GAPZ+3)-1),(L-2))) || __iter_28__ < (FORMA_MAX((__iter_1__-3),1)) || __iter_28__ > (FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) || __iter_29__ < (FORMA_MAX((__iter_0__-3),1)) || __iter_29__ > (FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)))) {
	  __tilevar_5__[__iter_29__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_28__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_27__+(EXTENT-__iter_2__)))] = __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+GAPZ+4)-1),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
    for(; __iter_34__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)); __iter_34__+=(int)(blockDim.y)) {
      int __iter_35__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
	// __iter_34__
        float __temp_a120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a122__ = (0.161000f * __temp_a120__ + 0.162000f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a124__ = (__temp_a122__ + 0.163000f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a126__ = (__temp_a124__ + 0.164000f * __temp_a125__);
        float __temp_a127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+1+(EXTENT-__iter_2__)))]);
        float __temp_a128__ = (__temp_a126__ + 0.165000f * __temp_a127__);
        float __temp_a129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__-1+(EXTENT-__iter_2__)))]);
        float __temp_a130__ = (__temp_a128__ + 0.166000f * __temp_a129__);
        float __temp_a131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*(__iter_33__+(EXTENT-__iter_2__)))]);
        float __temp_a132__ = (__temp_a130__ - 1.670000f * __temp_a131__);
        __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d7pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*(L*M*N), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_2___kernel___forma_kernel__0__ = ((L-1) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 16;
  int __block_2___kernel___forma_kernel__0__ = 12;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y+GAPY);
  int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.z+GAPZ);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 unrollConfig (__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/4, __blockConfig___kernel___forma_kernel__0__.z/3);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");

  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  __kernel___forma_kernel__4__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__4__\n");

  __kernel___forma_kernel__5__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__5__\n");

  __kernel___forma_kernel__6__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__6__\n");

  __kernel___forma_kernel__7__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__7__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

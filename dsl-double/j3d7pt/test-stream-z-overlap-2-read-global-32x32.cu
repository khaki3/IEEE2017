#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
/* X, Y, Z */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);

  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)-2);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)-2);
  float b3=0.0f, t3=0.0f;

  // Initialize the values
  int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
  int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
  // Rest of the computation
  for (int __iter_2__ = 1; __iter_2__ < L-1; __iter_2__++) {
    if(__iter_4__ >= FORMA_MAX((__iter_1__),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2))) {
        float __temp_a3__ = input[__iter_5__+1+N*(__iter_4__+M*__iter_2__)]; 
        float __temp_a7__ = input[__iter_5__-1+N*(__iter_4__+M*__iter_2__)];
        float __temp_a8__ = (0.161f * __temp_a3__ + 0.162f * __temp_a7__);
        float __temp_a12__ = input[__iter_5__+N*(__iter_4__+1+M*__iter_2__)];
        float __temp_a13__ = (__temp_a8__ + 0.163f * __temp_a12__);
        float __temp_a17__ = input[__iter_5__+N*(__iter_4__-1+M*__iter_2__)];
        float __temp_a18__ = (__temp_a13__ + 0.164f * __temp_a17__);
        float __temp_a23__ = __temp_a18__ + 0.165f * input[__iter_5__+N*(__iter_4__+M*(__iter_2__+1))];
        float __temp_a28__ = __temp_a23__ + 0.166f * input[__iter_5__+N*(__iter_4__+M*(__iter_2__-1))];
        float __temp_a32__ = input[__iter_5__+N*(__iter_4__+M*__iter_2__)];
        float __temp_a33__ = (__temp_a28__ - 1.670f * __temp_a32__);
	b3 = __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]; 
        __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t3;
        t3 = __temp_a33__;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+1),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))) {
        float __temp_a50__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a54__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a55__ = (0.161f * __temp_a50__ + 0.162f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_a60__ = (__temp_a55__ + 0.163f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_a65__ = (__temp_a60__ + 0.164f * __temp_a64__);
        float __temp_a70__ = (__temp_a65__ + 0.165f * t3);
        float __temp_a75__ = (__temp_a70__ + 0.166f * b3);
        float __temp_a79__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a80__ = (__temp_a75__ - 1.670f * __temp_a79__);
        __var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-1,0))] = __temp_a80__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(1*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d7pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*(L*M*N), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = N;
  int __size_1___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 32;
  int __block_2___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-2);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y-2);
  int __grid_2___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 unrollConfig (__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_2__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

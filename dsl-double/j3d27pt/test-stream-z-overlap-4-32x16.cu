#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
/* X, Y, Z */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);

  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)-8);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)-8);
  float t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f, out = 0.0f;
  float b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;

  // Initialize the values
  int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
  int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
      __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = 0.0f;
      __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = 0.0f;
      __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = 0.0f;
  }
  // Rest of the computation
  for (int __iter_2__ = 0; __iter_2__ <= L-1; __iter_2__++) {
    if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1))){
        __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = input[__iter_5__+N*(__iter_4__+M*(__iter_2__))]; 
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))){
	// Bottom 
	float __temp_3__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_7__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_8__ = (0.5f * __temp_3__ + 0.7f *__temp_7__);
        float __temp_12__ = (__tilevar_0__[ __iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_13__ = (__temp_8__ + 0.9f * __temp_12__);
        float __temp_17__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_18__ = (__temp_13__ + 1.2f * __temp_17__);
        float __temp_22__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_23__ = (__temp_18__ + 1.5f * __temp_22__);
        float __temp_27__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_28__ = (__temp_23__ + 1.2f * __temp_27__);
        float __temp_32__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_33__ = (__temp_28__ + 0.9f * __temp_32__);
        float __temp_37__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_38__ = (__temp_33__ + 0.7f * __temp_37__);
        float __temp_42__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_43__ = (__temp_38__ + 0.5f * __temp_42__) / 159;
	t2 += __temp_43__;
	//Mid 
        float __temp_47__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_52__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_53__ = (0.51f * __temp_47__ + 0.71f * __temp_52__);
        float __temp_57__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_58__ = (__temp_53__ + 0.91f * __temp_57__);
        float __temp_62__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_63__ = (__temp_58__ + 1.21f * __temp_62__);
        float __temp_67__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_68__ = (__temp_63__ + 1.51f * __temp_67__);
        float __temp_72__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_73__ = (__temp_68__ + 1.21f * __temp_72__);
        float __temp_77__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_78__ = (__temp_73__ + 0.91f * __temp_77__);
        float __temp_82__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_83__ = (__temp_78__ + 0.71f * __temp_82__);
        float __temp_87__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_88__ = (__temp_83__ + 0.51f * __temp_87__) / 159;
	b2 += __temp_88__;	
	// Top
        float __temp_92__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_97__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_98__ = (0.52f * __temp_92__ + 0.72f * __temp_97__);
        float __temp_102__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_103__ = (__temp_98__ + 0.92f * __temp_102__);
        float __temp_107__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_108__ = (__temp_103__ + 1.22f * __temp_107__);
        float __temp_112__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_113__ = (__temp_108__ + 1.52f * __temp_112__);
        float __temp_117__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_118__ = (__temp_113__ + 1.22f * __temp_117__);
        float __temp_122__ = (__tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_123__ = (__temp_118__ + 0.92f * __temp_122__);
        float __temp_127__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_128__ = (__temp_123__ + 0.72f * __temp_127__);
        float __temp_132__ = (__tilevar_0__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_133__ = (__temp_128__ + 0.52f * __temp_132__) / 159;
        __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] += __temp_133__;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+2),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))){
	float __temp_3__ = (__tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_7__ = (__tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_8__ = (0.5f * __temp_3__ + 0.7f *__temp_7__);
        float __temp_12__ = (__tilevar_1__[ __iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_13__ = (__temp_8__ + 0.9f * __temp_12__);
        float __temp_17__ = (__tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_18__ = (__temp_13__ + 1.2f * __temp_17__);
        float __temp_22__ = (__tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_23__ = (__temp_18__ + 1.5f * __temp_22__);
        float __temp_27__ = (__tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_28__ = (__temp_23__ + 1.2f * __temp_27__);
        float __temp_32__ = (__tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_33__ = (__temp_28__ + 0.9f * __temp_32__);
        float __temp_37__ = (__tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_38__ = (__temp_33__ + 0.7f * __temp_37__);
        float __temp_42__ = (__tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_43__ = (__temp_38__ + 0.5f * __temp_42__) / 159;
	t3 += __temp_43__;
	//Mid 
        float __temp_47__ = (__tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_52__ = (__tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_53__ = (0.51f * __temp_47__ + 0.71f * __temp_52__);
        float __temp_57__ = (__tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_58__ = (__temp_53__ + 0.91f * __temp_57__);
        float __temp_62__ = (__tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_63__ = (__temp_58__ + 1.21f * __temp_62__);
        float __temp_67__ = (__tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_68__ = (__temp_63__ + 1.51f * __temp_67__);
        float __temp_72__ = (__tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_73__ = (__temp_68__ + 1.21f * __temp_72__);
        float __temp_77__ = (__tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_78__ = (__temp_73__ + 0.91f * __temp_77__);
        float __temp_82__ = (__tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_83__ = (__temp_78__ + 0.71f * __temp_82__);
        float __temp_87__ = (__tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_88__ = (__temp_83__ + 0.51f * __temp_87__) / 159;
	b3 += __temp_88__;	
	// Top
        float __temp_92__ = (__tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_97__ = (__tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_98__ = (0.52f * __temp_92__ + 0.72f * __temp_97__);
        float __temp_102__ = (__tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_103__ = (__temp_98__ + 0.92f * __temp_102__);
        float __temp_107__ = (__tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_108__ = (__temp_103__ + 1.22f * __temp_107__);
        float __temp_112__ = (__tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_113__ = (__temp_108__ + 1.52f * __temp_112__);
        float __temp_117__ = (__tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_118__ = (__temp_113__ + 1.22f * __temp_117__);
        float __temp_122__ = (__tilevar_1__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_123__ = (__temp_118__ + 0.92f * __temp_122__);
        float __temp_127__ = (__tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_128__ = (__temp_123__ + 0.72f * __temp_127__);
        float __temp_132__ = (__tilevar_1__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_133__ = (__temp_128__ + 0.52f * __temp_132__) / 159;
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] += __temp_133__;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+3),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+3),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))){
	float __temp_3__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_7__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_8__ = (0.5f * __temp_3__ + 0.7f *__temp_7__);
        float __temp_12__ = (__tilevar_2__[ __iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_13__ = (__temp_8__ + 0.9f * __temp_12__);
        float __temp_17__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_18__ = (__temp_13__ + 1.2f * __temp_17__);
        float __temp_22__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_23__ = (__temp_18__ + 1.5f * __temp_22__);
        float __temp_27__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_28__ = (__temp_23__ + 1.2f * __temp_27__);
        float __temp_32__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_33__ = (__temp_28__ + 0.9f * __temp_32__);
        float __temp_37__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_38__ = (__temp_33__ + 0.7f * __temp_37__);
        float __temp_42__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_43__ = (__temp_38__ + 0.5f * __temp_42__) / 159;
	t4 += __temp_43__;
	//Mid 
        float __temp_47__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_52__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_53__ = (0.51f * __temp_47__ + 0.71f * __temp_52__);
        float __temp_57__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_58__ = (__temp_53__ + 0.91f * __temp_57__);
        float __temp_62__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_63__ = (__temp_58__ + 1.21f * __temp_62__);
        float __temp_67__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_68__ = (__temp_63__ + 1.51f * __temp_67__);
        float __temp_72__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_73__ = (__temp_68__ + 1.21f * __temp_72__);
        float __temp_77__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_78__ = (__temp_73__ + 0.91f * __temp_77__);
        float __temp_82__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_83__ = (__temp_78__ + 0.71f * __temp_82__);
        float __temp_87__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_88__ = (__temp_83__ + 0.51f * __temp_87__) / 159;
	b4 += __temp_88__;	
	// Top
        float __temp_92__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_97__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_98__ = (0.52f * __temp_92__ + 0.72f * __temp_97__);
        float __temp_102__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_103__ = (__temp_98__ + 0.92f * __temp_102__);
        float __temp_107__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_108__ = (__temp_103__ + 1.22f * __temp_107__);
        float __temp_112__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_113__ = (__temp_108__ + 1.52f * __temp_112__);
        float __temp_117__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_118__ = (__temp_113__ + 1.22f * __temp_117__);
        float __temp_122__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_123__ = (__temp_118__ + 0.92f * __temp_122__);
        float __temp_127__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_128__ = (__temp_123__ + 0.72f * __temp_127__);
        float __temp_132__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_133__ = (__temp_128__ + 0.52f * __temp_132__) / 159;
        __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] += __temp_133__;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+4),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+4),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2))){
	float __temp_3__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_7__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_8__ = (0.5f * __temp_3__ + 0.7f *__temp_7__);
        float __temp_12__ = (__tilevar_3__[ __iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_13__ = (__temp_8__ + 0.9f * __temp_12__);
        float __temp_17__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_18__ = (__temp_13__ + 1.2f * __temp_17__);
        float __temp_22__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_23__ = (__temp_18__ + 1.5f * __temp_22__);
        float __temp_27__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_28__ = (__temp_23__ + 1.2f * __temp_27__);
        float __temp_32__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_33__ = (__temp_28__ + 0.9f * __temp_32__);
        float __temp_37__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_38__ = (__temp_33__ + 0.7f * __temp_37__);
        float __temp_42__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_43__ = (__temp_38__ + 0.5f * __temp_42__) / 159;
	t5 += __temp_43__;
	//Mid 
        float __temp_47__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_52__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_53__ = (0.51f * __temp_47__ + 0.71f * __temp_52__);
        float __temp_57__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_58__ = (__temp_53__ + 0.91f * __temp_57__);
        float __temp_62__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_63__ = (__temp_58__ + 1.21f * __temp_62__);
        float __temp_67__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_68__ = (__temp_63__ + 1.51f * __temp_67__);
        float __temp_72__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_73__ = (__temp_68__ + 1.21f * __temp_72__);
        float __temp_77__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_78__ = (__temp_73__ + 0.91f * __temp_77__);
        float __temp_82__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_83__ = (__temp_78__ + 0.71f * __temp_82__);
        float __temp_87__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_88__ = (__temp_83__ + 0.51f * __temp_87__) / 159;
	b5 += __temp_88__;	
	// Top
        float __temp_92__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_97__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_98__ = (0.52f * __temp_92__ + 0.72f * __temp_97__);
        float __temp_102__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_103__ = (__temp_98__ + 0.92f * __temp_102__);
        float __temp_107__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_108__ = (__temp_103__ + 1.22f * __temp_107__);
        float __temp_112__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_113__ = (__temp_108__ + 1.52f * __temp_112__);
        float __temp_117__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_118__ = (__temp_113__ + 1.22f * __temp_117__);
        float __temp_122__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_123__ = (__temp_118__ + 0.92f * __temp_122__);
        float __temp_127__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_128__ = (__temp_123__ + 0.72f * __temp_127__);
        float __temp_132__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_133__ = (__temp_128__ + 0.52f * __temp_132__) / 159;
	out += __temp_133__;
	__var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-4,0))] = out;
    }
    __syncthreads ();
    // Now rotate 
    __tilevar_1__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = b2;
    b2 = t2;
    t2 = 0.0f;
    __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = b3;
    b3 = t3;
    t3 = 0.0f;
    __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = b4;
    b4 = t4;
    t4 = 0.0f;
    out= b5; 
    b5 = t5;
    t5 = 0.0f;
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(4*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d27pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*(L*M*N), memcpy_kind_h_input);
  }

  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = N;
  int __size_1___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 16;
  int __block_2___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-8);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y-8);
  int __grid_2___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 unrollConfig (__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

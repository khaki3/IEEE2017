#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel_j3d27pt0__(float * __restrict__ input, int L, int M, int N, float * __var_2__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (1-(-1));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-2))*(FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 1;
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 1;
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + 1;
  {
    int __iter_3__;
    __iter_3__ = (FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)) + (int)(threadIdx.z*1); 
    for (;__iter_3__+0 <= (FORMA_MIN((FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2))+1),(L-2))+1); __iter_3__ += (int)(blockDim.z*1)) {
      int __iter_4__;
      __iter_4__ = (FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)) + (int)(threadIdx.y*1); 
      if (__iter_4__ <= (FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2))+1),(M-2))+1)) {
        int __iter_5__;
        __iter_5__ = (FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)) + (int)(threadIdx.x); 
        if (__iter_5__ <= (FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2))+1),(N-2))+1)) {
          int __temp_0__;
          __temp_0__ = (__iter_3__+0);
          int __temp_1__;
          __temp_1__ = (__iter_4__+0);
          int __temp_2__;
          __temp_2__ = (__iter_5__+0);
          __tilevar_1__[(__iter_5__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_4__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_3__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))] = input[__temp_2__+(N-0)*(__temp_1__+(M-0)*(__temp_0__))];
        }
      }
    }
  }
  __syncthreads();
  {
    int __iter_6__;
    __iter_6__ = FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1) + (int)(threadIdx.z*1); 
    if (__iter_6__ <= FORMA_MIN((FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2))+1),(L-2))) {
      int __iter_7__;
      __iter_7__ = FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1) + (int)(threadIdx.y*1); 
      if (__iter_7__ <= FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2))+1),(M-2))) {
        int __iter_8__;
        __iter_8__ = FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1) + (int)(threadIdx.x); 
        if (__iter_8__ <= FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2))+1),(N-2))) {
          float __temp_3__;
          __temp_3__ = (0.500000 * __tilevar_1__[(__iter_8__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_4__;
          __temp_4__ = (0.700000 * __tilevar_1__[(__iter_8__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_5__;
          __temp_5__ = (__temp_3__ + __temp_4__);
          float __temp_6__;
          __temp_6__ = (0.900000 * __tilevar_1__[(__iter_8__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_7__;
          __temp_7__ = (__temp_5__ + __temp_6__);
          float __temp_8__;
          __temp_8__ = (1.200000 * __tilevar_1__[(__iter_8__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_9__;
          __temp_9__ = (__temp_7__ + __temp_8__);
          float __temp_10__;
          __temp_10__ = (1.500000 * __tilevar_1__[(__iter_8__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_11__;
          __temp_11__ = (__temp_9__ + __temp_10__);
          float __temp_12__;
          __temp_12__ = (1.200000 * __tilevar_1__[(__iter_8__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_13__;
          __temp_13__ = (__temp_11__ + __temp_12__);
          float __temp_14__;
          __temp_14__ = (0.900000 * __tilevar_1__[(__iter_8__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_15__;
          __temp_15__ = (__temp_13__ + __temp_14__);
          float __temp_16__;
          __temp_16__ = (0.700000 * __tilevar_1__[(__iter_8__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_17__;
          __temp_17__ = (__temp_15__ + __temp_16__);
          float __temp_18__;
          __temp_18__ = (0.500000 * __tilevar_1__[(__iter_8__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_19__;
          __temp_19__ = (__temp_17__ + __temp_18__);
          float __temp_20__;
          __temp_20__ = (0.500000 * __tilevar_1__[(__iter_8__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_21__;
          __temp_21__ = (__temp_19__ + __temp_20__);
          float __temp_22__;
          __temp_22__ = (0.700000 * __tilevar_1__[(__iter_8__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_23__;
          __temp_23__ = (__temp_21__ + __temp_22__);
          float __temp_24__;
          __temp_24__ = (0.900000 * __tilevar_1__[(__iter_8__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_25__;
          __temp_25__ = (__temp_23__ + __temp_24__);
          float __temp_26__;
          __temp_26__ = (1.200000 * __tilevar_1__[(__iter_8__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_27__;
          __temp_27__ = (__temp_25__ + __temp_26__);
          float __temp_28__;
          __temp_28__ = (1.500000 * __tilevar_1__[(__iter_8__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_29__;
          __temp_29__ = (__temp_27__ + __temp_28__);
          float __temp_30__;
          __temp_30__ = (1.200000 * __tilevar_1__[(__iter_8__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_31__;
          __temp_31__ = (__temp_29__ + __temp_30__);
          float __temp_32__;
          __temp_32__ = (0.900000 * __tilevar_1__[(__iter_8__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_33__;
          __temp_33__ = (__temp_31__ + __temp_32__);
          float __temp_34__;
          __temp_34__ = (0.700000 * __tilevar_1__[(__iter_8__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_35__;
          __temp_35__ = (__temp_33__ + __temp_34__);
          float __temp_36__;
          __temp_36__ = (0.500000 * __tilevar_1__[(__iter_8__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_37__;
          __temp_37__ = (__temp_35__ + __temp_36__);
          float __temp_38__;
          __temp_38__ = (0.500000 * __tilevar_1__[(__iter_8__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_39__;
          __temp_39__ = (__temp_37__ + __temp_38__);
          float __temp_40__;
          __temp_40__ = (0.700000 * __tilevar_1__[(__iter_8__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_41__;
          __temp_41__ = (__temp_39__ + __temp_40__);
          float __temp_42__;
          __temp_42__ = (0.900000 * __tilevar_1__[(__iter_8__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_43__;
          __temp_43__ = (__temp_41__ + __temp_42__);
          float __temp_44__;
          __temp_44__ = (1.200000 * __tilevar_1__[(__iter_8__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_45__;
          __temp_45__ = (__temp_43__ + __temp_44__);
          float __temp_46__;
          __temp_46__ = (1.500000 * __tilevar_1__[(__iter_8__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_47__;
          __temp_47__ = (__temp_45__ + __temp_46__);
          float __temp_48__;
          __temp_48__ = (1.200000 * __tilevar_1__[(__iter_8__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_49__;
          __temp_49__ = (__temp_47__ + __temp_48__);
          float __temp_50__;
          __temp_50__ = (0.900000 * __tilevar_1__[(__iter_8__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_51__;
          __temp_51__ = (__temp_49__ + __temp_50__);
          float __temp_52__;
          __temp_52__ = (0.700000 * __tilevar_1__[(__iter_8__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_53__;
          __temp_53__ = (__temp_51__ + __temp_52__);
          float __temp_54__;
          __temp_54__ = (0.500000 * __tilevar_1__[(__iter_8__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_7__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_6__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)))))]);
          float __temp_55__;
          __temp_55__ = (__temp_53__ + __temp_54__);
          float __temp_56__;
          __temp_56__ = (__temp_55__ / 159);
          __tilevar_0__[(__iter_8__+0)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_7__+0)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_6__+0)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))] = __temp_56__;
        }
      }
    }
  }
  __syncthreads();
  {
    int __iter_9__;
    __iter_9__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z*1); 
    if (__iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2))) {
      int __iter_10__;
      __iter_10__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y*1); 
      if (__iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2))) {
        int __iter_11__;
        __iter_11__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x); 
        if (__iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2))) {
          float __temp_57__;
          __temp_57__ = (0.500000 * __tilevar_0__[(__iter_11__+0)+(-1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(-1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(-1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_58__;
          __temp_58__ = (0.700000 * __tilevar_0__[(__iter_11__+0)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(-1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(-1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_59__;
          __temp_59__ = (__temp_57__ + __temp_58__);
          float __temp_60__;
          __temp_60__ = (0.900000 * __tilevar_0__[(__iter_11__+0)+(1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(-1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(-1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_61__;
          __temp_61__ = (__temp_59__ + __temp_60__);
          float __temp_62__;
          __temp_62__ = (1.200000 * __tilevar_0__[(__iter_11__+0)+(-1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(-1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_63__;
          __temp_63__ = (__temp_61__ + __temp_62__);
          float __temp_64__;
          __temp_64__ = (1.500000 * __tilevar_0__[(__iter_11__+0)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(-1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_65__;
          __temp_65__ = (__temp_63__ + __temp_64__);
          float __temp_66__;
          __temp_66__ = (1.200000 * __tilevar_0__[(__iter_11__+0)+(1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(-1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_67__;
          __temp_67__ = (__temp_65__ + __temp_66__);
          float __temp_68__;
          __temp_68__ = (0.900000 * __tilevar_0__[(__iter_11__+0)+(-1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(-1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_69__;
          __temp_69__ = (__temp_67__ + __temp_68__);
          float __temp_70__;
          __temp_70__ = (0.700000 * __tilevar_0__[(__iter_11__+0)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(-1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_71__;
          __temp_71__ = (__temp_69__ + __temp_70__);
          float __temp_72__;
          __temp_72__ = (0.500000 * __tilevar_0__[(__iter_11__+0)+(1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(-1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_73__;
          __temp_73__ = (__temp_71__ + __temp_72__);
          float __temp_74__;
          __temp_74__ = (0.500000 * __tilevar_0__[(__iter_11__+0)+(-1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(-1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_75__;
          __temp_75__ = (__temp_73__ + __temp_74__);
          float __temp_76__;
          __temp_76__ = (0.700000 * __tilevar_0__[(__iter_11__+0)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(-1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_77__;
          __temp_77__ = (__temp_75__ + __temp_76__);
          float __temp_78__;
          __temp_78__ = (0.900000 * __tilevar_0__[(__iter_11__+0)+(1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(-1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_79__;
          __temp_79__ = (__temp_77__ + __temp_78__);
          float __temp_80__;
          __temp_80__ = (1.200000 * __tilevar_0__[(__iter_11__+0)+(-1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_81__;
          __temp_81__ = (__temp_79__ + __temp_80__);
          float __temp_82__;
          __temp_82__ = (1.500000 * __tilevar_0__[(__iter_11__+0)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_83__;
          __temp_83__ = (__temp_81__ + __temp_82__);
          float __temp_84__;
          __temp_84__ = (1.200000 * __tilevar_0__[(__iter_11__+0)+(1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_85__;
          __temp_85__ = (__temp_83__ + __temp_84__);
          float __temp_86__;
          __temp_86__ = (0.900000 * __tilevar_0__[(__iter_11__+0)+(-1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_87__;
          __temp_87__ = (__temp_85__ + __temp_86__);
          float __temp_88__;
          __temp_88__ = (0.700000 * __tilevar_0__[(__iter_11__+0)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_89__;
          __temp_89__ = (__temp_87__ + __temp_88__);
          float __temp_90__;
          __temp_90__ = (0.500000 * __tilevar_0__[(__iter_11__+0)+(1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_91__;
          __temp_91__ = (__temp_89__ + __temp_90__);
          float __temp_92__;
          __temp_92__ = (0.500000 * __tilevar_0__[(__iter_11__+0)+(-1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(-1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_93__;
          __temp_93__ = (__temp_91__ + __temp_92__);
          float __temp_94__;
          __temp_94__ = (0.700000 * __tilevar_0__[(__iter_11__+0)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(-1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_95__;
          __temp_95__ = (__temp_93__ + __temp_94__);
          float __temp_96__;
          __temp_96__ = (0.900000 * __tilevar_0__[(__iter_11__+0)+(1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(-1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_97__;
          __temp_97__ = (__temp_95__ + __temp_96__);
          float __temp_98__;
          __temp_98__ = (1.200000 * __tilevar_0__[(__iter_11__+0)+(-1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_99__;
          __temp_99__ = (__temp_97__ + __temp_98__);
          float __temp_100__;
          __temp_100__ = (1.500000 * __tilevar_0__[(__iter_11__+0)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_101__;
          __temp_101__ = (__temp_99__ + __temp_100__);
          float __temp_102__;
          __temp_102__ = (1.200000 * __tilevar_0__[(__iter_11__+0)+(1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_103__;
          __temp_103__ = (__temp_101__ + __temp_102__);
          float __temp_104__;
          __temp_104__ = (0.900000 * __tilevar_0__[(__iter_11__+0)+(-1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_105__;
          __temp_105__ = (__temp_103__ + __temp_104__);
          float __temp_106__;
          __temp_106__ = (0.700000 * __tilevar_0__[(__iter_11__+0)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_107__;
          __temp_107__ = (__temp_105__ + __temp_106__);
          float __temp_108__;
          __temp_108__ = (0.500000 * __tilevar_0__[(__iter_11__+0)+(1)+(0-(FORMA_MAX(__iter_0__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_10__+0)+(1)+(0-(FORMA_MAX(__iter_1__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_9__+0)+(1)+(0-(FORMA_MAX(__iter_2__,1)+(-1)))))]);
          float __temp_109__;
          __temp_109__ = (__temp_107__ + __temp_108__);
          float __temp_110__;
          __temp_110__ = (__temp_109__ / 159);
          __var_2__[(__iter_11__+0)+(N-0)*((__iter_10__+0)+(M-0)*((__iter_9__+0)))] = __temp_110__;
        }
      }
    }
  }
}
int __blockSizeToSMemSize___kernel_j3d27pt0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (1-(-1));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-2))*(FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  return SMemSize;
}
__global__ void __kernel_j3d27pt1__(float * __restrict__ __var_2__, int L, int M, int N, float * __var_1__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (1-(-1));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-2))*(FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  int __iter_12__;
  __iter_12__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 1;
  int __iter_13__;
  __iter_13__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 1;
  int __iter_14__;
  __iter_14__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + 1;
  {
    int __iter_15__;
    __iter_15__ = (FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)) + (int)(threadIdx.z*1); 
    for (;__iter_15__+0 <= (FORMA_MIN((FORMA_MIN(((__iter_14__+FORMA_BLOCKDIM_Z)-1),(L-2))+1),(L-2))+1); __iter_15__ += (int)(blockDim.z*1)) {
      int __iter_16__;
      __iter_16__ = (FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)) + (int)(threadIdx.y*1); 
      if (__iter_16__ <= (FORMA_MIN((FORMA_MIN(((__iter_13__+FORMA_BLOCKDIM_Y)-1),(M-2))+1),(M-2))+1)) {
        int __iter_17__;
        __iter_17__ = (FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)) + (int)(threadIdx.x); 
        if (__iter_17__ <= (FORMA_MIN((FORMA_MIN(((__iter_12__+FORMA_BLOCKDIM_X)-1),(N-2))+1),(N-2))+1)) {
          int __temp_111__;
          __temp_111__ = (__iter_15__+0);
          int __temp_112__;
          __temp_112__ = (__iter_16__+0);
          int __temp_113__;
          __temp_113__ = (__iter_17__+0);
          __tilevar_3__[(__iter_17__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_16__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_15__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))] = __var_2__[__temp_113__+(N-0)*(__temp_112__+(M-0)*(__temp_111__))];
        }
      }
    }
  }
  __syncthreads();
  {
    int __iter_18__;
    __iter_18__ = FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1) + (int)(threadIdx.z*1); 
    if (__iter_18__ <= FORMA_MIN((FORMA_MIN(((__iter_14__+FORMA_BLOCKDIM_Z)-1),(L-2))+1),(L-2))) {
      int __iter_19__;
      __iter_19__ = FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1) + (int)(threadIdx.y*1); 
      if (__iter_19__ <= FORMA_MIN((FORMA_MIN(((__iter_13__+FORMA_BLOCKDIM_Y)-1),(M-2))+1),(M-2))) {
        int __iter_20__;
        __iter_20__ = FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1) + (int)(threadIdx.x); 
        if (__iter_20__ <= FORMA_MIN((FORMA_MIN(((__iter_12__+FORMA_BLOCKDIM_X)-1),(N-2))+1),(N-2))) {
          float __temp_114__;
          __temp_114__ = (0.500000 * __tilevar_3__[(__iter_20__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_115__;
          __temp_115__ = (0.700000 * __tilevar_3__[(__iter_20__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_116__;
          __temp_116__ = (__temp_114__ + __temp_115__);
          float __temp_117__;
          __temp_117__ = (0.900000 * __tilevar_3__[(__iter_20__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_118__;
          __temp_118__ = (__temp_116__ + __temp_117__);
          float __temp_119__;
          __temp_119__ = (1.200000 * __tilevar_3__[(__iter_20__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_120__;
          __temp_120__ = (__temp_118__ + __temp_119__);
          float __temp_121__;
          __temp_121__ = (1.500000 * __tilevar_3__[(__iter_20__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_122__;
          __temp_122__ = (__temp_120__ + __temp_121__);
          float __temp_123__;
          __temp_123__ = (1.200000 * __tilevar_3__[(__iter_20__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_124__;
          __temp_124__ = (__temp_122__ + __temp_123__);
          float __temp_125__;
          __temp_125__ = (0.900000 * __tilevar_3__[(__iter_20__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_126__;
          __temp_126__ = (__temp_124__ + __temp_125__);
          float __temp_127__;
          __temp_127__ = (0.700000 * __tilevar_3__[(__iter_20__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_128__;
          __temp_128__ = (__temp_126__ + __temp_127__);
          float __temp_129__;
          __temp_129__ = (0.500000 * __tilevar_3__[(__iter_20__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_130__;
          __temp_130__ = (__temp_128__ + __temp_129__);
          float __temp_131__;
          __temp_131__ = (0.500000 * __tilevar_3__[(__iter_20__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_132__;
          __temp_132__ = (__temp_130__ + __temp_131__);
          float __temp_133__;
          __temp_133__ = (0.700000 * __tilevar_3__[(__iter_20__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_134__;
          __temp_134__ = (__temp_132__ + __temp_133__);
          float __temp_135__;
          __temp_135__ = (0.900000 * __tilevar_3__[(__iter_20__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_136__;
          __temp_136__ = (__temp_134__ + __temp_135__);
          float __temp_137__;
          __temp_137__ = (1.200000 * __tilevar_3__[(__iter_20__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_138__;
          __temp_138__ = (__temp_136__ + __temp_137__);
          float __temp_139__;
          __temp_139__ = (1.500000 * __tilevar_3__[(__iter_20__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_140__;
          __temp_140__ = (__temp_138__ + __temp_139__);
          float __temp_141__;
          __temp_141__ = (1.200000 * __tilevar_3__[(__iter_20__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_142__;
          __temp_142__ = (__temp_140__ + __temp_141__);
          float __temp_143__;
          __temp_143__ = (0.900000 * __tilevar_3__[(__iter_20__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_144__;
          __temp_144__ = (__temp_142__ + __temp_143__);
          float __temp_145__;
          __temp_145__ = (0.700000 * __tilevar_3__[(__iter_20__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_146__;
          __temp_146__ = (__temp_144__ + __temp_145__);
          float __temp_147__;
          __temp_147__ = (0.500000 * __tilevar_3__[(__iter_20__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_148__;
          __temp_148__ = (__temp_146__ + __temp_147__);
          float __temp_149__;
          __temp_149__ = (0.500000 * __tilevar_3__[(__iter_20__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_150__;
          __temp_150__ = (__temp_148__ + __temp_149__);
          float __temp_151__;
          __temp_151__ = (0.700000 * __tilevar_3__[(__iter_20__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_152__;
          __temp_152__ = (__temp_150__ + __temp_151__);
          float __temp_153__;
          __temp_153__ = (0.900000 * __tilevar_3__[(__iter_20__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_154__;
          __temp_154__ = (__temp_152__ + __temp_153__);
          float __temp_155__;
          __temp_155__ = (1.200000 * __tilevar_3__[(__iter_20__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_156__;
          __temp_156__ = (__temp_154__ + __temp_155__);
          float __temp_157__;
          __temp_157__ = (1.500000 * __tilevar_3__[(__iter_20__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_158__;
          __temp_158__ = (__temp_156__ + __temp_157__);
          float __temp_159__;
          __temp_159__ = (1.200000 * __tilevar_3__[(__iter_20__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_160__;
          __temp_160__ = (__temp_158__ + __temp_159__);
          float __temp_161__;
          __temp_161__ = (0.900000 * __tilevar_3__[(__iter_20__+0)+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_162__;
          __temp_162__ = (__temp_160__ + __temp_161__);
          float __temp_163__;
          __temp_163__ = (0.700000 * __tilevar_3__[(__iter_20__+0)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_164__;
          __temp_164__ = (__temp_162__ + __temp_163__);
          float __temp_165__;
          __temp_165__ = (0.500000 * __tilevar_3__[(__iter_20__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_X-(-4))*((__iter_19__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)))+(FORMA_BLOCKDIM_Y-(-4))*((__iter_18__+0)+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)))))]);
          float __temp_166__;
          __temp_166__ = (__temp_164__ + __temp_165__);
          float __temp_167__;
          __temp_167__ = (__temp_166__ / 159);
          __tilevar_2__[(__iter_20__+0)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_19__+0)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_18__+0)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))] = __temp_167__;
        }
      }
    }
  }
  __syncthreads();
  {
    int __iter_21__;
    __iter_21__ = FORMA_MAX(__iter_14__,1) + (int)(threadIdx.z*1); 
    if (__iter_21__ <= FORMA_MIN(((__iter_14__+FORMA_BLOCKDIM_Z)-1),(L-2))) {
      int __iter_22__;
      __iter_22__ = FORMA_MAX(__iter_13__,1) + (int)(threadIdx.y*1); 
      if (__iter_22__ <= FORMA_MIN(((__iter_13__+FORMA_BLOCKDIM_Y)-1),(M-2))) {
        int __iter_23__;
        __iter_23__ = FORMA_MAX(__iter_12__,1) + (int)(threadIdx.x); 
        if (__iter_23__ <= FORMA_MIN(((__iter_12__+FORMA_BLOCKDIM_X)-1),(N-2))) {
          float __temp_168__;
          __temp_168__ = (0.500000 * __tilevar_2__[(__iter_23__+0)+(-1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(-1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(-1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_169__;
          __temp_169__ = (0.700000 * __tilevar_2__[(__iter_23__+0)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(-1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(-1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_170__;
          __temp_170__ = (__temp_168__ + __temp_169__);
          float __temp_171__;
          __temp_171__ = (0.900000 * __tilevar_2__[(__iter_23__+0)+(1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(-1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(-1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_172__;
          __temp_172__ = (__temp_170__ + __temp_171__);
          float __temp_173__;
          __temp_173__ = (1.200000 * __tilevar_2__[(__iter_23__+0)+(-1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(-1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_174__;
          __temp_174__ = (__temp_172__ + __temp_173__);
          float __temp_175__;
          __temp_175__ = (1.500000 * __tilevar_2__[(__iter_23__+0)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(-1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_176__;
          __temp_176__ = (__temp_174__ + __temp_175__);
          float __temp_177__;
          __temp_177__ = (1.200000 * __tilevar_2__[(__iter_23__+0)+(1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(-1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_178__;
          __temp_178__ = (__temp_176__ + __temp_177__);
          float __temp_179__;
          __temp_179__ = (0.900000 * __tilevar_2__[(__iter_23__+0)+(-1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(-1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_180__;
          __temp_180__ = (__temp_178__ + __temp_179__);
          float __temp_181__;
          __temp_181__ = (0.700000 * __tilevar_2__[(__iter_23__+0)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(-1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_182__;
          __temp_182__ = (__temp_180__ + __temp_181__);
          float __temp_183__;
          __temp_183__ = (0.500000 * __tilevar_2__[(__iter_23__+0)+(1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(-1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_184__;
          __temp_184__ = (__temp_182__ + __temp_183__);
          float __temp_185__;
          __temp_185__ = (0.500000 * __tilevar_2__[(__iter_23__+0)+(-1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(-1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_186__;
          __temp_186__ = (__temp_184__ + __temp_185__);
          float __temp_187__;
          __temp_187__ = (0.700000 * __tilevar_2__[(__iter_23__+0)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(-1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_188__;
          __temp_188__ = (__temp_186__ + __temp_187__);
          float __temp_189__;
          __temp_189__ = (0.900000 * __tilevar_2__[(__iter_23__+0)+(1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(-1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_190__;
          __temp_190__ = (__temp_188__ + __temp_189__);
          float __temp_191__;
          __temp_191__ = (1.200000 * __tilevar_2__[(__iter_23__+0)+(-1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_192__;
          __temp_192__ = (__temp_190__ + __temp_191__);
          float __temp_193__;
          __temp_193__ = (1.500000 * __tilevar_2__[(__iter_23__+0)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_194__;
          __temp_194__ = (__temp_192__ + __temp_193__);
          float __temp_195__;
          __temp_195__ = (1.200000 * __tilevar_2__[(__iter_23__+0)+(1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_196__;
          __temp_196__ = (__temp_194__ + __temp_195__);
          float __temp_197__;
          __temp_197__ = (0.900000 * __tilevar_2__[(__iter_23__+0)+(-1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_198__;
          __temp_198__ = (__temp_196__ + __temp_197__);
          float __temp_199__;
          __temp_199__ = (0.700000 * __tilevar_2__[(__iter_23__+0)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_200__;
          __temp_200__ = (__temp_198__ + __temp_199__);
          float __temp_201__;
          __temp_201__ = (0.500000 * __tilevar_2__[(__iter_23__+0)+(1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_202__;
          __temp_202__ = (__temp_200__ + __temp_201__);
          float __temp_203__;
          __temp_203__ = (0.500000 * __tilevar_2__[(__iter_23__+0)+(-1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(-1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_204__;
          __temp_204__ = (__temp_202__ + __temp_203__);
          float __temp_205__;
          __temp_205__ = (0.700000 * __tilevar_2__[(__iter_23__+0)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(-1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_206__;
          __temp_206__ = (__temp_204__ + __temp_205__);
          float __temp_207__;
          __temp_207__ = (0.900000 * __tilevar_2__[(__iter_23__+0)+(1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(-1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_208__;
          __temp_208__ = (__temp_206__ + __temp_207__);
          float __temp_209__;
          __temp_209__ = (1.200000 * __tilevar_2__[(__iter_23__+0)+(-1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_210__;
          __temp_210__ = (__temp_208__ + __temp_209__);
          float __temp_211__;
          __temp_211__ = (1.500000 * __tilevar_2__[(__iter_23__+0)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_212__;
          __temp_212__ = (__temp_210__ + __temp_211__);
          float __temp_213__;
          __temp_213__ = (1.200000 * __tilevar_2__[(__iter_23__+0)+(1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_214__;
          __temp_214__ = (__temp_212__ + __temp_213__);
          float __temp_215__;
          __temp_215__ = (0.900000 * __tilevar_2__[(__iter_23__+0)+(-1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_216__;
          __temp_216__ = (__temp_214__ + __temp_215__);
          float __temp_217__;
          __temp_217__ = (0.700000 * __tilevar_2__[(__iter_23__+0)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_218__;
          __temp_218__ = (__temp_216__ + __temp_217__);
          float __temp_219__;
          __temp_219__ = (0.500000 * __tilevar_2__[(__iter_23__+0)+(1)+(0-(FORMA_MAX(__iter_12__,1)+(-1)))+(FORMA_BLOCKDIM_X-(-2))*((__iter_22__+0)+(1)+(0-(FORMA_MAX(__iter_13__,1)+(-1)))+(FORMA_BLOCKDIM_Y-(-2))*((__iter_21__+0)+(1)+(0-(FORMA_MAX(__iter_14__,1)+(-1)))))]);
          float __temp_220__;
          __temp_220__ = (__temp_218__ + __temp_219__);
          float __temp_221__;
          __temp_221__ = (__temp_220__ / 159);
          __var_1__[(__iter_23__+0)+(N-0)*((__iter_22__+0)+(M-0)*((__iter_21__+0)))] = __temp_221__;
        }
      }
    }
  }
}
int __blockSizeToSMemSize___kernel_j3d27pt1__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (1-(-1));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-2))*(FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  return SMemSize;
}
/*Device code End */
/* Host Code Begin */
extern "C" void j3d27pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel_j3d27pt0__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel_j3d27pt0__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel_j3d27pt0__ = ((L-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel_j3d27pt0__;
  int _max_occupancy_gridsize___kernel_j3d27pt0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel_j3d27pt0__,&__max_occupancy_blocksize___kernel_j3d27pt0__,(const void*)__kernel_j3d27pt0__,0,0);
  int __max_occupancy_blocksize___kernel_j3d27pt0___0 = pow((float)__max_occupancy_blocksize___kernel_j3d27pt0__, (float)(1.0/(float)3));
  __max_occupancy_blocksize___kernel_j3d27pt0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel_j3d27pt0___0/32, 1)*32;
  int __block_0___kernel_j3d27pt0__ = 16;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt0___0,FORMA_MAX((__size_0___kernel_j3d27pt0__/1)/32,1)*32),FORMA_MAX_BLOCKDIM_0),5);
  __max_occupancy_blocksize___kernel_j3d27pt0__ /= __block_0___kernel_j3d27pt0__;
  int __max_occupancy_blocksize___kernel_j3d27pt0___1 = pow((float)__max_occupancy_blocksize___kernel_j3d27pt0__, (float)(1.0/(float)2));
  int __block_1___kernel_j3d27pt0__ = 8;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt0___1,__size_1___kernel_j3d27pt0__/1),FORMA_MAX_BLOCKDIM_1),5);
  __max_occupancy_blocksize___kernel_j3d27pt0__ /= __block_1___kernel_j3d27pt0__;
  int __max_occupancy_blocksize___kernel_j3d27pt0___2 = __max_occupancy_blocksize___kernel_j3d27pt0__;
  int __block_2___kernel_j3d27pt0__ = 8;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt0___2,__size_2___kernel_j3d27pt0__/1),FORMA_MAX_BLOCKDIM_2),3);
  __max_occupancy_blocksize___kernel_j3d27pt0__ /= __block_2___kernel_j3d27pt0__;
  dim3 __blockConfig___kernel_j3d27pt0__(__block_0___kernel_j3d27pt0__,__block_1___kernel_j3d27pt0__,__block_2___kernel_j3d27pt0__);
  int __SMemSize___kernel_j3d27pt0__ = 0;
  __SMemSize___kernel_j3d27pt0__ = __blockSizeToSMemSize___kernel_j3d27pt0__(__blockConfig___kernel_j3d27pt0__);
  while( __SMemSize___kernel_j3d27pt0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel_j3d27pt0__.z/2 > 3)
      __blockConfig___kernel_j3d27pt0__.z /= 2;
    __SMemSize___kernel_j3d27pt0__ = __blockSizeToSMemSize___kernel_j3d27pt0__(__blockConfig___kernel_j3d27pt0__);
    if( __SMemSize___kernel_j3d27pt0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel_j3d27pt0__.y/2 > 5)
      __blockConfig___kernel_j3d27pt0__.y /= 2;
    __SMemSize___kernel_j3d27pt0__ = __blockSizeToSMemSize___kernel_j3d27pt0__(__blockConfig___kernel_j3d27pt0__);
    if( __SMemSize___kernel_j3d27pt0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel_j3d27pt0__.x/2 > FORMA_MIN(32,5))
      __blockConfig___kernel_j3d27pt0__.x /= 2;
    __SMemSize___kernel_j3d27pt0__ = __blockSizeToSMemSize___kernel_j3d27pt0__(__blockConfig___kernel_j3d27pt0__);
  }
  __block_0___kernel_j3d27pt0__ = __blockConfig___kernel_j3d27pt0__.x-(2-(-2));
  __block_1___kernel_j3d27pt0__ = __blockConfig___kernel_j3d27pt0__.y-(2-(-2));
  __block_2___kernel_j3d27pt0__ = __blockConfig___kernel_j3d27pt0__.z-(1-(-1));
  int __grid_0___kernel_j3d27pt0__ = FORMA_CEIL(__size_0___kernel_j3d27pt0__,__block_0___kernel_j3d27pt0__*1);
  int __grid_1___kernel_j3d27pt0__ = FORMA_CEIL(__size_1___kernel_j3d27pt0__,__block_1___kernel_j3d27pt0__*1);
  int __grid_2___kernel_j3d27pt0__ = FORMA_CEIL(__size_2___kernel_j3d27pt0__,__block_2___kernel_j3d27pt0__*1);
  dim3 __gridConfig___kernel_j3d27pt0__(__grid_0___kernel_j3d27pt0__,__grid_1___kernel_j3d27pt0__,__grid_2___kernel_j3d27pt0__);
  __kernel_j3d27pt0__<<<__gridConfig___kernel_j3d27pt0__,__blockConfig___kernel_j3d27pt0__,__SMemSize___kernel_j3d27pt0__>>>(input,   L,  M,  N,__var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel_j3d27pt0__\n");
  int __size_0___kernel_j3d27pt1__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel_j3d27pt1__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel_j3d27pt1__ = ((L-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel_j3d27pt1__;
  int _max_occupancy_gridsize___kernel_j3d27pt1__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel_j3d27pt1__,&__max_occupancy_blocksize___kernel_j3d27pt1__,(const void*)__kernel_j3d27pt1__,0,0);
  int __max_occupancy_blocksize___kernel_j3d27pt1___0 = pow((double)__max_occupancy_blocksize___kernel_j3d27pt1__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel_j3d27pt1___0 = FORMA_MAX(__max_occupancy_blocksize___kernel_j3d27pt1___0/32, 1)*32;
  int __block_0___kernel_j3d27pt1__ = 16;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt1___0,FORMA_MAX((__size_0___kernel_j3d27pt1__/1)/32,1)*32),FORMA_MAX_BLOCKDIM_0),5);
  __max_occupancy_blocksize___kernel_j3d27pt1__ /= __block_0___kernel_j3d27pt1__;
  int __max_occupancy_blocksize___kernel_j3d27pt1___1 = pow((double)__max_occupancy_blocksize___kernel_j3d27pt1__, (double)(1.0/(double)2));
  int __block_1___kernel_j3d27pt1__ = 8;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt1___1,__size_1___kernel_j3d27pt1__/1),FORMA_MAX_BLOCKDIM_1),5);
  __max_occupancy_blocksize___kernel_j3d27pt1__ /= __block_1___kernel_j3d27pt1__;
  int __max_occupancy_blocksize___kernel_j3d27pt1___2 = __max_occupancy_blocksize___kernel_j3d27pt1__;
  int __block_2___kernel_j3d27pt1__ = 8;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt1___2,__size_2___kernel_j3d27pt1__/1),FORMA_MAX_BLOCKDIM_2),3);
  __max_occupancy_blocksize___kernel_j3d27pt1__ /= __block_2___kernel_j3d27pt1__;
  dim3 __blockConfig___kernel_j3d27pt1__(__block_0___kernel_j3d27pt1__,__block_1___kernel_j3d27pt1__,__block_2___kernel_j3d27pt1__);
  int __SMemSize___kernel_j3d27pt1__ = 0;
  __SMemSize___kernel_j3d27pt1__ = __blockSizeToSMemSize___kernel_j3d27pt1__(__blockConfig___kernel_j3d27pt1__);
  while( __SMemSize___kernel_j3d27pt1__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel_j3d27pt1__.z/2 > 3)
      __blockConfig___kernel_j3d27pt1__.z /= 2;
    __SMemSize___kernel_j3d27pt1__ = __blockSizeToSMemSize___kernel_j3d27pt1__(__blockConfig___kernel_j3d27pt1__);
    if( __SMemSize___kernel_j3d27pt1__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel_j3d27pt1__.y/2 > 5)
      __blockConfig___kernel_j3d27pt1__.y /= 2;
    __SMemSize___kernel_j3d27pt1__ = __blockSizeToSMemSize___kernel_j3d27pt1__(__blockConfig___kernel_j3d27pt1__);
    if( __SMemSize___kernel_j3d27pt1__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel_j3d27pt1__.x/2 > FORMA_MIN(32,5))
      __blockConfig___kernel_j3d27pt1__.x /= 2;
    __SMemSize___kernel_j3d27pt1__ = __blockSizeToSMemSize___kernel_j3d27pt1__(__blockConfig___kernel_j3d27pt1__);
  }
  __block_0___kernel_j3d27pt1__ = __blockConfig___kernel_j3d27pt1__.x-(2-(-2));
  __block_1___kernel_j3d27pt1__ = __blockConfig___kernel_j3d27pt1__.y-(2-(-2));
  __block_2___kernel_j3d27pt1__ = __blockConfig___kernel_j3d27pt1__.z-(1-(-1));
  int __grid_0___kernel_j3d27pt1__ = FORMA_CEIL(__size_0___kernel_j3d27pt1__,__block_0___kernel_j3d27pt1__*1);
  int __grid_1___kernel_j3d27pt1__ = FORMA_CEIL(__size_1___kernel_j3d27pt1__,__block_1___kernel_j3d27pt1__*1);
  int __grid_2___kernel_j3d27pt1__ = FORMA_CEIL(__size_2___kernel_j3d27pt1__,__block_2___kernel_j3d27pt1__*1);
  dim3 __gridConfig___kernel_j3d27pt1__(__grid_0___kernel_j3d27pt1__,__grid_1___kernel_j3d27pt1__,__grid_2___kernel_j3d27pt1__);
  __kernel_j3d27pt1__<<<__gridConfig___kernel_j3d27pt1__,__blockConfig___kernel_j3d27pt1__,__SMemSize___kernel_j3d27pt1__>>>(__var_2__,   L,  M,  N,__var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel_j3d27pt1__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

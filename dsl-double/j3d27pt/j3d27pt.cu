#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel_j3d27pt0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __var_4__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_0__ <= (N-2)){
    int __iter_1__;
    __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_1__ <= (M-2)){
      int __iter_2__;
      __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_2__ <= (L-2)){
        float __temp_0__;
        __temp_0__ = (0.5f * input[__iter_0__-1+N*(__iter_1__-1+M*(__iter_2__-1))]);
        float __temp_1__;
        __temp_1__ = (0.7f * input[__iter_0__+N*(__iter_1__-1+M*(__iter_2__-1))]);
        float __temp_2__;
        __temp_2__ = (__temp_0__ + __temp_1__);
        float __temp_3__;
        __temp_3__ = (0.9f * input[__iter_0__+1+N*(__iter_1__-1+M*(__iter_2__-1))]);
        float __temp_4__;
        __temp_4__ = (__temp_2__ + __temp_3__);
        float __temp_5__;
        __temp_5__ = (1.2f * input[__iter_0__-1+N*(__iter_1__+M*(__iter_2__-1))]);
        float __temp_6__;
        __temp_6__ = (__temp_4__ + __temp_5__);
        float __temp_7__;
        __temp_7__ = (1.5f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__-1))]);
        float __temp_8__;
        __temp_8__ = (__temp_6__ + __temp_7__);
        float __temp_9__;
        __temp_9__ = (1.2f * input[__iter_0__+1+N*(__iter_1__+M*(__iter_2__-1))]);
        float __temp_10__;
        __temp_10__ = (__temp_8__ + __temp_9__);
        float __temp_11__;
        __temp_11__ = (0.9f * input[__iter_0__-1+N*(__iter_1__+1+M*(__iter_2__-1))]);
        float __temp_12__;
        __temp_12__ = (__temp_10__ + __temp_11__);
        float __temp_13__;
        __temp_13__ = (0.7f * input[__iter_0__+N*(__iter_1__+1+M*(__iter_2__-1))]);
        float __temp_14__;
        __temp_14__ = (__temp_12__ + __temp_13__);
        float __temp_15__;
        __temp_15__ = (0.5f * input[__iter_0__+1+N*(__iter_1__+1+M*(__iter_2__-1))]);
        float __temp_16__;
        __temp_16__ = (__temp_14__ + __temp_15__);
        float __temp_17__;
        __temp_17__ = (0.5f * input[__iter_0__-1+N*(__iter_1__-1+M*(__iter_2__))]);
        float __temp_18__;
        __temp_18__ = (__temp_16__ + __temp_17__);
        float __temp_19__;
        __temp_19__ = (0.7f * input[__iter_0__+N*(__iter_1__-1+M*(__iter_2__))]);
        float __temp_20__;
        __temp_20__ = (__temp_18__ + __temp_19__);
        float __temp_21__;
        __temp_21__ = (0.9f * input[__iter_0__+1+N*(__iter_1__-1+M*(__iter_2__))]);
        float __temp_22__;
        __temp_22__ = (__temp_20__ + __temp_21__);
        float __temp_23__;
        __temp_23__ = (1.2f * input[__iter_0__-1+N*(__iter_1__+M*(__iter_2__))]);
        float __temp_24__;
        __temp_24__ = (__temp_22__ + __temp_23__);
        float __temp_25__;
        __temp_25__ = (1.5f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__))]);
        float __temp_26__;
        __temp_26__ = (__temp_24__ + __temp_25__);
        float __temp_27__;
        __temp_27__ = (1.2f * input[__iter_0__+1+N*(__iter_1__+M*(__iter_2__))]);
        float __temp_28__;
        __temp_28__ = (__temp_26__ + __temp_27__);
        float __temp_29__;
        __temp_29__ = (0.9f * input[__iter_0__-1+N*(__iter_1__+1+M*(__iter_2__))]);
        float __temp_30__;
        __temp_30__ = (__temp_28__ + __temp_29__);
        float __temp_31__;
        __temp_31__ = (0.7f * input[__iter_0__+N*(__iter_1__+1+M*(__iter_2__))]);
        float __temp_32__;
        __temp_32__ = (__temp_30__ + __temp_31__);
        float __temp_33__;
        __temp_33__ = (0.5f * input[__iter_0__+1+N*(__iter_1__+1+M*(__iter_2__))]);
        float __temp_34__;
        __temp_34__ = (__temp_32__ + __temp_33__);
        float __temp_35__;
        __temp_35__ = (0.5f * input[__iter_0__-1+N*(__iter_1__-1+M*(__iter_2__+1))]);
        float __temp_36__;
        __temp_36__ = (__temp_34__ + __temp_35__);
        float __temp_37__;
        __temp_37__ = (0.7f * input[__iter_0__+N*(__iter_1__-1+M*(__iter_2__+1))]);
        float __temp_38__;
        __temp_38__ = (__temp_36__ + __temp_37__);
        float __temp_39__;
        __temp_39__ = (0.9f * input[__iter_0__+1+N*(__iter_1__-1+M*(__iter_2__+1))]);
        float __temp_40__;
        __temp_40__ = (__temp_38__ + __temp_39__);
        float __temp_41__;
        __temp_41__ = (1.2f * input[__iter_0__-1+N*(__iter_1__+M*(__iter_2__+1))]);
        float __temp_42__;
        __temp_42__ = (__temp_40__ + __temp_41__);
        float __temp_43__;
        __temp_43__ = (1.5f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__+1))]);
        float __temp_44__;
        __temp_44__ = (__temp_42__ + __temp_43__);
        float __temp_45__;
        __temp_45__ = (1.2f * input[__iter_0__+1+N*(__iter_1__+M*(__iter_2__+1))]);
        float __temp_46__;
        __temp_46__ = (__temp_44__ + __temp_45__);
        float __temp_47__;
        __temp_47__ = (0.9f * input[__iter_0__-1+N*(__iter_1__+1+M*(__iter_2__+1))]);
        float __temp_48__;
        __temp_48__ = (__temp_46__ + __temp_47__);
        float __temp_49__;
        __temp_49__ = (0.7f * input[__iter_0__+N*(__iter_1__+1+M*(__iter_2__+1))]);
        float __temp_50__;
        __temp_50__ = (__temp_48__ + __temp_49__);
        float __temp_51__;
        __temp_51__ = (0.5f * input[__iter_0__+1+N*(__iter_1__+1+M*(__iter_2__+1))]);
        float __temp_52__;
        __temp_52__ = (__temp_50__ + __temp_51__);
        float __temp_53__;
        __temp_53__ = (__temp_52__ / 159);
        __var_4__[__iter_0__+N*(__iter_1__+M*(__iter_2__))] = __temp_53__;
      }
    }
  }
}
__global__ void __kernel_j3d27pt1__(float * __restrict__ __var_4__, int L, int M, int N, float * __restrict__ __var_3__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_3__;
  __iter_3__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_3__ <= (N-2)){
    int __iter_4__;
    __iter_4__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_4__ <= (M-2)){
      int __iter_5__;
      __iter_5__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_5__ <= (L-2)){
        float __temp_54__;
        __temp_54__ = (0.5f * __var_4__[__iter_3__-1+N*(__iter_4__-1+M*(__iter_5__-1))]);
        float __temp_55__;
        __temp_55__ = (0.7f * __var_4__[__iter_3__+N*(__iter_4__-1+M*(__iter_5__-1))]);
        float __temp_56__;
        __temp_56__ = (__temp_54__ + __temp_55__);
        float __temp_57__;
        __temp_57__ = (0.9f * __var_4__[__iter_3__+1+N*(__iter_4__-1+M*(__iter_5__-1))]);
        float __temp_58__;
        __temp_58__ = (__temp_56__ + __temp_57__);
        float __temp_59__;
        __temp_59__ = (1.2f * __var_4__[__iter_3__-1+N*(__iter_4__+M*(__iter_5__-1))]);
        float __temp_60__;
        __temp_60__ = (__temp_58__ + __temp_59__);
        float __temp_61__;
        __temp_61__ = (1.5f * __var_4__[__iter_3__+N*(__iter_4__+M*(__iter_5__-1))]);
        float __temp_62__;
        __temp_62__ = (__temp_60__ + __temp_61__);
        float __temp_63__;
        __temp_63__ = (1.2f * __var_4__[__iter_3__+1+N*(__iter_4__+M*(__iter_5__-1))]);
        float __temp_64__;
        __temp_64__ = (__temp_62__ + __temp_63__);
        float __temp_65__;
        __temp_65__ = (0.9f * __var_4__[__iter_3__-1+N*(__iter_4__+1+M*(__iter_5__-1))]);
        float __temp_66__;
        __temp_66__ = (__temp_64__ + __temp_65__);
        float __temp_67__;
        __temp_67__ = (0.7f * __var_4__[__iter_3__+N*(__iter_4__+1+M*(__iter_5__-1))]);
        float __temp_68__;
        __temp_68__ = (__temp_66__ + __temp_67__);
        float __temp_69__;
        __temp_69__ = (0.5f * __var_4__[__iter_3__+1+N*(__iter_4__+1+M*(__iter_5__-1))]);
        float __temp_70__;
        __temp_70__ = (__temp_68__ + __temp_69__);
        float __temp_71__;
        __temp_71__ = (0.5f * __var_4__[__iter_3__-1+N*(__iter_4__-1+M*(__iter_5__))]);
        float __temp_72__;
        __temp_72__ = (__temp_70__ + __temp_71__);
        float __temp_73__;
        __temp_73__ = (0.7f * __var_4__[__iter_3__+N*(__iter_4__-1+M*(__iter_5__))]);
        float __temp_74__;
        __temp_74__ = (__temp_72__ + __temp_73__);
        float __temp_75__;
        __temp_75__ = (0.9f * __var_4__[__iter_3__+1+N*(__iter_4__-1+M*(__iter_5__))]);
        float __temp_76__;
        __temp_76__ = (__temp_74__ + __temp_75__);
        float __temp_77__;
        __temp_77__ = (1.2f * __var_4__[__iter_3__-1+N*(__iter_4__+M*(__iter_5__))]);
        float __temp_78__;
        __temp_78__ = (__temp_76__ + __temp_77__);
        float __temp_79__;
        __temp_79__ = (1.5f * __var_4__[__iter_3__+N*(__iter_4__+M*(__iter_5__))]);
        float __temp_80__;
        __temp_80__ = (__temp_78__ + __temp_79__);
        float __temp_81__;
        __temp_81__ = (1.2f * __var_4__[__iter_3__+1+N*(__iter_4__+M*(__iter_5__))]);
        float __temp_82__;
        __temp_82__ = (__temp_80__ + __temp_81__);
        float __temp_83__;
        __temp_83__ = (0.9f * __var_4__[__iter_3__-1+N*(__iter_4__+1+M*(__iter_5__))]);
        float __temp_84__;
        __temp_84__ = (__temp_82__ + __temp_83__);
        float __temp_85__;
        __temp_85__ = (0.7f * __var_4__[__iter_3__+N*(__iter_4__+1+M*(__iter_5__))]);
        float __temp_86__;
        __temp_86__ = (__temp_84__ + __temp_85__);
        float __temp_87__;
        __temp_87__ = (0.5f * __var_4__[__iter_3__+1+N*(__iter_4__+1+M*(__iter_5__))]);
        float __temp_88__;
        __temp_88__ = (__temp_86__ + __temp_87__);
        float __temp_89__;
        __temp_89__ = (0.5f * __var_4__[__iter_3__-1+N*(__iter_4__-1+M*(__iter_5__+1))]);
        float __temp_90__;
        __temp_90__ = (__temp_88__ + __temp_89__);
        float __temp_91__;
        __temp_91__ = (0.7f * __var_4__[__iter_3__+N*(__iter_4__-1+M*(__iter_5__+1))]);
        float __temp_92__;
        __temp_92__ = (__temp_90__ + __temp_91__);
        float __temp_93__;
        __temp_93__ = (0.9f * __var_4__[__iter_3__+1+N*(__iter_4__-1+M*(__iter_5__+1))]);
        float __temp_94__;
        __temp_94__ = (__temp_92__ + __temp_93__);
        float __temp_95__;
        __temp_95__ = (1.2f * __var_4__[__iter_3__-1+N*(__iter_4__+M*(__iter_5__+1))]);
        float __temp_96__;
        __temp_96__ = (__temp_94__ + __temp_95__);
        float __temp_97__;
        __temp_97__ = (1.5f * __var_4__[__iter_3__+N*(__iter_4__+M*(__iter_5__+1))]);
        float __temp_98__;
        __temp_98__ = (__temp_96__ + __temp_97__);
        float __temp_99__;
        __temp_99__ = (1.2f * __var_4__[__iter_3__+1+N*(__iter_4__+M*(__iter_5__+1))]);
        float __temp_100__;
        __temp_100__ = (__temp_98__ + __temp_99__);
        float __temp_101__;
        __temp_101__ = (0.9f * __var_4__[__iter_3__-1+N*(__iter_4__+1+M*(__iter_5__+1))]);
        float __temp_102__;
        __temp_102__ = (__temp_100__ + __temp_101__);
        float __temp_103__;
        __temp_103__ = (0.7f * __var_4__[__iter_3__+N*(__iter_4__+1+M*(__iter_5__+1))]);
        float __temp_104__;
        __temp_104__ = (__temp_102__ + __temp_103__);
        float __temp_105__;
        __temp_105__ = (0.5f * __var_4__[__iter_3__+1+N*(__iter_4__+1+M*(__iter_5__+1))]);
        float __temp_106__;
        __temp_106__ = (__temp_104__ + __temp_105__);
        float __temp_107__;
        __temp_107__ = (__temp_106__ / 159);
        __var_3__[__iter_3__+N*(__iter_4__+M*(__iter_5__))] = __temp_107__;
      }
    }
  }
}
__global__ void __kernel_j3d27pt2__(float * __restrict__ __var_3__, int L, int M, int N, float * __restrict__ __var_2__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_6__;
  __iter_6__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_6__ <= (N-2)){
    int __iter_7__;
    __iter_7__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_7__ <= (M-2)){
      int __iter_8__;
      __iter_8__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_8__ <= (L-2)){
        float __temp_108__;
        __temp_108__ = (0.5f * __var_3__[__iter_6__-1+N*(__iter_7__-1+M*(__iter_8__-1))]);
        float __temp_109__;
        __temp_109__ = (0.7f * __var_3__[__iter_6__+N*(__iter_7__-1+M*(__iter_8__-1))]);
        float __temp_110__;
        __temp_110__ = (__temp_108__ + __temp_109__);
        float __temp_111__;
        __temp_111__ = (0.9f * __var_3__[__iter_6__+1+N*(__iter_7__-1+M*(__iter_8__-1))]);
        float __temp_112__;
        __temp_112__ = (__temp_110__ + __temp_111__);
        float __temp_113__;
        __temp_113__ = (1.2f * __var_3__[__iter_6__-1+N*(__iter_7__+M*(__iter_8__-1))]);
        float __temp_114__;
        __temp_114__ = (__temp_112__ + __temp_113__);
        float __temp_115__;
        __temp_115__ = (1.5f * __var_3__[__iter_6__+N*(__iter_7__+M*(__iter_8__-1))]);
        float __temp_116__;
        __temp_116__ = (__temp_114__ + __temp_115__);
        float __temp_117__;
        __temp_117__ = (1.2f * __var_3__[__iter_6__+1+N*(__iter_7__+M*(__iter_8__-1))]);
        float __temp_118__;
        __temp_118__ = (__temp_116__ + __temp_117__);
        float __temp_119__;
        __temp_119__ = (0.9f * __var_3__[__iter_6__-1+N*(__iter_7__+1+M*(__iter_8__-1))]);
        float __temp_120__;
        __temp_120__ = (__temp_118__ + __temp_119__);
        float __temp_121__;
        __temp_121__ = (0.7f * __var_3__[__iter_6__+N*(__iter_7__+1+M*(__iter_8__-1))]);
        float __temp_122__;
        __temp_122__ = (__temp_120__ + __temp_121__);
        float __temp_123__;
        __temp_123__ = (0.5f * __var_3__[__iter_6__+1+N*(__iter_7__+1+M*(__iter_8__-1))]);
        float __temp_124__;
        __temp_124__ = (__temp_122__ + __temp_123__);
        float __temp_125__;
        __temp_125__ = (0.5f * __var_3__[__iter_6__-1+N*(__iter_7__-1+M*(__iter_8__))]);
        float __temp_126__;
        __temp_126__ = (__temp_124__ + __temp_125__);
        float __temp_127__;
        __temp_127__ = (0.7f * __var_3__[__iter_6__+N*(__iter_7__-1+M*(__iter_8__))]);
        float __temp_128__;
        __temp_128__ = (__temp_126__ + __temp_127__);
        float __temp_129__;
        __temp_129__ = (0.9f * __var_3__[__iter_6__+1+N*(__iter_7__-1+M*(__iter_8__))]);
        float __temp_130__;
        __temp_130__ = (__temp_128__ + __temp_129__);
        float __temp_131__;
        __temp_131__ = (1.2f * __var_3__[__iter_6__-1+N*(__iter_7__+M*(__iter_8__))]);
        float __temp_132__;
        __temp_132__ = (__temp_130__ + __temp_131__);
        float __temp_133__;
        __temp_133__ = (1.5f * __var_3__[__iter_6__+N*(__iter_7__+M*(__iter_8__))]);
        float __temp_134__;
        __temp_134__ = (__temp_132__ + __temp_133__);
        float __temp_135__;
        __temp_135__ = (1.2f * __var_3__[__iter_6__+1+N*(__iter_7__+M*(__iter_8__))]);
        float __temp_136__;
        __temp_136__ = (__temp_134__ + __temp_135__);
        float __temp_137__;
        __temp_137__ = (0.9f * __var_3__[__iter_6__-1+N*(__iter_7__+1+M*(__iter_8__))]);
        float __temp_138__;
        __temp_138__ = (__temp_136__ + __temp_137__);
        float __temp_139__;
        __temp_139__ = (0.7f * __var_3__[__iter_6__+N*(__iter_7__+1+M*(__iter_8__))]);
        float __temp_140__;
        __temp_140__ = (__temp_138__ + __temp_139__);
        float __temp_141__;
        __temp_141__ = (0.5f * __var_3__[__iter_6__+1+N*(__iter_7__+1+M*(__iter_8__))]);
        float __temp_142__;
        __temp_142__ = (__temp_140__ + __temp_141__);
        float __temp_143__;
        __temp_143__ = (0.5f * __var_3__[__iter_6__-1+N*(__iter_7__-1+M*(__iter_8__+1))]);
        float __temp_144__;
        __temp_144__ = (__temp_142__ + __temp_143__);
        float __temp_145__;
        __temp_145__ = (0.7f * __var_3__[__iter_6__+N*(__iter_7__-1+M*(__iter_8__+1))]);
        float __temp_146__;
        __temp_146__ = (__temp_144__ + __temp_145__);
        float __temp_147__;
        __temp_147__ = (0.9f * __var_3__[__iter_6__+1+N*(__iter_7__-1+M*(__iter_8__+1))]);
        float __temp_148__;
        __temp_148__ = (__temp_146__ + __temp_147__);
        float __temp_149__;
        __temp_149__ = (1.2f * __var_3__[__iter_6__-1+N*(__iter_7__+M*(__iter_8__+1))]);
        float __temp_150__;
        __temp_150__ = (__temp_148__ + __temp_149__);
        float __temp_151__;
        __temp_151__ = (1.5f * __var_3__[__iter_6__+N*(__iter_7__+M*(__iter_8__+1))]);
        float __temp_152__;
        __temp_152__ = (__temp_150__ + __temp_151__);
        float __temp_153__;
        __temp_153__ = (1.2f * __var_3__[__iter_6__+1+N*(__iter_7__+M*(__iter_8__+1))]);
        float __temp_154__;
        __temp_154__ = (__temp_152__ + __temp_153__);
        float __temp_155__;
        __temp_155__ = (0.9f * __var_3__[__iter_6__-1+N*(__iter_7__+1+M*(__iter_8__+1))]);
        float __temp_156__;
        __temp_156__ = (__temp_154__ + __temp_155__);
        float __temp_157__;
        __temp_157__ = (0.7f * __var_3__[__iter_6__+N*(__iter_7__+1+M*(__iter_8__+1))]);
        float __temp_158__;
        __temp_158__ = (__temp_156__ + __temp_157__);
        float __temp_159__;
        __temp_159__ = (0.5f * __var_3__[__iter_6__+1+N*(__iter_7__+1+M*(__iter_8__+1))]);
        float __temp_160__;
        __temp_160__ = (__temp_158__ + __temp_159__);
        float __temp_161__;
        __temp_161__ = (__temp_160__ / 159);
        __var_2__[__iter_6__+N*(__iter_7__+M*(__iter_8__))] = __temp_161__;
      }
    }
  }
}
__global__ void __kernel_j3d27pt3__(float * __restrict__ __var_2__, int L, int M, int N, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_9__;
  __iter_9__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_9__ <= (N-2)){
    int __iter_10__;
    __iter_10__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_10__ <= (M-2)){
      int __iter_11__;
      __iter_11__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_11__ <= (L-2)){
        float __temp_162__;
        __temp_162__ = (0.5f * __var_2__[__iter_9__-1+N*(__iter_10__-1+M*(__iter_11__-1))]);
        float __temp_163__;
        __temp_163__ = (0.7f * __var_2__[__iter_9__+N*(__iter_10__-1+M*(__iter_11__-1))]);
        float __temp_164__;
        __temp_164__ = (__temp_162__ + __temp_163__);
        float __temp_165__;
        __temp_165__ = (0.9f * __var_2__[__iter_9__+1+N*(__iter_10__-1+M*(__iter_11__-1))]);
        float __temp_166__;
        __temp_166__ = (__temp_164__ + __temp_165__);
        float __temp_167__;
        __temp_167__ = (1.2f * __var_2__[__iter_9__-1+N*(__iter_10__+M*(__iter_11__-1))]);
        float __temp_168__;
        __temp_168__ = (__temp_166__ + __temp_167__);
        float __temp_169__;
        __temp_169__ = (1.5f * __var_2__[__iter_9__+N*(__iter_10__+M*(__iter_11__-1))]);
        float __temp_170__;
        __temp_170__ = (__temp_168__ + __temp_169__);
        float __temp_171__;
        __temp_171__ = (1.2f * __var_2__[__iter_9__+1+N*(__iter_10__+M*(__iter_11__-1))]);
        float __temp_172__;
        __temp_172__ = (__temp_170__ + __temp_171__);
        float __temp_173__;
        __temp_173__ = (0.9f * __var_2__[__iter_9__-1+N*(__iter_10__+1+M*(__iter_11__-1))]);
        float __temp_174__;
        __temp_174__ = (__temp_172__ + __temp_173__);
        float __temp_175__;
        __temp_175__ = (0.7f * __var_2__[__iter_9__+N*(__iter_10__+1+M*(__iter_11__-1))]);
        float __temp_176__;
        __temp_176__ = (__temp_174__ + __temp_175__);
        float __temp_177__;
        __temp_177__ = (0.5f * __var_2__[__iter_9__+1+N*(__iter_10__+1+M*(__iter_11__-1))]);
        float __temp_178__;
        __temp_178__ = (__temp_176__ + __temp_177__);
        float __temp_179__;
        __temp_179__ = (0.5f * __var_2__[__iter_9__-1+N*(__iter_10__-1+M*(__iter_11__))]);
        float __temp_180__;
        __temp_180__ = (__temp_178__ + __temp_179__);
        float __temp_181__;
        __temp_181__ = (0.7f * __var_2__[__iter_9__+N*(__iter_10__-1+M*(__iter_11__))]);
        float __temp_182__;
        __temp_182__ = (__temp_180__ + __temp_181__);
        float __temp_183__;
        __temp_183__ = (0.9f * __var_2__[__iter_9__+1+N*(__iter_10__-1+M*(__iter_11__))]);
        float __temp_184__;
        __temp_184__ = (__temp_182__ + __temp_183__);
        float __temp_185__;
        __temp_185__ = (1.2f * __var_2__[__iter_9__-1+N*(__iter_10__+M*(__iter_11__))]);
        float __temp_186__;
        __temp_186__ = (__temp_184__ + __temp_185__);
        float __temp_187__;
        __temp_187__ = (1.5f * __var_2__[__iter_9__+N*(__iter_10__+M*(__iter_11__))]);
        float __temp_188__;
        __temp_188__ = (__temp_186__ + __temp_187__);
        float __temp_189__;
        __temp_189__ = (1.2f * __var_2__[__iter_9__+1+N*(__iter_10__+M*(__iter_11__))]);
        float __temp_190__;
        __temp_190__ = (__temp_188__ + __temp_189__);
        float __temp_191__;
        __temp_191__ = (0.9f * __var_2__[__iter_9__-1+N*(__iter_10__+1+M*(__iter_11__))]);
        float __temp_192__;
        __temp_192__ = (__temp_190__ + __temp_191__);
        float __temp_193__;
        __temp_193__ = (0.7f * __var_2__[__iter_9__+N*(__iter_10__+1+M*(__iter_11__))]);
        float __temp_194__;
        __temp_194__ = (__temp_192__ + __temp_193__);
        float __temp_195__;
        __temp_195__ = (0.5f * __var_2__[__iter_9__+1+N*(__iter_10__+1+M*(__iter_11__))]);
        float __temp_196__;
        __temp_196__ = (__temp_194__ + __temp_195__);
        float __temp_197__;
        __temp_197__ = (0.5f * __var_2__[__iter_9__-1+N*(__iter_10__-1+M*(__iter_11__+1))]);
        float __temp_198__;
        __temp_198__ = (__temp_196__ + __temp_197__);
        float __temp_199__;
        __temp_199__ = (0.7f * __var_2__[__iter_9__+N*(__iter_10__-1+M*(__iter_11__+1))]);
        float __temp_200__;
        __temp_200__ = (__temp_198__ + __temp_199__);
        float __temp_201__;
        __temp_201__ = (0.9f * __var_2__[__iter_9__+1+N*(__iter_10__-1+M*(__iter_11__+1))]);
        float __temp_202__;
        __temp_202__ = (__temp_200__ + __temp_201__);
        float __temp_203__;
        __temp_203__ = (1.2f * __var_2__[__iter_9__-1+N*(__iter_10__+M*(__iter_11__+1))]);
        float __temp_204__;
        __temp_204__ = (__temp_202__ + __temp_203__);
        float __temp_205__;
        __temp_205__ = (1.5f * __var_2__[__iter_9__+N*(__iter_10__+M*(__iter_11__+1))]);
        float __temp_206__;
        __temp_206__ = (__temp_204__ + __temp_205__);
        float __temp_207__;
        __temp_207__ = (1.2f * __var_2__[__iter_9__+1+N*(__iter_10__+M*(__iter_11__+1))]);
        float __temp_208__;
        __temp_208__ = (__temp_206__ + __temp_207__);
        float __temp_209__;
        __temp_209__ = (0.9f * __var_2__[__iter_9__-1+N*(__iter_10__+1+M*(__iter_11__+1))]);
        float __temp_210__;
        __temp_210__ = (__temp_208__ + __temp_209__);
        float __temp_211__;
        __temp_211__ = (0.7f * __var_2__[__iter_9__+N*(__iter_10__+1+M*(__iter_11__+1))]);
        float __temp_212__;
        __temp_212__ = (__temp_210__ + __temp_211__);
        float __temp_213__;
        __temp_213__ = (0.5f * __var_2__[__iter_9__+1+N*(__iter_10__+1+M*(__iter_11__+1))]);
        float __temp_214__;
        __temp_214__ = (__temp_212__ + __temp_213__);
        float __temp_215__;
        __temp_215__ = (__temp_214__ / 159);
        __var_1__[__iter_9__+N*(__iter_10__+M*(__iter_11__))] = __temp_215__;
      }
    }
  }
}
/*Device code End */
/* Host Code Begin */
extern "C" void j3d27pt (float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*(L*M*N), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
  float * __var_3__;
  hipMalloc(&__var_3__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_3__\n");
  float * __var_4__;
  hipMalloc(&__var_4__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_4__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel_j3d27pt0__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel_j3d27pt0__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel_j3d27pt0__ = ((L-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel_j3d27pt0__;
  int _max_occupancy_gridsize___kernel_j3d27pt0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel_j3d27pt0__,&__max_occupancy_blocksize___kernel_j3d27pt0__,(const void*)__kernel_j3d27pt0__,0,0);
  int __max_occupancy_blocksize___kernel_j3d27pt0___0 = pow((double)__max_occupancy_blocksize___kernel_j3d27pt0__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel_j3d27pt0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel_j3d27pt0___0/32, 1)*32;
  int __block_0___kernel_j3d27pt0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt0___0,FORMA_MAX((__size_0___kernel_j3d27pt0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),1);
  __max_occupancy_blocksize___kernel_j3d27pt0__ /= __block_0___kernel_j3d27pt0__;
  int __max_occupancy_blocksize___kernel_j3d27pt0___1 = pow((double)__max_occupancy_blocksize___kernel_j3d27pt0__, (double)(1.0/(double)2));
  int __block_1___kernel_j3d27pt0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt0___1,__size_1___kernel_j3d27pt0__),FORMA_MAX_BLOCKDIM_1),1);
  __max_occupancy_blocksize___kernel_j3d27pt0__ /= __block_1___kernel_j3d27pt0__;
  int __max_occupancy_blocksize___kernel_j3d27pt0___2 = __max_occupancy_blocksize___kernel_j3d27pt0__;
  int __block_2___kernel_j3d27pt0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt0___2,__size_2___kernel_j3d27pt0__),FORMA_MAX_BLOCKDIM_2),1);
  __max_occupancy_blocksize___kernel_j3d27pt0__ /= __block_2___kernel_j3d27pt0__;
  dim3 __blockConfig___kernel_j3d27pt0__(__block_0___kernel_j3d27pt0__,__block_1___kernel_j3d27pt0__,__block_2___kernel_j3d27pt0__);
  int __SMemSize___kernel_j3d27pt0__ = 0;
  int __grid_0___kernel_j3d27pt0__ = FORMA_CEIL(__size_0___kernel_j3d27pt0__,__block_0___kernel_j3d27pt0__);
  int __grid_1___kernel_j3d27pt0__ = FORMA_CEIL(__size_1___kernel_j3d27pt0__,__block_1___kernel_j3d27pt0__);
  int __grid_2___kernel_j3d27pt0__ = FORMA_CEIL(__size_2___kernel_j3d27pt0__,__block_2___kernel_j3d27pt0__);
  dim3 __gridConfig___kernel_j3d27pt0__(__grid_0___kernel_j3d27pt0__,__grid_1___kernel_j3d27pt0__,__grid_2___kernel_j3d27pt0__);
  __kernel_j3d27pt0__<<<__gridConfig___kernel_j3d27pt0__, __blockConfig___kernel_j3d27pt0__, __SMemSize___kernel_j3d27pt0__>>> (input, L, M, N, __var_4__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel_j3d27pt0__\n");
  int __size_0___kernel_j3d27pt1__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel_j3d27pt1__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel_j3d27pt1__ = ((L-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel_j3d27pt1__;
  int _max_occupancy_gridsize___kernel_j3d27pt1__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel_j3d27pt1__,&__max_occupancy_blocksize___kernel_j3d27pt1__,(const void*)__kernel_j3d27pt1__,0,0);
  int __max_occupancy_blocksize___kernel_j3d27pt1___0 = pow((double)__max_occupancy_blocksize___kernel_j3d27pt1__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel_j3d27pt1___0 = FORMA_MAX(__max_occupancy_blocksize___kernel_j3d27pt1___0/32, 1)*32;
  int __block_0___kernel_j3d27pt1__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt1___0,FORMA_MAX((__size_0___kernel_j3d27pt1__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),1);
  __max_occupancy_blocksize___kernel_j3d27pt1__ /= __block_0___kernel_j3d27pt1__;
  int __max_occupancy_blocksize___kernel_j3d27pt1___1 = pow((double)__max_occupancy_blocksize___kernel_j3d27pt1__, (double)(1.0/(double)2));
  int __block_1___kernel_j3d27pt1__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt1___1,__size_1___kernel_j3d27pt1__),FORMA_MAX_BLOCKDIM_1),1);
  __max_occupancy_blocksize___kernel_j3d27pt1__ /= __block_1___kernel_j3d27pt1__;
  int __max_occupancy_blocksize___kernel_j3d27pt1___2 = __max_occupancy_blocksize___kernel_j3d27pt1__;
  int __block_2___kernel_j3d27pt1__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt1___2,__size_2___kernel_j3d27pt1__),FORMA_MAX_BLOCKDIM_2),1);
  __max_occupancy_blocksize___kernel_j3d27pt1__ /= __block_2___kernel_j3d27pt1__;
  dim3 __blockConfig___kernel_j3d27pt1__(__block_0___kernel_j3d27pt1__,__block_1___kernel_j3d27pt1__,__block_2___kernel_j3d27pt1__);
  int __SMemSize___kernel_j3d27pt1__ = 0;
  int __grid_0___kernel_j3d27pt1__ = FORMA_CEIL(__size_0___kernel_j3d27pt1__,__block_0___kernel_j3d27pt1__);
  int __grid_1___kernel_j3d27pt1__ = FORMA_CEIL(__size_1___kernel_j3d27pt1__,__block_1___kernel_j3d27pt1__);
  int __grid_2___kernel_j3d27pt1__ = FORMA_CEIL(__size_2___kernel_j3d27pt1__,__block_2___kernel_j3d27pt1__);
  dim3 __gridConfig___kernel_j3d27pt1__(__grid_0___kernel_j3d27pt1__,__grid_1___kernel_j3d27pt1__,__grid_2___kernel_j3d27pt1__);
  __kernel_j3d27pt1__<<<__gridConfig___kernel_j3d27pt1__, __blockConfig___kernel_j3d27pt1__, __SMemSize___kernel_j3d27pt1__>>> (__var_4__, L, M, N, __var_3__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel_j3d27pt1__\n");
  int __size_0___kernel_j3d27pt2__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel_j3d27pt2__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel_j3d27pt2__ = ((L-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel_j3d27pt2__;
  int _max_occupancy_gridsize___kernel_j3d27pt2__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel_j3d27pt2__,&__max_occupancy_blocksize___kernel_j3d27pt2__,(const void*)__kernel_j3d27pt2__,0,0);
  int __max_occupancy_blocksize___kernel_j3d27pt2___0 = pow((double)__max_occupancy_blocksize___kernel_j3d27pt2__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel_j3d27pt2___0 = FORMA_MAX(__max_occupancy_blocksize___kernel_j3d27pt2___0/32, 1)*32;
  int __block_0___kernel_j3d27pt2__ = 32;
  __max_occupancy_blocksize___kernel_j3d27pt2__ /= __block_0___kernel_j3d27pt2__;
  int __max_occupancy_blocksize___kernel_j3d27pt2___1 = pow((double)__max_occupancy_blocksize___kernel_j3d27pt2__, (double)(1.0/(double)2));
  int __block_1___kernel_j3d27pt2__ = 32;
  __max_occupancy_blocksize___kernel_j3d27pt2__ /= __block_1___kernel_j3d27pt2__;
  int __max_occupancy_blocksize___kernel_j3d27pt2___2 = __max_occupancy_blocksize___kernel_j3d27pt2__;
  int __block_2___kernel_j3d27pt2__ = 1;
  __max_occupancy_blocksize___kernel_j3d27pt2__ /= __block_2___kernel_j3d27pt2__;
  dim3 __blockConfig___kernel_j3d27pt2__(__block_0___kernel_j3d27pt2__,__block_1___kernel_j3d27pt2__,__block_2___kernel_j3d27pt2__);
  int __SMemSize___kernel_j3d27pt2__ = 0;
  int __grid_0___kernel_j3d27pt2__ = FORMA_CEIL(__size_0___kernel_j3d27pt2__,__block_0___kernel_j3d27pt2__);
  int __grid_1___kernel_j3d27pt2__ = FORMA_CEIL(__size_1___kernel_j3d27pt2__,__block_1___kernel_j3d27pt2__);
  int __grid_2___kernel_j3d27pt2__ = FORMA_CEIL(__size_2___kernel_j3d27pt2__,__block_2___kernel_j3d27pt2__);
  dim3 __gridConfig___kernel_j3d27pt2__(__grid_0___kernel_j3d27pt2__,__grid_1___kernel_j3d27pt2__,__grid_2___kernel_j3d27pt2__);
  __kernel_j3d27pt2__<<<__gridConfig___kernel_j3d27pt2__, __blockConfig___kernel_j3d27pt2__, __SMemSize___kernel_j3d27pt2__>>> (__var_3__, L, M, N, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel_j3d27pt2__\n");
  int __size_0___kernel_j3d27pt3__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel_j3d27pt3__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel_j3d27pt3__ = ((L-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel_j3d27pt3__;
  int _max_occupancy_gridsize___kernel_j3d27pt3__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel_j3d27pt3__,&__max_occupancy_blocksize___kernel_j3d27pt3__,(const void*)__kernel_j3d27pt3__,0,0);
  int __max_occupancy_blocksize___kernel_j3d27pt3___0 = pow((double)__max_occupancy_blocksize___kernel_j3d27pt3__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel_j3d27pt3___0 = FORMA_MAX(__max_occupancy_blocksize___kernel_j3d27pt3___0/32, 1)*32;
  int __block_0___kernel_j3d27pt3__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt3___0,FORMA_MAX((__size_0___kernel_j3d27pt3__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),1);
  __max_occupancy_blocksize___kernel_j3d27pt3__ /= __block_0___kernel_j3d27pt3__;
  int __max_occupancy_blocksize___kernel_j3d27pt3___1 = pow((double)__max_occupancy_blocksize___kernel_j3d27pt3__, (double)(1.0/(double)2));
  int __block_1___kernel_j3d27pt3__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt3___1,__size_1___kernel_j3d27pt3__),FORMA_MAX_BLOCKDIM_1),1);
  __max_occupancy_blocksize___kernel_j3d27pt3__ /= __block_1___kernel_j3d27pt3__;
  int __max_occupancy_blocksize___kernel_j3d27pt3___2 = __max_occupancy_blocksize___kernel_j3d27pt3__;
  int __block_2___kernel_j3d27pt3__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel_j3d27pt3___2,__size_2___kernel_j3d27pt3__),FORMA_MAX_BLOCKDIM_2),1);
  __max_occupancy_blocksize___kernel_j3d27pt3__ /= __block_2___kernel_j3d27pt3__;
  dim3 __blockConfig___kernel_j3d27pt3__(__block_0___kernel_j3d27pt3__,__block_1___kernel_j3d27pt3__,__block_2___kernel_j3d27pt3__);
  int __SMemSize___kernel_j3d27pt3__ = 0;
  int __grid_0___kernel_j3d27pt3__ = FORMA_CEIL(__size_0___kernel_j3d27pt3__,__block_0___kernel_j3d27pt3__);
  int __grid_1___kernel_j3d27pt3__ = FORMA_CEIL(__size_1___kernel_j3d27pt3__,__block_1___kernel_j3d27pt3__);
  int __grid_2___kernel_j3d27pt3__ = FORMA_CEIL(__size_2___kernel_j3d27pt3__,__block_2___kernel_j3d27pt3__);
  dim3 __gridConfig___kernel_j3d27pt3__(__grid_0___kernel_j3d27pt3__,__grid_1___kernel_j3d27pt3__,__grid_2___kernel_j3d27pt3__);
  __kernel_j3d27pt3__<<<__gridConfig___kernel_j3d27pt3__, __blockConfig___kernel_j3d27pt3__, __SMemSize___kernel_j3d27pt3__>>> (__var_2__, L, M, N, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel_j3d27pt3__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
  hipFree(__var_3__);
  hipFree(__var_4__);
}
/*Host Free End*/

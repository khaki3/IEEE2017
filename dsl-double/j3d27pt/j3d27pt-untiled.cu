#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel_j3d27pt0__(double * __restrict__ input, int L, int M, int N, double * __restrict__ __var_4__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_0__ <= (N-2)){
    int __iter_1__;
    __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_1__ <= (M-2)){
      int __iter_2__;
      __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_2__ <= (L-2)){
        double __temp_0__;
        __temp_0__ = (0.5f * input[__iter_0__-1+N*(__iter_1__-1+M*(__iter_2__-1))]);
        double __temp_1__;
        __temp_1__ = (0.7f * input[__iter_0__+N*(__iter_1__-1+M*(__iter_2__-1))]);
        double __temp_2__;
        __temp_2__ = (__temp_0__ + __temp_1__);
        double __temp_3__;
        __temp_3__ = (0.9f * input[__iter_0__+1+N*(__iter_1__-1+M*(__iter_2__-1))]);
        double __temp_4__;
        __temp_4__ = (__temp_2__ + __temp_3__);
        double __temp_5__;
        __temp_5__ = (1.2f * input[__iter_0__-1+N*(__iter_1__+M*(__iter_2__-1))]);
        double __temp_6__;
        __temp_6__ = (__temp_4__ + __temp_5__);
        double __temp_7__;
        __temp_7__ = (1.5f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__-1))]);
        double __temp_8__;
        __temp_8__ = (__temp_6__ + __temp_7__);
        double __temp_9__;
        __temp_9__ = (1.2f * input[__iter_0__+1+N*(__iter_1__+M*(__iter_2__-1))]);
        double __temp_10__;
        __temp_10__ = (__temp_8__ + __temp_9__);
        double __temp_11__;
        __temp_11__ = (0.9f * input[__iter_0__-1+N*(__iter_1__+1+M*(__iter_2__-1))]);
        double __temp_12__;
        __temp_12__ = (__temp_10__ + __temp_11__);
        double __temp_13__;
        __temp_13__ = (0.7f * input[__iter_0__+N*(__iter_1__+1+M*(__iter_2__-1))]);
        double __temp_14__;
        __temp_14__ = (__temp_12__ + __temp_13__);
        double __temp_15__;
        __temp_15__ = (0.5f * input[__iter_0__+1+N*(__iter_1__+1+M*(__iter_2__-1))]);
        double __temp_16__;
        __temp_16__ = (__temp_14__ + __temp_15__);
        double __temp_17__;
        __temp_17__ = (0.51f * input[__iter_0__-1+N*(__iter_1__-1+M*(__iter_2__))]);
        double __temp_18__;
        __temp_18__ = (__temp_16__ + __temp_17__);
        double __temp_19__;
        __temp_19__ = (0.71f * input[__iter_0__+N*(__iter_1__-1+M*(__iter_2__))]);
        double __temp_20__;
        __temp_20__ = (__temp_18__ + __temp_19__);
        double __temp_21__;
        __temp_21__ = (0.91f * input[__iter_0__+1+N*(__iter_1__-1+M*(__iter_2__))]);
        double __temp_22__;
        __temp_22__ = (__temp_20__ + __temp_21__);
        double __temp_23__;
        __temp_23__ = (1.21f * input[__iter_0__-1+N*(__iter_1__+M*(__iter_2__))]);
        double __temp_24__;
        __temp_24__ = (__temp_22__ + __temp_23__);
        double __temp_25__;
        __temp_25__ = (1.51f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__))]);
        double __temp_26__;
        __temp_26__ = (__temp_24__ + __temp_25__);
        double __temp_27__;
        __temp_27__ = (1.21f * input[__iter_0__+1+N*(__iter_1__+M*(__iter_2__))]);
        double __temp_28__;
        __temp_28__ = (__temp_26__ + __temp_27__);
        double __temp_29__;
        __temp_29__ = (0.91f * input[__iter_0__-1+N*(__iter_1__+1+M*(__iter_2__))]);
        double __temp_30__;
        __temp_30__ = (__temp_28__ + __temp_29__);
        double __temp_31__;
        __temp_31__ = (0.71f * input[__iter_0__+N*(__iter_1__+1+M*(__iter_2__))]);
        double __temp_32__;
        __temp_32__ = (__temp_30__ + __temp_31__);
        double __temp_33__;
        __temp_33__ = (0.51f * input[__iter_0__+1+N*(__iter_1__+1+M*(__iter_2__))]);
        double __temp_34__;
        __temp_34__ = (__temp_32__ + __temp_33__);
        double __temp_35__;
        __temp_35__ = (0.52f * input[__iter_0__-1+N*(__iter_1__-1+M*(__iter_2__+1))]);
        double __temp_36__;
        __temp_36__ = (__temp_34__ + __temp_35__);
        double __temp_37__;
        __temp_37__ = (0.72f * input[__iter_0__+N*(__iter_1__-1+M*(__iter_2__+1))]);
        double __temp_38__;
        __temp_38__ = (__temp_36__ + __temp_37__);
        double __temp_39__;
        __temp_39__ = (0.92f * input[__iter_0__+1+N*(__iter_1__-1+M*(__iter_2__+1))]);
        double __temp_40__;
        __temp_40__ = (__temp_38__ + __temp_39__);
        double __temp_41__;
        __temp_41__ = (1.22f * input[__iter_0__-1+N*(__iter_1__+M*(__iter_2__+1))]);
        double __temp_42__;
        __temp_42__ = (__temp_40__ + __temp_41__);
        double __temp_43__;
        __temp_43__ = (1.52f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__+1))]);
        double __temp_44__;
        __temp_44__ = (__temp_42__ + __temp_43__);
        double __temp_45__;
        __temp_45__ = (1.22f * input[__iter_0__+1+N*(__iter_1__+M*(__iter_2__+1))]);
        double __temp_46__;
        __temp_46__ = (__temp_44__ + __temp_45__);
        double __temp_47__;
        __temp_47__ = (0.92f * input[__iter_0__-1+N*(__iter_1__+1+M*(__iter_2__+1))]);
        double __temp_48__;
        __temp_48__ = (__temp_46__ + __temp_47__);
        double __temp_49__;
        __temp_49__ = (0.72f * input[__iter_0__+N*(__iter_1__+1+M*(__iter_2__+1))]);
        double __temp_50__;
        __temp_50__ = (__temp_48__ + __temp_49__);
        double __temp_51__;
        __temp_51__ = (0.52f * input[__iter_0__+1+N*(__iter_1__+1+M*(__iter_2__+1))]);
        double __temp_52__;
        __temp_52__ = (__temp_50__ + __temp_51__);
        double __temp_53__;
        __temp_53__ = (__temp_52__ / 159);
        __var_4__[__iter_0__+N*(__iter_1__+M*(__iter_2__))] = __temp_53__;
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d27pt (double * h_input, int L, int M, int N, double * __var_0__){

/* Host allocation Begin */
  double * input;
  hipMalloc(&input,sizeof(double)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(double)*(L*M*N), memcpy_kind_h_input);
  }
  double * __var_1__;
  hipMalloc(&__var_1__,sizeof(double)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  double * __var_2__;
  hipMalloc(&__var_2__,sizeof(double)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel_j3d27pt0__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel_j3d27pt0__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel_j3d27pt0__ = ((L-2) - 1 ) + 1;
  int __block_0___kernel_j3d27pt0__ = 16;
  int __block_1___kernel_j3d27pt0__ = 4;
  int __block_2___kernel_j3d27pt0__ = 4;
  dim3 __blockConfig___kernel_j3d27pt0__(__block_0___kernel_j3d27pt0__,__block_1___kernel_j3d27pt0__,__block_2___kernel_j3d27pt0__);
  int __SMemSize___kernel_j3d27pt0__ = 0;
  int __grid_0___kernel_j3d27pt0__ = FORMA_CEIL(__size_0___kernel_j3d27pt0__,__block_0___kernel_j3d27pt0__);
  int __grid_1___kernel_j3d27pt0__ = FORMA_CEIL(__size_1___kernel_j3d27pt0__,__block_1___kernel_j3d27pt0__);
  int __grid_2___kernel_j3d27pt0__ = FORMA_CEIL(__size_2___kernel_j3d27pt0__,__block_2___kernel_j3d27pt0__);
  dim3 __gridConfig___kernel_j3d27pt0__(__grid_0___kernel_j3d27pt0__,__grid_1___kernel_j3d27pt0__,__grid_2___kernel_j3d27pt0__);

  __kernel_j3d27pt0__<<<__gridConfig___kernel_j3d27pt0__, __blockConfig___kernel_j3d27pt0__, __SMemSize___kernel_j3d27pt0__>>> (input, L, M, N, __var_2__);
  __kernel_j3d27pt0__<<<__gridConfig___kernel_j3d27pt0__, __blockConfig___kernel_j3d27pt0__, __SMemSize___kernel_j3d27pt0__>>> (__var_2__, L, M, N, __var_1__);
  __kernel_j3d27pt0__<<<__gridConfig___kernel_j3d27pt0__, __blockConfig___kernel_j3d27pt0__, __SMemSize___kernel_j3d27pt0__>>> (__var_1__, L, M, N, __var_2__);
  __kernel_j3d27pt0__<<<__gridConfig___kernel_j3d27pt0__, __blockConfig___kernel_j3d27pt0__, __SMemSize___kernel_j3d27pt0__>>> (__var_2__, L, M, N, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel_j3d27pt0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(double)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

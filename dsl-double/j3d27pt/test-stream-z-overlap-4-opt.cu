#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/* Device code Begin */
/* X, Y, Z */
__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, double * __restrict__ __var_1__){
  __shared__ double tilevar[4][32*32];
  double t[4], b[4], out=0.0f;

  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)-8);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)-8);
 
  // Initialize the values
  int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
  int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
      tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = 0.0f;
      tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = 0.0f;
      tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = 0.0f;
  }
  // Rest of the computation
  for (int __iter_2__ = 0; __iter_2__ <= L-1; __iter_2__++) {
    if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1))){
        tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = input[__iter_5__+N*(__iter_4__+M*(__iter_2__))]; 
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))){
	// Bottom 
	double __temp_3__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_7__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_8__ = (0.5f * __temp_3__ + 0.7f *__temp_7__);
        double __temp_12__ = (tilevar[0][ __iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_13__ = (__temp_8__ + 0.9f * __temp_12__);
        double __temp_17__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_18__ = (__temp_13__ + 1.2f * __temp_17__);
        double __temp_22__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_23__ = (__temp_18__ + 1.5f * __temp_22__);
        double __temp_27__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_28__ = (__temp_23__ + 1.2f * __temp_27__);
        double __temp_32__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_33__ = (__temp_28__ + 0.9f * __temp_32__);
        double __temp_37__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_38__ = (__temp_33__ + 0.7f * __temp_37__);
        double __temp_42__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_43__ = (__temp_38__ + 0.5f * __temp_42__) / 159;
	t[0] += __temp_43__;
	//Mid 
        double __temp_47__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_52__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_53__ = (0.51f * __temp_47__ + 0.71f * __temp_52__);
        double __temp_57__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_58__ = (__temp_53__ + 0.91f * __temp_57__);
        double __temp_62__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_63__ = (__temp_58__ + 1.21f * __temp_62__);
        double __temp_67__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_68__ = (__temp_63__ + 1.51f * __temp_67__);
        double __temp_72__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_73__ = (__temp_68__ + 1.21f * __temp_72__);
        double __temp_77__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_78__ = (__temp_73__ + 0.91f * __temp_77__);
        double __temp_82__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_83__ = (__temp_78__ + 0.71f * __temp_82__);
        double __temp_87__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_88__ = (__temp_83__ + 0.51f * __temp_87__) / 159;
	b[0] += __temp_88__;	
	// Top
        double __temp_92__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_97__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_98__ = (0.52f * __temp_92__ + 0.72f * __temp_97__);
        double __temp_102__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_103__ = (__temp_98__ + 0.92f * __temp_102__);
        double __temp_107__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_108__ = (__temp_103__ + 1.22f * __temp_107__);
        double __temp_112__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_113__ = (__temp_108__ + 1.52f * __temp_112__);
        double __temp_117__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_118__ = (__temp_113__ + 1.22f * __temp_117__);
        double __temp_122__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_123__ = (__temp_118__ + 0.92f * __temp_122__);
        double __temp_127__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_128__ = (__temp_123__ + 0.72f * __temp_127__);
        double __temp_132__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_133__ = (__temp_128__ + 0.52f * __temp_132__) / 159;
        tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] += __temp_133__;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+2),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))){
	double __temp_3__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_7__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_8__ = (0.5f * __temp_3__ + 0.7f *__temp_7__);
        double __temp_12__ = (tilevar[1][ __iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_13__ = (__temp_8__ + 0.9f * __temp_12__);
        double __temp_17__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_18__ = (__temp_13__ + 1.2f * __temp_17__);
        double __temp_22__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_23__ = (__temp_18__ + 1.5f * __temp_22__);
        double __temp_27__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_28__ = (__temp_23__ + 1.2f * __temp_27__);
        double __temp_32__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_33__ = (__temp_28__ + 0.9f * __temp_32__);
        double __temp_37__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_38__ = (__temp_33__ + 0.7f * __temp_37__);
        double __temp_42__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_43__ = (__temp_38__ + 0.5f * __temp_42__) / 159;
	t[1] += __temp_43__;
	//Mid 
        double __temp_47__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_52__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_53__ = (0.51f * __temp_47__ + 0.71f * __temp_52__);
        double __temp_57__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_58__ = (__temp_53__ + 0.91f * __temp_57__);
        double __temp_62__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_63__ = (__temp_58__ + 1.21f * __temp_62__);
        double __temp_67__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_68__ = (__temp_63__ + 1.51f * __temp_67__);
        double __temp_72__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_73__ = (__temp_68__ + 1.21f * __temp_72__);
        double __temp_77__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_78__ = (__temp_73__ + 0.91f * __temp_77__);
        double __temp_82__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_83__ = (__temp_78__ + 0.71f * __temp_82__);
        double __temp_87__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_88__ = (__temp_83__ + 0.51f * __temp_87__) / 159;
	b[1] += __temp_88__;	
	// Top
        double __temp_92__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_97__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_98__ = (0.52f * __temp_92__ + 0.72f * __temp_97__);
        double __temp_102__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_103__ = (__temp_98__ + 0.92f * __temp_102__);
        double __temp_107__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_108__ = (__temp_103__ + 1.22f * __temp_107__);
        double __temp_112__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_113__ = (__temp_108__ + 1.52f * __temp_112__);
        double __temp_117__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_118__ = (__temp_113__ + 1.22f * __temp_117__);
        double __temp_122__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_123__ = (__temp_118__ + 0.92f * __temp_122__);
        double __temp_127__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_128__ = (__temp_123__ + 0.72f * __temp_127__);
        double __temp_132__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_133__ = (__temp_128__ + 0.52f * __temp_132__) / 159;
        tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] += __temp_133__;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+3),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+3),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))){
	double __temp_3__ = (tilevar[2][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_7__ = (tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_8__ = (0.5f * __temp_3__ + 0.7f *__temp_7__);
        double __temp_12__ = (tilevar[2][ __iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_13__ = (__temp_8__ + 0.9f * __temp_12__);
        double __temp_17__ = (tilevar[2][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_18__ = (__temp_13__ + 1.2f * __temp_17__);
        double __temp_22__ = (tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_23__ = (__temp_18__ + 1.5f * __temp_22__);
        double __temp_27__ = (tilevar[2][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_28__ = (__temp_23__ + 1.2f * __temp_27__);
        double __temp_32__ = (tilevar[2][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_33__ = (__temp_28__ + 0.9f * __temp_32__);
        double __temp_37__ = (tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_38__ = (__temp_33__ + 0.7f * __temp_37__);
        double __temp_42__ = (tilevar[2][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_43__ = (__temp_38__ + 0.5f * __temp_42__) / 159;
	t[2] += __temp_43__;
	//Mid 
        double __temp_47__ = (tilevar[2][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_52__ = (tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_53__ = (0.51f * __temp_47__ + 0.71f * __temp_52__);
        double __temp_57__ = (tilevar[2][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_58__ = (__temp_53__ + 0.91f * __temp_57__);
        double __temp_62__ = (tilevar[2][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_63__ = (__temp_58__ + 1.21f * __temp_62__);
        double __temp_67__ = (tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_68__ = (__temp_63__ + 1.51f * __temp_67__);
        double __temp_72__ = (tilevar[2][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_73__ = (__temp_68__ + 1.21f * __temp_72__);
        double __temp_77__ = (tilevar[2][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_78__ = (__temp_73__ + 0.91f * __temp_77__);
        double __temp_82__ = (tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_83__ = (__temp_78__ + 0.71f * __temp_82__);
        double __temp_87__ = (tilevar[2][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_88__ = (__temp_83__ + 0.51f * __temp_87__) / 159;
	b[2] += __temp_88__;	
	// Top
        double __temp_92__ = (tilevar[2][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_97__ = (tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_98__ = (0.52f * __temp_92__ + 0.72f * __temp_97__);
        double __temp_102__ = (tilevar[2][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_103__ = (__temp_98__ + 0.92f * __temp_102__);
        double __temp_107__ = (tilevar[2][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_108__ = (__temp_103__ + 1.22f * __temp_107__);
        double __temp_112__ = (tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_113__ = (__temp_108__ + 1.52f * __temp_112__);
        double __temp_117__ = (tilevar[2][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_118__ = (__temp_113__ + 1.22f * __temp_117__);
        double __temp_122__ = (tilevar[2][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_123__ = (__temp_118__ + 0.92f * __temp_122__);
        double __temp_127__ = (tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_128__ = (__temp_123__ + 0.72f * __temp_127__);
        double __temp_132__ = (tilevar[2][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_133__ = (__temp_128__ + 0.52f * __temp_132__) / 159;
        tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] += __temp_133__;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+4),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+4),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2))){
	double __temp_3__ = (tilevar[3][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_7__ = (tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_8__ = (0.5f * __temp_3__ + 0.7f *__temp_7__);
        double __temp_12__ = (tilevar[3][ __iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_13__ = (__temp_8__ + 0.9f * __temp_12__);
        double __temp_17__ = (tilevar[3][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_18__ = (__temp_13__ + 1.2f * __temp_17__);
        double __temp_22__ = (tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_23__ = (__temp_18__ + 1.5f * __temp_22__);
        double __temp_27__ = (tilevar[3][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_28__ = (__temp_23__ + 1.2f * __temp_27__);
        double __temp_32__ = (tilevar[3][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_33__ = (__temp_28__ + 0.9f * __temp_32__);
        double __temp_37__ = (tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_38__ = (__temp_33__ + 0.7f * __temp_37__);
        double __temp_42__ = (tilevar[3][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_43__ = (__temp_38__ + 0.5f * __temp_42__) / 159;
	t[3] += __temp_43__;
	//Mid 
        double __temp_47__ = (tilevar[3][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_52__ = (tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_53__ = (0.51f * __temp_47__ + 0.71f * __temp_52__);
        double __temp_57__ = (tilevar[3][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_58__ = (__temp_53__ + 0.91f * __temp_57__);
        double __temp_62__ = (tilevar[3][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_63__ = (__temp_58__ + 1.21f * __temp_62__);
        double __temp_67__ = (tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_68__ = (__temp_63__ + 1.51f * __temp_67__);
        double __temp_72__ = (tilevar[3][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_73__ = (__temp_68__ + 1.21f * __temp_72__);
        double __temp_77__ = (tilevar[3][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_78__ = (__temp_73__ + 0.91f * __temp_77__);
        double __temp_82__ = (tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_83__ = (__temp_78__ + 0.71f * __temp_82__);
        double __temp_87__ = (tilevar[3][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_88__ = (__temp_83__ + 0.51f * __temp_87__) / 159;
	b[3] += __temp_88__;	
	// Top
        double __temp_92__ = (tilevar[3][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_97__ = (tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_98__ = (0.52f * __temp_92__ + 0.72f * __temp_97__);
        double __temp_102__ = (tilevar[3][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        double __temp_103__ = (__temp_98__ + 0.92f * __temp_102__);
        double __temp_107__ = (tilevar[3][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_108__ = (__temp_103__ + 1.22f * __temp_107__);
        double __temp_112__ = (tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_113__ = (__temp_108__ + 1.52f * __temp_112__);
        double __temp_117__ = (tilevar[3][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        double __temp_118__ = (__temp_113__ + 1.22f * __temp_117__);
        double __temp_122__ = (tilevar[3][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_123__ = (__temp_118__ + 0.92f * __temp_122__);
        double __temp_127__ = (tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_128__ = (__temp_123__ + 0.72f * __temp_127__);
        double __temp_132__ = (tilevar[3][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        double __temp_133__ = (__temp_128__ + 0.52f * __temp_132__) / 159;
	out += __temp_133__;
	__var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-4,0))] = out;
    }
    __syncthreads ();
    // Now rotate 
    tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = b[0];
    b[0] = t[0];
    t[0] = 0.0f;
    tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = b[1];
    b[1] = t[1];
    t[1] = 0.0f;
    tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = b[2];
    b[2] = t[2];
    t[2] = 0.0f;
    out= b[3]; 
    b[3] = t[3];
    t[3] = 0.0f;
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d27pt(double * h_input, int L, int M, int N, double * __var_0__){

/* Host allocation Begin */
  double * input;
  hipMalloc(&input,sizeof(double)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(double)*(L*M*N), memcpy_kind_h_input);
  }

  double * __var_1__;
  hipMalloc(&__var_1__,sizeof(double)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
  int __size_0___kernel___forma_kernel__0__ = N;
  int __size_1___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 32;
  int __block_2___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-8);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y-8);
  int __grid_2___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 unrollConfig (__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z);

  for (int i = 0 ; i < 125; i++) {
    __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
    __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig>>> (__var_1__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, input);
  }

  for (int n = 0; n < 5; n++) {

#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif


  for (int i = 0 ; i < 125; i++) {
    __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
    __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig>>> (__var_1__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, input);
  }

  #ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif

  }
    
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(double)*(L*M*N), memcpy_kind___var_0__);
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
  hipError_t error = hipGetLastError ();
  if (error != hipSuccess) {
    printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
    exit(-1);
  }
}

__global__ void hypterm (double * flux_0, double * flux_1, double * flux_2, double * flux_3, double * flux_4, double * cons_1, double * cons_2, double * cons_3, double * cons_4, double * q_1, double * q_2, double * q_3, double * q_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
  //Determining the block's indices
  int blockdim_i= (int)(blockDim.x);
  int i0 = (int)(blockIdx.x)*(blockdim_i-8);
  int i = i0 + (int)(threadIdx.x);
  int blockdim_j= (int)(blockDim.y);
  int j0 = (int)(blockIdx.y)*(blockdim_j)+4;
  int j = j0 + (int)(threadIdx.y);

  //Declarations
  double reg_cons_1_m4=0, reg_cons_1_m3=0, reg_cons_1_m2=0, reg_cons_1_m1=0, __shared__ sh_cons_1_c0[24][16], reg_cons_1_p1=0, reg_cons_1_p2=0, reg_cons_1_p3=0, reg_cons_1_p4=0;
  double reg_cons_2_m4=0, reg_cons_2_m3=0, reg_cons_2_m2=0, reg_cons_2_m1=0, __shared__ sh_cons_2_c0[24][16], reg_cons_2_p1=0, reg_cons_2_p2=0, reg_cons_2_p3=0, reg_cons_2_p4=0;
  double reg_cons_3_m4=0, reg_cons_3_m3=0, reg_cons_3_m2=0, reg_cons_3_m1=0, __shared__ sh_cons_3_c0[24][16], reg_cons_3_p1=0, reg_cons_3_p2=0, reg_cons_3_p3=0, reg_cons_3_p4=0;
  double reg_cons_4_m4=0, reg_cons_4_m3=0, reg_cons_4_m2=0, reg_cons_4_m1=0, __shared__ sh_cons_4_c0[24][16], reg_cons_4_p1=0, reg_cons_4_p2=0, reg_cons_4_p3=0, reg_cons_4_p4=0;
  double __shared__ sh_q_1_c0[24][16];
  double __shared__ sh_q_2_c0[24][16];
  double reg_q_3_m4=0, reg_q_3_m3=0, reg_q_3_m2=0, reg_q_3_m1=0, reg_q_3_c0=0, reg_q_3_p1=0, reg_q_3_p2=0, reg_q_3_p3=0, reg_q_3_p4=0;
  double reg_q_4_m4=0, reg_q_4_m3=0, reg_q_4_m2=0, reg_q_4_m1=0, __shared__ sh_q_4_c0[24][16], reg_q_4_p1=0, reg_q_4_p2=0, reg_q_4_p3=0, reg_q_4_p4=0;

  //Value Initialization
  if (j <= M-1 & i <= min (i0+blockdim_i-1, N-1)) {
    reg_cons_1_m4 = cons_1[0 + j*N + i];
    reg_cons_1_m3 = cons_1[1*M*N + j*N + i];
    reg_cons_1_m2 = cons_1[2*M*N + j*N + i];
    reg_cons_1_m1 = cons_1[3*M*N + j*N + i];
    sh_cons_1_c0[j-j0+4][i-i0] = cons_1[4*M*N + j*N + i];
    reg_cons_1_p1 = cons_1[5*M*N + j*N + i];
    reg_cons_1_p2 = cons_1[6*M*N + j*N + i];
    reg_cons_1_p3 = cons_1[7*M*N + j*N + i];
    reg_cons_2_m4 = cons_2[0 + j*N + i];
    reg_cons_2_m3 = cons_2[1*M*N + j*N + i];
    reg_cons_2_m2 = cons_2[2*M*N + j*N + i];
    reg_cons_2_m1 = cons_2[3*M*N + j*N + i];
    sh_cons_2_c0[j-j0+4][i-i0] = cons_2[4*M*N + j*N + i];
    reg_cons_2_p1 = cons_2[5*M*N + j*N + i];
    reg_cons_2_p2 = cons_2[6*M*N + j*N + i];
    reg_cons_2_p3 = cons_2[7*M*N + j*N + i];
    reg_cons_3_m4 = cons_3[0 + j*N + i];
    reg_cons_3_m3 = cons_3[1*M*N + j*N + i];
    reg_cons_3_m2 = cons_3[2*M*N + j*N + i];
    reg_cons_3_m1 = cons_3[3*M*N + j*N + i];
    sh_cons_3_c0[j-j0+4][i-i0] = cons_3[4*M*N + j*N + i];
    reg_cons_3_p1 = cons_3[5*M*N + j*N + i];
    reg_cons_3_p2 = cons_3[6*M*N + j*N + i];
    reg_cons_3_p3 = cons_3[7*M*N + j*N + i];
    reg_cons_4_m4 = cons_4[0 + j*N + i];
    reg_cons_4_m3 = cons_4[1*M*N + j*N + i];
    reg_cons_4_m2 = cons_4[2*M*N + j*N + i];
    reg_cons_4_m1 = cons_4[3*M*N + j*N + i];
    sh_cons_4_c0[j-j0+4][i-i0] = cons_4[4*M*N + j*N + i];
    reg_cons_4_p1 = cons_4[5*M*N + j*N + i];
    reg_cons_4_p2 = cons_4[6*M*N + j*N + i];
    reg_cons_4_p3 = cons_4[7*M*N + j*N + i];
    reg_q_3_m4 = q_3[0 + j*N + i];
    reg_q_3_m3 = q_3[1*M*N + j*N + i];
    reg_q_3_m2 = q_3[2*M*N + j*N + i];
    reg_q_3_m1 = q_3[3*M*N + j*N + i];
    reg_q_3_c0 = q_3[4*M*N + j*N + i];
    reg_q_3_p1 = q_3[5*M*N + j*N + i];
    reg_q_3_p2 = q_3[6*M*N + j*N + i];
    reg_q_3_p3 = q_3[7*M*N + j*N + i];
    reg_q_4_m4 = q_4[0 + j*N + i];
    reg_q_4_m3 = q_4[1*M*N + j*N + i];
    reg_q_4_m2 = q_4[2*M*N + j*N + i];
    reg_q_4_m1 = q_4[3*M*N + j*N + i];
    sh_q_4_c0[j-j0+4][i-i0] = q_4[4*M*N + j*N + i];
    reg_q_4_p1 = q_4[5*M*N + j*N + i];
    reg_q_4_p2 = q_4[6*M*N + j*N + i];
    reg_q_4_p3 = q_4[7*M*N + j*N + i];
  }
  if(threadIdx.y < 4) {
   int jj = (j-4);
   if (jj <= M-1 & i <= min (i0+blockdim_i-1, N-1)) {
    sh_cons_1_c0[jj-j0+4][i-i0] = cons_1[4*M*N + jj*N + i];
    sh_cons_2_c0[jj-j0+4][i-i0] = cons_2[4*M*N + jj*N + i];
    sh_cons_3_c0[jj-j0+4][i-i0] = cons_3[4*M*N + jj*N + i];
    sh_cons_4_c0[jj-j0+4][i-i0] = cons_4[4*M*N + jj*N + i];
    sh_q_4_c0[jj-j0+4][i-i0] = q_4[4*M*N + jj*N + i];
   }
  } else if(threadIdx.y < 8) {
   int jj = (j-4)+16;
   if (jj <= M-1 & i <= min (i0+blockdim_i-1, N-1)) {
    sh_cons_1_c0[jj-j0+4][i-i0] = cons_1[4*M*N + jj*N + i];
    sh_cons_2_c0[jj-j0+4][i-i0] = cons_2[4*M*N + jj*N + i];
    sh_cons_3_c0[jj-j0+4][i-i0] = cons_3[4*M*N + jj*N + i];
    sh_cons_4_c0[jj-j0+4][i-i0] = cons_4[4*M*N + jj*N + i];
    sh_q_4_c0[jj-j0+4][i-i0] = q_4[4*M*N + jj*N + i];
   }
  }

  //Rest of the computation
  for (int k=4; k<=L-5; ++k) {
    //Fetch new plane
    if (j <= M-1 & i <= min (i0+blockdim_i-1, N-1)) {
        reg_cons_1_p4 = cons_1[(k+4)*M*N + j*N + i];
        reg_cons_2_p4 = cons_2[(k+4)*M*N + j*N + i];
        reg_cons_3_p4 = cons_3[(k+4)*M*N + j*N + i];
        reg_cons_4_p4 = cons_4[(k+4)*M*N + j*N + i];
        sh_q_1_c0[j-j0+4][i-i0] = q_1[k*M*N + j*N + i];
        sh_q_2_c0[j-j0+4][i-i0] = q_2[k*M*N + j*N + i];
        reg_q_3_p4 = q_3[(k+4)*M*N + j*N + i];
        reg_q_4_p4 = q_4[(k+4)*M*N + j*N + i];
    }
    if(threadIdx.y < 4) {
     int jj = (j-4);
     if (jj <= M-1 & i <= min (i0+blockdim_i-1, N-1)) {
        sh_cons_1_c0[jj-j0+4][i-i0] = cons_1[k*M*N + jj*N + i];
        sh_cons_2_c0[jj-j0+4][i-i0] = cons_2[k*M*N + jj*N + i];
        sh_cons_3_c0[jj-j0+4][i-i0] = cons_3[k*M*N + jj*N + i];
        sh_cons_4_c0[jj-j0+4][i-i0] = cons_4[k*M*N + jj*N + i];
     }
    } else if(threadIdx.y < 8) {
     int jj = (j-4)+16;
     if (jj <= M-1 & i <= min (i0+blockdim_i-1, N-1)) {
        sh_cons_1_c0[jj-j0+4][i-i0] = cons_1[k*M*N + jj*N + i];
        sh_cons_2_c0[jj-j0+4][i-i0] = cons_2[k*M*N + jj*N + i];
        sh_cons_3_c0[jj-j0+4][i-i0] = cons_3[k*M*N + jj*N + i];
        sh_cons_4_c0[jj-j0+4][i-i0] = cons_4[k*M*N + jj*N + i];
     }
    } else if(threadIdx.y < 12) {
     int jj = (j-12);
     if (jj <= M-1 & i <= min (i0+blockdim_i-1, N-1)) {
        sh_q_1_c0[jj-j0+4][i-i0] = q_1[k*M*N + jj*N + i];
        sh_q_2_c0[jj-j0+4][i-i0] = q_2[k*M*N + jj*N + i];
        sh_q_4_c0[jj-j0+4][i-i0] = q_4[k*M*N + jj*N + i];
     }
    } else {
     int jj = (j-12)+16;
     if (jj <= M-1 & i <= min (i0+blockdim_i-1, N-1)) {
        sh_q_1_c0[jj-j0+4][i-i0] = q_1[k*M*N + jj*N + i];
        sh_q_2_c0[jj-j0+4][i-i0] = q_2[k*M*N + jj*N + i];
        sh_q_4_c0[jj-j0+4][i-i0] = q_4[k*M*N + jj*N + i];
     }
    }
    __syncthreads ();
	double r0,r1,r2,r3,r4;
    if ((j <= M-5) & i >= max (i0+4, 4) & i <= min (i0+blockdim_i-5, N-5)) {
	r0=flux_0[k*M*N + j*N + i], r1=flux_1[k*M*N + j*N + i], r2=flux_2[k*M*N + j*N + i], r3=flux_3[k*M*N + j*N + i], r4 = flux_4[k*M*N + j*N + i];
//	double r0=0.0f,r1=0.0f,r2=0.0f,r3=0.0f,r4=0.0f;
        r0 -= (((((0.8f * (sh_cons_1_c0[j-j0+4][i-i0+1] - sh_cons_1_c0[j-j0+4][i-i0-1])) - (0.2f * (sh_cons_1_c0[j-j0+4][i-i0+2] - sh_cons_1_c0[j-j0+4][i-i0-2]))) + (0.038f * (sh_cons_1_c0[j-j0+4][i-i0+3] - sh_cons_1_c0[j-j0+4][i-i0-3]))) - (0.0035f * (sh_cons_1_c0[j-j0+4][i-i0+4] - sh_cons_1_c0[j-j0+4][i-i0-4]))) * dxinv0);
        r0 -= (((((0.8f * (sh_cons_2_c0[j-j0+4+1][i-i0] - sh_cons_2_c0[j-j0+4-1][i-i0])) - (0.2f * (sh_cons_2_c0[j-j0+4+2][i-i0] - sh_cons_2_c0[j-j0+4-2][i-i0]))) + (0.038f * (sh_cons_2_c0[j-j0+4+3][i-i0] - sh_cons_2_c0[j-j0+4-3][i-i0]))) - (0.0035f * (sh_cons_2_c0[j-j0+4+4][i-i0] - sh_cons_2_c0[j-j0+4-4][i-i0]))) * dxinv1);
        r0 -= (((((0.8f * (reg_cons_3_p1 - reg_cons_3_m1)) - (0.2f * (reg_cons_3_p2 - reg_cons_3_m2))) + (0.038f * (reg_cons_3_p3 - reg_cons_3_m3))) - (0.0035f * (reg_cons_3_p4 - reg_cons_3_m4))) * dxinv2);
	flux_0[k*M*N + j*N + i] = r0;
        r1 -= (((((0.8f * (((sh_cons_1_c0[j-j0+4][i-i0+1] * sh_q_1_c0[j-j0+4][i-i0+1]) - (sh_cons_1_c0[j-j0+4][i-i0-1] * sh_q_1_c0[j-j0+4][i-i0-1])) + (sh_q_4_c0[j-j0+4][i-i0+1] - sh_q_4_c0[j-j0+4][i-i0-1]))) - (0.2f * (((sh_cons_1_c0[j-j0+4][i-i0+2] * sh_q_1_c0[j-j0+4][i-i0+2]) - (sh_cons_1_c0[j-j0+4][i-i0-2] * sh_q_1_c0[j-j0+4][i-i0-2])) + (sh_q_4_c0[j-j0+4][i-i0+2] - sh_q_4_c0[j-j0+4][i-i0-2])))) + (0.038f * (((sh_cons_1_c0[j-j0+4][i-i0+3] * sh_q_1_c0[j-j0+4][i-i0+3]) - (sh_cons_1_c0[j-j0+4][i-i0-3] * sh_q_1_c0[j-j0+4][i-i0-3])) + (sh_q_4_c0[j-j0+4][i-i0+3] - sh_q_4_c0[j-j0+4][i-i0-3])))) - (0.0035f * (((sh_cons_1_c0[j-j0+4][i-i0+4] * sh_q_1_c0[j-j0+4][i-i0+4]) - (sh_cons_1_c0[j-j0+4][i-i0-4] * sh_q_1_c0[j-j0+4][i-i0-4])) + (sh_q_4_c0[j-j0+4][i-i0+4] - sh_q_4_c0[j-j0+4][i-i0-4])))) * dxinv0);
        r1 -= (((((0.8f * ((sh_cons_1_c0[j-j0+4+1][i-i0] * sh_q_2_c0[j-j0+4+1][i-i0]) - (sh_cons_1_c0[j-j0+4-1][i-i0] * sh_q_2_c0[j-j0+4-1][i-i0]))) - (0.2f * ((sh_cons_1_c0[j-j0+4+2][i-i0] * sh_q_2_c0[j-j0+4+2][i-i0]) - (sh_cons_1_c0[j-j0+4-2][i-i0] * sh_q_2_c0[j-j0+4-2][i-i0])))) + (0.038f * ((sh_cons_1_c0[j-j0+4+3][i-i0] * sh_q_2_c0[j-j0+4+3][i-i0]) - (sh_cons_1_c0[j-j0+4-3][i-i0] * sh_q_2_c0[j-j0+4-3][i-i0])))) - (0.0035f * ((sh_cons_1_c0[j-j0+4+4][i-i0] * sh_q_2_c0[j-j0+4+4][i-i0]) - (sh_cons_1_c0[j-j0+4-4][i-i0] * sh_q_2_c0[j-j0+4-4][i-i0])))) * dxinv1);
        r1 -= (((((0.8f * ((reg_cons_1_p1 * reg_q_3_p1) - (reg_cons_1_m1 * reg_q_3_m1))) - (0.2f * ((reg_cons_1_p2 * reg_q_3_p2) - (reg_cons_1_m2 * reg_q_3_m2)))) + (0.038f * ((reg_cons_1_p3 * reg_q_3_p3) - (reg_cons_1_m3 * reg_q_3_m3)))) - (0.0035f * ((reg_cons_1_p4 * reg_q_3_p4) - (reg_cons_1_m4 * reg_q_3_m4)))) * dxinv2);
	flux_1[k*M*N + j*N + i] = r1;
        r2 -= (((((0.8f * ((sh_cons_2_c0[j-j0+4][i-i0+1] * sh_q_1_c0[j-j0+4][i-i0+1]) - (sh_cons_2_c0[j-j0+4][i-i0-1] * sh_q_1_c0[j-j0+4][i-i0-1]))) - (0.2f * ((sh_cons_2_c0[j-j0+4][i-i0+2] * sh_q_1_c0[j-j0+4][i-i0+2]) - (sh_cons_2_c0[j-j0+4][i-i0-2] * sh_q_1_c0[j-j0+4][i-i0-2])))) + (0.038f * ((sh_cons_2_c0[j-j0+4][i-i0+3] * sh_q_1_c0[j-j0+4][i-i0+3]) - (sh_cons_2_c0[j-j0+4][i-i0-3] * sh_q_1_c0[j-j0+4][i-i0-3])))) - (0.0035f * ((sh_cons_2_c0[j-j0+4][i-i0+4] * sh_q_1_c0[j-j0+4][i-i0+4]) - (sh_cons_2_c0[j-j0+4][i-i0-4] * sh_q_1_c0[j-j0+4][i-i0-4])))) * dxinv0);
        r2 -= (((((0.8f * (((sh_cons_2_c0[j-j0+4+1][i-i0] * sh_q_2_c0[j-j0+4+1][i-i0]) - (sh_cons_2_c0[j-j0+4-1][i-i0] * sh_q_2_c0[j-j0+4-1][i-i0])) + (sh_q_4_c0[j-j0+4+1][i-i0] - sh_q_4_c0[j-j0+4-1][i-i0]))) - (0.2f * (((sh_cons_2_c0[j-j0+4+2][i-i0] * sh_q_2_c0[j-j0+4+2][i-i0]) - (sh_cons_2_c0[j-j0+4-2][i-i0] * sh_q_2_c0[j-j0+4-2][i-i0])) + (sh_q_4_c0[j-j0+4+2][i-i0] - sh_q_4_c0[j-j0+4-2][i-i0])))) + (0.038f * (((sh_cons_2_c0[j-j0+4+3][i-i0] * sh_q_2_c0[j-j0+4+3][i-i0]) - (sh_cons_2_c0[j-j0+4-3][i-i0] * sh_q_2_c0[j-j0+4-3][i-i0])) + (sh_q_4_c0[j-j0+4+3][i-i0] - sh_q_4_c0[j-j0+4-3][i-i0])))) - (0.0035f * (((sh_cons_2_c0[j-j0+4+4][i-i0] * sh_q_2_c0[j-j0+4+4][i-i0]) - (sh_cons_2_c0[j-j0+4-4][i-i0] * sh_q_2_c0[j-j0+4-4][i-i0])) + (sh_q_4_c0[j-j0+4+4][i-i0] - sh_q_4_c0[j-j0+4-4][i-i0])))) * dxinv1);
        r2 -= (((((0.8f * ((reg_cons_2_p1 * reg_q_3_p1) - (reg_cons_2_m1 * reg_q_3_m1))) - (0.2f * ((reg_cons_2_p2 * reg_q_3_p2) - (reg_cons_2_m2 * reg_q_3_m2)))) + (0.038f * ((reg_cons_2_p3 * reg_q_3_p3) - (reg_cons_2_m3 * reg_q_3_m3)))) - (0.0035f * ((reg_cons_2_p4 * reg_q_3_p4) - (reg_cons_2_m4 * reg_q_3_m4)))) * dxinv2);
	flux_2[k*M*N + j*N + i] = r2;
        r3 -= (((((0.8f * ((sh_cons_3_c0[j-j0+4][i-i0+1] * sh_q_1_c0[j-j0+4][i-i0+1]) - (sh_cons_3_c0[j-j0+4][i-i0-1] * sh_q_1_c0[j-j0+4][i-i0-1]))) - (0.2f * ((sh_cons_3_c0[j-j0+4][i-i0+2] * sh_q_1_c0[j-j0+4][i-i0+2]) - (sh_cons_3_c0[j-j0+4][i-i0-2] * sh_q_1_c0[j-j0+4][i-i0-2])))) + (0.038f * ((sh_cons_3_c0[j-j0+4][i-i0+3] * sh_q_1_c0[j-j0+4][i-i0+3]) - (sh_cons_3_c0[j-j0+4][i-i0-3] * sh_q_1_c0[j-j0+4][i-i0-3])))) - (0.0035f * ((sh_cons_3_c0[j-j0+4][i-i0+4] * sh_q_1_c0[j-j0+4][i-i0+4]) - (sh_cons_3_c0[j-j0+4][i-i0-4] * sh_q_1_c0[j-j0+4][i-i0-4])))) * dxinv0);
        r3 -= (((((0.8f * ((sh_cons_3_c0[j-j0+4+1][i-i0] * sh_q_2_c0[j-j0+4+1][i-i0]) - (sh_cons_3_c0[j-j0+4-1][i-i0] * sh_q_2_c0[j-j0+4-1][i-i0]))) - (0.2f * ((sh_cons_3_c0[j-j0+4+2][i-i0] * sh_q_2_c0[j-j0+4+2][i-i0]) - (sh_cons_3_c0[j-j0+4-2][i-i0] * sh_q_2_c0[j-j0+4-2][i-i0])))) + (0.038f * ((sh_cons_3_c0[j-j0+4+3][i-i0] * sh_q_2_c0[j-j0+4+3][i-i0]) - (sh_cons_3_c0[j-j0+4-3][i-i0] * sh_q_2_c0[j-j0+4-3][i-i0])))) - (0.0035f * ((sh_cons_3_c0[j-j0+4+4][i-i0] * sh_q_2_c0[j-j0+4+4][i-i0]) - (sh_cons_3_c0[j-j0+4-4][i-i0] * sh_q_2_c0[j-j0+4-4][i-i0])))) * dxinv1);
        r3 -= (((((0.8f * (((reg_cons_3_p1 * reg_q_3_p1) - (reg_cons_3_m1 * reg_q_3_m1)) + (reg_q_4_p1 - reg_q_4_m1))) - (0.2f * (((reg_cons_3_p2 * reg_q_3_p2) - (reg_cons_3_m2 * reg_q_3_m2)) + (reg_q_4_p2 - reg_q_4_m2)))) + (0.038f * (((reg_cons_3_p3 * reg_q_3_p3) - (reg_cons_3_m3 * reg_q_3_m3)) + (reg_q_4_p3 - reg_q_4_m3)))) - (0.0035f * (((reg_cons_3_p4 * reg_q_3_p4) - (reg_cons_3_m4 * reg_q_3_m4)) + (reg_q_4_p4 - reg_q_4_m4)))) * dxinv2);
	flux_3[k*M*N + j*N + i] = r3;
        r4 -= (((((0.8f * (((sh_cons_4_c0[j-j0+4][i-i0+1] * sh_q_1_c0[j-j0+4][i-i0+1]) - (sh_cons_4_c0[j-j0+4][i-i0-1] * sh_q_1_c0[j-j0+4][i-i0-1])) + ((sh_q_4_c0[j-j0+4][i-i0+1] * sh_q_1_c0[j-j0+4][i-i0+1]) - (sh_q_4_c0[j-j0+4][i-i0-1] * sh_q_1_c0[j-j0+4][i-i0-1])))) - (0.2f * (((sh_cons_4_c0[j-j0+4][i-i0+2] * sh_q_1_c0[j-j0+4][i-i0+2]) - (sh_cons_4_c0[j-j0+4][i-i0-2] * sh_q_1_c0[j-j0+4][i-i0-2])) + ((sh_q_4_c0[j-j0+4][i-i0+2] * sh_q_1_c0[j-j0+4][i-i0+2]) - (sh_q_4_c0[j-j0+4][i-i0-2] * sh_q_1_c0[j-j0+4][i-i0-2]))))) + (0.038f * (((sh_cons_4_c0[j-j0+4][i-i0+3] * sh_q_1_c0[j-j0+4][i-i0+3]) - (sh_cons_4_c0[j-j0+4][i-i0-3] * sh_q_1_c0[j-j0+4][i-i0-3])) + ((sh_q_4_c0[j-j0+4][i-i0+3] * sh_q_1_c0[j-j0+4][i-i0+3]) - (sh_q_4_c0[j-j0+4][i-i0-3] * sh_q_1_c0[j-j0+4][i-i0-3]))))) - (0.0035f * (((sh_cons_4_c0[j-j0+4][i-i0+4] * sh_q_1_c0[j-j0+4][i-i0+4]) - (sh_cons_4_c0[j-j0+4][i-i0-4] * sh_q_1_c0[j-j0+4][i-i0-4])) + ((sh_q_4_c0[j-j0+4][i-i0+4] * sh_q_1_c0[j-j0+4][i-i0+4]) - (sh_q_4_c0[j-j0+4][i-i0-4] * sh_q_1_c0[j-j0+4][i-i0-4]))))) * dxinv0);
        r4 -= (((((0.8f * (((sh_cons_4_c0[j-j0+4+1][i-i0] * sh_q_2_c0[j-j0+4+1][i-i0]) - (sh_cons_4_c0[j-j0+4-1][i-i0] * sh_q_2_c0[j-j0+4-1][i-i0])) + ((sh_q_4_c0[j-j0+4+1][i-i0] * sh_q_2_c0[j-j0+4+1][i-i0]) - (sh_q_4_c0[j-j0+4-1][i-i0] * sh_q_2_c0[j-j0+4-1][i-i0])))) - (0.2f * (((sh_cons_4_c0[j-j0+4+2][i-i0] * sh_q_2_c0[j-j0+4+2][i-i0]) - (sh_cons_4_c0[j-j0+4-2][i-i0] * sh_q_2_c0[j-j0+4-2][i-i0])) + ((sh_q_4_c0[j-j0+4+2][i-i0] * sh_q_2_c0[j-j0+4+2][i-i0]) - (sh_q_4_c0[j-j0+4-2][i-i0] * sh_q_2_c0[j-j0+4-2][i-i0]))))) + (0.038f * (((sh_cons_4_c0[j-j0+4+3][i-i0] * sh_q_2_c0[j-j0+4+3][i-i0]) - (sh_cons_4_c0[j-j0+4-3][i-i0] * sh_q_2_c0[j-j0+4-3][i-i0])) + ((sh_q_4_c0[j-j0+4+3][i-i0] * sh_q_2_c0[j-j0+4+3][i-i0]) - (sh_q_4_c0[j-j0+4-3][i-i0] * sh_q_2_c0[j-j0+4-3][i-i0]))))) - (0.0035f * (((sh_cons_4_c0[j-j0+4+4][i-i0] * sh_q_2_c0[j-j0+4+4][i-i0]) - (sh_cons_4_c0[j-j0+4-4][i-i0] * sh_q_2_c0[j-j0+4-4][i-i0])) + ((sh_q_4_c0[j-j0+4+4][i-i0] * sh_q_2_c0[j-j0+4+4][i-i0]) - (sh_q_4_c0[j-j0+4-4][i-i0] * sh_q_2_c0[j-j0+4-4][i-i0]))))) * dxinv1);
        r4 -= (((((0.8f * (((reg_cons_4_p1 * reg_q_3_p1) - (reg_cons_4_m1 * reg_q_3_m1)) + ((reg_q_4_p1 * reg_q_3_p1) - (reg_q_4_m1 * reg_q_3_m1)))) - (0.2f * (((reg_cons_4_p2 * reg_q_3_p2) - (reg_cons_4_m2 * reg_q_3_m2)) + ((reg_q_4_p2 * reg_q_3_p2) - (reg_q_4_m2 * reg_q_3_m2))))) + (0.038f * (((reg_cons_4_p3 * reg_q_3_p3) - (reg_cons_4_m3 * reg_q_3_m3)) + ((reg_q_4_p3 * reg_q_3_p3) - (reg_q_4_m3 * reg_q_3_m3))))) - (0.0035f * (((reg_cons_4_p4 * reg_q_3_p4) - (reg_cons_4_m4 * reg_q_3_m4)) + ((reg_q_4_p4 * reg_q_3_p4) - (reg_q_4_m4 * reg_q_3_m4))))) * dxinv2);
	flux_4[k*M*N + j*N + i] = r4;
    }
    __syncthreads ();
    //Value rotation


  if (j <= M-1 & i <= min (i0+blockdim_i-1, N-1)) {
    reg_cons_1_m4 = reg_cons_1_m3;
    reg_cons_1_m3 = reg_cons_1_m2;
    reg_cons_1_m2 = reg_cons_1_m1;
    reg_cons_1_m1 = sh_cons_1_c0[j-j0+4][i-i0];
    sh_cons_1_c0[j-j0+4][i-i0] = reg_cons_1_p1;
    reg_cons_1_p1 = reg_cons_1_p2;
    reg_cons_1_p2 = reg_cons_1_p3;
    reg_cons_1_p3 = reg_cons_1_p4;
    reg_cons_2_m4 = reg_cons_2_m3;
    reg_cons_2_m3 = reg_cons_2_m2;
    reg_cons_2_m2 = reg_cons_2_m1;
    reg_cons_2_m1 = sh_cons_2_c0[j-j0+4][i-i0];
    sh_cons_2_c0[j-j0+4][i-i0] = reg_cons_2_p1;
    reg_cons_2_p1 = reg_cons_2_p2;
    reg_cons_2_p2 = reg_cons_2_p3;
    reg_cons_2_p3 = reg_cons_2_p4;
    reg_cons_3_m4 = reg_cons_3_m3;
    reg_cons_3_m3 = reg_cons_3_m2;
    reg_cons_3_m2 = reg_cons_3_m1;
    reg_cons_3_m1 = sh_cons_3_c0[j-j0+4][i-i0];
    sh_cons_3_c0[j-j0+4][i-i0] = reg_cons_3_p1;
    reg_cons_3_p1 = reg_cons_3_p2;
    reg_cons_3_p2 = reg_cons_3_p3;
    reg_cons_3_p3 = reg_cons_3_p4;
    reg_cons_4_m4 = reg_cons_4_m3;
    reg_cons_4_m3 = reg_cons_4_m2;
    reg_cons_4_m2 = reg_cons_4_m1;
    reg_cons_4_m1 = sh_cons_4_c0[j-j0+4][i-i0];
    sh_cons_4_c0[j-j0+4][i-i0] = reg_cons_4_p1;
    reg_cons_4_p1 = reg_cons_4_p2;
    reg_cons_4_p2 = reg_cons_4_p3;
    reg_cons_4_p3 = reg_cons_4_p4;
    reg_q_3_m4 = reg_q_3_m3;
    reg_q_3_m3 = reg_q_3_m2;
    reg_q_3_m2 = reg_q_3_m1;
    reg_q_3_m1 = reg_q_3_c0;
    reg_q_3_c0 = reg_q_3_p1;
    reg_q_3_p1 = reg_q_3_p2;
    reg_q_3_p2 = reg_q_3_p3;
    reg_q_3_p3 = reg_q_3_p4;
    reg_q_4_m4 = reg_q_4_m3;
    reg_q_4_m3 = reg_q_4_m2;
    reg_q_4_m2 = reg_q_4_m1;
    reg_q_4_m1 = sh_q_4_c0[j-j0+4][i-i0];
    sh_q_4_c0[j-j0+4][i-i0] = reg_q_4_p1;
    reg_q_4_p1 = reg_q_4_p2;
    reg_q_4_p2 = reg_q_4_p3;
    reg_q_4_p3 = reg_q_4_p4;
   }
  }
}

extern "C" void host_code (double *h_flux_0, double *h_flux_1, double *h_flux_2, double *h_flux_3, double *h_flux_4, double *h_cons_1, double *h_cons_2, double *h_cons_3, double *h_cons_4, double *h_q_1, double *h_q_2, double *h_q_3, double *h_q_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
  double *flux_0;
  hipMalloc (&flux_0, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for flux_0\n");
  hipMemcpy (flux_0, h_flux_0, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *flux_1;
  hipMalloc (&flux_1, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for flux_1\n");
  hipMemcpy (flux_1, h_flux_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *flux_2;
  hipMalloc (&flux_2, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for flux_2\n");
  hipMemcpy (flux_2, h_flux_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *flux_3;
  hipMalloc (&flux_3, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for flux_3\n");
  hipMemcpy (flux_3, h_flux_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *flux_4;
  hipMalloc (&flux_4, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for flux_4\n");
  hipMemcpy (flux_4, h_flux_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *cons_1;
  hipMalloc (&cons_1, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for cons_1\n");
  hipMemcpy (cons_1, h_cons_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *cons_2;
  hipMalloc (&cons_2, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for cons_2\n");
  hipMemcpy (cons_2, h_cons_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *cons_3;
  hipMalloc (&cons_3, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for cons_3\n");
  hipMemcpy (cons_3, h_cons_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *cons_4;
  hipMalloc (&cons_4, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for cons_4\n");
  hipMemcpy (cons_4, h_cons_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *q_1;
  hipMalloc (&q_1, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for q_1\n");
  hipMemcpy (q_1, h_q_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *q_2;
  hipMalloc (&q_2, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for q_2\n");
  hipMemcpy (q_2, h_q_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *q_3;
  hipMalloc (&q_3, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for q_3\n");
  hipMemcpy (q_3, h_q_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *q_4;
  hipMalloc (&q_4, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for q_4\n");
  hipMemcpy (q_4, h_q_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);

  dim3 blockconfig_1 (16, 16, 1);
  dim3 gridconfig_1 (ceil(N, blockconfig_1.x-8), ceil(M-8, blockconfig_1.y), 1);
  hypterm <<<gridconfig_1, blockconfig_1>>> (flux_0, flux_1, flux_2, flux_3, flux_4, cons_1, cons_2, cons_3, cons_4, q_1, q_2, q_3, q_4, dxinv0, dxinv1, dxinv2, L, M, N);

  hipMemcpy (h_flux_0, flux_0, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
  hipMemcpy (h_flux_1, flux_1, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
  hipMemcpy (h_flux_2, flux_2, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
  hipMemcpy (h_flux_3, flux_3, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
  hipMemcpy (h_flux_4, flux_4, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);

  //Free allocated memory
  hipFree (flux_0);
  hipFree (flux_1);
  hipFree (flux_2);
  hipFree (flux_3);
  hipFree (flux_4);
  hipFree (cons_1);
  hipFree (cons_2);
  hipFree (cons_3);
  hipFree (cons_4);
  hipFree (q_1);
  hipFree (q_2);
  hipFree (q_3);
  hipFree (q_4);
}

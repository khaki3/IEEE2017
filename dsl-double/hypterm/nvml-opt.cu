#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)
#include <nvml.h>
#include <assert.h>


void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void hypterm (double * __restrict__ flux_0, double * __restrict__ flux_1, double * __restrict__ flux_2, double * __restrict__ flux_3, double * __restrict__ flux_4, double * __restrict__ cons_1, double * __restrict__ cons_2, double * __restrict__ cons_3, double * __restrict__ cons_4, double * __restrict__ q_1, double * __restrict__ q_2, double * __restrict__ q_3, double * __restrict__ q_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	//Determining the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i-8);
	int i = i0 + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j-8);
	int j = j0 + (int)(threadIdx.y);

	//Declarations
	double reg_cons_1_m4=0, reg_cons_1_m3=0, reg_cons_1_m2=0, reg_cons_1_m1=0, __shared__ sh_cons_1_c0[16][16], reg_cons_1_p1=0, reg_cons_1_p2=0, reg_cons_1_p3=0, reg_cons_1_p4=0;
	double reg_cons_2_m4=0, reg_cons_2_m3=0, reg_cons_2_m2=0, reg_cons_2_m1=0, __shared__ sh_cons_2_c0[16][16], reg_cons_2_p1=0, reg_cons_2_p2=0, reg_cons_2_p3=0, reg_cons_2_p4=0;
	double reg_cons_3_m4=0, reg_cons_3_m3=0, reg_cons_3_m2=0, reg_cons_3_m1=0, __shared__ sh_cons_3_c0[16][16], reg_cons_3_p1=0, reg_cons_3_p2=0, reg_cons_3_p3=0, reg_cons_3_p4=0;
	double reg_cons_4_m4=0, reg_cons_4_m3=0, reg_cons_4_m2=0, reg_cons_4_m1=0, __shared__ sh_cons_4_c0[16][16], reg_cons_4_p1=0, reg_cons_4_p2=0, reg_cons_4_p3=0, reg_cons_4_p4=0;
	double __shared__ sh_q_1_c0[16][16];
	double __shared__ sh_q_2_c0[16][16];
	double reg_q_3_m4=0, reg_q_3_m3=0, reg_q_3_m2=0, reg_q_3_m1=0, reg_q_3_c0=0, reg_q_3_p1=0, reg_q_3_p2=0, reg_q_3_p3=0, reg_q_3_p4=0;
	double reg_q_4_m4=0, reg_q_4_m3=0, reg_q_4_m2=0, reg_q_4_m1=0, __shared__ sh_q_4_c0[16][16], reg_q_4_p1=0, reg_q_4_p2=0, reg_q_4_p3=0, reg_q_4_p4=0;

	//Value Initialization
	if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
		reg_cons_1_m4 = cons_1[0 + j*N + i];
		reg_cons_1_m3 = cons_1[1*M*N + j*N + i];
		reg_cons_1_m2 = cons_1[2*M*N + j*N + i];
		reg_cons_1_m1 = cons_1[3*M*N + j*N + i];
		sh_cons_1_c0[j-j0][i-i0] = cons_1[4*M*N + j*N + i];
		reg_cons_1_p1 = cons_1[5*M*N + j*N + i];
		reg_cons_1_p2 = cons_1[6*M*N + j*N + i];
		reg_cons_1_p3 = cons_1[7*M*N + j*N + i];
		reg_cons_2_m4 = cons_2[0 + j*N + i];
		reg_cons_2_m3 = cons_2[1*M*N + j*N + i];
		reg_cons_2_m2 = cons_2[2*M*N + j*N + i];
		reg_cons_2_m1 = cons_2[3*M*N + j*N + i];
		sh_cons_2_c0[j-j0][i-i0] = cons_2[4*M*N + j*N + i];
		reg_cons_2_p1 = cons_2[5*M*N + j*N + i];
		reg_cons_2_p2 = cons_2[6*M*N + j*N + i];
		reg_cons_2_p3 = cons_2[7*M*N + j*N + i];
		reg_cons_3_m4 = cons_3[0 + j*N + i];
		reg_cons_3_m3 = cons_3[1*M*N + j*N + i];
		reg_cons_3_m2 = cons_3[2*M*N + j*N + i];
		reg_cons_3_m1 = cons_3[3*M*N + j*N + i];
		sh_cons_3_c0[j-j0][i-i0] = cons_3[4*M*N + j*N + i];
		reg_cons_3_p1 = cons_3[5*M*N + j*N + i];
		reg_cons_3_p2 = cons_3[6*M*N + j*N + i];
		reg_cons_3_p3 = cons_3[7*M*N + j*N + i];
		reg_cons_4_m4 = cons_4[0 + j*N + i];
		reg_cons_4_m3 = cons_4[1*M*N + j*N + i];
		reg_cons_4_m2 = cons_4[2*M*N + j*N + i];
		reg_cons_4_m1 = cons_4[3*M*N + j*N + i];
		sh_cons_4_c0[j-j0][i-i0] = cons_4[4*M*N + j*N + i];
		reg_cons_4_p1 = cons_4[5*M*N + j*N + i];
		reg_cons_4_p2 = cons_4[6*M*N + j*N + i];
		reg_cons_4_p3 = cons_4[7*M*N + j*N + i];
		reg_q_3_m4 = q_3[0 + j*N + i];
		reg_q_3_m3 = q_3[1*M*N + j*N + i];
		reg_q_3_m2 = q_3[2*M*N + j*N + i];
		reg_q_3_m1 = q_3[3*M*N + j*N + i];
		reg_q_3_c0 = q_3[4*M*N + j*N + i];
		reg_q_3_p1 = q_3[5*M*N + j*N + i];
		reg_q_3_p2 = q_3[6*M*N + j*N + i];
		reg_q_3_p3 = q_3[7*M*N + j*N + i];
		reg_q_4_m4 = q_4[0 + j*N + i];
		reg_q_4_m3 = q_4[1*M*N + j*N + i];
		reg_q_4_m2 = q_4[2*M*N + j*N + i];
		reg_q_4_m1 = q_4[3*M*N + j*N + i];
		sh_q_4_c0[j-j0][i-i0] = q_4[4*M*N + j*N + i];
		reg_q_4_p1 = q_4[5*M*N + j*N + i];
		reg_q_4_p2 = q_4[6*M*N + j*N + i];
		reg_q_4_p3 = q_4[7*M*N + j*N + i];
	}

	//Rest of the computation
	for (int k=4; k<=L-5; ++k) {
		//Fetch new plane
		if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
			reg_cons_1_p4 = cons_1[(k+4)*M*N + j*N + i];
			reg_cons_2_p4 = cons_2[(k+4)*M*N + j*N + i];
			reg_cons_3_p4 = cons_3[(k+4)*M*N + j*N + i];
			reg_cons_4_p4 = cons_4[(k+4)*M*N + j*N + i];
			sh_q_1_c0[j-j0][i-i0] = q_1[k*M*N + j*N + i];
			sh_q_2_c0[j-j0][i-i0] = q_2[k*M*N + j*N + i];
			reg_q_3_p4 = q_3[(k+4)*M*N + j*N + i];
			reg_q_4_p4 = q_4[(k+4)*M*N + j*N + i];
		}
		__syncthreads ();
		if (j >= j0+4 & j <= min (j0+blockdim_j-5, M-1) & i >= i0+4 & i <= min (i0+blockdim_i-5, N-5)) {
			double f0 = -(((((0.8f * (sh_cons_1_c0[j-j0][i-i0+1] - sh_cons_1_c0[j-j0][i-i0-1])) - (0.2f * (sh_cons_1_c0[j-j0][i-i0+2] - sh_cons_1_c0[j-j0][i-i0-2]))) + (0.038f * (sh_cons_1_c0[j-j0][i-i0+3] - sh_cons_1_c0[j-j0][i-i0-3]))) - (0.0035f * (sh_cons_1_c0[j-j0][i-i0+4] - sh_cons_1_c0[j-j0][i-i0-4]))) * dxinv0);
			f0 -= (((((0.8f * (sh_cons_2_c0[j-j0+1][i-i0] - sh_cons_2_c0[j-j0-1][i-i0])) - (0.2f * (sh_cons_2_c0[j-j0+2][i-i0] - sh_cons_2_c0[j-j0-2][i-i0]))) + (0.038f * (sh_cons_2_c0[j-j0+3][i-i0] - sh_cons_2_c0[j-j0-3][i-i0]))) - (0.0035f * (sh_cons_2_c0[j-j0+4][i-i0] - sh_cons_2_c0[j-j0-4][i-i0]))) * dxinv1);
			f0 -= (((((0.8f * (reg_cons_3_p1 - reg_cons_3_m1)) - (0.2f * (reg_cons_3_p2 - reg_cons_3_m2))) + (0.038f * (reg_cons_3_p3 - reg_cons_3_m3))) - (0.0035f * (reg_cons_3_p4 - reg_cons_3_m4))) * dxinv2);
			flux_0[k*M*N + j*N + i] = f0;
			double f1 = -(((((0.8f * (((sh_cons_1_c0[j-j0][i-i0+1] * sh_q_1_c0[j-j0][i-i0+1]) - (sh_cons_1_c0[j-j0][i-i0-1] * sh_q_1_c0[j-j0][i-i0-1])) + (sh_q_4_c0[j-j0][i-i0+1] - sh_q_4_c0[j-j0][i-i0-1]))) - (0.2f * (((sh_cons_1_c0[j-j0][i-i0+2] * sh_q_1_c0[j-j0][i-i0+2]) - (sh_cons_1_c0[j-j0][i-i0-2] * sh_q_1_c0[j-j0][i-i0-2])) + (sh_q_4_c0[j-j0][i-i0+2] - sh_q_4_c0[j-j0][i-i0-2])))) + (0.038f * (((sh_cons_1_c0[j-j0][i-i0+3] * sh_q_1_c0[j-j0][i-i0+3]) - (sh_cons_1_c0[j-j0][i-i0-3] * sh_q_1_c0[j-j0][i-i0-3])) + (sh_q_4_c0[j-j0][i-i0+3] - sh_q_4_c0[j-j0][i-i0-3])))) - (0.0035f * (((sh_cons_1_c0[j-j0][i-i0+4] * sh_q_1_c0[j-j0][i-i0+4]) - (sh_cons_1_c0[j-j0][i-i0-4] * sh_q_1_c0[j-j0][i-i0-4])) + (sh_q_4_c0[j-j0][i-i0+4] - sh_q_4_c0[j-j0][i-i0-4])))) * dxinv0);
			f1 -= (((((0.8f * ((sh_cons_1_c0[j-j0+1][i-i0] * sh_q_2_c0[j-j0+1][i-i0]) - (sh_cons_1_c0[j-j0-1][i-i0] * sh_q_2_c0[j-j0-1][i-i0]))) - (0.2f * ((sh_cons_1_c0[j-j0+2][i-i0] * sh_q_2_c0[j-j0+2][i-i0]) - (sh_cons_1_c0[j-j0-2][i-i0] * sh_q_2_c0[j-j0-2][i-i0])))) + (0.038f * ((sh_cons_1_c0[j-j0+3][i-i0] * sh_q_2_c0[j-j0+3][i-i0]) - (sh_cons_1_c0[j-j0-3][i-i0] * sh_q_2_c0[j-j0-3][i-i0])))) - (0.0035f * ((sh_cons_1_c0[j-j0+4][i-i0] * sh_q_2_c0[j-j0+4][i-i0]) - (sh_cons_1_c0[j-j0-4][i-i0] * sh_q_2_c0[j-j0-4][i-i0])))) * dxinv1);
			f1 -= (((((0.8f * ((reg_cons_1_p1 * reg_q_3_p1) - (reg_cons_1_m1 * reg_q_3_m1))) - (0.2f * ((reg_cons_1_p2 * reg_q_3_p2) - (reg_cons_1_m2 * reg_q_3_m2)))) + (0.038f * ((reg_cons_1_p3 * reg_q_3_p3) - (reg_cons_1_m3 * reg_q_3_m3)))) - (0.0035f * ((reg_cons_1_p4 * reg_q_3_p4) - (reg_cons_1_m4 * reg_q_3_m4)))) * dxinv2);
			flux_1[k*M*N + j*N + i] = f1;
			double f2 = -(((((0.8f * ((sh_cons_2_c0[j-j0][i-i0+1] * sh_q_1_c0[j-j0][i-i0+1]) - (sh_cons_2_c0[j-j0][i-i0-1] * sh_q_1_c0[j-j0][i-i0-1]))) - (0.2f * ((sh_cons_2_c0[j-j0][i-i0+2] * sh_q_1_c0[j-j0][i-i0+2]) - (sh_cons_2_c0[j-j0][i-i0-2] * sh_q_1_c0[j-j0][i-i0-2])))) + (0.038f * ((sh_cons_2_c0[j-j0][i-i0+3] * sh_q_1_c0[j-j0][i-i0+3]) - (sh_cons_2_c0[j-j0][i-i0-3] * sh_q_1_c0[j-j0][i-i0-3])))) - (0.0035f * ((sh_cons_2_c0[j-j0][i-i0+4] * sh_q_1_c0[j-j0][i-i0+4]) - (sh_cons_2_c0[j-j0][i-i0-4] * sh_q_1_c0[j-j0][i-i0-4])))) * dxinv0);
			f2 -= (((((0.8f * (((sh_cons_2_c0[j-j0+1][i-i0] * sh_q_2_c0[j-j0+1][i-i0]) - (sh_cons_2_c0[j-j0-1][i-i0] * sh_q_2_c0[j-j0-1][i-i0])) + (sh_q_4_c0[j-j0+1][i-i0] - sh_q_4_c0[j-j0-1][i-i0]))) - (0.2f * (((sh_cons_2_c0[j-j0+2][i-i0] * sh_q_2_c0[j-j0+2][i-i0]) - (sh_cons_2_c0[j-j0-2][i-i0] * sh_q_2_c0[j-j0-2][i-i0])) + (sh_q_4_c0[j-j0+2][i-i0] - sh_q_4_c0[j-j0-2][i-i0])))) + (0.038f * (((sh_cons_2_c0[j-j0+3][i-i0] * sh_q_2_c0[j-j0+3][i-i0]) - (sh_cons_2_c0[j-j0-3][i-i0] * sh_q_2_c0[j-j0-3][i-i0])) + (sh_q_4_c0[j-j0+3][i-i0] - sh_q_4_c0[j-j0-3][i-i0])))) - (0.0035f * (((sh_cons_2_c0[j-j0+4][i-i0] * sh_q_2_c0[j-j0+4][i-i0]) - (sh_cons_2_c0[j-j0-4][i-i0] * sh_q_2_c0[j-j0-4][i-i0])) + (sh_q_4_c0[j-j0+4][i-i0] - sh_q_4_c0[j-j0-4][i-i0])))) * dxinv1);
			f2 -= (((((0.8f * ((reg_cons_2_p1 * reg_q_3_p1) - (reg_cons_2_m1 * reg_q_3_m1))) - (0.2f * ((reg_cons_2_p2 * reg_q_3_p2) - (reg_cons_2_m2 * reg_q_3_m2)))) + (0.038f * ((reg_cons_2_p3 * reg_q_3_p3) - (reg_cons_2_m3 * reg_q_3_m3)))) - (0.0035f * ((reg_cons_2_p4 * reg_q_3_p4) - (reg_cons_2_m4 * reg_q_3_m4)))) * dxinv2);
			flux_2[k*M*N + j*N + i] = f2;
			double f3 = -(((((0.8f * ((sh_cons_3_c0[j-j0][i-i0+1] * sh_q_1_c0[j-j0][i-i0+1]) - (sh_cons_3_c0[j-j0][i-i0-1] * sh_q_1_c0[j-j0][i-i0-1]))) - (0.2f * ((sh_cons_3_c0[j-j0][i-i0+2] * sh_q_1_c0[j-j0][i-i0+2]) - (sh_cons_3_c0[j-j0][i-i0-2] * sh_q_1_c0[j-j0][i-i0-2])))) + (0.038f * ((sh_cons_3_c0[j-j0][i-i0+3] * sh_q_1_c0[j-j0][i-i0+3]) - (sh_cons_3_c0[j-j0][i-i0-3] * sh_q_1_c0[j-j0][i-i0-3])))) - (0.0035f * ((sh_cons_3_c0[j-j0][i-i0+4] * sh_q_1_c0[j-j0][i-i0+4]) - (sh_cons_3_c0[j-j0][i-i0-4] * sh_q_1_c0[j-j0][i-i0-4])))) * dxinv0);
			f3 -= (((((0.8f * ((sh_cons_3_c0[j-j0+1][i-i0] * sh_q_2_c0[j-j0+1][i-i0]) - (sh_cons_3_c0[j-j0-1][i-i0] * sh_q_2_c0[j-j0-1][i-i0]))) - (0.2f * ((sh_cons_3_c0[j-j0+2][i-i0] * sh_q_2_c0[j-j0+2][i-i0]) - (sh_cons_3_c0[j-j0-2][i-i0] * sh_q_2_c0[j-j0-2][i-i0])))) + (0.038f * ((sh_cons_3_c0[j-j0+3][i-i0] * sh_q_2_c0[j-j0+3][i-i0]) - (sh_cons_3_c0[j-j0-3][i-i0] * sh_q_2_c0[j-j0-3][i-i0])))) - (0.0035f * ((sh_cons_3_c0[j-j0+4][i-i0] * sh_q_2_c0[j-j0+4][i-i0]) - (sh_cons_3_c0[j-j0-4][i-i0] * sh_q_2_c0[j-j0-4][i-i0])))) * dxinv1);
			f3 -= (((((0.8f * (((reg_cons_3_p1 * reg_q_3_p1) - (reg_cons_3_m1 * reg_q_3_m1)) + (reg_q_4_p1 - reg_q_4_m1))) - (0.2f * (((reg_cons_3_p2 * reg_q_3_p2) - (reg_cons_3_m2 * reg_q_3_m2)) + (reg_q_4_p2 - reg_q_4_m2)))) + (0.038f * (((reg_cons_3_p3 * reg_q_3_p3) - (reg_cons_3_m3 * reg_q_3_m3)) + (reg_q_4_p3 - reg_q_4_m3)))) - (0.0035f * (((reg_cons_3_p4 * reg_q_3_p4) - (reg_cons_3_m4 * reg_q_3_m4)) + (reg_q_4_p4 - reg_q_4_m4)))) * dxinv2);
			flux_3[k*M*N + j*N + i] = f3;
			double f4 = -(((((0.8f * (((sh_cons_4_c0[j-j0][i-i0+1] * sh_q_1_c0[j-j0][i-i0+1]) - (sh_cons_4_c0[j-j0][i-i0-1] * sh_q_1_c0[j-j0][i-i0-1])) + ((sh_q_4_c0[j-j0][i-i0+1] * sh_q_1_c0[j-j0][i-i0+1]) - (sh_q_4_c0[j-j0][i-i0-1] * sh_q_1_c0[j-j0][i-i0-1])))) - (0.2f * (((sh_cons_4_c0[j-j0][i-i0+2] * sh_q_1_c0[j-j0][i-i0+2]) - (sh_cons_4_c0[j-j0][i-i0-2] * sh_q_1_c0[j-j0][i-i0-2])) + ((sh_q_4_c0[j-j0][i-i0+2] * sh_q_1_c0[j-j0][i-i0+2]) - (sh_q_4_c0[j-j0][i-i0-2] * sh_q_1_c0[j-j0][i-i0-2]))))) + (0.038f * (((sh_cons_4_c0[j-j0][i-i0+3] * sh_q_1_c0[j-j0][i-i0+3]) - (sh_cons_4_c0[j-j0][i-i0-3] * sh_q_1_c0[j-j0][i-i0-3])) + ((sh_q_4_c0[j-j0][i-i0+3] * sh_q_1_c0[j-j0][i-i0+3]) - (sh_q_4_c0[j-j0][i-i0-3] * sh_q_1_c0[j-j0][i-i0-3]))))) - (0.0035f * (((sh_cons_4_c0[j-j0][i-i0+4] * sh_q_1_c0[j-j0][i-i0+4]) - (sh_cons_4_c0[j-j0][i-i0-4] * sh_q_1_c0[j-j0][i-i0-4])) + ((sh_q_4_c0[j-j0][i-i0+4] * sh_q_1_c0[j-j0][i-i0+4]) - (sh_q_4_c0[j-j0][i-i0-4] * sh_q_1_c0[j-j0][i-i0-4]))))) * dxinv0);
			f4 -= (((((0.8f * (((sh_cons_4_c0[j-j0+1][i-i0] * sh_q_2_c0[j-j0+1][i-i0]) - (sh_cons_4_c0[j-j0-1][i-i0] * sh_q_2_c0[j-j0-1][i-i0])) + ((sh_q_4_c0[j-j0+1][i-i0] * sh_q_2_c0[j-j0+1][i-i0]) - (sh_q_4_c0[j-j0-1][i-i0] * sh_q_2_c0[j-j0-1][i-i0])))) - (0.2f * (((sh_cons_4_c0[j-j0+2][i-i0] * sh_q_2_c0[j-j0+2][i-i0]) - (sh_cons_4_c0[j-j0-2][i-i0] * sh_q_2_c0[j-j0-2][i-i0])) + ((sh_q_4_c0[j-j0+2][i-i0] * sh_q_2_c0[j-j0+2][i-i0]) - (sh_q_4_c0[j-j0-2][i-i0] * sh_q_2_c0[j-j0-2][i-i0]))))) + (0.038f * (((sh_cons_4_c0[j-j0+3][i-i0] * sh_q_2_c0[j-j0+3][i-i0]) - (sh_cons_4_c0[j-j0-3][i-i0] * sh_q_2_c0[j-j0-3][i-i0])) + ((sh_q_4_c0[j-j0+3][i-i0] * sh_q_2_c0[j-j0+3][i-i0]) - (sh_q_4_c0[j-j0-3][i-i0] * sh_q_2_c0[j-j0-3][i-i0]))))) - (0.0035f * (((sh_cons_4_c0[j-j0+4][i-i0] * sh_q_2_c0[j-j0+4][i-i0]) - (sh_cons_4_c0[j-j0-4][i-i0] * sh_q_2_c0[j-j0-4][i-i0])) + ((sh_q_4_c0[j-j0+4][i-i0] * sh_q_2_c0[j-j0+4][i-i0]) - (sh_q_4_c0[j-j0-4][i-i0] * sh_q_2_c0[j-j0-4][i-i0]))))) * dxinv1);
			f4 -= (((((0.8f * (((reg_cons_4_p1 * reg_q_3_p1) - (reg_cons_4_m1 * reg_q_3_m1)) + ((reg_q_4_p1 * reg_q_3_p1) - (reg_q_4_m1 * reg_q_3_m1)))) - (0.2f * (((reg_cons_4_p2 * reg_q_3_p2) - (reg_cons_4_m2 * reg_q_3_m2)) + ((reg_q_4_p2 * reg_q_3_p2) - (reg_q_4_m2 * reg_q_3_m2))))) + (0.038f * (((reg_cons_4_p3 * reg_q_3_p3) - (reg_cons_4_m3 * reg_q_3_m3)) + ((reg_q_4_p3 * reg_q_3_p3) - (reg_q_4_m3 * reg_q_3_m3))))) - (0.0035f * (((reg_cons_4_p4 * reg_q_3_p4) - (reg_cons_4_m4 * reg_q_3_m4)) + ((reg_q_4_p4 * reg_q_3_p4) - (reg_q_4_m4 * reg_q_3_m4))))) * dxinv2);
			flux_4[k*M*N + j*N + i] = f4;
		}
		__syncthreads ();
		//Value rotation
		reg_cons_1_m4 = reg_cons_1_m3;
		reg_cons_1_m3 = reg_cons_1_m2;
		reg_cons_1_m2 = reg_cons_1_m1;
		reg_cons_1_m1 = sh_cons_1_c0[j-j0][i-i0];
		sh_cons_1_c0[j-j0][i-i0] = reg_cons_1_p1;
		reg_cons_1_p1 = reg_cons_1_p2;
		reg_cons_1_p2 = reg_cons_1_p3;
		reg_cons_1_p3 = reg_cons_1_p4;
		reg_cons_2_m4 = reg_cons_2_m3;
		reg_cons_2_m3 = reg_cons_2_m2;
		reg_cons_2_m2 = reg_cons_2_m1;
		reg_cons_2_m1 = sh_cons_2_c0[j-j0][i-i0];
		sh_cons_2_c0[j-j0][i-i0] = reg_cons_2_p1;
		reg_cons_2_p1 = reg_cons_2_p2;
		reg_cons_2_p2 = reg_cons_2_p3;
		reg_cons_2_p3 = reg_cons_2_p4;
		reg_cons_3_m4 = reg_cons_3_m3;
		reg_cons_3_m3 = reg_cons_3_m2;
		reg_cons_3_m2 = reg_cons_3_m1;
		reg_cons_3_m1 = sh_cons_3_c0[j-j0][i-i0];
		sh_cons_3_c0[j-j0][i-i0] = reg_cons_3_p1;
		reg_cons_3_p1 = reg_cons_3_p2;
		reg_cons_3_p2 = reg_cons_3_p3;
		reg_cons_3_p3 = reg_cons_3_p4;
		reg_cons_4_m4 = reg_cons_4_m3;
		reg_cons_4_m3 = reg_cons_4_m2;
		reg_cons_4_m2 = reg_cons_4_m1;
		reg_cons_4_m1 = sh_cons_4_c0[j-j0][i-i0];
		sh_cons_4_c0[j-j0][i-i0] = reg_cons_4_p1;
		reg_cons_4_p1 = reg_cons_4_p2;
		reg_cons_4_p2 = reg_cons_4_p3;
		reg_cons_4_p3 = reg_cons_4_p4;
		reg_q_3_m4 = reg_q_3_m3;
		reg_q_3_m3 = reg_q_3_m2;
		reg_q_3_m2 = reg_q_3_m1;
		reg_q_3_m1 = reg_q_3_c0;
		reg_q_3_c0 = reg_q_3_p1;
		reg_q_3_p1 = reg_q_3_p2;
		reg_q_3_p2 = reg_q_3_p3;
		reg_q_3_p3 = reg_q_3_p4;
		reg_q_4_m4 = reg_q_4_m3;
		reg_q_4_m3 = reg_q_4_m2;
		reg_q_4_m2 = reg_q_4_m1;
		reg_q_4_m1 = sh_q_4_c0[j-j0][i-i0];
		sh_q_4_c0[j-j0][i-i0] = reg_q_4_p1;
		reg_q_4_p1 = reg_q_4_p2;
		reg_q_4_p2 = reg_q_4_p3;
		reg_q_4_p3 = reg_q_4_p4;
	}
}

extern "C" void host_code (double *h_flux_0, double *h_flux_1, double *h_flux_2, double *h_flux_3, double *h_flux_4, double *h_cons_1, double *h_cons_2, double *h_cons_3, double *h_cons_4, double *h_q_1, double *h_q_2, double *h_q_3, double *h_q_4, double dxinv0, double dxinv1, double dxinv2, int L, int M, int N) {
	double *flux_0;
	hipMalloc (&flux_0, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_0\n");
	hipMemcpy (flux_0, h_flux_0, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_1;
	hipMalloc (&flux_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_1\n");
	hipMemcpy (flux_1, h_flux_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_2;
	hipMalloc (&flux_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_2\n");
	hipMemcpy (flux_2, h_flux_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_3;
	hipMalloc (&flux_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_3\n");
	hipMemcpy (flux_3, h_flux_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *flux_4;
	hipMalloc (&flux_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for flux_4\n");
	hipMemcpy (flux_4, h_flux_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_1;
	hipMalloc (&cons_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_1\n");
	hipMemcpy (cons_1, h_cons_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_2;
	hipMalloc (&cons_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_2\n");
	hipMemcpy (cons_2, h_cons_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_3;
	hipMalloc (&cons_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_3\n");
	hipMemcpy (cons_3, h_cons_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *cons_4;
	hipMalloc (&cons_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for cons_4\n");
	hipMemcpy (cons_4, h_cons_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_1;
	hipMalloc (&q_1, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_1\n");
	hipMemcpy (q_1, h_q_1, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_2;
	hipMalloc (&q_2, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_2\n");
	hipMemcpy (q_2, h_q_2, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_3;
	hipMalloc (&q_3, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_3\n");
	hipMemcpy (q_3, h_q_3, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
	double *q_4;
	hipMalloc (&q_4, sizeof(double)*L*M*N);
	check_error ("Failed to allocate device memory for q_4\n");
	hipMemcpy (q_4, h_q_4, sizeof(double)*L*M*N, hipMemcpyHostToDevice);

	dim3 blockconfig_1 (16, 16, 1);
	dim3 gridconfig_1 (ceil(N, blockconfig_1.x-8), ceil(M, blockconfig_1.y-8), 1);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<500; x++) {
		hypterm <<<gridconfig_1, blockconfig_1>>> (flux_0, flux_1, flux_2, flux_3, flux_4, cons_1, cons_2, cons_3, cons_4, q_1, q_2, q_3, q_4, dxinv0, dxinv1, dxinv2, L, M, N);
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();

	hipMemcpy (h_flux_0, flux_0, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_1, flux_1, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_2, flux_2, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_3, flux_3, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
	hipMemcpy (h_flux_4, flux_4, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);

	//Free allocated memory
	hipFree (flux_0);
	hipFree (flux_1);
	hipFree (flux_2);
	hipFree (flux_3);
	hipFree (flux_4);
	hipFree (cons_1);
	hipFree (cons_2);
	hipFree (cons_3);
	hipFree (cons_4);
	hipFree (q_1);
	hipFree (q_2);
	hipFree (q_3);
	hipFree (q_4);
}

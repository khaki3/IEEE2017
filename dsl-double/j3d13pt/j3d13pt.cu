#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_3__;
  __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1)) ; __iter_3__ += (int)(blockDim.z) ){
    int __iter_4__;
    __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
    if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) ){
      int __iter_5__;
      __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        __tilevar_2__[__iter_5__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-__iter_1__)+(FORMA_BLOCKDIM_Y-0)*(__iter_3__+(0-__iter_2__)))] = input[__iter_5__+(N-0)*(__iter_4__+(M-0)*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__;
  __iter_6__ = FORMA_MAX((__iter_2__+2),2) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-3)) ; __iter_6__ += (int)(blockDim.z) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_1__+2),2) + (int)(threadIdx.y) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-3)) ){
      int __iter_8__;
      __iter_8__ = FORMA_MAX((__iter_0__+2),2) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-3)) ){
        float __temp_3__ = (__tilevar_2__[__iter_8__+2-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_7__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_8__ = (0.083000f * __temp_3__ + 0.083000f * __temp_7__);
        float __temp_12__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_13__ = (__temp_8__ + 0.083000f * __temp_12__);
        float __temp_17__ = (__tilevar_2__[__iter_8__-2-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_18__ = (__temp_13__ + 0.083000f * __temp_17__);
        float __temp_22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+2-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_23__ = (__temp_18__ + 0.083000f * __temp_22__);
        float __temp_27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_28__ = (__temp_23__ + 0.083000f * __temp_27__);
        float __temp_32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_33__ = (__temp_28__ + 0.083000f * __temp_32__);
        float __temp_37__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-2-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_38__ = (__temp_33__ + 0.083000f * __temp_37__);
        float __temp_42__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+2-__iter_2__))]);
        float __temp_43__ = (__temp_38__ + 0.083000f * __temp_42__);
        float __temp_47__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__+1-__iter_2__))]);
        float __temp_48__ = (__temp_43__ + 0.083000f * __temp_47__);
        float __temp_52__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-1-__iter_2__))]);
        float __temp_53__ = (__temp_48__ + 0.083000f * __temp_52__);
        float __temp_57__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-2-__iter_2__))]);
        float __temp_58__ = (__temp_53__ + 0.083000f * __temp_57__);
        float __temp_62__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))]);
        float __temp_63__ = (__temp_58__ - 0.996000f * __temp_62__);
        __tilevar_3__[__iter_8__+(0-(__iter_0__+2))+FORMA_BLOCKDIM_X*(__iter_7__+(0-(__iter_1__+2))+FORMA_BLOCKDIM_Y*(__iter_6__+(0-(__iter_2__+2))))] = __temp_63__;

        //int __temp_0__;
        //__temp_0__ = __iter_6__;
        //int __temp_1__;
        //__temp_1__ = __iter_7__;
        //int __temp_2__;
        //__temp_2__ = __iter_8__+(2);
        //float __temp_3__;
        //__temp_3__ = (0.083000f * __tilevar_2__[__temp_2__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_1__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_0__+(0-(__iter_2__+0))))]);
        //int __temp_4__;
        //__temp_4__ = __iter_6__;
        //int __temp_5__;
        //__temp_5__ = __iter_7__;
        //int __temp_6__;
        //__temp_6__ = __iter_8__+(1);
        //float __temp_7__;
        //__temp_7__ = (0.083000f * __tilevar_2__[__temp_6__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_5__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_4__+(0-(__iter_2__+0))))]);
        //float __temp_8__;
        //__temp_8__ = (__temp_3__ + __temp_7__);
        //int __temp_9__;
        //__temp_9__ = __iter_6__;
        //int __temp_10__;
        //__temp_10__ = __iter_7__;
        //int __temp_11__;
        //__temp_11__ = __iter_8__+(-1);
        //float __temp_12__;
        //__temp_12__ = (0.083000f * __tilevar_2__[__temp_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_10__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_9__+(0-(__iter_2__+0))))]);
        //float __temp_13__;
        //__temp_13__ = (__temp_8__ + __temp_12__);
        //int __temp_14__;
        //__temp_14__ = __iter_6__;
        //int __temp_15__;
        //__temp_15__ = __iter_7__;
        //int __temp_16__;
        //__temp_16__ = __iter_8__+(-2);
        //float __temp_17__;
        //__temp_17__ = (0.083000f * __tilevar_2__[__temp_16__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_15__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_14__+(0-(__iter_2__+0))))]);
        //float __temp_18__;
        //__temp_18__ = (__temp_13__ + __temp_17__);
        //int __temp_19__;
        //__temp_19__ = __iter_6__;
        //int __temp_20__;
        //__temp_20__ = __iter_7__+(2);
        //int __temp_21__;
        //__temp_21__ = __iter_8__;
        //float __temp_22__;
        //__temp_22__ = (0.083000f * __tilevar_2__[__temp_21__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_20__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_19__+(0-(__iter_2__+0))))]);
        //float __temp_23__;
        //__temp_23__ = (__temp_18__ + __temp_22__);
        //int __temp_24__;
        //__temp_24__ = __iter_6__;
        //int __temp_25__;
        //__temp_25__ = __iter_7__+(1);
        //int __temp_26__;
        //__temp_26__ = __iter_8__;
        //float __temp_27__;
        //__temp_27__ = (0.083000f * __tilevar_2__[__temp_26__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_25__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_24__+(0-(__iter_2__+0))))]);
        //float __temp_28__;
        //__temp_28__ = (__temp_23__ + __temp_27__);
        //int __temp_29__;
        //__temp_29__ = __iter_6__;
        //int __temp_30__;
        //__temp_30__ = __iter_7__+(-1);
        //int __temp_31__;
        //__temp_31__ = __iter_8__;
        //float __temp_32__;
        //__temp_32__ = (0.083000f * __tilevar_2__[__temp_31__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_30__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_29__+(0-(__iter_2__+0))))]);
        //float __temp_33__;
        //__temp_33__ = (__temp_28__ + __temp_32__);
        //int __temp_34__;
        //__temp_34__ = __iter_6__;
        //int __temp_35__;
        //__temp_35__ = __iter_7__+(-2);
        //int __temp_36__;
        //__temp_36__ = __iter_8__;
        //float __temp_37__;
        //__temp_37__ = (0.083000f * __tilevar_2__[__temp_36__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_35__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_34__+(0-(__iter_2__+0))))]);
        //float __temp_38__;
        //__temp_38__ = (__temp_33__ + __temp_37__);
        //int __temp_39__;
        //__temp_39__ = __iter_6__+(2);
        //int __temp_40__;
        //__temp_40__ = __iter_7__;
        //int __temp_41__;
        //__temp_41__ = __iter_8__;
        //float __temp_42__;
        //__temp_42__ = (0.083000f * __tilevar_2__[__temp_41__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_40__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_39__+(0-(__iter_2__+0))))]);
        //float __temp_43__;
        //__temp_43__ = (__temp_38__ + __temp_42__);
        //int __temp_44__;
        //__temp_44__ = __iter_6__+(1);
        //int __temp_45__;
        //__temp_45__ = __iter_7__;
        //int __temp_46__;
        //__temp_46__ = __iter_8__;
        //float __temp_47__;
        //__temp_47__ = (0.083000f * __tilevar_2__[__temp_46__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_45__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_44__+(0-(__iter_2__+0))))]);
        //float __temp_48__;
        //__temp_48__ = (__temp_43__ + __temp_47__);
        //int __temp_49__;
        //__temp_49__ = __iter_6__+(-1);
        //int __temp_50__;
        //__temp_50__ = __iter_7__;
        //int __temp_51__;
        //__temp_51__ = __iter_8__;
        //float __temp_52__;
        //__temp_52__ = (0.083000f * __tilevar_2__[__temp_51__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_50__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_49__+(0-(__iter_2__+0))))]);
        //float __temp_53__;
        //__temp_53__ = (__temp_48__ + __temp_52__);
        //int __temp_54__;
        //__temp_54__ = __iter_6__+(-2);
        //int __temp_55__;
        //__temp_55__ = __iter_7__;
        //int __temp_56__;
        //__temp_56__ = __iter_8__;
        //float __temp_57__;
        //__temp_57__ = (0.083000f * __tilevar_2__[__temp_56__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_55__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_54__+(0-(__iter_2__+0))))]);
        //float __temp_58__;
        //__temp_58__ = (__temp_53__ + __temp_57__);
        //int __temp_59__;
        //__temp_59__ = __iter_6__;
        //int __temp_60__;
        //__temp_60__ = __iter_7__;
        //int __temp_61__;
        //__temp_61__ = __iter_8__;
        //float __temp_62__;
        //__temp_62__ = (0.996000f * __tilevar_2__[__temp_61__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_60__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_59__+(0-(__iter_2__+0))))]);
        //float __temp_63__;
        //__temp_63__ = (__temp_58__ - __temp_62__);
        //__tilevar_3__[__iter_8__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+2))))] = __temp_63__;
      }
    }
  }
  int __iter_9__;
  __iter_9__ = FORMA_MAX((__iter_2__+2),2) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-3)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__;
    __iter_10__ = FORMA_MAX((__iter_1__+2),2) + (int)(threadIdx.y) ; 
    if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-3)) ){
      int __iter_11__;
      __iter_11__ = FORMA_MAX((__iter_0__+2),2) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-3)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__+2),2)+4) || __iter_9__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-3))-4) || __iter_10__ < (FORMA_MAX((__iter_1__+2),2)+4) || __iter_10__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-3))-4) || __iter_11__ < (FORMA_MAX((__iter_0__+2),2)+4) || __iter_11__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-3))-4)) {
          __copy_arr_0__[__iter_11__+(N-0)*(__iter_10__+(M-0)*(__iter_9__))] = __tilevar_3__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_9__+(0-(__iter_2__+2))))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_15__;
  __iter_15__ = FORMA_MAX((__iter_2__+4),2) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-3)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__;
    __iter_16__ = FORMA_MAX((__iter_1__+4),2) + (int)(threadIdx.y) ; 
    if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-3)) ){
      int __iter_17__;
      __iter_17__ = FORMA_MAX((__iter_0__+4),2) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-3)) ){
        int __temp_89__;
        __temp_89__ = __iter_15__;
        int __temp_90__;
        __temp_90__ = __iter_16__;
        int __temp_91__;
        __temp_91__ = __iter_17__+(2);
        float __temp_92__;
        __temp_92__ = (0.083000f * __tilevar_3__[__temp_91__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_90__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_89__+(0-(__iter_2__+2))))]);
        int __temp_93__;
        __temp_93__ = __iter_15__;
        int __temp_94__;
        __temp_94__ = __iter_16__;
        int __temp_95__;
        __temp_95__ = __iter_17__+(1);
        float __temp_96__;
        __temp_96__ = (0.083000f * __tilevar_3__[__temp_95__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_94__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_93__+(0-(__iter_2__+2))))]);
        float __temp_97__;
        __temp_97__ = (__temp_92__ + __temp_96__);
        int __temp_98__;
        __temp_98__ = __iter_15__;
        int __temp_99__;
        __temp_99__ = __iter_16__;
        int __temp_100__;
        __temp_100__ = __iter_17__+(-1);
        float __temp_101__;
        __temp_101__ = (0.083000f * __tilevar_3__[__temp_100__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_99__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_98__+(0-(__iter_2__+2))))]);
        float __temp_102__;
        __temp_102__ = (__temp_97__ + __temp_101__);
        int __temp_103__;
        __temp_103__ = __iter_15__;
        int __temp_104__;
        __temp_104__ = __iter_16__;
        int __temp_105__;
        __temp_105__ = __iter_17__+(-2);
        float __temp_106__;
        __temp_106__ = (0.083000f * __tilevar_3__[__temp_105__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_104__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_103__+(0-(__iter_2__+2))))]);
        float __temp_107__;
        __temp_107__ = (__temp_102__ + __temp_106__);
        int __temp_108__;
        __temp_108__ = __iter_15__;
        int __temp_109__;
        __temp_109__ = __iter_16__+(2);
        int __temp_110__;
        __temp_110__ = __iter_17__;
        float __temp_111__;
        __temp_111__ = (0.083000f * __tilevar_3__[__temp_110__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_109__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_108__+(0-(__iter_2__+2))))]);
        float __temp_112__;
        __temp_112__ = (__temp_107__ + __temp_111__);
        int __temp_113__;
        __temp_113__ = __iter_15__;
        int __temp_114__;
        __temp_114__ = __iter_16__+(1);
        int __temp_115__;
        __temp_115__ = __iter_17__;
        float __temp_116__;
        __temp_116__ = (0.083000f * __tilevar_3__[__temp_115__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_114__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_113__+(0-(__iter_2__+2))))]);
        float __temp_117__;
        __temp_117__ = (__temp_112__ + __temp_116__);
        int __temp_118__;
        __temp_118__ = __iter_15__;
        int __temp_119__;
        __temp_119__ = __iter_16__+(-1);
        int __temp_120__;
        __temp_120__ = __iter_17__;
        float __temp_121__;
        __temp_121__ = (0.083000f * __tilevar_3__[__temp_120__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_119__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_118__+(0-(__iter_2__+2))))]);
        float __temp_122__;
        __temp_122__ = (__temp_117__ + __temp_121__);
        int __temp_123__;
        __temp_123__ = __iter_15__;
        int __temp_124__;
        __temp_124__ = __iter_16__+(-2);
        int __temp_125__;
        __temp_125__ = __iter_17__;
        float __temp_126__;
        __temp_126__ = (0.083000f * __tilevar_3__[__temp_125__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_124__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_123__+(0-(__iter_2__+2))))]);
        float __temp_127__;
        __temp_127__ = (__temp_122__ + __temp_126__);
        int __temp_128__;
        __temp_128__ = __iter_15__+(2);
        int __temp_129__;
        __temp_129__ = __iter_16__;
        int __temp_130__;
        __temp_130__ = __iter_17__;
        float __temp_131__;
        __temp_131__ = (0.083000f * __tilevar_3__[__temp_130__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_129__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_128__+(0-(__iter_2__+2))))]);
        float __temp_132__;
        __temp_132__ = (__temp_127__ + __temp_131__);
        int __temp_133__;
        __temp_133__ = __iter_15__+(1);
        int __temp_134__;
        __temp_134__ = __iter_16__;
        int __temp_135__;
        __temp_135__ = __iter_17__;
        float __temp_136__;
        __temp_136__ = (0.083000f * __tilevar_3__[__temp_135__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_134__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_133__+(0-(__iter_2__+2))))]);
        float __temp_137__;
        __temp_137__ = (__temp_132__ + __temp_136__);
        int __temp_138__;
        __temp_138__ = __iter_15__+(-1);
        int __temp_139__;
        __temp_139__ = __iter_16__;
        int __temp_140__;
        __temp_140__ = __iter_17__;
        float __temp_141__;
        __temp_141__ = (0.083000f * __tilevar_3__[__temp_140__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_139__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_138__+(0-(__iter_2__+2))))]);
        float __temp_142__;
        __temp_142__ = (__temp_137__ + __temp_141__);
        int __temp_143__;
        __temp_143__ = __iter_15__+(-2);
        int __temp_144__;
        __temp_144__ = __iter_16__;
        int __temp_145__;
        __temp_145__ = __iter_17__;
        float __temp_146__;
        __temp_146__ = (0.083000f * __tilevar_3__[__temp_145__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_144__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_143__+(0-(__iter_2__+2))))]);
        float __temp_147__;
        __temp_147__ = (__temp_142__ + __temp_146__);
        int __temp_148__;
        __temp_148__ = __iter_15__;
        int __temp_149__;
        __temp_149__ = __iter_16__;
        int __temp_150__;
        __temp_150__ = __iter_17__;
        float __temp_151__;
        __temp_151__ = (0.996000f * __tilevar_3__[__temp_150__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_149__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__temp_148__+(0-(__iter_2__+2))))]);
        float __temp_152__;
        __temp_152__ = (__temp_147__ - __temp_151__);
        __tilevar_4__[__iter_17__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+4))))] = __temp_152__;
      }
    }
  }
  int __iter_18__;
  __iter_18__ = FORMA_MAX((__iter_2__+4),2) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-3)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_1__+4),2) + (int)(threadIdx.y) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-3)) ){
      int __iter_20__;
      __iter_20__ = FORMA_MAX((__iter_0__+4),2) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-3)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__+4),2)+4) || __iter_18__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-3))-4) || __iter_19__ < (FORMA_MAX((__iter_1__+4),2)+4) || __iter_19__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-3))-4) || __iter_20__ < (FORMA_MAX((__iter_0__+4),2)+4) || __iter_20__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-3))-4)) {
          __copy_arr_1__[__iter_20__+(N-0)*(__iter_19__+(M-0)*(__iter_18__))] = __tilevar_4__[__iter_20__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_19__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_18__+(0-(__iter_2__+4))))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__;
  __iter_24__ = FORMA_MAX((__iter_2__+6),2) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-7),(L-3)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__;
    __iter_25__ = FORMA_MAX((__iter_1__+6),2) + (int)(threadIdx.y) ; 
    if( __iter_25__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-7),(M-3)) ){
      int __iter_26__;
      __iter_26__ = FORMA_MAX((__iter_0__+6),2) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(N-3)) ){
        float __temp_178__;
        __temp_178__ = (0.083000f * __tilevar_4__[__iter_26__+(2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+4))))]);
        float __temp_179__;
        __temp_179__ = (0.083000f * __tilevar_4__[__iter_26__+(1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+4))))]);
        float __temp_180__;
        __temp_180__ = (__temp_178__ + __temp_179__);
        float __temp_181__;
        __temp_181__ = (0.083000f * __tilevar_4__[__iter_26__+(-1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+4))))]);
        float __temp_182__;
        __temp_182__ = (__temp_180__ + __temp_181__);
        float __temp_183__;
        __temp_183__ = (0.083000f * __tilevar_4__[__iter_26__+(-2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+4))))]);
        float __temp_184__;
        __temp_184__ = (__temp_182__ + __temp_183__);
        float __temp_185__;
        __temp_185__ = (0.083000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(2)+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+4))))]);
        float __temp_186__;
        __temp_186__ = (__temp_184__ + __temp_185__);
        float __temp_187__;
        __temp_187__ = (0.083000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(1)+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+4))))]);
        float __temp_188__;
        __temp_188__ = (__temp_186__ + __temp_187__);
        float __temp_189__;
        __temp_189__ = (0.083000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(-1)+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+4))))]);
        float __temp_190__;
        __temp_190__ = (__temp_188__ + __temp_189__);
        float __temp_191__;
        __temp_191__ = (0.083000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(-2)+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+4))))]);
        float __temp_192__;
        __temp_192__ = (__temp_190__ + __temp_191__);
        float __temp_193__;
        __temp_193__ = (0.083000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(2)+(0-(__iter_2__+4))))]);
        float __temp_194__;
        __temp_194__ = (__temp_192__ + __temp_193__);
        float __temp_195__;
        __temp_195__ = (0.083000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(1)+(0-(__iter_2__+4))))]);
        float __temp_196__;
        __temp_196__ = (__temp_194__ + __temp_195__);
        float __temp_197__;
        __temp_197__ = (0.083000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(-1)+(0-(__iter_2__+4))))]);
        float __temp_198__;
        __temp_198__ = (__temp_196__ + __temp_197__);
        float __temp_199__;
        __temp_199__ = (0.083000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(-2)+(0-(__iter_2__+4))))]);
        float __temp_200__;
        __temp_200__ = (__temp_198__ + __temp_199__);
        float __temp_201__;
        __temp_201__ = (0.996000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+4))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+4))))]);
        float __temp_202__;
        __temp_202__ = (__temp_200__ - __temp_201__);
        __tilevar_5__[__iter_26__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+6))))] = __temp_202__;
      }
    }
  }
  int __iter_27__;
  __iter_27__ = FORMA_MAX((__iter_2__+6),2) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-7),(L-3)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__;
    __iter_28__ = FORMA_MAX((__iter_1__+6),2) + (int)(threadIdx.y) ; 
    if( __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-7),(M-3)) ){
      int __iter_29__;
      __iter_29__ = FORMA_MAX((__iter_0__+6),2) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(N-3)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__+6),2)+4) || __iter_27__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-7),(L-3))-4) || __iter_28__ < (FORMA_MAX((__iter_1__+6),2)+4) || __iter_28__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-7),(M-3))-4) || __iter_29__ < (FORMA_MAX((__iter_0__+6),2)+4) || __iter_29__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(N-3))-4)) {
          __copy_arr_2__[__iter_29__+(N-0)*(__iter_28__+(M-0)*(__iter_27__))] = __tilevar_5__[__iter_29__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_28__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_27__+(0-(__iter_2__+6))))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__;
  __iter_33__ = FORMA_MAX((__iter_2__+8),2) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-9),(L-3)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__;
    __iter_34__ = FORMA_MAX((__iter_1__+8),2) + (int)(threadIdx.y) ; 
    if( __iter_34__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-9),(M-3)) ){
      int __iter_35__;
      __iter_35__ = FORMA_MAX((__iter_0__+8),2) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-9),(N-3)) ){
        float __temp_228__;
        __temp_228__ = (0.083000f * __tilevar_5__[__iter_35__+(2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+6))))]);
        float __temp_229__;
        __temp_229__ = (0.083000f * __tilevar_5__[__iter_35__+(1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+6))))]);
        float __temp_230__;
        __temp_230__ = (__temp_228__ + __temp_229__);
        float __temp_231__;
        __temp_231__ = (0.083000f * __tilevar_5__[__iter_35__+(-1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+6))))]);
        float __temp_232__;
        __temp_232__ = (__temp_230__ + __temp_231__);
        float __temp_233__;
        __temp_233__ = (0.083000f * __tilevar_5__[__iter_35__+(-2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+6))))]);
        float __temp_234__;
        __temp_234__ = (__temp_232__ + __temp_233__);
        float __temp_235__;
        __temp_235__ = (0.083000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(2)+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+6))))]);
        float __temp_236__;
        __temp_236__ = (__temp_234__ + __temp_235__);
        float __temp_237__;
        __temp_237__ = (0.083000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(1)+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+6))))]);
        float __temp_238__;
        __temp_238__ = (__temp_236__ + __temp_237__);
        float __temp_239__;
        __temp_239__ = (0.083000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(-1)+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+6))))]);
        float __temp_240__;
        __temp_240__ = (__temp_238__ + __temp_239__);
        float __temp_241__;
        __temp_241__ = (0.083000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(-2)+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+6))))]);
        float __temp_242__;
        __temp_242__ = (__temp_240__ + __temp_241__);
        float __temp_243__;
        __temp_243__ = (0.083000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(2)+(0-(__iter_2__+6))))]);
        float __temp_244__;
        __temp_244__ = (__temp_242__ + __temp_243__);
        float __temp_245__;
        __temp_245__ = (0.083000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(1)+(0-(__iter_2__+6))))]);
        float __temp_246__;
        __temp_246__ = (__temp_244__ + __temp_245__);
        float __temp_247__;
        __temp_247__ = (0.083000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(-1)+(0-(__iter_2__+6))))]);
        float __temp_248__;
        __temp_248__ = (__temp_246__ + __temp_247__);
        float __temp_249__;
        __temp_249__ = (0.083000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(-2)+(0-(__iter_2__+6))))]);
        float __temp_250__;
        __temp_250__ = (__temp_248__ + __temp_249__);
        float __temp_251__;
        __temp_251__ = (0.996000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+6))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+6))))]);
        float __temp_252__;
        __temp_252__ = (__temp_250__ - __temp_251__);
        __var_1__[__iter_35__+(N-0)*(__iter_34__+(M-0)*(__iter_33__))] = __temp_252__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __copy_arr_0__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_12__;
  __iter_12__ = FORMA_MAX(__iter_2__,2) + (int)(threadIdx.z) ; 
  for( ; __iter_12__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-3)) ; __iter_12__ += (int)(blockDim.z) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX(__iter_1__,2) + (int)(threadIdx.y) ; 
    for( ; __iter_13__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-3)) ; __iter_13__ += (int)(blockDim.y) ){
      int __iter_14__;
      __iter_14__ = FORMA_MAX(__iter_0__,2) + (int)(threadIdx.x) ; 
      if( __iter_14__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-3)) ){
        if (__iter_12__ < FORMA_MAX((__iter_2__+2),2) || __iter_12__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-3)) || __iter_13__ < FORMA_MAX((__iter_1__+2),2) || __iter_13__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-3)) || __iter_14__ < FORMA_MAX((__iter_0__+2),2) || __iter_14__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-3))) {
          float __temp_64__;
          __temp_64__ = (0.083000f * input[__iter_14__+(2)+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_65__;
          __temp_65__ = (0.083000f * input[__iter_14__+(1)+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_66__;
          __temp_66__ = (__temp_64__ + __temp_65__);
          float __temp_67__;
          __temp_67__ = (0.083000f * input[__iter_14__+(-1)+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_68__;
          __temp_68__ = (__temp_66__ + __temp_67__);
          float __temp_69__;
          __temp_69__ = (0.083000f * input[__iter_14__+(-2)+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_70__;
          __temp_70__ = (__temp_68__ + __temp_69__);
          float __temp_71__;
          __temp_71__ = (0.083000f * input[__iter_14__+(N-0)*(__iter_13__+(2)+(M-0)*(__iter_12__))]);
          float __temp_72__;
          __temp_72__ = (__temp_70__ + __temp_71__);
          float __temp_73__;
          __temp_73__ = (0.083000f * input[__iter_14__+(N-0)*(__iter_13__+(1)+(M-0)*(__iter_12__))]);
          float __temp_74__;
          __temp_74__ = (__temp_72__ + __temp_73__);
          float __temp_75__;
          __temp_75__ = (0.083000f * input[__iter_14__+(N-0)*(__iter_13__+(-1)+(M-0)*(__iter_12__))]);
          float __temp_76__;
          __temp_76__ = (__temp_74__ + __temp_75__);
          float __temp_77__;
          __temp_77__ = (0.083000f * input[__iter_14__+(N-0)*(__iter_13__+(-2)+(M-0)*(__iter_12__))]);
          float __temp_78__;
          __temp_78__ = (__temp_76__ + __temp_77__);
          float __temp_79__;
          __temp_79__ = (0.083000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__+(2)))]);
          float __temp_80__;
          __temp_80__ = (__temp_78__ + __temp_79__);
          float __temp_81__;
          __temp_81__ = (0.083000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__+(1)))]);
          float __temp_82__;
          __temp_82__ = (__temp_80__ + __temp_81__);
          float __temp_83__;
          __temp_83__ = (0.083000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__+(-1)))]);
          float __temp_84__;
          __temp_84__ = (__temp_82__ + __temp_83__);
          float __temp_85__;
          __temp_85__ = (0.083000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__+(-2)))]);
          float __temp_86__;
          __temp_86__ = (__temp_84__ + __temp_85__);
          float __temp_87__;
          __temp_87__ = (0.996000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_88__;
          __temp_88__ = (__temp_86__ - __temp_87__);
          __copy_arr_0__[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__))] = __temp_88__;
        }
      }
    }
  }
}

__global__ void __kernel___forma_kernel__2__(float * __restrict__ __copy_arr_0__, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __copy_arr_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_21__;
  __iter_21__ = FORMA_MAX(__iter_2__,2) + (int)(threadIdx.z) ; 
  for( ; __iter_21__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-3)) ; __iter_21__ += (int)(blockDim.z) ){
    int __iter_22__;
    __iter_22__ = FORMA_MAX(__iter_1__,2) + (int)(threadIdx.y) ; 
    for( ; __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-3)) ; __iter_22__ += (int)(blockDim.y) ){
      int __iter_23__;
      __iter_23__ = FORMA_MAX(__iter_0__,2) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-3)) ){
        if (__iter_21__ < FORMA_MAX((__iter_2__+4),2) || __iter_21__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-3)) || __iter_22__ < FORMA_MAX((__iter_1__+4),2) || __iter_22__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-3)) || __iter_23__ < FORMA_MAX((__iter_0__+4),2) || __iter_23__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-3))) {
          float __temp_153__;
          __temp_153__ = (0.083000f * __copy_arr_0__[__iter_23__+(2)+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_154__;
          __temp_154__ = (0.083000f * __copy_arr_0__[__iter_23__+(1)+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_155__;
          __temp_155__ = (__temp_153__ + __temp_154__);
          float __temp_156__;
          __temp_156__ = (0.083000f * __copy_arr_0__[__iter_23__+(-1)+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_157__;
          __temp_157__ = (__temp_155__ + __temp_156__);
          float __temp_158__;
          __temp_158__ = (0.083000f * __copy_arr_0__[__iter_23__+(-2)+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_159__;
          __temp_159__ = (__temp_157__ + __temp_158__);
          float __temp_160__;
          __temp_160__ = (0.083000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(2)+(M-0)*(__iter_21__))]);
          float __temp_161__;
          __temp_161__ = (__temp_159__ + __temp_160__);
          float __temp_162__;
          __temp_162__ = (0.083000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(1)+(M-0)*(__iter_21__))]);
          float __temp_163__;
          __temp_163__ = (__temp_161__ + __temp_162__);
          float __temp_164__;
          __temp_164__ = (0.083000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(-1)+(M-0)*(__iter_21__))]);
          float __temp_165__;
          __temp_165__ = (__temp_163__ + __temp_164__);
          float __temp_166__;
          __temp_166__ = (0.083000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(-2)+(M-0)*(__iter_21__))]);
          float __temp_167__;
          __temp_167__ = (__temp_165__ + __temp_166__);
          float __temp_168__;
          __temp_168__ = (0.083000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__+(2)))]);
          float __temp_169__;
          __temp_169__ = (__temp_167__ + __temp_168__);
          float __temp_170__;
          __temp_170__ = (0.083000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__+(1)))]);
          float __temp_171__;
          __temp_171__ = (__temp_169__ + __temp_170__);
          float __temp_172__;
          __temp_172__ = (0.083000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__+(-1)))]);
          float __temp_173__;
          __temp_173__ = (__temp_171__ + __temp_172__);
          float __temp_174__;
          __temp_174__ = (0.083000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__+(-2)))]);
          float __temp_175__;
          __temp_175__ = (__temp_173__ + __temp_174__);
          float __temp_176__;
          __temp_176__ = (0.996000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_177__;
          __temp_177__ = (__temp_175__ - __temp_176__);
          __copy_arr_1__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__))] = __temp_177__;
        }
      }
    }
  }
}

__global__ void __kernel___forma_kernel__3__(float * __restrict__ __copy_arr_1__, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __copy_arr_2__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_30__;
  __iter_30__ = FORMA_MAX(__iter_2__,2) + (int)(threadIdx.z) ; 
  for( ; __iter_30__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-3)) ; __iter_30__ += (int)(blockDim.z) ){
    int __iter_31__;
    __iter_31__ = FORMA_MAX(__iter_1__,2) + (int)(threadIdx.y) ; 
    for( ; __iter_31__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-3)) ; __iter_31__ += (int)(blockDim.y) ){
      int __iter_32__;
      __iter_32__ = FORMA_MAX(__iter_0__,2) + (int)(threadIdx.x) ; 
      if( __iter_32__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-3)) ){
        if (__iter_30__ < FORMA_MAX((__iter_2__+6),2) || __iter_30__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-7),(L-3)) || __iter_31__ < FORMA_MAX((__iter_1__+6),2) || __iter_31__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-7),(M-3)) || __iter_32__ < FORMA_MAX((__iter_0__+6),2) || __iter_32__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(N-3))) {
          float __temp_203__;
          __temp_203__ = (0.083000f * __copy_arr_1__[__iter_32__+(2)+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_204__;
          __temp_204__ = (0.083000f * __copy_arr_1__[__iter_32__+(1)+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_205__;
          __temp_205__ = (__temp_203__ + __temp_204__);
          float __temp_206__;
          __temp_206__ = (0.083000f * __copy_arr_1__[__iter_32__+(-1)+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_207__;
          __temp_207__ = (__temp_205__ + __temp_206__);
          float __temp_208__;
          __temp_208__ = (0.083000f * __copy_arr_1__[__iter_32__+(-2)+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_209__;
          __temp_209__ = (__temp_207__ + __temp_208__);
          float __temp_210__;
          __temp_210__ = (0.083000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(2)+(M-0)*(__iter_30__))]);
          float __temp_211__;
          __temp_211__ = (__temp_209__ + __temp_210__);
          float __temp_212__;
          __temp_212__ = (0.083000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(1)+(M-0)*(__iter_30__))]);
          float __temp_213__;
          __temp_213__ = (__temp_211__ + __temp_212__);
          float __temp_214__;
          __temp_214__ = (0.083000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(-1)+(M-0)*(__iter_30__))]);
          float __temp_215__;
          __temp_215__ = (__temp_213__ + __temp_214__);
          float __temp_216__;
          __temp_216__ = (0.083000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(-2)+(M-0)*(__iter_30__))]);
          float __temp_217__;
          __temp_217__ = (__temp_215__ + __temp_216__);
          float __temp_218__;
          __temp_218__ = (0.083000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__+(2)))]);
          float __temp_219__;
          __temp_219__ = (__temp_217__ + __temp_218__);
          float __temp_220__;
          __temp_220__ = (0.083000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__+(1)))]);
          float __temp_221__;
          __temp_221__ = (__temp_219__ + __temp_220__);
          float __temp_222__;
          __temp_222__ = (0.083000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__+(-1)))]);
          float __temp_223__;
          __temp_223__ = (__temp_221__ + __temp_222__);
          float __temp_224__;
          __temp_224__ = (0.083000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__+(-2)))]);
          float __temp_225__;
          __temp_225__ = (__temp_223__ + __temp_224__);
          float __temp_226__;
          __temp_226__ = (0.996000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_227__;
          __temp_227__ = (__temp_225__ - __temp_226__);
          __copy_arr_2__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__))] = __temp_227__;
        }
      }
    }
  }
}

__global__ void __kernel___forma_kernel__4__(float * __restrict__ __copy_arr_2__, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_36__;
  __iter_36__ = FORMA_MAX(__iter_2__,2) + (int)(threadIdx.z) ; 
  for( ; __iter_36__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-3)) ; __iter_36__ += (int)(blockDim.z) ){
    int __iter_37__;
    __iter_37__ = FORMA_MAX(__iter_1__,2) + (int)(threadIdx.y) ; 
    for( ; __iter_37__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-3)) ; __iter_37__ += (int)(blockDim.y) ){
      int __iter_38__;
      __iter_38__ = FORMA_MAX(__iter_0__,2) + (int)(threadIdx.x) ; 
      if( __iter_38__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-3)) ){
        if (__iter_36__ < FORMA_MAX((__iter_2__+8),2) || __iter_36__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-9),(L-3)) || __iter_37__ < FORMA_MAX((__iter_1__+8),2) || __iter_37__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-9),(M-3)) || __iter_38__ < FORMA_MAX((__iter_0__+8),2) || __iter_38__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-9),(N-3))) {
          float __temp_253__;
          __temp_253__ = (0.083000f * __copy_arr_2__[__iter_38__+(2)+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_254__;
          __temp_254__ = (0.083000f * __copy_arr_2__[__iter_38__+(1)+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_255__;
          __temp_255__ = (__temp_253__ + __temp_254__);
          float __temp_256__;
          __temp_256__ = (0.083000f * __copy_arr_2__[__iter_38__+(-1)+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_257__;
          __temp_257__ = (__temp_255__ + __temp_256__);
          float __temp_258__;
          __temp_258__ = (0.083000f * __copy_arr_2__[__iter_38__+(-2)+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_259__;
          __temp_259__ = (__temp_257__ + __temp_258__);
          float __temp_260__;
          __temp_260__ = (0.083000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(2)+(M-0)*(__iter_36__))]);
          float __temp_261__;
          __temp_261__ = (__temp_259__ + __temp_260__);
          float __temp_262__;
          __temp_262__ = (0.083000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(1)+(M-0)*(__iter_36__))]);
          float __temp_263__;
          __temp_263__ = (__temp_261__ + __temp_262__);
          float __temp_264__;
          __temp_264__ = (0.083000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(-1)+(M-0)*(__iter_36__))]);
          float __temp_265__;
          __temp_265__ = (__temp_263__ + __temp_264__);
          float __temp_266__;
          __temp_266__ = (0.083000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(-2)+(M-0)*(__iter_36__))]);
          float __temp_267__;
          __temp_267__ = (__temp_265__ + __temp_266__);
          float __temp_268__;
          __temp_268__ = (0.083000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__+(2)))]);
          float __temp_269__;
          __temp_269__ = (__temp_267__ + __temp_268__);
          float __temp_270__;
          __temp_270__ = (0.083000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__+(1)))]);
          float __temp_271__;
          __temp_271__ = (__temp_269__ + __temp_270__);
          float __temp_272__;
          __temp_272__ = (0.083000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__+(-1)))]);
          float __temp_273__;
          __temp_273__ = (__temp_271__ + __temp_272__);
          float __temp_274__;
          __temp_274__ = (0.083000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__+(-2)))]);
          float __temp_275__;
          __temp_275__ = (__temp_273__ + __temp_274__);
          float __temp_276__;
          __temp_276__ = (0.996000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_277__;
          __temp_277__ = (__temp_275__ - __temp_276__);
          __var_1__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__))] = __temp_277__;
        }
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d13pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_2___kernel___forma_kernel__0__ = ((L-1) - 0 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),13);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),13);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___2 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_2___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___2,__size_2___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_2),13);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_2___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.z/2 > 13)
      __blockConfig___kernel___forma_kernel__0__.z /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 13)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,13))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y);
  int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.z);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 __blockConfig_st__kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, 2);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig_st__kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  dim3 __blockConfig___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__1__>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __copy_arr_0__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  dim3 __blockConfig___kernel___forma_kernel__2__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__2__>>> (__copy_arr_0__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __copy_arr_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");

  dim3 __blockConfig___kernel___forma_kernel__3__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__3__>>> (__copy_arr_1__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __copy_arr_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  dim3 __blockConfig___kernel___forma_kernel__4__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__4__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__4__>>> (__copy_arr_2__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__4__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

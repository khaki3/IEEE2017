#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"
#include <nvml.h>
#include <assert.h>


#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

	template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
	int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
	input[loc] = value;
}


	template<typename T>
void initialize_array(T* d_input, int size, T value)
{
	dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
	dim3 init_block(FORMA_MAX_BLOCKDIM_0);
	__kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, double * __restrict__ __var_1__){
	__shared__ double tilevar[4][32*32];
	double t[4], b[4];

	int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)-8);
	int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)-8);

	// Initialize the values
	int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
	int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
	if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1))) {
		tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = input[__iter_5__+N*__iter_4__];
		t[0] = input[__iter_5__+N*(__iter_4__+M)];  
	}
	// Rest of the computation
	for (int __iter_2__ = 1; __iter_2__ < L-1; __iter_2__++) {
		if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1))){
			b[0] = tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t[0]; 
			t[0] = input[__iter_5__+N*(__iter_4__+M*(__iter_2__+1))]; 
		}
		__syncthreads ();
		if(__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))) {
			double a0 = t[0] - 2.0f*tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + b[0];
			double a1 = tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)] - 2.0f*tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)];
			double a2 = (0.125f*a0 + 0.125f*a1);
			double a3 = tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] - 2.0f*tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double a4 = (a2 + 0.125f*a3);
			double a5 = a4 + tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]; 
			b[1] = tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]; 
			tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t[1];
			t[1] = a5;
		}
		__syncthreads ();
		if(__iter_4__ >= FORMA_MAX((__iter_1__+2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+2),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))) {
			double a0 = t[1] - 2.0f*tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + b[1];
			double a1 = tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)] - 2.0f*tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)];
			double a2 = (0.125f*a0 + 0.125f*a1);
			double a3 = tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] - 2.0f*tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double a4 = (a2 + 0.125f*a3);
			double a5 = a4 + tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			b[2] = tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t[2];
			t[2] = a5;
		}
		__syncthreads ();
		if(__iter_4__ >= FORMA_MAX((__iter_1__+3),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) &  __iter_5__ >= FORMA_MAX((__iter_0__+3),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))) {
			double a0 = t[2] - 2.0f*tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + b[2];
			double a1 = tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)] - 2.0f*tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)];
			double a2 = (0.125f*a0 + 0.125f*a1);
			double a3 = tilevar[2][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] - 2.0f*tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + tilevar[2][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double a4 = (a2 + 0.125f*a3);
			double a5 = a4 + tilevar[2][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			b[3] = tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t[3];
			t[3] = a5;
		}
		__syncthreads (); 
		if(__iter_4__ >= FORMA_MAX((__iter_1__+4),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+4),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2))) {
			double a0 = t[3] - 2.0f*tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + b[3];
			double a1 = tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)] - 2.0f*tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)];
			double a2 = (0.125f*a0 + 0.125f*a1);
			double a3 = tilevar[3][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] - 2.0f*tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] + tilevar[3][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			double a4 = (a2 + 0.125f*a3);
			double a5 = a4 + tilevar[3][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
			__var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-3,0))] = a5;
		}
	}
}

/*Device code End */
/* Host Code Begin */
extern "C" void host_code(double * h_input, double * __var_0__, int L, int M, int N) {

	/* Host allocation Begin */
	double * input;
	hipMalloc(&input,sizeof(double)*(L*M*N));
	Check_CUDA_Error("Allocation Error!! : input\n");
	hipPointerAttribute_t ptrAttrib_h_input;
	hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
	if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
		if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
			memcpy_kind_h_input = hipMemcpyDeviceToDevice;
	hipGetLastError();
	if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
		hipMemcpy(input,h_input,sizeof(double)*(L*M*N), memcpy_kind_h_input);
	}
	double * __var_1__;
	hipMalloc(&__var_1__,sizeof(double)*(L*M*N));
	Check_CUDA_Error("Allocation Error!! : __var_1__\n");
	/*Host Allocation End */
	/* Kernel Launch Begin */
	int __FORMA_MAX_SHARED_MEM__;
	hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
	hipEvent_t _forma_timer_start_,_forma_timer_stop_;
	hipEventCreate(&_forma_timer_start_);
	hipEventCreate(&_forma_timer_stop_);
	hipEventRecord(_forma_timer_start_,0);
#endif
	int __size_0___kernel___forma_kernel__0__ = N;
	int __size_1___kernel___forma_kernel__0__ = M;
	int __block_0___kernel___forma_kernel__0__ = 32;
	int __block_1___kernel___forma_kernel__0__ = 32;
	int __block_2___kernel___forma_kernel__0__ = 1;
	dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
	int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-8);
	int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y-8);
	int __grid_2___kernel___forma_kernel__0__ = 1;
	dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<500; x++) {
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();

	hipPointerAttribute_t ptrAttrib___var_0__;
	hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
	if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
		if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
			memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
	hipGetLastError();
	hipMemcpy(__var_0__,__var_1__, sizeof(double)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
	hipEventRecord(_forma_timer_stop_,0);
	hipEventSynchronize(_forma_timer_stop_);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
	printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
	hipEventDestroy(_forma_timer_start_);
	hipEventDestroy(_forma_timer_stop_);
#endif
	/*Kernel Launch End */
	/* Host Free Begin */
	hipFree(input);
	hipFree(__var_1__);
}
/*Host Free End*/

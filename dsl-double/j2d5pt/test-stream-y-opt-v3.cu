#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define GAPX (22) 
#define EXTENT (5)
#define mod(x,y) ( (x) & (y-1))

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float * __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+16)*FORMA_BLOCKDIM_X);
  float * __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+16)*FORMA_BLOCKDIM_X);
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int rowy = FORMA_BLOCKDIM_Y+16;
  //int threadIdx_y = mod((int)threadIdx.y,2);

  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X + GAPX);
  for (int __iter_1__ = 0; __iter_1__ <= N-1; __iter_1__ += FORMA_BLOCKDIM_Y) {
    int __iter_2__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
    int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
    if(__iter_2__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
      __tilevar_2__[__iter_3__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_2__,rowy)] = input[__iter_3__+M*__iter_2__];
    }
    __syncthreads();
    int __iter_4__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        float __temp_2__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_4__-1),rowy)]);
        float __temp_5__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_6__ = (5 * __temp_2__ + 12 * __temp_5__);
        float __temp_9__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_10__ = (__temp_6__ + 15 * __temp_9__);
        float __temp_13__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
        float __temp_17__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_4__+1),rowy)]);
        float __temp_18__ = (__temp_14__ + 5 * __temp_17__);
        float __temp_19__ = (__temp_18__ / 118);
        __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)] = __temp_19__;
      }
    }
    int __iter_6__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        if (__iter_7__ < (FORMA_MAX((__iter_0__+1),1)+2) | __iter_7__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-2)) {
	  __copy_arr_0__[__iter_6__+(M)*(__iter_7__)] = __tilevar_3__[__iter_7__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_6__,rowy)];
        }
      }
    }
    __syncthreads();
    int __iter_10__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
      int __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        float __temp_32__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_10__-1),rowy)]);
        float __temp_35__ = (__tilevar_3__[__iter_11__+(-1)-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_36__ = (5 * __temp_32__ + 12 * __temp_35__);
        float __temp_39__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_40__ = (__temp_36__ + 15 * __temp_39__);
        float __temp_43__ = (__tilevar_3__[__iter_11__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_44__ = (__temp_40__ + 12 * __temp_43__);
        float __temp_47__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_10__+1),rowy)]);
        float __temp_48__ = (__temp_44__ + 5 * __temp_47__);
        float __temp_49__ = (__temp_48__ / 118);
        __tilevar_4__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)] = __temp_49__;
      }
    }
    int __iter_12__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
      int __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        if (__iter_13__ < (FORMA_MAX((__iter_0__+2),1)+2) | __iter_13__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-2)) {
	  __copy_arr_1__[__iter_12__+(M)*(__iter_13__)] = __tilevar_4__[__iter_13__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_12__,rowy)]; 
        }
      }
    }
    __syncthreads();
    int __iter_16__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
      int __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        float __temp_60__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_16__-1),rowy)]);
        float __temp_61__ = (__tilevar_4__[__iter_17__+(-1)-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_62__ = (5 * __temp_60__ + 12 * __temp_61__);
        float __temp_63__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_64__ = (__temp_62__ + 15 * __temp_63__);
        float __temp_65__ = (__tilevar_4__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_66__ = (__temp_64__ + 12 * __temp_65__);
        float __temp_67__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_16__+1),rowy)]);
        float __temp_68__ = (__temp_66__ + 5 * __temp_67__);
        float __temp_69__ = (__temp_68__ / 118);
        __tilevar_5__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)] = __temp_69__;
      }
    }
    int __iter_18__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
      int __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        if (__iter_19__ < (FORMA_MAX((__iter_0__+3),1)+2) | __iter_19__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-2)) {
	  __copy_arr_2__[__iter_18__+(M)*(__iter_19__)] = __tilevar_5__[__iter_19__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_18__,rowy)];
        }
      }
    }
    __syncthreads();
    int __iter_22__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
    if( __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) ){
      int __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
        float __temp_80__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__-1),rowy)]);
        float __temp_81__ = (__tilevar_5__[__iter_23__+(-1)-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_82__ = (5 * __temp_80__ + 12 * __temp_81__);
        float __temp_83__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_84__ = (__temp_82__ + 15 * __temp_83__);
        float __temp_85__ = (__tilevar_5__[__iter_23__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_86__ = (__temp_84__ + 12 * __temp_85__);
        float __temp_87__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__+1),rowy)]);
        float __temp_88__ = (__temp_86__ + 5 * __temp_87__);
        float __temp_89__ = (__temp_88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__)] = __temp_89__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(2*(FORMA_BLOCKDIM_Y+16)*FORMA_BLOCKDIM_X);
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float * __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+16)*FORMA_BLOCKDIM_X);
  float * __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+16)*FORMA_BLOCKDIM_X);
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int rowy = FORMA_BLOCKDIM_Y+16;
  //int threadIdx_y = mod(threadIdx.y,2);
 
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X + GAPX) + (int)FORMA_BLOCKDIM_X;
  for (int __iter_1__ = 0; __iter_1__ <= N-1; __iter_1__ += FORMA_BLOCKDIM_Y) {
    int __iter_2__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
    int __iter_3__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
    if( __iter_2__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1))){
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_2__,rowy)] = input[__iter_3__+(M)*(__iter_2__)];
    }
    __syncthreads();
    int __iter_4__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_5__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
        float __temp_2__ = (5 * __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_4__-1),rowy)]);
        float __temp_5__ = (12 * __tilevar_2__[__iter_5__+(-1)+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_6__ = (__temp_2__ + __temp_5__);
        float __temp_9__ = (15 * __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_10__ = (__temp_6__ + __temp_9__);
        float __temp_13__ = (12 * __tilevar_2__[__iter_5__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_14__ = (__temp_10__ + __temp_13__);
        float __temp_17__ = (5 * __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_4__+1),rowy)]);
        float __temp_18__ = (__temp_14__ + __temp_17__);
        float __temp_19__ = (__temp_18__ / 118);
        __tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)] = __temp_19__;
      }
    }
    int __iter_6__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
    if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_7__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
        if (__iter_7__ < FORMA_MAX((__iter_0__-1),1) | __iter_7__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2))) {
	  __tilevar_3__[__iter_7__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_6__,rowy)] = __copy_arr_0__[__iter_6__+(M)*(__iter_7__)];
        }
      }
    }
    __syncthreads();
    int __iter_10__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
      int __iter_11__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
        float __temp_32__ = (5 * __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_10__-1),rowy)]);
        float __temp_35__ = (12 * __tilevar_3__[__iter_11__+(-1)+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_36__ = (__temp_32__ + __temp_35__);
        float __temp_39__ = (15 * __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_40__ = (__temp_36__ + __temp_39__);
        float __temp_43__ = (12 * __tilevar_3__[__iter_11__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_44__ = (__temp_40__ + __temp_43__);
        float __temp_47__ = (5 * __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_10__+1),rowy)]);
        float __temp_48__ = (__temp_44__ + __temp_47__);
        float __temp_49__ = (__temp_48__ / 118);
        __tilevar_4__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)] = __temp_49__;
      }
    }
    int __iter_12__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
    if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
      int __iter_13__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
        if (__iter_13__ < FORMA_MAX((__iter_0__-2),1) | __iter_13__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ) {
	  __tilevar_4__[__iter_13__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_12__,rowy)] = __copy_arr_1__[__iter_12__+(M)*(__iter_13__)];
        }
      }
    }
    __syncthreads();
    int __iter_16__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
      int __iter_17__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
        float __temp_60__ = (5 * __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_16__-1),rowy)]);
        float __temp_61__ = (12 * __tilevar_4__[__iter_17__+(-1)+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_62__ = (__temp_60__ + __temp_61__);
        float __temp_63__ = (15 * __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_64__ = (__temp_62__ + __temp_63__);
        float __temp_65__ = (12 * __tilevar_4__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_66__ = (__temp_64__ + __temp_65__);
        float __temp_67__ = (5 * __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_16__+1),rowy)]);
        float __temp_68__ = (__temp_66__ + __temp_67__);
        float __temp_69__ = (__temp_68__ / 118);
        __tilevar_5__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)] = __temp_69__;
      }
    }
    int  __iter_18__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
    if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
      int __iter_19__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(M-2)) ){
        if (__iter_19__ < FORMA_MAX((__iter_0__-3),1) | __iter_19__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2))) {
	  __tilevar_5__[__iter_19__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_18__,rowy)] = __copy_arr_2__[__iter_18__+(M)*(__iter_19__)];
        }
      }
    }
    __syncthreads();
    int __iter_22__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
    if( __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) ){
      int __iter_23__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
        float __temp_80__ = (5 * __tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_22__-1),rowy)]);
        float __temp_81__ = (12 * __tilevar_5__[__iter_23__+(-1)+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_82__ = (__temp_80__ + __temp_81__);
        float __temp_83__ = (15 * __tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_84__ = (__temp_82__ + __temp_83__);
        float __temp_85__ = (12 * __tilevar_5__[__iter_23__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_86__ = (__temp_84__ + __temp_85__);
        float __temp_87__ = (5 * __tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_22__+1),rowy)]);
        float __temp_88__ = (__temp_86__ + __temp_87__);
        float __temp_89__ = (__temp_88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__)] = __temp_89__;
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N)*(M)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 16;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

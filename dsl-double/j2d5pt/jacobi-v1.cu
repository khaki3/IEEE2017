#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__,0) + 4*(int)(threadIdx.y) ; 
  if( __iter_2__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(0-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(1-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+1)];
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(2-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+2)];
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(3-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+3)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((__iter_1__+1),1) + 4*(int)(threadIdx.y) ;
  if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      // iter 0 : __iter_4__ 
      float __temp_a2__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_a5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a6__ = (5*__temp_a2__ + 12*__temp_a5__);
      float __temp_a9__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a10__ = (__temp_a6__ + 15*__temp_a9__);
      float __temp_a13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a14__ = (__temp_a10__ + 12*__temp_a13__);
      float __temp_a17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_a18__ = (__temp_a14__ + 5*__temp_a17__);
      float __temp_a19__ = (__temp_a18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+1)))] = __temp_a19__;
      // iter 1 : __iter_4__ + 1
      float __temp_b5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b6__ = (5*__temp_a9__+ 12*__temp_b5__);
      float __temp_b10__ = (__temp_b6__ + 15*__temp_a17__);
      float __temp_b13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b14__ = (__temp_b10__ + 12*__temp_b13__);
      float __temp_b17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_b18__ = (__temp_b14__ + 5*__temp_b17__);
      float __temp_b19__ = (__temp_b18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1-(__iter_1__+1)))] = __temp_b19__;
      // iter 2 : __iter_4__ + 2
      float __temp_c5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c6__ = (5*__temp_a17__+ 12*__temp_c5__);
      float __temp_c10__ = (__temp_c6__ + 15*__temp_b17__);
      float __temp_c13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c14__ = (__temp_c10__ + 12*__temp_c13__);
      float __temp_c17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(0-(__iter_1__+0)))]);
      float __temp_c18__ = (__temp_c14__ + 5*__temp_c17__);
      float __temp_c19__ = (__temp_c18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2-(__iter_1__+1)))] = __temp_c19__;
      // iter 3 : __iter_4__ + 3
      float __temp_d5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d6__ = (5*__temp_b17__+ 12*__temp_d5__);
      float __temp_d10__ = (__temp_d6__ + 15*__temp_c17__);
      float __temp_d13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d14__ = (__temp_d10__ + 12*__temp_d13__);
      float __temp_d17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(4)+(0-(__iter_1__+0)))]);
      float __temp_d18__ = (__temp_d14__ + 5*__temp_d17__);
      float __temp_d19__ = (__temp_d18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3-(__iter_1__+1)))] = __temp_d19__;
    }
  }
  __syncthreads ();
  int __iter_6__;
  __iter_6__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
  for(; __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ; __iter_6__ += (int)(blockDim.y)){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      if (__iter_6__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_6__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))-2) || __iter_7__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_7__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-2)) {
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+1)))];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__+2),1) + 4*(int)(threadIdx.y) ;
  if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      // iter 1 : __iter_10__ 
      float __temp_a32__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(-1)+(0-(__iter_1__+1)))]);
      float __temp_a35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+1)))]);
      float __temp_a36__ = (5*__temp_a32__ + 12*__temp_a35__);
      float __temp_a39__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+1)))]);
      float __temp_a40__ = (__temp_a36__ + 15*__temp_a39__);
      float __temp_a43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+1)))]);
      float __temp_a44__ = (__temp_a40__ + 12*__temp_a43__);
      float __temp_a47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(0-(__iter_1__+1)))]);
      float __temp_a48__ = (__temp_a44__ + 5*__temp_a47__);
      float __temp_a49__ = (__temp_a48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+2)))] = __temp_a49__;
      // iter 2 : __iter_10__ + 1
      float __temp_b35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(0-(__iter_1__+1)))]);
      float __temp_b36__ = (5*__temp_a39__+ 12*__temp_b35__);
      float __temp_b40__ = (__temp_b36__ + 15*__temp_a47__);
      float __temp_b43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(0-(__iter_1__+1)))]);
      float __temp_b44__ = (__temp_b40__ + 12*__temp_b43__);
      float __temp_b47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(0-(__iter_1__+1)))]);
      float __temp_b48__ = (__temp_b44__ + 5*__temp_b47__);
      float __temp_b49__ = (__temp_b48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1-(__iter_1__+2)))] = __temp_b49__;
      // iter 3 : __iter_10__ + 2
      float __temp_c35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(0-(__iter_1__+1)))]);
      float __temp_c36__ = (5*__temp_a47__+ 12*__temp_c35__);
      float __temp_c40__ = (__temp_c36__ + 15*__temp_b47__);
      float __temp_c43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(0-(__iter_1__+1)))]);
      float __temp_c44__ = (__temp_c40__ + 12*__temp_c43__);
      float __temp_c47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(0-(__iter_1__+1)))]);
      float __temp_c48__ = (__temp_c44__ + 5*__temp_c47__);
      float __temp_c49__ = (__temp_c48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2-(__iter_1__+2)))] = __temp_c49__;
      // iter 4 : __iter_10__ + 3
      float __temp_d35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(0-(__iter_1__+1)))]);
      float __temp_d36__ = (5*__temp_b47__+ 12*__temp_d35__);
      float __temp_d40__ = (__temp_d36__ + 15*__temp_c47__);
      float __temp_d43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(0-(__iter_1__+1)))]);
      float __temp_d44__ = (__temp_d40__ + 12*__temp_d43__);
      float __temp_d47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(4)+(0-(__iter_1__+1)))]);
      float __temp_d48__ = (__temp_d44__ + 5*__temp_d47__);
      float __temp_d49__ = (__temp_d48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3-(__iter_1__+2)))] = __temp_d49__;
    }
  }
  __syncthreads ();
  int __iter_12__;
  __iter_12__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
  for(; __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ; __iter_12__ += (int)(blockDim.y)){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      if (__iter_12__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_12__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))-2) || __iter_13__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_13__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-2)) {
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+2)))];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__+3),1) + 4*(int)(threadIdx.y) ;
  if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      // iter 1 : __iter_16__ 
      float __temp_a60__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+2)))]);
      float __temp_a61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_a62__ = (5*__temp_a60__ + 12*__temp_a61__);
      float __temp_a63__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_a64__ = (__temp_a62__ + 15*__temp_a63__);
      float __temp_a65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_a66__ = (__temp_a64__ + 12*__temp_a65__);
      float __temp_a67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_a68__ = (__temp_a66__ + 5*__temp_a67__);
      float __temp_a69__ = (__temp_a68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+3)))] = __temp_a69__;
      // iter 2 : __iter_16__ + 1
      float __temp_b61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_b62__ = (5*__temp_a63__+ 12*__temp_b61__);
      float __temp_b64__ = (__temp_b62__ + 15*__temp_a67__);
      float __temp_b65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_b66__ = (__temp_b64__ + 12*__temp_b65__);
      float __temp_b67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+2)))]);
      float __temp_b68__ = (__temp_b66__ + 5*__temp_b67__);
      float __temp_b69__ = (__temp_b68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1-(__iter_1__+3)))] = __temp_b69__;
      // iter 3 : __iter_16__ + 2
      float __temp_c61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+2)))]);
      float __temp_c62__ = (5*__temp_a67__+ 12*__temp_c61__);
      float __temp_c64__ = (__temp_c62__ + 15*__temp_b67__);
      float __temp_c65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+2)))]);
      float __temp_c66__ = (__temp_c64__ + 12*__temp_c65__);
      float __temp_c67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(0-(__iter_1__+2)))]);
      float __temp_c68__ = (__temp_c66__ + 5*__temp_c67__);
      float __temp_c69__ = (__temp_c68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2-(__iter_1__+3)))] = __temp_c69__;
      // iter 4 : __iter_16__ + 3
      float __temp_d61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(0-(__iter_1__+2)))]);
      float __temp_d62__ = (5*__temp_b67__+ 12*__temp_d61__);
      float __temp_d64__ = (__temp_d62__ + 15*__temp_c67__);
      float __temp_d65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(0-(__iter_1__+2)))]);
      float __temp_d66__ = (__temp_d64__ + 12*__temp_d65__);
      float __temp_d67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(4)+(0-(__iter_1__+2)))]);
      float __temp_d68__ = (__temp_d66__ + 5*__temp_d67__);
      float __temp_d69__ = (__temp_d68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3-(__iter_1__+3)))] = __temp_d69__;
    }
  }
  __syncthreads ();
  int __iter_18__;
  __iter_18__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
  for(; __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ; __iter_18__ += (int)(blockDim.y)){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      if (__iter_18__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))-2) || __iter_19__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-2)) {
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+3)))];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__+4),1) + 4*(int)(threadIdx.y) ; 
  if( __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
      // iter 0 : __iter_22__
      float __temp_a80__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+3)))]);
      float __temp_a81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_a82__ = (5*__temp_a80__ + 12*__temp_a81__);
      float __temp_a83__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_a84__ = (__temp_a82__ + 15*__temp_a83__);
      float __temp_a85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_a86__ = (__temp_a84__ + 12*__temp_a85__);
      float __temp_a87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+3)))]);
      float __temp_a88__ = (__temp_a86__ + 5*__temp_a87__);
      float __temp_a89__ = (__temp_a88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_a89__;
      // iter 1 : __iter_22__ + 1
      float __temp_b81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+3)))]);
      float __temp_b82__ = (5*__temp_a83__+ 12*__temp_b81__);
      float __temp_b84__ = (__temp_b82__ + 15*__temp_a87__);
      float __temp_b85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+3)))]);
      float __temp_b86__ = (__temp_b84__ + 12*__temp_b85__);
      float __temp_b87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+3)))]);
      float __temp_b88__ = (__temp_b86__ + 5*__temp_b87__);
      float __temp_b89__ = (__temp_b88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+1)] = __temp_b89__;
      // iter 2 : __iter_22__ + 2
      float __temp_c81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+3)))]);
      float __temp_c82__ = (5*__temp_a87__+ 12*__temp_c81__);
      float __temp_c84__ = (__temp_c82__ + 15*__temp_b87__);
      float __temp_c85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+3)))]);
      float __temp_c86__ = (__temp_c84__ + 12*__temp_c85__);
      float __temp_c87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(0-(__iter_1__+3)))]);
      float __temp_c88__ = (__temp_c86__ + 5*__temp_c87__);
      float __temp_c89__ = (__temp_c88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+2)] = __temp_c89__;
      // iter 3 : __iter_22__ + 3
      float __temp_d81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(0-(__iter_1__+3)))]);
      float __temp_d82__ = (5*__temp_b87__+ 12*__temp_d81__);
      float __temp_d84__ = (__temp_d82__ + 15*__temp_c87__);
      float __temp_d85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(0-(__iter_1__+3)))]);
      float __temp_d86__ = (__temp_d84__ + 12*__temp_d85__);
      float __temp_d87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(4)+(0-(__iter_1__+3)))]);
      float __temp_d88__ = (__temp_d86__ + 5*__temp_d87__);
      float __temp_d89__ = (__temp_d88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+3)] = __temp_d89__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __copy_arr_0__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  if (threadIdx.y == 0) {
    int __iter_8__ = FORMA_MAX(__iter_1__,1);
    int __iter_9__;
    __iter_9__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_9__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      float __temp_20__ = (input[__iter_9__+(M-0)*(__iter_8__+(-1))]);
      float __temp_21__ = (input[__iter_9__+(-1)+(M-0)*(__iter_8__)]);
      float __temp_22__ = (5 * __temp_20__ + 12 * __temp_21__);
      float __temp_23__ = (input[__iter_9__+(M-0)*(__iter_8__)]);
      float __temp_24__ = (__temp_22__ + 15 * __temp_23__);
      float __temp_25__ = (input[__iter_9__+(1)+(M-0)*(__iter_8__)]);
      float __temp_26__ = (__temp_24__ + 12 * __temp_25__);
      float __temp_27__ = (input[__iter_9__+(M-0)*(__iter_8__+(1))]);
      float __temp_28__ = (__temp_26__ + 5  * __temp_27__);
      float __temp_29__ = (__temp_28__ / 118);
      __copy_arr_0__[__iter_9__+(M-0)*(__iter_8__)] = __temp_29__;
    }
  }
  if (threadIdx.y == 1) {
    int __iter_8__ = FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2));
    int __iter_9__;
    __iter_9__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_9__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
       float __temp_20__ = (input[__iter_9__+(M-0)*(__iter_8__+(-1))]);
       float __temp_21__ = (input[__iter_9__+(-1)+(M-0)*(__iter_8__)]);
       float __temp_22__ = (5 * __temp_20__ + 12 * __temp_21__);
       float __temp_23__ = (input[__iter_9__+(M-0)*(__iter_8__)]);
       float __temp_24__ = (__temp_22__ + 15 * __temp_23__);
       float __temp_25__ = (input[__iter_9__+(1)+(M-0)*(__iter_8__)]);
       float __temp_26__ = (__temp_24__ + 12 * __temp_25__);
       float __temp_27__ = (input[__iter_9__+(M-0)*(__iter_8__+(1))]);
       float __temp_28__ = (__temp_26__ + 5  * __temp_27__);
       float __temp_29__ = (__temp_28__ / 118);
       __copy_arr_0__[__iter_9__+(M-0)*(__iter_8__)] = __temp_29__;
    }
  }
  if (threadIdx.y == 2) {
    int __iter_8__ = FORMA_MAX(__iter_1__+1,1) + (int)(threadIdx.x) ; 
    if(__iter_8__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_9__;
      __iter_9__ = FORMA_MAX(__iter_0__,1);
      if( __iter_9__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
        float __temp_20__ = (input[__iter_9__+(M-0)*(__iter_8__+(-1))]);
        float __temp_21__ = (input[__iter_9__+(-1)+(M-0)*(__iter_8__)]);
        float __temp_22__ = (5 * __temp_20__ + 12 * __temp_21__);
        float __temp_23__ = (input[__iter_9__+(M-0)*(__iter_8__)]);
        float __temp_24__ = (__temp_22__ + 15 * __temp_23__);
        float __temp_25__ = (input[__iter_9__+(1)+(M-0)*(__iter_8__)]);
        float __temp_26__ = (__temp_24__ + 12 * __temp_25__);
        float __temp_27__ = (input[__iter_9__+(M-0)*(__iter_8__+(1))]);
        float __temp_28__ = (__temp_26__ + 5  * __temp_27__);
        float __temp_29__ = (__temp_28__ / 118);
        __copy_arr_0__[__iter_9__+(M-0)*(__iter_8__)] = __temp_29__;
      }
    }
  }
  if (threadIdx.y == 3) {
    int __iter_8__ = FORMA_MAX(__iter_1__+1,1) + (int)(threadIdx.x);
    if(__iter_8__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_9__;
      __iter_9__ = FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2));
      if( __iter_9__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
        float __temp_20__ = (input[__iter_9__+(M-0)*(__iter_8__+(-1))]);
        float __temp_21__ = (input[__iter_9__+(-1)+(M-0)*(__iter_8__)]);
        float __temp_22__ = (5 * __temp_20__ + 12 * __temp_21__);
        float __temp_23__ = (input[__iter_9__+(M-0)*(__iter_8__)]);
        float __temp_24__ = (__temp_22__ + 15 * __temp_23__);
        float __temp_25__ = (input[__iter_9__+(1)+(M-0)*(__iter_8__)]);
        float __temp_26__ = (__temp_24__ + 12 * __temp_25__);
        float __temp_27__ = (input[__iter_9__+(M-0)*(__iter_8__+(1))]);
        float __temp_28__ = (__temp_26__ + 5  * __temp_27__);
        float __temp_29__ = (__temp_28__ / 118);
        __copy_arr_0__[__iter_9__+(M-0)*(__iter_8__)] = __temp_29__;
      }
    }
  }
}

__global__ void __kernel___forma_kernel__2__(float * __restrict__ __copy_arr_0__, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __copy_arr_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  if (threadIdx.y == 0) {
    int __iter_14__ = FORMA_MAX(__iter_1__,1);
    for( ; __iter_14__ <= FORMA_MAX((__iter_1__+1),1); __iter_14__++) {
      int __iter_15__;
      __iter_15__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_15__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
          float __temp_50__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(-1))]);
          float __temp_51__ = (__copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__)]);
          float __temp_52__ = (5 * __temp_50__ + 12 * __temp_51__);
          float __temp_53__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__)]);
          float __temp_54__ = (__temp_52__ + 15 * __temp_53__);
          float __temp_55__ = (__copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__)]);
          float __temp_56__ = (__temp_54__ + 12 * __temp_55__);
          float __temp_57__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(1))]);
          float __temp_58__ = (__temp_56__ + 5 * __temp_57__);
          float __temp_59__ = (__temp_58__ / 118);
          __copy_arr_1__[__iter_15__+(M-0)*(__iter_14__)] = __temp_59__;
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_14__ =  FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)); 
    for( ; __iter_14__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2)) ; __iter_14__++) {
      int __iter_15__;
      __iter_15__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_15__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
          float __temp_50__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(-1))]);
          float __temp_51__ = (__copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__)]);
          float __temp_52__ = (5 * __temp_50__ + 12 * __temp_51__);
          float __temp_53__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__)]);
          float __temp_54__ = (__temp_52__ + 15 * __temp_53__);
          float __temp_55__ = (__copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__)]);
          float __temp_56__ = (__temp_54__ + 12 * __temp_55__);
          float __temp_57__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(1))]);
          float __temp_58__ = (__temp_56__ + 5 * __temp_57__);
          float __temp_59__ = (__temp_58__ / 118);
          __copy_arr_1__[__iter_15__+(M-0)*(__iter_14__)] = __temp_59__;
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_14__ = FORMA_MAX(__iter_1__+1,1) + (int)(threadIdx.x) ; 
    if(__iter_14__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_15__;
      __iter_15__ = FORMA_MAX(__iter_0__,1);
      for(; __iter_15__ <= FORMA_MAX((__iter_0__+1),1); __iter_15__++) {
          float __temp_50__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(-1))]);
          float __temp_51__ = (__copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__)]);
          float __temp_52__ = (5 * __temp_50__ + 12 * __temp_51__);
          float __temp_53__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__)]);
          float __temp_54__ = (__temp_52__ + 15 * __temp_53__);
          float __temp_55__ = (__copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__)]);
          float __temp_56__ = (__temp_54__ + 12 * __temp_55__);
          float __temp_57__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(1))]);
          float __temp_58__ = (__temp_56__ + 5 * __temp_57__);
          float __temp_59__ = (__temp_58__ / 118);
          __copy_arr_1__[__iter_15__+(M-0)*(__iter_14__)] = __temp_59__;
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_14__ = FORMA_MAX(__iter_1__+1,1) + (int)(threadIdx.x) ; 
    if(__iter_14__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_15__;
      __iter_15__ = FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2));
      for(; __iter_15__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)); __iter_15__++ ){
          float __temp_50__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(-1))]);
          float __temp_51__ = (__copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__)]);
          float __temp_52__ = (5 * __temp_50__ + 12 * __temp_51__);
          float __temp_53__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__)]);
          float __temp_54__ = (__temp_52__ + 15 * __temp_53__);
          float __temp_55__ = (__copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__)]);
          float __temp_56__ = (__temp_54__ + 12 * __temp_55__);
          float __temp_57__ = (__copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(1))]);
          float __temp_58__ = (__temp_56__ + 5 * __temp_57__);
          float __temp_59__ = (__temp_58__ / 118);
          __copy_arr_1__[__iter_15__+(M-0)*(__iter_14__)] = __temp_59__;
      }
    }
  }
}

__global__ void __kernel___forma_kernel__3__(float * __restrict__ __copy_arr_1__, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __copy_arr_2__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  if (threadIdx.y == 0) {
     int __iter_20__ = FORMA_MAX(__iter_1__,1);
     for( ; __iter_20__ <= FORMA_MAX((__iter_1__+2),1) ; __iter_20__++) {
       int __iter_21__;
       __iter_21__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
       if( __iter_21__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
           float __temp_70__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(-1))]);
           float __temp_71__ = (__copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__)]);
           float __temp_72__ = (5 * __temp_70__ + 12 * __temp_71__);
           float __temp_73__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__)]);
           float __temp_74__ = (__temp_72__ + 15 * __temp_73__);
           float __temp_75__ = (__copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__)]);
           float __temp_76__ = (__temp_74__ + 12 * __temp_75__);
           float __temp_77__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(1))]);
           float __temp_78__ = (__temp_76__ + 5 * __temp_77__);
           float __temp_79__ = (__temp_78__ / 118);
           __copy_arr_2__[__iter_21__+(M-0)*(__iter_20__)] = __temp_79__;
       }
     }
  }
  else if (threadIdx.y == 1) {
     int __iter_20__ =  FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2));
     for( ; __iter_20__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2)) ; __iter_20__++) {
       int __iter_21__;
       __iter_21__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
       if( __iter_21__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
           float __temp_70__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(-1))]);
           float __temp_71__ = (__copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__)]);
           float __temp_72__ = (5 * __temp_70__ + 12 * __temp_71__);
           float __temp_73__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__)]);
           float __temp_74__ = (__temp_72__ + 15 * __temp_73__);
           float __temp_75__ = (__copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__)]);
           float __temp_76__ = (__temp_74__ + 12 * __temp_75__);
           float __temp_77__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(1))]);
           float __temp_78__ = (__temp_76__ + 5 * __temp_77__);
           float __temp_79__ = (__temp_78__ / 118);
           __copy_arr_2__[__iter_21__+(M-0)*(__iter_20__)] = __temp_79__;
       }
     }
  }
  else if (threadIdx.y == 2) {
     int __iter_20__ = FORMA_MAX(__iter_1__+3,1) + (int)(threadIdx.x) ; 
     if(__iter_20__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))) {
       int __iter_21__;
       __iter_21__ = FORMA_MAX(__iter_0__,1);
       for(; __iter_21__ <= FORMA_MAX((__iter_0__+2),1); __iter_21__++) {
           float __temp_70__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(-1))]);
           float __temp_71__ = (__copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__)]);
           float __temp_72__ = (5 * __temp_70__ + 12 * __temp_71__);
           float __temp_73__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__)]);
           float __temp_74__ = (__temp_72__ + 15 * __temp_73__);
           float __temp_75__ = (__copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__)]);
           float __temp_76__ = (__temp_74__ + 12 * __temp_75__);
           float __temp_77__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(1))]);
           float __temp_78__ = (__temp_76__ + 5 * __temp_77__);
           float __temp_79__ = (__temp_78__ / 118);
           __copy_arr_2__[__iter_21__+(M-0)*(__iter_20__)] = __temp_79__;
       }
     }
  }
  else if (threadIdx.y == 3) {
     int __iter_20__ = FORMA_MAX(__iter_1__+3,1) + (int)(threadIdx.x) ; 
     if(__iter_20__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))) {
       int __iter_21__;
       __iter_21__ = FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2));
       for(; __iter_21__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)); __iter_21__++){
           float __temp_70__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(-1))]);
           float __temp_71__ = (__copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__)]);
           float __temp_72__ = (5 * __temp_70__ + 12 * __temp_71__);
           float __temp_73__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__)]);
           float __temp_74__ = (__temp_72__ + 15 * __temp_73__);
           float __temp_75__ = (__copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__)]);
           float __temp_76__ = (__temp_74__ + 12 * __temp_75__);
           float __temp_77__ = (__copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(1))]);
           float __temp_78__ = (__temp_76__ + 5 * __temp_77__);
           float __temp_79__ = (__temp_78__ / 118);
           __copy_arr_2__[__iter_21__+(M-0)*(__iter_20__)] = __temp_79__;
       }
     }
  }
}

__global__ void __kernel___forma_kernel__4__(float * __restrict__ __copy_arr_2__, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  if (threadIdx.y == 0) {
    int __iter_24__ = FORMA_MAX(__iter_1__,1);
    for( ; __iter_24__ <= FORMA_MAX((__iter_1__+3),1) ; __iter_24__ ++) {
      int __iter_25__;
      __iter_25__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_25__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
          float __temp_90__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(-1))]);
          float __temp_91__  = (__copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__)]);
          float __temp_92__ = (5 * __temp_90__ + 12 * __temp_91__);
          float __temp_93__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__)]);
          float __temp_94__ = (__temp_92__ + 15 * __temp_93__);
          float __temp_95__ = (__copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__)]);
          float __temp_96__ = (__temp_94__ + 12 * __temp_95__);
          float __temp_97__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(1))]);
          float __temp_98__ = (__temp_96__ + 5 * __temp_97__);
          float __temp_99__ = (__temp_98__ / 118);
          __var_1__[__iter_25__+(M-0)*(__iter_24__)] = __temp_99__;
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_24__ = FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2));
    for( ; __iter_24__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2)) ; __iter_24__ ++) {
      int __iter_25__;
      __iter_25__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_25__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
          float __temp_90__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(-1))]);
          float __temp_91__  = (__copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__)]);
          float __temp_92__ = (5 * __temp_90__ + 12 * __temp_91__);
          float __temp_93__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__)]);
          float __temp_94__ = (__temp_92__ + 15 * __temp_93__);
          float __temp_95__ = (__copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__)]);
          float __temp_96__ = (__temp_94__ + 12 * __temp_95__);
          float __temp_97__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(1))]);
          float __temp_98__ = (__temp_96__ + 5 * __temp_97__);
          float __temp_99__ = (__temp_98__ / 118);
          __var_1__[__iter_25__+(M-0)*(__iter_24__)] = __temp_99__;
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_24__ = FORMA_MAX(__iter_1__+4,1) + (int)(threadIdx.x) ; 
    if(__iter_24__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))) {
      int __iter_25__;
      __iter_25__ = FORMA_MAX(__iter_0__,1);
      for(; __iter_25__ <= FORMA_MAX((__iter_0__+3),1); __iter_25__++) {
          float __temp_90__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(-1))]);
          float __temp_91__  = (__copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__)]);
          float __temp_92__ = (5 * __temp_90__ + 12 * __temp_91__);
          float __temp_93__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__)]);
          float __temp_94__ = (__temp_92__ + 15 * __temp_93__);
          float __temp_95__ = (__copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__)]);
          float __temp_96__ = (__temp_94__ + 12 * __temp_95__);
          float __temp_97__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(1))]);
          float __temp_98__ = (__temp_96__ + 5 * __temp_97__);
          float __temp_99__ = (__temp_98__ / 118);
          __var_1__[__iter_25__+(M-0)*(__iter_24__)] = __temp_99__;
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_24__ = FORMA_MAX(__iter_1__+4,1) + (int)(threadIdx.x) ; 
    if(__iter_24__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))) {
      int __iter_25__;
      __iter_25__ = FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2));
      for(; __iter_25__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)); __iter_25__++ ){
          float __temp_90__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(-1))]);
          float __temp_91__  = (__copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__)]);
          float __temp_92__ = (5 * __temp_90__ + 12 * __temp_91__);
          float __temp_93__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__)]);
          float __temp_94__ = (__temp_92__ + 15 * __temp_93__);
          float __temp_95__ = (__copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__)]);
          float __temp_96__ = (__temp_94__ + 12 * __temp_95__);
          float __temp_97__ = (__copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(1))]);
          float __temp_98__ = (__temp_96__ + 5 * __temp_97__);
          float __temp_99__ = (__temp_98__ / 118);
          __var_1__[__iter_25__+(M-0)*(__iter_24__)] = __temp_99__;
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 32;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  dim3 blockConfig (__blockConfig___kernel___forma_kernel__0__.x, 8);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, blockConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  
  dim3 __blockConfig___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__.x, 4);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__1__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __copy_arr_0__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  dim3 __blockConfig___kernel___forma_kernel__2__(__blockConfig___kernel___forma_kernel__0__.x, 4);
  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__2__>>> (__copy_arr_0__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __copy_arr_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");

  dim3 __blockConfig___kernel___forma_kernel__3__(__blockConfig___kernel___forma_kernel__0__.x, 4);
  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__3__>>> (__copy_arr_1__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __copy_arr_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  dim3 __blockConfig___kernel___forma_kernel__4__(__blockConfig___kernel___forma_kernel__0__.x, 4);
  __kernel___forma_kernel__4__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__4__>>> (__copy_arr_2__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__4__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

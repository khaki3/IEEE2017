#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __var_2__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 1;
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 1;
  int __iter_2__;
  __iter_2__ = (FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= (FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2))+1) ){
    int __iter_3__;
    __iter_3__ = (FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= (FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2))+1) ){
      int __temp_0__;
      __temp_0__ = __iter_2__;
      int __temp_1__;
      __temp_1__ = __iter_3__;
      __tilevar_1__[__iter_3__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_2__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] = input[__temp_1__+(M-0)*(__temp_0__)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2)) ){
      float __temp_2__;
      __temp_2__ = (5 * __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_3__;
      __temp_3__ = (12 * __tilevar_1__[__iter_5__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_4__;
      __temp_4__ = (__temp_2__ + __temp_3__);
      float __temp_5__;
      __temp_5__ = (15 * __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_6__;
      __temp_6__ = (__temp_4__ + __temp_5__);
      float __temp_7__;
      __temp_7__ = (12 * __tilevar_1__[__iter_5__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_8__;
      __temp_8__ = (__temp_6__ + __temp_7__);
      float __temp_9__;
      __temp_9__ = (5 * __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_10__;
      __temp_10__ = (__temp_8__ + __temp_9__);
      float __temp_11__;
      __temp_11__ = (__temp_10__ / 118);
      __tilevar_0__[__iter_5__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_4__+(0-(FORMA_MAX(__iter_1__,1)-1)))] = __temp_11__;
    }
  }
  __syncthreads();
  int __iter_6__;
  __iter_6__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      float __temp_12__;
      __temp_12__ = (5 * __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(-1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_13__;
      __temp_13__ = (12 * __tilevar_0__[__iter_7__+(-1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_14__;
      __temp_14__ = (__temp_12__ + __temp_13__);
      float __temp_15__;
      __temp_15__ = (15 * __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_16__;
      __temp_16__ = (__temp_14__ + __temp_15__);
      float __temp_17__;
      __temp_17__ = (12 * __tilevar_0__[__iter_7__+(1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_18__;
      __temp_18__ = (__temp_16__ + __temp_17__);
      float __temp_19__;
      __temp_19__ = (5 * __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_20__;
      __temp_20__ = (__temp_18__ + __temp_19__);
      float __temp_21__;
      __temp_21__ = (__temp_20__ / 118);
      __var_2__[__iter_7__+(M-0)*(__iter_6__)] = __temp_21__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ __var_2__, int N, int M, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  int __iter_8__;
  __iter_8__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 1;
  int __iter_9__;
  __iter_9__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 1;
  int __iter_10__;
  __iter_10__ = (FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)+(-1)) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= (FORMA_MIN((FORMA_MIN(((__iter_9__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2))+1) ){
    int __iter_11__;
    __iter_11__ = (FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)+(-1)) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= (FORMA_MIN((FORMA_MIN(((__iter_8__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2))+1) ){
      int __temp_22__;
      __temp_22__ = __iter_10__;
      int __temp_23__;
      __temp_23__ = __iter_11__;
      __tilevar_3__[__iter_11__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] = __var_2__[__temp_23__+(M-0)*(__temp_22__)];
    }
  }
  __syncthreads();
  int __iter_12__;
  __iter_12__ = FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN((FORMA_MIN(((__iter_9__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN((FORMA_MIN(((__iter_8__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2)) ){
      float __temp_24__;
      __temp_24__ = (5 * __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_25__;
      __temp_25__ = (12 * __tilevar_3__[__iter_13__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_26__;
      __temp_26__ = (__temp_24__ + __temp_25__);
      float __temp_27__;
      __temp_27__ = (15 * __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_28__;
      __temp_28__ = (__temp_26__ + __temp_27__);
      float __temp_29__;
      __temp_29__ = (12 * __tilevar_3__[__iter_13__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_30__;
      __temp_30__ = (__temp_28__ + __temp_29__);
      float __temp_31__;
      __temp_31__ = (5 * __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_32__;
      __temp_32__ = (__temp_30__ + __temp_31__);
      float __temp_33__;
      __temp_33__ = (__temp_32__ / 118);
      __tilevar_2__[__iter_13__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_12__+(0-(FORMA_MAX(__iter_9__,1)-1)))] = __temp_33__;
    }
  }
  __syncthreads();
  int __iter_14__;
  __iter_14__ = FORMA_MAX(__iter_9__,1) + (int)(threadIdx.y) ; 
  if( __iter_14__ <= FORMA_MIN(((__iter_9__+FORMA_BLOCKDIM_Y)-1),(N-2)) ){
    int __iter_15__;
    __iter_15__ = FORMA_MAX(__iter_8__,1) + (int)(threadIdx.x) ; 
    if( __iter_15__ <= FORMA_MIN(((__iter_8__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      float __temp_34__;
      __temp_34__ = (5 * __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(-1)+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_35__;
      __temp_35__ = (12 * __tilevar_2__[__iter_15__+(-1)+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_36__;
      __temp_36__ = (__temp_34__ + __temp_35__);
      float __temp_37__;
      __temp_37__ = (15 * __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_38__;
      __temp_38__ = (__temp_36__ + __temp_37__);
      float __temp_39__;
      __temp_39__ = (12 * __tilevar_2__[__iter_15__+(1)+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_40__;
      __temp_40__ = (__temp_38__ + __temp_39__);
      float __temp_41__;
      __temp_41__ = (5 * __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(1)+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_42__;
      __temp_42__ = (__temp_40__ + __temp_41__);
      float __temp_43__;
      __temp_43__ = (__temp_42__ / 118);
      __var_1__[__iter_15__+(M-0)*(__iter_14__)] = __temp_43__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__1__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),5);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),5);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 5)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,5))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  __block_0___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.x-(2-(-2));
  __block_1___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.y-(2-(-2));
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  int __size_0___kernel___forma_kernel__1__ = ((M-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__1__ = ((N-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__1__;
  int _max_occupancy_gridsize___kernel___forma_kernel__1__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__1__,&__max_occupancy_blocksize___kernel___forma_kernel__1__,(const void*)__kernel___forma_kernel__1__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__1___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__1__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__1___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__1___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__1__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__1___0,FORMA_MAX((__size_0___kernel___forma_kernel__1__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),5);
  __max_occupancy_blocksize___kernel___forma_kernel__1__ /= __block_0___kernel___forma_kernel__1__;
  int __max_occupancy_blocksize___kernel___forma_kernel__1___1 = __max_occupancy_blocksize___kernel___forma_kernel__1__;
  int __block_1___kernel___forma_kernel__1__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__1___1,__size_1___kernel___forma_kernel__1__),FORMA_MAX_BLOCKDIM_1),5);
  __max_occupancy_blocksize___kernel___forma_kernel__1__ /= __block_1___kernel___forma_kernel__1__;
  dim3 __blockConfig___kernel___forma_kernel__1__(__block_0___kernel___forma_kernel__1__,__block_1___kernel___forma_kernel__1__);
  int __SMemSize___kernel___forma_kernel__1__ = 0;
  __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
  while( __SMemSize___kernel___forma_kernel__1__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__1__.y/2 > 5)
      __blockConfig___kernel___forma_kernel__1__.y /= 2;
    __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
    if( __SMemSize___kernel___forma_kernel__1__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__1__.x/2 > FORMA_MIN(32,5))
      __blockConfig___kernel___forma_kernel__1__.x /= 2;
    __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
  }
  __block_0___kernel___forma_kernel__1__ = __blockConfig___kernel___forma_kernel__1__.x-(2-(-2));
  __block_1___kernel___forma_kernel__1__ = __blockConfig___kernel___forma_kernel__1__.y-(2-(-2));
  int __grid_0___kernel___forma_kernel__1__ = FORMA_CEIL(__size_0___kernel___forma_kernel__1__,__block_0___kernel___forma_kernel__1__);
  int __grid_1___kernel___forma_kernel__1__ = FORMA_CEIL(__size_1___kernel___forma_kernel__1__,__block_1___kernel___forma_kernel__1__);
  dim3 __gridConfig___kernel___forma_kernel__1__(__grid_0___kernel___forma_kernel__1__,__grid_1___kernel___forma_kernel__1__);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__1__, __blockConfig___kernel___forma_kernel__1__, __SMemSize___kernel___forma_kernel__1__>>> (__var_2__, N, M, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

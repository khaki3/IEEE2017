#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define mod(x,y) ( (x) & (y-1))

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, double * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  double * __tilevar_2__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double * __tilevar_3__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double * __tilevar_4__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double * __tilevar_5__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f;
  double b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;
  int __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X-8);
  int __iter_y__ = FORMA_MAX((int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y)-4, 0);
  // Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
    __tilevar_2__[__iter_3__-__iter_0__] = input[__iter_3__+M*__iter_y__];
    t2 = input[__iter_3__+M*(__iter_y__+1)]; 
  }

  // Initial computation
  for (int __iter_1__ = FORMA_MAX(1,__iter_y__+1); __iter_1__ < FORMA_MIN(N-1,__iter_y__+7); __iter_1__++) {
    if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
      b2 = __tilevar_2__[__iter_3__-__iter_0__];
      __tilevar_2__[__iter_3__-__iter_0__] = t2;
      t2 = input[__iter_3__+M*(__iter_1__+1)];  
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      double __temp_2__ = b2;
      double __temp_5__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
      double __temp_6__ = (5 * __temp_2__ + 12 * __temp_5__);
      double __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__]);
      double __temp_10__ = (__temp_6__ + 15 * __temp_9__);
      double __temp_13__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
      double __temp_14__ = (__temp_10__ + 12 * __temp_13__);
      double __temp_17__ = t2;
      double __temp_18__ = (__temp_14__ + 5 * __temp_17__);
      double __temp_19__ = (__temp_18__ / 118);
      b3 = __tilevar_3__[__iter_3__-__iter_0__];
      __tilevar_3__[__iter_3__-__iter_0__] = t3;
      t3 = __temp_19__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      double __temp_32__ = b3;
      double __temp_35__ = (__tilevar_3__[__iter_3__+(-1)-__iter_0__]);
      double __temp_36__ = (5 * __temp_32__ + 12 * __temp_35__);
      double __temp_39__ = (__tilevar_3__[__iter_3__-__iter_0__]);
      double __temp_40__ = (__temp_36__ + 15 * __temp_39__);
      double __temp_43__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
      double __temp_44__ = (__temp_40__ + 12 * __temp_43__);
      double __temp_47__ = t3;
      double __temp_48__ = (__temp_44__ + 5 * __temp_47__);
      double __temp_49__ = (__temp_48__ / 118);
      b4 = __tilevar_4__[__iter_3__-__iter_0__];
      __tilevar_4__[__iter_3__-__iter_0__] = t4;
      t4 = __temp_49__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      double __temp_60__ = b4;
      double __temp_61__ = (__tilevar_4__[__iter_3__+(-1)-__iter_0__]);
      double __temp_62__ = (5 * __temp_60__ + 12 * __temp_61__);
      double __temp_63__ = (__tilevar_4__[__iter_3__-__iter_0__]);
      double __temp_64__ = (__temp_62__ + 15 * __temp_63__);
      double __temp_65__ = (__tilevar_4__[__iter_3__+1-__iter_0__]);
      double __temp_66__ = (__temp_64__ + 12 * __temp_65__);
      double __temp_67__ = t4;
      double __temp_68__ = (__temp_66__ + 5 * __temp_67__);
      double __temp_69__ = (__temp_68__ / 118);
      b5 = __tilevar_5__[__iter_3__-__iter_0__];
      __tilevar_5__[__iter_3__-__iter_0__] = t5;
      t5 = __temp_69__;
    }
  }
  // Rest of the computation
  for (int __iter_1__ = FORMA_MAX(1,__iter_y__+7); __iter_1__ < FORMA_MIN(N-1,__iter_y__+FORMA_BLOCKDIM_Y+8); __iter_1__++) {
    if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
      b2 = __tilevar_2__[__iter_3__-__iter_0__];
      __tilevar_2__[__iter_3__-__iter_0__] = t2;
      t2 = input[__iter_3__+M*(__iter_1__+1)];  
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      double __temp_2__ = b2;
      double __temp_5__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
      double __temp_6__ = (5 * __temp_2__ + 12 * __temp_5__);
      double __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__]);
      double __temp_10__ = (__temp_6__ + 15 * __temp_9__);
      double __temp_13__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
      double __temp_14__ = (__temp_10__ + 12 * __temp_13__);
      double __temp_17__ = t2;
      double __temp_18__ = (__temp_14__ + 5 * __temp_17__);
      double __temp_19__ = (__temp_18__ / 118);
      b3 = __tilevar_3__[__iter_3__-__iter_0__];
      __tilevar_3__[__iter_3__-__iter_0__] = t3;
      t3 = __temp_19__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      double __temp_32__ = b3;
      double __temp_35__ = (__tilevar_3__[__iter_3__+(-1)-__iter_0__]);
      double __temp_36__ = (5 * __temp_32__ + 12 * __temp_35__);
      double __temp_39__ = (__tilevar_3__[__iter_3__-__iter_0__]);
      double __temp_40__ = (__temp_36__ + 15 * __temp_39__);
      double __temp_43__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
      double __temp_44__ = (__temp_40__ + 12 * __temp_43__);
      double __temp_47__ = t3;
      double __temp_48__ = (__temp_44__ + 5 * __temp_47__);
      double __temp_49__ = (__temp_48__ / 118);
      b4 = __tilevar_4__[__iter_3__-__iter_0__];
      __tilevar_4__[__iter_3__-__iter_0__] = t4;
      t4 = __temp_49__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      double __temp_60__ = b4;
      double __temp_61__ = (__tilevar_4__[__iter_3__+(-1)-__iter_0__]);
      double __temp_62__ = (5 * __temp_60__ + 12 * __temp_61__);
      double __temp_63__ = (__tilevar_4__[__iter_3__-__iter_0__]);
      double __temp_64__ = (__temp_62__ + 15 * __temp_63__);
      double __temp_65__ = (__tilevar_4__[__iter_3__+1-__iter_0__]);
      double __temp_66__ = (__temp_64__ + 12 * __temp_65__);
      double __temp_67__ = t4;
      double __temp_68__ = (__temp_66__ + 5 * __temp_67__);
      double __temp_69__ = (__temp_68__ / 118);
      b5 = __tilevar_5__[__iter_3__-__iter_0__];
      __tilevar_5__[__iter_3__-__iter_0__] = t5;
      t5 = __temp_69__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
      double __temp_80__ = b5;
      double __temp_81__ = (__tilevar_5__[__iter_3__+(-1)-__iter_0__]);
      double __temp_82__ = (5 * __temp_80__ + 12 * __temp_81__);
      double __temp_83__ = (__tilevar_5__[__iter_3__-__iter_0__]);
      double __temp_84__ = (__temp_82__ + 15 * __temp_83__);
      double __temp_85__ = (__tilevar_5__[__iter_3__+1-__iter_0__]);
      double __temp_86__ = (__temp_84__ + 12 * __temp_85__);
      double __temp_87__ = t5;
      double __temp_88__ = (__temp_86__ + 5 * __temp_87__);
      double __temp_89__ = (__temp_88__ / 118);
      __var_1__[__iter_3__+(M)*FORMA_MAX(__iter_1__-3,0)] = __temp_89__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(double)*(4*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(double * h_input, int N, int M, double * __var_0__){

/* Host allocation Begin */
  double * input;
  hipMalloc(&input,sizeof(double)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(double)*((N)*(M)), memcpy_kind_h_input);
  }
  double * __var_1__;
  hipMalloc(&__var_1__,sizeof(double)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);

  int __size_0___kernel___forma_kernel__0__ = M;
  int __size_1___kernel___forma_kernel__0__ = N;
  int __block_0___kernel___forma_kernel__0__ = 128;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-8);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__, 128);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);

  for (int i = 0; i < 125; i++) {
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, 128, __var_1__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_1__, N, M, __blockConfig___kernel___forma_kernel__0__.x, 128, input);
  }
  
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  for (int n = 0 ; n < 5; n++) {
  #ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  for (int i = 0; i < 125; i++) {
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, 128, __var_1__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_1__, N, M, __blockConfig___kernel___forma_kernel__0__.x, 128, input);
  }

  #ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
  }


  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(double)*((N)*(M)), memcpy_kind___var_0__);
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

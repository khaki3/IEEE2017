#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int N, int M, double * __restrict__ __var_4__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_0__ <= (M-2)){
    int __iter_1__;
    __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_1__ <= (N-2)){
      double __temp_0__;
      __temp_0__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(-1))]);
      double __temp_1__;
      __temp_1__ = (12 * input[__iter_0__+(-1)+(M-0)*(__iter_1__)]);
      double __temp_2__;
      __temp_2__ = (__temp_0__ + __temp_1__);
      double __temp_3__;
      __temp_3__ = (15 * input[__iter_0__+(M-0)*(__iter_1__)]);
      double __temp_4__;
      __temp_4__ = (__temp_2__ + __temp_3__);
      double __temp_5__;
      __temp_5__ = (12 * input[__iter_0__+(1)+(M-0)*(__iter_1__)]);
      double __temp_6__;
      __temp_6__ = (__temp_4__ + __temp_5__);
      double __temp_7__;
      __temp_7__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(1))]);
      double __temp_8__;
      __temp_8__ = (__temp_6__ + __temp_7__);
      double __temp_9__;
      __temp_9__ = (__temp_8__ / 118);
      __var_4__[__iter_0__+(M-0)*(__iter_1__)] = __temp_9__;
    }
  }
}
__global__ void __kernel___forma_kernel__1__(double * __restrict__ __var_4__, int N, int M, double * __restrict__ __var_3__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_2__ <= (M-2)){
    int __iter_3__;
    __iter_3__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_3__ <= (N-2)){
      double __temp_10__;
      __temp_10__ = (5 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(-1))]);
      double __temp_11__;
      __temp_11__ = (12 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__)]);
      double __temp_12__;
      __temp_12__ = (__temp_10__ + __temp_11__);
      double __temp_13__;
      __temp_13__ = (15 * __var_4__[__iter_2__+(M-0)*(__iter_3__)]);
      double __temp_14__;
      __temp_14__ = (__temp_12__ + __temp_13__);
      double __temp_15__;
      __temp_15__ = (12 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__)]);
      double __temp_16__;
      __temp_16__ = (__temp_14__ + __temp_15__);
      double __temp_17__;
      __temp_17__ = (5 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(1))]);
      double __temp_18__;
      __temp_18__ = (__temp_16__ + __temp_17__);
      double __temp_19__;
      __temp_19__ = (__temp_18__ / 118);
      __var_3__[__iter_2__+(M-0)*(__iter_3__)] = __temp_19__;
    }
  }
}
__global__ void __kernel___forma_kernel__2__(double * __restrict__ __var_3__, int N, int M, double * __restrict__ __var_2__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_4__;
  __iter_4__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_4__ <= (M-2)){
    int __iter_5__;
    __iter_5__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_5__ <= (N-2)){
      double __temp_20__;
      __temp_20__ = (5 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(-1))]);
      double __temp_21__;
      __temp_21__ = (12 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__)]);
      double __temp_22__;
      __temp_22__ = (__temp_20__ + __temp_21__);
      double __temp_23__;
      __temp_23__ = (15 * __var_3__[__iter_4__+(M-0)*(__iter_5__)]);
      double __temp_24__;
      __temp_24__ = (__temp_22__ + __temp_23__);
      double __temp_25__;
      __temp_25__ = (12 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__)]);
      double __temp_26__;
      __temp_26__ = (__temp_24__ + __temp_25__);
      double __temp_27__;
      __temp_27__ = (5 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(1))]);
      double __temp_28__;
      __temp_28__ = (__temp_26__ + __temp_27__);
      double __temp_29__;
      __temp_29__ = (__temp_28__ / 118);
      __var_2__[__iter_4__+(M-0)*(__iter_5__)] = __temp_29__;
    }
  }
}
__global__ void __kernel___forma_kernel__3__(double * __restrict__ __var_2__, int N, int M, double * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_6__;
  __iter_6__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_6__ <= (M-2)){
    int __iter_7__;
    __iter_7__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_7__ <= (N-2)){
      double __temp_30__;
      __temp_30__ = (5 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(-1))]);
      double __temp_31__;
      __temp_31__ = (12 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__)]);
      double __temp_32__;
      __temp_32__ = (__temp_30__ + __temp_31__);
      double __temp_33__;
      __temp_33__ = (15 * __var_2__[__iter_6__+(M-0)*(__iter_7__)]);
      double __temp_34__;
      __temp_34__ = (__temp_32__ + __temp_33__);
      double __temp_35__;
      __temp_35__ = (12 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__)]);
      double __temp_36__;
      __temp_36__ = (__temp_34__ + __temp_35__);
      double __temp_37__;
      __temp_37__ = (5 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(1))]);
      double __temp_38__;
      __temp_38__ = (__temp_36__ + __temp_37__);
      double __temp_39__;
      __temp_39__ = (__temp_38__ / 118);
      __var_1__[__iter_6__+(M-0)*(__iter_7__)] = __temp_39__;
    }
  }
}
/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(double * h_input, int N, int M, double * __var_0__){

/* Host allocation Begin */
  double * input;
  hipMalloc(&input,sizeof(double)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(double)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  double * __var_1__;
  hipMalloc(&__var_1__,sizeof(double)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  double * __var_2__;
  hipMalloc(&__var_2__,sizeof(double)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
  double * __var_3__;
  hipMalloc(&__var_3__,sizeof(double)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_3__\n");
  double * __var_4__;
  hipMalloc(&__var_4__,sizeof(double)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_4__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 16;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_4__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_4__, N, M, __var_3__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_3__, N, M, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, N, M, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(double)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
  hipFree(__var_3__);
  hipFree(__var_4__);
}
/*Host Free End*/

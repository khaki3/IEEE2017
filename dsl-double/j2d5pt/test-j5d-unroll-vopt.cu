#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define GAPX (22) 
#define GAPY (22)
#define EXTENT (5)

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, float * __restrict__ __copy_arr_t0__, float * __restrict__ __copy_arr_t1__, float * __restrict__ __copy_arr_t2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X) + GAPX);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y) + GAPY);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__,0) + 4*(int)(threadIdx.y) ; 
  if( __iter_2__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(0-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(1-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+1)];
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(2-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+2)];
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(3-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+3)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((__iter_1__+1),1) + 4*(int)(threadIdx.y) ;
  if( __iter_4__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      // iter 0 : __iter_4__ 
      float __temp_a2__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_a5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a6__ = (5*__temp_a2__ + 12*__temp_a5__);
      float __temp_a9__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a10__ = (__temp_a6__ + 15*__temp_a9__);
      float __temp_a13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a14__ = (__temp_a10__ + 12*__temp_a13__);
      float __temp_a17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_a18__ = (__temp_a14__ + 5*__temp_a17__);
      float __temp_a19__ = (__temp_a18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+1)))] = __temp_a19__;
      // iter 1 : __iter_4__ + 1
      float __temp_b5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b6__ = (5*__temp_a9__+ 12*__temp_b5__);
      float __temp_b10__ = (__temp_b6__ + 15*__temp_a17__);
      float __temp_b13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b14__ = (__temp_b10__ + 12*__temp_b13__);
      float __temp_b17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_b18__ = (__temp_b14__ + 5*__temp_b17__);
      float __temp_b19__ = (__temp_b18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1-(__iter_1__+1)))] = __temp_b19__;
      // iter 2 : __iter_4__ + 2
      float __temp_c5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c6__ = (5*__temp_a17__+ 12*__temp_c5__);
      float __temp_c10__ = (__temp_c6__ + 15*__temp_b17__);
      float __temp_c13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c14__ = (__temp_c10__ + 12*__temp_c13__);
      float __temp_c17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(0-(__iter_1__+0)))]);
      float __temp_c18__ = (__temp_c14__ + 5*__temp_c17__);
      float __temp_c19__ = (__temp_c18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2-(__iter_1__+1)))] = __temp_c19__;
      // iter 3 : __iter_4__ + 3
      float __temp_d5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d6__ = (5*__temp_b17__+ 12*__temp_d5__);
      float __temp_d10__ = (__temp_d6__ + 15*__temp_c17__);
      float __temp_d13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d14__ = (__temp_d10__ + 12*__temp_d13__);
      float __temp_d17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(4)+(0-(__iter_1__+0)))]);
      float __temp_d18__ = (__temp_d14__ + 5*__temp_d17__);
      float __temp_d19__ = (__temp_d18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3-(__iter_1__+1)))] = __temp_d19__;
    }
  }
  else if( __iter_4__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      // iter 0 : __iter_4__ 
      float __temp_a2__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_a5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a6__ = (5*__temp_a2__ + 12*__temp_a5__);
      float __temp_a9__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a10__ = (__temp_a6__ + 15*__temp_a9__);
      float __temp_a13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a14__ = (__temp_a10__ + 12*__temp_a13__);
      float __temp_a17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_a18__ = (__temp_a14__ + 5*__temp_a17__);
      float __temp_a19__ = (__temp_a18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+1)))] = __temp_a19__;
      // iter 1 : __iter_4__ + 1
      float __temp_b5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b6__ = (5*__temp_a9__+ 12*__temp_b5__);
      float __temp_b10__ = (__temp_b6__ + 15*__temp_a17__);
      float __temp_b13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b14__ = (__temp_b10__ + 12*__temp_b13__);
      float __temp_b17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_b18__ = (__temp_b14__ + 5*__temp_b17__);
      float __temp_b19__ = (__temp_b18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1-(__iter_1__+1)))] = __temp_b19__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) {
    int __iter_6__ = FORMA_MAX((__iter_1__+1),1);
    for(; __iter_6__ < (FORMA_MAX((__iter_1__+1),1)+2); __iter_6__++) {
      int __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+1)))];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_6__ = (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))-1);
    for(; __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ; __iter_6__++) {
      int __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+1)))];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_6__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.x) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_7__ = FORMA_MAX((__iter_0__+1),1);
      for(; __iter_7__ < (FORMA_MAX((__iter_0__+1),1)+2); __iter_7__++) {
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+1)))];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_6__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.x) ;
    if (__iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_7__ = (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-1);
      for(; __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ; __iter_7__++ ){
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+1)))];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__+2),1) + 4*(int)(threadIdx.y) ; 
  if( __iter_10__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      // iter 1 : __iter_10__ 
      float __temp_a32__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(-1)+(0-(__iter_1__+1)))]);
      float __temp_a35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+1)))]);
      float __temp_a36__ = (5*__temp_a32__ + 12*__temp_a35__);
      float __temp_a39__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+1)))]);
      float __temp_a40__ = (__temp_a36__ + 15*__temp_a39__);
      float __temp_a43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+1)))]);
      float __temp_a44__ = (__temp_a40__ + 12*__temp_a43__);
      float __temp_a47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(0-(__iter_1__+1)))]);
      float __temp_a48__ = (__temp_a44__ + 5*__temp_a47__);
      float __temp_a49__ = (__temp_a48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+2)))] = __temp_a49__;
      // iter 2 : __iter_10__ + 1
      float __temp_b35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(0-(__iter_1__+1)))]);
      float __temp_b36__ = (5*__temp_a39__+ 12*__temp_b35__);
      float __temp_b40__ = (__temp_b36__ + 15*__temp_a47__);
      float __temp_b43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(0-(__iter_1__+1)))]);
      float __temp_b44__ = (__temp_b40__ + 12*__temp_b43__);
      float __temp_b47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(0-(__iter_1__+1)))]);
      float __temp_b48__ = (__temp_b44__ + 5*__temp_b47__);
      float __temp_b49__ = (__temp_b48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1-(__iter_1__+2)))] = __temp_b49__;
      // iter 3 : __iter_10__ + 2
      float __temp_c35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(0-(__iter_1__+1)))]);
      float __temp_c36__ = (5*__temp_a47__+ 12*__temp_c35__);
      float __temp_c40__ = (__temp_c36__ + 15*__temp_b47__);
      float __temp_c43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(0-(__iter_1__+1)))]);
      float __temp_c44__ = (__temp_c40__ + 12*__temp_c43__);
      float __temp_c47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(0-(__iter_1__+1)))]);
      float __temp_c48__ = (__temp_c44__ + 5*__temp_c47__);
      float __temp_c49__ = (__temp_c48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2-(__iter_1__+2)))] = __temp_c49__;
      // iter 4 : __iter_10__ + 3
      float __temp_d35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(0-(__iter_1__+1)))]);
      float __temp_d36__ = (5*__temp_b47__+ 12*__temp_d35__);
      float __temp_d40__ = (__temp_d36__ + 15*__temp_c47__);
      float __temp_d43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(0-(__iter_1__+1)))]);
      float __temp_d44__ = (__temp_d40__ + 12*__temp_d43__);
      float __temp_d47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(4)+(0-(__iter_1__+1)))]);
      float __temp_d48__ = (__temp_d44__ + 5*__temp_d47__);
      float __temp_d49__ = (__temp_d48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3-(__iter_1__+2)))] = __temp_d49__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) {
    int __iter_12__ = FORMA_MAX((__iter_1__+2),1);
    for(; __iter_12__ < (FORMA_MAX((__iter_1__+2),1)+2); __iter_12__++) {
      int __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+2)))];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_12__ = (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))-1);
    for(; __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ; __iter_12__++) {
      int __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+2)))];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_12__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.x) ; 
    if (__iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))) {
      int __iter_13__ = FORMA_MAX((__iter_0__+2),1);
      for(; __iter_13__ < (FORMA_MAX((__iter_0__+2),1)+2); __iter_13__++) {
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+2)))];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_12__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.x) ; 
    if (__iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))) {
      int __iter_13__ = (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-1);
      for(; __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ; __iter_13__++){
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+2)))];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__+3),1) + 4*(int)(threadIdx.y) ;
  if( __iter_16__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      // iter 1 : __iter_16__ 
      float __temp_a60__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+2)))]);
      float __temp_a61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_a62__ = (5*__temp_a60__ + 12*__temp_a61__);
      float __temp_a63__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_a64__ = (__temp_a62__ + 15*__temp_a63__);
      float __temp_a65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_a66__ = (__temp_a64__ + 12*__temp_a65__);
      float __temp_a67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_a68__ = (__temp_a66__ + 5*__temp_a67__);
      float __temp_a69__ = (__temp_a68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+3)))] = __temp_a69__;
      // iter 2 : __iter_16__ + 1
      float __temp_b61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_b62__ = (5*__temp_a63__+ 12*__temp_b61__);
      float __temp_b64__ = (__temp_b62__ + 15*__temp_a67__);
      float __temp_b65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_b66__ = (__temp_b64__ + 12*__temp_b65__);
      float __temp_b67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+2)))]);
      float __temp_b68__ = (__temp_b66__ + 5*__temp_b67__);
      float __temp_b69__ = (__temp_b68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1-(__iter_1__+3)))] = __temp_b69__;
      // iter 3 : __iter_16__ + 2
      float __temp_c61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+2)))]);
      float __temp_c62__ = (5*__temp_a67__+ 12*__temp_c61__);
      float __temp_c64__ = (__temp_c62__ + 15*__temp_b67__);
      float __temp_c65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+2)))]);
      float __temp_c66__ = (__temp_c64__ + 12*__temp_c65__);
      float __temp_c67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(0-(__iter_1__+2)))]);
      float __temp_c68__ = (__temp_c66__ + 5*__temp_c67__);
      float __temp_c69__ = (__temp_c68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2-(__iter_1__+3)))] = __temp_c69__;
      // iter 4 : __iter_16__ + 3
      float __temp_d61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(0-(__iter_1__+2)))]);
      float __temp_d62__ = (5*__temp_b67__+ 12*__temp_d61__);
      float __temp_d64__ = (__temp_d62__ + 15*__temp_c67__);
      float __temp_d65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(0-(__iter_1__+2)))]);
      float __temp_d66__ = (__temp_d64__ + 12*__temp_d65__);
      float __temp_d67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(4)+(0-(__iter_1__+2)))]);
      float __temp_d68__ = (__temp_d66__ + 5*__temp_d67__);
      float __temp_d69__ = (__temp_d68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3-(__iter_1__+3)))] = __temp_d69__;
    }
  }
  else if( __iter_16__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      // iter 1 : __iter_16__ 
      float __temp_a60__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+2)))]);
      float __temp_a61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_a62__ = (5*__temp_a60__ + 12*__temp_a61__);
      float __temp_a63__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_a64__ = (__temp_a62__ + 15*__temp_a63__);
      float __temp_a65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_a66__ = (__temp_a64__ + 12*__temp_a65__);
      float __temp_a67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_a68__ = (__temp_a66__ + 5*__temp_a67__);
      float __temp_a69__ = (__temp_a68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+3)))] = __temp_a69__;
      // iter 2 : __iter_16__ + 1
      float __temp_b61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_b62__ = (5*__temp_a63__+ 12*__temp_b61__);
      float __temp_b64__ = (__temp_b62__ + 15*__temp_a67__);
      float __temp_b65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_b66__ = (__temp_b64__ + 12*__temp_b65__);
      float __temp_b67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+2)))]);
      float __temp_b68__ = (__temp_b66__ + 5*__temp_b67__);
      float __temp_b69__ = (__temp_b68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1-(__iter_1__+3)))] = __temp_b69__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) {
    int __iter_18__ = FORMA_MAX((__iter_1__+3),1);
    for(; __iter_18__ < (FORMA_MAX((__iter_1__+3),1)+2); __iter_18__++) {
      int __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+3)))];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_18__ = (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))-1);
    for(; __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ; __iter_18__++) {
      int __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+3)))];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_18__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.x) ; 
    if (__iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))) {
      int __iter_19__ = FORMA_MAX((__iter_0__+3),1);
      for(; __iter_19__ < (FORMA_MAX((__iter_0__+3),1)+2); __iter_19__++) {
         __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+3)))];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_18__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.x) ; 
    if (__iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))) {
      int __iter_19__ = (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-1);
      for(; __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ; __iter_19__++){
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+3)))];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__+4),1) + 4*(int)(threadIdx.y) ; 
  if( __iter_22__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
      // iter 0 : __iter_22__
      float __temp_a80__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+3)))]);
      float __temp_a81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_a82__ = (5*__temp_a80__ + 12*__temp_a81__);
      float __temp_a83__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_a84__ = (__temp_a82__ + 15*__temp_a83__);
      float __temp_a85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_a86__ = (__temp_a84__ + 12*__temp_a85__);
      float __temp_a87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+3)))]);
      float __temp_a88__ = (__temp_a86__ + 5*__temp_a87__);
      float __temp_a89__ = (__temp_a88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_a89__;
      // iter 1 : __iter_22__ + 1
      float __temp_b81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+3)))]);
      float __temp_b82__ = (5*__temp_a83__+ 12*__temp_b81__);
      float __temp_b84__ = (__temp_b82__ + 15*__temp_a87__);
      float __temp_b85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+3)))]);
      float __temp_b86__ = (__temp_b84__ + 12*__temp_b85__);
      float __temp_b87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+3)))]);
      float __temp_b88__ = (__temp_b86__ + 5*__temp_b87__);
      float __temp_b89__ = (__temp_b88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+1)] = __temp_b89__;
      // iter 2 : __iter_22__ + 2
      float __temp_c81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+3)))]);
      float __temp_c82__ = (5*__temp_a87__+ 12*__temp_c81__);
      float __temp_c84__ = (__temp_c82__ + 15*__temp_b87__);
      float __temp_c85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+3)))]);
      float __temp_c86__ = (__temp_c84__ + 12*__temp_c85__);
      float __temp_c87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(0-(__iter_1__+3)))]);
      float __temp_c88__ = (__temp_c86__ + 5*__temp_c87__);
      float __temp_c89__ = (__temp_c88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+2)] = __temp_c89__;
      // iter 3 : __iter_22__ + 3
      float __temp_d81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(0-(__iter_1__+3)))]);
      float __temp_d82__ = (5*__temp_b87__+ 12*__temp_d81__);
      float __temp_d84__ = (__temp_d82__ + 15*__temp_c87__);
      float __temp_d85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(0-(__iter_1__+3)))]);
      float __temp_d86__ = (__temp_d84__ + 12*__temp_d85__);
      float __temp_d87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(4)+(0-(__iter_1__+3)))]);
      float __temp_d88__ = (__temp_d86__ + 5*__temp_d87__);
      float __temp_d89__ = (__temp_d88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+3)] = __temp_d89__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, float * __restrict__ __copy_arr_t0__, float * __restrict__ __copy_arr_t1__, float * __restrict__ __copy_arr_t2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X) + GAPX) + (int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y) + GAPY);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__,0) + 4*(int)(threadIdx.y) ; 
  if( __iter_2__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(0-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(1-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+1)];
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(2-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+2)];
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(3-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+3)];
    }
  }
  __syncthreads();
  int __iter_4__; 
  __iter_4__ = FORMA_MAX((__iter_1__+1),1) + 4*(int)(threadIdx.y) ; 
  if( __iter_4__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
      // iter 0 : __iter_4__ 
      float __temp_a2__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_a5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a6__ = (5 * __temp_a2__ + 12 * __temp_a5__);
      float __temp_a9__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a10__ = (__temp_a6__ + 15 * __temp_a9__);
      float __temp_a13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a14__ = (__temp_a10__ + 12 * __temp_a13__);
      float __temp_a17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_a18__ = (__temp_a14__ + 5 * __temp_a17__);
      float __temp_a19__ = (__temp_a18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))] = __temp_a19__;
      // iter 0 : __iter_4__ + 1
      float __temp_b5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b6__ = (5 * __temp_a9__+ 12 * __temp_b5__);
      float __temp_b10__ = (__temp_b6__ + 15 * __temp_a17__);
      float __temp_b13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b14__ = (__temp_b10__ + 12 * __temp_b13__);
      float __temp_b17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_b18__ = (__temp_b14__ + 5 * __temp_b17__);
      float __temp_b19__ = (__temp_b18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1-(__iter_1__+0)))] = __temp_b19__;
      // iter 0 : __iter_4__ + 2
      float __temp_c5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c6__ = (5 * __temp_a17__+ 12 * __temp_c5__);
      float __temp_c10__ = (__temp_c6__ + 15 * __temp_b17__);
      float __temp_c13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c14__ = (__temp_c10__ + 12 * __temp_c13__);
      float __temp_c17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(0-(__iter_1__+0)))]);
      float __temp_c18__ = (__temp_c14__ + 5 * __temp_c17__);
      float __temp_c19__ = (__temp_c18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2-(__iter_1__+0)))] = __temp_c19__;
      // iter 3 : __iter_4__ + 3
      float __temp_d5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d6__ = (5 * __temp_b17__ + 12 * __temp_d5__);
      float __temp_d10__ = (__temp_d6__ + 15 * __temp_c17__);
      float __temp_d13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d14__ = (__temp_d10__ + 12 * __temp_d13__);
      float __temp_d17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(4)+(0-(__iter_1__+0)))]);
      float __temp_d18__ = (__temp_d14__ + 5 * __temp_d17__);
      float __temp_d19__ = (__temp_d18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3-(__iter_1__+0)))] = __temp_d19__;
    }
  }
  else if( __iter_4__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
      // iter 0 : __iter_4__ 
      float __temp_a2__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_a5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a6__ = (5 * __temp_a2__ + 12 * __temp_a5__);
      float __temp_a9__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a10__ = (__temp_a6__ + 15 * __temp_a9__);
      float __temp_a13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))]);
      float __temp_a14__ = (__temp_a10__ + 12 * __temp_a13__);
      float __temp_a17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_a18__ = (__temp_a14__ + 5 * __temp_a17__);
      float __temp_a19__ = (__temp_a18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))] = __temp_a19__;
      // iter 0 : __iter_4__ + 1
      float __temp_b5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b6__ = (5 * __temp_a9__ + 12 * __temp_b5__);
      float __temp_b10__ = (__temp_b6__ + 15 * __temp_a17__);
      float __temp_b13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b14__ = (__temp_b10__ + 12 * __temp_b13__);
      float __temp_b17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(0-(__iter_1__+0)))]);
      float __temp_b18__ = (__temp_b14__ + 5 * __temp_b17__);
      float __temp_b19__ = (__temp_b18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1-(__iter_1__+0)))] = __temp_b19__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) {
    int __iter_6__ = FORMA_MAX((__iter_1__+1),1);
    for(; __iter_6__ < (FORMA_MAX((__iter_1__+1),1)+2); __iter_6__++) {
      int __iter_7__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_6__ = FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))-1;
    for(; __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ; __iter_6__++) {
      int __iter_7__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_6__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.x) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) { 
      int __iter_7__ = FORMA_MAX((__iter_0__-3),1);
      for(; __iter_7__ < FORMA_MAX((__iter_0__-1),1); __iter_7__++) {
        __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_6__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.x) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_7__ = FORMA_MIN(((__iter_0__+GAPX+1)),(M-2));
      for(; __iter_7__ < FORMA_MIN(((__iter_0__+GAPX+3)),(M-2)); __iter_7__++){
         __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__+2),1) + 4*(int)(threadIdx.y) ; 
  if( __iter_10__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
      // iter 0 : __iter_10__ 
      float __temp_a32__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*( __iter_10__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_a35__ = (__tilevar_3__[__iter_11__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+0)))]);
      float __temp_a36__ = (5 * __temp_a32__ + 12 * __temp_a35__);
      float __temp_a39__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+0)))]);
      float __temp_a40__ = (__temp_a36__ + 15 * __temp_a39__);
      float __temp_a43__ = (__tilevar_3__[ __iter_11__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+0)))]);
      float __temp_a44__ = (__temp_a40__ + 12 * __temp_a43__);
      float __temp_a47__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(0-(__iter_1__+0)))]);
      float __temp_a48__ = (__temp_a44__ + 5 * __temp_a47__);
      float __temp_a49__ = (__temp_a48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+0)))] = __temp_a49__;
      // iter 2 : __iter_10__ + 1
      float __temp_b35__ = (__tilevar_3__[__iter_11__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b36__ = (5 * __temp_a39__ + 12 * __temp_b35__);
      float __temp_b40__ = (__temp_b36__ + 15 * __temp_a47__);
      float __temp_b43__ = (__tilevar_3__[ __iter_11__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b44__ = (__temp_b40__ + 12 * __temp_b43__);
      float __temp_b47__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(0-(__iter_1__+0)))]);
      float __temp_b48__ = (__temp_b44__ + 5 * __temp_b47__);
      float __temp_b49__ = (__temp_b48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1-(__iter_1__+0)))] = __temp_b49__;
      // iter 3 : __iter_10__ + 2
      float __temp_c35__ = (__tilevar_3__[__iter_11__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c36__ = (5 * __temp_a47__ + 12 * __temp_c35__);
      float __temp_c40__ = (__temp_c36__ + 15 * __temp_b47__);
      float __temp_c43__ = (__tilevar_3__[ __iter_11__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c44__ = (__temp_c40__ + 12 * __temp_c43__);
      float __temp_c47__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(0-(__iter_1__+0)))]);
      float __temp_c48__ = (__temp_c44__ + 5 * __temp_c47__);
      float __temp_c49__ = (__temp_c48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2-(__iter_1__+0)))] = __temp_c49__;
      // iter 4 : __iter_10__ + 3
      float __temp_d35__ = (__tilevar_3__[__iter_11__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d36__ = (5 * __temp_b47__ + 12 * __temp_d35__);
      float __temp_d40__ = (__temp_d36__ + 15 * __temp_c47__);
      float __temp_d43__ = (__tilevar_3__[ __iter_11__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d44__ = (__temp_d40__ + 12 * __temp_d43__);
      float __temp_d47__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(4)+(0-(__iter_1__+0)))]);
      float __temp_d48__ = (__temp_d44__ + 5 * __temp_d47__);
      float __temp_d49__ = (__temp_d48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3-(__iter_1__+0)))] = __temp_d49__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) {
    int __iter_12__ = FORMA_MAX((__iter_1__+2),1);
    for(; __iter_12__ < (FORMA_MAX((__iter_1__+2),1)+2); __iter_12__++) {
      int __iter_13__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_12__ = (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))-1);
    for(; __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ; __iter_12__++) {
      int __iter_13__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_12__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.x) ; 
    if(__iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))) {
      int __iter_13__ = FORMA_MAX((__iter_0__-4),1);
      for(; __iter_13__ < FORMA_MAX((__iter_0__-2),1); __iter_13__++) {
        __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_12__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.x) ; 
    if(__iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))) {
      int __iter_13__ = FORMA_MIN((__iter_0__+GAPX+2),(M-2));
      for(; __iter_13__ < FORMA_MIN((__iter_0__+GAPX+4),(M-2)) ; __iter_13__++){
         __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__+3),1) + 4*(int)(threadIdx.y) ;
  if( __iter_16__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
      // iter 0 : __iter_16__
      float __temp_a60__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_a61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))]);
      float __temp_a62__ = (5 * __temp_a60__ + 12 * __temp_a61__);
      float __temp_a63__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))]);
      float __temp_a64__ = (__temp_a62__ + 15 * __temp_a63__);
      float __temp_a65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))]);
      float __temp_a66__ = (__temp_a64__ + 12 * __temp_a65__);
      float __temp_a67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+0)))]);
      float __temp_a68__ = (__temp_a66__ + 5 * __temp_a67__);
      float __temp_a69__ = (__temp_a68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))] = __temp_a69__;
      // iter 1 : __iter_16__ + 1
      float __temp_b61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b62__ = (5 * __temp_a63__ + 12 * __temp_b61__);
      float __temp_b64__ = (__temp_b62__ + 15 * __temp_a67__);
      float __temp_b65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b66__ = (__temp_b64__ + 12 * __temp_b65__);
      float __temp_b67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+0)))]);
      float __temp_b68__ = (__temp_b66__ + 5 * __temp_b67__);
      float __temp_b69__ = (__temp_b68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1-(__iter_1__+0)))] = __temp_b69__;
      // iter 2 : __iter_16__ + 2
      float __temp_c61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c62__ = (5 * __temp_a67__ + 12 * __temp_c61__);
      float __temp_c64__ = (__temp_c62__ + 15 * __temp_b67__);
      float __temp_c65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c66__ = (__temp_c64__ + 12 * __temp_c65__);
      float __temp_c67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(0-(__iter_1__+0)))]);
      float __temp_c68__ = (__temp_c66__ + 5 * __temp_c67__);
      float __temp_c69__ = (__temp_c68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2-(__iter_1__+0)))] = __temp_c69__;
      // iter 3 : __iter_16__ + 3
      float __temp_d61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d62__ = (5 * __temp_b67__ + 12 * __temp_d61__);
      float __temp_d64__ = (__temp_d62__ + 15 * __temp_c67__);
      float __temp_d65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d66__ = (__temp_d64__ + 12 * __temp_d65__);
      float __temp_d67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(4)+(0-(__iter_1__+0)))]);
      float __temp_d68__ = (__temp_d66__ + 5 * __temp_d67__);
      float __temp_d69__ = (__temp_d68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3-(__iter_1__+0)))] = __temp_d69__;
    }
  }
  else if( __iter_16__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
      // iter 0 : __iter_16__
      float __temp_a60__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_a61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))]);
      float __temp_a62__ = (5 * __temp_a60__ + 12 * __temp_a61__);
      float __temp_a63__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))]);
      float __temp_a64__ = (__temp_a62__ + 15 * __temp_a63__);
      float __temp_a65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))]);
      float __temp_a66__ = (__temp_a64__ + 12 * __temp_a65__);
      float __temp_a67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+0)))]);
      float __temp_a68__ = (__temp_a66__ + 5 * __temp_a67__);
      float __temp_a69__ = (__temp_a68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))] = __temp_a69__;
      // iter 1 : __iter_16__ + 1
      float __temp_b61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b62__ = (5 * __temp_a63__+ 12 * __temp_b61__);
      float __temp_b64__ = (__temp_b62__ + 15 * __temp_a67__);
      float __temp_b65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b66__ = (__temp_b64__ + 12 * __temp_b65__);
      float __temp_b67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+0)))]);
      float __temp_b68__ = (__temp_b66__ + 5 * __temp_b67__);
      float __temp_b69__ = (__temp_b68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1-(__iter_1__+0)))] = __temp_b69__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) {
    int __iter_18__ = FORMA_MAX((__iter_1__+3),1);
    for(; __iter_18__ < (FORMA_MAX((__iter_1__+3),1)+2); __iter_18__++) {
      int __iter_19__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_18__ = FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))-1;
    for(; __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ; __iter_18__++) {
      int __iter_19__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_18__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.x) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))) {
      int __iter_19__ = FORMA_MAX((__iter_0__-5),1);
      for(; __iter_19__ < FORMA_MAX((__iter_0__-3),1); __iter_19__++) {
        __tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_18__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.x) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))) {
      int __iter_19__ = FORMA_MIN(((__iter_0__+GAPX+3)),(M-2));
      for(; __iter_19__ < FORMA_MIN(((__iter_0__+GAPX+5)),(M-2)) ; __iter_19__++ ){
        __tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__+4),1) + 4*(int)(threadIdx.y) ; 
  if( __iter_22__ + 3 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
      // iter 0 : __iter_22__
      float __temp_a80__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_a81__ = (__tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+0)))]);
      float __temp_a82__ = (5 * __temp_a80__ + 12 * __temp_a81__);
      float __temp_a83__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+0)))]);
      float __temp_a84__ = (__temp_a82__ + 15 * __temp_a83__);
      float __temp_a85__ = (__tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+0)))]);
      float __temp_a86__ = (__temp_a84__ + 12 * __temp_a85__);
      float __temp_a87__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+0)))]);
      float __temp_a88__ = (__temp_a86__ + 5 * __temp_a87__);
      float __temp_a89__ = (__temp_a88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_a89__;
      // iter 1 : __iter_22__ + 1
      float __temp_b81__ = (__tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b82__ = (5 * __temp_a83__ + 12 * __temp_b81__);
      float __temp_b84__ = (__temp_b82__ + 15 * __temp_a87__);
      float __temp_b85__ = (__tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+0)))]);
      float __temp_b86__ = (__temp_b84__ + 12 * __temp_b85__);
      float __temp_b87__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+0)))]);
      float __temp_b88__ = (__temp_b86__ + 5 * __temp_b87__);
      float __temp_b89__ = (__temp_b88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+1)] = __temp_b89__;
      // iter 2 : __iter_22__ + 2
      float __temp_c81__ = (__tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c82__ = (5 * __temp_a87__ + 12 * __temp_c81__);
      float __temp_c84__ = (__temp_c82__ + 15 * __temp_b87__);
      float __temp_c85__ = (__tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+0)))]);
      float __temp_c86__ = (__temp_c84__ + 12 * __temp_c85__);
      float __temp_c87__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(0-(__iter_1__+0)))]);
      float __temp_c88__ = (__temp_c86__ + 5 * __temp_c87__);
      float __temp_c89__ = (__temp_c88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+2)] = __temp_c89__;
      // iter 3 : __iter_22__ + 3
      float __temp_d81__ = (__tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d82__ = (5 * __temp_b87__ + 12 * __temp_d81__);
      float __temp_d84__ = (__temp_d82__ + 15 * __temp_c87__);
      float __temp_d85__ = (__tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(0-(__iter_1__+0)))]);
      float __temp_d86__ = (__temp_d84__ + 12 * __temp_d85__);
      float __temp_d87__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(4)+(0-(__iter_1__+0)))]);
      float __temp_d88__ = (__temp_d86__ + 5 * __temp_d87__);
      float __temp_d89__ = (__temp_d88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+3)] = __temp_d89__;
    }
  }
}

__global__ void __kernel___forma_kernel__2__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, float * __restrict__ __copy_arr_t0__, float * __restrict__ __copy_arr_t1__, float * __restrict__ __copy_arr_t2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X) + GAPX);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y) + GAPY) + (int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__-2,0) + 4*(int)(threadIdx.y) ; 
  if( __iter_2__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(1)+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+1)];
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(2)+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+2)];
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(3)+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+3)];
    }
  }
  else if( __iter_2__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(1)+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+1)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((__iter_1__-1),1) + 4*(int)(threadIdx.y) ;
  if( __iter_4__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      // iter 0 : __iter_4__ 
      float __temp_a2__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a6__ = (5 * __temp_a2__ + 12 * __temp_a5__);
      float __temp_a9__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a10__ = (__temp_a6__ + 15 * __temp_a9__);
      float __temp_a13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a14__ = (__temp_a10__ + 12 * __temp_a13__);
      float __temp_a17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a18__ = (__temp_a14__ + 5 * __temp_a17__);
      float __temp_a19__ = (__temp_a18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))] = __temp_a19__;
      // iter 1 : __iter_4__ + 1
      float __temp_b5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b6__ = (5 * __temp_a9__ + 12 * __temp_b5__);
      float __temp_b10__ = (__temp_b6__ + 15 * __temp_a17__);
      float __temp_b13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b14__ = (__temp_b10__ + 12 * __temp_b13__);
      float __temp_b17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b18__ = (__temp_b14__ + 5 * __temp_b17__);
      float __temp_b19__ = (__temp_b18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b19__;
      // iter 2 : __iter_4__ + 2
      float __temp_c5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c6__ = (5 * __temp_a17__ + 12 * __temp_c5__);
      float __temp_c10__ = (__temp_c6__ + 15 * __temp_b17__);
      float __temp_c13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c14__ = (__temp_c10__ + 12 * __temp_c13__);
      float __temp_c17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c18__ = (__temp_c14__ + 5 * __temp_c17__);
      float __temp_c19__ = (__temp_c18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(EXTENT-(__iter_1__+0)))] = __temp_c19__;
      // iter 3 : __iter_4__ + 3
      float __temp_d5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d6__ = (5 * __temp_b17__ + 12 * __temp_d5__);
      float __temp_d10__ = (__temp_d6__ + 15 * __temp_c17__);
      float __temp_d13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d14__ = (__temp_d10__ + 12 * __temp_d13__);
      float __temp_d17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(4)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d18__ = (__temp_d14__ + 5 * __temp_d17__);
      float __temp_d19__ = (__temp_d18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(EXTENT-(__iter_1__+0)))] = __temp_d19__;
    }
  }
  else if( __iter_4__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ;
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      // iter 0 : __iter_4__ 
      float __temp_a2__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a6__ = (5 * __temp_a2__ + 12 * __temp_a5__);
      float __temp_a9__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a10__ = (__temp_a6__ + 15 * __temp_a9__);
      float __temp_a13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a14__ = (__temp_a10__ + 12 * __temp_a13__);
      float __temp_a17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a18__ = (__temp_a14__ + 5 * __temp_a17__);
      float __temp_a19__ = (__temp_a18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))] = __temp_a19__;
      // iter 1 : __iter_4__ + 1
      float __temp_b5__ = (__tilevar_2__[__iter_5__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b6__ = (5 * __temp_a9__ + 12 * __temp_b5__);
      float __temp_b10__ = (__temp_b6__ + 15 * __temp_a17__);
      float __temp_b13__ = (__tilevar_2__[__iter_5__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b14__ = (__temp_b10__ + 12 * __temp_b13__);
      float __temp_b17__ = (__tilevar_2__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b18__ = (__temp_b14__ + 5 * __temp_b17__);
      float __temp_b19__ = (__temp_b18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b19__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) {
    int __iter_6__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.x) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2))) {
      int __iter_7__ = FORMA_MAX((__iter_0__+1),1);
      for(; __iter_7__ < (FORMA_MAX((__iter_0__+1),1)+2); __iter_7__++) {
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_6__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.x) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2))) {
      int __iter_7__ = (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-1);
      for(; __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ; __iter_7__++){
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_6__ = FORMA_MAX((__iter_1__-3),1);
    for(; __iter_6__ < FORMA_MAX((__iter_1__-1),1); __iter_6__++) {
      int __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        __tilevar_3__[__iter_7__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_6__ = FORMA_MIN(((__iter_1__+GAPY+1)),(N-2));
    for(; __iter_6__ < FORMA_MIN(((__iter_1__+GAPY+3)),(N-2)) ; __iter_6__++) {
      int __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        __tilevar_3__[__iter_7__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__-2),1) + 4*(int)(threadIdx.y) ;
  if( __iter_10__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      // iter 0 : __iter_10__ 
      float __temp_a32__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*( __iter_10__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a36__ = (5 * __temp_a32__ + 12 * __temp_a35__);
      float __temp_a39__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a40__ = (__temp_a36__ + 15 * __temp_a39__);
      float __temp_a43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a44__ = (__temp_a40__ + 12 * __temp_a43__);
      float __temp_a47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a48__ = (__temp_a44__ + 5 * __temp_a47__);
      float __temp_a49__ = (__temp_a48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))] = __temp_a49__;
      // iter 1 : __iter_10__ + 1
      float __temp_b35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b36__ = (5 * __temp_a39__ + 12 * __temp_b35__);
      float __temp_b40__ = (__temp_b36__ + 15 * __temp_a47__);
      float __temp_b43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b44__ = (__temp_b40__ + 12 * __temp_b43__);
      float __temp_b47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b48__ = (__temp_b44__ + 5 * __temp_b47__);
      float __temp_b49__ = (__temp_b48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b49__;
      // iter 2 : __iter_10__ + 2
      float __temp_c35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c36__ = (5 * __temp_a47__ + 12 * __temp_c35__);
      float __temp_c40__ = (__temp_c36__ + 15 * __temp_b47__);
      float __temp_c43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c44__ = (__temp_c40__ + 12 * __temp_c43__);
      float __temp_c47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c48__ = (__temp_c44__ + 5 * __temp_c47__);
      float __temp_c49__ = (__temp_c48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(EXTENT-(__iter_1__+0)))] = __temp_c49__;
      // iter 3 : __iter_10__ + 3
      float __temp_d35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d36__ = (5 * __temp_b47__ + 12 * __temp_d35__);
      float __temp_d40__ = (__temp_d36__ + 15 * __temp_c47__);
      float __temp_d43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d44__ = (__temp_d40__ + 12 * __temp_d43__);
      float __temp_d47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(4)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d48__ = (__temp_d44__ + 5 * __temp_d47__);
      float __temp_d49__ = (__temp_d48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(EXTENT-(__iter_1__+0)))] = __temp_d49__;
    }
  }
  else if( __iter_10__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      // iter 0 : __iter_10__ 
      float __temp_a32__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*( __iter_10__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a36__ = (5 * __temp_a32__ + 12 * __temp_a35__);
      float __temp_a39__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a40__ = (__temp_a36__ + 15 * __temp_a39__);
      float __temp_a43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a44__ = (__temp_a40__ + 12 * __temp_a43__);
      float __temp_a47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a48__ = (__temp_a44__ + 5 * __temp_a47__);
      float __temp_a49__ = (__temp_a48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))] = __temp_a49__;
      // iter 1 : __iter_10__ + 1
      float __temp_b35__ = (__tilevar_3__[__iter_11__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b36__ = (5 * __temp_a39__ + 12 * __temp_b35__);
      float __temp_b40__ = (__temp_b36__ + 15 * __temp_a47__);
      float __temp_b43__ = (__tilevar_3__[ __iter_11__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b44__ = (__temp_b40__ + 12 * __temp_b43__);
      float __temp_b47__ = (__tilevar_3__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b48__ = (__temp_b44__ + 5 * __temp_b47__);
      float __temp_b49__ = (__temp_b48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b49__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) {
    int __iter_12__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.x) ; 
    if(__iter_12__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2))) { 
      int __iter_13__ = FORMA_MAX((__iter_0__+2),1);
      for(; __iter_13__ < (FORMA_MAX((__iter_0__+2),1)+2); __iter_13__++) {
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_12__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.x) ; 
    if(__iter_12__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2))) { 
      int __iter_13__ = (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-1);
      for(; __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ; __iter_13__++){
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_12__ = FORMA_MAX((__iter_1__-4),1);
    for(; __iter_12__ < FORMA_MAX((__iter_1__-2),1); __iter_12__++) {
      int __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        __tilevar_4__[__iter_13__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_12__ = FORMA_MIN(((__iter_1__+GAPY+2)),(N-2));
    for(; __iter_12__ < FORMA_MIN(((__iter_1__+GAPY+4)),(N-2)) ; __iter_12__++) {
      int __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        __tilevar_4__[__iter_13__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__-3),1) + 4*(int)(threadIdx.y) ;
  if( __iter_16__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      //iter 0 : __iter_16__
      float __temp_a60__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a62__ = (5 * __temp_a60__ + 12 * __temp_a61__);
      float __temp_a63__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a64__ = (__temp_a62__ + 15 * __temp_a63__);
      float __temp_a65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a66__ = (__temp_a64__ + 12 * __temp_a65__);
      float __temp_a67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a68__ = (__temp_a66__ + 5 * __temp_a67__);
      float __temp_a69__ = (__temp_a68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))] = __temp_a69__;
      // iter 1 : __iter_16__ + 1
      float __temp_b61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b62__ = (5 * __temp_a63__ + 12 * __temp_b61__);
      float __temp_b64__ = (__temp_b62__ + 15 * __temp_a67__);
      float __temp_b65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b66__ = (__temp_b64__ + 12 * __temp_b65__);
      float __temp_b67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b68__ = (__temp_b66__ + 5 * __temp_b67__);
      float __temp_b69__ = (__temp_b68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b69__;
      // iter 2 : __iter_16__ + 2
      float __temp_c61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c62__ = (5 * __temp_a67__ + 12 * __temp_c61__);
      float __temp_c64__ = (__temp_c62__ + 15 * __temp_b67__);
      float __temp_c65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c66__ = (__temp_c64__ + 12 * __temp_c65__);
      float __temp_c67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c68__ = (__temp_c66__ + 5 * __temp_c67__);
      float __temp_c69__ = (__temp_c68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(EXTENT-(__iter_1__+0)))] = __temp_c69__;
      // iter 3 : __iter_16__ + 3
      float __temp_d61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d62__ = (5 * __temp_b67__ + 12 * __temp_d61__);
      float __temp_d64__ = (__temp_d62__ + 15 * __temp_c67__);
      float __temp_d65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d66__ = (__temp_d64__ + 12 * __temp_d65__);
      float __temp_d67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(4)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d68__ = (__temp_d66__ + 5 * __temp_d67__);
      float __temp_d69__ = (__temp_d68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(EXTENT-(__iter_1__+0)))] = __temp_d69__;
    }
  }
  else if( __iter_16__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ;
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      //iter 0 : __iter_16__
      float __temp_a60__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a62__ = (5 * __temp_a60__ + 12 * __temp_a61__);
      float __temp_a63__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a64__ = (__temp_a62__ + 15 * __temp_a63__);
      float __temp_a65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a66__ = (__temp_a64__ + 12 * __temp_a65__);
      float __temp_a67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a68__ = (__temp_a66__ + 5 * __temp_a67__);
      float __temp_a69__ = (__temp_a68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))] = __temp_a69__;
      // iter 1 : __iter_16__ + 1
      float __temp_b61__ = (__tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b62__ = (5 * __temp_a63__ + 12 * __temp_b61__);
      float __temp_b64__ = (__temp_b62__ + 15 * __temp_a67__);
      float __temp_b65__ = (__tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b66__ = (__temp_b64__ + 12 * __temp_b65__);
      float __temp_b67__ = (__tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b68__ = (__temp_b66__ + 5 * __temp_b67__);
      float __temp_b69__ = (__temp_b68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b69__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) {
    int __iter_18__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.x) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2))) {
      int __iter_19__ = FORMA_MAX((__iter_0__+3),1);
      for(; __iter_19__ < (FORMA_MAX((__iter_0__+3),1)+2); __iter_19__++) {
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_18__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.x) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2))) {
      int __iter_19__ = (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-1);
      for(; __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ; __iter_19__++ ){
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))];
      }
    }
  }
  else if (threadIdx.y == 2) { 
    int __iter_18__ = FORMA_MAX((__iter_1__-5),1);
    for(; __iter_18__ < FORMA_MAX((__iter_1__-3),1); __iter_18__++) {
      int __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        __tilevar_5__[__iter_19__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  else if (threadIdx.y == 3) { 
    int __iter_18__ = FORMA_MIN(((__iter_1__+GAPY+3)),(N-2));
    for(; __iter_18__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(N-2)) ; __iter_18__++) {
      int __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
         __tilevar_5__[__iter_19__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__-4),1) + 4*(int)(threadIdx.y) ; 
  if( __iter_22__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
      // iter 0 : __iter_22__
      float __temp_a80__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a82__ = (5 * __temp_a80__ + 12 * __temp_a81__);
      float __temp_a83__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a84__ = (__temp_a82__ + 15 * __temp_a83__);
      float __temp_a85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a86__ = (__temp_a84__ + 12 * __temp_a85__);
      float __temp_a87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a88__ = (__temp_a86__ + 5 * __temp_a87__);
      float __temp_a89__ = (__temp_a88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_a89__;
      // iter 1 : __iter_22__ + 1
      float __temp_b81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b82__ = (5 * __temp_a83__ + 12 * __temp_b81__);
      float __temp_b84__ = (__temp_b82__ + 15 * __temp_a87__);
      float __temp_b85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b86__ = (__temp_b84__ + 12 * __temp_b85__);
      float __temp_b87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b88__ = (__temp_b86__ + 5 * __temp_b87__);
      float __temp_b89__ = (__temp_b88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+1)] = __temp_b89__;
      // iter 2 : __iter_22__ + 2
      float __temp_c81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c82__ = (5 * __temp_a87__ + 12 * __temp_c81__);
      float __temp_c84__ = (__temp_c82__ + 15 * __temp_b87__);
      float __temp_c85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c86__ = (__temp_c84__ + 12 * __temp_c85__);
      float __temp_c87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c88__ = (__temp_c86__ + 5 * __temp_c87__);
      float __temp_c89__ = (__temp_c88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+2)] = __temp_c89__;
      // iter 3 : __iter_22__ + 3
      float __temp_d81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d82__ = (5 * __temp_b87__ + 12 * __temp_d81__);
      float __temp_d84__ = (__temp_d82__ + 15 * __temp_c87__);
      float __temp_d85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d86__ = (__temp_d84__ + 12 * __temp_d85__);
      float __temp_d87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(4)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d88__ = (__temp_d86__ + 5 * __temp_d87__);
      float __temp_d89__ = (__temp_d88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+3)] = __temp_d89__;
    }
  }
  else if( __iter_22__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
      // iter 0 : __iter_22__
      float __temp_a80__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a82__ = (5 * __temp_a80__ + 12 * __temp_a81__);
      float __temp_a83__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a84__ = (__temp_a82__ + 15 * __temp_a83__);
      float __temp_a85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a86__ = (__temp_a84__ + 12 * __temp_a85__);
      float __temp_a87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a88__ = (__temp_a86__ + 5 * __temp_a87__);
      float __temp_a89__ = (__temp_a88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_a89__;
      // iter 1 : __iter_22__ + 1
      float __temp_b81__ = (__tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b82__ = (5 * __temp_a83__ + 12 * __temp_b81__);
      float __temp_b84__ = (__temp_b82__ + 15 * __temp_a87__);
      float __temp_b85__ = (__tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b86__ = (__temp_b84__ + 12 * __temp_b85__);
      float __temp_b87__ = (__tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b88__ = (__temp_b86__ + 5 * __temp_b87__);
      float __temp_b89__ = (__temp_b88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+1)] = __temp_b89__;
    }
  }
}

__global__ void __kernel___forma_kernel__3__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, float * __restrict__ __copy_arr_t0__, float * __restrict__ __copy_arr_t1__, float * __restrict__ __copy_arr_t2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X) + GAPX) + (int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y) + GAPY) + (int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__-2,0) + 4*(int)(threadIdx.y) ;
  if( __iter_2__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(1)+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+1)];
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(2)+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+2)];
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(3)+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+3)];
    }
  }
  else if( __iter_2__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(1)+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__+1)];
    }
  }
  __syncthreads ();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((__iter_1__-1),1) + 4*(int)(threadIdx.y) ; 
  if( __iter_4__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
      // iter 0 : __iter_4__
      float __temp_a2__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a6__ = (5 * __temp_a2__ + 12 * __temp_a5__);
      float __temp_a9__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a10__ = (__temp_a6__ + 15 * __temp_a9__);
      float __temp_a13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a14__ = (__temp_a10__ + 12 * __temp_a13__);
      float __temp_a17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a18__ = (__temp_a14__ + 5 * __temp_a17__);
      float __temp_a19__ = (__temp_a18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))] = __temp_a19__;
      // iter 1 : __iter_4__ + 1
      float __temp_b5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b6__ = (5 * __temp_a9__ + 12 * __temp_b5__);
      float __temp_b10__ = (__temp_b6__ + 15 * __temp_a17__);
      float __temp_b13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b14__ = (__temp_b10__ + 12 * __temp_b13__);
      float __temp_b17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b18__ = (__temp_b14__ + 5 * __temp_b17__);
      float __temp_b19__ = (__temp_b18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b19__;
      // iter 2 : __iter_4__ + 2
      float __temp_c5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c6__ = (5 * __temp_a17__ + 12 * __temp_c5__);
      float __temp_c10__ = (__temp_c6__ + 15 * __temp_b17__);
      float __temp_c13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c14__ = (__temp_c10__ + 12 * __temp_c13__);
      float __temp_c17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c18__ = (__temp_c14__ + 5 * __temp_c17__);
      float __temp_c19__ = (__temp_c18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(EXTENT-(__iter_1__+0)))] = __temp_c19__;
      // iter 3 : __iter_4__ + 3
      float __temp_d5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d6__ = (5 * __temp_b17__ + 12 * __temp_d5__);
      float __temp_d10__ = (__temp_d6__ + 15 * __temp_c17__);
      float __temp_d13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d14__ = (__temp_d10__ + 12 * __temp_d13__);
      float __temp_d17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(4)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d18__ = (__temp_d14__ + 5 * __temp_d17__);
      float __temp_d19__ = (__temp_d18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(3)+(EXTENT-(__iter_1__+0)))] = __temp_d19__;
    }
  }
  else if( __iter_4__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
      // iter 0 : __iter_4__
      float __temp_a2__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a6__ = (5 * __temp_a2__ + 12 * __temp_a5__);
      float __temp_a9__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a10__ = (__temp_a6__ + 15 * __temp_a9__);
      float __temp_a13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a14__ = (__temp_a10__ + 12 * __temp_a13__);
      float __temp_a17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a18__ = (__temp_a14__ + 5 * __temp_a17__);
      float __temp_a19__ = (__temp_a18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))] = __temp_a19__;
      // iter 1 : __iter_4__ + 1
      float __temp_b5__ = (__tilevar_2__[__iter_5__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b6__ = (5 * __temp_a9__ + 12 * __temp_b5__);
      float __temp_b10__ = (__temp_b6__ + 15 * __temp_a17__);
      float __temp_b13__ = (__tilevar_2__[__iter_5__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b14__ = (__temp_b10__ + 12 * __temp_b13__);
      float __temp_b17__ = (__tilevar_2__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b18__ = (__temp_b14__ + 5 * __temp_b17__);
      float __temp_b19__ = (__temp_b18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b19__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) {
    int __iter_6__ = FORMA_MAX((__iter_1__-3),1);
    for(; __iter_6__ < FORMA_MAX((__iter_1__-1),1); __iter_6__++) {
      int __iter_7__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
         __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_6__ = FORMA_MIN(((__iter_1__+GAPY+1)),(N-2));
    for(; __iter_6__ < FORMA_MIN(((__iter_1__+GAPY+3)),(N-2)) ; __iter_6__++) {
      int __iter_7__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
         __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_6__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.x) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2))) {
      int __iter_7__ = FORMA_MAX((__iter_0__-3),1);
      for(; __iter_7__ < FORMA_MAX((__iter_0__-1),1); __iter_7__++) {
        __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_6__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.x) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2))) {
      int __iter_7__ = FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2))+1;
      for(; __iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ; __iter_7__++){
        __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__-2),1) + 4*(int)(threadIdx.y) ;
  if( __iter_10__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
      // iter 0 : __iter_10__ 
      float __temp_a32__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*( __iter_10__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a35__ = (__tilevar_3__[__iter_11__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a36__ = (5 * __temp_a32__ + 12 * __temp_a35__);
      float __temp_a39__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a40__ = (__temp_a36__ + 15 * __temp_a39__);
      float __temp_a43__ = (__tilevar_3__[ __iter_11__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a44__ = (__temp_a40__ + 12 * __temp_a43__);
      float __temp_a47__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a48__ = (__temp_a44__ + 5 * __temp_a47__);
      float __temp_a49__ = (__temp_a48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))] = __temp_a49__;
      // iter 1 : __iter_10__ + 1
      float __temp_b35__ = (__tilevar_3__[__iter_11__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b36__ = (5 * __temp_a39__ + 12 * __temp_b35__);
      float __temp_b40__ = (__temp_b36__ + 15 * __temp_a47__);
      float __temp_b43__ = (__tilevar_3__[ __iter_11__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b44__ = (__temp_b40__ + 12 * __temp_b43__);
      float __temp_b47__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b48__ = (__temp_b44__ + 5 * __temp_b47__);
      float __temp_b49__ = (__temp_b48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b49__;
      // iter 2 : __iter_10__ + 2
      float __temp_c35__ = (__tilevar_3__[__iter_11__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c36__ = (5 * __temp_a47__ + 12 * __temp_c35__);
      float __temp_c40__ = (__temp_c36__ + 15 * __temp_b47__);
      float __temp_c43__ = (__tilevar_3__[ __iter_11__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c44__ = (__temp_c40__ + 12 * __temp_c43__);
      float __temp_c47__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c48__ = (__temp_c44__ + 5 * __temp_c47__);
      float __temp_c49__ = (__temp_c48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(EXTENT-(__iter_1__+0)))] = __temp_c49__;
      // iter 3 : __iter_10__ + 3
      float __temp_d35__ = (__tilevar_3__[__iter_11__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d36__ = (5 * __temp_b47__ + 12 * __temp_d35__);
      float __temp_d40__ = (__temp_d36__ + 15 * __temp_c47__);
      float __temp_d43__ = (__tilevar_3__[ __iter_11__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d44__ = (__temp_d40__ + 12 * __temp_d43__);
      float __temp_d47__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(4)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d48__ = (__temp_d44__ + 5 * __temp_d47__);
      float __temp_d49__ = (__temp_d48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(3)+(EXTENT-(__iter_1__+0)))] = __temp_d49__;
    }
  }
  else if( __iter_10__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
      // iter 0 : __iter_10__ 
      float __temp_a32__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*( __iter_10__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a35__ = (__tilevar_3__[__iter_11__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a36__ = (5 * __temp_a32__ + 12 * __temp_a35__);
      float __temp_a39__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a40__ = (__temp_a36__ + 15 * __temp_a39__);
      float __temp_a43__ = (__tilevar_3__[ __iter_11__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a44__ = (__temp_a40__ + 12 * __temp_a43__);
      float __temp_a47__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a48__ = (__temp_a44__ + 5 * __temp_a47__);
      float __temp_a49__ = (__temp_a48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))] = __temp_a49__;
      // iter 1 : __iter_10__ + 1
      float __temp_b35__ = (__tilevar_3__[__iter_11__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b36__ = (5 * __temp_a39__ + 12 * __temp_b35__);
      float __temp_b40__ = (__temp_b36__ + 15 * __temp_a47__);
      float __temp_b43__ = (__tilevar_3__[ __iter_11__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b44__ = (__temp_b40__ + 12 * __temp_b43__);
      float __temp_b47__ = (__tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b48__ = (__temp_b44__ + 5 * __temp_b47__);
      float __temp_b49__ = (__temp_b48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b49__;
    }
  }
  __syncthreads ();

  if (threadIdx.y == 0) {
    int __iter_12__ = FORMA_MAX((__iter_1__-4),1);
    for(; __iter_12__ < FORMA_MAX((__iter_1__-2),1); __iter_12__++) {
      int __iter_13__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
        __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  else if (threadIdx.y == 1) {
    int __iter_12__ = FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2))+1;
    for(; __iter_12__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2)) ; __iter_12__++) {
      int __iter_13__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
        __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_12__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.x) ; 
    if(__iter_12__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2))) {
      int __iter_13__ = FORMA_MAX((__iter_0__-4),1);
      for(; __iter_13__ < FORMA_MAX((__iter_0__-2),1); __iter_13__++) {
        __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_12__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.x) ; 
    if(__iter_12__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2))) {
      int __iter_13__ = FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2))+1;
      for(; __iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ; __iter_13__++){
        __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__-3),1) + 4*(int)(threadIdx.y) ; 
  if( __iter_16__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
      // iter 0 : __iter_16__
      float __temp_a60__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a62__ = (5 * __temp_a60__ + 12 * __temp_a61__);
      float __temp_a63__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a64__ = (__temp_a62__ + 15 * __temp_a63__);
      float __temp_a65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a66__ = (__temp_a64__ + 12 * __temp_a65__);
      float __temp_a67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a68__ = (__temp_a66__ + 5 * __temp_a67__);
      float __temp_a69__ = (__temp_a68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))] = __temp_a69__;
      // iter 1 : __iter_16__ + 1
      float __temp_b61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b62__ = (5 * __temp_a63__ + 12 * __temp_b61__);
      float __temp_b64__ = (__temp_b62__ + 15 * __temp_a67__);
      float __temp_b65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b66__ = (__temp_b64__ + 12 * __temp_b65__);
      float __temp_b67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b68__ = (__temp_b66__ + 5 * __temp_b67__);
      float __temp_b69__ = (__temp_b68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b69__;
      // iter 2 : __iter_16__ + 2
      float __temp_c61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c62__ = (5 * __temp_a67__ + 12 * __temp_c61__);
      float __temp_c64__ = (__temp_c62__ + 15 * __temp_b67__);
      float __temp_c65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c66__ = (__temp_c64__ + 12 * __temp_c65__);
      float __temp_c67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c68__ = (__temp_c66__ + 5 * __temp_c67__);
      float __temp_c69__ = (__temp_c68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(EXTENT-(__iter_1__+0)))] = __temp_c69__;
      // iter 3 : __iter_16__ + 3
      float __temp_d61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d62__ = (5 * __temp_b67__ + 12 * __temp_d61__);
      float __temp_d64__ = (__temp_d62__ + 15 * __temp_c67__);
      float __temp_d65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d66__ = (__temp_d64__ + 12 * __temp_d65__);
      float __temp_d67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(4)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d68__ = (__temp_d66__ + 5 * __temp_d67__);
      float __temp_d69__ = (__temp_d68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(3)+(EXTENT-(__iter_1__+0)))] = __temp_d69__;
    }
  }
  else if( __iter_16__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
      // iter 0 : __iter_16__
      float __temp_a60__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a62__ = (5 * __temp_a60__ + 12 * __temp_a61__);
      float __temp_a63__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a64__ = (__temp_a62__ + 15 * __temp_a63__);
      float __temp_a65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a66__ = (__temp_a64__ + 12 * __temp_a65__);
      float __temp_a67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a68__ = (__temp_a66__ + 5 * __temp_a67__);
      float __temp_a69__ = (__temp_a68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))] = __temp_a69__;
      // iter 1 : __iter_16__ + 1
      float __temp_b61__ = (__tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b62__ = (5 * __temp_a63__ + 12 * __temp_b61__);
      float __temp_b64__ = (__temp_b62__ + 15 * __temp_a67__);
      float __temp_b65__ = (__tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b66__ = (__temp_b64__ + 12 * __temp_b65__);
      float __temp_b67__ = (__tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b68__ = (__temp_b66__ + 5 * __temp_b67__);
      float __temp_b69__ = (__temp_b68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))] = __temp_b69__;
    }
  }
  __syncthreads ();
  if (threadIdx.y == 0) { 
    int __iter_18__ = FORMA_MAX((__iter_1__-5),1);
    for(; __iter_18__ < FORMA_MAX((__iter_1__-3),1); __iter_18__++) {
      int __iter_19__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(M-2)) ){
        __tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  else if (threadIdx.y == 1) { 
    int __iter_18__ = FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2))+1;
    for(; __iter_18__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(N-2)) ; __iter_18__++) {
      int __iter_19__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(M-2)) ){
        __tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  else if (threadIdx.y == 2) {
    int __iter_18__ = FORMA_MAX((__iter_1__-5),1) + (int)(threadIdx.x) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(N-2))) {
      int __iter_19__ = FORMA_MAX((__iter_0__-5),1);
      for(; __iter_19__ < FORMA_MAX((__iter_0__-3),1); __iter_19__++) {
        __tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  else if (threadIdx.y == 3) {
    int __iter_18__ = FORMA_MAX((__iter_1__-5),1) + (int)(threadIdx.x) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(N-2))) {
      int __iter_19__ = FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2))+1;
      for(; __iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(M-2)) ; __iter_19__++){
        __tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__-4),1) + 4*(int)(threadIdx.y) ; 
  if( __iter_22__ + 3 <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
      // iter 0 : __iter_22__ 
      float __temp_a80__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a81__ = (__tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a82__ = (5 * __temp_a80__ + 12 * __temp_a81__);
      float __temp_a83__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a84__ = (__temp_a82__ + 15 * __temp_a83__);
      float __temp_a85__ = (__tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a86__ = (__temp_a84__ + 12 * __temp_a85__);
      float __temp_a87__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a88__ = (__temp_a86__ + 5 * __temp_a87__);
      float __temp_a89__ = (__temp_a88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_a89__;
      // iter 1 : __iter_22__ + 1
      float __temp_b81__ = (__tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b82__ = (5 * __temp_a83__ + 12 * __temp_b81__);
      float __temp_b84__ = (__temp_b82__ + 15 * __temp_a87__);
      float __temp_b85__ = (__tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b86__ = (__temp_b84__ + 12 * __temp_b85__);
      float __temp_b87__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b88__ = (__temp_b86__ + 5 * __temp_b87__);
      float __temp_b89__ = (__temp_b88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+1)] = __temp_b89__;
      // iter 2 : __iter_22__ + 2
      float __temp_c81__ = (__tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c82__ = (5 * __temp_a87__ + 12 * __temp_c81__);
      float __temp_c84__ = (__temp_c82__ + 15 * __temp_b87__);
      float __temp_c85__ = (__tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c86__ = (__temp_c84__ + 12 * __temp_c85__);
      float __temp_c87__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_c88__ = (__temp_c86__ + 5 * __temp_c87__);
      float __temp_c89__ = (__temp_c88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+2)] = __temp_c89__;
      // iter 3 : __iter_22__ + 3
      float __temp_d81__ = (__tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d82__ = (5 * __temp_b87__ + 12 * __temp_d81__);
      float __temp_d84__ = (__temp_d82__ + 15 * __temp_c87__);
      float __temp_d85__ = (__tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(3)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d86__ = (__temp_d84__ + 12 * __temp_d85__);
      float __temp_d87__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(4)+(EXTENT-(__iter_1__+0)))]);
      float __temp_d88__ = (__temp_d86__ + 5 * __temp_d87__);
      float __temp_d89__ = (__temp_d88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+3)] = __temp_d89__;
    }
  }
  else if( __iter_22__ + 1 <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
      // iter 0 : __iter_22__ 
      float __temp_a80__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a81__ = (__tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a82__ = (5 * __temp_a80__ + 12 * __temp_a81__);
      float __temp_a83__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a84__ = (__temp_a82__ + 15 * __temp_a83__);
      float __temp_a85__ = (__tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_a86__ = (__temp_a84__ + 12 * __temp_a85__);
      float __temp_a87__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_a88__ = (__temp_a86__ + 5 * __temp_a87__);
      float __temp_a89__ = (__temp_a88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_a89__;
      // iter 1 : __iter_22__ + 1
      float __temp_b81__ = (__tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b82__ = (5 * __temp_a83__ + 12 * __temp_b81__);
      float __temp_b84__ = (__temp_b82__ + 15 * __temp_a87__);
      float __temp_b85__ = (__tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b86__ = (__temp_b84__ + 12 * __temp_b85__);
      float __temp_b87__ = (__tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(EXTENT-(__iter_1__+0)))]);
      float __temp_b88__ = (__temp_b86__ + 5 * __temp_b87__);
      float __temp_b89__ = (__temp_b88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__+1)] = __temp_b89__;
    }
  }
}


/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
  float * __copy_arr_t0__;
  hipMalloc(&__copy_arr_t0__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_t0__\n");
  float * __copy_arr_t1__;
  hipMalloc(&__copy_arr_t1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_t1__\n");
  float * __copy_arr_t2__;
  hipMalloc(&__copy_arr_t2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_t2__\n");

/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,9))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y+GAPY);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  dim3 unrollConfig(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/4);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __copy_arr_t0__, __copy_arr_t1__, __copy_arr_t2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __copy_arr_t0__, __copy_arr_t1__, __copy_arr_t2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");
  dim3 __blockConfig___kernel___forma_kernel__2__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y);
  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __copy_arr_t0__, __copy_arr_t1__, __copy_arr_t2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");
  dim3 __blockConfig___kernel___forma_kernel__3__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y);
  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __copy_arr_t0__, __copy_arr_t1__, __copy_arr_t2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(0-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      int __temp_0__;
      __temp_0__ = __iter_4__+(-1);
      int __temp_1__;
      __temp_1__ = __iter_5__;
      float __temp_2__;
      __temp_2__ = (5 * __tilevar_2__[__temp_1__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_0__+(0-(__iter_1__+0)))]);
      int __temp_3__;
      __temp_3__ = __iter_4__;
      int __temp_4__;
      __temp_4__ = __iter_5__+(-1);
      float __temp_5__;
      __temp_5__ = (12 * __tilevar_2__[__temp_4__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_3__+(0-(__iter_1__+0)))]);
      float __temp_6__;
      __temp_6__ = (__temp_2__ + __temp_5__);
      int __temp_7__;
      __temp_7__ = __iter_4__;
      int __temp_8__;
      __temp_8__ = __iter_5__;
      float __temp_9__;
      __temp_9__ = (15 * __tilevar_2__[__temp_8__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_7__+(0-(__iter_1__+0)))]);
      float __temp_10__;
      __temp_10__ = (__temp_6__ + __temp_9__);
      int __temp_11__;
      __temp_11__ = __iter_4__;
      int __temp_12__;
      __temp_12__ = __iter_5__+(1);
      float __temp_13__;
      __temp_13__ = (12 * __tilevar_2__[__temp_12__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_11__+(0-(__iter_1__+0)))]);
      float __temp_14__;
      __temp_14__ = (__temp_10__ + __temp_13__);
      int __temp_15__;
      __temp_15__ = __iter_4__+(1);
      int __temp_16__;
      __temp_16__ = __iter_5__;
      float __temp_17__;
      __temp_17__ = (5 * __tilevar_2__[__temp_16__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_15__+(0-(__iter_1__+0)))]);
      float __temp_18__;
      __temp_18__ = (__temp_14__ + __temp_17__);
      float __temp_19__;
      __temp_19__ = (__temp_18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+1)))] = __temp_19__;
    }
  }
  int __iter_6__;
  __iter_6__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      if (__iter_6__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_6__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))-2) || __iter_7__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_7__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-2)) {
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+1)))];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      int __temp_30__;
      __temp_30__ = __iter_10__+(-1);
      int __temp_31__;
      __temp_31__ = __iter_11__;
      float __temp_32__;
      __temp_32__ = (5 * __tilevar_3__[__temp_31__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_30__+(0-(__iter_1__+1)))]);
      int __temp_33__;
      __temp_33__ = __iter_10__;
      int __temp_34__;
      __temp_34__ = __iter_11__+(-1);
      float __temp_35__;
      __temp_35__ = (12 * __tilevar_3__[__temp_34__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_33__+(0-(__iter_1__+1)))]);
      float __temp_36__;
      __temp_36__ = (__temp_32__ + __temp_35__);
      int __temp_37__;
      __temp_37__ = __iter_10__;
      int __temp_38__;
      __temp_38__ = __iter_11__;
      float __temp_39__;
      __temp_39__ = (15 * __tilevar_3__[__temp_38__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_37__+(0-(__iter_1__+1)))]);
      float __temp_40__;
      __temp_40__ = (__temp_36__ + __temp_39__);
      int __temp_41__;
      __temp_41__ = __iter_10__;
      int __temp_42__;
      __temp_42__ = __iter_11__+(1);
      float __temp_43__;
      __temp_43__ = (12 * __tilevar_3__[__temp_42__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_41__+(0-(__iter_1__+1)))]);
      float __temp_44__;
      __temp_44__ = (__temp_40__ + __temp_43__);
      int __temp_45__;
      __temp_45__ = __iter_10__+(1);
      int __temp_46__;
      __temp_46__ = __iter_11__;
      float __temp_47__;
      __temp_47__ = (5 * __tilevar_3__[__temp_46__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_45__+(0-(__iter_1__+1)))]);
      float __temp_48__;
      __temp_48__ = (__temp_44__ + __temp_47__);
      float __temp_49__;
      __temp_49__ = (__temp_48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+2)))] = __temp_49__;
    }
  }
  int __iter_12__;
  __iter_12__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      if (__iter_12__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_12__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))-2) || __iter_13__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_13__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-2)) {
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+2)))];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
  if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      float __temp_60__;
      __temp_60__ = (5 * __tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+2)))]);
      float __temp_61__;
      __temp_61__ = (12 * __tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_62__;
      __temp_62__ = (__temp_60__ + __temp_61__);
      float __temp_63__;
      __temp_63__ = (15 * __tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_64__;
      __temp_64__ = (__temp_62__ + __temp_63__);
      float __temp_65__;
      __temp_65__ = (12 * __tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_66__;
      __temp_66__ = (__temp_64__ + __temp_65__);
      float __temp_67__;
      __temp_67__ = (5 * __tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_68__;
      __temp_68__ = (__temp_66__ + __temp_67__);
      float __temp_69__;
      __temp_69__ = (__temp_68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+3)))] = __temp_69__;
    }
  }
  int __iter_18__;
  __iter_18__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
  if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      if (__iter_18__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))-2) || __iter_19__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-2)) {
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+3)))];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__+4),1) + (int)(threadIdx.y) ; 
  if( __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
      float __temp_80__;
      __temp_80__ = (5 * __tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+3)))]);
      float __temp_81__;
      __temp_81__ = (12 * __tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_82__;
      __temp_82__ = (__temp_80__ + __temp_81__);
      float __temp_83__;
      __temp_83__ = (15 * __tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_84__;
      __temp_84__ = (__temp_82__ + __temp_83__);
      float __temp_85__;
      __temp_85__ = (12 * __tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_86__;
      __temp_86__ = (__temp_84__ + __temp_85__);
      float __temp_87__;
      __temp_87__ = (5 * __tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+3)))]);
      float __temp_88__;
      __temp_88__ = (__temp_86__ + __temp_87__);
      float __temp_89__;
      __temp_89__ = (__temp_88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_89__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __copy_arr_0__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_8__;
  __iter_8__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ;
  {
    int __iter_9__;
    __iter_9__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_9__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      {
        float __temp_20__;
        __temp_20__ = (5 * input[__iter_9__+(M-0)*(__iter_8__+(-1))]);
        float __temp_21__;
        __temp_21__ = (12 * input[__iter_9__+(-1)+(M-0)*(__iter_8__)]);
        float __temp_22__;
        __temp_22__ = (__temp_20__ + __temp_21__);
        float __temp_23__;
        __temp_23__ = (15 * input[__iter_9__+(M-0)*(__iter_8__)]);
        float __temp_24__;
        __temp_24__ = (__temp_22__ + __temp_23__);
        float __temp_25__;
        __temp_25__ = (12 * input[__iter_9__+(1)+(M-0)*(__iter_8__)]);
        float __temp_26__;
        __temp_26__ = (__temp_24__ + __temp_25__);
        float __temp_27__;
        __temp_27__ = (5 * input[__iter_9__+(M-0)*(__iter_8__+(1))]);
        float __temp_28__;
        __temp_28__ = (__temp_26__ + __temp_27__);
        float __temp_29__;
        __temp_29__ = (__temp_28__ / 118);
        __copy_arr_0__[__iter_9__+(M-0)*(__iter_8__)] = __temp_29__;
      }
    }
  }
  __iter_8__ = FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2));
  {
    int __iter_9__;
    __iter_9__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_9__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      {
        float __temp_20__;
        __temp_20__ = (5 * input[__iter_9__+(M-0)*(__iter_8__+(-1))]);
        float __temp_21__;
        __temp_21__ = (12 * input[__iter_9__+(-1)+(M-0)*(__iter_8__)]);
        float __temp_22__;
        __temp_22__ = (__temp_20__ + __temp_21__);
        float __temp_23__;
        __temp_23__ = (15 * input[__iter_9__+(M-0)*(__iter_8__)]);
        float __temp_24__;
        __temp_24__ = (__temp_22__ + __temp_23__);
        float __temp_25__;
        __temp_25__ = (12 * input[__iter_9__+(1)+(M-0)*(__iter_8__)]);
        float __temp_26__;
        __temp_26__ = (__temp_24__ + __temp_25__);
        float __temp_27__;
        __temp_27__ = (5 * input[__iter_9__+(M-0)*(__iter_8__+(1))]);
        float __temp_28__;
        __temp_28__ = (__temp_26__ + __temp_27__);
        float __temp_29__;
        __temp_29__ = (__temp_28__ / 118);
        __copy_arr_0__[__iter_9__+(M-0)*(__iter_8__)] = __temp_29__;
      }
    }
  }
}

__global__ void __kernel___forma_kernel__2__(float * __restrict__ __copy_arr_0__, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __copy_arr_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_14__;
  __iter_14__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
  {
    int __iter_15__;
    __iter_15__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_15__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      {
        float __temp_50__;
        __temp_50__ = (5 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(-1))]);
        float __temp_51__;
        __temp_51__ = (12 * __copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__)]);
        float __temp_52__;
        __temp_52__ = (__temp_50__ + __temp_51__);
        float __temp_53__;
        __temp_53__ = (15 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__)]);
        float __temp_54__;
        __temp_54__ = (__temp_52__ + __temp_53__);
        float __temp_55__;
        __temp_55__ = (12 * __copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__)]);
        float __temp_56__;
        __temp_56__ = (__temp_54__ + __temp_55__);
        float __temp_57__;
        __temp_57__ = (5 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(1))]);
        float __temp_58__;
        __temp_58__ = (__temp_56__ + __temp_57__);
        float __temp_59__;
        __temp_59__ = (__temp_58__ / 118);
        __copy_arr_1__[__iter_15__+(M-0)*(__iter_14__)] = __temp_59__;
      }
    }
  }
  __iter_14__ = FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) + (int)(threadIdx.y);
  {
    int __iter_15__;
    __iter_15__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_15__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      {
        float __temp_50__;
        __temp_50__ = (5 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(-1))]);
        float __temp_51__;
        __temp_51__ = (12 * __copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__)]);
        float __temp_52__;
        __temp_52__ = (__temp_50__ + __temp_51__);
        float __temp_53__;
        __temp_53__ = (15 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__)]);
        float __temp_54__;
        __temp_54__ = (__temp_52__ + __temp_53__);
        float __temp_55__;
        __temp_55__ = (12 * __copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__)]);
        float __temp_56__;
        __temp_56__ = (__temp_54__ + __temp_55__);
        float __temp_57__;
        __temp_57__ = (5 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(1))]);
        float __temp_58__;
        __temp_58__ = (__temp_56__ + __temp_57__);
        float __temp_59__;
        __temp_59__ = (__temp_58__ / 118);
        __copy_arr_1__[__iter_15__+(M-0)*(__iter_14__)] = __temp_59__;
      }
    }
  }
}

__global__ void __kernel___forma_kernel__3__(float * __restrict__ __copy_arr_1__, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __copy_arr_2__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_20__;
  __iter_20__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ;
  {
    int __iter_21__;
    __iter_21__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_21__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      {
        float __temp_70__;
        __temp_70__ = (5 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(-1))]);
        float __temp_71__;
        __temp_71__ = (12 * __copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__)]);
        float __temp_72__;
        __temp_72__ = (__temp_70__ + __temp_71__);
        float __temp_73__;
        __temp_73__ = (15 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__)]);
        float __temp_74__;
        __temp_74__ = (__temp_72__ + __temp_73__);
        float __temp_75__;
        __temp_75__ = (12 * __copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__)]);
        float __temp_76__;
        __temp_76__ = (__temp_74__ + __temp_75__);
        float __temp_77__;
        __temp_77__ = (5 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(1))]);
        float __temp_78__;
        __temp_78__ = (__temp_76__ + __temp_77__);
        float __temp_79__;
        __temp_79__ = (__temp_78__ / 118);
        __copy_arr_2__[__iter_21__+(M-0)*(__iter_20__)] = __temp_79__;
      }
    }
  }
  __iter_20__ = FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) + (int)(threadIdx.y) ;
  {
    int __iter_21__;
    __iter_21__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_21__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      {
        float __temp_70__;
        __temp_70__ = (5 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(-1))]);
        float __temp_71__;
        __temp_71__ = (12 * __copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__)]);
        float __temp_72__;
        __temp_72__ = (__temp_70__ + __temp_71__);
        float __temp_73__;
        __temp_73__ = (15 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__)]);
        float __temp_74__;
        __temp_74__ = (__temp_72__ + __temp_73__);
        float __temp_75__;
        __temp_75__ = (12 * __copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__)]);
        float __temp_76__;
        __temp_76__ = (__temp_74__ + __temp_75__);
        float __temp_77__;
        __temp_77__ = (5 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(1))]);
        float __temp_78__;
        __temp_78__ = (__temp_76__ + __temp_77__);
        float __temp_79__;
        __temp_79__ = (__temp_78__ / 118);
        __copy_arr_2__[__iter_21__+(M-0)*(__iter_20__)] = __temp_79__;
      }
    }
  }
}

__global__ void __kernel___forma_kernel__4__(float * __restrict__ __copy_arr_2__, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_24__;
  __iter_24__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ;
  {
    int __iter_25__;
    __iter_25__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_25__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      {
        float __temp_90__;
        __temp_90__ = (5 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(-1))]);
        float __temp_91__;
        __temp_91__ = (12 * __copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__)]);
        float __temp_92__;
        __temp_92__ = (__temp_90__ + __temp_91__);
        float __temp_93__;
        __temp_93__ = (15 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__)]);
        float __temp_94__;
        __temp_94__ = (__temp_92__ + __temp_93__);
        float __temp_95__;
        __temp_95__ = (12 * __copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__)]);
        float __temp_96__;
        __temp_96__ = (__temp_94__ + __temp_95__);
        float __temp_97__;
        __temp_97__ = (5 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(1))]);
        float __temp_98__;
        __temp_98__ = (__temp_96__ + __temp_97__);
        float __temp_99__;
        __temp_99__ = (__temp_98__ / 118);
        __var_1__[__iter_25__+(M-0)*(__iter_24__)] = __temp_99__;
      }
    }
  }
  __iter_24__ = FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) + (int)(threadIdx.y) ;
  {
    int __iter_25__;
    __iter_25__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_25__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      {
        float __temp_90__;
        __temp_90__ = (5 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(-1))]);
        float __temp_91__;
        __temp_91__ = (12 * __copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__)]);
        float __temp_92__;
        __temp_92__ = (__temp_90__ + __temp_91__);
        float __temp_93__;
        __temp_93__ = (15 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__)]);
        float __temp_94__;
        __temp_94__ = (__temp_92__ + __temp_93__);
        float __temp_95__;
        __temp_95__ = (12 * __copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__)]);
        float __temp_96__;
        __temp_96__ = (__temp_94__ + __temp_95__);
        float __temp_97__;
        __temp_97__ = (5 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(1))]);
        float __temp_98__;
        __temp_98__ = (__temp_96__ + __temp_97__);
        float __temp_99__;
        __temp_99__ = (__temp_98__ / 118);
        __var_1__[__iter_25__+(M-0)*(__iter_24__)] = __temp_99__;
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),7);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),7);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 7)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,7))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  dim3 __blockConfig___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__.x, 1);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__1__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __copy_arr_0__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  dim3 __blockConfig___kernel___forma_kernel__2__(__blockConfig___kernel___forma_kernel__0__.x, 2);
  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__2__>>> (__copy_arr_0__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __copy_arr_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");

  dim3 __blockConfig___kernel___forma_kernel__3__(__blockConfig___kernel___forma_kernel__0__.x, 3);
  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__3__>>> (__copy_arr_1__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __copy_arr_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  dim3 __blockConfig___kernel___forma_kernel__4__(__blockConfig___kernel___forma_kernel__0__.x, 4);
  __kernel___forma_kernel__4__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__4__>>> (__copy_arr_2__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__4__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

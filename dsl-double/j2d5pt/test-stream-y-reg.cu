#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define GAPX (118) 
#define EXTENT (5)

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}

template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}

void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float * __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_4__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_5__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);

  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X + GAPX);
  float t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f;
  float b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;

  // Initialize the value
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
     __tilevar_2__[__iter_3__-__iter_0__] = input[__iter_3__+M*0];
     t2 = input[__iter_3__+M*1];
  }
  // Rest of the computation
  for (int __iter_1__ = 1; __iter_1__ <= N-1; __iter_1__++) {
    if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
      b2 = __tilevar_2__[__iter_3__-__iter_0__];
      __tilevar_2__[__iter_3__-__iter_0__] = t2; 
      t2 = input[__iter_3__+M*(__iter_1__+1)];
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) &  __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      float __temp_2__ = b2;
      float __temp_5__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
      float __temp_6__ = (5 * __temp_2__ + 12 * __temp_5__);
      float __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__]);
      float __temp_10__ = (__temp_6__ + 15 * __temp_9__);
      float __temp_13__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
      float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
      float __temp_17__ = t2;
      float __temp_18__ = (__temp_14__ + 5 * __temp_17__);
      float __temp_19__ = (__temp_18__ / 118);
      b3 = __tilevar_3__[__iter_3__-__iter_0__];
      __tilevar_3__[__iter_3__-__iter_0__] = t3;
      t3 = __temp_19__;
      //printf ("var0[%d][%d] = %.6f\n", __iter_1__, __iter_3__, t3);
    }
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      if (__iter_3__ < (FORMA_MAX((__iter_0__+1),1)+2) | __iter_3__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-2)) {
        __copy_arr_0__[__iter_1__+(M)*(__iter_3__)] = t3;
      }
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      float __temp_32__ = b3;
      float __temp_35__ = (__tilevar_3__[__iter_3__+(-1)-__iter_0__]);
      float __temp_36__ = (5 * __temp_32__ + 12 * __temp_35__);
      float __temp_39__ = (__tilevar_3__[__iter_3__-__iter_0__]);
      float __temp_40__ = (__temp_36__ + 15 * __temp_39__);
      float __temp_43__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
      float __temp_44__ = (__temp_40__ + 12 * __temp_43__);
      float __temp_47__ = t3;
      float __temp_48__ = (__temp_44__ + 5 * __temp_47__);
      float __temp_49__ = (__temp_48__ / 118);
      b4 = __tilevar_4__[__iter_3__-__iter_0__];
      __tilevar_4__[__iter_3__-__iter_0__] = t4;
      t4 = __temp_49__;
      //printf ("var1[%d][%d] = %.6f\n", __iter_1__-1, __iter_3__, t4);
    }
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) &  __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      if (__iter_3__ < (FORMA_MAX((__iter_0__+2),1)+2) | __iter_3__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-2)) {
        __copy_arr_1__[__iter_1__+(M)*(__iter_3__)] = t4;
      }
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      float __temp_60__ = b4;
      float __temp_61__ = (__tilevar_4__[__iter_3__+(-1)-__iter_0__]);
      float __temp_62__ = (5 * __temp_60__ + 12 * __temp_61__);
      float __temp_63__ = (__tilevar_4__[__iter_3__-__iter_0__]);
      float __temp_64__ = (__temp_62__ + 15 * __temp_63__);
      float __temp_65__ = (__tilevar_4__[__iter_3__+1-__iter_0__]);
      float __temp_66__ = (__temp_64__ + 12 * __temp_65__);
      float __temp_67__ = t4;
      float __temp_68__ = (__temp_66__ + 5 * __temp_67__);
      float __temp_69__ = (__temp_68__ / 118);
      b5 = __tilevar_5__[__iter_3__-__iter_0__];
      __tilevar_5__[__iter_3__-__iter_0__] = t5;
      t5 = __temp_69__;
      //printf ("var2[%d][%d] = %.6f\n", __iter_1__-2, __iter_3__, t5);
    }
    if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      if (__iter_3__ < (FORMA_MAX((__iter_0__+3),1)+2) | __iter_3__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-2)) {
        __copy_arr_2__[__iter_1__+(M)*(__iter_3__)] = t5;
      }
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
      float __temp_80__ = b5;
      float __temp_81__ = (__tilevar_5__[__iter_3__+(-1)-__iter_0__]);
      float __temp_82__ = (5 * __temp_80__ + 12 * __temp_81__);
      float __temp_83__ = (__tilevar_5__[__iter_3__-__iter_0__]);
      float __temp_84__ = (__temp_82__ + 15 * __temp_83__);
      float __temp_85__ = (__tilevar_5__[__iter_3__+1-__iter_0__]);
      float __temp_86__ = (__temp_84__ + 12 * __temp_85__);
      float __temp_87__ = t5;
      float __temp_88__ = (__temp_86__ + 5 * __temp_87__);
      float __temp_89__ = (__temp_88__ / 118);
      __var_1__[__iter_3__+(M)*FORMA_MAX(__iter_1__-3,0)] = __temp_89__;
      //printf ("var3[%d][%d] = %.6f\n", __iter_1__-3, __iter_3__, __temp_89__);
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(4*FORMA_BLOCKDIM_X);
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float * __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_4__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_5__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
 
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X + GAPX) + (int)FORMA_BLOCKDIM_X;
  float t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f;
  float b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;

  // Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__-EXTENT,0) + (int)(threadIdx.x) ;
  if (__iter_3__ >= FORMA_MAX(__iter_0__-2,0) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1))){
    __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)] = input[__iter_3__+(M)*(0)];
    t2 = input[__iter_3__+(M)*(1)];
  }
  // Rest of the computation
  for (int __iter_1__ = 1; __iter_1__ <= N-1; __iter_1__++) {
    if(__iter_3__ >= FORMA_MAX(__iter_0__-2,0) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1))){
      b2 = __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)];
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)] = t2;
      t2 = input[__iter_3__+(M)*(__iter_1__+1)];
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__-1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
      float __temp_2__ = b2;
      float __temp_5__ = (__tilevar_2__[__iter_3__+(-1)+(EXTENT-__iter_0__)]);
      float __temp_6__ = (5 * __temp_2__ + 12 * __temp_5__);
      float __temp_9__ = (__tilevar_2__[__iter_3__+(EXTENT-__iter_0__)]);
      float __temp_10__ = (__temp_6__ + 15 * __temp_9__);
      float __temp_13__ = (__tilevar_2__[__iter_3__+1+(EXTENT-__iter_0__)]);
      float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
      float __temp_17__ = t2;
      float __temp_18__ = (__temp_14__ + 5 * __temp_17__);
      float __temp_19__ = (__temp_18__ / 118);
      b3 = __tilevar_3__[__iter_3__+(EXTENT-__iter_0__)];
      __tilevar_3__[__iter_3__+(EXTENT-__iter_0__)] = t3;
      t3 = __temp_19__;
      //printf ("var0[%d][%d] = %.6f\n", __iter_1__-0, __iter_3__, t3); 
    }
    if (__iter_3__ >= FORMA_MAX((__iter_0__-3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) & (__iter_3__ < FORMA_MAX((__iter_0__-1),1) | __iter_3__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)))) {
      b3 = __copy_arr_0__[__iter_1__-2+(M)*(__iter_3__)];
      __tilevar_3__[__iter_3__+(EXTENT-__iter_0__)] = __copy_arr_0__[__iter_1__-1+(M)*(__iter_3__)];
      t3 = __copy_arr_0__[__iter_1__+(M)*(__iter_3__)]; 
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__-2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
      float __temp_32__ = b3;
      float __temp_35__ = (__tilevar_3__[__iter_3__+(-1)+(EXTENT-__iter_0__)]);
      float __temp_36__ = (5 * __temp_32__ + 12 * __temp_35__);
      float __temp_39__ = (__tilevar_3__[__iter_3__+(EXTENT-__iter_0__)]);
      float __temp_40__ = (__temp_36__ + 15 * __temp_39__);
      float __temp_43__ = (__tilevar_3__[__iter_3__+1+(EXTENT-__iter_0__)]);
      float __temp_44__ = (__temp_40__ + 12 * __temp_43__);
      float __temp_47__ = t3;
      float __temp_48__ = (__temp_44__ + 5 * __temp_47__);
      float __temp_49__ = (__temp_48__ / 118);
      b4 = __tilevar_4__[__iter_3__+(EXTENT-__iter_0__)];
      __tilevar_4__[__iter_3__+(EXTENT-__iter_0__)] = t4;
      t4 = __temp_49__;
      //printf ("var1[%d][%d] = %.6f\n", __iter_1__-1, __iter_3__, t4);
    }
    if (__iter_3__ >= FORMA_MAX((__iter_0__-4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) & (__iter_3__ < FORMA_MAX((__iter_0__-2),1) | __iter_3__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)))) {
      b4 = __copy_arr_1__[__iter_1__-2+(M)*(__iter_3__)];
      __tilevar_4__[__iter_3__+(EXTENT-__iter_0__)] = __copy_arr_1__[__iter_1__-1+(M)*(__iter_3__)];
      t4 = __copy_arr_1__[__iter_1__+(M)*(__iter_3__)];
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__-3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
      float __temp_60__ = b4;
      float __temp_61__ = (__tilevar_4__[__iter_3__+(-1)+(EXTENT-__iter_0__)]);
      float __temp_62__ = (5 * __temp_60__ + 12 * __temp_61__);
      float __temp_63__ = (__tilevar_4__[__iter_3__+(EXTENT-__iter_0__)]);
      float __temp_64__ = (__temp_62__ + 15 * __temp_63__);
      float __temp_65__ = (__tilevar_4__[__iter_3__+1+(EXTENT-__iter_0__)]);
      float __temp_66__ = (__temp_64__ + 12 * __temp_65__);
      float __temp_67__ = t4;
      float __temp_68__ = (__temp_66__ + 5 * __temp_67__);
      float __temp_69__ = (__temp_68__ / 118);
      b5 = __tilevar_5__[__iter_3__+(EXTENT-__iter_0__)];
      __tilevar_5__[__iter_3__+(EXTENT-__iter_0__)] = t5;
      t5 = __temp_69__;
    }
    if (__iter_3__ >= FORMA_MAX((__iter_0__-5),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(M-2)) & (__iter_3__ < FORMA_MAX((__iter_0__-3),1) | __iter_3__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)))) {
      b5 = __copy_arr_2__[__iter_1__-2+(M)*(__iter_3__)];
      __tilevar_5__[__iter_3__+(EXTENT-__iter_0__)] = __copy_arr_2__[__iter_1__-1+(M)*(__iter_3__)];
      t5 = __copy_arr_2__[__iter_1__+(M)*(__iter_3__)];
    }
    __syncthreads();
    if( __iter_3__ >= FORMA_MAX((__iter_0__-4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
      float __temp_80__ = b5;
      float __temp_81__ = (__tilevar_5__[__iter_3__+(-1)+(EXTENT-__iter_0__)]);
      float __temp_82__ = (5 * __temp_80__ + 12 * __temp_81__);
      float __temp_83__ = (__tilevar_5__[__iter_3__+(EXTENT-__iter_0__)]);
      float __temp_84__ = (__temp_82__ + 15 * __temp_83__);
      float __temp_85__ = (__tilevar_5__[__iter_3__+1+(EXTENT-__iter_0__)]);
      float __temp_86__ = (__temp_84__ + 12 * __temp_85__);
      float __temp_87__ = t5;
      float __temp_88__ = (__temp_86__ + 5 * __temp_87__);
      float __temp_89__ = (__temp_88__ / 118);
      __var_1__[__iter_3__+(M)*FORMA_MAX(__iter_1__-3,0)] = __temp_89__;
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N)*(M)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 128;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

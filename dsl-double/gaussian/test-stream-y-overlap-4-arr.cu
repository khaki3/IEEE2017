#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  __shared__ float __tilevar_0__[5][128];
  __shared__ float __tilevar_1__[5][128];
  __shared__ float __tilevar_2__[5][128];
  __shared__ float __tilevar_3__[5][128];
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X-16);

  //Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){ 
    __tilevar_0__[1][__iter_3__-__iter_0__] = input[__iter_3__+M*0];
    __tilevar_0__[2][__iter_3__-__iter_0__] = input[__iter_3__+M*1];
    __tilevar_0__[3][__iter_3__-__iter_0__] = input[__iter_3__+M*2];
    __tilevar_0__[4][__iter_3__-__iter_0__] = input[__iter_3__+M*3];
  }
  // Rest of the computation
  for (int __iter_1__ = 2; __iter_1__ < N-2; __iter_1__++) {
    if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
	__tilevar_0__[0][__iter_3__-__iter_0__] = __tilevar_0__[1][__iter_3__-__iter_0__];
      	__tilevar_0__[1][__iter_3__-__iter_0__] = __tilevar_0__[2][__iter_3__-__iter_0__]; 
      	__tilevar_0__[2][__iter_3__-__iter_0__] = __tilevar_0__[3][__iter_3__-__iter_0__];
      	__tilevar_0__[3][__iter_3__-__iter_0__] = __tilevar_0__[4][__iter_3__-__iter_0__];
	__tilevar_0__[4][__iter_3__-__iter_0__] = input[__iter_3__+M*(__iter_1__+2)];
    }
    __syncthreads ();
    if (__iter_3__ >= FORMA_MAX((__iter_0__+2),2) &  __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-3)) ){
        float __temp_2__ = (__tilevar_0__[0][__iter_3__-2-__iter_0__]);
        float __temp_5__ = (__tilevar_0__[0][__iter_3__-1-__iter_0__]);
        float __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        float __temp_9__ = (__tilevar_0__[0][__iter_3__-__iter_0__]);
        float __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        float __temp_13__ = (__tilevar_0__[0][__iter_3__+1-__iter_0__]);
        float __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        float __temp_17__ = (__tilevar_0__[0][__iter_3__+2-__iter_0__]);
        float __temp_18__ = (__temp_14__ + 2 * __temp_17__);
        float __temp_21__ = (__tilevar_0__[1][__iter_3__-2-__iter_0__]);
        float __temp_22__ = (__temp_18__ + 4 * __temp_21__);
        float __temp_25__ = (__tilevar_0__[1][__iter_3__-1-__iter_0__]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_0__[1][__iter_3__-__iter_0__]);
        float __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        float __temp_33__ = (__tilevar_0__[1][__iter_3__+1-__iter_0__]);
        float __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        float __temp_37__ = (__tilevar_0__[1][__iter_3__+2-__iter_0__]);
        float __temp_38__ = (__temp_34__ + 4 * __temp_37__);
        float __temp_41__ = (__tilevar_0__[2][__iter_3__-2-__iter_0__]);
        float __temp_42__ = (__temp_38__ + 5 * __temp_41__);
        float __temp_45__ = (__tilevar_0__[2][__iter_3__-1-__iter_0__]);
        float __temp_46__ = (__temp_42__ + 12 * __temp_45__);
        float __temp_49__ = (__tilevar_0__[2][__iter_3__-__iter_0__]);
        float __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        float __temp_53__ = (__tilevar_0__[2][__iter_3__+1-__iter_0__]);
        float __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        float __temp_57__ = (__tilevar_0__[2][__iter_3__+2-__iter_0__]);
        float __temp_58__ = (__temp_54__ + 5 * __temp_57__);
        float __temp_61__ = (__tilevar_0__[3][__iter_3__-2-__iter_0__]);
        float __temp_62__ = (__temp_58__ + 4 * __temp_61__);
        float __temp_65__ = (__tilevar_0__[3][__iter_3__-1-__iter_0__]);
        float __temp_66__ = (__temp_62__ + 9 * __temp_65__);
        float __temp_69__ = (__tilevar_0__[3][__iter_3__-__iter_0__]);
        float __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        float __temp_73__ = (__tilevar_0__[3][__iter_3__+1-__iter_0__]);
        float __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        float __temp_77__ = (__tilevar_0__[3][__iter_3__+2-__iter_0__]);
        float __temp_78__ = (__temp_74__ + 4 * __temp_77__);
        float __temp_81__ = (__tilevar_0__[4][__iter_3__-2-__iter_0__]);
        float __temp_82__ = (__temp_78__ + 2 * __temp_81__);
        float __temp_85__ = (__tilevar_0__[4][__iter_3__-1-__iter_0__]);
        float __temp_86__ = (__temp_82__ + 4 * __temp_85__);
        float __temp_89__ = (__tilevar_0__[4][__iter_3__-__iter_0__]);
        float __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        float __temp_93__ = (__tilevar_0__[4][__iter_3__+1-__iter_0__]);
        float __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        float __temp_97__ = (__tilevar_0__[4][__iter_3__+2-__iter_0__]);
        float __temp_98__ = (__temp_94__ + 2 * __temp_97__);
        float __temp_99__ = (__temp_98__ / 159);
	__tilevar_1__[0][__iter_3__-__iter_0__] = __tilevar_1__[1][__iter_3__-__iter_0__];
	__tilevar_1__[1][__iter_3__-__iter_0__] = __tilevar_1__[2][__iter_3__-__iter_0__];   
        __tilevar_1__[2][__iter_3__-__iter_0__] = __tilevar_1__[3][__iter_3__-__iter_0__];
	__tilevar_1__[3][__iter_3__-__iter_0__] = __tilevar_1__[4][__iter_3__-__iter_0__];
	__tilevar_1__[4][__iter_3__-__iter_0__] = __temp_99__;
    }
    __syncthreads();
    if (__iter_3__ >= FORMA_MAX((__iter_0__+4),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-3))) {
        float __temp_2__ = (__tilevar_1__[0][__iter_3__-2-__iter_0__]);
        float __temp_5__ = (__tilevar_1__[0][__iter_3__-1-__iter_0__]);
        float __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        float __temp_9__ = (__tilevar_1__[0][__iter_3__-__iter_0__]);
        float __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        float __temp_13__ = (__tilevar_1__[0][__iter_3__+1-__iter_0__]);
        float __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        float __temp_17__ = (__tilevar_1__[0][__iter_3__+2-__iter_0__]);
        float __temp_18__ = (__temp_14__ + 2 * __temp_17__);
        float __temp_21__ = (__tilevar_1__[1][__iter_3__-2-__iter_0__]);
        float __temp_22__ = (__temp_18__ + 4 * __temp_21__);
        float __temp_25__ = (__tilevar_1__[1][__iter_3__-1-__iter_0__]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_1__[1][__iter_3__-__iter_0__]);
        float __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        float __temp_33__ = (__tilevar_1__[1][__iter_3__+1-__iter_0__]);
        float __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        float __temp_37__ = (__tilevar_1__[1][__iter_3__+2-__iter_0__]);
        float __temp_38__ = (__temp_34__ + 4 * __temp_37__);
        float __temp_41__ = (__tilevar_1__[2][__iter_3__-2-__iter_0__]);
        float __temp_42__ = (__temp_38__ + 5 * __temp_41__);
        float __temp_45__ = (__tilevar_1__[2][__iter_3__-1-__iter_0__]);
        float __temp_46__ = (__temp_42__ + 12 * __temp_45__);
        float __temp_49__ = (__tilevar_1__[2][__iter_3__-__iter_0__]);
        float __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        float __temp_53__ = (__tilevar_1__[2][__iter_3__+1-__iter_0__]);
        float __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        float __temp_57__ = (__tilevar_1__[2][__iter_3__+2-__iter_0__]);
        float __temp_58__ = (__temp_54__ + 5 * __temp_57__);
        float __temp_61__ = (__tilevar_1__[3][__iter_3__-2-__iter_0__]);
        float __temp_62__ = (__temp_58__ + 4 * __temp_61__);
        float __temp_65__ = (__tilevar_1__[3][__iter_3__-1-__iter_0__]);
        float __temp_66__ = (__temp_62__ + 9 * __temp_65__);
        float __temp_69__ = (__tilevar_1__[3][__iter_3__-__iter_0__]);
        float __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        float __temp_73__ = (__tilevar_1__[3][__iter_3__+1-__iter_0__]);
        float __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        float __temp_77__ = (__tilevar_1__[3][__iter_3__+2-__iter_0__]);
        float __temp_78__ = (__temp_74__ + 4 * __temp_77__);
        float __temp_81__ = (__tilevar_1__[4][__iter_3__-2-__iter_0__]);
        float __temp_82__ = (__temp_78__ + 2 * __temp_81__);
        float __temp_85__ = (__tilevar_1__[4][__iter_3__-1-__iter_0__]);
        float __temp_86__ = (__temp_82__ + 4 * __temp_85__);
        float __temp_89__ = (__tilevar_1__[4][__iter_3__-__iter_0__]);
        float __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        float __temp_93__ = (__tilevar_1__[4][__iter_3__+1-__iter_0__]);
        float __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        float __temp_97__ = (__tilevar_1__[4][__iter_3__+2-__iter_0__]);
        float __temp_98__ = (__temp_94__ + 2 * __temp_97__);
        float __temp_99__ = (__temp_98__ / 159);
	__tilevar_2__[0][__iter_3__-__iter_0__] = __tilevar_2__[1][__iter_3__-__iter_0__];
	__tilevar_2__[1][__iter_3__-__iter_0__] = __tilevar_2__[2][__iter_3__-__iter_0__];   
        __tilevar_2__[2][__iter_3__-__iter_0__] = __tilevar_2__[3][__iter_3__-__iter_0__];
	__tilevar_2__[3][__iter_3__-__iter_0__] = __tilevar_2__[4][__iter_3__-__iter_0__];
	__tilevar_2__[4][__iter_3__-__iter_0__] = __temp_99__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+6),2) &  __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(M-3))) {
        float __temp_2__ = (__tilevar_2__[0][__iter_3__-2-__iter_0__]);
        float __temp_5__ = (__tilevar_2__[0][__iter_3__-1-__iter_0__]);
        float __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        float __temp_9__ = (__tilevar_2__[0][__iter_3__-__iter_0__]);
        float __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        float __temp_13__ = (__tilevar_2__[0][__iter_3__+1-__iter_0__]);
        float __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        float __temp_17__ = (__tilevar_2__[0][__iter_3__+2-__iter_0__]);
        float __temp_18__ = (__temp_14__ + 2 * __temp_17__);
        float __temp_21__ = (__tilevar_2__[1][__iter_3__-2-__iter_0__]);
        float __temp_22__ = (__temp_18__ + 4 * __temp_21__);
        float __temp_25__ = (__tilevar_2__[1][__iter_3__-1-__iter_0__]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_2__[1][__iter_3__-__iter_0__]);
        float __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        float __temp_33__ = (__tilevar_2__[1][__iter_3__+1-__iter_0__]);
        float __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        float __temp_37__ = (__tilevar_2__[1][__iter_3__+2-__iter_0__]);
        float __temp_38__ = (__temp_34__ + 4 * __temp_37__);
        float __temp_41__ = (__tilevar_2__[2][__iter_3__-2-__iter_0__]);
        float __temp_42__ = (__temp_38__ + 5 * __temp_41__);
        float __temp_45__ = (__tilevar_2__[2][__iter_3__-1-__iter_0__]);
        float __temp_46__ = (__temp_42__ + 12 * __temp_45__);
        float __temp_49__ = (__tilevar_2__[2][__iter_3__-__iter_0__]);
        float __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        float __temp_53__ = (__tilevar_2__[2][__iter_3__+1-__iter_0__]);
        float __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        float __temp_57__ = (__tilevar_2__[2][__iter_3__+2-__iter_0__]);
        float __temp_58__ = (__temp_54__ + 5 * __temp_57__);
        float __temp_61__ = (__tilevar_2__[3][__iter_3__-2-__iter_0__]);
        float __temp_62__ = (__temp_58__ + 4 * __temp_61__);
        float __temp_65__ = (__tilevar_2__[3][__iter_3__-1-__iter_0__]);
        float __temp_66__ = (__temp_62__ + 9 * __temp_65__);
        float __temp_69__ = (__tilevar_2__[3][__iter_3__-__iter_0__]);
        float __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        float __temp_73__ = (__tilevar_2__[3][__iter_3__+1-__iter_0__]);
        float __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        float __temp_77__ = (__tilevar_2__[3][__iter_3__+2-__iter_0__]);
        float __temp_78__ = (__temp_74__ + 4 * __temp_77__);
        float __temp_81__ = (__tilevar_2__[4][__iter_3__-2-__iter_0__]);
        float __temp_82__ = (__temp_78__ + 2 * __temp_81__);
        float __temp_85__ = (__tilevar_2__[4][__iter_3__-1-__iter_0__]);
        float __temp_86__ = (__temp_82__ + 4 * __temp_85__);
        float __temp_89__ = (__tilevar_2__[4][__iter_3__-__iter_0__]);
        float __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        float __temp_93__ = (__tilevar_2__[4][__iter_3__+1-__iter_0__]);
        float __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        float __temp_97__ = (__tilevar_2__[4][__iter_3__+2-__iter_0__]);
        float __temp_98__ = (__temp_94__ + 2 * __temp_97__);
        float __temp_99__ = (__temp_98__ / 159);
	__tilevar_3__[0][__iter_3__-__iter_0__] = __tilevar_3__[1][__iter_3__-__iter_0__];
	__tilevar_3__[1][__iter_3__-__iter_0__] = __tilevar_3__[2][__iter_3__-__iter_0__];   
        __tilevar_3__[2][__iter_3__-__iter_0__] = __tilevar_3__[3][__iter_3__-__iter_0__];
	__tilevar_3__[3][__iter_3__-__iter_0__] = __tilevar_3__[4][__iter_3__-__iter_0__];
	__tilevar_3__[4][__iter_3__-__iter_0__] = __temp_99__;
    }
    __syncthreads();
    if (__iter_3__ >= FORMA_MAX((__iter_0__+8),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-9),(M-3))){
        float __temp_2__ = (__tilevar_3__[0][__iter_3__-2-__iter_0__]);
        float __temp_5__ = (__tilevar_3__[0][__iter_3__-1-__iter_0__]);
        float __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        float __temp_9__ = (__tilevar_3__[0][__iter_3__-__iter_0__]);
        float __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        float __temp_13__ = (__tilevar_3__[0][__iter_3__+1-__iter_0__]);
        float __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        float __temp_17__ = (__tilevar_3__[0][__iter_3__+2-__iter_0__]);
        float __temp_18__ = (__temp_14__ + 2 * __temp_17__);
        float __temp_21__ = (__tilevar_3__[1][__iter_3__-2-__iter_0__]);
        float __temp_22__ = (__temp_18__ + 4 * __temp_21__);
        float __temp_25__ = (__tilevar_3__[1][__iter_3__-1-__iter_0__]);
        float __temp_26__ = (__temp_22__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_3__[1][__iter_3__-__iter_0__]);
        float __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        float __temp_33__ = (__tilevar_3__[1][__iter_3__+1-__iter_0__]);
        float __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        float __temp_37__ = (__tilevar_3__[1][__iter_3__+2-__iter_0__]);
        float __temp_38__ = (__temp_34__ + 4 * __temp_37__);
        float __temp_41__ = (__tilevar_3__[2][__iter_3__-2-__iter_0__]);
        float __temp_42__ = (__temp_38__ + 5 * __temp_41__);
        float __temp_45__ = (__tilevar_3__[2][__iter_3__-1-__iter_0__]);
        float __temp_46__ = (__temp_42__ + 12 * __temp_45__);
        float __temp_49__ = (__tilevar_3__[2][__iter_3__-__iter_0__]);
        float __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        float __temp_53__ = (__tilevar_3__[2][__iter_3__+1-__iter_0__]);
        float __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        float __temp_57__ = (__tilevar_3__[2][__iter_3__+2-__iter_0__]);
        float __temp_58__ = (__temp_54__ + 5 * __temp_57__);
        float __temp_61__ = (__tilevar_3__[3][__iter_3__-2-__iter_0__]);
        float __temp_62__ = (__temp_58__ + 4 * __temp_61__);
        float __temp_65__ = (__tilevar_3__[3][__iter_3__-1-__iter_0__]);
        float __temp_66__ = (__temp_62__ + 9 * __temp_65__);
        float __temp_69__ = (__tilevar_3__[3][__iter_3__-__iter_0__]);
        float __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        float __temp_73__ = (__tilevar_3__[3][__iter_3__+1-__iter_0__]);
        float __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        float __temp_77__ = (__tilevar_3__[3][__iter_3__+2-__iter_0__]);
        float __temp_78__ = (__temp_74__ + 4 * __temp_77__);
        float __temp_81__ = (__tilevar_3__[4][__iter_3__-2-__iter_0__]);
        float __temp_82__ = (__temp_78__ + 2 * __temp_81__);
        float __temp_85__ = (__tilevar_3__[4][__iter_3__-1-__iter_0__]);
        float __temp_86__ = (__temp_82__ + 4 * __temp_85__);
        float __temp_89__ = (__tilevar_3__[4][__iter_3__-__iter_0__]);
        float __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        float __temp_93__ = (__tilevar_3__[4][__iter_3__+1-__iter_0__]);
        float __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        float __temp_97__ = (__tilevar_3__[4][__iter_3__+2-__iter_0__]);
        float __temp_98__ = (__temp_94__ + 2 * __temp_97__);
        float __temp_99__ = (__temp_98__ / 159);
        __var_1__[__iter_3__+M*FORMA_MAX(__iter_1__-6,0)] = __temp_99__;
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void gaussian(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*(N*M));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*(N*M), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*(N*M));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 128;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-16);
  int __grid_1___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*(N*M), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

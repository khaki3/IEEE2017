#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"
#include <nvml.h>
#include <assert.h>


#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
	template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
	int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
	input[loc] = value;
}


	template<typename T>
void initialize_array(T* d_input, int size, T value)
{
	dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
	dim3 init_block(FORMA_MAX_BLOCKDIM_0);
	__kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int N, int M, double * __restrict__ __var_4__){
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_0__;
	__iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 2;
	if(__iter_0__ <= (M-3)){
		int __iter_1__;
		__iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 2;
		if(__iter_1__ <= (N-3)){
			double __temp_0__;
			__temp_0__ = (2 * input[__iter_0__+(-2)+(M-0)*(__iter_1__+(-2))]);
			double __temp_1__;
			__temp_1__ = (4 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(-2))]);
			double __temp_2__;
			__temp_2__ = (__temp_0__ + __temp_1__);
			double __temp_3__;
			__temp_3__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(-2))]);
			double __temp_4__;
			__temp_4__ = (__temp_2__ + __temp_3__);
			double __temp_5__;
			__temp_5__ = (4 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(-2))]);
			double __temp_6__;
			__temp_6__ = (__temp_4__ + __temp_5__);
			double __temp_7__;
			__temp_7__ = (2 * input[__iter_0__+(2)+(M-0)*(__iter_1__+(-2))]);
			double __temp_8__;
			__temp_8__ = (__temp_6__ + __temp_7__);
			double __temp_9__;
			__temp_9__ = (4 * input[__iter_0__+(-2)+(M-0)*(__iter_1__+(-1))]);
			double __temp_10__;
			__temp_10__ = (__temp_8__ + __temp_9__);
			double __temp_11__;
			__temp_11__ = (9 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(-1))]);
			double __temp_12__;
			__temp_12__ = (__temp_10__ + __temp_11__);
			double __temp_13__;
			__temp_13__ = (12 * input[__iter_0__+(M-0)*(__iter_1__+(-1))]);
			double __temp_14__;
			__temp_14__ = (__temp_12__ + __temp_13__);
			double __temp_15__;
			__temp_15__ = (9 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(-1))]);
			double __temp_16__;
			__temp_16__ = (__temp_14__ + __temp_15__);
			double __temp_17__;
			__temp_17__ = (4 * input[__iter_0__+(2)+(M-0)*(__iter_1__+(-1))]);
			double __temp_18__;
			__temp_18__ = (__temp_16__ + __temp_17__);
			double __temp_19__;
			__temp_19__ = (5 * input[__iter_0__+(-2)+(M-0)*(__iter_1__)]);
			double __temp_20__;
			__temp_20__ = (__temp_18__ + __temp_19__);
			double __temp_21__;
			__temp_21__ = (12 * input[__iter_0__+(-1)+(M-0)*(__iter_1__)]);
			double __temp_22__;
			__temp_22__ = (__temp_20__ + __temp_21__);
			double __temp_23__;
			__temp_23__ = (15 * input[__iter_0__+(M-0)*(__iter_1__)]);
			double __temp_24__;
			__temp_24__ = (__temp_22__ + __temp_23__);
			double __temp_25__;
			__temp_25__ = (12 * input[__iter_0__+(1)+(M-0)*(__iter_1__)]);
			double __temp_26__;
			__temp_26__ = (__temp_24__ + __temp_25__);
			double __temp_27__;
			__temp_27__ = (5 * input[__iter_0__+(2)+(M-0)*(__iter_1__)]);
			double __temp_28__;
			__temp_28__ = (__temp_26__ + __temp_27__);
			double __temp_29__;
			__temp_29__ = (4 * input[__iter_0__+(-2)+(M-0)*(__iter_1__+(1))]);
			double __temp_30__;
			__temp_30__ = (__temp_28__ + __temp_29__);
			double __temp_31__;
			__temp_31__ = (9 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(1))]);
			double __temp_32__;
			__temp_32__ = (__temp_30__ + __temp_31__);
			double __temp_33__;
			__temp_33__ = (12 * input[__iter_0__+(M-0)*(__iter_1__+(1))]);
			double __temp_34__;
			__temp_34__ = (__temp_32__ + __temp_33__);
			double __temp_35__;
			__temp_35__ = (9 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(1))]);
			double __temp_36__;
			__temp_36__ = (__temp_34__ + __temp_35__);
			double __temp_37__;
			__temp_37__ = (4 * input[__iter_0__+(2)+(M-0)*(__iter_1__+(1))]);
			double __temp_38__;
			__temp_38__ = (__temp_36__ + __temp_37__);
			double __temp_39__;
			__temp_39__ = (2 * input[__iter_0__+(-2)+(M-0)*(__iter_1__+(2))]);
			double __temp_40__;
			__temp_40__ = (__temp_38__ + __temp_39__);
			double __temp_41__;
			__temp_41__ = (4 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(2))]);
			double __temp_42__;
			__temp_42__ = (__temp_40__ + __temp_41__);
			double __temp_43__;
			__temp_43__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(2))]);
			double __temp_44__;
			__temp_44__ = (__temp_42__ + __temp_43__);
			double __temp_45__;
			__temp_45__ = (4 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(2))]);
			double __temp_46__;
			__temp_46__ = (__temp_44__ + __temp_45__);
			double __temp_47__;
			__temp_47__ = (2 * input[__iter_0__+(2)+(M-0)*(__iter_1__+(2))]);
			double __temp_48__;
			__temp_48__ = (__temp_46__ + __temp_47__);
			double __temp_49__;
			__temp_49__ = (__temp_48__ / 159);
			__var_4__[__iter_0__+(M-0)*(__iter_1__)] = __temp_49__;
		}
	}
}
__global__ void __kernel___forma_kernel__1__(double * __restrict__ __var_4__, int N, int M, double * __restrict__ __var_3__){
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_2__;
	__iter_2__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 2;
	if(__iter_2__ <= (M-3)){
		int __iter_3__;
		__iter_3__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 2;
		if(__iter_3__ <= (N-3)){
			double __temp_50__;
			__temp_50__ = (2 * __var_4__[__iter_2__+(-2)+(M-0)*(__iter_3__+(-2))]);
			double __temp_51__;
			__temp_51__ = (4 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(-2))]);
			double __temp_52__;
			__temp_52__ = (__temp_50__ + __temp_51__);
			double __temp_53__;
			__temp_53__ = (5 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(-2))]);
			double __temp_54__;
			__temp_54__ = (__temp_52__ + __temp_53__);
			double __temp_55__;
			__temp_55__ = (4 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(-2))]);
			double __temp_56__;
			__temp_56__ = (__temp_54__ + __temp_55__);
			double __temp_57__;
			__temp_57__ = (2 * __var_4__[__iter_2__+(2)+(M-0)*(__iter_3__+(-2))]);
			double __temp_58__;
			__temp_58__ = (__temp_56__ + __temp_57__);
			double __temp_59__;
			__temp_59__ = (4 * __var_4__[__iter_2__+(-2)+(M-0)*(__iter_3__+(-1))]);
			double __temp_60__;
			__temp_60__ = (__temp_58__ + __temp_59__);
			double __temp_61__;
			__temp_61__ = (9 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(-1))]);
			double __temp_62__;
			__temp_62__ = (__temp_60__ + __temp_61__);
			double __temp_63__;
			__temp_63__ = (12 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(-1))]);
			double __temp_64__;
			__temp_64__ = (__temp_62__ + __temp_63__);
			double __temp_65__;
			__temp_65__ = (9 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(-1))]);
			double __temp_66__;
			__temp_66__ = (__temp_64__ + __temp_65__);
			double __temp_67__;
			__temp_67__ = (4 * __var_4__[__iter_2__+(2)+(M-0)*(__iter_3__+(-1))]);
			double __temp_68__;
			__temp_68__ = (__temp_66__ + __temp_67__);
			double __temp_69__;
			__temp_69__ = (5 * __var_4__[__iter_2__+(-2)+(M-0)*(__iter_3__)]);
			double __temp_70__;
			__temp_70__ = (__temp_68__ + __temp_69__);
			double __temp_71__;
			__temp_71__ = (12 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__)]);
			double __temp_72__;
			__temp_72__ = (__temp_70__ + __temp_71__);
			double __temp_73__;
			__temp_73__ = (15 * __var_4__[__iter_2__+(M-0)*(__iter_3__)]);
			double __temp_74__;
			__temp_74__ = (__temp_72__ + __temp_73__);
			double __temp_75__;
			__temp_75__ = (12 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__)]);
			double __temp_76__;
			__temp_76__ = (__temp_74__ + __temp_75__);
			double __temp_77__;
			__temp_77__ = (5 * __var_4__[__iter_2__+(2)+(M-0)*(__iter_3__)]);
			double __temp_78__;
			__temp_78__ = (__temp_76__ + __temp_77__);
			double __temp_79__;
			__temp_79__ = (4 * __var_4__[__iter_2__+(-2)+(M-0)*(__iter_3__+(1))]);
			double __temp_80__;
			__temp_80__ = (__temp_78__ + __temp_79__);
			double __temp_81__;
			__temp_81__ = (9 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(1))]);
			double __temp_82__;
			__temp_82__ = (__temp_80__ + __temp_81__);
			double __temp_83__;
			__temp_83__ = (12 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(1))]);
			double __temp_84__;
			__temp_84__ = (__temp_82__ + __temp_83__);
			double __temp_85__;
			__temp_85__ = (9 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(1))]);
			double __temp_86__;
			__temp_86__ = (__temp_84__ + __temp_85__);
			double __temp_87__;
			__temp_87__ = (4 * __var_4__[__iter_2__+(2)+(M-0)*(__iter_3__+(1))]);
			double __temp_88__;
			__temp_88__ = (__temp_86__ + __temp_87__);
			double __temp_89__;
			__temp_89__ = (2 * __var_4__[__iter_2__+(-2)+(M-0)*(__iter_3__+(2))]);
			double __temp_90__;
			__temp_90__ = (__temp_88__ + __temp_89__);
			double __temp_91__;
			__temp_91__ = (4 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(2))]);
			double __temp_92__;
			__temp_92__ = (__temp_90__ + __temp_91__);
			double __temp_93__;
			__temp_93__ = (5 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(2))]);
			double __temp_94__;
			__temp_94__ = (__temp_92__ + __temp_93__);
			double __temp_95__;
			__temp_95__ = (4 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(2))]);
			double __temp_96__;
			__temp_96__ = (__temp_94__ + __temp_95__);
			double __temp_97__;
			__temp_97__ = (2 * __var_4__[__iter_2__+(2)+(M-0)*(__iter_3__+(2))]);
			double __temp_98__;
			__temp_98__ = (__temp_96__ + __temp_97__);
			double __temp_99__;
			__temp_99__ = (__temp_98__ / 159);
			__var_3__[__iter_2__+(M-0)*(__iter_3__)] = __temp_99__;
		}
	}
}
__global__ void __kernel___forma_kernel__2__(double * __restrict__ __var_3__, int N, int M, double * __restrict__ __var_2__){
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_4__;
	__iter_4__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 2;
	if(__iter_4__ <= (M-3)){
		int __iter_5__;
		__iter_5__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 2;
		if(__iter_5__ <= (N-3)){
			double __temp_100__;
			__temp_100__ = (2 * __var_3__[__iter_4__+(-2)+(M-0)*(__iter_5__+(-2))]);
			double __temp_101__;
			__temp_101__ = (4 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(-2))]);
			double __temp_102__;
			__temp_102__ = (__temp_100__ + __temp_101__);
			double __temp_103__;
			__temp_103__ = (5 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(-2))]);
			double __temp_104__;
			__temp_104__ = (__temp_102__ + __temp_103__);
			double __temp_105__;
			__temp_105__ = (4 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(-2))]);
			double __temp_106__;
			__temp_106__ = (__temp_104__ + __temp_105__);
			double __temp_107__;
			__temp_107__ = (2 * __var_3__[__iter_4__+(2)+(M-0)*(__iter_5__+(-2))]);
			double __temp_108__;
			__temp_108__ = (__temp_106__ + __temp_107__);
			double __temp_109__;
			__temp_109__ = (4 * __var_3__[__iter_4__+(-2)+(M-0)*(__iter_5__+(-1))]);
			double __temp_110__;
			__temp_110__ = (__temp_108__ + __temp_109__);
			double __temp_111__;
			__temp_111__ = (9 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(-1))]);
			double __temp_112__;
			__temp_112__ = (__temp_110__ + __temp_111__);
			double __temp_113__;
			__temp_113__ = (12 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(-1))]);
			double __temp_114__;
			__temp_114__ = (__temp_112__ + __temp_113__);
			double __temp_115__;
			__temp_115__ = (9 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(-1))]);
			double __temp_116__;
			__temp_116__ = (__temp_114__ + __temp_115__);
			double __temp_117__;
			__temp_117__ = (4 * __var_3__[__iter_4__+(2)+(M-0)*(__iter_5__+(-1))]);
			double __temp_118__;
			__temp_118__ = (__temp_116__ + __temp_117__);
			double __temp_119__;
			__temp_119__ = (5 * __var_3__[__iter_4__+(-2)+(M-0)*(__iter_5__)]);
			double __temp_120__;
			__temp_120__ = (__temp_118__ + __temp_119__);
			double __temp_121__;
			__temp_121__ = (12 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__)]);
			double __temp_122__;
			__temp_122__ = (__temp_120__ + __temp_121__);
			double __temp_123__;
			__temp_123__ = (15 * __var_3__[__iter_4__+(M-0)*(__iter_5__)]);
			double __temp_124__;
			__temp_124__ = (__temp_122__ + __temp_123__);
			double __temp_125__;
			__temp_125__ = (12 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__)]);
			double __temp_126__;
			__temp_126__ = (__temp_124__ + __temp_125__);
			double __temp_127__;
			__temp_127__ = (5 * __var_3__[__iter_4__+(2)+(M-0)*(__iter_5__)]);
			double __temp_128__;
			__temp_128__ = (__temp_126__ + __temp_127__);
			double __temp_129__;
			__temp_129__ = (4 * __var_3__[__iter_4__+(-2)+(M-0)*(__iter_5__+(1))]);
			double __temp_130__;
			__temp_130__ = (__temp_128__ + __temp_129__);
			double __temp_131__;
			__temp_131__ = (9 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(1))]);
			double __temp_132__;
			__temp_132__ = (__temp_130__ + __temp_131__);
			double __temp_133__;
			__temp_133__ = (12 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(1))]);
			double __temp_134__;
			__temp_134__ = (__temp_132__ + __temp_133__);
			double __temp_135__;
			__temp_135__ = (9 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(1))]);
			double __temp_136__;
			__temp_136__ = (__temp_134__ + __temp_135__);
			double __temp_137__;
			__temp_137__ = (4 * __var_3__[__iter_4__+(2)+(M-0)*(__iter_5__+(1))]);
			double __temp_138__;
			__temp_138__ = (__temp_136__ + __temp_137__);
			double __temp_139__;
			__temp_139__ = (2 * __var_3__[__iter_4__+(-2)+(M-0)*(__iter_5__+(2))]);
			double __temp_140__;
			__temp_140__ = (__temp_138__ + __temp_139__);
			double __temp_141__;
			__temp_141__ = (4 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(2))]);
			double __temp_142__;
			__temp_142__ = (__temp_140__ + __temp_141__);
			double __temp_143__;
			__temp_143__ = (5 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(2))]);
			double __temp_144__;
			__temp_144__ = (__temp_142__ + __temp_143__);
			double __temp_145__;
			__temp_145__ = (4 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(2))]);
			double __temp_146__;
			__temp_146__ = (__temp_144__ + __temp_145__);
			double __temp_147__;
			__temp_147__ = (2 * __var_3__[__iter_4__+(2)+(M-0)*(__iter_5__+(2))]);
			double __temp_148__;
			__temp_148__ = (__temp_146__ + __temp_147__);
			double __temp_149__;
			__temp_149__ = (__temp_148__ / 159);
			__var_2__[__iter_4__+(M-0)*(__iter_5__)] = __temp_149__;
		}
	}
}
__global__ void __kernel___forma_kernel__3__(double * __restrict__ __var_2__, int N, int M, double * __restrict__ __var_1__){
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_6__;
	__iter_6__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 2;
	if(__iter_6__ <= (M-3)){
		int __iter_7__;
		__iter_7__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 2;
		if(__iter_7__ <= (N-3)){
			double __temp_150__;
			__temp_150__ = (2 * __var_2__[__iter_6__+(-2)+(M-0)*(__iter_7__+(-2))]);
			double __temp_151__;
			__temp_151__ = (4 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(-2))]);
			double __temp_152__;
			__temp_152__ = (__temp_150__ + __temp_151__);
			double __temp_153__;
			__temp_153__ = (5 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(-2))]);
			double __temp_154__;
			__temp_154__ = (__temp_152__ + __temp_153__);
			double __temp_155__;
			__temp_155__ = (4 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(-2))]);
			double __temp_156__;
			__temp_156__ = (__temp_154__ + __temp_155__);
			double __temp_157__;
			__temp_157__ = (2 * __var_2__[__iter_6__+(2)+(M-0)*(__iter_7__+(-2))]);
			double __temp_158__;
			__temp_158__ = (__temp_156__ + __temp_157__);
			double __temp_159__;
			__temp_159__ = (4 * __var_2__[__iter_6__+(-2)+(M-0)*(__iter_7__+(-1))]);
			double __temp_160__;
			__temp_160__ = (__temp_158__ + __temp_159__);
			double __temp_161__;
			__temp_161__ = (9 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(-1))]);
			double __temp_162__;
			__temp_162__ = (__temp_160__ + __temp_161__);
			double __temp_163__;
			__temp_163__ = (12 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(-1))]);
			double __temp_164__;
			__temp_164__ = (__temp_162__ + __temp_163__);
			double __temp_165__;
			__temp_165__ = (9 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(-1))]);
			double __temp_166__;
			__temp_166__ = (__temp_164__ + __temp_165__);
			double __temp_167__;
			__temp_167__ = (4 * __var_2__[__iter_6__+(2)+(M-0)*(__iter_7__+(-1))]);
			double __temp_168__;
			__temp_168__ = (__temp_166__ + __temp_167__);
			double __temp_169__;
			__temp_169__ = (5 * __var_2__[__iter_6__+(-2)+(M-0)*(__iter_7__)]);
			double __temp_170__;
			__temp_170__ = (__temp_168__ + __temp_169__);
			double __temp_171__;
			__temp_171__ = (12 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__)]);
			double __temp_172__;
			__temp_172__ = (__temp_170__ + __temp_171__);
			double __temp_173__;
			__temp_173__ = (15 * __var_2__[__iter_6__+(M-0)*(__iter_7__)]);
			double __temp_174__;
			__temp_174__ = (__temp_172__ + __temp_173__);
			double __temp_175__;
			__temp_175__ = (12 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__)]);
			double __temp_176__;
			__temp_176__ = (__temp_174__ + __temp_175__);
			double __temp_177__;
			__temp_177__ = (5 * __var_2__[__iter_6__+(2)+(M-0)*(__iter_7__)]);
			double __temp_178__;
			__temp_178__ = (__temp_176__ + __temp_177__);
			double __temp_179__;
			__temp_179__ = (4 * __var_2__[__iter_6__+(-2)+(M-0)*(__iter_7__+(1))]);
			double __temp_180__;
			__temp_180__ = (__temp_178__ + __temp_179__);
			double __temp_181__;
			__temp_181__ = (9 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(1))]);
			double __temp_182__;
			__temp_182__ = (__temp_180__ + __temp_181__);
			double __temp_183__;
			__temp_183__ = (12 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(1))]);
			double __temp_184__;
			__temp_184__ = (__temp_182__ + __temp_183__);
			double __temp_185__;
			__temp_185__ = (9 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(1))]);
			double __temp_186__;
			__temp_186__ = (__temp_184__ + __temp_185__);
			double __temp_187__;
			__temp_187__ = (4 * __var_2__[__iter_6__+(2)+(M-0)*(__iter_7__+(1))]);
			double __temp_188__;
			__temp_188__ = (__temp_186__ + __temp_187__);
			double __temp_189__;
			__temp_189__ = (2 * __var_2__[__iter_6__+(-2)+(M-0)*(__iter_7__+(2))]);
			double __temp_190__;
			__temp_190__ = (__temp_188__ + __temp_189__);
			double __temp_191__;
			__temp_191__ = (4 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(2))]);
			double __temp_192__;
			__temp_192__ = (__temp_190__ + __temp_191__);
			double __temp_193__;
			__temp_193__ = (5 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(2))]);
			double __temp_194__;
			__temp_194__ = (__temp_192__ + __temp_193__);
			double __temp_195__;
			__temp_195__ = (4 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(2))]);
			double __temp_196__;
			__temp_196__ = (__temp_194__ + __temp_195__);
			double __temp_197__;
			__temp_197__ = (2 * __var_2__[__iter_6__+(2)+(M-0)*(__iter_7__+(2))]);
			double __temp_198__;
			__temp_198__ = (__temp_196__ + __temp_197__);
			double __temp_199__;
			__temp_199__ = (__temp_198__ / 159);
			__var_1__[__iter_6__+(M-0)*(__iter_7__)] = __temp_199__;
		}
	}
}
/*Device code End */
/* Host Code Begin */
extern "C" void gaussian(double * h_input, int N, int M, double * __var_0__){

	/* Host allocation Begin */
	double * input;
	hipMalloc(&input,sizeof(double)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : input\n");
	hipPointerAttribute_t ptrAttrib_h_input;
	hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
	if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
		if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
			memcpy_kind_h_input = hipMemcpyDeviceToDevice;
	hipGetLastError();
	if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
		hipMemcpy(input,h_input,sizeof(double)*((N-0)*(M-0)), memcpy_kind_h_input);
	}
	double * __var_1__;
	hipMalloc(&__var_1__,sizeof(double)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : __var_1__\n");
	double * __var_2__;
	hipMalloc(&__var_2__,sizeof(double)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : __var_2__\n");
	double * __var_3__;
	hipMalloc(&__var_3__,sizeof(double)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : __var_3__\n");
	double * __var_4__;
	hipMalloc(&__var_4__,sizeof(double)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : __var_4__\n");
	/*Host Allocation End */
	/* Kernel Launch Begin */
#ifdef _TIMER_
	hipEvent_t _forma_timer_start_,_forma_timer_stop_;
	hipEventCreate(&_forma_timer_start_);
	hipEventCreate(&_forma_timer_stop_);
	hipEventRecord(_forma_timer_start_,0);
#endif
	int __size_0___kernel___forma_kernel__0__ = ((M-3) - 2 ) + 1;
	int __size_1___kernel___forma_kernel__0__ = ((N-3) - 2 ) + 1;
	int __block_0___kernel___forma_kernel__0__ = 32;
	int __block_1___kernel___forma_kernel__0__ = 32;
	dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
	int __SMemSize___kernel___forma_kernel__0__ = 0;
	int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
	int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
	dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<1000; x++) {
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_4__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_4__, N, M, __var_3__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_3__, N, M, __var_2__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, N, M, __var_1__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();

	hipPointerAttribute_t ptrAttrib___var_0__;
	hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
	if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
		if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
			memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
	hipGetLastError();
	hipMemcpy(__var_0__,__var_1__, sizeof(double)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
	hipEventRecord(_forma_timer_stop_,0);
	hipEventSynchronize(_forma_timer_stop_);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
	printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
	hipEventDestroy(_forma_timer_start_);
	hipEventDestroy(_forma_timer_stop_);
#endif
	/*Kernel Launch End */
	/* Host Free Begin */
	hipFree(input);
	hipFree(__var_1__);
	hipFree(__var_2__);
	hipFree(__var_3__);
	hipFree(__var_4__);
}
/*Host Free End*/

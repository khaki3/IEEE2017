#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define mod(x,y) ( (x) & (y-1))

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, double * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  double * __tilevar_0__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double * __tilevar_1__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);

  double t2_0=0.0f, t2_1=0.0f, t3_0=0.0f, t3_1=0.0f, out=0.0f;
  double b2_0=0.0f, b2_1=0.0f, b3_0=0.0f, b3_1=0.0f;
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X-8);
  int __iter_y__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y));

  //Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
    __tilevar_1__[__iter_3__-__iter_0__] = 0.0f;
  }
  // Initial loop 
  for (int __iter_1__ = FORMA_MAX(0,__iter_y__-4); __iter_1__ <= __iter_y__+3; __iter_1__++) {
    if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
	__tilevar_0__[__iter_3__-__iter_0__] = input[__iter_3__+M*(__iter_1__)];
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-3))) {
    	// Bottom -2
        double __temp_2__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        double __temp_5__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        double __temp_9__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        double __temp_13__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        double __temp_17__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        double __temp_18__ = (__temp_14__ + 2 * __temp_17__) / 159;
	t2_1 += __temp_18__;
	// Bottom -1	
        double __temp_21__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        double __temp_25__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_26__ = (4 * __temp_21__ + 9 * __temp_25__);
        double __temp_29__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        double __temp_33__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        double __temp_37__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        double __temp_38__ = (__temp_34__ + 4 * __temp_37__) / 159;
	t2_0 += __temp_38__;
	// Mid
        double __temp_41__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        double __temp_45__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_46__ = (5 * __temp_41__ + 12 * __temp_45__);
        double __temp_49__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        double __temp_53__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        double __temp_57__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        double __temp_58__ = (__temp_54__ + 5 * __temp_57__) / 159;
	b2_0 += __temp_58__;
	// Top +1
        double __temp_61__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        double __temp_65__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_66__ = (4 * __temp_61__ + 9 * __temp_65__);
        double __temp_69__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        double __temp_73__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        double __temp_77__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        double __temp_78__ = (__temp_74__ + 4 * __temp_77__) / 159;
	b2_1 += __temp_78__;
	// Top +2
        double __temp_81__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        double __temp_85__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_86__ = (2 * __temp_81__ + 4 * __temp_85__);
        double __temp_89__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        double __temp_93__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        double __temp_97__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        double __temp_98__ = (__temp_94__ + 2 * __temp_97__) / 159;
        __tilevar_1__[__iter_3__-__iter_0__] += __temp_98__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+4),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-3))) {
    	// Bottom -2
        double __temp_2__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        double __temp_5__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        double __temp_9__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        double __temp_13__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        double __temp_17__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        double __temp_18__ = (__temp_14__ + 2 * __temp_17__) / 159;
	t3_1 += __temp_18__;
	// Bottom -1	
        double __temp_21__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        double __temp_25__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_26__ = (4 * __temp_21__ + 9 * __temp_25__);
        double __temp_29__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        double __temp_33__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        double __temp_37__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        double __temp_38__ = (__temp_34__ + 4 * __temp_37__) / 159;
	t3_0 += __temp_38__;
	// Mid
        double __temp_41__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        double __temp_45__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_46__ = (5 * __temp_41__ + 12 * __temp_45__);
        double __temp_49__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        double __temp_53__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        double __temp_57__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        double __temp_58__ = (__temp_54__ + 5 * __temp_57__) / 159;
	b3_0 += __temp_58__;
	// Top +1
        double __temp_61__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        double __temp_65__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_66__ = (4 * __temp_61__ + 9 * __temp_65__);
        double __temp_69__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        double __temp_73__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        double __temp_77__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        double __temp_78__ = (__temp_74__ + 4 * __temp_77__) / 159;
	b3_1 += __temp_78__;
	// Top +2
        double __temp_81__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        double __temp_85__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_86__ = (2 * __temp_81__ + 4 * __temp_85__);
        double __temp_89__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        double __temp_93__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        double __temp_97__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        double __temp_98__ = (__temp_94__ + 2 * __temp_97__) / 159;
	out += __temp_98__;
    }
    __syncthreads();
    // Now rotate
    __tilevar_1__[__iter_3__-__iter_0__] = b2_1; b2_1 = b2_0; b2_0=t2_0; t2_0=t2_1; t2_1=0.0f; 
    out=b3_1; b3_1=b3_0; b3_0=t3_0; t3_0=t3_1; t3_1=0.0f;
  }
  // Rest of the computation
  for (int __iter_1__ = __iter_y__+4; __iter_1__ <= FORMA_MIN(N-1,__iter_y__+FORMA_BLOCKDIM_Y+3); __iter_1__++) {
    if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
	__tilevar_0__[__iter_3__-__iter_0__] = input[__iter_3__+M*(__iter_1__)];
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-3))) {
    	// Bottom -2
        double __temp_2__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        double __temp_5__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        double __temp_9__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        double __temp_13__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        double __temp_17__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        double __temp_18__ = (__temp_14__ + 2 * __temp_17__) / 159;
	t2_1 += __temp_18__;
	// Bottom -1	
        double __temp_21__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        double __temp_25__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_26__ = (4 * __temp_21__ + 9 * __temp_25__);
        double __temp_29__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        double __temp_33__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        double __temp_37__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        double __temp_38__ = (__temp_34__ + 4 * __temp_37__) / 159;
	t2_0 += __temp_38__;
	// Mid
        double __temp_41__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        double __temp_45__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_46__ = (5 * __temp_41__ + 12 * __temp_45__);
        double __temp_49__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        double __temp_53__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        double __temp_57__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        double __temp_58__ = (__temp_54__ + 5 * __temp_57__) / 159;
	b2_0 += __temp_58__;
	// Top +1
        double __temp_61__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        double __temp_65__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_66__ = (4 * __temp_61__ + 9 * __temp_65__);
        double __temp_69__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        double __temp_73__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        double __temp_77__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        double __temp_78__ = (__temp_74__ + 4 * __temp_77__) / 159;
	b2_1 += __temp_78__;
	// Top +2
        double __temp_81__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        double __temp_85__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        double __temp_86__ = (2 * __temp_81__ + 4 * __temp_85__);
        double __temp_89__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        double __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        double __temp_93__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        double __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        double __temp_97__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        double __temp_98__ = (__temp_94__ + 2 * __temp_97__) / 159;
        __tilevar_1__[__iter_3__-__iter_0__] += __temp_98__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+4),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-3))) {
    	// Bottom -2
        double __temp_2__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        double __temp_5__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        double __temp_9__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        double __temp_13__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        double __temp_17__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        double __temp_18__ = (__temp_14__ + 2 * __temp_17__) / 159;
	t3_1 += __temp_18__;
	// Bottom -1	
        double __temp_21__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        double __temp_25__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_26__ = (4 * __temp_21__ + 9 * __temp_25__);
        double __temp_29__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        double __temp_33__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        double __temp_37__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        double __temp_38__ = (__temp_34__ + 4 * __temp_37__) / 159;
	t3_0 += __temp_38__;
	// Mid
        double __temp_41__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        double __temp_45__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_46__ = (5 * __temp_41__ + 12 * __temp_45__);
        double __temp_49__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        double __temp_53__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        double __temp_57__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        double __temp_58__ = (__temp_54__ + 5 * __temp_57__) / 159;
	b3_0 += __temp_58__;
	// Top +1
        double __temp_61__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        double __temp_65__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_66__ = (4 * __temp_61__ + 9 * __temp_65__);
        double __temp_69__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        double __temp_73__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        double __temp_77__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        double __temp_78__ = (__temp_74__ + 4 * __temp_77__) / 159;
	b3_1 += __temp_78__;
	// Top +2
        double __temp_81__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        double __temp_85__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        double __temp_86__ = (2 * __temp_81__ + 4 * __temp_85__);
        double __temp_89__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        double __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        double __temp_93__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        double __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        double __temp_97__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        double __temp_98__ = (__temp_94__ + 2 * __temp_97__) / 159;
	out += __temp_98__;
	__var_1__[__iter_3__+M*FORMA_MAX(__iter_1__-4,0)] = out;
    }
    __syncthreads();
    // Now rotate
    __tilevar_1__[__iter_3__-__iter_0__] = b2_1; b2_1 = b2_0; b2_0=t2_0; t2_0=t2_1; t2_1=0.0f; 
    out=b3_1; b3_1=b3_0; b3_0=t3_0; t3_0=t3_1; t3_1=0.0f;
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(double)*(2*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void gaussian(double * h_input, int N, int M, double * __var_0__){

/* Host allocation Begin */
  double * input;
  hipMalloc(&input,sizeof(double)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(double)*((N)*(M)), memcpy_kind_h_input);
  }
  double * __var_1__;
  hipMalloc(&__var_1__,sizeof(double)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  double * __var_2__;
  hipMalloc(&__var_2__,sizeof(double)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");

/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = M;
  int __size_1___kernel___forma_kernel__0__ = N;

  int __block_0___kernel___forma_kernel__0__ = 128;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-8);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__size_0___kernel___forma_kernel__0__/32);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __size_0___kernel___forma_kernel__0__/32, __var_2__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __size_0___kernel___forma_kernel__0__/32, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(double)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

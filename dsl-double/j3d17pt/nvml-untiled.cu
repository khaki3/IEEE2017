#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"
#include <nvml.h>
#include <assert.h>


#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
	template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
	int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
	input[loc] = value;
}


	template<typename T>
void initialize_array(T* d_input, int size, T value)
{
	dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
	dim3 init_block(FORMA_MAX_BLOCKDIM_0);
	__kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int L, int M, int N, double * __restrict__ __var_4__){
	int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_0__;
	__iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
	if(__iter_0__ <= (N-2)){
		int __iter_1__;
		__iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
		if(__iter_1__ <= (M-2)){
			int __iter_2__;
			__iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
			if(__iter_2__ <= (L-2)){
				double __temp_0__;
				__temp_0__ = (input[__iter_0__+(-1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__+(-1)))] + input[__iter_0__+(1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__+(-1)))]);
				double __temp_1__;
				__temp_1__ = (__temp_0__ + input[__iter_0__+(-1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__+(-1)))]);
				double __temp_2__;
				__temp_2__ = (__temp_1__ + input[__iter_0__+(1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__+(-1)))]);
				double __temp_3__;
				__temp_3__ = (0.500000f * __temp_2__);
				double __temp_4__;
				__temp_4__ = (0.510000f * input[__iter_0__+(-1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__))]);
				double __temp_5__;
				__temp_5__ = (__temp_3__ + __temp_4__);
				double __temp_6__;
				__temp_6__ = (0.710000f * input[__iter_0__+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__))]);
				double __temp_7__;
				__temp_7__ = (__temp_5__ + __temp_6__);
				double __temp_8__;
				__temp_8__ = (0.910000f * input[__iter_0__+(1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__))]);
				double __temp_9__;
				__temp_9__ = (__temp_7__ + __temp_8__);
				double __temp_10__;
				__temp_10__ = (1.210000f * input[__iter_0__+(-1)+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
				double __temp_11__;
				__temp_11__ = (__temp_9__ + __temp_10__);
				double __temp_12__;
				__temp_12__ = (1.510000f * input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
				double __temp_13__;
				__temp_13__ = (__temp_11__ + __temp_12__);
				double __temp_14__;
				__temp_14__ = (1.210000f * input[__iter_0__+(1)+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
				double __temp_15__;
				__temp_15__ = (__temp_13__ + __temp_14__);
				double __temp_16__;
				__temp_16__ = (0.910000f * input[__iter_0__+(-1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__))]);
				double __temp_17__;
				__temp_17__ = (__temp_15__ + __temp_16__);
				double __temp_18__;
				__temp_18__ = (0.710000f * input[__iter_0__+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__))]);
				double __temp_19__;
				__temp_19__ = (__temp_17__ + __temp_18__);
				double __temp_20__;
				__temp_20__ = (0.510000f * input[__iter_0__+(1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__))]);
				double __temp_21__;
				__temp_21__ = (__temp_19__ + __temp_20__);
				double __temp_22__;
				__temp_22__ = (input[__iter_0__+(-1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__+(1)))] + input[__iter_0__+(1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__+(1)))]);
				double __temp_23__;
				__temp_23__ = (__temp_22__ + input[__iter_0__+(-1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__+(1)))]);
				double __temp_24__;
				__temp_24__ = (__temp_23__ + input[__iter_0__+(1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__+(1)))]);
				double __temp_25__;
				__temp_25__ = (0.520000f * __temp_24__);
				double __temp_26__;
				__temp_26__ = (__temp_21__ + __temp_25__);
				double __temp_27__;
				__temp_27__ = (__temp_26__ / 159);
				__var_4__[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))] = __temp_27__;
			}
		}
	}
}
__global__ void __kernel___forma_kernel__1__(double * __restrict__ __var_4__, int L, int M, int N, double * __restrict__ __var_3__){
	int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_3__;
	__iter_3__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
	if(__iter_3__ <= (N-2)){
		int __iter_4__;
		__iter_4__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
		if(__iter_4__ <= (M-2)){
			int __iter_5__;
			__iter_5__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
			if(__iter_5__ <= (L-2)){
				double __temp_28__;
				__temp_28__ = (__var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__+(-1)))] + __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__+(-1)))]);
				double __temp_29__;
				__temp_29__ = (__temp_28__ + __var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__+(-1)))]);
				double __temp_30__;
				__temp_30__ = (__temp_29__ + __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__+(-1)))]);
				double __temp_31__;
				__temp_31__ = (0.500000f * __temp_30__);
				double __temp_32__;
				__temp_32__ = (0.510000f * __var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__))]);
				double __temp_33__;
				__temp_33__ = (__temp_31__ + __temp_32__);
				double __temp_34__;
				__temp_34__ = (0.710000f * __var_4__[__iter_3__+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__))]);
				double __temp_35__;
				__temp_35__ = (__temp_33__ + __temp_34__);
				double __temp_36__;
				__temp_36__ = (0.910000f * __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__))]);
				double __temp_37__;
				__temp_37__ = (__temp_35__ + __temp_36__);
				double __temp_38__;
				__temp_38__ = (1.210000f * __var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(M-0)*(__iter_5__))]);
				double __temp_39__;
				__temp_39__ = (__temp_37__ + __temp_38__);
				double __temp_40__;
				__temp_40__ = (1.510000f * __var_4__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))]);
				double __temp_41__;
				__temp_41__ = (__temp_39__ + __temp_40__);
				double __temp_42__;
				__temp_42__ = (1.210000f * __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(M-0)*(__iter_5__))]);
				double __temp_43__;
				__temp_43__ = (__temp_41__ + __temp_42__);
				double __temp_44__;
				__temp_44__ = (0.910000f * __var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__))]);
				double __temp_45__;
				__temp_45__ = (__temp_43__ + __temp_44__);
				double __temp_46__;
				__temp_46__ = (0.710000f * __var_4__[__iter_3__+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__))]);
				double __temp_47__;
				__temp_47__ = (__temp_45__ + __temp_46__);
				double __temp_48__;
				__temp_48__ = (0.510000f * __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__))]);
				double __temp_49__;
				__temp_49__ = (__temp_47__ + __temp_48__);
				double __temp_50__;
				__temp_50__ = (__var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__+(1)))] + __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__+(1)))]);
				double __temp_51__;
				__temp_51__ = (__temp_50__ + __var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__+(1)))]);
				double __temp_52__;
				__temp_52__ = (__temp_51__ + __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__+(1)))]);
				double __temp_53__;
				__temp_53__ = (0.520000f * __temp_52__);
				double __temp_54__;
				__temp_54__ = (__temp_49__ + __temp_53__);
				double __temp_55__;
				__temp_55__ = (__temp_54__ / 159);
				__var_3__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))] = __temp_55__;
			}
		}
	}
}
__global__ void __kernel___forma_kernel__2__(double * __restrict__ __var_3__, int L, int M, int N, double * __restrict__ __var_2__){
	int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_6__;
	__iter_6__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
	if(__iter_6__ <= (N-2)){
		int __iter_7__;
		__iter_7__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
		if(__iter_7__ <= (M-2)){
			int __iter_8__;
			__iter_8__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
			if(__iter_8__ <= (L-2)){
				double __temp_56__;
				__temp_56__ = (__var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__+(-1)))] + __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__+(-1)))]);
				double __temp_57__;
				__temp_57__ = (__temp_56__ + __var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__+(-1)))]);
				double __temp_58__;
				__temp_58__ = (__temp_57__ + __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__+(-1)))]);
				double __temp_59__;
				__temp_59__ = (0.500000f * __temp_58__);
				double __temp_60__;
				__temp_60__ = (0.510000f * __var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__))]);
				double __temp_61__;
				__temp_61__ = (__temp_59__ + __temp_60__);
				double __temp_62__;
				__temp_62__ = (0.710000f * __var_3__[__iter_6__+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__))]);
				double __temp_63__;
				__temp_63__ = (__temp_61__ + __temp_62__);
				double __temp_64__;
				__temp_64__ = (0.910000f * __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__))]);
				double __temp_65__;
				__temp_65__ = (__temp_63__ + __temp_64__);
				double __temp_66__;
				__temp_66__ = (1.210000f * __var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(M-0)*(__iter_8__))]);
				double __temp_67__;
				__temp_67__ = (__temp_65__ + __temp_66__);
				double __temp_68__;
				__temp_68__ = (1.510000f * __var_3__[__iter_6__+(N-0)*(__iter_7__+(M-0)*(__iter_8__))]);
				double __temp_69__;
				__temp_69__ = (__temp_67__ + __temp_68__);
				double __temp_70__;
				__temp_70__ = (1.210000f * __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(M-0)*(__iter_8__))]);
				double __temp_71__;
				__temp_71__ = (__temp_69__ + __temp_70__);
				double __temp_72__;
				__temp_72__ = (0.910000f * __var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__))]);
				double __temp_73__;
				__temp_73__ = (__temp_71__ + __temp_72__);
				double __temp_74__;
				__temp_74__ = (0.710000f * __var_3__[__iter_6__+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__))]);
				double __temp_75__;
				__temp_75__ = (__temp_73__ + __temp_74__);
				double __temp_76__;
				__temp_76__ = (0.510000f * __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__))]);
				double __temp_77__;
				__temp_77__ = (__temp_75__ + __temp_76__);
				double __temp_78__;
				__temp_78__ = (__var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__+(1)))] + __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__+(1)))]);
				double __temp_79__;
				__temp_79__ = (__temp_78__ + __var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__+(1)))]);
				double __temp_80__;
				__temp_80__ = (__temp_79__ + __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__+(1)))]);
				double __temp_81__;
				__temp_81__ = (0.520000f * __temp_80__);
				double __temp_82__;
				__temp_82__ = (__temp_77__ + __temp_81__);
				double __temp_83__;
				__temp_83__ = (__temp_82__ / 159);
				__var_2__[__iter_6__+(N-0)*(__iter_7__+(M-0)*(__iter_8__))] = __temp_83__;
			}
		}
	}
}
__global__ void __kernel___forma_kernel__3__(double * __restrict__ __var_2__, int L, int M, int N, double * __restrict__ __var_1__){
	int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_9__;
	__iter_9__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
	if(__iter_9__ <= (N-2)){
		int __iter_10__;
		__iter_10__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
		if(__iter_10__ <= (M-2)){
			int __iter_11__;
			__iter_11__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
			if(__iter_11__ <= (L-2)){
				double __temp_84__;
				__temp_84__ = (__var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__+(-1)))] + __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__+(-1)))]);
				double __temp_85__;
				__temp_85__ = (__temp_84__ + __var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__+(-1)))]);
				double __temp_86__;
				__temp_86__ = (__temp_85__ + __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__+(-1)))]);
				double __temp_87__;
				__temp_87__ = (0.500000f * __temp_86__);
				double __temp_88__;
				__temp_88__ = (0.510000f * __var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__))]);
				double __temp_89__;
				__temp_89__ = (__temp_87__ + __temp_88__);
				double __temp_90__;
				__temp_90__ = (0.710000f * __var_2__[__iter_9__+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__))]);
				double __temp_91__;
				__temp_91__ = (__temp_89__ + __temp_90__);
				double __temp_92__;
				__temp_92__ = (0.910000f * __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__))]);
				double __temp_93__;
				__temp_93__ = (__temp_91__ + __temp_92__);
				double __temp_94__;
				__temp_94__ = (1.210000f * __var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(M-0)*(__iter_11__))]);
				double __temp_95__;
				__temp_95__ = (__temp_93__ + __temp_94__);
				double __temp_96__;
				__temp_96__ = (1.510000f * __var_2__[__iter_9__+(N-0)*(__iter_10__+(M-0)*(__iter_11__))]);
				double __temp_97__;
				__temp_97__ = (__temp_95__ + __temp_96__);
				double __temp_98__;
				__temp_98__ = (1.210000f * __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(M-0)*(__iter_11__))]);
				double __temp_99__;
				__temp_99__ = (__temp_97__ + __temp_98__);
				double __temp_100__;
				__temp_100__ = (0.910000f * __var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__))]);
				double __temp_101__;
				__temp_101__ = (__temp_99__ + __temp_100__);
				double __temp_102__;
				__temp_102__ = (0.710000f * __var_2__[__iter_9__+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__))]);
				double __temp_103__;
				__temp_103__ = (__temp_101__ + __temp_102__);
				double __temp_104__;
				__temp_104__ = (0.510000f * __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__))]);
				double __temp_105__;
				__temp_105__ = (__temp_103__ + __temp_104__);
				double __temp_106__;
				__temp_106__ = (__var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__+(1)))] + __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__+(1)))]);
				double __temp_107__;
				__temp_107__ = (__temp_106__ + __var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__+(1)))]);
				double __temp_108__;
				__temp_108__ = (__temp_107__ + __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__+(1)))]);
				double __temp_109__;
				__temp_109__ = (0.520000f * __temp_108__);
				double __temp_110__;
				__temp_110__ = (__temp_105__ + __temp_109__);
				double __temp_111__;
				__temp_111__ = (__temp_110__ / 159);
				__var_1__[__iter_9__+(N-0)*(__iter_10__+(M-0)*(__iter_11__))] = __temp_111__;
			}
		}
	}
}
/*Device code End */
/* Host Code Begin */
extern "C" void j3d17pt(double * h_input, int L, int M, int N, double * __var_0__){

	/* Host allocation Begin */
	double * input;
	hipMalloc(&input,sizeof(double)*((L-0)*(M-0)*(N-0)));
	Check_CUDA_Error("Allocation Error!! : input\n");
	hipPointerAttribute_t ptrAttrib_h_input;
	hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
	if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
		if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
			memcpy_kind_h_input = hipMemcpyDeviceToDevice;
	hipGetLastError();
	if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
		hipMemcpy(input,h_input,sizeof(double)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
	}
	double * __var_1__;
	hipMalloc(&__var_1__,sizeof(double)*((L-0)*(M-0)*(N-0)));
	Check_CUDA_Error("Allocation Error!! : __var_1__\n");
	double * __var_2__;
	hipMalloc(&__var_2__,sizeof(double)*((L-0)*(M-0)*(N-0)));
	Check_CUDA_Error("Allocation Error!! : __var_2__\n");
	double * __var_3__;
	hipMalloc(&__var_3__,sizeof(double)*((L-0)*(M-0)*(N-0)));
	Check_CUDA_Error("Allocation Error!! : __var_3__\n");
	double * __var_4__;
	hipMalloc(&__var_4__,sizeof(double)*((L-0)*(M-0)*(N-0)));
	Check_CUDA_Error("Allocation Error!! : __var_4__\n");
	/*Host Allocation End */
	/* Kernel Launch Begin */
#ifdef _TIMER_
	hipEvent_t _forma_timer_start_,_forma_timer_stop_;
	hipEventCreate(&_forma_timer_start_);
	hipEventCreate(&_forma_timer_stop_);
	hipEventRecord(_forma_timer_start_,0);
#endif
	int __size_0___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
	int __size_1___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
	int __size_2___kernel___forma_kernel__0__ = ((L-2) - 1 ) + 1;
	int __block_0___kernel___forma_kernel__0__ = 16;
	int __block_1___kernel___forma_kernel__0__ = 4;
	int __block_2___kernel___forma_kernel__0__ = 4;
	dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
	int __SMemSize___kernel___forma_kernel__0__ = 0;
	int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
	int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
	int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
	dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<500; x++) {

		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __var_4__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_4__, L, M, N, __var_3__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_3__, L, M, N, __var_2__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, L, M, N, __var_1__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();


	hipPointerAttribute_t ptrAttrib___var_0__;
	hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
	if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
		if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
			memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
	hipGetLastError();
	hipMemcpy(__var_0__,__var_1__, sizeof(double)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
	hipEventRecord(_forma_timer_stop_,0);
	hipEventSynchronize(_forma_timer_stop_);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
	printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
	hipEventDestroy(_forma_timer_start_);
	hipEventDestroy(_forma_timer_stop_);
#endif
	/*Kernel Launch End */
	/* Host Free Begin */
	hipFree(input);
	hipFree(__var_1__);
	hipFree(__var_2__);
	hipFree(__var_3__);
	hipFree(__var_4__);
}
/*Host Free End*/

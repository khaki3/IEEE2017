#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define mod(x,y) ( (x) & (y-1))

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, double * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  double * __tilevar_2__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double * __tilevar_3__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double * __tilevar_4__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double * __tilevar_5__ = (double*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(double)*(FORMA_BLOCKDIM_X);
  double t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f;
  double b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;
  int __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X-8);
  int __iter_y__ = FORMA_MAX((int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y)-4, 0);
  // Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
    __tilevar_2__[__iter_3__-__iter_0__] = input[__iter_3__+M*__iter_y__];
    t2 = input[__iter_3__+M*(__iter_y__+1)]; 
  }

  // Initial computation
  for (int __iter_1__ = FORMA_MAX(1,__iter_y__+1); __iter_1__ < FORMA_MIN(N-1,__iter_y__+7); __iter_1__++) {
    if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
      b2 = __tilevar_2__[__iter_3__-__iter_0__];
      __tilevar_2__[__iter_3__-__iter_0__] = t2;
      t2 = input[__iter_3__+M*(__iter_1__+1)];  
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        double __temp_1__ = (__tilevar_2__[__iter_3__-__iter_0__] - b2);
        double __temp_2__ = (__temp_1__ * __temp_1__);
        double __temp_3__ = (0.000100f + __temp_2__);
        double __temp_5__ = (__tilevar_2__[__iter_3__-__iter_0__] - t2);
        double __temp_6__ = (__temp_5__ * __temp_5__);
        double __temp_7__ = (__temp_3__ + __temp_6__);
        double __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__] - __tilevar_2__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_9__ * __temp_9__);
        double __temp_11__ = (__temp_7__ + __temp_10__);
        double __temp_13__ = (__tilevar_2__[__iter_3__-__iter_0__] - __tilevar_2__[__iter_3__-1-__iter_0__]);
        double __temp_14__ = (__temp_13__ * __temp_13__);
        double __temp_15__ = (__temp_11__ + __temp_14__);
        double __temp_16__ = sqrt(__temp_15__);
        double __temp_17__ = (1.000000f / __temp_16__);
        double __temp_18__ = (__tilevar_2__[__iter_3__-__iter_0__] + __temp_17__);
        b3 = __tilevar_3__[__iter_3__-__iter_0__];
        __tilevar_3__[__iter_3__-__iter_0__] = t3;
        t3 = __temp_18__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        double __temp_1__ = (__tilevar_3__[__iter_3__-__iter_0__] - b3);
        double __temp_2__ = (__temp_1__ * __temp_1__);
        double __temp_3__ = (0.000100f + __temp_2__);
        double __temp_5__ = (__tilevar_3__[__iter_3__-__iter_0__] - t3);
        double __temp_6__ = (__temp_5__ * __temp_5__);
        double __temp_7__ = (__temp_3__ + __temp_6__);
        double __temp_9__ = (__tilevar_3__[__iter_3__-__iter_0__] - __tilevar_3__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_9__ * __temp_9__);
        double __temp_11__ = (__temp_7__ + __temp_10__);
        double __temp_13__ = (__tilevar_3__[__iter_3__-__iter_0__] - __tilevar_3__[__iter_3__-1-__iter_0__]);
        double __temp_14__ = (__temp_13__ * __temp_13__);
        double __temp_15__ = (__temp_11__ + __temp_14__);
        double __temp_16__ = sqrt(__temp_15__);
        double __temp_17__ = (1.000000f / __temp_16__);
        double __temp_18__ = (__tilevar_3__[__iter_3__-__iter_0__] + __temp_17__);
        b4 = __tilevar_4__[__iter_3__-__iter_0__];
        __tilevar_4__[__iter_3__-__iter_0__] = t4;
        t4 = __temp_18__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        double __temp_1__ = (__tilevar_4__[__iter_3__-__iter_0__] - b4);
        double __temp_2__ = (__temp_1__ * __temp_1__);
        double __temp_3__ = (0.000100f + __temp_2__);
        double __temp_5__ = (__tilevar_4__[__iter_3__-__iter_0__] - t4);
        double __temp_6__ = (__temp_5__ * __temp_5__);
        double __temp_7__ = (__temp_3__ + __temp_6__);
        double __temp_9__ = (__tilevar_4__[__iter_3__-__iter_0__] - __tilevar_4__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_9__ * __temp_9__);
        double __temp_11__ = (__temp_7__ + __temp_10__);
        double __temp_13__ = (__tilevar_4__[__iter_3__-__iter_0__] - __tilevar_4__[__iter_3__-1-__iter_0__]);
        double __temp_14__ = (__temp_13__ * __temp_13__);
        double __temp_15__ = (__temp_11__ + __temp_14__);
        double __temp_16__ = sqrt(__temp_15__);
        double __temp_17__ = (1.000000f / __temp_16__);
        double __temp_18__ = (__tilevar_4__[__iter_3__-__iter_0__] + __temp_17__);
        b5 = __tilevar_5__[__iter_3__-__iter_0__];
        __tilevar_5__[__iter_3__-__iter_0__] = t5;
        t5 = __temp_18__;
    }
  }
  // Rest of the computation
  for (int __iter_1__ = FORMA_MAX(1,__iter_y__+7); __iter_1__ < FORMA_MIN(N-1,__iter_y__+FORMA_BLOCKDIM_Y+8); __iter_1__++) {
    if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
      b2 = __tilevar_2__[__iter_3__-__iter_0__];
      __tilevar_2__[__iter_3__-__iter_0__] = t2;
      t2 = input[__iter_3__+M*(__iter_1__+1)];  
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        double __temp_1__ = (__tilevar_2__[__iter_3__-__iter_0__] - b2);
        double __temp_2__ = (__temp_1__ * __temp_1__);
        double __temp_3__ = (0.000100f + __temp_2__);
        double __temp_5__ = (__tilevar_2__[__iter_3__-__iter_0__] - t2);
        double __temp_6__ = (__temp_5__ * __temp_5__);
        double __temp_7__ = (__temp_3__ + __temp_6__);
        double __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__] - __tilevar_2__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_9__ * __temp_9__);
        double __temp_11__ = (__temp_7__ + __temp_10__);
        double __temp_13__ = (__tilevar_2__[__iter_3__-__iter_0__] - __tilevar_2__[__iter_3__-1-__iter_0__]);
        double __temp_14__ = (__temp_13__ * __temp_13__);
        double __temp_15__ = (__temp_11__ + __temp_14__);
        double __temp_16__ = sqrt(__temp_15__);
        double __temp_17__ = (1.000000f / __temp_16__);
        double __temp_18__ = (__tilevar_2__[__iter_3__-__iter_0__] + __temp_17__);
        b3 = __tilevar_3__[__iter_3__-__iter_0__];
        __tilevar_3__[__iter_3__-__iter_0__] = t3;
        t3 = __temp_18__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        double __temp_1__ = (__tilevar_3__[__iter_3__-__iter_0__] - b3);
        double __temp_2__ = (__temp_1__ * __temp_1__);
        double __temp_3__ = (0.000100f + __temp_2__);
        double __temp_5__ = (__tilevar_3__[__iter_3__-__iter_0__] - t3);
        double __temp_6__ = (__temp_5__ * __temp_5__);
        double __temp_7__ = (__temp_3__ + __temp_6__);
        double __temp_9__ = (__tilevar_3__[__iter_3__-__iter_0__] - __tilevar_3__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_9__ * __temp_9__);
        double __temp_11__ = (__temp_7__ + __temp_10__);
        double __temp_13__ = (__tilevar_3__[__iter_3__-__iter_0__] - __tilevar_3__[__iter_3__-1-__iter_0__]);
        double __temp_14__ = (__temp_13__ * __temp_13__);
        double __temp_15__ = (__temp_11__ + __temp_14__);
        double __temp_16__ = sqrt(__temp_15__);
        double __temp_17__ = (1.000000f / __temp_16__);
        double __temp_18__ = (__tilevar_3__[__iter_3__-__iter_0__] + __temp_17__);
        b4 = __tilevar_4__[__iter_3__-__iter_0__];
        __tilevar_4__[__iter_3__-__iter_0__] = t4;
        t4 = __temp_18__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        double __temp_1__ = (__tilevar_4__[__iter_3__-__iter_0__] - b4);
        double __temp_2__ = (__temp_1__ * __temp_1__);
        double __temp_3__ = (0.000100f + __temp_2__);
        double __temp_5__ = (__tilevar_4__[__iter_3__-__iter_0__] - t4);
        double __temp_6__ = (__temp_5__ * __temp_5__);
        double __temp_7__ = (__temp_3__ + __temp_6__);
        double __temp_9__ = (__tilevar_4__[__iter_3__-__iter_0__] - __tilevar_4__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_9__ * __temp_9__);
        double __temp_11__ = (__temp_7__ + __temp_10__);
        double __temp_13__ = (__tilevar_4__[__iter_3__-__iter_0__] - __tilevar_4__[__iter_3__-1-__iter_0__]);
        double __temp_14__ = (__temp_13__ * __temp_13__);
        double __temp_15__ = (__temp_11__ + __temp_14__);
        double __temp_16__ = sqrt(__temp_15__);
        double __temp_17__ = (1.000000f / __temp_16__);
        double __temp_18__ = (__tilevar_4__[__iter_3__-__iter_0__] + __temp_17__);
        b5 = __tilevar_5__[__iter_3__-__iter_0__];
        __tilevar_5__[__iter_3__-__iter_0__] = t5;
        t5 = __temp_18__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
        double __temp_1__ = (__tilevar_5__[__iter_3__-__iter_0__] - b5);
        double __temp_2__ = (__temp_1__ * __temp_1__);
        double __temp_3__ = (0.000100f + __temp_2__);
        double __temp_5__ = (__tilevar_5__[__iter_3__-__iter_0__] - t5);
        double __temp_6__ = (__temp_5__ * __temp_5__);
        double __temp_7__ = (__temp_3__ + __temp_6__);
        double __temp_9__ = (__tilevar_5__[__iter_3__-__iter_0__] - __tilevar_5__[__iter_3__+1-__iter_0__]);
        double __temp_10__ = (__temp_9__ * __temp_9__);
        double __temp_11__ = (__temp_7__ + __temp_10__);
        double __temp_13__ = (__tilevar_5__[__iter_3__-__iter_0__] - __tilevar_5__[__iter_3__-1-__iter_0__]);
        double __temp_14__ = (__temp_13__ * __temp_13__);
        double __temp_15__ = (__temp_11__ + __temp_14__);
        double __temp_16__ = sqrt(__temp_15__);
        double __temp_17__ = (1.000000f / __temp_16__);
        double __temp_18__ = (__tilevar_5__[__iter_3__-__iter_0__] + __temp_17__);
        __var_1__[__iter_3__+(M)*FORMA_MAX(__iter_1__-3,0)] = __temp_18__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(double)*(4*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void gradient (double * h_input, int N, int M, double * __var_0__){

/* Host allocation Begin */
  double * input;
  hipMalloc(&input,sizeof(double)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(double)*((N)*(M)), memcpy_kind_h_input);
  }
  double * __var_1__;
  hipMalloc(&__var_1__,sizeof(double)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = M;
  int __size_1___kernel___forma_kernel__0__ = N;
  int __block_0___kernel___forma_kernel__0__ = 128;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-8);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__, __size_1___kernel___forma_kernel__0__/16);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __size_1___kernel___forma_kernel__0__/16, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(double)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

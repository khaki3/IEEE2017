#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define GAPX (118) 
#define EXTENT (5)

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float * __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_4__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_5__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);

  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X + GAPX);
  float t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f;
  float b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;

  // Initialize the value
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
     __tilevar_2__[__iter_3__-__iter_0__] = input[__iter_3__+M*0];
     t2 = input[__iter_3__+M*1];
  }
  // Rest of the computation
  for (int __iter_1__ = 1; __iter_1__ <= N-1; __iter_1__++) {
    if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
      b2 = __tilevar_2__[__iter_3__-__iter_0__];
      __tilevar_2__[__iter_3__-__iter_0__] = t2; 
      t2 = input[__iter_3__+M*(__iter_1__+1)];
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) &  __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        float __temp_0__ = (__tilevar_2__[__iter_3__-__iter_0__] - b2);
        float __temp_1__ = (__tilevar_2__[__iter_3__-__iter_0__] - b2);
        float __temp_2__ = (__temp_0__ * __temp_1__);
        float __temp_3__ = (0.000100f + __temp_2__);
        float __temp_4__ = (__tilevar_2__[__iter_3__-__iter_0__] - t2);
        float __temp_5__ = (__tilevar_2__[__iter_3__-__iter_0__] - t2);
        float __temp_6__ = (__temp_4__ * __temp_5__);
        float __temp_7__ = (__temp_3__ + __temp_6__);
        float __temp_8__ = (__tilevar_2__[__iter_3__-__iter_0__] - __tilevar_2__[__iter_3__+1-__iter_0__]);
        float __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__] - __tilevar_2__[__iter_3__+1-__iter_0__]);
        float __temp_10__ = (__temp_8__ * __temp_9__);
        float __temp_11__ = (__temp_7__ + __temp_10__);
        float __temp_12__ = (__tilevar_2__[__iter_3__-__iter_0__] - __tilevar_2__[__iter_3__-1-__iter_0__]);
        float __temp_13__ = (__tilevar_2__[__iter_3__-__iter_0__] - __tilevar_2__[__iter_3__-1-__iter_0__]);
        float __temp_14__ = (__temp_12__ * __temp_13__);
        float __temp_15__ = (__temp_11__ + __temp_14__);
        float __temp_16__ = sqrt(__temp_15__);
        float __temp_17__ = (1.000000f / __temp_16__);
        float __temp_18__ = (__tilevar_2__[__iter_3__-__iter_0__] + __temp_17__);
        b3 = __tilevar_3__[__iter_3__-__iter_0__];
        __tilevar_3__[__iter_3__-__iter_0__] = t3;
        t3 = __temp_18__;
    }
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      if (__iter_3__ < (FORMA_MAX((__iter_0__+1),1)+2) | __iter_3__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-2)) {
        __copy_arr_0__[__iter_1__+(M)*(__iter_3__)] = t3;
      }
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        float __temp_0__ = (__tilevar_3__[__iter_3__-__iter_0__] - b3);
        float __temp_1__ = (__tilevar_3__[__iter_3__-__iter_0__] - b3);
        float __temp_2__ = (__temp_0__ * __temp_1__);
        float __temp_3__ = (0.000100f + __temp_2__);
        float __temp_4__ = (__tilevar_3__[__iter_3__-__iter_0__] - t3);
        float __temp_5__ = (__tilevar_3__[__iter_3__-__iter_0__] - t3);
        float __temp_6__ = (__temp_4__ * __temp_5__);
        float __temp_7__ = (__temp_3__ + __temp_6__);
        float __temp_8__ = (__tilevar_3__[__iter_3__-__iter_0__] - __tilevar_3__[__iter_3__+1-__iter_0__]);
        float __temp_9__ = (__tilevar_3__[__iter_3__-__iter_0__] - __tilevar_3__[__iter_3__+1-__iter_0__]);
        float __temp_10__ = (__temp_8__ * __temp_9__);
        float __temp_11__ = (__temp_7__ + __temp_10__);
        float __temp_12__ = (__tilevar_3__[__iter_3__-__iter_0__] - __tilevar_3__[__iter_3__-1-__iter_0__]);
        float __temp_13__ = (__tilevar_3__[__iter_3__-__iter_0__] - __tilevar_3__[__iter_3__-1-__iter_0__]);
        float __temp_14__ = (__temp_12__ * __temp_13__);
        float __temp_15__ = (__temp_11__ + __temp_14__);
        float __temp_16__ = sqrt(__temp_15__);
        float __temp_17__ = (1.000000f / __temp_16__);
        float __temp_18__ = (__tilevar_3__[__iter_3__-__iter_0__] + __temp_17__);
        b4 = __tilevar_4__[__iter_3__-__iter_0__];
        __tilevar_4__[__iter_3__-__iter_0__] = t4;
        t4 = __temp_18__;
    }
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) &  __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      if (__iter_3__ < (FORMA_MAX((__iter_0__+2),1)+2) | __iter_3__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-2)) {
        __copy_arr_1__[__iter_1__+(M)*(__iter_3__)] = t4;
      }
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        float __temp_0__ = (__tilevar_4__[__iter_3__-__iter_0__] - b4);
        float __temp_1__ = (__tilevar_4__[__iter_3__-__iter_0__] - b4);
        float __temp_2__ = (__temp_0__ * __temp_1__);
        float __temp_3__ = (0.000100f + __temp_2__);
        float __temp_4__ = (__tilevar_4__[__iter_3__-__iter_0__] - t4);
        float __temp_5__ = (__tilevar_4__[__iter_3__-__iter_0__] - t4);
        float __temp_6__ = (__temp_4__ * __temp_5__);
        float __temp_7__ = (__temp_3__ + __temp_6__);
        float __temp_8__ = (__tilevar_4__[__iter_3__-__iter_0__] - __tilevar_4__[__iter_3__+1-__iter_0__]);
        float __temp_9__ = (__tilevar_4__[__iter_3__-__iter_0__] - __tilevar_4__[__iter_3__+1-__iter_0__]);
        float __temp_10__ = (__temp_8__ * __temp_9__);
        float __temp_11__ = (__temp_7__ + __temp_10__);
        float __temp_12__ = (__tilevar_4__[__iter_3__-__iter_0__] - __tilevar_4__[__iter_3__-1-__iter_0__]);
        float __temp_13__ = (__tilevar_4__[__iter_3__-__iter_0__] - __tilevar_4__[__iter_3__-1-__iter_0__]);
        float __temp_14__ = (__temp_12__ * __temp_13__);
        float __temp_15__ = (__temp_11__ + __temp_14__);
        float __temp_16__ = sqrt(__temp_15__);
        float __temp_17__ = (1.000000f / __temp_16__);
        float __temp_18__ = (__tilevar_4__[__iter_3__-__iter_0__] + __temp_17__);
        b5 = __tilevar_5__[__iter_3__-__iter_0__];
        __tilevar_5__[__iter_3__-__iter_0__] = t5;
        t5 = __temp_18__;
    }
    if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      if (__iter_3__ < (FORMA_MAX((__iter_0__+3),1)+2) | __iter_3__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-2)) {
        __copy_arr_2__[__iter_1__+(M)*(__iter_3__)] = t5;
      }
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
        float __temp_0__ = (__tilevar_5__[__iter_3__-__iter_0__] - b5);
        float __temp_1__ = (__tilevar_5__[__iter_3__-__iter_0__] - b5);
        float __temp_2__ = (__temp_0__ * __temp_1__);
        float __temp_3__ = (0.000100f + __temp_2__);
        float __temp_4__ = (__tilevar_5__[__iter_3__-__iter_0__] - t5);
        float __temp_5__ = (__tilevar_5__[__iter_3__-__iter_0__] - t5);
        float __temp_6__ = (__temp_4__ * __temp_5__);
        float __temp_7__ = (__temp_3__ + __temp_6__);
        float __temp_8__ = (__tilevar_5__[__iter_3__-__iter_0__] - __tilevar_5__[__iter_3__+1-__iter_0__]);
        float __temp_9__ = (__tilevar_5__[__iter_3__-__iter_0__] - __tilevar_5__[__iter_3__+1-__iter_0__]);
        float __temp_10__ = (__temp_8__ * __temp_9__);
        float __temp_11__ = (__temp_7__ + __temp_10__);
        float __temp_12__ = (__tilevar_5__[__iter_3__-__iter_0__] - __tilevar_5__[__iter_3__-1-__iter_0__]);
        float __temp_13__ = (__tilevar_5__[__iter_3__-__iter_0__] - __tilevar_5__[__iter_3__-1-__iter_0__]);
        float __temp_14__ = (__temp_12__ * __temp_13__);
        float __temp_15__ = (__temp_11__ + __temp_14__);
        float __temp_16__ = sqrt(__temp_15__);
        float __temp_17__ = (1.000000f / __temp_16__);
        float __temp_18__ = (__tilevar_5__[__iter_3__-__iter_0__] + __temp_17__);
        __var_1__[__iter_3__+(M)*FORMA_MAX(__iter_1__-3,0)] = __temp_18__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(4*FORMA_BLOCKDIM_X);
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float * __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_4__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_5__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
 
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X + GAPX) + (int)FORMA_BLOCKDIM_X;
  float t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f;
  float b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;

  // Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__-EXTENT,0) + (int)(threadIdx.x) ;
  if (__iter_3__ >= FORMA_MAX(__iter_0__-2,0) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1))){
    __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)] = input[__iter_3__+(M)*(0)];
    t2 = input[__iter_3__+(M)*(1)];
  }
  // Rest of the computation
  for (int __iter_1__ = 1; __iter_1__ <= N-1; __iter_1__++) {
    if(__iter_3__ >= FORMA_MAX(__iter_0__-2,0) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1))){
      b2 = __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)];
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)] = t2;
      t2 = input[__iter_3__+(M)*(__iter_1__+1)];
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__-1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
	float __temp_0__ = (__tilevar_2__[__iter_3__+EXTENT-__iter_0__] - b2);
        float __temp_1__ = (__tilevar_2__[__iter_3__+EXTENT-__iter_0__] - b2);
        float __temp_2__ = (__temp_0__ * __temp_1__);
        float __temp_3__ = (0.000100f + __temp_2__);
        float __temp_4__ = (__tilevar_2__[__iter_3__+EXTENT-__iter_0__] - t2);
        float __temp_5__ = (__tilevar_2__[__iter_3__+EXTENT-__iter_0__] - t2);
        float __temp_6__ = (__temp_4__ * __temp_5__);
        float __temp_7__ = (__temp_3__ + __temp_6__);
        float __temp_8__ = (__tilevar_2__[__iter_3__+EXTENT-__iter_0__] - __tilevar_2__[__iter_3__+1+EXTENT-__iter_0__]);
        float __temp_9__ = (__tilevar_2__[__iter_3__+EXTENT-__iter_0__] - __tilevar_2__[__iter_3__+1+EXTENT-__iter_0__]);
        float __temp_10__ = (__temp_8__ * __temp_9__);
        float __temp_11__ = (__temp_7__ + __temp_10__);
        float __temp_12__ = (__tilevar_2__[__iter_3__+EXTENT-__iter_0__] - __tilevar_2__[__iter_3__-1+EXTENT-__iter_0__]);
        float __temp_13__ = (__tilevar_2__[__iter_3__+EXTENT-__iter_0__] - __tilevar_2__[__iter_3__-1+EXTENT-__iter_0__]);
        float __temp_14__ = (__temp_12__ * __temp_13__);
        float __temp_15__ = (__temp_11__ + __temp_14__);
        float __temp_16__ = sqrt(__temp_15__);
        float __temp_17__ = (1.000000f / __temp_16__);
        float __temp_18__ = (__tilevar_2__[__iter_3__+EXTENT-__iter_0__] + __temp_17__);
        b3 = __tilevar_3__[__iter_3__+EXTENT-__iter_0__];
        __tilevar_3__[__iter_3__+EXTENT-__iter_0__] = t3;
        t3 = __temp_18__;
    }
    if (__iter_3__ >= FORMA_MAX((__iter_0__-3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) & (__iter_3__ < FORMA_MAX((__iter_0__-1),1) | __iter_3__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)))) {
      b3 = __copy_arr_0__[__iter_1__-2+(M)*(__iter_3__)];
      __tilevar_3__[__iter_3__+(EXTENT-__iter_0__)] = __copy_arr_0__[__iter_1__-1+(M)*(__iter_3__)];
      t3 = __copy_arr_0__[__iter_1__+(M)*(__iter_3__)]; 
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__-2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
        float __temp_0__ = (__tilevar_3__[__iter_3__+EXTENT-__iter_0__] - b3);
        float __temp_1__ = (__tilevar_3__[__iter_3__+EXTENT-__iter_0__] - b3);
        float __temp_2__ = (__temp_0__ * __temp_1__);
        float __temp_3__ = (0.000100f + __temp_2__);
        float __temp_4__ = (__tilevar_3__[__iter_3__+EXTENT-__iter_0__] - t3);
        float __temp_5__ = (__tilevar_3__[__iter_3__+EXTENT-__iter_0__] - t3);
        float __temp_6__ = (__temp_4__ * __temp_5__);
        float __temp_7__ = (__temp_3__ + __temp_6__);
        float __temp_8__ = (__tilevar_3__[__iter_3__+EXTENT-__iter_0__] - __tilevar_3__[__iter_3__+1+EXTENT-__iter_0__]);
        float __temp_9__ = (__tilevar_3__[__iter_3__+EXTENT-__iter_0__] - __tilevar_3__[__iter_3__+1+EXTENT-__iter_0__]);
        float __temp_10__ = (__temp_8__ * __temp_9__);
        float __temp_11__ = (__temp_7__ + __temp_10__);
        float __temp_12__ = (__tilevar_3__[__iter_3__+EXTENT-__iter_0__] - __tilevar_3__[__iter_3__-1+EXTENT-__iter_0__]);
        float __temp_13__ = (__tilevar_3__[__iter_3__+EXTENT-__iter_0__] - __tilevar_3__[__iter_3__-1+EXTENT-__iter_0__]);
        float __temp_14__ = (__temp_12__ * __temp_13__);
        float __temp_15__ = (__temp_11__ + __temp_14__);
        float __temp_16__ = sqrt(__temp_15__);
        float __temp_17__ = (1.000000f / __temp_16__);
        float __temp_18__ = (__tilevar_3__[__iter_3__+EXTENT-__iter_0__] + __temp_17__);
        b4 = __tilevar_4__[__iter_3__+EXTENT-__iter_0__];
        __tilevar_4__[__iter_3__+EXTENT-__iter_0__] = t4;
        t4 = __temp_18__;
    }
    if (__iter_3__ >= FORMA_MAX((__iter_0__-4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) & (__iter_3__ < FORMA_MAX((__iter_0__-2),1) | __iter_3__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)))) {
      b4 = __copy_arr_1__[__iter_1__-2+(M)*(__iter_3__)];
      __tilevar_4__[__iter_3__+(EXTENT-__iter_0__)] = __copy_arr_1__[__iter_1__-1+(M)*(__iter_3__)];
      t4 = __copy_arr_1__[__iter_1__+(M)*(__iter_3__)];
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__-3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
	float __temp_0__ = (__tilevar_4__[__iter_3__+EXTENT-__iter_0__] - b4);
        float __temp_1__ = (__tilevar_4__[__iter_3__+EXTENT-__iter_0__] - b4);
        float __temp_2__ = (__temp_0__ * __temp_1__);
        float __temp_3__ = (0.000100f + __temp_2__);
        float __temp_4__ = (__tilevar_4__[__iter_3__+EXTENT-__iter_0__] - t4);
        float __temp_5__ = (__tilevar_4__[__iter_3__+EXTENT-__iter_0__] - t4);
        float __temp_6__ = (__temp_4__ * __temp_5__);
        float __temp_7__ = (__temp_3__ + __temp_6__);
        float __temp_8__ = (__tilevar_4__[__iter_3__+EXTENT-__iter_0__] - __tilevar_4__[__iter_3__+1+EXTENT-__iter_0__]);
        float __temp_9__ = (__tilevar_4__[__iter_3__+EXTENT-__iter_0__] - __tilevar_4__[__iter_3__+1+EXTENT-__iter_0__]);
        float __temp_10__ = (__temp_8__ * __temp_9__);
        float __temp_11__ = (__temp_7__ + __temp_10__);
        float __temp_12__ = (__tilevar_4__[__iter_3__+EXTENT-__iter_0__] - __tilevar_4__[__iter_3__-1+EXTENT-__iter_0__]);
        float __temp_13__ = (__tilevar_4__[__iter_3__+EXTENT-__iter_0__] - __tilevar_4__[__iter_3__-1+EXTENT-__iter_0__]);
        float __temp_14__ = (__temp_12__ * __temp_13__);
        float __temp_15__ = (__temp_11__ + __temp_14__);
        float __temp_16__ = sqrt(__temp_15__);
        float __temp_17__ = (1.000000f / __temp_16__);
        float __temp_18__ = (__tilevar_4__[__iter_3__+EXTENT-__iter_0__] + __temp_17__);
        b5 = __tilevar_5__[__iter_3__+EXTENT-__iter_0__];
        __tilevar_5__[__iter_3__+EXTENT-__iter_0__] = t5;
        t5 = __temp_18__;
    }
    if (__iter_3__ >= FORMA_MAX((__iter_0__-5),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(M-2)) & (__iter_3__ < FORMA_MAX((__iter_0__-3),1) | __iter_3__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)))) {
      b5 = __copy_arr_2__[__iter_1__-2+(M)*(__iter_3__)];
      __tilevar_5__[__iter_3__+(EXTENT-__iter_0__)] = __copy_arr_2__[__iter_1__-1+(M)*(__iter_3__)];
      t5 = __copy_arr_2__[__iter_1__+(M)*(__iter_3__)];
    }
    __syncthreads();
    if( __iter_3__ >= FORMA_MAX((__iter_0__-4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
        float __temp_0__ = (__tilevar_5__[__iter_3__+EXTENT-__iter_0__] - b5);
        float __temp_1__ = (__tilevar_5__[__iter_3__+EXTENT-__iter_0__] - b5);
        float __temp_2__ = (__temp_0__ * __temp_1__);
        float __temp_3__ = (0.000100f + __temp_2__);
        float __temp_4__ = (__tilevar_5__[__iter_3__+EXTENT-__iter_0__] - t5);
        float __temp_5__ = (__tilevar_5__[__iter_3__+EXTENT-__iter_0__] - t5);
        float __temp_6__ = (__temp_4__ * __temp_5__);
        float __temp_7__ = (__temp_3__ + __temp_6__);
        float __temp_8__ = (__tilevar_5__[__iter_3__+EXTENT-__iter_0__] - __tilevar_5__[__iter_3__+1+EXTENT-__iter_0__]);
        float __temp_9__ = (__tilevar_5__[__iter_3__+EXTENT-__iter_0__] - __tilevar_5__[__iter_3__+1+EXTENT-__iter_0__]);
        float __temp_10__ = (__temp_8__ * __temp_9__);
        float __temp_11__ = (__temp_7__ + __temp_10__);
        float __temp_12__ = (__tilevar_5__[__iter_3__+EXTENT-__iter_0__] - __tilevar_5__[__iter_3__-1+EXTENT-__iter_0__]);
        float __temp_13__ = (__tilevar_5__[__iter_3__+EXTENT-__iter_0__] - __tilevar_5__[__iter_3__-1+EXTENT-__iter_0__]);
        float __temp_14__ = (__temp_12__ * __temp_13__);
        float __temp_15__ = (__temp_11__ + __temp_14__);
        float __temp_16__ = sqrt(__temp_15__);
        float __temp_17__ = (1.000000f / __temp_16__);
        float __temp_18__ = (__tilevar_5__[__iter_3__+EXTENT-__iter_0__] + __temp_17__);
        __var_1__[__iter_3__+(M)*FORMA_MAX(__iter_1__-3,0)] = __temp_18__;
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void gradient (float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N)*(M)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 128;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

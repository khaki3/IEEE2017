#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"
#include <nvml.h>
#include <assert.h>


#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
	template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
	int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
	input[loc] = value;
}


	template<typename T>
void initialize_array(T* d_input, int size, T value)
{
	dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
	dim3 init_block(FORMA_MAX_BLOCKDIM_0);
	__kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(double * __restrict__ input, int N, int M, double * __restrict__ __var_4__){
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_0__;
	__iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
	if(__iter_0__ <= (M-2)){
		int __iter_1__;
		__iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
		if(__iter_1__ <= (N-2)){
			double __temp_0__;
			__temp_0__ = (input[__iter_0__+(M-0)*(__iter_1__)] - input[__iter_0__+(M-0)*(__iter_1__+(-1))]);
			double __temp_1__;
			__temp_1__ = (input[__iter_0__+(M-0)*(__iter_1__)] - input[__iter_0__+(M-0)*(__iter_1__+(-1))]);
			double __temp_2__;
			__temp_2__ = (__temp_0__ * __temp_1__);
			double __temp_3__;
			__temp_3__ = (0.000100f + __temp_2__);
			double __temp_4__;
			__temp_4__ = (input[__iter_0__+(M-0)*(__iter_1__)] - input[__iter_0__+(M-0)*(__iter_1__+(1))]);
			double __temp_5__;
			__temp_5__ = (input[__iter_0__+(M-0)*(__iter_1__)] - input[__iter_0__+(M-0)*(__iter_1__+(1))]);
			double __temp_6__;
			__temp_6__ = (__temp_4__ * __temp_5__);
			double __temp_7__;
			__temp_7__ = (__temp_3__ + __temp_6__);
			double __temp_8__;
			__temp_8__ = (input[__iter_0__+(M-0)*(__iter_1__)] - input[__iter_0__+(1)+(M-0)*(__iter_1__)]);
			double __temp_9__;
			__temp_9__ = (input[__iter_0__+(M-0)*(__iter_1__)] - input[__iter_0__+(1)+(M-0)*(__iter_1__)]);
			double __temp_10__;
			__temp_10__ = (__temp_8__ * __temp_9__);
			double __temp_11__;
			__temp_11__ = (__temp_7__ + __temp_10__);
			double __temp_12__;
			__temp_12__ = (input[__iter_0__+(M-0)*(__iter_1__)] - input[__iter_0__+(-1)+(M-0)*(__iter_1__)]);
			double __temp_13__;
			__temp_13__ = (input[__iter_0__+(M-0)*(__iter_1__)] - input[__iter_0__+(-1)+(M-0)*(__iter_1__)]);
			double __temp_14__;
			__temp_14__ = (__temp_12__ * __temp_13__);
			double __temp_15__;
			__temp_15__ = (__temp_11__ + __temp_14__);
			double __temp_16__;
			__temp_16__ = sqrt(__temp_15__);
			double __temp_17__;
			__temp_17__ = (1.000000f / __temp_16__);
			double __temp_18__;
			__temp_18__ = (input[__iter_0__+(M-0)*(__iter_1__)] + __temp_17__);
			__var_4__[__iter_0__+(M-0)*(__iter_1__)] = __temp_18__;
		}
	}
}
__global__ void __kernel___forma_kernel__1__(double * __restrict__ __var_4__, int N, int M, double * __restrict__ __var_3__){
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_2__;
	__iter_2__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
	if(__iter_2__ <= (M-2)){
		int __iter_3__;
		__iter_3__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
		if(__iter_3__ <= (N-2)){
			double __temp_19__;
			__temp_19__ = (__var_4__[__iter_2__+(M-0)*(__iter_3__)] - __var_4__[__iter_2__+(M-0)*(__iter_3__+(-1))]);
			double __temp_20__;
			__temp_20__ = (__var_4__[__iter_2__+(M-0)*(__iter_3__)] - __var_4__[__iter_2__+(M-0)*(__iter_3__+(-1))]);
			double __temp_21__;
			__temp_21__ = (__temp_19__ * __temp_20__);
			double __temp_22__;
			__temp_22__ = (0.000100f + __temp_21__);
			double __temp_23__;
			__temp_23__ = (__var_4__[__iter_2__+(M-0)*(__iter_3__)] - __var_4__[__iter_2__+(M-0)*(__iter_3__+(1))]);
			double __temp_24__;
			__temp_24__ = (__var_4__[__iter_2__+(M-0)*(__iter_3__)] - __var_4__[__iter_2__+(M-0)*(__iter_3__+(1))]);
			double __temp_25__;
			__temp_25__ = (__temp_23__ * __temp_24__);
			double __temp_26__;
			__temp_26__ = (__temp_22__ + __temp_25__);
			double __temp_27__;
			__temp_27__ = (__var_4__[__iter_2__+(M-0)*(__iter_3__)] - __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__)]);
			double __temp_28__;
			__temp_28__ = (__var_4__[__iter_2__+(M-0)*(__iter_3__)] - __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__)]);
			double __temp_29__;
			__temp_29__ = (__temp_27__ * __temp_28__);
			double __temp_30__;
			__temp_30__ = (__temp_26__ + __temp_29__);
			double __temp_31__;
			__temp_31__ = (__var_4__[__iter_2__+(M-0)*(__iter_3__)] - __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__)]);
			double __temp_32__;
			__temp_32__ = (__var_4__[__iter_2__+(M-0)*(__iter_3__)] - __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__)]);
			double __temp_33__;
			__temp_33__ = (__temp_31__ * __temp_32__);
			double __temp_34__;
			__temp_34__ = (__temp_30__ + __temp_33__);
			double __temp_35__;
			__temp_35__ = sqrt(__temp_34__);
			double __temp_36__;
			__temp_36__ = (1.000000f / __temp_35__);
			double __temp_37__;
			__temp_37__ = (__var_4__[__iter_2__+(M-0)*(__iter_3__)] + __temp_36__);
			__var_3__[__iter_2__+(M-0)*(__iter_3__)] = __temp_37__;
		}
	}
}
__global__ void __kernel___forma_kernel__2__(double * __restrict__ __var_3__, int N, int M, double * __restrict__ __var_2__){
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_4__;
	__iter_4__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
	if(__iter_4__ <= (M-2)){
		int __iter_5__;
		__iter_5__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
		if(__iter_5__ <= (N-2)){
			double __temp_38__;
			__temp_38__ = (__var_3__[__iter_4__+(M-0)*(__iter_5__)] - __var_3__[__iter_4__+(M-0)*(__iter_5__+(-1))]);
			double __temp_39__;
			__temp_39__ = (__var_3__[__iter_4__+(M-0)*(__iter_5__)] - __var_3__[__iter_4__+(M-0)*(__iter_5__+(-1))]);
			double __temp_40__;
			__temp_40__ = (__temp_38__ * __temp_39__);
			double __temp_41__;
			__temp_41__ = (0.000100f + __temp_40__);
			double __temp_42__;
			__temp_42__ = (__var_3__[__iter_4__+(M-0)*(__iter_5__)] - __var_3__[__iter_4__+(M-0)*(__iter_5__+(1))]);
			double __temp_43__;
			__temp_43__ = (__var_3__[__iter_4__+(M-0)*(__iter_5__)] - __var_3__[__iter_4__+(M-0)*(__iter_5__+(1))]);
			double __temp_44__;
			__temp_44__ = (__temp_42__ * __temp_43__);
			double __temp_45__;
			__temp_45__ = (__temp_41__ + __temp_44__);
			double __temp_46__;
			__temp_46__ = (__var_3__[__iter_4__+(M-0)*(__iter_5__)] - __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__)]);
			double __temp_47__;
			__temp_47__ = (__var_3__[__iter_4__+(M-0)*(__iter_5__)] - __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__)]);
			double __temp_48__;
			__temp_48__ = (__temp_46__ * __temp_47__);
			double __temp_49__;
			__temp_49__ = (__temp_45__ + __temp_48__);
			double __temp_50__;
			__temp_50__ = (__var_3__[__iter_4__+(M-0)*(__iter_5__)] - __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__)]);
			double __temp_51__;
			__temp_51__ = (__var_3__[__iter_4__+(M-0)*(__iter_5__)] - __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__)]);
			double __temp_52__;
			__temp_52__ = (__temp_50__ * __temp_51__);
			double __temp_53__;
			__temp_53__ = (__temp_49__ + __temp_52__);
			double __temp_54__;
			__temp_54__ = sqrt(__temp_53__);
			double __temp_55__;
			__temp_55__ = (1.000000f / __temp_54__);
			double __temp_56__;
			__temp_56__ = (__var_3__[__iter_4__+(M-0)*(__iter_5__)] + __temp_55__);
			__var_2__[__iter_4__+(M-0)*(__iter_5__)] = __temp_56__;
		}
	}
}
__global__ void __kernel___forma_kernel__3__(double * __restrict__ __var_2__, int N, int M, double * __restrict__ __var_1__){
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_6__;
	__iter_6__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
	if(__iter_6__ <= (M-2)){
		int __iter_7__;
		__iter_7__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
		if(__iter_7__ <= (N-2)){
			double __temp_57__;
			__temp_57__ = (__var_2__[__iter_6__+(M-0)*(__iter_7__)] - __var_2__[__iter_6__+(M-0)*(__iter_7__+(-1))]);
			double __temp_58__;
			__temp_58__ = (__var_2__[__iter_6__+(M-0)*(__iter_7__)] - __var_2__[__iter_6__+(M-0)*(__iter_7__+(-1))]);
			double __temp_59__;
			__temp_59__ = (__temp_57__ * __temp_58__);
			double __temp_60__;
			__temp_60__ = (0.000100f + __temp_59__);
			double __temp_61__;
			__temp_61__ = (__var_2__[__iter_6__+(M-0)*(__iter_7__)] - __var_2__[__iter_6__+(M-0)*(__iter_7__+(1))]);
			double __temp_62__;
			__temp_62__ = (__var_2__[__iter_6__+(M-0)*(__iter_7__)] - __var_2__[__iter_6__+(M-0)*(__iter_7__+(1))]);
			double __temp_63__;
			__temp_63__ = (__temp_61__ * __temp_62__);
			double __temp_64__;
			__temp_64__ = (__temp_60__ + __temp_63__);
			double __temp_65__;
			__temp_65__ = (__var_2__[__iter_6__+(M-0)*(__iter_7__)] - __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__)]);
			double __temp_66__;
			__temp_66__ = (__var_2__[__iter_6__+(M-0)*(__iter_7__)] - __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__)]);
			double __temp_67__;
			__temp_67__ = (__temp_65__ * __temp_66__);
			double __temp_68__;
			__temp_68__ = (__temp_64__ + __temp_67__);
			double __temp_69__;
			__temp_69__ = (__var_2__[__iter_6__+(M-0)*(__iter_7__)] - __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__)]);
			double __temp_70__;
			__temp_70__ = (__var_2__[__iter_6__+(M-0)*(__iter_7__)] - __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__)]);
			double __temp_71__;
			__temp_71__ = (__temp_69__ * __temp_70__);
			double __temp_72__;
			__temp_72__ = (__temp_68__ + __temp_71__);
			double __temp_73__;
			__temp_73__ = sqrt(__temp_72__);
			double __temp_74__;
			__temp_74__ = (1.000000f / __temp_73__);
			double __temp_75__;
			__temp_75__ = (__var_2__[__iter_6__+(M-0)*(__iter_7__)] + __temp_74__);
			__var_1__[__iter_6__+(M-0)*(__iter_7__)] = __temp_75__;
		}
	}
}
/*Device code End */
/* Host Code Begin */
extern "C" void gradient(double * h_input, int N, int M, double * __var_0__){

	/* Host allocation Begin */
	double * input;
	hipMalloc(&input,sizeof(double)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : input\n");
	hipPointerAttribute_t ptrAttrib_h_input;
	hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
	if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
		if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
			memcpy_kind_h_input = hipMemcpyDeviceToDevice;
	hipGetLastError();
	if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
		hipMemcpy(input,h_input,sizeof(double)*((N-0)*(M-0)), memcpy_kind_h_input);
	}
	double * __var_1__;
	hipMalloc(&__var_1__,sizeof(double)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : __var_1__\n");
	double * __var_2__;
	hipMalloc(&__var_2__,sizeof(double)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : __var_2__\n");
	double * __var_3__;
	hipMalloc(&__var_3__,sizeof(double)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : __var_3__\n");
	double * __var_4__;
	hipMalloc(&__var_4__,sizeof(double)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : __var_4__\n");
	/*Host Allocation End */
	/* Kernel Launch Begin */
#ifdef _TIMER_
	hipEvent_t _forma_timer_start_,_forma_timer_stop_;
	hipEventCreate(&_forma_timer_start_);
	hipEventCreate(&_forma_timer_stop_);
	hipEventRecord(_forma_timer_start_,0);
#endif
	int __size_0___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
	int __size_1___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
	int __block_0___kernel___forma_kernel__0__ = 16;
	int __block_1___kernel___forma_kernel__0__ = 32;
	dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
	int __SMemSize___kernel___forma_kernel__0__ = 0;
	int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
	int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
	dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<1000; x++) {
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_4__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_4__, N, M, __var_3__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_3__, N, M, __var_2__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, N, M, __var_1__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();
	hipPointerAttribute_t ptrAttrib___var_0__;
	hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
	if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
		if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
			memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
	hipGetLastError();
	hipMemcpy(__var_0__,__var_1__, sizeof(double)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
	hipEventRecord(_forma_timer_stop_,0);
	hipEventSynchronize(_forma_timer_stop_);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
	printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
	hipEventDestroy(_forma_timer_start_);
	hipEventDestroy(_forma_timer_stop_);
#endif
	/*Kernel Launch End */
	/* Host Free Begin */
	hipFree(input);
	hipFree(__var_1__);
	hipFree(__var_2__);
	hipFree(__var_3__);
	hipFree(__var_4__);
}
/*Host Free End*/

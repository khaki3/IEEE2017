#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)
#define epsilon (1.0E-20f)

void check_error (const char* message) {
  hipError_t error = hipGetLastError ();
  if (error != hipSuccess) {
    printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
    exit(-1);
  }
}

__global__ void denoise (double * __restrict__ u, double * __restrict__ f, double * __restrict__ out, double gamma, double r, double sigma2, int L, int M, int N) {
  //Determing the block's indices
  int blockdim_i= (int)(blockDim.x);
  int i0 = (int)(blockIdx.x)*(blockdim_i-4);
  int i = max (i0, 0) + (int)(threadIdx.x);
  int blockdim_j= (int)(blockDim.y);
  int j0 = (int)(blockIdx.y)*(blockdim_j-4);
  int j = max (j0, 0) + (int)(threadIdx.y);

  //Declarations
  double reg_f_m1=0;
  double reg_g_m2=0, __shared__ sh_g_m1[16][32];
  double reg_g_c0=0;
  double reg_u_m2=0, __shared__ sh_u_m1[16][32], __shared__ sh_u_c0[16][32];
  double reg_u_p1=0;

  //Value Initialization
  if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
    reg_u_m2 = u[0 + j*N + i];
    sh_u_m1[j-j0][i-i0] = u[0 + j*N + i];
    sh_u_c0[j-j0][i-i0] = u[1*M*N + j*N + i];
  }

  //Rest of the computation
  for (int k=1; k<=L-2; ++k) {
    //Fetch new plane
    if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
        reg_f_m1 = f[max(k-1,0)*M*N + j*N + i];
        reg_u_p1 = u[(k+1)*M*N + j*N + i];
    }
    __syncthreads ();
    if (j >= max (j0+1, 1) & j <= min (j0+blockdim_j-2, M-2) & i >= max (i0+1, 1) & i <= min (i0+blockdim_i-2, N-2)) {
        reg_g_c0 = (1.0f / sqrt (((((((epsilon + ((sh_u_c0[j-j0][i-i0] - sh_u_c0[j-j0+1][i-i0]) * (sh_u_c0[j-j0][i-i0] - sh_u_c0[j-j0+1][i-i0]))) + ((sh_u_c0[j-j0][i-i0] - sh_u_c0[j-j0-1][i-i0]) * (sh_u_c0[j-j0][i-i0] - sh_u_c0[j-j0-1][i-i0]))) + ((sh_u_c0[j-j0][i-i0] - sh_u_c0[j-j0][i-i0+1]) * (sh_u_c0[j-j0][i-i0] - sh_u_c0[j-j0][i-i0+1]))) + ((sh_u_c0[j-j0][i-i0] - sh_u_c0[j-j0][i-i0-1]) * (sh_u_c0[j-j0][i-i0] - sh_u_c0[j-j0][i-i0-1]))) + ((sh_u_c0[j-j0][i-i0] - reg_u_p1) * (sh_u_c0[j-j0][i-i0] - reg_u_p1))) + ((sh_u_c0[j-j0][i-i0] - sh_u_m1[j-j0][i-i0]) * (sh_u_c0[j-j0][i-i0] - sh_u_m1[j-j0][i-i0])))));
        r = ((sh_u_m1[j-j0][i-i0] * reg_f_m1) / sigma2);
        r = ((r * (2.38944f + (r * (0.950037f + r)))) / (4.65314f + (r * (2.57541f + (r * (1.48937f + r))))));
    }
    __syncthreads ();
    if (j >= max (j0+2, 1) & j <= min (j0+blockdim_j-3, M-2) & i >= max (i0+2, 1) & i <= min (i0+blockdim_i-3, N-2)) {
        out[max(k-1,0)*M*N + j*N + i] = ((sh_u_m1[j-j0][i-i0] + (5.0f * (((((((sh_u_m1[j-j0+1][i-i0] * sh_g_m1[j-j0+1][i-i0]) + (sh_u_m1[j-j0-1][i-i0] * sh_g_m1[j-j0-1][i-i0])) + (sh_u_m1[j-j0][i-i0+1] * sh_g_m1[j-j0][i-i0+1])) + (sh_u_m1[j-j0][i-i0-1] * sh_g_m1[j-j0][i-i0-1])) + (sh_u_c0[j-j0][i-i0] * reg_g_c0)) + (reg_u_m2 * reg_g_m2)) + ((gamma * reg_f_m1) * r)))) / (1.0f + (5.0f * ((((((sh_g_m1[j-j0+1][i-i0] + sh_g_m1[j-j0-1][i-i0]) + sh_g_m1[j-j0][i-i0+1]) + sh_g_m1[j-j0][i-i0-1]) + reg_g_c0) + reg_g_m2) + gamma))));
    }
    __syncthreads ();
    //Value rotation
    if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
        reg_g_m2 = sh_g_m1[j-j0][i-i0];
        sh_g_m1[j-j0][i-i0] = reg_g_c0;
        reg_u_m2 = sh_u_m1[j-j0][i-i0];
        sh_u_m1[j-j0][i-i0] = sh_u_c0[j-j0][i-i0];
        sh_u_c0[j-j0][i-i0] = reg_u_p1;
    }
  }
}

extern "C" void host_code (double *h_u, double *h_f, double gamma, double r, double sigma2, int L, int M, int N) {
  double *u;
  hipMalloc (&u, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for u\n");
  hipMemcpy (u, h_u, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *f;
  hipMalloc (&f, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for f\n");
  hipMemcpy (f, h_f, sizeof(double)*L*M*N, hipMemcpyHostToDevice);
  double *out;
  hipMalloc (&out, sizeof(double)*L*M*N);
  check_error ("Failed to allocate device memory for out\n");

  dim3 blockconfig_1 (32, 16, 1);
  dim3 gridconfig_1 (ceil(N, blockconfig_1.x-4), ceil(M, blockconfig_1.y-4), 1);
  denoise <<<gridconfig_1, blockconfig_1>>> (u, f, out, gamma, r, sigma2, L, M, N);
  denoise <<<gridconfig_1, blockconfig_1>>> (out, f, u, gamma, r, sigma2, L, M, N);
  denoise <<<gridconfig_1, blockconfig_1>>> (u, f, out, gamma, r, sigma2, L, M, N);
  denoise <<<gridconfig_1, blockconfig_1>>> (out, f, u, gamma, r, sigma2, L, M, N);
  hipMemcpy (h_u, u, sizeof(double)*L*M*N, hipMemcpyDeviceToHost);
}

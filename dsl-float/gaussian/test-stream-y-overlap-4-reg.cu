#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define mod(x,y) ( (x) & (y-1))

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float * __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
  float * __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);

  float t2_0=0.0f, t2_1=0.0f, t3_0=0.0f, t3_1=0.0f, t4_0=0.0f, t4_1=0.0f, t5_0=0.0f, t5_1=0.0f, out=0.0f;
  float b2_0=0.0f, b2_1=0.0f, b3_0=0.0f, b3_1=0.0f, b4_0=0.0f, b4_1=0.0f, b5_0=0.0f, b5_1=0.0f;
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X-16);

  //Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
    __tilevar_1__[__iter_3__-__iter_0__] = 0.0f;
    __tilevar_2__[__iter_3__-__iter_0__] = 0.0f;
    __tilevar_3__[__iter_3__-__iter_0__] = 0.0f;
  }
  // Rest of the computation
  for (int __iter_1__ = 0; __iter_1__ <= N-1; __iter_1__++) {
    if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
	__tilevar_0__[__iter_3__-__iter_0__] = input[__iter_3__+M*(__iter_1__)];
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-3))) {
    	// Bottom -2
        float __temp_2__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        float __temp_5__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        float __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        float __temp_9__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        float __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        float __temp_13__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        float __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        float __temp_17__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        float __temp_18__ = (__temp_14__ + 2 * __temp_17__) / 159;
	t2_1 += __temp_18__;
	// Bottom -1	
        float __temp_21__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        float __temp_25__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        float __temp_26__ = (4 * __temp_21__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        float __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        float __temp_33__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        float __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        float __temp_37__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        float __temp_38__ = (__temp_34__ + 4 * __temp_37__) / 159;
	t2_0 += __temp_38__;
	// Mid
        float __temp_41__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        float __temp_45__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        float __temp_46__ = (5 * __temp_41__ + 12 * __temp_45__);
        float __temp_49__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        float __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        float __temp_53__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        float __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        float __temp_57__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        float __temp_58__ = (__temp_54__ + 5 * __temp_57__) / 159;
	b2_0 += __temp_58__;
	// Top +1
        float __temp_61__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        float __temp_65__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        float __temp_66__ = (4 * __temp_61__ + 9 * __temp_65__);
        float __temp_69__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        float __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        float __temp_73__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        float __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        float __temp_77__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        float __temp_78__ = (__temp_74__ + 4 * __temp_77__) / 159;
	b2_1 += __temp_78__;
	// Top +2
        float __temp_81__ = (__tilevar_0__[__iter_3__-2-__iter_0__]);
        float __temp_85__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        float __temp_86__ = (2 * __temp_81__ + 4 * __temp_85__);
        float __temp_89__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        float __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        float __temp_93__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        float __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        float __temp_97__ = (__tilevar_0__[__iter_3__+2-__iter_0__]);
        float __temp_98__ = (__temp_94__ + 2 * __temp_97__) / 159;
        __tilevar_1__[__iter_3__-__iter_0__] += __temp_98__;
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+4),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-3))) {
    	// Bottom -2
        float __temp_2__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        float __temp_5__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        float __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        float __temp_9__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        float __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        float __temp_13__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        float __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        float __temp_17__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        float __temp_18__ = (__temp_14__ + 2 * __temp_17__) / 159;
	t3_1 += __temp_18__;
	// Bottom -1	
        float __temp_21__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        float __temp_25__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        float __temp_26__ = (4 * __temp_21__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        float __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        float __temp_33__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        float __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        float __temp_37__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        float __temp_38__ = (__temp_34__ + 4 * __temp_37__) / 159;
	t3_0 += __temp_38__;
	// Mid
        float __temp_41__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        float __temp_45__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        float __temp_46__ = (5 * __temp_41__ + 12 * __temp_45__);
        float __temp_49__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        float __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        float __temp_53__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        float __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        float __temp_57__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        float __temp_58__ = (__temp_54__ + 5 * __temp_57__) / 159;
	b3_0 += __temp_58__;
	// Top +1
        float __temp_61__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        float __temp_65__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        float __temp_66__ = (4 * __temp_61__ + 9 * __temp_65__);
        float __temp_69__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        float __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        float __temp_73__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        float __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        float __temp_77__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        float __temp_78__ = (__temp_74__ + 4 * __temp_77__) / 159;
	b3_1 += __temp_78__;
	// Top +2
        float __temp_81__ = (__tilevar_1__[__iter_3__-2-__iter_0__]);
        float __temp_85__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        float __temp_86__ = (2 * __temp_81__ + 4 * __temp_85__);
        float __temp_89__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        float __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        float __temp_93__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        float __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        float __temp_97__ = (__tilevar_1__[__iter_3__+2-__iter_0__]);
        float __temp_98__ = (__temp_94__ + 2 * __temp_97__) / 159;
        __tilevar_2__[__iter_3__-__iter_0__] += __temp_98__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+6),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(M-3))) {
    	// Bottom -2
        float __temp_2__ = (__tilevar_2__[__iter_3__-2-__iter_0__]);
        float __temp_5__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        float __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        float __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        float __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        float __temp_13__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        float __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        float __temp_17__ = (__tilevar_2__[__iter_3__+2-__iter_0__]);
        float __temp_18__ = (__temp_14__ + 2 * __temp_17__) / 159;
	t4_1 += __temp_18__;
	// Bottom -1	
        float __temp_21__ = (__tilevar_2__[__iter_3__-2-__iter_0__]);
        float __temp_25__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        float __temp_26__ = (4 * __temp_21__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        float __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        float __temp_33__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        float __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        float __temp_37__ = (__tilevar_2__[__iter_3__+2-__iter_0__]);
        float __temp_38__ = (__temp_34__ + 4 * __temp_37__) / 159;
	t4_0 += __temp_38__;
	// Mid
        float __temp_41__ = (__tilevar_2__[__iter_3__-2-__iter_0__]);
        float __temp_45__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        float __temp_46__ = (5 * __temp_41__ + 12 * __temp_45__);
        float __temp_49__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        float __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        float __temp_53__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        float __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        float __temp_57__ = (__tilevar_2__[__iter_3__+2-__iter_0__]);
        float __temp_58__ = (__temp_54__ + 5 * __temp_57__) / 159;
	b4_0 += __temp_58__;
	// Top +1
        float __temp_61__ = (__tilevar_2__[__iter_3__-2-__iter_0__]);
        float __temp_65__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        float __temp_66__ = (4 * __temp_61__ + 9 * __temp_65__);
        float __temp_69__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        float __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        float __temp_73__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        float __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        float __temp_77__ = (__tilevar_2__[__iter_3__+2-__iter_0__]);
        float __temp_78__ = (__temp_74__ + 4 * __temp_77__) / 159;
	b4_1 += __temp_78__;
	// Top +2
        float __temp_81__ = (__tilevar_2__[__iter_3__-2-__iter_0__]);
        float __temp_85__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
        float __temp_86__ = (2 * __temp_81__ + 4 * __temp_85__);
        float __temp_89__ = (__tilevar_2__[__iter_3__-__iter_0__]);
        float __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        float __temp_93__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
        float __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        float __temp_97__ = (__tilevar_2__[__iter_3__+2-__iter_0__]);
        float __temp_98__ = (__temp_94__ + 2 * __temp_97__) / 159;
        __tilevar_3__[__iter_3__-__iter_0__] += __temp_98__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+8),2) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-9),(M-3))) {
    	// Bottom -2
        float __temp_2__ = (__tilevar_3__[__iter_3__-2-__iter_0__]);
        float __temp_5__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        float __temp_6__ = (2 * __temp_2__ + 4 * __temp_5__);
        float __temp_9__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        float __temp_10__ = (__temp_6__ + 5 * __temp_9__);
        float __temp_13__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        float __temp_14__ = (__temp_10__ + 4 * __temp_13__);
        float __temp_17__ = (__tilevar_3__[__iter_3__+2-__iter_0__]);
        float __temp_18__ = (__temp_14__ + 2 * __temp_17__) / 159;
	t5_1 += __temp_18__;
	// Bottom -1	
        float __temp_21__ = (__tilevar_3__[__iter_3__-2-__iter_0__]);
        float __temp_25__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        float __temp_26__ = (4 * __temp_21__ + 9 * __temp_25__);
        float __temp_29__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        float __temp_30__ = (__temp_26__ + 12 * __temp_29__);
        float __temp_33__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        float __temp_34__ = (__temp_30__ + 9 * __temp_33__);
        float __temp_37__ = (__tilevar_3__[__iter_3__+2-__iter_0__]);
        float __temp_38__ = (__temp_34__ + 4 * __temp_37__) / 159;
	t5_0 += __temp_38__;
	// Mid
        float __temp_41__ = (__tilevar_3__[__iter_3__-2-__iter_0__]);
        float __temp_45__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        float __temp_46__ = (5 * __temp_41__ + 12 * __temp_45__);
        float __temp_49__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        float __temp_50__ = (__temp_46__ + 15 * __temp_49__);
        float __temp_53__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        float __temp_54__ = (__temp_50__ + 12 * __temp_53__);
        float __temp_57__ = (__tilevar_3__[__iter_3__+2-__iter_0__]);
        float __temp_58__ = (__temp_54__ + 5 * __temp_57__) / 159;
	b5_0 += __temp_58__;
	// Top +1
        float __temp_61__ = (__tilevar_3__[__iter_3__-2-__iter_0__]);
        float __temp_65__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        float __temp_66__ = (4 * __temp_61__ + 9 * __temp_65__);
        float __temp_69__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        float __temp_70__ = (__temp_66__ + 12 * __temp_69__);
        float __temp_73__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        float __temp_74__ = (__temp_70__ + 9 * __temp_73__);
        float __temp_77__ = (__tilevar_3__[__iter_3__+2-__iter_0__]);
        float __temp_78__ = (__temp_74__ + 4 * __temp_77__) / 159;
	b5_1 += __temp_78__;
	// Top +2
        float __temp_81__ = (__tilevar_3__[__iter_3__-2-__iter_0__]);
        float __temp_85__ = (__tilevar_3__[__iter_3__-1-__iter_0__]);
        float __temp_86__ = (2 * __temp_81__ + 4 * __temp_85__);
        float __temp_89__ = (__tilevar_3__[__iter_3__-__iter_0__]);
        float __temp_90__ = (__temp_86__ + 5 * __temp_89__);
        float __temp_93__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
        float __temp_94__ = (__temp_90__ + 4 * __temp_93__);
        float __temp_97__ = (__tilevar_3__[__iter_3__+2-__iter_0__]);
        float __temp_98__ = (__temp_94__ + 2 * __temp_97__) / 159;
	out += __temp_98__;
	__var_1__[__iter_3__+M*FORMA_MAX(__iter_1__-8,0)] = out;
    }
    __syncthreads();
    // Now rotate
    __tilevar_1__[__iter_3__-__iter_0__] = b2_1; b2_1 = b2_0; b2_0=t2_0; t2_0=t2_1; t2_1=0.0f; 
    __tilevar_2__[__iter_3__-__iter_0__] = b3_1; b3_1 = b3_0; b3_0=t3_0; t3_0=t3_1; t3_1=0.0f;
    __tilevar_3__[__iter_3__-__iter_0__] = b4_1; b4_1 = b4_0; b4_0=t4_0; t4_0=t4_1; t4_1=0.0f;
    out=b5_1; b5_1=b5_0; b5_0=t5_0; t5_0=t5_1; t5_1=0.0f;
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(4*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void gaussian(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N)*(M)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 128;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-16);
  int __grid_1___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(0-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((__iter_1__+2),2) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__+2),2) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-3)) ){
      int __temp_0__;
      __temp_0__ = __iter_4__+(-2);
      int __temp_1__;
      __temp_1__ = __iter_5__+(-2);
      float __temp_2__;
      __temp_2__ = (2 * __tilevar_2__[__temp_1__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_0__+(0-(__iter_1__+0)))]);
      int __temp_3__;
      __temp_3__ = __iter_4__+(-2);
      int __temp_4__;
      __temp_4__ = __iter_5__+(-1);
      float __temp_5__;
      __temp_5__ = (4 * __tilevar_2__[__temp_4__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_3__+(0-(__iter_1__+0)))]);
      float __temp_6__;
      __temp_6__ = (__temp_2__ + __temp_5__);
      int __temp_7__;
      __temp_7__ = __iter_4__+(-2);
      int __temp_8__;
      __temp_8__ = __iter_5__;
      float __temp_9__;
      __temp_9__ = (5 * __tilevar_2__[__temp_8__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_7__+(0-(__iter_1__+0)))]);
      float __temp_10__;
      __temp_10__ = (__temp_6__ + __temp_9__);
      int __temp_11__;
      __temp_11__ = __iter_4__+(-2);
      int __temp_12__;
      __temp_12__ = __iter_5__+(1);
      float __temp_13__;
      __temp_13__ = (4 * __tilevar_2__[__temp_12__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_11__+(0-(__iter_1__+0)))]);
      float __temp_14__;
      __temp_14__ = (__temp_10__ + __temp_13__);
      int __temp_15__;
      __temp_15__ = __iter_4__+(-2);
      int __temp_16__;
      __temp_16__ = __iter_5__+(2);
      float __temp_17__;
      __temp_17__ = (2 * __tilevar_2__[__temp_16__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_15__+(0-(__iter_1__+0)))]);
      float __temp_18__;
      __temp_18__ = (__temp_14__ + __temp_17__);
      int __temp_19__;
      __temp_19__ = __iter_4__+(-1);
      int __temp_20__;
      __temp_20__ = __iter_5__+(-2);
      float __temp_21__;
      __temp_21__ = (4 * __tilevar_2__[__temp_20__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_19__+(0-(__iter_1__+0)))]);
      float __temp_22__;
      __temp_22__ = (__temp_18__ + __temp_21__);
      int __temp_23__;
      __temp_23__ = __iter_4__+(-1);
      int __temp_24__;
      __temp_24__ = __iter_5__+(-1);
      float __temp_25__;
      __temp_25__ = (9 * __tilevar_2__[__temp_24__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_23__+(0-(__iter_1__+0)))]);
      float __temp_26__;
      __temp_26__ = (__temp_22__ + __temp_25__);
      int __temp_27__;
      __temp_27__ = __iter_4__+(-1);
      int __temp_28__;
      __temp_28__ = __iter_5__;
      float __temp_29__;
      __temp_29__ = (12 * __tilevar_2__[__temp_28__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_27__+(0-(__iter_1__+0)))]);
      float __temp_30__;
      __temp_30__ = (__temp_26__ + __temp_29__);
      int __temp_31__;
      __temp_31__ = __iter_4__+(-1);
      int __temp_32__;
      __temp_32__ = __iter_5__+(1);
      float __temp_33__;
      __temp_33__ = (9 * __tilevar_2__[__temp_32__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_31__+(0-(__iter_1__+0)))]);
      float __temp_34__;
      __temp_34__ = (__temp_30__ + __temp_33__);
      int __temp_35__;
      __temp_35__ = __iter_4__+(-1);
      int __temp_36__;
      __temp_36__ = __iter_5__+(2);
      float __temp_37__;
      __temp_37__ = (4 * __tilevar_2__[__temp_36__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_35__+(0-(__iter_1__+0)))]);
      float __temp_38__;
      __temp_38__ = (__temp_34__ + __temp_37__);
      int __temp_39__;
      __temp_39__ = __iter_4__;
      int __temp_40__;
      __temp_40__ = __iter_5__+(-2);
      float __temp_41__;
      __temp_41__ = (5 * __tilevar_2__[__temp_40__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_39__+(0-(__iter_1__+0)))]);
      float __temp_42__;
      __temp_42__ = (__temp_38__ + __temp_41__);
      int __temp_43__;
      __temp_43__ = __iter_4__;
      int __temp_44__;
      __temp_44__ = __iter_5__+(-1);
      float __temp_45__;
      __temp_45__ = (12 * __tilevar_2__[__temp_44__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_43__+(0-(__iter_1__+0)))]);
      float __temp_46__;
      __temp_46__ = (__temp_42__ + __temp_45__);
      int __temp_47__;
      __temp_47__ = __iter_4__;
      int __temp_48__;
      __temp_48__ = __iter_5__;
      float __temp_49__;
      __temp_49__ = (15 * __tilevar_2__[__temp_48__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_47__+(0-(__iter_1__+0)))]);
      float __temp_50__;
      __temp_50__ = (__temp_46__ + __temp_49__);
      int __temp_51__;
      __temp_51__ = __iter_4__;
      int __temp_52__;
      __temp_52__ = __iter_5__+(1);
      float __temp_53__;
      __temp_53__ = (12 * __tilevar_2__[__temp_52__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_51__+(0-(__iter_1__+0)))]);
      float __temp_54__;
      __temp_54__ = (__temp_50__ + __temp_53__);
      int __temp_55__;
      __temp_55__ = __iter_4__;
      int __temp_56__;
      __temp_56__ = __iter_5__+(2);
      float __temp_57__;
      __temp_57__ = (5 * __tilevar_2__[__temp_56__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_55__+(0-(__iter_1__+0)))]);
      float __temp_58__;
      __temp_58__ = (__temp_54__ + __temp_57__);
      int __temp_59__;
      __temp_59__ = __iter_4__+(1);
      int __temp_60__;
      __temp_60__ = __iter_5__+(-2);
      float __temp_61__;
      __temp_61__ = (4 * __tilevar_2__[__temp_60__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_59__+(0-(__iter_1__+0)))]);
      float __temp_62__;
      __temp_62__ = (__temp_58__ + __temp_61__);
      int __temp_63__;
      __temp_63__ = __iter_4__+(1);
      int __temp_64__;
      __temp_64__ = __iter_5__+(-1);
      float __temp_65__;
      __temp_65__ = (9 * __tilevar_2__[__temp_64__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_63__+(0-(__iter_1__+0)))]);
      float __temp_66__;
      __temp_66__ = (__temp_62__ + __temp_65__);
      int __temp_67__;
      __temp_67__ = __iter_4__+(1);
      int __temp_68__;
      __temp_68__ = __iter_5__;
      float __temp_69__;
      __temp_69__ = (12 * __tilevar_2__[__temp_68__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_67__+(0-(__iter_1__+0)))]);
      float __temp_70__;
      __temp_70__ = (__temp_66__ + __temp_69__);
      int __temp_71__;
      __temp_71__ = __iter_4__+(1);
      int __temp_72__;
      __temp_72__ = __iter_5__+(1);
      float __temp_73__;
      __temp_73__ = (9 * __tilevar_2__[__temp_72__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_71__+(0-(__iter_1__+0)))]);
      float __temp_74__;
      __temp_74__ = (__temp_70__ + __temp_73__);
      int __temp_75__;
      __temp_75__ = __iter_4__+(1);
      int __temp_76__;
      __temp_76__ = __iter_5__+(2);
      float __temp_77__;
      __temp_77__ = (4 * __tilevar_2__[__temp_76__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_75__+(0-(__iter_1__+0)))]);
      float __temp_78__;
      __temp_78__ = (__temp_74__ + __temp_77__);
      int __temp_79__;
      __temp_79__ = __iter_4__+(2);
      int __temp_80__;
      __temp_80__ = __iter_5__+(-2);
      float __temp_81__;
      __temp_81__ = (2 * __tilevar_2__[__temp_80__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_79__+(0-(__iter_1__+0)))]);
      float __temp_82__;
      __temp_82__ = (__temp_78__ + __temp_81__);
      int __temp_83__;
      __temp_83__ = __iter_4__+(2);
      int __temp_84__;
      __temp_84__ = __iter_5__+(-1);
      float __temp_85__;
      __temp_85__ = (4 * __tilevar_2__[__temp_84__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_83__+(0-(__iter_1__+0)))]);
      float __temp_86__;
      __temp_86__ = (__temp_82__ + __temp_85__);
      int __temp_87__;
      __temp_87__ = __iter_4__+(2);
      int __temp_88__;
      __temp_88__ = __iter_5__;
      float __temp_89__;
      __temp_89__ = (5 * __tilevar_2__[__temp_88__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_87__+(0-(__iter_1__+0)))]);
      float __temp_90__;
      __temp_90__ = (__temp_86__ + __temp_89__);
      int __temp_91__;
      __temp_91__ = __iter_4__+(2);
      int __temp_92__;
      __temp_92__ = __iter_5__+(1);
      float __temp_93__;
      __temp_93__ = (4 * __tilevar_2__[__temp_92__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_91__+(0-(__iter_1__+0)))]);
      float __temp_94__;
      __temp_94__ = (__temp_90__ + __temp_93__);
      int __temp_95__;
      __temp_95__ = __iter_4__+(2);
      int __temp_96__;
      __temp_96__ = __iter_5__+(2);
      float __temp_97__;
      __temp_97__ = (2 * __tilevar_2__[__temp_96__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_95__+(0-(__iter_1__+0)))]);
      float __temp_98__;
      __temp_98__ = (__temp_94__ + __temp_97__);
      float __temp_99__;
      __temp_99__ = (__temp_98__ / 159);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+2)))] = __temp_99__;
    }
  }
  int __iter_6__;
  __iter_6__ = FORMA_MAX((__iter_1__+2),2) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_0__+2),2) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-3)) ){
      if (__iter_6__ < (FORMA_MAX((__iter_1__+2),2)+4) || __iter_6__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3))-4) || __iter_7__ < (FORMA_MAX((__iter_0__+2),2)+4) || __iter_7__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-3))-4)) {
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+2)))];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__+4),2) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-3)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__+4),2) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-3)) ){
      int __temp_150__;
      __temp_150__ = __iter_10__+(-2);
      int __temp_151__;
      __temp_151__ = __iter_11__+(-2);
      float __temp_152__;
      __temp_152__ = (2 * __tilevar_3__[__temp_151__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_150__+(0-(__iter_1__+2)))]);
      int __temp_153__;
      __temp_153__ = __iter_10__+(-2);
      int __temp_154__;
      __temp_154__ = __iter_11__+(-1);
      float __temp_155__;
      __temp_155__ = (4 * __tilevar_3__[__temp_154__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_153__+(0-(__iter_1__+2)))]);
      float __temp_156__;
      __temp_156__ = (__temp_152__ + __temp_155__);
      int __temp_157__;
      __temp_157__ = __iter_10__+(-2);
      int __temp_158__;
      __temp_158__ = __iter_11__;
      float __temp_159__;
      __temp_159__ = (5 * __tilevar_3__[__temp_158__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_157__+(0-(__iter_1__+2)))]);
      float __temp_160__;
      __temp_160__ = (__temp_156__ + __temp_159__);
      int __temp_161__;
      __temp_161__ = __iter_10__+(-2);
      int __temp_162__;
      __temp_162__ = __iter_11__+(1);
      float __temp_163__;
      __temp_163__ = (4 * __tilevar_3__[__temp_162__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_161__+(0-(__iter_1__+2)))]);
      float __temp_164__;
      __temp_164__ = (__temp_160__ + __temp_163__);
      int __temp_165__;
      __temp_165__ = __iter_10__+(-2);
      int __temp_166__;
      __temp_166__ = __iter_11__+(2);
      float __temp_167__;
      __temp_167__ = (2 * __tilevar_3__[__temp_166__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_165__+(0-(__iter_1__+2)))]);
      float __temp_168__;
      __temp_168__ = (__temp_164__ + __temp_167__);
      int __temp_169__;
      __temp_169__ = __iter_10__+(-1);
      int __temp_170__;
      __temp_170__ = __iter_11__+(-2);
      float __temp_171__;
      __temp_171__ = (4 * __tilevar_3__[__temp_170__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_169__+(0-(__iter_1__+2)))]);
      float __temp_172__;
      __temp_172__ = (__temp_168__ + __temp_171__);
      int __temp_173__;
      __temp_173__ = __iter_10__+(-1);
      int __temp_174__;
      __temp_174__ = __iter_11__+(-1);
      float __temp_175__;
      __temp_175__ = (9 * __tilevar_3__[__temp_174__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_173__+(0-(__iter_1__+2)))]);
      float __temp_176__;
      __temp_176__ = (__temp_172__ + __temp_175__);
      int __temp_177__;
      __temp_177__ = __iter_10__+(-1);
      int __temp_178__;
      __temp_178__ = __iter_11__;
      float __temp_179__;
      __temp_179__ = (12 * __tilevar_3__[__temp_178__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_177__+(0-(__iter_1__+2)))]);
      float __temp_180__;
      __temp_180__ = (__temp_176__ + __temp_179__);
      int __temp_181__;
      __temp_181__ = __iter_10__+(-1);
      int __temp_182__;
      __temp_182__ = __iter_11__+(1);
      float __temp_183__;
      __temp_183__ = (9 * __tilevar_3__[__temp_182__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_181__+(0-(__iter_1__+2)))]);
      float __temp_184__;
      __temp_184__ = (__temp_180__ + __temp_183__);
      int __temp_185__;
      __temp_185__ = __iter_10__+(-1);
      int __temp_186__;
      __temp_186__ = __iter_11__+(2);
      float __temp_187__;
      __temp_187__ = (4 * __tilevar_3__[__temp_186__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_185__+(0-(__iter_1__+2)))]);
      float __temp_188__;
      __temp_188__ = (__temp_184__ + __temp_187__);
      int __temp_189__;
      __temp_189__ = __iter_10__;
      int __temp_190__;
      __temp_190__ = __iter_11__+(-2);
      float __temp_191__;
      __temp_191__ = (5 * __tilevar_3__[__temp_190__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_189__+(0-(__iter_1__+2)))]);
      float __temp_192__;
      __temp_192__ = (__temp_188__ + __temp_191__);
      int __temp_193__;
      __temp_193__ = __iter_10__;
      int __temp_194__;
      __temp_194__ = __iter_11__+(-1);
      float __temp_195__;
      __temp_195__ = (12 * __tilevar_3__[__temp_194__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_193__+(0-(__iter_1__+2)))]);
      float __temp_196__;
      __temp_196__ = (__temp_192__ + __temp_195__);
      int __temp_197__;
      __temp_197__ = __iter_10__;
      int __temp_198__;
      __temp_198__ = __iter_11__;
      float __temp_199__;
      __temp_199__ = (15 * __tilevar_3__[__temp_198__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_197__+(0-(__iter_1__+2)))]);
      float __temp_200__;
      __temp_200__ = (__temp_196__ + __temp_199__);
      int __temp_201__;
      __temp_201__ = __iter_10__;
      int __temp_202__;
      __temp_202__ = __iter_11__+(1);
      float __temp_203__;
      __temp_203__ = (12 * __tilevar_3__[__temp_202__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_201__+(0-(__iter_1__+2)))]);
      float __temp_204__;
      __temp_204__ = (__temp_200__ + __temp_203__);
      int __temp_205__;
      __temp_205__ = __iter_10__;
      int __temp_206__;
      __temp_206__ = __iter_11__+(2);
      float __temp_207__;
      __temp_207__ = (5 * __tilevar_3__[__temp_206__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_205__+(0-(__iter_1__+2)))]);
      float __temp_208__;
      __temp_208__ = (__temp_204__ + __temp_207__);
      int __temp_209__;
      __temp_209__ = __iter_10__+(1);
      int __temp_210__;
      __temp_210__ = __iter_11__+(-2);
      float __temp_211__;
      __temp_211__ = (4 * __tilevar_3__[__temp_210__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_209__+(0-(__iter_1__+2)))]);
      float __temp_212__;
      __temp_212__ = (__temp_208__ + __temp_211__);
      int __temp_213__;
      __temp_213__ = __iter_10__+(1);
      int __temp_214__;
      __temp_214__ = __iter_11__+(-1);
      float __temp_215__;
      __temp_215__ = (9 * __tilevar_3__[__temp_214__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_213__+(0-(__iter_1__+2)))]);
      float __temp_216__;
      __temp_216__ = (__temp_212__ + __temp_215__);
      int __temp_217__;
      __temp_217__ = __iter_10__+(1);
      int __temp_218__;
      __temp_218__ = __iter_11__;
      float __temp_219__;
      __temp_219__ = (12 * __tilevar_3__[__temp_218__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_217__+(0-(__iter_1__+2)))]);
      float __temp_220__;
      __temp_220__ = (__temp_216__ + __temp_219__);
      int __temp_221__;
      __temp_221__ = __iter_10__+(1);
      int __temp_222__;
      __temp_222__ = __iter_11__+(1);
      float __temp_223__;
      __temp_223__ = (9 * __tilevar_3__[__temp_222__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_221__+(0-(__iter_1__+2)))]);
      float __temp_224__;
      __temp_224__ = (__temp_220__ + __temp_223__);
      int __temp_225__;
      __temp_225__ = __iter_10__+(1);
      int __temp_226__;
      __temp_226__ = __iter_11__+(2);
      float __temp_227__;
      __temp_227__ = (4 * __tilevar_3__[__temp_226__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_225__+(0-(__iter_1__+2)))]);
      float __temp_228__;
      __temp_228__ = (__temp_224__ + __temp_227__);
      int __temp_229__;
      __temp_229__ = __iter_10__+(2);
      int __temp_230__;
      __temp_230__ = __iter_11__+(-2);
      float __temp_231__;
      __temp_231__ = (2 * __tilevar_3__[__temp_230__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_229__+(0-(__iter_1__+2)))]);
      float __temp_232__;
      __temp_232__ = (__temp_228__ + __temp_231__);
      int __temp_233__;
      __temp_233__ = __iter_10__+(2);
      int __temp_234__;
      __temp_234__ = __iter_11__+(-1);
      float __temp_235__;
      __temp_235__ = (4 * __tilevar_3__[__temp_234__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_233__+(0-(__iter_1__+2)))]);
      float __temp_236__;
      __temp_236__ = (__temp_232__ + __temp_235__);
      int __temp_237__;
      __temp_237__ = __iter_10__+(2);
      int __temp_238__;
      __temp_238__ = __iter_11__;
      float __temp_239__;
      __temp_239__ = (5 * __tilevar_3__[__temp_238__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_237__+(0-(__iter_1__+2)))]);
      float __temp_240__;
      __temp_240__ = (__temp_236__ + __temp_239__);
      int __temp_241__;
      __temp_241__ = __iter_10__+(2);
      int __temp_242__;
      __temp_242__ = __iter_11__+(1);
      float __temp_243__;
      __temp_243__ = (4 * __tilevar_3__[__temp_242__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_241__+(0-(__iter_1__+2)))]);
      float __temp_244__;
      __temp_244__ = (__temp_240__ + __temp_243__);
      int __temp_245__;
      __temp_245__ = __iter_10__+(2);
      int __temp_246__;
      __temp_246__ = __iter_11__+(2);
      float __temp_247__;
      __temp_247__ = (2 * __tilevar_3__[__temp_246__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__temp_245__+(0-(__iter_1__+2)))]);
      float __temp_248__;
      __temp_248__ = (__temp_244__ + __temp_247__);
      float __temp_249__;
      __temp_249__ = (__temp_248__ / 159);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+4)))] = __temp_249__;
    }
  }
  int __iter_12__;
  __iter_12__ = FORMA_MAX((__iter_1__+4),2) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-3)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((__iter_0__+4),2) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-3)) ){
      if (__iter_12__ < (FORMA_MAX((__iter_1__+4),2)+4) || __iter_12__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-3))-4) || __iter_13__ < (FORMA_MAX((__iter_0__+4),2)+4) || __iter_13__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-3))-4)) {
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+4)))];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__+6),2) + (int)(threadIdx.y) ; 
  if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-7),(N-3)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__+6),2) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(M-3)) ){
      float __temp_300__;
      __temp_300__ = (2 * __tilevar_4__[__iter_17__+(-2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-2)+(0-(__iter_1__+4)))]);
      float __temp_301__;
      __temp_301__ = (4 * __tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-2)+(0-(__iter_1__+4)))]);
      float __temp_302__;
      __temp_302__ = (__temp_300__ + __temp_301__);
      float __temp_303__;
      __temp_303__ = (5 * __tilevar_4__[__iter_17__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-2)+(0-(__iter_1__+4)))]);
      float __temp_304__;
      __temp_304__ = (__temp_302__ + __temp_303__);
      float __temp_305__;
      __temp_305__ = (4 * __tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-2)+(0-(__iter_1__+4)))]);
      float __temp_306__;
      __temp_306__ = (__temp_304__ + __temp_305__);
      float __temp_307__;
      __temp_307__ = (2 * __tilevar_4__[__iter_17__+(2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-2)+(0-(__iter_1__+4)))]);
      float __temp_308__;
      __temp_308__ = (__temp_306__ + __temp_307__);
      float __temp_309__;
      __temp_309__ = (4 * __tilevar_4__[__iter_17__+(-2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+4)))]);
      float __temp_310__;
      __temp_310__ = (__temp_308__ + __temp_309__);
      float __temp_311__;
      __temp_311__ = (9 * __tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+4)))]);
      float __temp_312__;
      __temp_312__ = (__temp_310__ + __temp_311__);
      float __temp_313__;
      __temp_313__ = (12 * __tilevar_4__[__iter_17__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+4)))]);
      float __temp_314__;
      __temp_314__ = (__temp_312__ + __temp_313__);
      float __temp_315__;
      __temp_315__ = (9 * __tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+4)))]);
      float __temp_316__;
      __temp_316__ = (__temp_314__ + __temp_315__);
      float __temp_317__;
      __temp_317__ = (4 * __tilevar_4__[__iter_17__+(2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+4)))]);
      float __temp_318__;
      __temp_318__ = (__temp_316__ + __temp_317__);
      float __temp_319__;
      __temp_319__ = (5 * __tilevar_4__[__iter_17__+(-2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+4)))]);
      float __temp_320__;
      __temp_320__ = (__temp_318__ + __temp_319__);
      float __temp_321__;
      __temp_321__ = (12 * __tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+4)))]);
      float __temp_322__;
      __temp_322__ = (__temp_320__ + __temp_321__);
      float __temp_323__;
      __temp_323__ = (15 * __tilevar_4__[__iter_17__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+4)))]);
      float __temp_324__;
      __temp_324__ = (__temp_322__ + __temp_323__);
      float __temp_325__;
      __temp_325__ = (12 * __tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+4)))]);
      float __temp_326__;
      __temp_326__ = (__temp_324__ + __temp_325__);
      float __temp_327__;
      __temp_327__ = (5 * __tilevar_4__[__iter_17__+(2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+4)))]);
      float __temp_328__;
      __temp_328__ = (__temp_326__ + __temp_327__);
      float __temp_329__;
      __temp_329__ = (4 * __tilevar_4__[__iter_17__+(-2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+4)))]);
      float __temp_330__;
      __temp_330__ = (__temp_328__ + __temp_329__);
      float __temp_331__;
      __temp_331__ = (9 * __tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+4)))]);
      float __temp_332__;
      __temp_332__ = (__temp_330__ + __temp_331__);
      float __temp_333__;
      __temp_333__ = (12 * __tilevar_4__[__iter_17__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+4)))]);
      float __temp_334__;
      __temp_334__ = (__temp_332__ + __temp_333__);
      float __temp_335__;
      __temp_335__ = (9 * __tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+4)))]);
      float __temp_336__;
      __temp_336__ = (__temp_334__ + __temp_335__);
      float __temp_337__;
      __temp_337__ = (4 * __tilevar_4__[__iter_17__+(2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+4)))]);
      float __temp_338__;
      __temp_338__ = (__temp_336__ + __temp_337__);
      float __temp_339__;
      __temp_339__ = (2 * __tilevar_4__[__iter_17__+(-2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+4)))]);
      float __temp_340__;
      __temp_340__ = (__temp_338__ + __temp_339__);
      float __temp_341__;
      __temp_341__ = (4 * __tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+4)))]);
      float __temp_342__;
      __temp_342__ = (__temp_340__ + __temp_341__);
      float __temp_343__;
      __temp_343__ = (5 * __tilevar_4__[__iter_17__+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+4)))]);
      float __temp_344__;
      __temp_344__ = (__temp_342__ + __temp_343__);
      float __temp_345__;
      __temp_345__ = (4 * __tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+4)))]);
      float __temp_346__;
      __temp_346__ = (__temp_344__ + __temp_345__);
      float __temp_347__;
      __temp_347__ = (2 * __tilevar_4__[__iter_17__+(2)+(0-(__iter_0__+4))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(2)+(0-(__iter_1__+4)))]);
      float __temp_348__;
      __temp_348__ = (__temp_346__ + __temp_347__);
      float __temp_349__;
      __temp_349__ = (__temp_348__ / 159);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+6)))] = __temp_349__;
    }
  }
  int __iter_18__;
  __iter_18__ = FORMA_MAX((__iter_1__+6),2) + (int)(threadIdx.y) ; 
  if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-7),(N-3)) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_0__+6),2) + (int)(threadIdx.x) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(M-3)) ){
      if (__iter_18__ < (FORMA_MAX((__iter_1__+6),2)+4) || __iter_18__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-7),(N-3))-4) || __iter_19__ < (FORMA_MAX((__iter_0__+6),2)+4) || __iter_19__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(M-3))-4)) {
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+6)))];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__+8),2) + (int)(threadIdx.y) ; 
  if( __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-9),(N-3)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__+8),2) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-9),(M-3)) ){
      float __temp_400__;
      __temp_400__ = (2 * __tilevar_5__[__iter_23__+(-2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-2)+(0-(__iter_1__+6)))]);
      float __temp_401__;
      __temp_401__ = (4 * __tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-2)+(0-(__iter_1__+6)))]);
      float __temp_402__;
      __temp_402__ = (__temp_400__ + __temp_401__);
      float __temp_403__;
      __temp_403__ = (5 * __tilevar_5__[__iter_23__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-2)+(0-(__iter_1__+6)))]);
      float __temp_404__;
      __temp_404__ = (__temp_402__ + __temp_403__);
      float __temp_405__;
      __temp_405__ = (4 * __tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-2)+(0-(__iter_1__+6)))]);
      float __temp_406__;
      __temp_406__ = (__temp_404__ + __temp_405__);
      float __temp_407__;
      __temp_407__ = (2 * __tilevar_5__[__iter_23__+(2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-2)+(0-(__iter_1__+6)))]);
      float __temp_408__;
      __temp_408__ = (__temp_406__ + __temp_407__);
      float __temp_409__;
      __temp_409__ = (4 * __tilevar_5__[__iter_23__+(-2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+6)))]);
      float __temp_410__;
      __temp_410__ = (__temp_408__ + __temp_409__);
      float __temp_411__;
      __temp_411__ = (9 * __tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+6)))]);
      float __temp_412__;
      __temp_412__ = (__temp_410__ + __temp_411__);
      float __temp_413__;
      __temp_413__ = (12 * __tilevar_5__[__iter_23__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+6)))]);
      float __temp_414__;
      __temp_414__ = (__temp_412__ + __temp_413__);
      float __temp_415__;
      __temp_415__ = (9 * __tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+6)))]);
      float __temp_416__;
      __temp_416__ = (__temp_414__ + __temp_415__);
      float __temp_417__;
      __temp_417__ = (4 * __tilevar_5__[__iter_23__+(2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+6)))]);
      float __temp_418__;
      __temp_418__ = (__temp_416__ + __temp_417__);
      float __temp_419__;
      __temp_419__ = (5 * __tilevar_5__[__iter_23__+(-2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+6)))]);
      float __temp_420__;
      __temp_420__ = (__temp_418__ + __temp_419__);
      float __temp_421__;
      __temp_421__ = (12 * __tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+6)))]);
      float __temp_422__;
      __temp_422__ = (__temp_420__ + __temp_421__);
      float __temp_423__;
      __temp_423__ = (15 * __tilevar_5__[__iter_23__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+6)))]);
      float __temp_424__;
      __temp_424__ = (__temp_422__ + __temp_423__);
      float __temp_425__;
      __temp_425__ = (12 * __tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+6)))]);
      float __temp_426__;
      __temp_426__ = (__temp_424__ + __temp_425__);
      float __temp_427__;
      __temp_427__ = (5 * __tilevar_5__[__iter_23__+(2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+6)))]);
      float __temp_428__;
      __temp_428__ = (__temp_426__ + __temp_427__);
      float __temp_429__;
      __temp_429__ = (4 * __tilevar_5__[__iter_23__+(-2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+6)))]);
      float __temp_430__;
      __temp_430__ = (__temp_428__ + __temp_429__);
      float __temp_431__;
      __temp_431__ = (9 * __tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+6)))]);
      float __temp_432__;
      __temp_432__ = (__temp_430__ + __temp_431__);
      float __temp_433__;
      __temp_433__ = (12 * __tilevar_5__[__iter_23__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+6)))]);
      float __temp_434__;
      __temp_434__ = (__temp_432__ + __temp_433__);
      float __temp_435__;
      __temp_435__ = (9 * __tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+6)))]);
      float __temp_436__;
      __temp_436__ = (__temp_434__ + __temp_435__);
      float __temp_437__;
      __temp_437__ = (4 * __tilevar_5__[__iter_23__+(2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+6)))]);
      float __temp_438__;
      __temp_438__ = (__temp_436__ + __temp_437__);
      float __temp_439__;
      __temp_439__ = (2 * __tilevar_5__[__iter_23__+(-2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+6)))]);
      float __temp_440__;
      __temp_440__ = (__temp_438__ + __temp_439__);
      float __temp_441__;
      __temp_441__ = (4 * __tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+6)))]);
      float __temp_442__;
      __temp_442__ = (__temp_440__ + __temp_441__);
      float __temp_443__;
      __temp_443__ = (5 * __tilevar_5__[__iter_23__+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+6)))]);
      float __temp_444__;
      __temp_444__ = (__temp_442__ + __temp_443__);
      float __temp_445__;
      __temp_445__ = (4 * __tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+6)))]);
      float __temp_446__;
      __temp_446__ = (__temp_444__ + __temp_445__);
      float __temp_447__;
      __temp_447__ = (2 * __tilevar_5__[__iter_23__+(2)+(0-(__iter_0__+6))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(2)+(0-(__iter_1__+6)))]);
      float __temp_448__;
      __temp_448__ = (__temp_446__ + __temp_447__);
      float __temp_449__;
      __temp_449__ = (__temp_448__ / 159);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_449__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __copy_arr_0__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_8__;
  __iter_8__ = FORMA_MAX(__iter_1__,2) + (int)(threadIdx.y) ; 
  for( ; __iter_8__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-3)) ; __iter_8__ += (int)(blockDim.y) ){
    int __iter_9__;
    __iter_9__ = FORMA_MAX(__iter_0__,2) + (int)(threadIdx.x) ; 
    if( __iter_9__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-3)) ){
      if (__iter_8__ < FORMA_MAX((__iter_1__+2),2) || __iter_8__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3)) || __iter_9__ < FORMA_MAX((__iter_0__+2),2) || __iter_9__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-3))) {
        float __temp_100__;
        __temp_100__ = (2 * input[__iter_9__+(-2)+(M-0)*(__iter_8__+(-2))]);
        float __temp_101__;
        __temp_101__ = (4 * input[__iter_9__+(-1)+(M-0)*(__iter_8__+(-2))]);
        float __temp_102__;
        __temp_102__ = (__temp_100__ + __temp_101__);
        float __temp_103__;
        __temp_103__ = (5 * input[__iter_9__+(M-0)*(__iter_8__+(-2))]);
        float __temp_104__;
        __temp_104__ = (__temp_102__ + __temp_103__);
        float __temp_105__;
        __temp_105__ = (4 * input[__iter_9__+(1)+(M-0)*(__iter_8__+(-2))]);
        float __temp_106__;
        __temp_106__ = (__temp_104__ + __temp_105__);
        float __temp_107__;
        __temp_107__ = (2 * input[__iter_9__+(2)+(M-0)*(__iter_8__+(-2))]);
        float __temp_108__;
        __temp_108__ = (__temp_106__ + __temp_107__);
        float __temp_109__;
        __temp_109__ = (4 * input[__iter_9__+(-2)+(M-0)*(__iter_8__+(-1))]);
        float __temp_110__;
        __temp_110__ = (__temp_108__ + __temp_109__);
        float __temp_111__;
        __temp_111__ = (9 * input[__iter_9__+(-1)+(M-0)*(__iter_8__+(-1))]);
        float __temp_112__;
        __temp_112__ = (__temp_110__ + __temp_111__);
        float __temp_113__;
        __temp_113__ = (12 * input[__iter_9__+(M-0)*(__iter_8__+(-1))]);
        float __temp_114__;
        __temp_114__ = (__temp_112__ + __temp_113__);
        float __temp_115__;
        __temp_115__ = (9 * input[__iter_9__+(1)+(M-0)*(__iter_8__+(-1))]);
        float __temp_116__;
        __temp_116__ = (__temp_114__ + __temp_115__);
        float __temp_117__;
        __temp_117__ = (4 * input[__iter_9__+(2)+(M-0)*(__iter_8__+(-1))]);
        float __temp_118__;
        __temp_118__ = (__temp_116__ + __temp_117__);
        float __temp_119__;
        __temp_119__ = (5 * input[__iter_9__+(-2)+(M-0)*(__iter_8__)]);
        float __temp_120__;
        __temp_120__ = (__temp_118__ + __temp_119__);
        float __temp_121__;
        __temp_121__ = (12 * input[__iter_9__+(-1)+(M-0)*(__iter_8__)]);
        float __temp_122__;
        __temp_122__ = (__temp_120__ + __temp_121__);
        float __temp_123__;
        __temp_123__ = (15 * input[__iter_9__+(M-0)*(__iter_8__)]);
        float __temp_124__;
        __temp_124__ = (__temp_122__ + __temp_123__);
        float __temp_125__;
        __temp_125__ = (12 * input[__iter_9__+(1)+(M-0)*(__iter_8__)]);
        float __temp_126__;
        __temp_126__ = (__temp_124__ + __temp_125__);
        float __temp_127__;
        __temp_127__ = (5 * input[__iter_9__+(2)+(M-0)*(__iter_8__)]);
        float __temp_128__;
        __temp_128__ = (__temp_126__ + __temp_127__);
        float __temp_129__;
        __temp_129__ = (4 * input[__iter_9__+(-2)+(M-0)*(__iter_8__+(1))]);
        float __temp_130__;
        __temp_130__ = (__temp_128__ + __temp_129__);
        float __temp_131__;
        __temp_131__ = (9 * input[__iter_9__+(-1)+(M-0)*(__iter_8__+(1))]);
        float __temp_132__;
        __temp_132__ = (__temp_130__ + __temp_131__);
        float __temp_133__;
        __temp_133__ = (12 * input[__iter_9__+(M-0)*(__iter_8__+(1))]);
        float __temp_134__;
        __temp_134__ = (__temp_132__ + __temp_133__);
        float __temp_135__;
        __temp_135__ = (9 * input[__iter_9__+(1)+(M-0)*(__iter_8__+(1))]);
        float __temp_136__;
        __temp_136__ = (__temp_134__ + __temp_135__);
        float __temp_137__;
        __temp_137__ = (4 * input[__iter_9__+(2)+(M-0)*(__iter_8__+(1))]);
        float __temp_138__;
        __temp_138__ = (__temp_136__ + __temp_137__);
        float __temp_139__;
        __temp_139__ = (2 * input[__iter_9__+(-2)+(M-0)*(__iter_8__+(2))]);
        float __temp_140__;
        __temp_140__ = (__temp_138__ + __temp_139__);
        float __temp_141__;
        __temp_141__ = (4 * input[__iter_9__+(-1)+(M-0)*(__iter_8__+(2))]);
        float __temp_142__;
        __temp_142__ = (__temp_140__ + __temp_141__);
        float __temp_143__;
        __temp_143__ = (5 * input[__iter_9__+(M-0)*(__iter_8__+(2))]);
        float __temp_144__;
        __temp_144__ = (__temp_142__ + __temp_143__);
        float __temp_145__;
        __temp_145__ = (4 * input[__iter_9__+(1)+(M-0)*(__iter_8__+(2))]);
        float __temp_146__;
        __temp_146__ = (__temp_144__ + __temp_145__);
        float __temp_147__;
        __temp_147__ = (2 * input[__iter_9__+(2)+(M-0)*(__iter_8__+(2))]);
        float __temp_148__;
        __temp_148__ = (__temp_146__ + __temp_147__);
        float __temp_149__;
        __temp_149__ = (__temp_148__ / 159);
        __copy_arr_0__[__iter_9__+(M-0)*(__iter_8__)] = __temp_149__;
      }
    }
  }
}

__global__ void __kernel___forma_kernel__2__(float * __restrict__ __copy_arr_0__, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __copy_arr_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_14__;
  __iter_14__ = FORMA_MAX(__iter_1__,2) + (int)(threadIdx.y) ; 
  for( ; __iter_14__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-3)) ; __iter_14__ += (int)(blockDim.y) ){
    int __iter_15__;
    __iter_15__ = FORMA_MAX(__iter_0__,2) + (int)(threadIdx.x) ; 
    if( __iter_15__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-3)) ){
      if (__iter_14__ < FORMA_MAX((__iter_1__+4),2) || __iter_14__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-3)) || __iter_15__ < FORMA_MAX((__iter_0__+4),2) || __iter_15__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-3))) {
        float __temp_250__;
        __temp_250__ = (2 * __copy_arr_0__[__iter_15__+(-2)+(M-0)*(__iter_14__+(-2))]);
        float __temp_251__;
        __temp_251__ = (4 * __copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__+(-2))]);
        float __temp_252__;
        __temp_252__ = (__temp_250__ + __temp_251__);
        float __temp_253__;
        __temp_253__ = (5 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(-2))]);
        float __temp_254__;
        __temp_254__ = (__temp_252__ + __temp_253__);
        float __temp_255__;
        __temp_255__ = (4 * __copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__+(-2))]);
        float __temp_256__;
        __temp_256__ = (__temp_254__ + __temp_255__);
        float __temp_257__;
        __temp_257__ = (2 * __copy_arr_0__[__iter_15__+(2)+(M-0)*(__iter_14__+(-2))]);
        float __temp_258__;
        __temp_258__ = (__temp_256__ + __temp_257__);
        float __temp_259__;
        __temp_259__ = (4 * __copy_arr_0__[__iter_15__+(-2)+(M-0)*(__iter_14__+(-1))]);
        float __temp_260__;
        __temp_260__ = (__temp_258__ + __temp_259__);
        float __temp_261__;
        __temp_261__ = (9 * __copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__+(-1))]);
        float __temp_262__;
        __temp_262__ = (__temp_260__ + __temp_261__);
        float __temp_263__;
        __temp_263__ = (12 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(-1))]);
        float __temp_264__;
        __temp_264__ = (__temp_262__ + __temp_263__);
        float __temp_265__;
        __temp_265__ = (9 * __copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__+(-1))]);
        float __temp_266__;
        __temp_266__ = (__temp_264__ + __temp_265__);
        float __temp_267__;
        __temp_267__ = (4 * __copy_arr_0__[__iter_15__+(2)+(M-0)*(__iter_14__+(-1))]);
        float __temp_268__;
        __temp_268__ = (__temp_266__ + __temp_267__);
        float __temp_269__;
        __temp_269__ = (5 * __copy_arr_0__[__iter_15__+(-2)+(M-0)*(__iter_14__)]);
        float __temp_270__;
        __temp_270__ = (__temp_268__ + __temp_269__);
        float __temp_271__;
        __temp_271__ = (12 * __copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__)]);
        float __temp_272__;
        __temp_272__ = (__temp_270__ + __temp_271__);
        float __temp_273__;
        __temp_273__ = (15 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__)]);
        float __temp_274__;
        __temp_274__ = (__temp_272__ + __temp_273__);
        float __temp_275__;
        __temp_275__ = (12 * __copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__)]);
        float __temp_276__;
        __temp_276__ = (__temp_274__ + __temp_275__);
        float __temp_277__;
        __temp_277__ = (5 * __copy_arr_0__[__iter_15__+(2)+(M-0)*(__iter_14__)]);
        float __temp_278__;
        __temp_278__ = (__temp_276__ + __temp_277__);
        float __temp_279__;
        __temp_279__ = (4 * __copy_arr_0__[__iter_15__+(-2)+(M-0)*(__iter_14__+(1))]);
        float __temp_280__;
        __temp_280__ = (__temp_278__ + __temp_279__);
        float __temp_281__;
        __temp_281__ = (9 * __copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__+(1))]);
        float __temp_282__;
        __temp_282__ = (__temp_280__ + __temp_281__);
        float __temp_283__;
        __temp_283__ = (12 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(1))]);
        float __temp_284__;
        __temp_284__ = (__temp_282__ + __temp_283__);
        float __temp_285__;
        __temp_285__ = (9 * __copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__+(1))]);
        float __temp_286__;
        __temp_286__ = (__temp_284__ + __temp_285__);
        float __temp_287__;
        __temp_287__ = (4 * __copy_arr_0__[__iter_15__+(2)+(M-0)*(__iter_14__+(1))]);
        float __temp_288__;
        __temp_288__ = (__temp_286__ + __temp_287__);
        float __temp_289__;
        __temp_289__ = (2 * __copy_arr_0__[__iter_15__+(-2)+(M-0)*(__iter_14__+(2))]);
        float __temp_290__;
        __temp_290__ = (__temp_288__ + __temp_289__);
        float __temp_291__;
        __temp_291__ = (4 * __copy_arr_0__[__iter_15__+(-1)+(M-0)*(__iter_14__+(2))]);
        float __temp_292__;
        __temp_292__ = (__temp_290__ + __temp_291__);
        float __temp_293__;
        __temp_293__ = (5 * __copy_arr_0__[__iter_15__+(M-0)*(__iter_14__+(2))]);
        float __temp_294__;
        __temp_294__ = (__temp_292__ + __temp_293__);
        float __temp_295__;
        __temp_295__ = (4 * __copy_arr_0__[__iter_15__+(1)+(M-0)*(__iter_14__+(2))]);
        float __temp_296__;
        __temp_296__ = (__temp_294__ + __temp_295__);
        float __temp_297__;
        __temp_297__ = (2 * __copy_arr_0__[__iter_15__+(2)+(M-0)*(__iter_14__+(2))]);
        float __temp_298__;
        __temp_298__ = (__temp_296__ + __temp_297__);
        float __temp_299__;
        __temp_299__ = (__temp_298__ / 159);
        __copy_arr_1__[__iter_15__+(M-0)*(__iter_14__)] = __temp_299__;
      }
    }
  }
}

__global__ void __kernel___forma_kernel__3__(float * __restrict__ __copy_arr_1__, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __copy_arr_2__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_20__;
  __iter_20__ = FORMA_MAX(__iter_1__,2) + (int)(threadIdx.y) ; 
  for( ; __iter_20__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-3)) ; __iter_20__ += (int)(blockDim.y) ){
    int __iter_21__;
    __iter_21__ = FORMA_MAX(__iter_0__,2) + (int)(threadIdx.x) ; 
    if( __iter_21__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-3)) ){
      if (__iter_20__ < FORMA_MAX((__iter_1__+6),2) || __iter_20__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-7),(N-3)) || __iter_21__ < FORMA_MAX((__iter_0__+6),2) || __iter_21__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-7),(M-3))) {
        float __temp_350__;
        __temp_350__ = (2 * __copy_arr_1__[__iter_21__+(-2)+(M-0)*(__iter_20__+(-2))]);
        float __temp_351__;
        __temp_351__ = (4 * __copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__+(-2))]);
        float __temp_352__;
        __temp_352__ = (__temp_350__ + __temp_351__);
        float __temp_353__;
        __temp_353__ = (5 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(-2))]);
        float __temp_354__;
        __temp_354__ = (__temp_352__ + __temp_353__);
        float __temp_355__;
        __temp_355__ = (4 * __copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__+(-2))]);
        float __temp_356__;
        __temp_356__ = (__temp_354__ + __temp_355__);
        float __temp_357__;
        __temp_357__ = (2 * __copy_arr_1__[__iter_21__+(2)+(M-0)*(__iter_20__+(-2))]);
        float __temp_358__;
        __temp_358__ = (__temp_356__ + __temp_357__);
        float __temp_359__;
        __temp_359__ = (4 * __copy_arr_1__[__iter_21__+(-2)+(M-0)*(__iter_20__+(-1))]);
        float __temp_360__;
        __temp_360__ = (__temp_358__ + __temp_359__);
        float __temp_361__;
        __temp_361__ = (9 * __copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__+(-1))]);
        float __temp_362__;
        __temp_362__ = (__temp_360__ + __temp_361__);
        float __temp_363__;
        __temp_363__ = (12 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(-1))]);
        float __temp_364__;
        __temp_364__ = (__temp_362__ + __temp_363__);
        float __temp_365__;
        __temp_365__ = (9 * __copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__+(-1))]);
        float __temp_366__;
        __temp_366__ = (__temp_364__ + __temp_365__);
        float __temp_367__;
        __temp_367__ = (4 * __copy_arr_1__[__iter_21__+(2)+(M-0)*(__iter_20__+(-1))]);
        float __temp_368__;
        __temp_368__ = (__temp_366__ + __temp_367__);
        float __temp_369__;
        __temp_369__ = (5 * __copy_arr_1__[__iter_21__+(-2)+(M-0)*(__iter_20__)]);
        float __temp_370__;
        __temp_370__ = (__temp_368__ + __temp_369__);
        float __temp_371__;
        __temp_371__ = (12 * __copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__)]);
        float __temp_372__;
        __temp_372__ = (__temp_370__ + __temp_371__);
        float __temp_373__;
        __temp_373__ = (15 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__)]);
        float __temp_374__;
        __temp_374__ = (__temp_372__ + __temp_373__);
        float __temp_375__;
        __temp_375__ = (12 * __copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__)]);
        float __temp_376__;
        __temp_376__ = (__temp_374__ + __temp_375__);
        float __temp_377__;
        __temp_377__ = (5 * __copy_arr_1__[__iter_21__+(2)+(M-0)*(__iter_20__)]);
        float __temp_378__;
        __temp_378__ = (__temp_376__ + __temp_377__);
        float __temp_379__;
        __temp_379__ = (4 * __copy_arr_1__[__iter_21__+(-2)+(M-0)*(__iter_20__+(1))]);
        float __temp_380__;
        __temp_380__ = (__temp_378__ + __temp_379__);
        float __temp_381__;
        __temp_381__ = (9 * __copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__+(1))]);
        float __temp_382__;
        __temp_382__ = (__temp_380__ + __temp_381__);
        float __temp_383__;
        __temp_383__ = (12 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(1))]);
        float __temp_384__;
        __temp_384__ = (__temp_382__ + __temp_383__);
        float __temp_385__;
        __temp_385__ = (9 * __copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__+(1))]);
        float __temp_386__;
        __temp_386__ = (__temp_384__ + __temp_385__);
        float __temp_387__;
        __temp_387__ = (4 * __copy_arr_1__[__iter_21__+(2)+(M-0)*(__iter_20__+(1))]);
        float __temp_388__;
        __temp_388__ = (__temp_386__ + __temp_387__);
        float __temp_389__;
        __temp_389__ = (2 * __copy_arr_1__[__iter_21__+(-2)+(M-0)*(__iter_20__+(2))]);
        float __temp_390__;
        __temp_390__ = (__temp_388__ + __temp_389__);
        float __temp_391__;
        __temp_391__ = (4 * __copy_arr_1__[__iter_21__+(-1)+(M-0)*(__iter_20__+(2))]);
        float __temp_392__;
        __temp_392__ = (__temp_390__ + __temp_391__);
        float __temp_393__;
        __temp_393__ = (5 * __copy_arr_1__[__iter_21__+(M-0)*(__iter_20__+(2))]);
        float __temp_394__;
        __temp_394__ = (__temp_392__ + __temp_393__);
        float __temp_395__;
        __temp_395__ = (4 * __copy_arr_1__[__iter_21__+(1)+(M-0)*(__iter_20__+(2))]);
        float __temp_396__;
        __temp_396__ = (__temp_394__ + __temp_395__);
        float __temp_397__;
        __temp_397__ = (2 * __copy_arr_1__[__iter_21__+(2)+(M-0)*(__iter_20__+(2))]);
        float __temp_398__;
        __temp_398__ = (__temp_396__ + __temp_397__);
        float __temp_399__;
        __temp_399__ = (__temp_398__ / 159);
        __copy_arr_2__[__iter_21__+(M-0)*(__iter_20__)] = __temp_399__;
      }
    }
  }
}

__global__ void __kernel___forma_kernel__4__(float * __restrict__ __copy_arr_2__, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_24__;
  __iter_24__ = FORMA_MAX(__iter_1__,2) + (int)(threadIdx.y) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-3)) ; __iter_24__ += (int)(blockDim.y) ){
    int __iter_25__;
    __iter_25__ = FORMA_MAX(__iter_0__,2) + (int)(threadIdx.x) ; 
    if( __iter_25__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-3)) ){
      if (__iter_24__ < FORMA_MAX((__iter_1__+8),2) || __iter_24__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-9),(N-3)) || __iter_25__ < FORMA_MAX((__iter_0__+8),2) || __iter_25__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-9),(M-3))) {
        float __temp_450__;
        __temp_450__ = (2 * __copy_arr_2__[__iter_25__+(-2)+(M-0)*(__iter_24__+(-2))]);
        float __temp_451__;
        __temp_451__ = (4 * __copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__+(-2))]);
        float __temp_452__;
        __temp_452__ = (__temp_450__ + __temp_451__);
        float __temp_453__;
        __temp_453__ = (5 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(-2))]);
        float __temp_454__;
        __temp_454__ = (__temp_452__ + __temp_453__);
        float __temp_455__;
        __temp_455__ = (4 * __copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__+(-2))]);
        float __temp_456__;
        __temp_456__ = (__temp_454__ + __temp_455__);
        float __temp_457__;
        __temp_457__ = (2 * __copy_arr_2__[__iter_25__+(2)+(M-0)*(__iter_24__+(-2))]);
        float __temp_458__;
        __temp_458__ = (__temp_456__ + __temp_457__);
        float __temp_459__;
        __temp_459__ = (4 * __copy_arr_2__[__iter_25__+(-2)+(M-0)*(__iter_24__+(-1))]);
        float __temp_460__;
        __temp_460__ = (__temp_458__ + __temp_459__);
        float __temp_461__;
        __temp_461__ = (9 * __copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__+(-1))]);
        float __temp_462__;
        __temp_462__ = (__temp_460__ + __temp_461__);
        float __temp_463__;
        __temp_463__ = (12 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(-1))]);
        float __temp_464__;
        __temp_464__ = (__temp_462__ + __temp_463__);
        float __temp_465__;
        __temp_465__ = (9 * __copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__+(-1))]);
        float __temp_466__;
        __temp_466__ = (__temp_464__ + __temp_465__);
        float __temp_467__;
        __temp_467__ = (4 * __copy_arr_2__[__iter_25__+(2)+(M-0)*(__iter_24__+(-1))]);
        float __temp_468__;
        __temp_468__ = (__temp_466__ + __temp_467__);
        float __temp_469__;
        __temp_469__ = (5 * __copy_arr_2__[__iter_25__+(-2)+(M-0)*(__iter_24__)]);
        float __temp_470__;
        __temp_470__ = (__temp_468__ + __temp_469__);
        float __temp_471__;
        __temp_471__ = (12 * __copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__)]);
        float __temp_472__;
        __temp_472__ = (__temp_470__ + __temp_471__);
        float __temp_473__;
        __temp_473__ = (15 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__)]);
        float __temp_474__;
        __temp_474__ = (__temp_472__ + __temp_473__);
        float __temp_475__;
        __temp_475__ = (12 * __copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__)]);
        float __temp_476__;
        __temp_476__ = (__temp_474__ + __temp_475__);
        float __temp_477__;
        __temp_477__ = (5 * __copy_arr_2__[__iter_25__+(2)+(M-0)*(__iter_24__)]);
        float __temp_478__;
        __temp_478__ = (__temp_476__ + __temp_477__);
        float __temp_479__;
        __temp_479__ = (4 * __copy_arr_2__[__iter_25__+(-2)+(M-0)*(__iter_24__+(1))]);
        float __temp_480__;
        __temp_480__ = (__temp_478__ + __temp_479__);
        float __temp_481__;
        __temp_481__ = (9 * __copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__+(1))]);
        float __temp_482__;
        __temp_482__ = (__temp_480__ + __temp_481__);
        float __temp_483__;
        __temp_483__ = (12 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(1))]);
        float __temp_484__;
        __temp_484__ = (__temp_482__ + __temp_483__);
        float __temp_485__;
        __temp_485__ = (9 * __copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__+(1))]);
        float __temp_486__;
        __temp_486__ = (__temp_484__ + __temp_485__);
        float __temp_487__;
        __temp_487__ = (4 * __copy_arr_2__[__iter_25__+(2)+(M-0)*(__iter_24__+(1))]);
        float __temp_488__;
        __temp_488__ = (__temp_486__ + __temp_487__);
        float __temp_489__;
        __temp_489__ = (2 * __copy_arr_2__[__iter_25__+(-2)+(M-0)*(__iter_24__+(2))]);
        float __temp_490__;
        __temp_490__ = (__temp_488__ + __temp_489__);
        float __temp_491__;
        __temp_491__ = (4 * __copy_arr_2__[__iter_25__+(-1)+(M-0)*(__iter_24__+(2))]);
        float __temp_492__;
        __temp_492__ = (__temp_490__ + __temp_491__);
        float __temp_493__;
        __temp_493__ = (5 * __copy_arr_2__[__iter_25__+(M-0)*(__iter_24__+(2))]);
        float __temp_494__;
        __temp_494__ = (__temp_492__ + __temp_493__);
        float __temp_495__;
        __temp_495__ = (4 * __copy_arr_2__[__iter_25__+(1)+(M-0)*(__iter_24__+(2))]);
        float __temp_496__;
        __temp_496__ = (__temp_494__ + __temp_495__);
        float __temp_497__;
        __temp_497__ = (2 * __copy_arr_2__[__iter_25__+(2)+(M-0)*(__iter_24__+(2))]);
        float __temp_498__;
        __temp_498__ = (__temp_496__ + __temp_497__);
        float __temp_499__;
        __temp_499__ = (__temp_498__ / 159);
        __var_1__[__iter_25__+(M-0)*(__iter_24__)] = __temp_499__;
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void gaussian(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),13);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),13);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 13)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,13))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  dim3 __blockConfig___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__1__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __copy_arr_0__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  dim3 __blockConfig___kernel___forma_kernel__2__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2);
  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__2__>>> (__copy_arr_0__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __copy_arr_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");

  dim3 __blockConfig___kernel___forma_kernel__3__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2);
  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__3__>>> (__copy_arr_1__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __copy_arr_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  dim3 __blockConfig___kernel___forma_kernel__4__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2);
  __kernel___forma_kernel__4__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__4__>>> (__copy_arr_2__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__4__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

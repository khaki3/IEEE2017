#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y)*(FORMA_BLOCKDIM_X)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y)*(FORMA_BLOCKDIM_X)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X)*(__iter_2__+(0-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
    }
  }
  __syncthreads();
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void gaussian(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),13);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),13);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  __block_1___kernel___forma_kernel__0__ = 16;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 13)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,13))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  __blockConfig___kernel___forma_kernel__0__.y = 32;
  __blockConfig___kernel___forma_kernel__0__.y = 32;
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

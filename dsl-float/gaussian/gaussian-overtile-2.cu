#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __var_2__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 2;
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 2;
  int __iter_2__;
  __iter_2__ = (FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)+(-2)) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= (FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-3))+2),(N-3))+2) ){
    int __iter_3__;
    __iter_3__ = (FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)+(-2)) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= (FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-3))+2),(M-3))+2) ){
      int __temp_0__;
      __temp_0__ = __iter_2__;
      int __temp_1__;
      __temp_1__ = __iter_3__;
      __tilevar_1__[__iter_3__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_2__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))] = input[__temp_1__+(M-0)*(__temp_0__)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-3))+2),(N-3)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-3))+2),(M-3)) ){
      float __temp_2__;
      __temp_2__ = (2 * __tilevar_1__[__iter_5__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_3__;
      __temp_3__ = (4 * __tilevar_1__[__iter_5__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_4__;
      __temp_4__ = (__temp_2__ + __temp_3__);
      float __temp_5__;
      __temp_5__ = (5 * __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_6__;
      __temp_6__ = (__temp_4__ + __temp_5__);
      float __temp_7__;
      __temp_7__ = (4 * __tilevar_1__[__iter_5__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_8__;
      __temp_8__ = (__temp_6__ + __temp_7__);
      float __temp_9__;
      __temp_9__ = (2 * __tilevar_1__[__iter_5__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_10__;
      __temp_10__ = (__temp_8__ + __temp_9__);
      float __temp_11__;
      __temp_11__ = (4 * __tilevar_1__[__iter_5__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_12__;
      __temp_12__ = (__temp_10__ + __temp_11__);
      float __temp_13__;
      __temp_13__ = (9 * __tilevar_1__[__iter_5__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_14__;
      __temp_14__ = (__temp_12__ + __temp_13__);
      float __temp_15__;
      __temp_15__ = (12 * __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_16__;
      __temp_16__ = (__temp_14__ + __temp_15__);
      float __temp_17__;
      __temp_17__ = (9 * __tilevar_1__[__iter_5__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_18__;
      __temp_18__ = (__temp_16__ + __temp_17__);
      float __temp_19__;
      __temp_19__ = (4 * __tilevar_1__[__iter_5__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_20__;
      __temp_20__ = (__temp_18__ + __temp_19__);
      float __temp_21__;
      __temp_21__ = (5 * __tilevar_1__[__iter_5__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_22__;
      __temp_22__ = (__temp_20__ + __temp_21__);
      float __temp_23__;
      __temp_23__ = (12 * __tilevar_1__[__iter_5__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_24__;
      __temp_24__ = (__temp_22__ + __temp_23__);
      float __temp_25__;
      __temp_25__ = (15 * __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_26__;
      __temp_26__ = (__temp_24__ + __temp_25__);
      float __temp_27__;
      __temp_27__ = (12 * __tilevar_1__[__iter_5__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_28__;
      __temp_28__ = (__temp_26__ + __temp_27__);
      float __temp_29__;
      __temp_29__ = (5 * __tilevar_1__[__iter_5__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_30__;
      __temp_30__ = (__temp_28__ + __temp_29__);
      float __temp_31__;
      __temp_31__ = (4 * __tilevar_1__[__iter_5__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_32__;
      __temp_32__ = (__temp_30__ + __temp_31__);
      float __temp_33__;
      __temp_33__ = (9 * __tilevar_1__[__iter_5__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_34__;
      __temp_34__ = (__temp_32__ + __temp_33__);
      float __temp_35__;
      __temp_35__ = (12 * __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_36__;
      __temp_36__ = (__temp_34__ + __temp_35__);
      float __temp_37__;
      __temp_37__ = (9 * __tilevar_1__[__iter_5__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_38__;
      __temp_38__ = (__temp_36__ + __temp_37__);
      float __temp_39__;
      __temp_39__ = (4 * __tilevar_1__[__iter_5__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_40__;
      __temp_40__ = (__temp_38__ + __temp_39__);
      float __temp_41__;
      __temp_41__ = (2 * __tilevar_1__[__iter_5__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_42__;
      __temp_42__ = (__temp_40__ + __temp_41__);
      float __temp_43__;
      __temp_43__ = (4 * __tilevar_1__[__iter_5__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_44__;
      __temp_44__ = (__temp_42__ + __temp_43__);
      float __temp_45__;
      __temp_45__ = (5 * __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_46__;
      __temp_46__ = (__temp_44__ + __temp_45__);
      float __temp_47__;
      __temp_47__ = (4 * __tilevar_1__[__iter_5__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_48__;
      __temp_48__ = (__temp_46__ + __temp_47__);
      float __temp_49__;
      __temp_49__ = (2 * __tilevar_1__[__iter_5__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2)))]);
      float __temp_50__;
      __temp_50__ = (__temp_48__ + __temp_49__);
      float __temp_51__;
      __temp_51__ = (__temp_50__ / 159);
      __tilevar_0__[__iter_5__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX(__iter_1__,2)-2)))] = __temp_51__;
    }
  }
  __syncthreads();
  int __iter_6__;
  __iter_6__ = FORMA_MAX(__iter_1__,2) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-3)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX(__iter_0__,2) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-3)) ){
      float __temp_52__;
      __temp_52__ = (2 * __tilevar_0__[__iter_7__+(-2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(-2)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_53__;
      __temp_53__ = (4 * __tilevar_0__[__iter_7__+(-1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(-2)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_54__;
      __temp_54__ = (__temp_52__ + __temp_53__);
      float __temp_55__;
      __temp_55__ = (5 * __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(-2)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_56__;
      __temp_56__ = (__temp_54__ + __temp_55__);
      float __temp_57__;
      __temp_57__ = (4 * __tilevar_0__[__iter_7__+(1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(-2)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_58__;
      __temp_58__ = (__temp_56__ + __temp_57__);
      float __temp_59__;
      __temp_59__ = (2 * __tilevar_0__[__iter_7__+(2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(-2)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_60__;
      __temp_60__ = (__temp_58__ + __temp_59__);
      float __temp_61__;
      __temp_61__ = (4 * __tilevar_0__[__iter_7__+(-2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(-1)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_62__;
      __temp_62__ = (__temp_60__ + __temp_61__);
      float __temp_63__;
      __temp_63__ = (9 * __tilevar_0__[__iter_7__+(-1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(-1)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_64__;
      __temp_64__ = (__temp_62__ + __temp_63__);
      float __temp_65__;
      __temp_65__ = (12 * __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(-1)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_66__;
      __temp_66__ = (__temp_64__ + __temp_65__);
      float __temp_67__;
      __temp_67__ = (9 * __tilevar_0__[__iter_7__+(1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(-1)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_68__;
      __temp_68__ = (__temp_66__ + __temp_67__);
      float __temp_69__;
      __temp_69__ = (4 * __tilevar_0__[__iter_7__+(2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(-1)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_70__;
      __temp_70__ = (__temp_68__ + __temp_69__);
      float __temp_71__;
      __temp_71__ = (5 * __tilevar_0__[__iter_7__+(-2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_72__;
      __temp_72__ = (__temp_70__ + __temp_71__);
      float __temp_73__;
      __temp_73__ = (12 * __tilevar_0__[__iter_7__+(-1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_74__;
      __temp_74__ = (__temp_72__ + __temp_73__);
      float __temp_75__;
      __temp_75__ = (15 * __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_76__;
      __temp_76__ = (__temp_74__ + __temp_75__);
      float __temp_77__;
      __temp_77__ = (12 * __tilevar_0__[__iter_7__+(1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_78__;
      __temp_78__ = (__temp_76__ + __temp_77__);
      float __temp_79__;
      __temp_79__ = (5 * __tilevar_0__[__iter_7__+(2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_80__;
      __temp_80__ = (__temp_78__ + __temp_79__);
      float __temp_81__;
      __temp_81__ = (4 * __tilevar_0__[__iter_7__+(-2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(1)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_82__;
      __temp_82__ = (__temp_80__ + __temp_81__);
      float __temp_83__;
      __temp_83__ = (9 * __tilevar_0__[__iter_7__+(-1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(1)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_84__;
      __temp_84__ = (__temp_82__ + __temp_83__);
      float __temp_85__;
      __temp_85__ = (12 * __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(1)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_86__;
      __temp_86__ = (__temp_84__ + __temp_85__);
      float __temp_87__;
      __temp_87__ = (9 * __tilevar_0__[__iter_7__+(1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(1)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_88__;
      __temp_88__ = (__temp_86__ + __temp_87__);
      float __temp_89__;
      __temp_89__ = (4 * __tilevar_0__[__iter_7__+(2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(1)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_90__;
      __temp_90__ = (__temp_88__ + __temp_89__);
      float __temp_91__;
      __temp_91__ = (2 * __tilevar_0__[__iter_7__+(-2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(2)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_92__;
      __temp_92__ = (__temp_90__ + __temp_91__);
      float __temp_93__;
      __temp_93__ = (4 * __tilevar_0__[__iter_7__+(-1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(2)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_94__;
      __temp_94__ = (__temp_92__ + __temp_93__);
      float __temp_95__;
      __temp_95__ = (5 * __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(2)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_96__;
      __temp_96__ = (__temp_94__ + __temp_95__);
      float __temp_97__;
      __temp_97__ = (4 * __tilevar_0__[__iter_7__+(1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(2)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_98__;
      __temp_98__ = (__temp_96__ + __temp_97__);
      float __temp_99__;
      __temp_99__ = (2 * __tilevar_0__[__iter_7__+(2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(2)+(0-(FORMA_MAX(__iter_1__,2)-2)))]);
      float __temp_100__;
      __temp_100__ = (__temp_98__ + __temp_99__);
      float __temp_101__;
      __temp_101__ = (__temp_100__ / 159);
      __var_2__[__iter_7__+(M-0)*(__iter_6__)] = __temp_101__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ __var_2__, int N, int M, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  int __iter_8__;
  __iter_8__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 2;
  int __iter_9__;
  __iter_9__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 2;
  int __iter_10__;
  __iter_10__ = (FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)+(-2)) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= (FORMA_MIN((FORMA_MIN(((__iter_9__+FORMA_BLOCKDIM_Y)-1),(N-3))+2),(N-3))+2) ){
    int __iter_11__;
    __iter_11__ = (FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)+(-2)) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= (FORMA_MIN((FORMA_MIN(((__iter_8__+FORMA_BLOCKDIM_X)-1),(M-3))+2),(M-3))+2) ){
      int __temp_102__;
      __temp_102__ = __iter_10__;
      int __temp_103__;
      __temp_103__ = __iter_11__;
      __tilevar_3__[__iter_11__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_10__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))] = __var_2__[__temp_103__+(M-0)*(__temp_102__)];
    }
  }
  __syncthreads();
  int __iter_12__;
  __iter_12__ = FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN((FORMA_MIN(((__iter_9__+FORMA_BLOCKDIM_Y)-1),(N-3))+2),(N-3)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN((FORMA_MIN(((__iter_8__+FORMA_BLOCKDIM_X)-1),(M-3))+2),(M-3)) ){
      float __temp_104__;
      __temp_104__ = (2 * __tilevar_3__[__iter_13__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_105__;
      __temp_105__ = (4 * __tilevar_3__[__iter_13__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_106__;
      __temp_106__ = (__temp_104__ + __temp_105__);
      float __temp_107__;
      __temp_107__ = (5 * __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_108__;
      __temp_108__ = (__temp_106__ + __temp_107__);
      float __temp_109__;
      __temp_109__ = (4 * __tilevar_3__[__iter_13__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_110__;
      __temp_110__ = (__temp_108__ + __temp_109__);
      float __temp_111__;
      __temp_111__ = (2 * __tilevar_3__[__iter_13__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_112__;
      __temp_112__ = (__temp_110__ + __temp_111__);
      float __temp_113__;
      __temp_113__ = (4 * __tilevar_3__[__iter_13__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_114__;
      __temp_114__ = (__temp_112__ + __temp_113__);
      float __temp_115__;
      __temp_115__ = (9 * __tilevar_3__[__iter_13__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_116__;
      __temp_116__ = (__temp_114__ + __temp_115__);
      float __temp_117__;
      __temp_117__ = (12 * __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_118__;
      __temp_118__ = (__temp_116__ + __temp_117__);
      float __temp_119__;
      __temp_119__ = (9 * __tilevar_3__[__iter_13__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_120__;
      __temp_120__ = (__temp_118__ + __temp_119__);
      float __temp_121__;
      __temp_121__ = (4 * __tilevar_3__[__iter_13__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_122__;
      __temp_122__ = (__temp_120__ + __temp_121__);
      float __temp_123__;
      __temp_123__ = (5 * __tilevar_3__[__iter_13__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_124__;
      __temp_124__ = (__temp_122__ + __temp_123__);
      float __temp_125__;
      __temp_125__ = (12 * __tilevar_3__[__iter_13__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_126__;
      __temp_126__ = (__temp_124__ + __temp_125__);
      float __temp_127__;
      __temp_127__ = (15 * __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_128__;
      __temp_128__ = (__temp_126__ + __temp_127__);
      float __temp_129__;
      __temp_129__ = (12 * __tilevar_3__[__iter_13__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_130__;
      __temp_130__ = (__temp_128__ + __temp_129__);
      float __temp_131__;
      __temp_131__ = (5 * __tilevar_3__[__iter_13__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_132__;
      __temp_132__ = (__temp_130__ + __temp_131__);
      float __temp_133__;
      __temp_133__ = (4 * __tilevar_3__[__iter_13__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_134__;
      __temp_134__ = (__temp_132__ + __temp_133__);
      float __temp_135__;
      __temp_135__ = (9 * __tilevar_3__[__iter_13__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_136__;
      __temp_136__ = (__temp_134__ + __temp_135__);
      float __temp_137__;
      __temp_137__ = (12 * __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_138__;
      __temp_138__ = (__temp_136__ + __temp_137__);
      float __temp_139__;
      __temp_139__ = (9 * __tilevar_3__[__iter_13__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_140__;
      __temp_140__ = (__temp_138__ + __temp_139__);
      float __temp_141__;
      __temp_141__ = (4 * __tilevar_3__[__iter_13__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_142__;
      __temp_142__ = (__temp_140__ + __temp_141__);
      float __temp_143__;
      __temp_143__ = (2 * __tilevar_3__[__iter_13__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_144__;
      __temp_144__ = (__temp_142__ + __temp_143__);
      float __temp_145__;
      __temp_145__ = (4 * __tilevar_3__[__iter_13__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_146__;
      __temp_146__ = (__temp_144__ + __temp_145__);
      float __temp_147__;
      __temp_147__ = (5 * __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_148__;
      __temp_148__ = (__temp_146__ + __temp_147__);
      float __temp_149__;
      __temp_149__ = (4 * __tilevar_3__[__iter_13__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_150__;
      __temp_150__ = (__temp_148__ + __temp_149__);
      float __temp_151__;
      __temp_151__ = (2 * __tilevar_3__[__iter_13__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_12__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,2)+(-2)),2)-2)))]);
      float __temp_152__;
      __temp_152__ = (__temp_150__ + __temp_151__);
      float __temp_153__;
      __temp_153__ = (__temp_152__ / 159);
      __tilevar_2__[__iter_13__+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX(__iter_9__,2)-2)))] = __temp_153__;
    }
  }
  __syncthreads();
  int __iter_14__;
  __iter_14__ = FORMA_MAX(__iter_9__,2) + (int)(threadIdx.y) ; 
  if( __iter_14__ <= FORMA_MIN(((__iter_9__+FORMA_BLOCKDIM_Y)-1),(N-3)) ){
    int __iter_15__;
    __iter_15__ = FORMA_MAX(__iter_8__,2) + (int)(threadIdx.x) ; 
    if( __iter_15__ <= FORMA_MIN(((__iter_8__+FORMA_BLOCKDIM_X)-1),(M-3)) ){
      float __temp_154__;
      __temp_154__ = (2 * __tilevar_2__[__iter_15__+(-2)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(-2)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_155__;
      __temp_155__ = (4 * __tilevar_2__[__iter_15__+(-1)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(-2)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_156__;
      __temp_156__ = (__temp_154__ + __temp_155__);
      float __temp_157__;
      __temp_157__ = (5 * __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(-2)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_158__;
      __temp_158__ = (__temp_156__ + __temp_157__);
      float __temp_159__;
      __temp_159__ = (4 * __tilevar_2__[__iter_15__+(1)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(-2)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_160__;
      __temp_160__ = (__temp_158__ + __temp_159__);
      float __temp_161__;
      __temp_161__ = (2 * __tilevar_2__[__iter_15__+(2)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(-2)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_162__;
      __temp_162__ = (__temp_160__ + __temp_161__);
      float __temp_163__;
      __temp_163__ = (4 * __tilevar_2__[__iter_15__+(-2)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(-1)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_164__;
      __temp_164__ = (__temp_162__ + __temp_163__);
      float __temp_165__;
      __temp_165__ = (9 * __tilevar_2__[__iter_15__+(-1)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(-1)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_166__;
      __temp_166__ = (__temp_164__ + __temp_165__);
      float __temp_167__;
      __temp_167__ = (12 * __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(-1)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_168__;
      __temp_168__ = (__temp_166__ + __temp_167__);
      float __temp_169__;
      __temp_169__ = (9 * __tilevar_2__[__iter_15__+(1)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(-1)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_170__;
      __temp_170__ = (__temp_168__ + __temp_169__);
      float __temp_171__;
      __temp_171__ = (4 * __tilevar_2__[__iter_15__+(2)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(-1)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_172__;
      __temp_172__ = (__temp_170__ + __temp_171__);
      float __temp_173__;
      __temp_173__ = (5 * __tilevar_2__[__iter_15__+(-2)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_174__;
      __temp_174__ = (__temp_172__ + __temp_173__);
      float __temp_175__;
      __temp_175__ = (12 * __tilevar_2__[__iter_15__+(-1)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_176__;
      __temp_176__ = (__temp_174__ + __temp_175__);
      float __temp_177__;
      __temp_177__ = (15 * __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_178__;
      __temp_178__ = (__temp_176__ + __temp_177__);
      float __temp_179__;
      __temp_179__ = (12 * __tilevar_2__[__iter_15__+(1)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_180__;
      __temp_180__ = (__temp_178__ + __temp_179__);
      float __temp_181__;
      __temp_181__ = (5 * __tilevar_2__[__iter_15__+(2)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_182__;
      __temp_182__ = (__temp_180__ + __temp_181__);
      float __temp_183__;
      __temp_183__ = (4 * __tilevar_2__[__iter_15__+(-2)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(1)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_184__;
      __temp_184__ = (__temp_182__ + __temp_183__);
      float __temp_185__;
      __temp_185__ = (9 * __tilevar_2__[__iter_15__+(-1)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(1)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_186__;
      __temp_186__ = (__temp_184__ + __temp_185__);
      float __temp_187__;
      __temp_187__ = (12 * __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(1)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_188__;
      __temp_188__ = (__temp_186__ + __temp_187__);
      float __temp_189__;
      __temp_189__ = (9 * __tilevar_2__[__iter_15__+(1)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(1)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_190__;
      __temp_190__ = (__temp_188__ + __temp_189__);
      float __temp_191__;
      __temp_191__ = (4 * __tilevar_2__[__iter_15__+(2)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(1)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_192__;
      __temp_192__ = (__temp_190__ + __temp_191__);
      float __temp_193__;
      __temp_193__ = (2 * __tilevar_2__[__iter_15__+(-2)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(2)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_194__;
      __temp_194__ = (__temp_192__ + __temp_193__);
      float __temp_195__;
      __temp_195__ = (4 * __tilevar_2__[__iter_15__+(-1)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(2)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_196__;
      __temp_196__ = (__temp_194__ + __temp_195__);
      float __temp_197__;
      __temp_197__ = (5 * __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(2)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_198__;
      __temp_198__ = (__temp_196__ + __temp_197__);
      float __temp_199__;
      __temp_199__ = (4 * __tilevar_2__[__iter_15__+(1)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(2)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_200__;
      __temp_200__ = (__temp_198__ + __temp_199__);
      float __temp_201__;
      __temp_201__ = (2 * __tilevar_2__[__iter_15__+(2)+(0-(FORMA_MAX(__iter_8__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_14__+(2)+(0-(FORMA_MAX(__iter_9__,2)-2)))]);
      float __temp_202__;
      __temp_202__ = (__temp_200__ + __temp_201__);
      float __temp_203__;
      __temp_203__ = (__temp_202__ / 159);
      __var_1__[__iter_15__+(M-0)*(__iter_14__)] = __temp_203__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__1__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void gaussian(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-3) - 2 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-3) - 2 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,9))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  __block_0___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.x-(4-(-4));
  __block_1___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.y-(4-(-4));
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  int __size_0___kernel___forma_kernel__1__ = ((M-3) - 2 ) + 1;
  int __size_1___kernel___forma_kernel__1__ = ((N-3) - 2 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__1__;
  int _max_occupancy_gridsize___kernel___forma_kernel__1__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__1__,&__max_occupancy_blocksize___kernel___forma_kernel__1__,(const void*)__kernel___forma_kernel__1__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__1___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__1__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__1___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__1___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__1__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__1___0,FORMA_MAX((__size_0___kernel___forma_kernel__1__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),9);
  __max_occupancy_blocksize___kernel___forma_kernel__1__ /= __block_0___kernel___forma_kernel__1__;
  int __max_occupancy_blocksize___kernel___forma_kernel__1___1 = __max_occupancy_blocksize___kernel___forma_kernel__1__;
  int __block_1___kernel___forma_kernel__1__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__1___1,__size_1___kernel___forma_kernel__1__),FORMA_MAX_BLOCKDIM_1),9);
  __max_occupancy_blocksize___kernel___forma_kernel__1__ /= __block_1___kernel___forma_kernel__1__;
  dim3 __blockConfig___kernel___forma_kernel__1__(__block_0___kernel___forma_kernel__1__,__block_1___kernel___forma_kernel__1__);
  int __SMemSize___kernel___forma_kernel__1__ = 0;
  __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
  while( __SMemSize___kernel___forma_kernel__1__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__1__.y/2 > 9)
      __blockConfig___kernel___forma_kernel__1__.y /= 2;
    __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
    if( __SMemSize___kernel___forma_kernel__1__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__1__.x/2 > FORMA_MIN(32,9))
      __blockConfig___kernel___forma_kernel__1__.x /= 2;
    __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
  }
  __block_0___kernel___forma_kernel__1__ = __blockConfig___kernel___forma_kernel__1__.x-(4-(-4));
  __block_1___kernel___forma_kernel__1__ = __blockConfig___kernel___forma_kernel__1__.y-(4-(-4));
  int __grid_0___kernel___forma_kernel__1__ = FORMA_CEIL(__size_0___kernel___forma_kernel__1__,__block_0___kernel___forma_kernel__1__);
  int __grid_1___kernel___forma_kernel__1__ = FORMA_CEIL(__size_1___kernel___forma_kernel__1__,__block_1___kernel___forma_kernel__1__);
  dim3 __gridConfig___kernel___forma_kernel__1__(__grid_0___kernel___forma_kernel__1__,__grid_1___kernel___forma_kernel__1__);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__1__, __blockConfig___kernel___forma_kernel__1__, __SMemSize___kernel___forma_kernel__1__>>> (__var_2__, N, M, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

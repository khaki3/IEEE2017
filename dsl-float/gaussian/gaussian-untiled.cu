#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __var_4__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 2;
  if(__iter_0__ <= (M-3)){
    int __iter_1__;
    __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 2;
    if(__iter_1__ <= (N-3)){
      float __temp_0__;
      __temp_0__ = (2 * input[__iter_0__+(-2)+(M-0)*(__iter_1__+(-2))]);
      float __temp_1__;
      __temp_1__ = (4 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(-2))]);
      float __temp_2__;
      __temp_2__ = (__temp_0__ + __temp_1__);
      float __temp_3__;
      __temp_3__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(-2))]);
      float __temp_4__;
      __temp_4__ = (__temp_2__ + __temp_3__);
      float __temp_5__;
      __temp_5__ = (4 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(-2))]);
      float __temp_6__;
      __temp_6__ = (__temp_4__ + __temp_5__);
      float __temp_7__;
      __temp_7__ = (2 * input[__iter_0__+(2)+(M-0)*(__iter_1__+(-2))]);
      float __temp_8__;
      __temp_8__ = (__temp_6__ + __temp_7__);
      float __temp_9__;
      __temp_9__ = (4 * input[__iter_0__+(-2)+(M-0)*(__iter_1__+(-1))]);
      float __temp_10__;
      __temp_10__ = (__temp_8__ + __temp_9__);
      float __temp_11__;
      __temp_11__ = (9 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(-1))]);
      float __temp_12__;
      __temp_12__ = (__temp_10__ + __temp_11__);
      float __temp_13__;
      __temp_13__ = (12 * input[__iter_0__+(M-0)*(__iter_1__+(-1))]);
      float __temp_14__;
      __temp_14__ = (__temp_12__ + __temp_13__);
      float __temp_15__;
      __temp_15__ = (9 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(-1))]);
      float __temp_16__;
      __temp_16__ = (__temp_14__ + __temp_15__);
      float __temp_17__;
      __temp_17__ = (4 * input[__iter_0__+(2)+(M-0)*(__iter_1__+(-1))]);
      float __temp_18__;
      __temp_18__ = (__temp_16__ + __temp_17__);
      float __temp_19__;
      __temp_19__ = (5 * input[__iter_0__+(-2)+(M-0)*(__iter_1__)]);
      float __temp_20__;
      __temp_20__ = (__temp_18__ + __temp_19__);
      float __temp_21__;
      __temp_21__ = (12 * input[__iter_0__+(-1)+(M-0)*(__iter_1__)]);
      float __temp_22__;
      __temp_22__ = (__temp_20__ + __temp_21__);
      float __temp_23__;
      __temp_23__ = (15 * input[__iter_0__+(M-0)*(__iter_1__)]);
      float __temp_24__;
      __temp_24__ = (__temp_22__ + __temp_23__);
      float __temp_25__;
      __temp_25__ = (12 * input[__iter_0__+(1)+(M-0)*(__iter_1__)]);
      float __temp_26__;
      __temp_26__ = (__temp_24__ + __temp_25__);
      float __temp_27__;
      __temp_27__ = (5 * input[__iter_0__+(2)+(M-0)*(__iter_1__)]);
      float __temp_28__;
      __temp_28__ = (__temp_26__ + __temp_27__);
      float __temp_29__;
      __temp_29__ = (4 * input[__iter_0__+(-2)+(M-0)*(__iter_1__+(1))]);
      float __temp_30__;
      __temp_30__ = (__temp_28__ + __temp_29__);
      float __temp_31__;
      __temp_31__ = (9 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(1))]);
      float __temp_32__;
      __temp_32__ = (__temp_30__ + __temp_31__);
      float __temp_33__;
      __temp_33__ = (12 * input[__iter_0__+(M-0)*(__iter_1__+(1))]);
      float __temp_34__;
      __temp_34__ = (__temp_32__ + __temp_33__);
      float __temp_35__;
      __temp_35__ = (9 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(1))]);
      float __temp_36__;
      __temp_36__ = (__temp_34__ + __temp_35__);
      float __temp_37__;
      __temp_37__ = (4 * input[__iter_0__+(2)+(M-0)*(__iter_1__+(1))]);
      float __temp_38__;
      __temp_38__ = (__temp_36__ + __temp_37__);
      float __temp_39__;
      __temp_39__ = (2 * input[__iter_0__+(-2)+(M-0)*(__iter_1__+(2))]);
      float __temp_40__;
      __temp_40__ = (__temp_38__ + __temp_39__);
      float __temp_41__;
      __temp_41__ = (4 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(2))]);
      float __temp_42__;
      __temp_42__ = (__temp_40__ + __temp_41__);
      float __temp_43__;
      __temp_43__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(2))]);
      float __temp_44__;
      __temp_44__ = (__temp_42__ + __temp_43__);
      float __temp_45__;
      __temp_45__ = (4 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(2))]);
      float __temp_46__;
      __temp_46__ = (__temp_44__ + __temp_45__);
      float __temp_47__;
      __temp_47__ = (2 * input[__iter_0__+(2)+(M-0)*(__iter_1__+(2))]);
      float __temp_48__;
      __temp_48__ = (__temp_46__ + __temp_47__);
      float __temp_49__;
      __temp_49__ = (__temp_48__ / 159);
      __var_4__[__iter_0__+(M-0)*(__iter_1__)] = __temp_49__;
    }
  }
}
__global__ void __kernel___forma_kernel__1__(float * __restrict__ __var_4__, int N, int M, float * __restrict__ __var_3__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 2;
  if(__iter_2__ <= (M-3)){
    int __iter_3__;
    __iter_3__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 2;
    if(__iter_3__ <= (N-3)){
      float __temp_50__;
      __temp_50__ = (2 * __var_4__[__iter_2__+(-2)+(M-0)*(__iter_3__+(-2))]);
      float __temp_51__;
      __temp_51__ = (4 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(-2))]);
      float __temp_52__;
      __temp_52__ = (__temp_50__ + __temp_51__);
      float __temp_53__;
      __temp_53__ = (5 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(-2))]);
      float __temp_54__;
      __temp_54__ = (__temp_52__ + __temp_53__);
      float __temp_55__;
      __temp_55__ = (4 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(-2))]);
      float __temp_56__;
      __temp_56__ = (__temp_54__ + __temp_55__);
      float __temp_57__;
      __temp_57__ = (2 * __var_4__[__iter_2__+(2)+(M-0)*(__iter_3__+(-2))]);
      float __temp_58__;
      __temp_58__ = (__temp_56__ + __temp_57__);
      float __temp_59__;
      __temp_59__ = (4 * __var_4__[__iter_2__+(-2)+(M-0)*(__iter_3__+(-1))]);
      float __temp_60__;
      __temp_60__ = (__temp_58__ + __temp_59__);
      float __temp_61__;
      __temp_61__ = (9 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(-1))]);
      float __temp_62__;
      __temp_62__ = (__temp_60__ + __temp_61__);
      float __temp_63__;
      __temp_63__ = (12 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(-1))]);
      float __temp_64__;
      __temp_64__ = (__temp_62__ + __temp_63__);
      float __temp_65__;
      __temp_65__ = (9 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(-1))]);
      float __temp_66__;
      __temp_66__ = (__temp_64__ + __temp_65__);
      float __temp_67__;
      __temp_67__ = (4 * __var_4__[__iter_2__+(2)+(M-0)*(__iter_3__+(-1))]);
      float __temp_68__;
      __temp_68__ = (__temp_66__ + __temp_67__);
      float __temp_69__;
      __temp_69__ = (5 * __var_4__[__iter_2__+(-2)+(M-0)*(__iter_3__)]);
      float __temp_70__;
      __temp_70__ = (__temp_68__ + __temp_69__);
      float __temp_71__;
      __temp_71__ = (12 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__)]);
      float __temp_72__;
      __temp_72__ = (__temp_70__ + __temp_71__);
      float __temp_73__;
      __temp_73__ = (15 * __var_4__[__iter_2__+(M-0)*(__iter_3__)]);
      float __temp_74__;
      __temp_74__ = (__temp_72__ + __temp_73__);
      float __temp_75__;
      __temp_75__ = (12 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__)]);
      float __temp_76__;
      __temp_76__ = (__temp_74__ + __temp_75__);
      float __temp_77__;
      __temp_77__ = (5 * __var_4__[__iter_2__+(2)+(M-0)*(__iter_3__)]);
      float __temp_78__;
      __temp_78__ = (__temp_76__ + __temp_77__);
      float __temp_79__;
      __temp_79__ = (4 * __var_4__[__iter_2__+(-2)+(M-0)*(__iter_3__+(1))]);
      float __temp_80__;
      __temp_80__ = (__temp_78__ + __temp_79__);
      float __temp_81__;
      __temp_81__ = (9 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(1))]);
      float __temp_82__;
      __temp_82__ = (__temp_80__ + __temp_81__);
      float __temp_83__;
      __temp_83__ = (12 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(1))]);
      float __temp_84__;
      __temp_84__ = (__temp_82__ + __temp_83__);
      float __temp_85__;
      __temp_85__ = (9 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(1))]);
      float __temp_86__;
      __temp_86__ = (__temp_84__ + __temp_85__);
      float __temp_87__;
      __temp_87__ = (4 * __var_4__[__iter_2__+(2)+(M-0)*(__iter_3__+(1))]);
      float __temp_88__;
      __temp_88__ = (__temp_86__ + __temp_87__);
      float __temp_89__;
      __temp_89__ = (2 * __var_4__[__iter_2__+(-2)+(M-0)*(__iter_3__+(2))]);
      float __temp_90__;
      __temp_90__ = (__temp_88__ + __temp_89__);
      float __temp_91__;
      __temp_91__ = (4 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(2))]);
      float __temp_92__;
      __temp_92__ = (__temp_90__ + __temp_91__);
      float __temp_93__;
      __temp_93__ = (5 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(2))]);
      float __temp_94__;
      __temp_94__ = (__temp_92__ + __temp_93__);
      float __temp_95__;
      __temp_95__ = (4 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(2))]);
      float __temp_96__;
      __temp_96__ = (__temp_94__ + __temp_95__);
      float __temp_97__;
      __temp_97__ = (2 * __var_4__[__iter_2__+(2)+(M-0)*(__iter_3__+(2))]);
      float __temp_98__;
      __temp_98__ = (__temp_96__ + __temp_97__);
      float __temp_99__;
      __temp_99__ = (__temp_98__ / 159);
      __var_3__[__iter_2__+(M-0)*(__iter_3__)] = __temp_99__;
    }
  }
}
__global__ void __kernel___forma_kernel__2__(float * __restrict__ __var_3__, int N, int M, float * __restrict__ __var_2__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_4__;
  __iter_4__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 2;
  if(__iter_4__ <= (M-3)){
    int __iter_5__;
    __iter_5__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 2;
    if(__iter_5__ <= (N-3)){
      float __temp_100__;
      __temp_100__ = (2 * __var_3__[__iter_4__+(-2)+(M-0)*(__iter_5__+(-2))]);
      float __temp_101__;
      __temp_101__ = (4 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(-2))]);
      float __temp_102__;
      __temp_102__ = (__temp_100__ + __temp_101__);
      float __temp_103__;
      __temp_103__ = (5 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(-2))]);
      float __temp_104__;
      __temp_104__ = (__temp_102__ + __temp_103__);
      float __temp_105__;
      __temp_105__ = (4 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(-2))]);
      float __temp_106__;
      __temp_106__ = (__temp_104__ + __temp_105__);
      float __temp_107__;
      __temp_107__ = (2 * __var_3__[__iter_4__+(2)+(M-0)*(__iter_5__+(-2))]);
      float __temp_108__;
      __temp_108__ = (__temp_106__ + __temp_107__);
      float __temp_109__;
      __temp_109__ = (4 * __var_3__[__iter_4__+(-2)+(M-0)*(__iter_5__+(-1))]);
      float __temp_110__;
      __temp_110__ = (__temp_108__ + __temp_109__);
      float __temp_111__;
      __temp_111__ = (9 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(-1))]);
      float __temp_112__;
      __temp_112__ = (__temp_110__ + __temp_111__);
      float __temp_113__;
      __temp_113__ = (12 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(-1))]);
      float __temp_114__;
      __temp_114__ = (__temp_112__ + __temp_113__);
      float __temp_115__;
      __temp_115__ = (9 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(-1))]);
      float __temp_116__;
      __temp_116__ = (__temp_114__ + __temp_115__);
      float __temp_117__;
      __temp_117__ = (4 * __var_3__[__iter_4__+(2)+(M-0)*(__iter_5__+(-1))]);
      float __temp_118__;
      __temp_118__ = (__temp_116__ + __temp_117__);
      float __temp_119__;
      __temp_119__ = (5 * __var_3__[__iter_4__+(-2)+(M-0)*(__iter_5__)]);
      float __temp_120__;
      __temp_120__ = (__temp_118__ + __temp_119__);
      float __temp_121__;
      __temp_121__ = (12 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__)]);
      float __temp_122__;
      __temp_122__ = (__temp_120__ + __temp_121__);
      float __temp_123__;
      __temp_123__ = (15 * __var_3__[__iter_4__+(M-0)*(__iter_5__)]);
      float __temp_124__;
      __temp_124__ = (__temp_122__ + __temp_123__);
      float __temp_125__;
      __temp_125__ = (12 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__)]);
      float __temp_126__;
      __temp_126__ = (__temp_124__ + __temp_125__);
      float __temp_127__;
      __temp_127__ = (5 * __var_3__[__iter_4__+(2)+(M-0)*(__iter_5__)]);
      float __temp_128__;
      __temp_128__ = (__temp_126__ + __temp_127__);
      float __temp_129__;
      __temp_129__ = (4 * __var_3__[__iter_4__+(-2)+(M-0)*(__iter_5__+(1))]);
      float __temp_130__;
      __temp_130__ = (__temp_128__ + __temp_129__);
      float __temp_131__;
      __temp_131__ = (9 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(1))]);
      float __temp_132__;
      __temp_132__ = (__temp_130__ + __temp_131__);
      float __temp_133__;
      __temp_133__ = (12 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(1))]);
      float __temp_134__;
      __temp_134__ = (__temp_132__ + __temp_133__);
      float __temp_135__;
      __temp_135__ = (9 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(1))]);
      float __temp_136__;
      __temp_136__ = (__temp_134__ + __temp_135__);
      float __temp_137__;
      __temp_137__ = (4 * __var_3__[__iter_4__+(2)+(M-0)*(__iter_5__+(1))]);
      float __temp_138__;
      __temp_138__ = (__temp_136__ + __temp_137__);
      float __temp_139__;
      __temp_139__ = (2 * __var_3__[__iter_4__+(-2)+(M-0)*(__iter_5__+(2))]);
      float __temp_140__;
      __temp_140__ = (__temp_138__ + __temp_139__);
      float __temp_141__;
      __temp_141__ = (4 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(2))]);
      float __temp_142__;
      __temp_142__ = (__temp_140__ + __temp_141__);
      float __temp_143__;
      __temp_143__ = (5 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(2))]);
      float __temp_144__;
      __temp_144__ = (__temp_142__ + __temp_143__);
      float __temp_145__;
      __temp_145__ = (4 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(2))]);
      float __temp_146__;
      __temp_146__ = (__temp_144__ + __temp_145__);
      float __temp_147__;
      __temp_147__ = (2 * __var_3__[__iter_4__+(2)+(M-0)*(__iter_5__+(2))]);
      float __temp_148__;
      __temp_148__ = (__temp_146__ + __temp_147__);
      float __temp_149__;
      __temp_149__ = (__temp_148__ / 159);
      __var_2__[__iter_4__+(M-0)*(__iter_5__)] = __temp_149__;
    }
  }
}
__global__ void __kernel___forma_kernel__3__(float * __restrict__ __var_2__, int N, int M, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_6__;
  __iter_6__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 2;
  if(__iter_6__ <= (M-3)){
    int __iter_7__;
    __iter_7__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 2;
    if(__iter_7__ <= (N-3)){
      float __temp_150__;
      __temp_150__ = (2 * __var_2__[__iter_6__+(-2)+(M-0)*(__iter_7__+(-2))]);
      float __temp_151__;
      __temp_151__ = (4 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(-2))]);
      float __temp_152__;
      __temp_152__ = (__temp_150__ + __temp_151__);
      float __temp_153__;
      __temp_153__ = (5 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(-2))]);
      float __temp_154__;
      __temp_154__ = (__temp_152__ + __temp_153__);
      float __temp_155__;
      __temp_155__ = (4 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(-2))]);
      float __temp_156__;
      __temp_156__ = (__temp_154__ + __temp_155__);
      float __temp_157__;
      __temp_157__ = (2 * __var_2__[__iter_6__+(2)+(M-0)*(__iter_7__+(-2))]);
      float __temp_158__;
      __temp_158__ = (__temp_156__ + __temp_157__);
      float __temp_159__;
      __temp_159__ = (4 * __var_2__[__iter_6__+(-2)+(M-0)*(__iter_7__+(-1))]);
      float __temp_160__;
      __temp_160__ = (__temp_158__ + __temp_159__);
      float __temp_161__;
      __temp_161__ = (9 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(-1))]);
      float __temp_162__;
      __temp_162__ = (__temp_160__ + __temp_161__);
      float __temp_163__;
      __temp_163__ = (12 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(-1))]);
      float __temp_164__;
      __temp_164__ = (__temp_162__ + __temp_163__);
      float __temp_165__;
      __temp_165__ = (9 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(-1))]);
      float __temp_166__;
      __temp_166__ = (__temp_164__ + __temp_165__);
      float __temp_167__;
      __temp_167__ = (4 * __var_2__[__iter_6__+(2)+(M-0)*(__iter_7__+(-1))]);
      float __temp_168__;
      __temp_168__ = (__temp_166__ + __temp_167__);
      float __temp_169__;
      __temp_169__ = (5 * __var_2__[__iter_6__+(-2)+(M-0)*(__iter_7__)]);
      float __temp_170__;
      __temp_170__ = (__temp_168__ + __temp_169__);
      float __temp_171__;
      __temp_171__ = (12 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__)]);
      float __temp_172__;
      __temp_172__ = (__temp_170__ + __temp_171__);
      float __temp_173__;
      __temp_173__ = (15 * __var_2__[__iter_6__+(M-0)*(__iter_7__)]);
      float __temp_174__;
      __temp_174__ = (__temp_172__ + __temp_173__);
      float __temp_175__;
      __temp_175__ = (12 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__)]);
      float __temp_176__;
      __temp_176__ = (__temp_174__ + __temp_175__);
      float __temp_177__;
      __temp_177__ = (5 * __var_2__[__iter_6__+(2)+(M-0)*(__iter_7__)]);
      float __temp_178__;
      __temp_178__ = (__temp_176__ + __temp_177__);
      float __temp_179__;
      __temp_179__ = (4 * __var_2__[__iter_6__+(-2)+(M-0)*(__iter_7__+(1))]);
      float __temp_180__;
      __temp_180__ = (__temp_178__ + __temp_179__);
      float __temp_181__;
      __temp_181__ = (9 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(1))]);
      float __temp_182__;
      __temp_182__ = (__temp_180__ + __temp_181__);
      float __temp_183__;
      __temp_183__ = (12 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(1))]);
      float __temp_184__;
      __temp_184__ = (__temp_182__ + __temp_183__);
      float __temp_185__;
      __temp_185__ = (9 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(1))]);
      float __temp_186__;
      __temp_186__ = (__temp_184__ + __temp_185__);
      float __temp_187__;
      __temp_187__ = (4 * __var_2__[__iter_6__+(2)+(M-0)*(__iter_7__+(1))]);
      float __temp_188__;
      __temp_188__ = (__temp_186__ + __temp_187__);
      float __temp_189__;
      __temp_189__ = (2 * __var_2__[__iter_6__+(-2)+(M-0)*(__iter_7__+(2))]);
      float __temp_190__;
      __temp_190__ = (__temp_188__ + __temp_189__);
      float __temp_191__;
      __temp_191__ = (4 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(2))]);
      float __temp_192__;
      __temp_192__ = (__temp_190__ + __temp_191__);
      float __temp_193__;
      __temp_193__ = (5 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(2))]);
      float __temp_194__;
      __temp_194__ = (__temp_192__ + __temp_193__);
      float __temp_195__;
      __temp_195__ = (4 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(2))]);
      float __temp_196__;
      __temp_196__ = (__temp_194__ + __temp_195__);
      float __temp_197__;
      __temp_197__ = (2 * __var_2__[__iter_6__+(2)+(M-0)*(__iter_7__+(2))]);
      float __temp_198__;
      __temp_198__ = (__temp_196__ + __temp_197__);
      float __temp_199__;
      __temp_199__ = (__temp_198__ / 159);
      __var_1__[__iter_6__+(M-0)*(__iter_7__)] = __temp_199__;
    }
  }
}
/*Device code End */
/* Host Code Begin */
extern "C" void gaussian(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
  float * __var_3__;
  hipMalloc(&__var_3__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_3__\n");
  float * __var_4__;
  hipMalloc(&__var_4__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_4__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-3) - 2 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-3) - 2 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 32;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_4__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_4__, N, M, __var_3__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_3__, N, M, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, N, M, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
  hipFree(__var_3__);
  hipFree(__var_4__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel_j3d27pt0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __var_4__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_0__ <= (N-2)){
    int __iter_1__;
    __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_1__ <= (M-2)){
      int __iter_2__;
      __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_2__ <= (L-2)){
        float __temp_0__;
        __temp_0__ = (0.5f * input[__iter_0__-1+N*(__iter_1__-1+M*(__iter_2__-1))]);
        float __temp_1__;
        __temp_1__ = (0.7f * input[__iter_0__+N*(__iter_1__-1+M*(__iter_2__-1))]);
        float __temp_2__;
        __temp_2__ = (__temp_0__ + __temp_1__);
        float __temp_3__;
        __temp_3__ = (0.9f * input[__iter_0__+1+N*(__iter_1__-1+M*(__iter_2__-1))]);
        float __temp_4__;
        __temp_4__ = (__temp_2__ + __temp_3__);
        float __temp_5__;
        __temp_5__ = (1.2f * input[__iter_0__-1+N*(__iter_1__+M*(__iter_2__-1))]);
        float __temp_6__;
        __temp_6__ = (__temp_4__ + __temp_5__);
        float __temp_7__;
        __temp_7__ = (1.5f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__-1))]);
        float __temp_8__;
        __temp_8__ = (__temp_6__ + __temp_7__);
        float __temp_9__;
        __temp_9__ = (1.2f * input[__iter_0__+1+N*(__iter_1__+M*(__iter_2__-1))]);
        float __temp_10__;
        __temp_10__ = (__temp_8__ + __temp_9__);
        float __temp_11__;
        __temp_11__ = (0.9f * input[__iter_0__-1+N*(__iter_1__+1+M*(__iter_2__-1))]);
        float __temp_12__;
        __temp_12__ = (__temp_10__ + __temp_11__);
        float __temp_13__;
        __temp_13__ = (0.7f * input[__iter_0__+N*(__iter_1__+1+M*(__iter_2__-1))]);
        float __temp_14__;
        __temp_14__ = (__temp_12__ + __temp_13__);
        float __temp_15__;
        __temp_15__ = (0.5f * input[__iter_0__+1+N*(__iter_1__+1+M*(__iter_2__-1))]);
        float __temp_16__;
        __temp_16__ = (__temp_14__ + __temp_15__);
        float __temp_17__;
        __temp_17__ = (0.51f * input[__iter_0__-1+N*(__iter_1__-1+M*(__iter_2__))]);
        float __temp_18__;
        __temp_18__ = (__temp_16__ + __temp_17__);
        float __temp_19__;
        __temp_19__ = (0.71f * input[__iter_0__+N*(__iter_1__-1+M*(__iter_2__))]);
        float __temp_20__;
        __temp_20__ = (__temp_18__ + __temp_19__);
        float __temp_21__;
        __temp_21__ = (0.91f * input[__iter_0__+1+N*(__iter_1__-1+M*(__iter_2__))]);
        float __temp_22__;
        __temp_22__ = (__temp_20__ + __temp_21__);
        float __temp_23__;
        __temp_23__ = (1.21f * input[__iter_0__-1+N*(__iter_1__+M*(__iter_2__))]);
        float __temp_24__;
        __temp_24__ = (__temp_22__ + __temp_23__);
        float __temp_25__;
        __temp_25__ = (1.51f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__))]);
        float __temp_26__;
        __temp_26__ = (__temp_24__ + __temp_25__);
        float __temp_27__;
        __temp_27__ = (1.21f * input[__iter_0__+1+N*(__iter_1__+M*(__iter_2__))]);
        float __temp_28__;
        __temp_28__ = (__temp_26__ + __temp_27__);
        float __temp_29__;
        __temp_29__ = (0.91f * input[__iter_0__-1+N*(__iter_1__+1+M*(__iter_2__))]);
        float __temp_30__;
        __temp_30__ = (__temp_28__ + __temp_29__);
        float __temp_31__;
        __temp_31__ = (0.71f * input[__iter_0__+N*(__iter_1__+1+M*(__iter_2__))]);
        float __temp_32__;
        __temp_32__ = (__temp_30__ + __temp_31__);
        float __temp_33__;
        __temp_33__ = (0.51f * input[__iter_0__+1+N*(__iter_1__+1+M*(__iter_2__))]);
        float __temp_34__;
        __temp_34__ = (__temp_32__ + __temp_33__);
        float __temp_35__;
        __temp_35__ = (0.52f * input[__iter_0__-1+N*(__iter_1__-1+M*(__iter_2__+1))]);
        float __temp_36__;
        __temp_36__ = (__temp_34__ + __temp_35__);
        float __temp_37__;
        __temp_37__ = (0.72f * input[__iter_0__+N*(__iter_1__-1+M*(__iter_2__+1))]);
        float __temp_38__;
        __temp_38__ = (__temp_36__ + __temp_37__);
        float __temp_39__;
        __temp_39__ = (0.92f * input[__iter_0__+1+N*(__iter_1__-1+M*(__iter_2__+1))]);
        float __temp_40__;
        __temp_40__ = (__temp_38__ + __temp_39__);
        float __temp_41__;
        __temp_41__ = (1.22f * input[__iter_0__-1+N*(__iter_1__+M*(__iter_2__+1))]);
        float __temp_42__;
        __temp_42__ = (__temp_40__ + __temp_41__);
        float __temp_43__;
        __temp_43__ = (1.52f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__+1))]);
        float __temp_44__;
        __temp_44__ = (__temp_42__ + __temp_43__);
        float __temp_45__;
        __temp_45__ = (1.22f * input[__iter_0__+1+N*(__iter_1__+M*(__iter_2__+1))]);
        float __temp_46__;
        __temp_46__ = (__temp_44__ + __temp_45__);
        float __temp_47__;
        __temp_47__ = (0.92f * input[__iter_0__-1+N*(__iter_1__+1+M*(__iter_2__+1))]);
        float __temp_48__;
        __temp_48__ = (__temp_46__ + __temp_47__);
        float __temp_49__;
        __temp_49__ = (0.72f * input[__iter_0__+N*(__iter_1__+1+M*(__iter_2__+1))]);
        float __temp_50__;
        __temp_50__ = (__temp_48__ + __temp_49__);
        float __temp_51__;
        __temp_51__ = (0.52f * input[__iter_0__+1+N*(__iter_1__+1+M*(__iter_2__+1))]);
        float __temp_52__;
        __temp_52__ = (__temp_50__ + __temp_51__);
        float __temp_53__;
        __temp_53__ = (__temp_52__ / 159);
        __var_4__[__iter_0__+N*(__iter_1__+M*(__iter_2__))] = __temp_53__;
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d27pt (float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*(L*M*N), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel_j3d27pt0__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel_j3d27pt0__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel_j3d27pt0__ = ((L-2) - 1 ) + 1;
  int __block_0___kernel_j3d27pt0__ = 16;
  int __block_1___kernel_j3d27pt0__ = 4;
  int __block_2___kernel_j3d27pt0__ = 4;
  dim3 __blockConfig___kernel_j3d27pt0__(__block_0___kernel_j3d27pt0__,__block_1___kernel_j3d27pt0__,__block_2___kernel_j3d27pt0__);
  int __SMemSize___kernel_j3d27pt0__ = 0;
  int __grid_0___kernel_j3d27pt0__ = FORMA_CEIL(__size_0___kernel_j3d27pt0__,__block_0___kernel_j3d27pt0__);
  int __grid_1___kernel_j3d27pt0__ = FORMA_CEIL(__size_1___kernel_j3d27pt0__,__block_1___kernel_j3d27pt0__);
  int __grid_2___kernel_j3d27pt0__ = FORMA_CEIL(__size_2___kernel_j3d27pt0__,__block_2___kernel_j3d27pt0__);
  dim3 __gridConfig___kernel_j3d27pt0__(__grid_0___kernel_j3d27pt0__,__grid_1___kernel_j3d27pt0__,__grid_2___kernel_j3d27pt0__);

  __kernel_j3d27pt0__<<<__gridConfig___kernel_j3d27pt0__, __blockConfig___kernel_j3d27pt0__, __SMemSize___kernel_j3d27pt0__>>> (input, L, M, N, __var_2__);
  __kernel_j3d27pt0__<<<__gridConfig___kernel_j3d27pt0__, __blockConfig___kernel_j3d27pt0__, __SMemSize___kernel_j3d27pt0__>>> (__var_2__, L, M, N, __var_1__);
  __kernel_j3d27pt0__<<<__gridConfig___kernel_j3d27pt0__, __blockConfig___kernel_j3d27pt0__, __SMemSize___kernel_j3d27pt0__>>> (__var_1__, L, M, N, __var_2__);
  __kernel_j3d27pt0__<<<__gridConfig___kernel_j3d27pt0__, __blockConfig___kernel_j3d27pt0__, __SMemSize___kernel_j3d27pt0__>>> (__var_2__, L, M, N, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel_j3d27pt0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

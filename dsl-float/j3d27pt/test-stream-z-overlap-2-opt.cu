#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/* Device code Begin */
/* X, Y, Z */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  __shared__ float tilevar[2][32*32];
  float t[2], b[2], out=0.0f;

  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)-4);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)-4);

  // Initialize the values
  int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
  int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
      tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = 0.0f;
  }
  // Rest of the computation
  for (int __iter_2__ = 0; __iter_2__ <= L-1; __iter_2__++) {
    if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1))){
        tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = input[__iter_5__+N*(__iter_4__+M*(__iter_2__))]; 
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))){
	// Bottom 
	float __temp_3__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_7__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_8__ = (0.5f * __temp_3__ + 0.7f *__temp_7__);
        float __temp_12__ = (tilevar[0][ __iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_13__ = (__temp_8__ + 0.9f * __temp_12__);
        float __temp_17__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_18__ = (__temp_13__ + 1.2f * __temp_17__);
        float __temp_22__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_23__ = (__temp_18__ + 1.5f * __temp_22__);
        float __temp_27__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_28__ = (__temp_23__ + 1.2f * __temp_27__);
        float __temp_32__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_33__ = (__temp_28__ + 0.9f * __temp_32__);
        float __temp_37__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_38__ = (__temp_33__ + 0.7f * __temp_37__);
        float __temp_42__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_43__ = (__temp_38__ + 0.5f * __temp_42__) / 159;
	t[0] += __temp_43__;
	//Mid 
        float __temp_47__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_52__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_53__ = (0.51f * __temp_47__ + 0.71f * __temp_52__);
        float __temp_57__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_58__ = (__temp_53__ + 0.91f * __temp_57__);
        float __temp_62__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_63__ = (__temp_58__ + 1.21f * __temp_62__);
        float __temp_67__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_68__ = (__temp_63__ + 1.51f * __temp_67__);
        float __temp_72__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_73__ = (__temp_68__ + 1.21f * __temp_72__);
        float __temp_77__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_78__ = (__temp_73__ + 0.91f * __temp_77__);
        float __temp_82__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_83__ = (__temp_78__ + 0.71f * __temp_82__);
        float __temp_87__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_88__ = (__temp_83__ + 0.51f * __temp_87__) / 159;
	b[0] += __temp_88__;	
	// Top
        float __temp_92__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_97__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_98__ = (0.52f * __temp_92__ + 0.72f * __temp_97__);
        float __temp_102__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_103__ = (__temp_98__ + 0.92f * __temp_102__);
        float __temp_107__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_108__ = (__temp_103__ + 1.22f * __temp_107__);
        float __temp_112__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_113__ = (__temp_108__ + 1.52f * __temp_112__);
        float __temp_117__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_118__ = (__temp_113__ + 1.22f * __temp_117__);
        float __temp_122__ = (tilevar[0][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_123__ = (__temp_118__ + 0.92f * __temp_122__);
        float __temp_127__ = (tilevar[0][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_128__ = (__temp_123__ + 0.72f * __temp_127__);
        float __temp_132__ = (tilevar[0][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_133__ = (__temp_128__ + 0.52f * __temp_132__) / 159;
        tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] += __temp_133__;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) & __iter_5__ >= FORMA_MAX((__iter_0__+2),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))){
	float __temp_3__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_7__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_8__ = (0.5f * __temp_3__ + 0.7f *__temp_7__);
        float __temp_12__ = (tilevar[1][ __iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_13__ = (__temp_8__ + 0.9f * __temp_12__);
        float __temp_17__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_18__ = (__temp_13__ + 1.2f * __temp_17__);
        float __temp_22__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_23__ = (__temp_18__ + 1.5f * __temp_22__);
        float __temp_27__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_28__ = (__temp_23__ + 1.2f * __temp_27__);
        float __temp_32__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_33__ = (__temp_28__ + 0.9f * __temp_32__);
        float __temp_37__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_38__ = (__temp_33__ + 0.7f * __temp_37__);
        float __temp_42__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_43__ = (__temp_38__ + 0.5f * __temp_42__) / 159;
	t[1] += __temp_43__;
	//Mid 
        float __temp_47__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_52__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_53__ = (0.51f * __temp_47__ + 0.71f * __temp_52__);
        float __temp_57__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_58__ = (__temp_53__ + 0.91f * __temp_57__);
        float __temp_62__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_63__ = (__temp_58__ + 1.21f * __temp_62__);
        float __temp_67__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_68__ = (__temp_63__ + 1.51f * __temp_67__);
        float __temp_72__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_73__ = (__temp_68__ + 1.21f * __temp_72__);
        float __temp_77__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_78__ = (__temp_73__ + 0.91f * __temp_77__);
        float __temp_82__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_83__ = (__temp_78__ + 0.71f * __temp_82__);
        float __temp_87__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_88__ = (__temp_83__ + 0.51f * __temp_87__) / 159;
	b[1] += __temp_88__;	
	// Top
        float __temp_92__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_97__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_98__ = (0.52f * __temp_92__ + 0.72f * __temp_97__);
        float __temp_102__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_103__ = (__temp_98__ + 0.92f * __temp_102__);
        float __temp_107__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_108__ = (__temp_103__ + 1.22f * __temp_107__);
        float __temp_112__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_113__ = (__temp_108__ + 1.52f * __temp_112__);
        float __temp_117__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_118__ = (__temp_113__ + 1.22f * __temp_117__);
        float __temp_122__ = (tilevar[1][__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_123__ = (__temp_118__ + 0.92f * __temp_122__);
        float __temp_127__ = (tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_128__ = (__temp_123__ + 0.72f * __temp_127__);
        float __temp_132__ = (tilevar[1][__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_133__ = (__temp_128__ + 0.52f * __temp_132__) / 159;
	out += __temp_133__;
        __var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-2,0))] = out;
    }
    __syncthreads ();
    // Now rotate 
    tilevar[1][__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = b[0];
    b[0] = t[0];
    t[0] = 0.0;
    out= b[1]; 
    b[1] = t[1];
    t[1] = 0.0;
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d27pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*(L*M*N), memcpy_kind_h_input);
  }

  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = N;
  int __size_1___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 16;
  int __block_2___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-4);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y-4);
  int __grid_2___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 unrollConfig (__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig>>> (__var_2__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

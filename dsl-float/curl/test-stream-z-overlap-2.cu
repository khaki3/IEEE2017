#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(struct float4 * __restrict__ input, float dx, float dy, float dz, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, struct float4 * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  struct float4* __tilevar_0__ = (struct float4*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(struct float4)*((FORMA_BLOCKDIM_Z*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  
  float b_x = 0.0f, b_y = 0.0f;

  int __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X-2);
  int __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y-2);
  int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
  int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;

#pragma unroll 4 
  for (int __iter_2__ = 0; __iter_2__ < L-1; __iter_2__++) {
    if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),M-2) ){
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),N-2) ){
        float __temp_0__ = (input[__iter_5__+N*(__iter_4__+M*(__iter_2__+1))].y - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].y);
        float __temp_1__ = (dz * __temp_0__);
        float __temp_2__ = (input[__iter_5__+N*(__iter_4__+1+M*__iter_2__)].z - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].z);
        float __temp_3__ = (dy * __temp_2__);
        float __temp_4__ = (__temp_1__ - __temp_3__);
        float __temp_5__ = (input[__iter_5__+N*(__iter_4__+M*(__iter_2__+1))].x - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].x);
        float __temp_6__ = (dz * __temp_5__);
        float __temp_7__ = (input[__iter_5__+1+N*(__iter_4__+M*__iter_2__)].z - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].z);
        float __temp_8__ = (dx * __temp_7__);
        float __temp_9__ = (__temp_6__ - __temp_8__);
        float __temp_10__ = (input[__iter_5__+N*(__iter_4__+1+M*__iter_2__)].x - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].x);
        float __temp_11__ = (dy * __temp_10__);
        float __temp_12__ = (input[__iter_5__+1+N*(__iter_4__+M*__iter_2__)].y - input[__iter_5__+N*(__iter_4__+M*__iter_2__)].y);
        float __temp_13__ = (dx * __temp_12__);
        float __temp_14__ = (__temp_11__ - __temp_13__);
        float __temp_15__ = (__temp_4__ + input[__iter_5__+N*(__iter_4__+M*__iter_2__)].x);
        float __temp_16__ = (__temp_9__ + input[__iter_5__+N*(__iter_4__+M*__iter_2__)].y);
        float __temp_17__ = (__temp_14__ + input[__iter_5__+N*(__iter_4__+M*__iter_2__)].z);
	b_x = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].x;
	b_y = __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y;
        __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].x = __temp_15__;
        __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y = __temp_16__;
        __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].z = __temp_17__;
        __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].w = 0.0f;
      }
    }
    __syncthreads();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-1))){
      if(__iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-1))){
        float __temp_36__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y - b_y);
        float __temp_37__ = (dz * __temp_36__);
        float __temp_38__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].z - __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)].z);
        float __temp_39__ = (dy * __temp_38__);
        float __temp_40__ = (__temp_37__ - __temp_39__);
        float __temp_41__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].x - b_x);
        float __temp_42__ = (dz * __temp_41__);
        float __temp_43__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].z - __tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].z);
        float __temp_44__ = (dx * __temp_43__);
        float __temp_45__ = (__temp_42__ - __temp_44__);
        float __temp_46__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].x - __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)].x);
        float __temp_47__ = (dy * __temp_46__);
        float __temp_48__ = (__tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y - __tilevar_0__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y);
        float __temp_49__ = (dx * __temp_48__);
        float __temp_50__ = (__temp_47__ - __temp_49__);
        float __temp_51__ = (__temp_40__ + __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].x);
        float __temp_52__ = (__temp_45__ + __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].y);
        float __temp_53__ = (__temp_50__ + __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)].z);
        __var_1__[__iter_5__+N*(__iter_4__+M*__iter_2__)].x = __temp_51__;
        __var_1__[__iter_5__+N*(__iter_4__+M*__iter_2__)].y = __temp_52__;
        __var_1__[__iter_5__+N*(__iter_4__+M*__iter_2__)].z = __temp_53__;
        __var_1__[__iter_5__+N*(__iter_4__+M*__iter_2__)].w = 0.0f;
      }
    }
    __syncthreads ();
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(struct float4)*((FORMA_BLOCKDIM_Z*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void curl(struct float4 * h_input, float dx, float dy, float dz, int L, int M, int N, struct float4 * __var_0__){

/* Host allocation Begin */
  struct float4 * input;
  hipMalloc(&input,sizeof(struct float4)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(struct float4)*(L*M*N), memcpy_kind_h_input);
  }
  struct float4 * __var_1__;
  hipMalloc(&__var_1__,sizeof(struct float4)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = N;
  int __size_1___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 16;
  int __block_2___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-2);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y-2);
  int __grid_2___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, dx, dy, dz, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(struct float4)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(struct float4 * __restrict__ input, float dx, float dy, float dz, int L, int M, int N, struct float4 * __restrict__ __copy_arr_0__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, struct float4 * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  struct float4* __tilevar_0__ = (struct float4*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(struct float4)*((FORMA_BLOCKDIM_Z*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));

  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  __syncthreads();
  int __iter_3__;
  __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_3__ += (int)(blockDim.z) ){
    int __iter_4__;
    __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
    if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ){
      int __iter_5__;
      __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        float __temp_0__;
        __temp_0__ = (input[__iter_5__+N*(__iter_4__+M*(__iter_3__+1))].y - input[__iter_5__+N*(__iter_4__+M*__iter_3__)].y);
        float __temp_1__;
        __temp_1__ = (dz * __temp_0__);
        float __temp_2__;
        __temp_2__ = (input[__iter_5__+N*(__iter_4__+1+M*__iter_3__)].z - input[__iter_5__+N*(__iter_4__+M*__iter_3__)].z);
        float __temp_3__;
        __temp_3__ = (dy * __temp_2__);
        float __temp_4__;
        __temp_4__ = (__temp_1__ - __temp_3__);
        float __temp_5__;
        __temp_5__ = (input[__iter_5__+N*(__iter_4__+M*(__iter_3__+1))].x - input[__iter_5__+N*(__iter_4__+M*__iter_3__)].x);
        float __temp_6__;
        __temp_6__ = (dz * __temp_5__);
        float __temp_7__;
        __temp_7__ = (input[__iter_5__+1+N*(__iter_4__+M*__iter_3__)].z - input[__iter_5__+N*(__iter_4__+M*__iter_3__)].z);
        float __temp_8__;
        __temp_8__ = (dx * __temp_7__);
        float __temp_9__;
        __temp_9__ = (__temp_6__ - __temp_8__);
        float __temp_10__;
        __temp_10__ = (input[__iter_5__+N*(__iter_4__+1+M*__iter_3__)].x - input[__iter_5__+N*(__iter_4__+M*__iter_3__)].x);
        float __temp_11__;
        __temp_11__ = (dy * __temp_10__);
        float __temp_12__;
        __temp_12__ = (input[__iter_5__+1+N*(__iter_4__+M*__iter_3__)].y - input[__iter_5__+N*(__iter_4__+M*__iter_3__)].y);
        float __temp_13__;
        __temp_13__ = (dx * __temp_12__);
        float __temp_14__;
        __temp_14__ = (__temp_11__ - __temp_13__);
        float __temp_15__;
        __temp_15__ = (__temp_4__ + input[__iter_5__+N*(__iter_4__+M*__iter_3__)].x);
        float __temp_16__;
        __temp_16__ = (__temp_9__ + input[__iter_5__+N*(__iter_4__+M*__iter_3__)].y);
        float __temp_17__;
        __temp_17__ = (__temp_14__ + input[__iter_5__+N*(__iter_4__+M*__iter_3__)].z);
        __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_3__-__iter_2__))].x = __temp_15__;
        __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_3__-__iter_2__))].y = __temp_16__;
        __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_3__-__iter_2__))].z = __temp_17__;
        __tilevar_0__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_3__-__iter_2__))].w = 0.000000f;
      }
    }
  }
  int __iter_6__;
  __iter_6__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_6__ += (int)(blockDim.z) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ){
      int __iter_8__;
      __iter_8__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        if (__iter_6__ < (FORMA_MAX(__iter_2__,0)+2) || __iter_6__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))-2) || __iter_7__ < (FORMA_MAX(__iter_1__,0)+2) || __iter_7__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2) || __iter_8__ < (FORMA_MAX(__iter_0__,0)+2) || __iter_8__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))-2)) {
          __copy_arr_0__[__iter_8__+N*(__iter_7__+M*__iter_6__)].x = __tilevar_0__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))].x;
          __copy_arr_0__[__iter_8__+N*(__iter_7__+M*__iter_6__)].y = __tilevar_0__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))].y;
          __copy_arr_0__[__iter_8__+N*(__iter_7__+M*__iter_6__)].z = __tilevar_0__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))].z;
          __copy_arr_0__[__iter_8__+N*(__iter_7__+M*__iter_6__)].w = __tilevar_0__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_6__-__iter_2__))].w;

        }
      }
    }
  }
  __syncthreads();
  int __iter_12__;
  __iter_12__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_12__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-1)) ; __iter_12__ += (int)(blockDim.z) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    if( __iter_13__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-1)) ){
      int __iter_14__;
      __iter_14__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_14__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-1)) ){
        float __temp_36__;
        __temp_36__ = (__tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].y - __tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-1-__iter_2__))].y);
        float __temp_37__;
        __temp_37__ = (dz * __temp_36__);
        float __temp_38__;
        __temp_38__ = (__tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].z - __tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].z);
        float __temp_39__;
        __temp_39__ = (dy * __temp_38__);
        float __temp_40__;
        __temp_40__ = (__temp_37__ - __temp_39__);
        float __temp_41__;
        __temp_41__ = (__tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].x - __tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-1-__iter_2__))].x);
        float __temp_42__;
        __temp_42__ = (dz * __temp_41__);
        float __temp_43__;
        __temp_43__ = (__tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].z - __tilevar_0__[__iter_14__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].z);
        float __temp_44__;
        __temp_44__ = (dx * __temp_43__);
        float __temp_45__;
        __temp_45__ = (__temp_42__ - __temp_44__);
        float __temp_46__;
        __temp_46__ = (__tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].x - __tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-1-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].x);
        float __temp_47__;
        __temp_47__ = (dy * __temp_46__);
        float __temp_48__;
        __temp_48__ = (__tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].y - __tilevar_0__[__iter_14__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].y);
        float __temp_49__;
        __temp_49__ = (dx * __temp_48__);
        float __temp_50__;
        __temp_50__ = (__temp_47__ - __temp_49__);
        float __temp_51__;
        __temp_51__ = (__temp_40__ + __tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].x);
        float __temp_52__;
        __temp_52__ = (__temp_45__ + __tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].y);
        float __temp_53__;
        __temp_53__ = (__temp_50__ + __tilevar_0__[__iter_14__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_13__-__iter_1__+FORMA_BLOCKDIM_Y*(__iter_12__-__iter_2__))].z);
        __var_1__[__iter_14__+N*(__iter_13__+M*__iter_12__)].x = __temp_51__;
        __var_1__[__iter_14__+N*(__iter_13__+M*__iter_12__)].y = __temp_52__;
        __var_1__[__iter_14__+N*(__iter_13__+M*__iter_12__)].z = __temp_53__;
        __var_1__[__iter_14__+N*(__iter_13__+M*__iter_12__)].w = 0.000000f;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(struct float4)*((FORMA_BLOCKDIM_Z*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(struct float4 * __restrict__ input, float dx, float dy, float dz, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, struct float4 * __restrict__ __copy_arr_0__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_9__;
  __iter_9__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__;
    __iter_10__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
    for( ; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y) ){
      int __iter_11__;
      __iter_11__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        if (__iter_9__ < FORMA_MAX(__iter_2__,0) || __iter_9__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) || __iter_10__ < FORMA_MAX(__iter_1__,0) || __iter_10__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) || __iter_11__ < FORMA_MAX(__iter_0__,0) || __iter_11__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))) {
          float __temp_18__;
          __temp_18__ = (input[__iter_11__+N*(__iter_10__+M*(__iter_9__+1))].y - input[__iter_11__+N*(__iter_10__+M*__iter_9__)].y);
          float __temp_19__;
          __temp_19__ = (dz * __temp_18__);
          float __temp_20__;
          __temp_20__ = (input[__iter_11__+N*(__iter_10__+1+M*__iter_9__)].z - input[__iter_11__+N*(__iter_10__+M*__iter_9__)].z);
          float __temp_21__;
          __temp_21__ = (dy * __temp_20__);
          float __temp_22__;
          __temp_22__ = (__temp_19__ - __temp_21__);
          float __temp_23__;
          __temp_23__ = (input[__iter_11__+N*(__iter_10__+M*(__iter_9__+1))].x - input[__iter_11__+N*(__iter_10__+M*__iter_9__)].x);
          float __temp_24__;
          __temp_24__ = (dz * __temp_23__);
          float __temp_25__;
          __temp_25__ = (input[__iter_11__+1+N*(__iter_10__+M*__iter_9__)].z - input[__iter_11__+N*(__iter_10__+M*__iter_9__)].z);
          float __temp_26__;
          __temp_26__ = (dx * __temp_25__);
          float __temp_27__;
          __temp_27__ = (__temp_24__ - __temp_26__);
          float __temp_28__;
          __temp_28__ = (input[__iter_11__+N*(__iter_10__+1+M*__iter_9__)].x - input[__iter_11__+N*(__iter_10__+M*__iter_9__)].x);
          float __temp_29__;
          __temp_29__ = (dy * __temp_28__);
          float __temp_30__;
          __temp_30__ = (input[__iter_11__+1+N*(__iter_10__+M*__iter_9__)].y - input[__iter_11__+N*(__iter_10__+M*__iter_9__)].y);
          float __temp_31__;
          __temp_31__ = (dx * __temp_30__);
          float __temp_32__;
          __temp_32__ = (__temp_29__ - __temp_31__);
          float __temp_33__;
          __temp_33__ = (__temp_22__ + input[__iter_11__+N*(__iter_10__+M*__iter_9__)].x);
          float __temp_34__;
          __temp_34__ = (__temp_27__ + input[__iter_11__+N*(__iter_10__+M*__iter_9__)].y);
          float __temp_35__;
          __temp_35__ = (__temp_32__ + input[__iter_11__+N*(__iter_10__+M*__iter_9__)].z);
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*__iter_9__)].x = __temp_33__;
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*__iter_9__)].y = __temp_34__;
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*__iter_9__)].z = __temp_35__;
          __copy_arr_0__[__iter_11__+N*(__iter_10__+M*__iter_9__)].w = 0.000000f;
        }
      }
    }
  }
}

__global__ void __kernel___forma_kernel__2__(struct float4 * __restrict__ __copy_arr_0__, float dx, float dy, float dz, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, struct float4 * __restrict__ __var_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_15__;
  __iter_15__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__;
    __iter_16__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
    for( ; __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) ; __iter_16__ += (int)(blockDim.y) ){
      int __iter_17__;
      __iter_17__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        if (__iter_15__ < FORMA_MAX((__iter_2__+1),1) || __iter_15__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-1)) || __iter_16__ < FORMA_MAX((__iter_1__+1),1) || __iter_16__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-1)) || __iter_17__ < FORMA_MAX((__iter_0__+1),1) || __iter_17__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-1))) {
          float __temp_54__;
          __temp_54__ = (__copy_arr_0__[__iter_17__+N*(__iter_16__+M*__iter_15__)].y - __copy_arr_0__[__iter_17__+N*(__iter_16__+M*(__iter_15__-1))].y);
          float __temp_55__;
          __temp_55__ = (dz * __temp_54__);
          float __temp_56__;
          __temp_56__ = (__copy_arr_0__[__iter_17__+N*(__iter_16__+M*__iter_15__)].z - __copy_arr_0__[__iter_17__+N*(__iter_16__-1+M*__iter_15__)].z);
          float __temp_57__;
          __temp_57__ = (dy * __temp_56__);
          float __temp_58__;
          __temp_58__ = (__temp_55__ - __temp_57__);
          float __temp_59__;
          __temp_59__ = (__copy_arr_0__[__iter_17__+N*(__iter_16__+M*__iter_15__)].x - __copy_arr_0__[__iter_17__+N*(__iter_16__+M*(__iter_15__-1))].x);
          float __temp_60__;
          __temp_60__ = (dz * __temp_59__);
          float __temp_61__;
          __temp_61__ = (__copy_arr_0__[__iter_17__+N*(__iter_16__+M*__iter_15__)].z - __copy_arr_0__[__iter_17__-1+N*(__iter_16__+M*__iter_15__)].z);
          float __temp_62__;
          __temp_62__ = (dx * __temp_61__);
          float __temp_63__;
          __temp_63__ = (__temp_60__ - __temp_62__);
          float __temp_64__;
          __temp_64__ = (__copy_arr_0__[__iter_17__+N*(__iter_16__+M*__iter_15__)].x - __copy_arr_0__[__iter_17__+N*(__iter_16__-1+M*__iter_15__)].x);
          float __temp_65__;
          __temp_65__ = (dy * __temp_64__);
          float __temp_66__;
          __temp_66__ = (__copy_arr_0__[__iter_17__+N*(__iter_16__+M*__iter_15__)].y - __copy_arr_0__[__iter_17__-1+N*(__iter_16__+M*__iter_15__)].y);
          float __temp_67__;
          __temp_67__ = (dx * __temp_66__);
          float __temp_68__;
          __temp_68__ = (__temp_65__ - __temp_67__);
          float __temp_69__;
          __temp_69__ = (__temp_58__ + __copy_arr_0__[__iter_17__+N*(__iter_16__+M*__iter_15__)].x);
          float __temp_70__;
          __temp_70__ = (__temp_63__ + __copy_arr_0__[__iter_17__+N*(__iter_16__+M*__iter_15__)].y);
          float __temp_71__;
          __temp_71__ = (__temp_68__ + __copy_arr_0__[__iter_17__+N*(__iter_16__+M*__iter_15__)].z);
          __var_1__[__iter_17__+N*(__iter_16__+M*__iter_15__)].x = __temp_69__;
          __var_1__[__iter_17__+N*(__iter_16__+M*__iter_15__)].y = __temp_70__;
          __var_1__[__iter_17__+N*(__iter_16__+M*__iter_15__)].z = __temp_71__;
          __var_1__[__iter_17__+N*(__iter_16__+M*__iter_15__)].w = 0.000000f;
        }
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void curl(struct float4 * h_input, float dx, float dy, float dz, int L, int M, int N, struct float4 * __var_0__){

/* Host allocation Begin */
  struct float4 * input;
  hipMalloc(&input,sizeof(struct float4)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(struct float4)*(L*M*N), memcpy_kind_h_input);
  }
  struct float4 * __var_1__;
  hipMalloc(&__var_1__,sizeof(struct float4)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  struct float4 * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(struct float4)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_2___kernel___forma_kernel__0__ = ((L-1) - 0 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),1);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),1);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___2 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_2___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___2,__size_2___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_2),1);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_2___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.z/2 > 1)
      __blockConfig___kernel___forma_kernel__0__.z /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 1)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,1))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y);
  int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.z);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 __blockConfig_st__kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, 2);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig_st__kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, dx, dy, dz, L, M, N, __copy_arr_0__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  dim3 __blockConfig___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__1__>>> (input, dx, dy, dz, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __copy_arr_0__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  dim3 __blockConfig___kernel___forma_kernel__2__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__2__>>> (__copy_arr_0__, dx, dy, dz, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(struct float4)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
}
/*Host Free End*/

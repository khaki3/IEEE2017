#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __var_2__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 1;
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 1;
  int __iter_2__;
  __iter_2__ = (FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= (FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2))+1) ){
    int __iter_3__;
    __iter_3__ = (FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= (FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2))+1) ){
      int __temp_0__;
      __temp_0__ = __iter_2__;
      int __temp_1__;
      __temp_1__ = __iter_3__;
      __tilevar_1__[__iter_3__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_2__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] = input[__temp_1__+(M-0)*(__temp_0__)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2)) ){
      float __temp_2__;
      __temp_2__ = (__tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_3__;
      __temp_3__ = (__tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_4__;
      __temp_4__ = (__temp_2__ * __temp_3__);
      float __temp_5__;
      __temp_5__ = (0.000100f + __temp_4__);
      float __temp_6__;
      __temp_6__ = (__tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_7__;
      __temp_7__ = (__tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_8__;
      __temp_8__ = (__temp_6__ * __temp_7__);
      float __temp_9__;
      __temp_9__ = (__temp_5__ + __temp_8__);
      float __temp_10__;
      __temp_10__ = (__tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_5__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_11__;
      __temp_11__ = (__tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_5__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_12__;
      __temp_12__ = (__temp_10__ * __temp_11__);
      float __temp_13__;
      __temp_13__ = (__temp_9__ + __temp_12__);
      float __temp_14__;
      __temp_14__ = (__tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_5__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_15__;
      __temp_15__ = (__tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_5__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_16__;
      __temp_16__ = (__temp_14__ * __temp_15__);
      float __temp_17__;
      __temp_17__ = (__temp_13__ + __temp_16__);
      float __temp_18__;
      __temp_18__ = sqrt(__temp_17__);
      float __temp_19__;
      __temp_19__ = (1.000000f / __temp_18__);
      float __temp_20__;
      __temp_20__ = (__tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] + __temp_19__);
      __tilevar_0__[__iter_5__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_4__+(0-(FORMA_MAX(__iter_1__,1)-1)))] = __temp_20__;
    }
  }
  __syncthreads();
  int __iter_6__;
  __iter_6__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      float __temp_21__;
      __temp_21__ = (__tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(-1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_22__;
      __temp_22__ = (__tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(-1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_23__;
      __temp_23__ = (__temp_21__ * __temp_22__);
      float __temp_24__;
      __temp_24__ = (0.000100f + __temp_23__);
      float __temp_25__;
      __temp_25__ = (__tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_26__;
      __temp_26__ = (__tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_27__;
      __temp_27__ = (__temp_25__ * __temp_26__);
      float __temp_28__;
      __temp_28__ = (__temp_24__ + __temp_27__);
      float __temp_29__;
      __temp_29__ = (__tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_7__+(1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_30__;
      __temp_30__ = (__tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_7__+(1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_31__;
      __temp_31__ = (__temp_29__ * __temp_30__);
      float __temp_32__;
      __temp_32__ = (__temp_28__ + __temp_31__);
      float __temp_33__;
      __temp_33__ = (__tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_7__+(-1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_34__;
      __temp_34__ = (__tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_7__+(-1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_35__;
      __temp_35__ = (__temp_33__ * __temp_34__);
      float __temp_36__;
      __temp_36__ = (__temp_32__ + __temp_35__);
      float __temp_37__;
      __temp_37__ = sqrt(__temp_36__);
      float __temp_38__;
      __temp_38__ = (1.000000f / __temp_37__);
      float __temp_39__;
      __temp_39__ = (__tilevar_0__[__iter_7__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_6__+(0-(FORMA_MAX(__iter_1__,1)-1)))] + __temp_38__);
      __var_2__[__iter_7__+(M-0)*(__iter_6__)] = __temp_39__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ __var_2__, int N, int M, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  int __iter_8__;
  __iter_8__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 1;
  int __iter_9__;
  __iter_9__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 1;
  int __iter_10__;
  __iter_10__ = (FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)+(-1)) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= (FORMA_MIN((FORMA_MIN(((__iter_9__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2))+1) ){
    int __iter_11__;
    __iter_11__ = (FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)+(-1)) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= (FORMA_MIN((FORMA_MIN(((__iter_8__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2))+1) ){
      int __temp_40__;
      __temp_40__ = __iter_10__;
      int __temp_41__;
      __temp_41__ = __iter_11__;
      __tilevar_3__[__iter_11__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] = __var_2__[__temp_41__+(M-0)*(__temp_40__)];
    }
  }
  __syncthreads();
  int __iter_12__;
  __iter_12__ = FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN((FORMA_MIN(((__iter_9__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN((FORMA_MIN(((__iter_8__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2)) ){
      float __temp_42__;
      __temp_42__ = (__tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] - __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_43__;
      __temp_43__ = (__tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] - __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_44__;
      __temp_44__ = (__temp_42__ * __temp_43__);
      float __temp_45__;
      __temp_45__ = (0.000100f + __temp_44__);
      float __temp_46__;
      __temp_46__ = (__tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] - __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_47__;
      __temp_47__ = (__tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] - __tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_48__;
      __temp_48__ = (__temp_46__ * __temp_47__);
      float __temp_49__;
      __temp_49__ = (__temp_45__ + __temp_48__);
      float __temp_50__;
      __temp_50__ = (__tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] - __tilevar_3__[__iter_13__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_51__;
      __temp_51__ = (__tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] - __tilevar_3__[__iter_13__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_52__;
      __temp_52__ = (__temp_50__ * __temp_51__);
      float __temp_53__;
      __temp_53__ = (__temp_49__ + __temp_52__);
      float __temp_54__;
      __temp_54__ = (__tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] - __tilevar_3__[__iter_13__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_55__;
      __temp_55__ = (__tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] - __tilevar_3__[__iter_13__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))]);
      float __temp_56__;
      __temp_56__ = (__temp_54__ * __temp_55__);
      float __temp_57__;
      __temp_57__ = (__temp_53__ + __temp_56__);
      float __temp_58__;
      __temp_58__ = sqrt(__temp_57__);
      float __temp_59__;
      __temp_59__ = (1.000000f / __temp_58__);
      float __temp_60__;
      __temp_60__ = (__tilevar_3__[__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_8__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_9__,1)+(-1)),1)-1)))] + __temp_59__);
      __tilevar_2__[__iter_13__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_12__+(0-(FORMA_MAX(__iter_9__,1)-1)))] = __temp_60__;
    }
  }
  __syncthreads();
  int __iter_14__;
  __iter_14__ = FORMA_MAX(__iter_9__,1) + (int)(threadIdx.y) ; 
  if( __iter_14__ <= FORMA_MIN(((__iter_9__+FORMA_BLOCKDIM_Y)-1),(N-2)) ){
    int __iter_15__;
    __iter_15__ = FORMA_MAX(__iter_8__,1) + (int)(threadIdx.x) ; 
    if( __iter_15__ <= FORMA_MIN(((__iter_8__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      float __temp_61__;
      __temp_61__ = (__tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))] - __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(-1)+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_62__;
      __temp_62__ = (__tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))] - __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(-1)+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_63__;
      __temp_63__ = (__temp_61__ * __temp_62__);
      float __temp_64__;
      __temp_64__ = (0.000100f + __temp_63__);
      float __temp_65__;
      __temp_65__ = (__tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))] - __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(1)+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_66__;
      __temp_66__ = (__tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))] - __tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(1)+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_67__;
      __temp_67__ = (__temp_65__ * __temp_66__);
      float __temp_68__;
      __temp_68__ = (__temp_64__ + __temp_67__);
      float __temp_69__;
      __temp_69__ = (__tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))] - __tilevar_2__[__iter_15__+(1)+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_70__;
      __temp_70__ = (__tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))] - __tilevar_2__[__iter_15__+(1)+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_71__;
      __temp_71__ = (__temp_69__ * __temp_70__);
      float __temp_72__;
      __temp_72__ = (__temp_68__ + __temp_71__);
      float __temp_73__;
      __temp_73__ = (__tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))] - __tilevar_2__[__iter_15__+(-1)+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_74__;
      __temp_74__ = (__tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))] - __tilevar_2__[__iter_15__+(-1)+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))]);
      float __temp_75__;
      __temp_75__ = (__temp_73__ * __temp_74__);
      float __temp_76__;
      __temp_76__ = (__temp_72__ + __temp_75__);
      float __temp_77__;
      __temp_77__ = sqrt(__temp_76__);
      float __temp_78__;
      __temp_78__ = (1.000000f / __temp_77__);
      float __temp_79__;
      __temp_79__ = (__tilevar_2__[__iter_15__+(0-(FORMA_MAX(__iter_8__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_14__+(0-(FORMA_MAX(__iter_9__,1)-1)))] + __temp_78__);
      __var_1__[__iter_15__+(M-0)*(__iter_14__)] = __temp_79__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__1__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void gradient (float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),5);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),5);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 5)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,5))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  __block_0___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.x-(2-(-2));
  __block_1___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.y-(2-(-2));
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  int __size_0___kernel___forma_kernel__1__ = ((M-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__1__ = ((N-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__1__;
  int _max_occupancy_gridsize___kernel___forma_kernel__1__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__1__,&__max_occupancy_blocksize___kernel___forma_kernel__1__,(const void*)__kernel___forma_kernel__1__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__1___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__1__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__1___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__1___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__1__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__1___0,FORMA_MAX((__size_0___kernel___forma_kernel__1__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),5);
  __max_occupancy_blocksize___kernel___forma_kernel__1__ /= __block_0___kernel___forma_kernel__1__;
  int __max_occupancy_blocksize___kernel___forma_kernel__1___1 = __max_occupancy_blocksize___kernel___forma_kernel__1__;
  int __block_1___kernel___forma_kernel__1__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__1___1,__size_1___kernel___forma_kernel__1__),FORMA_MAX_BLOCKDIM_1),5);
  __max_occupancy_blocksize___kernel___forma_kernel__1__ /= __block_1___kernel___forma_kernel__1__;
  dim3 __blockConfig___kernel___forma_kernel__1__(__block_0___kernel___forma_kernel__1__,__block_1___kernel___forma_kernel__1__);
  int __SMemSize___kernel___forma_kernel__1__ = 0;
  __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
  while( __SMemSize___kernel___forma_kernel__1__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__1__.y/2 > 5)
      __blockConfig___kernel___forma_kernel__1__.y /= 2;
    __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
    if( __SMemSize___kernel___forma_kernel__1__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__1__.x/2 > FORMA_MIN(32,5))
      __blockConfig___kernel___forma_kernel__1__.x /= 2;
    __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
  }
  __block_0___kernel___forma_kernel__1__ = __blockConfig___kernel___forma_kernel__1__.x-(2-(-2));
  __block_1___kernel___forma_kernel__1__ = __blockConfig___kernel___forma_kernel__1__.y-(2-(-2));
  int __grid_0___kernel___forma_kernel__1__ = FORMA_CEIL(__size_0___kernel___forma_kernel__1__,__block_0___kernel___forma_kernel__1__);
  int __grid_1___kernel___forma_kernel__1__ = FORMA_CEIL(__size_1___kernel___forma_kernel__1__,__block_1___kernel___forma_kernel__1__);
  dim3 __gridConfig___kernel___forma_kernel__1__(__grid_0___kernel___forma_kernel__1__,__grid_1___kernel___forma_kernel__1__);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__1__, __blockConfig___kernel___forma_kernel__1__, __SMemSize___kernel___forma_kernel__1__>>> (__var_2__, N, M, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

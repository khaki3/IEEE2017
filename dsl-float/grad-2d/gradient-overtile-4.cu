#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-6))*(FORMA_BLOCKDIM_X-(-6))));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 1;
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 1;
  int __iter_2__;
  __iter_2__ = (FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)+(-1)) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= (FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2))+1),(N-2))+1),(N-2))+1) ){
    int __iter_3__;
    __iter_3__ = (FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)+(-1)) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= (FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2))+1),(M-2))+1),(M-2))+1) ){
      int __temp_0__;
      __temp_0__ = __iter_2__;
      int __temp_1__;
      __temp_1__ = __iter_3__;
      __tilevar_3__[__iter_3__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__iter_2__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] = input[__temp_1__+(M-0)*(__temp_0__)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2))+1),(N-2))+1),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2))+1),(M-2))+1),(M-2)) ){
      int __temp_2__;
      __temp_2__ = __iter_4__;
      int __temp_3__;
      __temp_3__ = __iter_5__;
      int __temp_4__;
      __temp_4__ = __iter_4__;
      int __temp_5__;
      __temp_5__ = __iter_5__;
      int __temp_6__;
      __temp_6__ = __iter_4__+(-1);
      int __temp_7__;
      __temp_7__ = __iter_5__;
      float __temp_8__;
      __temp_8__ = (__tilevar_3__[__temp_5__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_4__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_3__[__temp_7__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_6__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      int __temp_9__;
      __temp_9__ = __iter_4__;
      int __temp_10__;
      __temp_10__ = __iter_5__;
      int __temp_11__;
      __temp_11__ = __iter_4__+(-1);
      int __temp_12__;
      __temp_12__ = __iter_5__;
      float __temp_13__;
      __temp_13__ = (__tilevar_3__[__temp_10__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_9__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_3__[__temp_12__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_11__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_14__;
      __temp_14__ = (__temp_8__ * __temp_13__);
      float __temp_15__;
      __temp_15__ = (0.000100f + __temp_14__);
      int __temp_16__;
      __temp_16__ = __iter_4__;
      int __temp_17__;
      __temp_17__ = __iter_5__;
      int __temp_18__;
      __temp_18__ = __iter_4__+(1);
      int __temp_19__;
      __temp_19__ = __iter_5__;
      float __temp_20__;
      __temp_20__ = (__tilevar_3__[__temp_17__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_16__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_3__[__temp_19__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_18__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      int __temp_21__;
      __temp_21__ = __iter_4__;
      int __temp_22__;
      __temp_22__ = __iter_5__;
      int __temp_23__;
      __temp_23__ = __iter_4__+(1);
      int __temp_24__;
      __temp_24__ = __iter_5__;
      float __temp_25__;
      __temp_25__ = (__tilevar_3__[__temp_22__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_21__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_3__[__temp_24__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_23__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_26__;
      __temp_26__ = (__temp_20__ * __temp_25__);
      float __temp_27__;
      __temp_27__ = (__temp_15__ + __temp_26__);
      int __temp_28__;
      __temp_28__ = __iter_4__;
      int __temp_29__;
      __temp_29__ = __iter_5__;
      int __temp_30__;
      __temp_30__ = __iter_4__;
      int __temp_31__;
      __temp_31__ = __iter_5__+(1);
      float __temp_32__;
      __temp_32__ = (__tilevar_3__[__temp_29__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_28__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_3__[__temp_31__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_30__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      int __temp_33__;
      __temp_33__ = __iter_4__;
      int __temp_34__;
      __temp_34__ = __iter_5__;
      int __temp_35__;
      __temp_35__ = __iter_4__;
      int __temp_36__;
      __temp_36__ = __iter_5__+(1);
      float __temp_37__;
      __temp_37__ = (__tilevar_3__[__temp_34__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_33__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_3__[__temp_36__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_35__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_38__;
      __temp_38__ = (__temp_32__ * __temp_37__);
      float __temp_39__;
      __temp_39__ = (__temp_27__ + __temp_38__);
      int __temp_40__;
      __temp_40__ = __iter_4__;
      int __temp_41__;
      __temp_41__ = __iter_5__;
      int __temp_42__;
      __temp_42__ = __iter_4__;
      int __temp_43__;
      __temp_43__ = __iter_5__+(-1);
      float __temp_44__;
      __temp_44__ = (__tilevar_3__[__temp_41__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_40__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_3__[__temp_43__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_42__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      int __temp_45__;
      __temp_45__ = __iter_4__;
      int __temp_46__;
      __temp_46__ = __iter_5__;
      int __temp_47__;
      __temp_47__ = __iter_4__;
      int __temp_48__;
      __temp_48__ = __iter_5__+(-1);
      float __temp_49__;
      __temp_49__ = (__tilevar_3__[__temp_46__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_45__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_3__[__temp_48__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_47__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_50__;
      __temp_50__ = (__temp_44__ * __temp_49__);
      float __temp_51__;
      __temp_51__ = (__temp_39__ + __temp_50__);
      float __temp_52__;
      __temp_52__ = sqrt(__temp_51__);
      float __temp_53__;
      __temp_53__ = (1.000000f / __temp_52__);
      float __temp_54__;
      __temp_54__ = (__tilevar_3__[__temp_3__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_2__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1)))] + __temp_53__);
      __tilevar_2__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] = __temp_54__;
    }
  }
  __syncthreads();
  int __iter_6__;
  __iter_6__ = FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2))+1),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2))+1),(M-2)) ){
      int __temp_55__;
      __temp_55__ = __iter_6__;
      int __temp_56__;
      __temp_56__ = __iter_7__;
      int __temp_57__;
      __temp_57__ = __iter_6__;
      int __temp_58__;
      __temp_58__ = __iter_7__;
      int __temp_59__;
      __temp_59__ = __iter_6__+(-1);
      int __temp_60__;
      __temp_60__ = __iter_7__;
      float __temp_61__;
      __temp_61__ = (__tilevar_2__[__temp_58__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_57__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_2__[__temp_60__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_59__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      int __temp_62__;
      __temp_62__ = __iter_6__;
      int __temp_63__;
      __temp_63__ = __iter_7__;
      int __temp_64__;
      __temp_64__ = __iter_6__+(-1);
      int __temp_65__;
      __temp_65__ = __iter_7__;
      float __temp_66__;
      __temp_66__ = (__tilevar_2__[__temp_63__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_62__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_2__[__temp_65__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_64__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_67__;
      __temp_67__ = (__temp_61__ * __temp_66__);
      float __temp_68__;
      __temp_68__ = (0.000100f + __temp_67__);
      int __temp_69__;
      __temp_69__ = __iter_6__;
      int __temp_70__;
      __temp_70__ = __iter_7__;
      int __temp_71__;
      __temp_71__ = __iter_6__+(1);
      int __temp_72__;
      __temp_72__ = __iter_7__;
      float __temp_73__;
      __temp_73__ = (__tilevar_2__[__temp_70__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_69__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_2__[__temp_72__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_71__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      int __temp_74__;
      __temp_74__ = __iter_6__;
      int __temp_75__;
      __temp_75__ = __iter_7__;
      int __temp_76__;
      __temp_76__ = __iter_6__+(1);
      int __temp_77__;
      __temp_77__ = __iter_7__;
      float __temp_78__;
      __temp_78__ = (__tilevar_2__[__temp_75__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_74__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_2__[__temp_77__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_76__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_79__;
      __temp_79__ = (__temp_73__ * __temp_78__);
      float __temp_80__;
      __temp_80__ = (__temp_68__ + __temp_79__);
      int __temp_81__;
      __temp_81__ = __iter_6__;
      int __temp_82__;
      __temp_82__ = __iter_7__;
      int __temp_83__;
      __temp_83__ = __iter_6__;
      int __temp_84__;
      __temp_84__ = __iter_7__+(1);
      float __temp_85__;
      __temp_85__ = (__tilevar_2__[__temp_82__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_81__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_2__[__temp_84__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_83__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      int __temp_86__;
      __temp_86__ = __iter_6__;
      int __temp_87__;
      __temp_87__ = __iter_7__;
      int __temp_88__;
      __temp_88__ = __iter_6__;
      int __temp_89__;
      __temp_89__ = __iter_7__+(1);
      float __temp_90__;
      __temp_90__ = (__tilevar_2__[__temp_87__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_86__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_2__[__temp_89__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_88__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_91__;
      __temp_91__ = (__temp_85__ * __temp_90__);
      float __temp_92__;
      __temp_92__ = (__temp_80__ + __temp_91__);
      int __temp_93__;
      __temp_93__ = __iter_6__;
      int __temp_94__;
      __temp_94__ = __iter_7__;
      int __temp_95__;
      __temp_95__ = __iter_6__;
      int __temp_96__;
      __temp_96__ = __iter_7__+(-1);
      float __temp_97__;
      __temp_97__ = (__tilevar_2__[__temp_94__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_93__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_2__[__temp_96__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_95__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      int __temp_98__;
      __temp_98__ = __iter_6__;
      int __temp_99__;
      __temp_99__ = __iter_7__;
      int __temp_100__;
      __temp_100__ = __iter_6__;
      int __temp_101__;
      __temp_101__ = __iter_7__+(-1);
      float __temp_102__;
      __temp_102__ = (__tilevar_2__[__temp_99__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_98__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] - __tilevar_2__[__temp_101__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_100__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))]);
      float __temp_103__;
      __temp_103__ = (__temp_97__ * __temp_102__);
      float __temp_104__;
      __temp_104__ = (__temp_92__ + __temp_103__);
      float __temp_105__;
      __temp_105__ = sqrt(__temp_104__);
      float __temp_106__;
      __temp_106__ = (1.000000f / __temp_105__);
      float __temp_107__;
      __temp_107__ = (__tilevar_2__[__temp_56__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_55__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1)))] + __temp_106__);
      __tilevar_1__[__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] = __temp_107__;
    }
  }
  __syncthreads();
  int __iter_8__;
  __iter_8__ = FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1) + (int)(threadIdx.y) ; 
  if( __iter_8__ <= FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2))+1),(N-2)) ){
    int __iter_9__;
    __iter_9__ = FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1) + (int)(threadIdx.x) ; 
    if( __iter_9__ <= FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2))+1),(M-2)) ){
      float __temp_108__;
      __temp_108__ = (__tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_109__;
      __temp_109__ = (__tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_110__;
      __temp_110__ = (__temp_108__ * __temp_109__);
      float __temp_111__;
      __temp_111__ = (0.000100f + __temp_110__);
      float __temp_112__;
      __temp_112__ = (__tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_113__;
      __temp_113__ = (__tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_114__;
      __temp_114__ = (__temp_112__ * __temp_113__);
      float __temp_115__;
      __temp_115__ = (__temp_111__ + __temp_114__);
      float __temp_116__;
      __temp_116__ = (__tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_9__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_117__;
      __temp_117__ = (__tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_9__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_118__;
      __temp_118__ = (__temp_116__ * __temp_117__);
      float __temp_119__;
      __temp_119__ = (__temp_115__ + __temp_118__);
      float __temp_120__;
      __temp_120__ = (__tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_9__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_121__;
      __temp_121__ = (__tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] - __tilevar_1__[__iter_9__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))]);
      float __temp_122__;
      __temp_122__ = (__temp_120__ * __temp_121__);
      float __temp_123__;
      __temp_123__ = (__temp_119__ + __temp_122__);
      float __temp_124__;
      __temp_124__ = sqrt(__temp_123__);
      float __temp_125__;
      __temp_125__ = (1.000000f / __temp_124__);
      float __temp_126__;
      __temp_126__ = (__tilevar_1__[__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1)))] + __temp_125__);
      __tilevar_0__[__iter_9__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_8__+(0-(FORMA_MAX(__iter_1__,1)-1)))] = __temp_126__;
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-2)) ){
      float __temp_127__;
      __temp_127__ = (__tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(-1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_128__;
      __temp_128__ = (__tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(-1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_129__;
      __temp_129__ = (__temp_127__ * __temp_128__);
      float __temp_130__;
      __temp_130__ = (0.000100f + __temp_129__);
      float __temp_131__;
      __temp_131__ = (__tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_132__;
      __temp_132__ = (__tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(1)+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_133__;
      __temp_133__ = (__temp_131__ * __temp_132__);
      float __temp_134__;
      __temp_134__ = (__temp_130__ + __temp_133__);
      float __temp_135__;
      __temp_135__ = (__tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_11__+(1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_136__;
      __temp_136__ = (__tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_11__+(1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_137__;
      __temp_137__ = (__temp_135__ * __temp_136__);
      float __temp_138__;
      __temp_138__ = (__temp_134__ + __temp_137__);
      float __temp_139__;
      __temp_139__ = (__tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_11__+(-1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_140__;
      __temp_140__ = (__tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))] - __tilevar_0__[__iter_11__+(-1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))]);
      float __temp_141__;
      __temp_141__ = (__temp_139__ * __temp_140__);
      float __temp_142__;
      __temp_142__ = (__temp_138__ + __temp_141__);
      float __temp_143__;
      __temp_143__ = sqrt(__temp_142__);
      float __temp_144__;
      __temp_144__ = (1.000000f / __temp_143__);
      float __temp_145__;
      __temp_145__ = (__tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,1)-1)))] + __temp_144__);
      __var_1__[__iter_11__+(M-0)*(__iter_10__)] = __temp_145__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-6))*(FORMA_BLOCKDIM_X-(-6))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void gradient (float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,9))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  __block_0___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.x-(4-(-4));
  __block_1___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.y-(4-(-4));
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
  hipError_t error = hipGetLastError ();
  if (error != hipSuccess) {
    printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
    exit(-1);
  }
}

__global__ void possion (float * __restrict__ in, float * __restrict__ out, float * __restrict__ out1, float * __restrict__ out2, float * __restrict__ out3, int L, int M, int N) {
  //Determining the block's indices
  int blockdim_i= (int)(blockDim.x);
  int i0 = (int)(blockIdx.x)*(blockdim_i-8);
  int i = i0 + (int)(threadIdx.x);
  int blockdim_j= (int)(blockDim.y);
  int j0 = (int)(blockIdx.y)*(blockdim_j-8);
  int j = j0 + (int)(threadIdx.y);

  //Declarations
  float __shared__ sh_in_c0[32][32];
  float __shared__ sh_out_m1[32][32];
  float reg_out_c0=0, reg_out_p1=0;
  float __shared__ sh_out1_m2[32][32];
  float reg_out1_m1=0, reg_out1_c0=0;
  float __shared__ sh_out2_m3[32][32];
  float reg_out2_m2=0, reg_out2_m1=0;
  float reg_out3_m4=0, reg_out3_m3=0, reg_out3_m2=0;

  //Value Initialization
  if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
    sh_in_c0[j-j0][i-i0] = 0;
    sh_out_m1[j-j0][i-i0] = 0;
    sh_out1_m2[j-j0][i-i0] = 0;
    sh_out2_m3[j-j0][i-i0] = 0;
  }
  //Rest of the computation
	#pragma unroll 2
  for (int k=0; k<=L-1; ++k) {
    //Fetch new plane
    if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
        sh_in_c0[j-j0][i-i0] = in[k*M*N + j*N + i];
    }
    __syncthreads ();
    if (j >= j0+1 & j <= min (j0+blockdim_j-2, M-2) & i >= i0+1 & i <= min (i0+blockdim_i-2, N-2)) {
        reg_out_c0 += (((2.666f * sh_in_c0[j-j0][i-i0]) - (0.166f * (((sh_in_c0[j-j0-1][i-i0] + sh_in_c0[j-j0+1][i-i0]) + sh_in_c0[j-j0][i-i0+1]) + sh_in_c0[j-j0][i-i0-1]))) - (0.0833f * (((sh_in_c0[j-j0-1][i-i0-1] + sh_in_c0[j-j0+1][i-i0-1]) + sh_in_c0[j-j0-1][i-i0+1]) + sh_in_c0[j-j0+1][i-i0+1])));
        reg_out_p1 += ((-(0.166f) * sh_in_c0[j-j0][i-i0]) - (0.0833f * (((sh_in_c0[j-j0-1][i-i0] + sh_in_c0[j-j0+1][i-i0]) + sh_in_c0[j-j0][i-i0-1]) + sh_in_c0[j-j0][i-i0+1])));
        sh_out_m1[j-j0][i-i0] += ((-(0.166f) * sh_in_c0[j-j0][i-i0]) - (0.0833f * (((sh_in_c0[j-j0-1][i-i0] + sh_in_c0[j-j0+1][i-i0]) + sh_in_c0[j-j0][i-i0-1]) + sh_in_c0[j-j0][i-i0+1])));
    }
    __syncthreads ();
    if (j >= j0+2 & j <= min (j0+blockdim_j-3, M-2) & i >= i0+2 & i <= min (i0+blockdim_i-3, N-2)) {
        reg_out1_m1 += (((2.666f * sh_out_m1[j-j0][i-i0]) - (0.166f * (((sh_out_m1[j-j0-1][i-i0] + sh_out_m1[j-j0+1][i-i0]) + sh_out_m1[j-j0][i-i0+1]) + sh_out_m1[j-j0][i-i0-1]))) - (0.0833f * (((sh_out_m1[j-j0-1][i-i0-1] + sh_out_m1[j-j0+1][i-i0-1]) + sh_out_m1[j-j0-1][i-i0+1]) + sh_out_m1[j-j0+1][i-i0+1])));
        reg_out1_c0 += ((-(0.166f) * sh_out_m1[j-j0][i-i0]) - (0.0833f * (((sh_out_m1[j-j0-1][i-i0] + sh_out_m1[j-j0+1][i-i0]) + sh_out_m1[j-j0][i-i0-1]) + sh_out_m1[j-j0][i-i0+1])));
        sh_out1_m2[j-j0][i-i0] += ((-(0.166f) * sh_out_m1[j-j0][i-i0]) - (0.0833f * (((sh_out_m1[j-j0-1][i-i0] + sh_out_m1[j-j0+1][i-i0]) + sh_out_m1[j-j0][i-i0-1]) + sh_out_m1[j-j0][i-i0+1])));
    }
    __syncthreads ();
    if (j >= j0+3 & j <= min (j0+blockdim_j-4, M-2) & i >= i0+3 & i <= min (i0+blockdim_i-4, N-2)) {
        reg_out2_m2 += (((2.666f * sh_out1_m2[j-j0][i-i0]) - (0.166f * (((sh_out1_m2[j-j0-1][i-i0] + sh_out1_m2[j-j0+1][i-i0]) + sh_out1_m2[j-j0][i-i0+1]) + sh_out1_m2[j-j0][i-i0-1]))) - (0.0833f * (((sh_out1_m2[j-j0-1][i-i0-1] + sh_out1_m2[j-j0+1][i-i0-1]) + sh_out1_m2[j-j0-1][i-i0+1]) + sh_out1_m2[j-j0+1][i-i0+1])));
        reg_out2_m1 += ((-(0.166f) * sh_out1_m2[j-j0][i-i0]) - (0.0833f * (((sh_out1_m2[j-j0-1][i-i0] + sh_out1_m2[j-j0+1][i-i0]) + sh_out1_m2[j-j0][i-i0-1]) + sh_out1_m2[j-j0][i-i0+1])));
        sh_out2_m3[j-j0][i-i0] += ((-(0.166f) * sh_out1_m2[j-j0][i-i0]) - (0.0833f * (((sh_out1_m2[j-j0-1][i-i0] + sh_out1_m2[j-j0+1][i-i0]) + sh_out1_m2[j-j0][i-i0-1]) + sh_out1_m2[j-j0][i-i0+1])));
    }
    __syncthreads ();
    if (j >= j0+4 & j <= min (j0+blockdim_j-5, M-2) & i >= i0+4 & i <= min (i0+blockdim_i-5, N-2)) {
        reg_out3_m3 += (((2.666f * sh_out2_m3[j-j0][i-i0]) - (0.166f * (((sh_out2_m3[j-j0-1][i-i0] + sh_out2_m3[j-j0+1][i-i0]) + sh_out2_m3[j-j0][i-i0+1]) + sh_out2_m3[j-j0][i-i0-1]))) - (0.0833f * (((sh_out2_m3[j-j0-1][i-i0-1] + sh_out2_m3[j-j0+1][i-i0-1]) + sh_out2_m3[j-j0-1][i-i0+1]) + sh_out2_m3[j-j0+1][i-i0+1])));
        reg_out3_m2 += ((-(0.166f) * sh_out2_m3[j-j0][i-i0]) - (0.0833f * (((sh_out2_m3[j-j0-1][i-i0] + sh_out2_m3[j-j0+1][i-i0]) + sh_out2_m3[j-j0][i-i0-1]) + sh_out2_m3[j-j0][i-i0+1])));
        reg_out3_m4 += ((-(0.166f) * sh_out2_m3[j-j0][i-i0]) - (0.0833f * (((sh_out2_m3[j-j0-1][i-i0] + sh_out2_m3[j-j0+1][i-i0]) + sh_out2_m3[j-j0][i-i0-1]) + sh_out2_m3[j-j0][i-i0+1])));
        out3[max(k-4,0)*M*N + j*N + i] = reg_out3_m4;
    }
    __syncthreads ();
    //Value rotation
    sh_out_m1[j-j0][i-i0] = reg_out_c0;
    reg_out_c0 = reg_out_p1;
    reg_out_p1 = 0;
    sh_out1_m2[j-j0][i-i0] = reg_out1_m1;
    reg_out1_m1 = reg_out1_c0;
    reg_out1_c0 = 0;
    sh_out2_m3[j-j0][i-i0] = reg_out2_m2;
    reg_out2_m2 = reg_out2_m1;
    reg_out2_m1 = 0;
    reg_out3_m4 = reg_out3_m3;
    reg_out3_m3 = reg_out3_m2;
    reg_out3_m2 = 0;
  }
}

extern "C" void host_code (float *h_in, float *h_out3, int L, int M, int N) {
  float *in;
  hipMalloc (&in, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for in\n");
  hipMemcpy (in, h_in, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *out1;
  hipMalloc (&out1, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for out1\n");
  float *out2;
  hipMalloc (&out2, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for out2\n");
  float *out3;
  hipMalloc (&out3, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for out3\n");
  float *out;
  hipMalloc (&out, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for out\n");

  dim3 blockconfig_1 (32, 32, 1);
  dim3 gridconfig_1 (ceil(N, blockconfig_1.x-8), ceil(M, blockconfig_1.y-8), 1);
  possion <<<gridconfig_1, blockconfig_1>>> (in, out, out1, out2, out3, L, M, N);

  hipMemcpy (h_out3, out3, sizeof(float)*L*M*N, hipMemcpyDeviceToHost);

  //Free allocated memory
  hipFree (in);
  hipFree (out1);
  hipFree (out2);
  hipFree (out3);
  hipFree (out);
}

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define GAPX (22) 
#define GAPY (22)
#define EXTENT (5)

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X) + GAPX);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y) + GAPY);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(0-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      int __temp_0__;
      __temp_0__ = __iter_4__+(-1);
      int __temp_1__;
      __temp_1__ = __iter_5__;
      float __temp_2__;
      __temp_2__ = (5 * __tilevar_2__[__temp_1__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_0__+(0-(__iter_1__+0)))]);
      int __temp_3__;
      __temp_3__ = __iter_4__;
      int __temp_4__;
      __temp_4__ = __iter_5__+(-1);
      float __temp_5__;
      __temp_5__ = (12 * __tilevar_2__[__temp_4__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_3__+(0-(__iter_1__+0)))]);
      float __temp_6__;
      __temp_6__ = (__temp_2__ + __temp_5__);
      int __temp_7__;
      __temp_7__ = __iter_4__;
      int __temp_8__;
      __temp_8__ = __iter_5__;
      float __temp_9__;
      __temp_9__ = (15 * __tilevar_2__[__temp_8__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_7__+(0-(__iter_1__+0)))]);
      float __temp_10__;
      __temp_10__ = (__temp_6__ + __temp_9__);
      int __temp_11__;
      __temp_11__ = __iter_4__;
      int __temp_12__;
      __temp_12__ = __iter_5__+(1);
      float __temp_13__;
      __temp_13__ = (12 * __tilevar_2__[__temp_12__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_11__+(0-(__iter_1__+0)))]);
      float __temp_14__;
      __temp_14__ = (__temp_10__ + __temp_13__);
      int __temp_15__;
      __temp_15__ = __iter_4__+(1);
      int __temp_16__;
      __temp_16__ = __iter_5__;
      float __temp_17__;
      __temp_17__ = (5 * __tilevar_2__[__temp_16__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_15__+(0-(__iter_1__+0)))]);
      float __temp_18__;
      __temp_18__ = (__temp_14__ + __temp_17__);
      float __temp_19__;
      __temp_19__ = (__temp_18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+1)))] = __temp_19__;
    }
  }
  int __iter_6__;
  __iter_6__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      if (__iter_6__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_6__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))-2) || __iter_7__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_7__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-2)) {
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+1)))];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      int __temp_30__;
      __temp_30__ = __iter_10__+(-1);
      int __temp_31__;
      __temp_31__ = __iter_11__;
      float __temp_32__;
      __temp_32__ = (5 * __tilevar_3__[__temp_31__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_30__+(0-(__iter_1__+1)))]);
      int __temp_33__;
      __temp_33__ = __iter_10__;
      int __temp_34__;
      __temp_34__ = __iter_11__+(-1);
      float __temp_35__;
      __temp_35__ = (12 * __tilevar_3__[__temp_34__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_33__+(0-(__iter_1__+1)))]);
      float __temp_36__;
      __temp_36__ = (__temp_32__ + __temp_35__);
      int __temp_37__;
      __temp_37__ = __iter_10__;
      int __temp_38__;
      __temp_38__ = __iter_11__;
      float __temp_39__;
      __temp_39__ = (15 * __tilevar_3__[__temp_38__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_37__+(0-(__iter_1__+1)))]);
      float __temp_40__;
      __temp_40__ = (__temp_36__ + __temp_39__);
      int __temp_41__;
      __temp_41__ = __iter_10__;
      int __temp_42__;
      __temp_42__ = __iter_11__+(1);
      float __temp_43__;
      __temp_43__ = (12 * __tilevar_3__[__temp_42__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_41__+(0-(__iter_1__+1)))]);
      float __temp_44__;
      __temp_44__ = (__temp_40__ + __temp_43__);
      int __temp_45__;
      __temp_45__ = __iter_10__+(1);
      int __temp_46__;
      __temp_46__ = __iter_11__;
      float __temp_47__;
      __temp_47__ = (5 * __tilevar_3__[__temp_46__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_45__+(0-(__iter_1__+1)))]);
      float __temp_48__;
      __temp_48__ = (__temp_44__ + __temp_47__);
      float __temp_49__;
      __temp_49__ = (__temp_48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+2)))] = __temp_49__;
    }
  }
  int __iter_12__;
  __iter_12__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      if (__iter_12__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_12__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))-2) || __iter_13__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_13__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-2)) {
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+2)))];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
  if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      float __temp_60__;
      __temp_60__ = (5 * __tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+2)))]);
      float __temp_61__;
      __temp_61__ = (12 * __tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_62__;
      __temp_62__ = (__temp_60__ + __temp_61__);
      float __temp_63__;
      __temp_63__ = (15 * __tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_64__;
      __temp_64__ = (__temp_62__ + __temp_63__);
      float __temp_65__;
      __temp_65__ = (12 * __tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2)))]);
      float __temp_66__;
      __temp_66__ = (__temp_64__ + __temp_65__);
      float __temp_67__;
      __temp_67__ = (5 * __tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+2)))]);
      float __temp_68__;
      __temp_68__ = (__temp_66__ + __temp_67__);
      float __temp_69__;
      __temp_69__ = (__temp_68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+3)))] = __temp_69__;
    }
  }
  int __iter_18__;
  __iter_18__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
  if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      if (__iter_18__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))-2) || __iter_19__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-2)) {
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+3)))];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__+4),1) + (int)(threadIdx.y) ; 
  if( __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
      float __temp_80__;
      __temp_80__ = (5 * __tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+3)))]);
      float __temp_81__;
      __temp_81__ = (12 * __tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_82__;
      __temp_82__ = (__temp_80__ + __temp_81__);
      float __temp_83__;
      __temp_83__ = (15 * __tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_84__;
      __temp_84__ = (__temp_82__ + __temp_83__);
      float __temp_85__;
      __temp_85__ = (12 * __tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+3)))]);
      float __temp_86__;
      __temp_86__ = (__temp_84__ + __temp_85__);
      float __temp_87__;
      __temp_87__ = (5 * __tilevar_5__[__iter_23__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+3)))]);
      float __temp_88__;
      __temp_88__ = (__temp_86__ + __temp_87__);
      float __temp_89__;
      __temp_89__ = (__temp_88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_89__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X) + GAPX) + (int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y) + GAPY);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(0-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
    }
  }
  __syncthreads();

  int __iter_4__;
  __iter_4__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
      int __temp_0__;
      __temp_0__ = __iter_4__+(-1);
      int __temp_1__;
      __temp_1__ = __iter_5__;
      float __temp_2__;
      __temp_2__ = (5 * __tilevar_2__[__temp_1__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_0__+(0-(__iter_1__+0)))]);
      int __temp_3__;
      __temp_3__ = __iter_4__;
      int __temp_4__;
      __temp_4__ = __iter_5__+(-1);
      float __temp_5__;
      __temp_5__ = (12 * __tilevar_2__[__temp_4__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_3__+(0-(__iter_1__+0)))]);
      float __temp_6__;
      __temp_6__ = (__temp_2__ + __temp_5__);
      int __temp_7__;
      __temp_7__ = __iter_4__;
      int __temp_8__;
      __temp_8__ = __iter_5__;
      float __temp_9__;
      __temp_9__ = (15 * __tilevar_2__[__temp_8__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_7__+(0-(__iter_1__+0)))]);
      float __temp_10__;
      __temp_10__ = (__temp_6__ + __temp_9__);
      int __temp_11__;
      __temp_11__ = __iter_4__;
      int __temp_12__;
      __temp_12__ = __iter_5__+(1);
      float __temp_13__;
      __temp_13__ = (12 * __tilevar_2__[__temp_12__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_11__+(0-(__iter_1__+0)))]);
      float __temp_14__;
      __temp_14__ = (__temp_10__ + __temp_13__);
      int __temp_15__;
      __temp_15__ = __iter_4__+(1);
      int __temp_16__;
      __temp_16__ = __iter_5__;
      float __temp_17__;
      __temp_17__ = (5 * __tilevar_2__[__temp_16__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_15__+(0-(__iter_1__+0)))]);
      float __temp_18__;
      __temp_18__ = (__temp_14__ + __temp_17__);
      float __temp_19__;
      __temp_19__ = (__temp_18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-(__iter_1__+0)))] = __temp_19__;
    }
  }
  int __iter_6__;
  __iter_6__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
      if (__iter_6__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_6__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))-2)) {
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+0)))];
      }
    }
  }
  __iter_6__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
      if (__iter_7__ < FORMA_MAX((__iter_0__-1),1) || __iter_7__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2))) {
       __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(0-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
      int __temp_30__;
      __temp_30__ = __iter_10__+(-1);
      int __temp_31__;
      __temp_31__ = __iter_11__;
      float __temp_32__;
      __temp_32__ = (5 * __tilevar_3__[__temp_31__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_30__+(0-(__iter_1__+0)))]);
      int __temp_33__;
      __temp_33__ = __iter_10__;
      int __temp_34__;
      __temp_34__ = __iter_11__+(-1);
      float __temp_35__;
      __temp_35__ = (12 * __tilevar_3__[__temp_34__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_33__+(0-(__iter_1__+0)))]);
      float __temp_36__;
      __temp_36__ = (__temp_32__ + __temp_35__);
      int __temp_37__;
      __temp_37__ = __iter_10__;
      int __temp_38__;
      __temp_38__ = __iter_11__;
      float __temp_39__;
      __temp_39__ = (15 * __tilevar_3__[__temp_38__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_37__+(0-(__iter_1__+0)))]);
      float __temp_40__;
      __temp_40__ = (__temp_36__ + __temp_39__);
      int __temp_41__;
      __temp_41__ = __iter_10__;
      int __temp_42__;
      __temp_42__ = __iter_11__+(1);
      float __temp_43__;
      __temp_43__ = (12 * __tilevar_3__[__temp_42__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_41__+(0-(__iter_1__+0)))]);
      float __temp_44__;
      __temp_44__ = (__temp_40__ + __temp_43__);
      int __temp_45__;
      __temp_45__ = __iter_10__+(1);
      int __temp_46__;
      __temp_46__ = __iter_11__;
      float __temp_47__;
      __temp_47__ = (5 * __tilevar_3__[__temp_46__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_45__+(0-(__iter_1__+0)))]);
      float __temp_48__;
      __temp_48__ = (__temp_44__ + __temp_47__);
      float __temp_49__;
      __temp_49__ = (__temp_48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+0)))] = __temp_49__;
    }
  }
  int __iter_12__;
  __iter_12__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
      if (__iter_12__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_12__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2))-2)) {
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+0)))];
      }
    }
  }
  __iter_12__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-2)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
      if (__iter_13__ < FORMA_MAX((__iter_0__-2),1) || __iter_13__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ) {
        __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(0-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
  if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
      float __temp_60__;
      __temp_60__ = (5 * __tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_61__;
      __temp_61__ = (12 * __tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))]);
      float __temp_62__;
      __temp_62__ = (__temp_60__ + __temp_61__);
      float __temp_63__;
      __temp_63__ = (15 * __tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))]);
      float __temp_64__;
      __temp_64__ = (__temp_62__ + __temp_63__);
      float __temp_65__;
      __temp_65__ = (12 * __tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))]);
      float __temp_66__;
      __temp_66__ = (__temp_64__ + __temp_65__);
      float __temp_67__;
      __temp_67__ = (5 * __tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+0)))]);
      float __temp_68__;
      __temp_68__ = (__temp_66__ + __temp_67__);
      float __temp_69__;
      __temp_69__ = (__temp_68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0)))] = __temp_69__;
    }
  }
  int __iter_18__;
  __iter_18__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
  if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
      if (__iter_18__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2))-2)) {
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+0)))];
      }
    }
  }
  __iter_18__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
  if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-2)) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(M-2)) ){
      if (__iter_19__ < FORMA_MAX((__iter_0__-3),1) || __iter_19__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2))) {
	__tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(0-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__+4),1) + (int)(threadIdx.y) ; 
  if( __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
      float __temp_80__;
      __temp_80__ = (5 * __tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(0-(__iter_1__+0)))]);
      float __temp_81__;
      __temp_81__ = (12 * __tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+0)))]);
      float __temp_82__;
      __temp_82__ = (__temp_80__ + __temp_81__);
      float __temp_83__;
      __temp_83__ = (15 * __tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+0)))]);
      float __temp_84__;
      __temp_84__ = (__temp_82__ + __temp_83__);
      float __temp_85__;
      __temp_85__ = (12 * __tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(0-(__iter_1__+0)))]);
      float __temp_86__;
      __temp_86__ = (__temp_84__ + __temp_85__);
      float __temp_87__;
      __temp_87__ = (5 * __tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(0-(__iter_1__+0)))]);
      float __temp_88__;
      __temp_88__ = (__temp_86__ + __temp_87__);
      float __temp_89__;
      __temp_89__ = (__temp_88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_89__;
    }
  }
}

__global__ void __kernel___forma_kernel__2__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X) + GAPX);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y) + GAPY) + (int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__-2,0) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      int __temp_0__;
      __temp_0__ = __iter_4__+(-1);
      int __temp_1__;
      __temp_1__ = __iter_5__;
      float __temp_2__;
      __temp_2__ = (5 * __tilevar_2__[__temp_1__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_0__+(EXTENT-(__iter_1__+0)))]);
      int __temp_3__;
      __temp_3__ = __iter_4__;
      int __temp_4__;
      __temp_4__ = __iter_5__+(-1);
      float __temp_5__;
      __temp_5__ = (12 * __tilevar_2__[__temp_4__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_3__+(EXTENT-(__iter_1__+0)))]);
      float __temp_6__;
      __temp_6__ = (__temp_2__ + __temp_5__);
      int __temp_7__;
      __temp_7__ = __iter_4__;
      int __temp_8__;
      __temp_8__ = __iter_5__;
      float __temp_9__;
      __temp_9__ = (15 * __tilevar_2__[__temp_8__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_7__+(EXTENT-(__iter_1__+0)))]);
      float __temp_10__;
      __temp_10__ = (__temp_6__ + __temp_9__);
      int __temp_11__;
      __temp_11__ = __iter_4__;
      int __temp_12__;
      __temp_12__ = __iter_5__+(1);
      float __temp_13__;
      __temp_13__ = (12 * __tilevar_2__[__temp_12__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_11__+(EXTENT-(__iter_1__+0)))]);
      float __temp_14__;
      __temp_14__ = (__temp_10__ + __temp_13__);
      int __temp_15__;
      __temp_15__ = __iter_4__+(1);
      int __temp_16__;
      __temp_16__ = __iter_5__;
      float __temp_17__;
      __temp_17__ = (5 * __tilevar_2__[__temp_16__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_15__+(EXTENT-(__iter_1__+0)))]);
      float __temp_18__;
      __temp_18__ = (__temp_14__ + __temp_17__);
      float __temp_19__;
      __temp_19__ = (__temp_18__ / 118);
      __tilevar_3__[__iter_5__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))] = __temp_19__;
    }
  }
  int __iter_6__;
  __iter_6__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      if (__iter_7__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_7__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-2)) {
        __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)] = __tilevar_3__[__iter_7__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))];
      }
    }
  }
  __iter_6__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
      if (__iter_6__ < FORMA_MAX((__iter_1__-1),1) || __iter_6__ > FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2))) {
       __tilevar_3__[__iter_7__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      int __temp_30__;
      __temp_30__ = __iter_10__+(-1);
      int __temp_31__;
      __temp_31__ = __iter_11__;
      float __temp_32__;
      __temp_32__ = (5 * __tilevar_3__[__temp_31__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_30__+(EXTENT-(__iter_1__+0)))]);
      int __temp_33__;
      __temp_33__ = __iter_10__;
      int __temp_34__;
      __temp_34__ = __iter_11__+(-1);
      float __temp_35__;
      __temp_35__ = (12 * __tilevar_3__[__temp_34__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_33__+(EXTENT-(__iter_1__+0)))]);
      float __temp_36__;
      __temp_36__ = (__temp_32__ + __temp_35__);
      int __temp_37__;
      __temp_37__ = __iter_10__;
      int __temp_38__;
      __temp_38__ = __iter_11__;
      float __temp_39__;
      __temp_39__ = (15 * __tilevar_3__[__temp_38__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_37__+(EXTENT-(__iter_1__+0)))]);
      float __temp_40__;
      __temp_40__ = (__temp_36__ + __temp_39__);
      int __temp_41__;
      __temp_41__ = __iter_10__;
      int __temp_42__;
      __temp_42__ = __iter_11__+(1);
      float __temp_43__;
      __temp_43__ = (12 * __tilevar_3__[__temp_42__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_41__+(EXTENT-(__iter_1__+0)))]);
      float __temp_44__;
      __temp_44__ = (__temp_40__ + __temp_43__);
      int __temp_45__;
      __temp_45__ = __iter_10__+(1);
      int __temp_46__;
      __temp_46__ = __iter_11__;
      float __temp_47__;
      __temp_47__ = (5 * __tilevar_3__[__temp_46__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_45__+(EXTENT-(__iter_1__+0)))]);
      float __temp_48__;
      __temp_48__ = (__temp_44__ + __temp_47__);
      float __temp_49__;
      __temp_49__ = (__temp_48__ / 118);
      __tilevar_4__[__iter_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))] = __temp_49__;
    }
  }
  int __iter_12__;
  __iter_12__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      if (__iter_13__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_13__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-2)) {
        __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)] = __tilevar_4__[__iter_13__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))];
      }
    }
  }
  __iter_12__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
      if (__iter_12__ < FORMA_MAX((__iter_1__-2),1) || __iter_12__ > FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2))) {
        __tilevar_4__[__iter_13__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
  if( __iter_16__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      float __temp_60__;
      __temp_60__ = (5 * __tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_61__;
      __temp_61__ = (12 * __tilevar_4__[__iter_17__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_62__;
      __temp_62__ = (__temp_60__ + __temp_61__);
      float __temp_63__;
      __temp_63__ = (15 * __tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_64__;
      __temp_64__ = (__temp_62__ + __temp_63__);
      float __temp_65__;
      __temp_65__ = (12 * __tilevar_4__[__iter_17__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_66__;
      __temp_66__ = (__temp_64__ + __temp_65__);
      float __temp_67__;
      __temp_67__ = (5 * __tilevar_4__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_68__;
      __temp_68__ = (__temp_66__ + __temp_67__);
      float __temp_69__;
      __temp_69__ = (__temp_68__ / 118);
      __tilevar_5__[__iter_17__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))] = __temp_69__;
    }
  }
  int __iter_18__;
  __iter_18__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
  if( __iter_18__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2)) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
      if (__iter_19__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-2)) {
        __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)] = __tilevar_5__[__iter_19__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))];
      }
    }
  }
  __iter_18__ = FORMA_MAX((__iter_1__-5),1) + (int)(threadIdx.y) ; 
  if( __iter_18__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(N-2)) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
     if (__iter_18__ < FORMA_MAX((__iter_1__-3),1) || __iter_18__ > FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2))) {
	__tilevar_5__[__iter_19__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
  if( __iter_22__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
      float __temp_80__;
      __temp_80__ = (5 * __tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_81__;
      __temp_81__ = (12 * __tilevar_5__[__iter_23__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_82__;
      __temp_82__ = (__temp_80__ + __temp_81__);
      float __temp_83__;
      __temp_83__ = (15 * __tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_84__;
      __temp_84__ = (__temp_82__ + __temp_83__);
      float __temp_85__;
      __temp_85__ = (12 * __tilevar_5__[__iter_23__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_86__;
      __temp_86__ = (__temp_84__ + __temp_85__);
      float __temp_87__;
      __temp_87__ = (5 * __tilevar_5__[__iter_23__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_88__;
      __temp_88__ = (__temp_86__ + __temp_87__);
      float __temp_89__;
      __temp_89__ = (__temp_88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_89__;
    }
  }
}

__global__ void __kernel___forma_kernel__3__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X) + GAPX) + (int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y) + GAPY) + (int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = FORMA_MAX(__iter_1__-2,0) + (int)(threadIdx.y) ; 
  if( __iter_2__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-1)) ){
    int __iter_3__;
    __iter_3__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
    if( __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1)) ){
      __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_2__+(EXTENT-__iter_1__))] = input[__iter_3__+(M-0)*(__iter_2__)];
    }
  }
  __syncthreads();
  int __iter_4__;
  __iter_4__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
  if( __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2)) ){
    int __iter_5__;
    __iter_5__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
      int __temp_0__;
      __temp_0__ = __iter_4__+(-1);
      int __temp_1__;
      __temp_1__ = __iter_5__;
      float __temp_2__;
      __temp_2__ = (5 * __tilevar_2__[__temp_1__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_0__+(EXTENT-(__iter_1__+0)))]);
      int __temp_3__;
      __temp_3__ = __iter_4__;
      int __temp_4__;
      __temp_4__ = __iter_5__+(-1);
      float __temp_5__;
      __temp_5__ = (12 * __tilevar_2__[__temp_4__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_3__+(EXTENT-(__iter_1__+0)))]);
      float __temp_6__;
      __temp_6__ = (__temp_2__ + __temp_5__);
      int __temp_7__;
      __temp_7__ = __iter_4__;
      int __temp_8__;
      __temp_8__ = __iter_5__;
      float __temp_9__;
      __temp_9__ = (15 * __tilevar_2__[__temp_8__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_7__+(EXTENT-(__iter_1__+0)))]);
      float __temp_10__;
      __temp_10__ = (__temp_6__ + __temp_9__);
      int __temp_11__;
      __temp_11__ = __iter_4__;
      int __temp_12__;
      __temp_12__ = __iter_5__+(1);
      float __temp_13__;
      __temp_13__ = (12 * __tilevar_2__[__temp_12__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_11__+(EXTENT-(__iter_1__+0)))]);
      float __temp_14__;
      __temp_14__ = (__temp_10__ + __temp_13__);
      int __temp_15__;
      __temp_15__ = __iter_4__+(1);
      int __temp_16__;
      __temp_16__ = __iter_5__;
      float __temp_17__;
      __temp_17__ = (5 * __tilevar_2__[__temp_16__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_15__+(EXTENT-(__iter_1__+0)))]);
      float __temp_18__;
      __temp_18__ = (__temp_14__ + __temp_17__);
      float __temp_19__;
      __temp_19__ = (__temp_18__ / 118);
      __tilevar_3__[__iter_5__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(EXTENT-(__iter_1__+0)))] = __temp_19__;
    }
  }
  int __iter_6__;
  __iter_6__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
  if( __iter_6__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
      if (__iter_6__ < FORMA_MAX((__iter_1__-1),1) || __iter_6__ > FORMA_MIN(((__iter_1__+GAPY+1)-1),(N-2)) || __iter_7__ < FORMA_MAX((__iter_0__-1),1) || __iter_7__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2))) {
       __tilevar_3__[__iter_7__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_6__+(EXTENT-(__iter_1__+0)))] = __copy_arr_0__[__iter_7__+(M-0)*(__iter_6__)];
      }
    }
  }
  __syncthreads();
  int __iter_10__;
  __iter_10__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
  if( __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2)) ){
    int __iter_11__;
    __iter_11__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
    if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
      int __temp_30__;
      __temp_30__ = __iter_10__+(-1);
      int __temp_31__;
      __temp_31__ = __iter_11__;
      float __temp_32__;
      __temp_32__ = (5 * __tilevar_3__[__temp_31__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_30__+(EXTENT-(__iter_1__+0)))]);
      int __temp_33__;
      __temp_33__ = __iter_10__;
      int __temp_34__;
      __temp_34__ = __iter_11__+(-1);
      float __temp_35__;
      __temp_35__ = (12 * __tilevar_3__[__temp_34__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_33__+(EXTENT-(__iter_1__+0)))]);
      float __temp_36__;
      __temp_36__ = (__temp_32__ + __temp_35__);
      int __temp_37__;
      __temp_37__ = __iter_10__;
      int __temp_38__;
      __temp_38__ = __iter_11__;
      float __temp_39__;
      __temp_39__ = (15 * __tilevar_3__[__temp_38__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_37__+(EXTENT-(__iter_1__+0)))]);
      float __temp_40__;
      __temp_40__ = (__temp_36__ + __temp_39__);
      int __temp_41__;
      __temp_41__ = __iter_10__;
      int __temp_42__;
      __temp_42__ = __iter_11__+(1);
      float __temp_43__;
      __temp_43__ = (12 * __tilevar_3__[__temp_42__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_41__+(EXTENT-(__iter_1__+0)))]);
      float __temp_44__;
      __temp_44__ = (__temp_40__ + __temp_43__);
      int __temp_45__;
      __temp_45__ = __iter_10__+(1);
      int __temp_46__;
      __temp_46__ = __iter_11__;
      float __temp_47__;
      __temp_47__ = (5 * __tilevar_3__[__temp_46__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_45__+(EXTENT-(__iter_1__+0)))]);
      float __temp_48__;
      __temp_48__ = (__temp_44__ + __temp_47__);
      float __temp_49__;
      __temp_49__ = (__temp_48__ / 118);
      __tilevar_4__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(EXTENT-(__iter_1__+0)))] = __temp_49__;
    }
  }
  int __iter_12__;
  __iter_12__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
  if( __iter_12__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
    if( __iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
      if (__iter_12__ < FORMA_MAX((__iter_1__-2),1) || __iter_12__ > FORMA_MIN(((__iter_1__+GAPY+2)-1),(N-2)) || __iter_13__ < FORMA_MAX((__iter_0__-2),1) || __iter_13__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2))) {
        __tilevar_4__[__iter_13__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_12__+(EXTENT-(__iter_1__+0)))] = __copy_arr_1__[__iter_13__+(M-0)*(__iter_12__)];
      }
    }
  }
  __syncthreads();
  int __iter_16__;
  __iter_16__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
  if( __iter_16__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2)) ){
    int __iter_17__;
    __iter_17__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
    if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
      float __temp_60__;
      __temp_60__ = (5 * __tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_61__;
      __temp_61__ = (12 * __tilevar_4__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_62__;
      __temp_62__ = (__temp_60__ + __temp_61__);
      float __temp_63__;
      __temp_63__ = (15 * __tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_64__;
      __temp_64__ = (__temp_62__ + __temp_63__);
      float __temp_65__;
      __temp_65__ = (12 * __tilevar_4__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))]);
      float __temp_66__;
      __temp_66__ = (__temp_64__ + __temp_65__);
      float __temp_67__;
      __temp_67__ = (5 * __tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_68__;
      __temp_68__ = (__temp_66__ + __temp_67__);
      float __temp_69__;
      __temp_69__ = (__temp_68__ / 118);
      __tilevar_5__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(EXTENT-(__iter_1__+0)))] = __temp_69__;
    }
  }
  int __iter_18__;
  __iter_18__ = FORMA_MAX((__iter_1__-5),1) + (int)(threadIdx.y) ; 
  if( __iter_18__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(N-2)) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(M-2)) ){
     if (__iter_18__ < FORMA_MAX((__iter_1__-3),1) || __iter_18__ > FORMA_MIN(((__iter_1__+GAPY+3)-1),(N-2)) || __iter_19__ < FORMA_MAX((__iter_0__-3),1) || __iter_19__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2))) {
	__tilevar_5__[__iter_19__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_18__+(EXTENT-(__iter_1__+0)))] = __copy_arr_2__[__iter_19__+(M-0)*(__iter_18__)];
      }
    }
  }
  __syncthreads();
  int __iter_22__;
  __iter_22__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
  if( __iter_22__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(N-2)) ){
    int __iter_23__;
    __iter_23__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
    if( __iter_23__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
      float __temp_80__;
      __temp_80__ = (5 * __tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(-1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_81__;
      __temp_81__ = (12 * __tilevar_5__[__iter_23__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_82__;
      __temp_82__ = (__temp_80__ + __temp_81__);
      float __temp_83__;
      __temp_83__ = (15 * __tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_84__;
      __temp_84__ = (__temp_82__ + __temp_83__);
      float __temp_85__;
      __temp_85__ = (12 * __tilevar_5__[__iter_23__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(EXTENT-(__iter_1__+0)))]);
      float __temp_86__;
      __temp_86__ = (__temp_84__ + __temp_85__);
      float __temp_87__;
      __temp_87__ = (5 * __tilevar_5__[__iter_23__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_22__+(1)+(EXTENT-(__iter_1__+0)))]);
      float __temp_88__;
      __temp_88__ = (__temp_86__ + __temp_87__);
      float __temp_89__;
      __temp_89__ = (__temp_88__ / 118);
      __var_1__[__iter_23__+(M-0)*(__iter_22__)] = __temp_89__;
    }
  }
}


/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,9))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y+GAPY);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  dim3 __blockConfig___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__1__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");
  dim3 __blockConfig___kernel___forma_kernel__2__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y);
  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__2__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");
  dim3 __blockConfig___kernel___forma_kernel__3__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y);
  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__3__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define GAPX (22) 
#define EXTENT (5)
#define mod(x,y) ( (x) & (y-1))

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_5__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X);
  float* __tilevar_4__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X);
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X);
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X);
  int rowy = FORMA_BLOCKDIM_Y + 2;
 
  int threadIdx_y = mod((int)threadIdx.y,2);

  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X + GAPX);
  for (int __iter_1__ = 0; __iter_1__ <= N-1; __iter_1__ += FORMA_BLOCKDIM_Y) {
    int __iter_2__ = FORMA_MAX(__iter_1__,0) + 3*(int)(threadIdx.y) ; 
    if(__iter_2__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1))) {
      int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1)) ){
        __tilevar_2__[__iter_3__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_2__,rowy)] = input[__iter_3__+M*__iter_2__];
        __tilevar_2__[__iter_3__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_2__+1,rowy)] = input[__iter_3__+M*(__iter_2__+1)];
        __tilevar_2__[__iter_3__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_2__+2,rowy)] = input[__iter_3__+M*(__iter_2__+2)];
      }
    }
    __syncthreads();
    int __iter_4__ = FORMA_MAX((__iter_1__-1),1) + 3*(int)(threadIdx.y) ;
    if(__iter_4__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
	// __iter_4__
        float __temp_a2__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_4__-1),rowy)]);
        float __temp_a5__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a6__ = (5 * __temp_a2__ + 12 * __temp_a5__);
        float __temp_a9__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a10__ = (__temp_a6__ + 15 * __temp_a9__);
        float __temp_a13__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a14__ = (__temp_a10__ + 12 * __temp_a13__);
        float __temp_a17__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_4__+1),rowy)]);
        float __temp_a18__ = (__temp_a14__ + 5 * __temp_a17__);
        float __temp_a19__ = (__temp_a18__ / 118);
        __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)] = __temp_a19__;
	// __iter_4__ + 1
        float __temp_b2__ = __temp_a9__;
        float __temp_b5__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)]);
        float __temp_b6__ = (5 * __temp_b2__ + 12 * __temp_b5__);
        float __temp_b9__ = __temp_a17__;
        float __temp_b10__ = (__temp_b6__ + 15 * __temp_b9__);
        float __temp_b13__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)]);
        float __temp_b14__ = (__temp_b10__ + 12 * __temp_b13__);
        float __temp_b17__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_4__+2),rowy)]);
        float __temp_b18__ = (__temp_b14__ + 5 * __temp_b17__);
        float __temp_b19__ = (__temp_b18__ / 118);
        __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)] = __temp_b19__;
	// __iter_4__ + 2
        float __temp_c2__ = __temp_a17__;
        float __temp_c5__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+2,rowy)]);
        float __temp_c6__ = (5 * __temp_c2__ + 12 * __temp_c5__);
        float __temp_c9__ = __temp_b17__;
        float __temp_c10__ = (__temp_c6__ + 15 * __temp_c9__);
        float __temp_c13__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+2,rowy)]);
        float __temp_c14__ = (__temp_c10__ + 12 * __temp_c13__);
        float __temp_c17__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_4__+3),rowy)]);
        float __temp_c18__ = (__temp_c14__ + 5 * __temp_c17__);
        float __temp_c19__ = (__temp_c18__ / 118);
        __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__+2,rowy)] = __temp_c19__;
      }
    }
    else if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_5__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
	// __iter_4__
        float __temp_a2__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_4__-1),rowy)]);
        float __temp_a5__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a6__ = (5 * __temp_a2__ + 12 * __temp_a5__);
        float __temp_a9__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a10__ = (__temp_a6__ + 15 * __temp_a9__);
        float __temp_a13__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a14__ = (__temp_a10__ + 12 * __temp_a13__);
        float __temp_a17__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_4__+1),rowy)]);
        float __temp_a18__ = (__temp_a14__ + 5 * __temp_a17__);
        float __temp_a19__ = (__temp_a18__ / 118);
        __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)] = __temp_a19__;
      }
    }
    __syncthreads ();
    int __iter_6__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.x) ; 
    if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_7__ = FORMA_MAX((__iter_0__+1),1) + threadIdx_y;
      if( __iter_7__ < (FORMA_MAX((__iter_0__+1),1)+2)) {
        __copy_arr_0__[__iter_6__+(M)*(__iter_7__)] = __tilevar_3__[__iter_7__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_6__,rowy)];
      }
    }
    if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_7__ = FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))-1 + threadIdx_y;
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
        __copy_arr_0__[__iter_6__+(M)*(__iter_7__)] = __tilevar_3__[__iter_7__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_6__,rowy)];
      }
    }
    __syncthreads();
    int __iter_10__ = FORMA_MAX((__iter_1__-2),2) + 3*(int)(threadIdx.y) ;
    if(__iter_10__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3))) {
      int __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
	// __iter_10__
        float __temp_a32__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_10__-1),rowy)]);
        float __temp_a35__ = (__tilevar_3__[__iter_11__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a36__ = (5 * __temp_a32__ + 12 * __temp_a35__);
        float __temp_a39__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a40__ = (__temp_a36__ + 15 * __temp_a39__);
        float __temp_a43__ = (__tilevar_3__[__iter_11__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a44__ = (__temp_a40__ + 12 * __temp_a43__);
        float __temp_a47__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_10__+1),rowy)]);
        float __temp_a48__ = (__temp_a44__ + 5 * __temp_a47__);
        float __temp_a49__ = (__temp_a48__ / 118);
        __tilevar_4__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)] = __temp_a49__;
	// __iter_10__ + 1
        float __temp_b32__ = __temp_a39__;
        float __temp_b35__ = (__tilevar_3__[__iter_11__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_b36__ = (5 * __temp_b32__ + 12 * __temp_b35__);
        float __temp_b39__ = __temp_a47__;
        float __temp_b40__ = (__temp_b36__ + 15 * __temp_b39__);
        float __temp_b43__ = (__tilevar_3__[__iter_11__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_b44__ = (__temp_b40__ + 12 * __temp_b43__);
        float __temp_b47__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_10__+2),rowy)]);
        float __temp_b48__ = (__temp_b44__ + 5 * __temp_b47__);
        float __temp_b49__ = (__temp_b48__ / 118);
        __tilevar_4__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)] = __temp_b49__;
	// __iter_10__ + 2
        float __temp_c32__ = __temp_a47__;
        float __temp_c35__ = (__tilevar_3__[__iter_11__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+2,rowy)]);
        float __temp_c36__ = (5 * __temp_c32__ + 12 * __temp_c35__);
        float __temp_c39__ = __temp_b47__;
        float __temp_c40__ = (__temp_c36__ + 15 * __temp_c39__);
        float __temp_c43__ = (__tilevar_3__[__iter_11__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+2,rowy)]);
        float __temp_c44__ = (__temp_c40__ + 12 * __temp_c43__);
        float __temp_c47__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_10__+3),rowy)]);
        float __temp_c48__ = (__temp_c44__ + 5 * __temp_c47__);
        float __temp_c49__ = (__temp_c48__ / 118);
        __tilevar_4__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+2,rowy)] = __temp_c49__;
      }
    }
    else if(__iter_10__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3))) {
      int __iter_11__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
	// __iter_10__
        float __temp_a32__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_10__-1),rowy)]);
        float __temp_a35__ = (__tilevar_3__[__iter_11__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a36__ = (5 * __temp_a32__ + 12 * __temp_a35__);
        float __temp_a39__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a40__ = (__temp_a36__ + 15 * __temp_a39__);
        float __temp_a43__ = (__tilevar_3__[__iter_11__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a44__ = (__temp_a40__ + 12 * __temp_a43__);
        float __temp_a47__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_10__+1),rowy)]);
        float __temp_a48__ = (__temp_a44__ + 5 * __temp_a47__);
        float __temp_a49__ = (__temp_a48__ / 118);
        __tilevar_4__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)] = __temp_a49__;
	// __iter_10__ + 1
        float __temp_b32__ = __temp_a39__;
        float __temp_b35__ = (__tilevar_3__[__iter_11__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_b36__ = (5 * __temp_b32__ + 12 * __temp_b35__);
        float __temp_b39__ = __temp_a47__;
        float __temp_b40__ = (__temp_b36__ + 15 * __temp_b39__);
        float __temp_b43__ = (__tilevar_3__[__iter_11__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_b44__ = (__temp_b40__ + 12 * __temp_b43__);
        float __temp_b47__ = (__tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_10__+2),rowy)]);
        float __temp_b48__ = (__temp_b44__ + 5 * __temp_b47__);
        float __temp_b49__ = (__temp_b48__ / 118);
        __tilevar_4__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)] = __temp_b49__;
      }
    }
    __syncthreads ();
    int __iter_12__ = FORMA_MAX((__iter_1__-2),2) + (int)(threadIdx.x) ; 
    if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3)) ){
      int __iter_13__ = FORMA_MAX((__iter_0__+2),1) + threadIdx_y;
      if( __iter_13__ < (FORMA_MAX((__iter_0__+2),1)+2)) {
        __copy_arr_1__[__iter_12__+(M)*(__iter_13__)] = __tilevar_4__[__iter_13__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_12__,rowy)];
      }
    }
    if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3)) ){
      int __iter_13__ = FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))-1 + threadIdx_y;
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
        __copy_arr_1__[__iter_12__+(M)*(__iter_13__)] = __tilevar_4__[__iter_13__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_12__,rowy)];
      }
    }
    __syncthreads();
    int __iter_16__ = FORMA_MAX((__iter_1__-3),3) + 3*(int)(threadIdx.y) ; 
    if(__iter_16__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-4))) {
      int __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
	// __iter_16__
        float __temp_a60__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_16__-1),rowy)]);
        float __temp_a61__ = (__tilevar_4__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a62__ = (5 * __temp_a60__ + 12 * __temp_a61__);
        float __temp_a63__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a64__ = (__temp_a62__ + 15 * __temp_a63__);
        float __temp_a65__ = (__tilevar_4__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a66__ = (__temp_a64__ + 12 * __temp_a65__);
        float __temp_a67__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_16__+1),rowy)]);
        float __temp_a68__ = (__temp_a66__ + 5 * __temp_a67__);
        float __temp_a69__ = (__temp_a68__ / 118);
        __tilevar_5__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)] = __temp_a69__;
	// __iter_16__ + 1
        float __temp_b60__ = __temp_a63__;
        float __temp_b61__ = (__tilevar_4__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_b62__ = (5 * __temp_b60__ + 12 * __temp_b61__);
        float __temp_b63__ = __temp_a67__;
        float __temp_b64__ = (__temp_b62__ + 15 * __temp_b63__);
        float __temp_b65__ = (__tilevar_4__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_b66__ = (__temp_b64__ + 12 * __temp_b65__);
        float __temp_b67__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_16__+2),rowy)]);
        float __temp_b68__ = (__temp_b66__ + 5 * __temp_b67__);
        float __temp_b69__ = (__temp_b68__ / 118);
        __tilevar_5__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)] = __temp_b69__;
	// __iter_16__ + 2
        float __temp_c60__ = __temp_a67__;
        float __temp_c61__ = (__tilevar_4__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+2,rowy)]);
        float __temp_c62__ = (5 * __temp_c60__ + 12 * __temp_c61__);
        float __temp_c63__ = __temp_b67__;
        float __temp_c64__ = (__temp_c62__ + 15 * __temp_c63__);
        float __temp_c65__ = (__tilevar_4__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+2,rowy)]);
        float __temp_c66__ = (__temp_c64__ + 12 * __temp_c65__);
        float __temp_c67__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_16__+3),rowy)]);
        float __temp_c68__ = (__temp_c66__ + 5 * __temp_c67__);
        float __temp_c69__ = (__temp_c68__ / 118);
        __tilevar_5__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+2,rowy)] = __temp_c69__;
      }
    }
    else if(__iter_16__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-4))) {
      int __iter_17__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
	// __iter_16__
        float __temp_a60__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_16__-1),rowy)]);
        float __temp_a61__ = (__tilevar_4__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a62__ = (5 * __temp_a60__ + 12 * __temp_a61__);
        float __temp_a63__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a64__ = (__temp_a62__ + 15 * __temp_a63__);
        float __temp_a65__ = (__tilevar_4__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a66__ = (__temp_a64__ + 12 * __temp_a65__);
        float __temp_a67__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_16__+1),rowy)]);
        float __temp_a68__ = (__temp_a66__ + 5 * __temp_a67__);
        float __temp_a69__ = (__temp_a68__ / 118);
        __tilevar_5__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)] = __temp_a69__;
	// __iter_16__ + 1
        float __temp_b60__ = __temp_a63__;
        float __temp_b61__ = (__tilevar_4__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_b62__ = (5 * __temp_b60__ + 12 * __temp_b61__);
        float __temp_b63__ = __temp_a67__;
        float __temp_b64__ = (__temp_b62__ + 15 * __temp_b63__);
        float __temp_b65__ = (__tilevar_4__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_b66__ = (__temp_b64__ + 12 * __temp_b65__);
        float __temp_b67__ = (__tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_16__+2),rowy)]);
        float __temp_b68__ = (__temp_b66__ + 5 * __temp_b67__);
        float __temp_b69__ = (__temp_b68__ / 118);
        __tilevar_5__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)] = __temp_b69__;
      }
    }
    __syncthreads ();
    int __iter_18__ = FORMA_MAX((__iter_1__-3),3) + (int)(threadIdx.x) ; 
    if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-4)) ){
      int __iter_19__ = FORMA_MAX((__iter_0__+3),1) + threadIdx_y;
      if( __iter_19__ < FORMA_MAX((__iter_0__+3),1)+2) {
        __copy_arr_2__[__iter_18__+(M)*(__iter_19__)] = __tilevar_5__[__iter_19__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_18__,rowy)];
      }
    }
    if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-4)) ){
      int __iter_19__ = FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2))-1 + threadIdx_y;
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
        __copy_arr_2__[__iter_18__+(M)*(__iter_19__)] = __tilevar_5__[__iter_19__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_18__,rowy)];
      }
    }
    __syncthreads();
    int __iter_22__ = FORMA_MAX((__iter_1__-4),4) + 3*(int)(threadIdx.y) ;
    if(__iter_22__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-5))) {
      int __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
	// __iter_22__
        float __temp_a80__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__-1),rowy)]);
        float __temp_a81__ = (__tilevar_5__[__iter_23__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a82__ = (5 * __temp_a80__ + 12 * __temp_a81__);
        float __temp_a83__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a84__ = (__temp_a82__ + 15 * __temp_a83__);
        float __temp_a85__ = (__tilevar_5__[__iter_23__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a86__ = (__temp_a84__ + 12 * __temp_a85__);
        float __temp_a87__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__+1),rowy)]);
        float __temp_a88__ = (__temp_a86__ + 5 * __temp_a87__);
        float __temp_a89__ = (__temp_a88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__)] = __temp_a89__;
	// __iter_22__ + 1
        float __temp_b80__ = __temp_a83__;
        float __temp_b81__ = (__tilevar_5__[__iter_23__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_b82__ = (5 * __temp_b80__ + 12 * __temp_b81__);
        float __temp_b83__ = __temp_a87__;
        float __temp_b84__ = (__temp_b82__ + 15 * __temp_b83__);
        float __temp_b85__ = (__tilevar_5__[__iter_23__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_b86__ = (__temp_b84__ + 12 * __temp_b85__);
        float __temp_b87__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__+2),rowy)]);
        float __temp_b88__ = (__temp_b86__ + 5 * __temp_b87__);
        float __temp_b89__ = (__temp_b88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__+1)] = __temp_b89__;
	// __iter_22__ + 2
        float __temp_c80__ = __temp_a87__;
        float __temp_c81__ = (__tilevar_5__[__iter_23__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+2,rowy)]);
        float __temp_c82__ = (5 * __temp_c80__ + 12 * __temp_c81__);
        float __temp_c83__ = __temp_b87__;
        float __temp_c84__ = (__temp_c82__ + 15 * __temp_c83__);
        float __temp_c85__ = (__tilevar_5__[__iter_23__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+2,rowy)]);
        float __temp_c86__ = (__temp_c84__ + 12 * __temp_c85__);
        float __temp_c87__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__+3),rowy)]);
        float __temp_c88__ = (__temp_c86__ + 5 * __temp_c87__);
        float __temp_c89__ = (__temp_c88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__+2)] = __temp_c89__;
      }
    }
    else if(__iter_22__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-5))) {
      int __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
	// __iter_22__
        float __temp_a80__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__-1),rowy)]);
        float __temp_a81__ = (__tilevar_5__[__iter_23__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a82__ = (5 * __temp_a80__ + 12 * __temp_a81__);
        float __temp_a83__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a84__ = (__temp_a82__ + 15 * __temp_a83__);
        float __temp_a85__ = (__tilevar_5__[__iter_23__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a86__ = (__temp_a84__ + 12 * __temp_a85__);
        float __temp_a87__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__+1),rowy)]);
        float __temp_a88__ = (__temp_a86__ + 5 * __temp_a87__);
        float __temp_a89__ = (__temp_a88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__)] = __temp_a89__;
	// __iter_22__ + 1
        float __temp_b80__ = __temp_a83__;
        float __temp_b81__ = (__tilevar_5__[__iter_23__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_b82__ = (5 * __temp_b80__ + 12 * __temp_b81__);
        float __temp_b83__ = __temp_a87__;
        float __temp_b84__ = (__temp_b82__ + 15 * __temp_b83__);
        float __temp_b85__ = (__tilevar_5__[__iter_23__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_b86__ = (__temp_b84__ + 12 * __temp_b85__);
        float __temp_b87__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__+2),rowy)]);
        float __temp_b88__ = (__temp_b86__ + 5 * __temp_b87__);
        float __temp_b89__ = (__temp_b88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__+1)] = __temp_b89__;
      }
    }
    else if(__iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-5))) {
      int __iter_23__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
	// __iter_22__
        float __temp_a80__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__-1),rowy)]);
        float __temp_a81__ = (__tilevar_5__[__iter_23__-1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a82__ = (5 * __temp_a80__ + 12 * __temp_a81__);
        float __temp_a83__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a84__ = (__temp_a82__ + 15 * __temp_a83__);
        float __temp_a85__ = (__tilevar_5__[__iter_23__+1-__iter_0__+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a86__ = (__temp_a84__ + 12 * __temp_a85__);
        float __temp_a87__ = (__tilevar_5__[__iter_23__-__iter_0__+FORMA_BLOCKDIM_X*mod((__iter_22__+1),rowy)]);
        float __temp_a88__ = (__temp_a86__ + 5 * __temp_a87__);
        float __temp_a89__ = (__temp_a88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__)] = __temp_a89__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int N, int M, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_5__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X));
  float* __tilevar_4__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X));
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X));
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Y+2)*FORMA_BLOCKDIM_X));
  int rowy = FORMA_BLOCKDIM_Y + 2;
  int threadIdx_y = mod(threadIdx.y,2);
 
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X + GAPX) + (int)FORMA_BLOCKDIM_X;
  for (int __iter_1__ = 0; __iter_1__ <= N-1; __iter_1__ += FORMA_BLOCKDIM_Y) {
    int __iter_2__ = FORMA_MAX(__iter_1__,0) + 3*(int)(threadIdx.y) ; 
    if(__iter_2__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(N-1))) {
      int __iter_3__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
      if( __iter_3__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-1)) ){
        __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_2__,rowy)] = input[__iter_3__+(M)*(__iter_2__)];
        __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_2__+1,rowy)] = input[__iter_3__+(M)*(__iter_2__+1)];
        __tilevar_2__[__iter_3__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_2__+2,rowy)] = input[__iter_3__+(M)*(__iter_2__+2)];
      }
    }
    __syncthreads();
    int __iter_4__ = FORMA_MAX((__iter_1__-1),1) + 3*(int)(threadIdx.y) ;
    if(__iter_4__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_5__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
	// __iter_4__
        float __temp_a2__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_4__-1),rowy)]);
        float __temp_a5__ = (__tilevar_2__[__iter_5__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a6__ = (5 * __temp_a2__ + 12 * __temp_a5__);
        float __temp_a9__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a10__ = (__temp_a6__ + 15 * __temp_a9__);
        float __temp_a13__ = (__tilevar_2__[__iter_5__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a14__ = (__temp_a10__ + 12 * __temp_a13__);
        float __temp_a17__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_4__+1),rowy)]);
        float __temp_a18__ = (__temp_a14__ + 5 * __temp_a17__);
        float __temp_a19__ = (__temp_a18__ / 118);
        __tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)] = __temp_a19__;
	// __iter_4__ + 1
        float __temp_b2__ = __temp_a9__;
        float __temp_b5__ = (__tilevar_2__[__iter_5__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)]);
        float __temp_b6__ = (5 * __temp_b2__ + 12 * __temp_b5__);
        float __temp_b9__ = __temp_a17__;
        float __temp_b10__ = (__temp_b6__ + 15 * __temp_b9__);
        float __temp_b13__ = (__tilevar_2__[__iter_5__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)]);
        float __temp_b14__ = (__temp_b10__ + 12 * __temp_b13__);
        float __temp_b17__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_4__+2),rowy)]);
        float __temp_b18__ = (__temp_b14__ + 5 * __temp_b17__);
        float __temp_b19__ = (__temp_b18__ / 118);
        __tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__+1,rowy)] = __temp_b19__;
	// __iter_4__ + 2
        float __temp_c2__ = __temp_a17__;
        float __temp_c5__ = (__tilevar_2__[__iter_5__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__+2,rowy)]);
        float __temp_c6__ = (5 * __temp_c2__ + 12 * __temp_c5__);
        float __temp_c9__ = __temp_b17__;
        float __temp_c10__ = (__temp_c6__ + 15 * __temp_c9__);
        float __temp_c13__ = (__tilevar_2__[__iter_5__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__+2,rowy)]);
        float __temp_c14__ = (__temp_c10__ + 12 * __temp_c13__);
        float __temp_c17__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_4__+3),rowy)]);
        float __temp_c18__ = (__temp_c14__ + 5 * __temp_c17__);
        float __temp_c19__ = (__temp_c18__ / 118);
        __tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__+2,rowy)] = __temp_c19__;
      }
    }
    else if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2))) {
      int __iter_5__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) ){
	// __iter_4__
        float __temp_a2__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_4__-1),rowy)]);
        float __temp_a5__ = (__tilevar_2__[__iter_5__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a6__ = (5 * __temp_a2__ + 12 * __temp_a5__);
        float __temp_a9__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a10__ = (__temp_a6__ + 15 * __temp_a9__);
        float __temp_a13__ = (__tilevar_2__[__iter_5__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)]);
        float __temp_a14__ = (__temp_a10__ + 12 * __temp_a13__);
        float __temp_a17__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_4__+1),rowy)]);
        float __temp_a18__ = (__temp_a14__ + 5 * __temp_a17__);
        float __temp_a19__ = (__temp_a18__ / 118);
        __tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_4__,rowy)] = __temp_a19__;
      }
    }
    __syncthreads ();
    int __iter_6__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.x) ; 
    if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_7__ = FORMA_MAX((__iter_0__-3),1) + threadIdx_y;
      if( __iter_7__ < FORMA_MAX((__iter_0__-1),1)) {
        __tilevar_3__[__iter_7__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_6__,rowy)] = __copy_arr_0__[__iter_6__+(M)*(__iter_7__)];
      }
    }
    if( __iter_6__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(N-2)) ){
      int __iter_7__ = FORMA_MIN(((__iter_0__+GAPX+1)-1),(M-2)) +1 + threadIdx_y;
      if( __iter_7__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
        __tilevar_3__[__iter_7__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_6__,rowy)] = __copy_arr_0__[__iter_6__+(M)*(__iter_7__)];
      }
    }
    __syncthreads();
    int __iter_10__ = FORMA_MAX((__iter_1__-2),2) + 3*(int)(threadIdx.y) ; 
    if(__iter_10__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3))) {
      int __iter_11__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
	// __iter_10__
        float __temp_a32__ = (__tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_10__-1),rowy)]);
        float __temp_a35__ = (__tilevar_3__[__iter_11__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a36__ = (5 * __temp_a32__ + 12 * __temp_a35__);
        float __temp_a39__ = (__tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a40__ = (__temp_a36__ + 15 * __temp_a39__);
        float __temp_a43__ = (__tilevar_3__[__iter_11__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a44__ = (__temp_a40__ + 12 * __temp_a43__);
        float __temp_a47__ = (__tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_10__+1),rowy)]);
        float __temp_a48__ = (__temp_a44__ + 5 * __temp_a47__);
        float __temp_a49__ = (__temp_a48__ / 118);
        __tilevar_4__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)] = __temp_a49__;
	// __iter_10__ + 1
        float __temp_b32__ = __temp_a39__;
        float __temp_b35__ = (__tilevar_3__[__iter_11__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_b36__ = (5 * __temp_b32__ + 12 * __temp_b35__);
        float __temp_b39__ = __temp_a47__;
        float __temp_b40__ = (__temp_b36__ + 15 * __temp_b39__);
        float __temp_b43__ = (__tilevar_3__[__iter_11__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_b44__ = (__temp_b40__ + 12 * __temp_b43__);
        float __temp_b47__ = (__tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_10__+2),rowy)]);
        float __temp_b48__ = (__temp_b44__ + 5 * __temp_b47__);
        float __temp_b49__ = (__temp_b48__ / 118);
        __tilevar_4__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)] = __temp_b49__;
	// __iter_10__ + 2
        float __temp_c32__ = __temp_a47__;
        float __temp_c35__ = (__tilevar_3__[__iter_11__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__+2,rowy)]);
        float __temp_c36__ = (5 * __temp_c32__ + 12 * __temp_c35__);
        float __temp_c39__ = __temp_b47__;
        float __temp_c40__ = (__temp_c36__ + 15 * __temp_c39__);
        float __temp_c43__ = (__tilevar_3__[__iter_11__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__+2,rowy)]);
        float __temp_c44__ = (__temp_c40__ + 12 * __temp_c43__);
        float __temp_c47__ = (__tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_10__+3),rowy)]);
        float __temp_c48__ = (__temp_c44__ + 5 * __temp_c47__);
        float __temp_c49__ = (__temp_c48__ / 118);
        __tilevar_4__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__+2,rowy)] = __temp_c49__;
      }
    }
    else if(__iter_10__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3))) {
      int __iter_11__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) ){
	// __iter_10__
        float __temp_a32__ = (__tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_10__-1),rowy)]);
        float __temp_a35__ = (__tilevar_3__[__iter_11__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a36__ = (5 * __temp_a32__ + 12 * __temp_a35__);
        float __temp_a39__ = (__tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a40__ = (__temp_a36__ + 15 * __temp_a39__);
        float __temp_a43__ = (__tilevar_3__[__iter_11__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)]);
        float __temp_a44__ = (__temp_a40__ + 12 * __temp_a43__);
        float __temp_a47__ = (__tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_10__+1),rowy)]);
        float __temp_a48__ = (__temp_a44__ + 5 * __temp_a47__);
        float __temp_a49__ = (__temp_a48__ / 118);
        __tilevar_4__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__,rowy)] = __temp_a49__;
	// __iter_10__ + 1
        float __temp_b32__ = __temp_a39__;
        float __temp_b35__ = (__tilevar_3__[__iter_11__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_b36__ = (5 * __temp_b32__ + 12 * __temp_b35__);
        float __temp_b39__ = __temp_a47__;
        float __temp_b40__ = (__temp_b36__ + 15 * __temp_b39__);
        float __temp_b43__ = (__tilevar_3__[__iter_11__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)]);
        float __temp_b44__ = (__temp_b40__ + 12 * __temp_b43__);
        float __temp_b47__ = (__tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_10__+2),rowy)]);
        float __temp_b48__ = (__temp_b44__ + 5 * __temp_b47__);
        float __temp_b49__ = (__temp_b48__ / 118);
        __tilevar_4__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_10__+1,rowy)] = __temp_b49__;
      }
    }
    __syncthreads ();
    int __iter_12__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.x) ; 
    if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3)) ){
      int __iter_13__ = FORMA_MAX((__iter_0__-4),1) + threadIdx_y;
      if( __iter_13__ < FORMA_MAX((__iter_0__-2),1)) {
        __tilevar_4__[__iter_13__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_12__,rowy)] = __copy_arr_1__[__iter_12__+(M)*(__iter_13__)];
      }
    }
    if( __iter_12__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(N-3)) ){
      int __iter_13__ = FORMA_MIN(((__iter_0__+GAPX+2)-1),(M-2)) + 1 + threadIdx_y;
      if( __iter_13__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
        __tilevar_4__[__iter_13__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_12__,rowy)] = __copy_arr_1__[__iter_12__+(M)*(__iter_13__)];
      }
    }
    __syncthreads();
    int __iter_16__ = FORMA_MAX((__iter_1__-3),1) + 3*(int)(threadIdx.y) ; 
    if(__iter_16__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-4))) {
      int __iter_17__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
	// __iter_16__
        float __temp_a60__ = (__tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_16__-1),rowy)]);
        float __temp_a61__ = (__tilevar_4__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a62__ = (5 * __temp_a60__ + 12 * __temp_a61__);
        float __temp_a63__ = (__tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a64__ = (__temp_a62__ + 15 * __temp_a63__);
        float __temp_a65__ = (__tilevar_4__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a66__ = (__temp_a64__ + 12 * __temp_a65__);
        float __temp_a67__ = (__tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_16__+1),rowy)]);
        float __temp_a68__ = (__temp_a66__ + 5 * __temp_a67__);
        float __temp_a69__ = (__temp_a68__ / 118);
        __tilevar_5__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)] = __temp_a69__;
	// __iter_16__ + 1
        float __temp_b60__ = __temp_a63__;
        float __temp_b61__ = (__tilevar_4__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_b62__ = (5 * __temp_b60__ + 12 * __temp_b61__);
        float __temp_b63__ = __temp_a67__;
        float __temp_b64__ = (__temp_b62__ + 15 * __temp_b63__);
        float __temp_b65__ = (__tilevar_4__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_b66__ = (__temp_b64__ + 12 * __temp_b65__);
        float __temp_b67__ = (__tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_16__+2),rowy)]);
        float __temp_b68__ = (__temp_b66__ + 5 * __temp_b67__);
        float __temp_b69__ = (__temp_b68__ / 118);
        __tilevar_5__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)] = __temp_b69__;
	// __iter_16__ + 2
        float __temp_c60__ = __temp_a67__;
        float __temp_c61__ = (__tilevar_4__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__+2,rowy)]);
        float __temp_c62__ = (5 * __temp_c60__ + 12 * __temp_c61__);
        float __temp_c63__ = __temp_b67__;
        float __temp_c64__ = (__temp_c62__ + 15 * __temp_c63__);
        float __temp_c65__ = (__tilevar_4__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__+2,rowy)]);
        float __temp_c66__ = (__temp_c64__ + 12 * __temp_c65__);
        float __temp_c67__ = (__tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_16__+3),rowy)]);
        float __temp_c68__ = (__temp_c66__ + 5 * __temp_c67__);
        float __temp_c69__ = (__temp_c68__ / 118);
        __tilevar_5__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__+2,rowy)] = __temp_c69__;
      }
    }
    if(__iter_16__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-4))) {
      int __iter_17__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) ){
	// __iter_16__
        float __temp_a60__ = (__tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_16__-1),rowy)]);
        float __temp_a61__ = (__tilevar_4__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a62__ = (5 * __temp_a60__ + 12 * __temp_a61__);
        float __temp_a63__ = (__tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a64__ = (__temp_a62__ + 15 * __temp_a63__);
        float __temp_a65__ = (__tilevar_4__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)]);
        float __temp_a66__ = (__temp_a64__ + 12 * __temp_a65__);
        float __temp_a67__ = (__tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_16__+1),rowy)]);
        float __temp_a68__ = (__temp_a66__ + 5 * __temp_a67__);
        float __temp_a69__ = (__temp_a68__ / 118);
        __tilevar_5__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__,rowy)] = __temp_a69__;
	// __iter_16__ + 1
        float __temp_b60__ = __temp_a63__;
        float __temp_b61__ = (__tilevar_4__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_b62__ = (5 * __temp_b60__ + 12 * __temp_b61__);
        float __temp_b63__ = __temp_a67__;
        float __temp_b64__ = (__temp_b62__ + 15 * __temp_b63__);
        float __temp_b65__ = (__tilevar_4__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)]);
        float __temp_b66__ = (__temp_b64__ + 12 * __temp_b65__);
        float __temp_b67__ = (__tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_16__+2),rowy)]);
        float __temp_b68__ = (__temp_b66__ + 5 * __temp_b67__);
        float __temp_b69__ = (__temp_b68__ / 118);
        __tilevar_5__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_16__+1,rowy)] = __temp_b69__;
      }
    }
    __syncthreads ();
    int __iter_18__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.x) ; 
    if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-4)) ){
      int __iter_19__ = FORMA_MAX((__iter_0__-5),1) + threadIdx_y;
      if( __iter_19__ < FORMA_MAX((__iter_0__-3),1)) {
        __tilevar_5__[__iter_19__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_18__,rowy)] = __copy_arr_2__[__iter_18__+(M)*(__iter_19__)];
      }
    }
    if( __iter_18__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(N-4)) ){
      int __iter_19__ = FORMA_MIN(((__iter_0__+GAPX+3)-1),(M-2)) + 1 + threadIdx_y;
      if( __iter_19__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(M-2)) ){
        __tilevar_5__[__iter_19__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_18__,rowy)] = __copy_arr_2__[__iter_18__+(M)*(__iter_19__)];
      }
    }
    __syncthreads();
    int __iter_22__ = FORMA_MAX((__iter_1__-4),1) + 3*(int)(threadIdx.y) ; 
    if(__iter_22__ + 2 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-5))) {
      int __iter_23__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
	// __iter_22__
        float __temp_a80__ = (__tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_22__-1),rowy)]);
        float __temp_a81__ = (__tilevar_5__[__iter_23__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a82__ = (5 * __temp_a80__ + 12 * __temp_a81__);
        float __temp_a83__ = (__tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a84__ = (__temp_a82__ + 15 * __temp_a83__);
        float __temp_a85__ = (__tilevar_5__[__iter_23__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a86__ = (__temp_a84__ + 12 * __temp_a85__);
        float __temp_a87__ = (__tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_22__+1),rowy)]);
        float __temp_a88__ = (__temp_a86__ + 5 * __temp_a87__);
        float __temp_a89__ = (__temp_a88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__)] = __temp_a89__;
	// __iter_22__ + 1
        float __temp_b80__ = __temp_a83__;
        float __temp_b81__ = (__tilevar_5__[__iter_23__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_b82__ = (5 * __temp_b80__ + 12 * __temp_b81__);
        float __temp_b83__ = __temp_a87__;
        float __temp_b84__ = (__temp_b82__ + 15 * __temp_b83__);
        float __temp_b85__ = (__tilevar_5__[__iter_23__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_b86__ = (__temp_b84__ + 12 * __temp_b85__);
        float __temp_b87__ = (__tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_22__+2),rowy)]);
        float __temp_b88__ = (__temp_b86__ + 5 * __temp_b87__);
        float __temp_b89__ = (__temp_b88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__+1)] = __temp_b89__;
	// __iter_22__ + 2
        float __temp_c80__ = __temp_a87__;
        float __temp_c81__ = (__tilevar_5__[__iter_23__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__+2,rowy)]);
        float __temp_c82__ = (5 * __temp_c80__ + 12 * __temp_c81__);
        float __temp_c83__ = __temp_b87__;
        float __temp_c84__ = (__temp_c82__ + 15 * __temp_c83__);
        float __temp_c85__ = (__tilevar_5__[__iter_23__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__+2,rowy)]);
        float __temp_c86__ = (__temp_c84__ + 12 * __temp_c85__);
        float __temp_c87__ = (__tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_22__+3),rowy)]);
        float __temp_c88__ = (__temp_c86__ + 5 * __temp_c87__);
        float __temp_c89__ = (__temp_c88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__+2)] = __temp_c89__;
      }
    }
    else if(__iter_22__ + 1 <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(N-5))) {
      int __iter_23__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(M-2)) ){
	// __iter_22__
        float __temp_a80__ = (__tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_22__-1),rowy)]);
        float __temp_a81__ = (__tilevar_5__[__iter_23__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a82__ = (5 * __temp_a80__ + 12 * __temp_a81__);
        float __temp_a83__ = (__tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a84__ = (__temp_a82__ + 15 * __temp_a83__);
        float __temp_a85__ = (__tilevar_5__[__iter_23__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__,rowy)]);
        float __temp_a86__ = (__temp_a84__ + 12 * __temp_a85__);
        float __temp_a87__ = (__tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_22__+1),rowy)]);
        float __temp_a88__ = (__temp_a86__ + 5 * __temp_a87__);
        float __temp_a89__ = (__temp_a88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__)] = __temp_a89__;
	// __iter_22__ + 1
        float __temp_b80__ = __temp_a83__;
        float __temp_b81__ = (__tilevar_5__[__iter_23__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_b82__ = (5 * __temp_b80__ + 12 * __temp_b81__);
        float __temp_b83__ = __temp_a87__;
        float __temp_b84__ = (__temp_b82__ + 15 * __temp_b83__);
        float __temp_b85__ = (__tilevar_5__[__iter_23__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod(__iter_22__+1,rowy)]);
        float __temp_b86__ = (__temp_b84__ + 12 * __temp_b85__);
        float __temp_b87__ = (__tilevar_5__[__iter_23__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*mod((__iter_22__+2),rowy)]);
        float __temp_b88__ = (__temp_b86__ + 5 * __temp_b87__);
        float __temp_b89__ = (__temp_b88__ / 118);
        __var_1__[__iter_23__+(M)*(__iter_22__+1)] = __temp_b89__;
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N)*(M)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 6;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  dim3 unrollConfig(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__/3);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"
#include <nvml.h>
#include <assert.h>

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define mod(x,y) ( (x) & (y-1))

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
	int __FORMA_SHARED_MEM_OFFSET__ = 0;
	float * __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
	__FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
	float * __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
	__FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
	float * __tilevar_4__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
	__FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
	float * __tilevar_5__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
	__FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_X);
	float t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f;
	float b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;
	int __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X-8);
	int __iter_y__ = FORMA_MAX((int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y)-4, 0);
	// Initialize the values
	int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
	if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
		__tilevar_2__[__iter_3__-__iter_0__] = input[__iter_3__+M*__iter_y__];
		t2 = input[__iter_3__+M*(__iter_y__+1)]; 
	}

	// Initial computation
	for (int __iter_1__ = FORMA_MAX(1,__iter_y__+1); __iter_1__ < FORMA_MIN(N-1,__iter_y__+7); __iter_1__++) {
		if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
			b2 = __tilevar_2__[__iter_3__-__iter_0__];
			__tilevar_2__[__iter_3__-__iter_0__] = t2;
			t2 = input[__iter_3__+M*(__iter_1__+1)];  
		}
		__syncthreads();
		if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
			float __temp_2__ = b2;
			float __temp_5__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
			float __temp_6__ = (5 * __temp_2__ + 12 * __temp_5__);
			float __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__]);
			float __temp_10__ = (__temp_6__ + 15 * __temp_9__);
			float __temp_13__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
			float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
			float __temp_17__ = t2;
			float __temp_18__ = (__temp_14__ + 5 * __temp_17__);
			float __temp_19__ = (__temp_18__ / 118);
			b3 = __tilevar_3__[__iter_3__-__iter_0__];
			__tilevar_3__[__iter_3__-__iter_0__] = t3;
			t3 = __temp_19__;
		}
		__syncthreads();
		if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
			float __temp_32__ = b3;
			float __temp_35__ = (__tilevar_3__[__iter_3__+(-1)-__iter_0__]);
			float __temp_36__ = (5 * __temp_32__ + 12 * __temp_35__);
			float __temp_39__ = (__tilevar_3__[__iter_3__-__iter_0__]);
			float __temp_40__ = (__temp_36__ + 15 * __temp_39__);
			float __temp_43__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
			float __temp_44__ = (__temp_40__ + 12 * __temp_43__);
			float __temp_47__ = t3;
			float __temp_48__ = (__temp_44__ + 5 * __temp_47__);
			float __temp_49__ = (__temp_48__ / 118);
			b4 = __tilevar_4__[__iter_3__-__iter_0__];
			__tilevar_4__[__iter_3__-__iter_0__] = t4;
			t4 = __temp_49__;
		}
		__syncthreads ();
		if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
			float __temp_60__ = b4;
			float __temp_61__ = (__tilevar_4__[__iter_3__+(-1)-__iter_0__]);
			float __temp_62__ = (5 * __temp_60__ + 12 * __temp_61__);
			float __temp_63__ = (__tilevar_4__[__iter_3__-__iter_0__]);
			float __temp_64__ = (__temp_62__ + 15 * __temp_63__);
			float __temp_65__ = (__tilevar_4__[__iter_3__+1-__iter_0__]);
			float __temp_66__ = (__temp_64__ + 12 * __temp_65__);
			float __temp_67__ = t4;
			float __temp_68__ = (__temp_66__ + 5 * __temp_67__);
			float __temp_69__ = (__temp_68__ / 118);
			b5 = __tilevar_5__[__iter_3__-__iter_0__];
			__tilevar_5__[__iter_3__-__iter_0__] = t5;
			t5 = __temp_69__;
		}
	}
	// Rest of the computation
	for (int __iter_1__ = FORMA_MAX(1,__iter_y__+7); __iter_1__ < FORMA_MIN(N-1,__iter_y__+FORMA_BLOCKDIM_Y+8); __iter_1__++) {
		if(__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))){
			b2 = __tilevar_2__[__iter_3__-__iter_0__];
			__tilevar_2__[__iter_3__-__iter_0__] = t2;
			t2 = input[__iter_3__+M*(__iter_1__+1)];  
		}
		__syncthreads();
		if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2)) ){
			float __temp_2__ = b2;
			float __temp_5__ = (__tilevar_2__[__iter_3__-1-__iter_0__]);
			float __temp_6__ = (5 * __temp_2__ + 12 * __temp_5__);
			float __temp_9__ = (__tilevar_2__[__iter_3__-__iter_0__]);
			float __temp_10__ = (__temp_6__ + 15 * __temp_9__);
			float __temp_13__ = (__tilevar_2__[__iter_3__+1-__iter_0__]);
			float __temp_14__ = (__temp_10__ + 12 * __temp_13__);
			float __temp_17__ = t2;
			float __temp_18__ = (__temp_14__ + 5 * __temp_17__);
			float __temp_19__ = (__temp_18__ / 118);
			b3 = __tilevar_3__[__iter_3__-__iter_0__];
			__tilevar_3__[__iter_3__-__iter_0__] = t3;
			t3 = __temp_19__;
		}
		__syncthreads();
		if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2)) ){
			float __temp_32__ = b3;
			float __temp_35__ = (__tilevar_3__[__iter_3__+(-1)-__iter_0__]);
			float __temp_36__ = (5 * __temp_32__ + 12 * __temp_35__);
			float __temp_39__ = (__tilevar_3__[__iter_3__-__iter_0__]);
			float __temp_40__ = (__temp_36__ + 15 * __temp_39__);
			float __temp_43__ = (__tilevar_3__[__iter_3__+1-__iter_0__]);
			float __temp_44__ = (__temp_40__ + 12 * __temp_43__);
			float __temp_47__ = t3;
			float __temp_48__ = (__temp_44__ + 5 * __temp_47__);
			float __temp_49__ = (__temp_48__ / 118);
			b4 = __tilevar_4__[__iter_3__-__iter_0__];
			__tilevar_4__[__iter_3__-__iter_0__] = t4;
			t4 = __temp_49__;
		}
		__syncthreads ();
		if(__iter_3__ >= FORMA_MAX((__iter_0__+3),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(M-2)) ){
			float __temp_60__ = b4;
			float __temp_61__ = (__tilevar_4__[__iter_3__+(-1)-__iter_0__]);
			float __temp_62__ = (5 * __temp_60__ + 12 * __temp_61__);
			float __temp_63__ = (__tilevar_4__[__iter_3__-__iter_0__]);
			float __temp_64__ = (__temp_62__ + 15 * __temp_63__);
			float __temp_65__ = (__tilevar_4__[__iter_3__+1-__iter_0__]);
			float __temp_66__ = (__temp_64__ + 12 * __temp_65__);
			float __temp_67__ = t4;
			float __temp_68__ = (__temp_66__ + 5 * __temp_67__);
			float __temp_69__ = (__temp_68__ / 118);
			b5 = __tilevar_5__[__iter_3__-__iter_0__];
			__tilevar_5__[__iter_3__-__iter_0__] = t5;
			t5 = __temp_69__;
		}
		__syncthreads ();
		if(__iter_3__ >= FORMA_MAX((__iter_0__+4),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(M-2)) ){
			float __temp_80__ = b5;
			float __temp_81__ = (__tilevar_5__[__iter_3__+(-1)-__iter_0__]);
			float __temp_82__ = (5 * __temp_80__ + 12 * __temp_81__);
			float __temp_83__ = (__tilevar_5__[__iter_3__-__iter_0__]);
			float __temp_84__ = (__temp_82__ + 15 * __temp_83__);
			float __temp_85__ = (__tilevar_5__[__iter_3__+1-__iter_0__]);
			float __temp_86__ = (__temp_84__ + 12 * __temp_85__);
			float __temp_87__ = t5;
			float __temp_88__ = (__temp_86__ + 5 * __temp_87__);
			float __temp_89__ = (__temp_88__ / 118);
			__var_1__[__iter_3__+(M)*FORMA_MAX(__iter_1__-3,0)] = __temp_89__;
		}
	}
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int SMemSize = 0;
	SMemSize += sizeof(float)*(4*FORMA_BLOCKDIM_X);
	return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

	/* Host allocation Begin */
	float * input;
	hipMalloc(&input,sizeof(float)*((N)*(M)));
	Check_CUDA_Error("Allocation Error!! : input\n");
	hipPointerAttribute_t ptrAttrib_h_input;
	hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
	if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
		if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
			memcpy_kind_h_input = hipMemcpyDeviceToDevice;
	hipGetLastError();
	if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
		hipMemcpy(input,h_input,sizeof(float)*((N)*(M)), memcpy_kind_h_input);
	}
	float * __var_1__;
	hipMalloc(&__var_1__,sizeof(float)*((N)*(M)));
	Check_CUDA_Error("Allocation Error!! : __var_1__\n");
	/*Host Allocation End */
	/* Kernel Launch Begin */
	int __FORMA_MAX_SHARED_MEM__;
	hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
	hipEvent_t _forma_timer_start_,_forma_timer_stop_;
	hipEventCreate(&_forma_timer_start_);
	hipEventCreate(&_forma_timer_stop_);
	hipEventRecord(_forma_timer_start_,0);
#endif
	int __size_0___kernel___forma_kernel__0__ = M;
	int __size_1___kernel___forma_kernel__0__ = N;
	int __block_0___kernel___forma_kernel__0__ = 128;
	int __block_1___kernel___forma_kernel__0__ = 1;
	dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
	int __SMemSize___kernel___forma_kernel__0__ = 0;
	__SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
	int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-8);
	int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__, __size_1___kernel___forma_kernel__0__/64);
	dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<1000; x++) {
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __size_1___kernel___forma_kernel__0__/64, __var_1__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();
	hipPointerAttribute_t ptrAttrib___var_0__;
	hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
	if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
		if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
			memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
	hipGetLastError();
	hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
	hipEventRecord(_forma_timer_stop_,0);
	hipEventSynchronize(_forma_timer_stop_);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
	printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
	hipEventDestroy(_forma_timer_start_);
	hipEventDestroy(_forma_timer_stop_);
#endif
	/*Kernel Launch End */
	/* Host Free Begin */
	hipFree(input);
	hipFree(__var_1__);
}
/*Host Free End*/

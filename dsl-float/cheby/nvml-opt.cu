/* --------------------------------------------------------------------
   OPTIMIZED CODE MAKING USE OF REGISTERS + SHARED MEMORY
   ----------------------------------------------------------------------*/


#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)
#include <nvml.h>
#include <assert.h>


void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void cheby (float h2inv, float a, float b, float c, float d, float * __restrict__ RHS, float * __restrict__ Ap, float * __restrict__ Dinv, float * __restrict__ Ac, float * __restrict__ out, int L, int M, int N, float * __restrict__ out1) {
	//Determing the block's indices
	int blockdim_i= (int)(blockDim.x);
	int i0 = (int)(blockIdx.x)*(blockdim_i-4);
	int i = max (i0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j-4);
	int j = max (j0, 0) + (int)(threadIdx.y);

	//Declarations
	float reg_Ac_m2=0, reg_Ac_m1=0, __shared__ sh_Ac_c0[32][32];
	float reg_Ap_m1=0;
	float reg_Dinv_m2=0, reg_Dinv_m1=0;
	float reg_RHS_m2=0, reg_RHS_m1=0;
	float reg_out1_m2=0, __shared__ sh_out1_m1[32][32];
	float reg_temp1_m1=0, reg_temp1_c0=0, reg_temp1_p1=0;
	float reg_temp2_m2=0, reg_temp2_m1=0, reg_temp2_c0=0;

	//Value Initialization
	if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
		sh_Ac_c0[j-j0][i-i0] = 0;
		sh_out1_m1[j-j0][i-i0] = 0;
	}
	//Rest of the computation
	for (int k=0; k<=L-1; ++k) {
		//Fetch new plane
		sh_Ac_c0[j-j0][i-i0] = Ac[k*M*N + j*N + i];
		reg_Ap_m1 = Ap[max(k-1,0)*M*N + j*N + i];
		reg_Dinv_m1 = Dinv[max(k-1,0)*M*N + j*N + i];
		reg_RHS_m1 = RHS[max(k-1,0)*M*N + j*N + i];

		__syncthreads ();
		if (j >= max (j0+1, 1) & j <= min (j0+blockdim_j-2, M-2) & i >= max (i0+1, 1) & i <= min (i0+blockdim_i-2, N-2)) {
			reg_temp1_p1 -= (h2inv * (((0.03f * (((sh_Ac_c0[j-j0-1][i-i0-1] + sh_Ac_c0[j-j0-1][i-i0+1]) + sh_Ac_c0[j-j0+1][i-i0-1]) + sh_Ac_c0[j-j0+1][i-i0+1])) + (0.1f * (((sh_Ac_c0[j-j0-1][i-i0] + sh_Ac_c0[j-j0][i-i0-1]) + sh_Ac_c0[j-j0][i-i0+1]) + sh_Ac_c0[j-j0+1][i-i0]))) + (0.46f * sh_Ac_c0[j-j0][i-i0])));
			reg_temp1_c0 += (sh_Ac_c0[j-j0][i-i0] - (h2inv * (((0.1f * (((sh_Ac_c0[j-j0-1][i-i0-1] + sh_Ac_c0[j-j0-1][i-i0+1]) + sh_Ac_c0[j-j0+1][i-i0-1]) + sh_Ac_c0[j-j0+1][i-i0+1])) + (0.46f * (((sh_Ac_c0[j-j0-1][i-i0] + sh_Ac_c0[j-j0][i-i0-1]) + sh_Ac_c0[j-j0][i-i0+1]) + sh_Ac_c0[j-j0+1][i-i0]))) - (4.26f * sh_Ac_c0[j-j0][i-i0]))));
			reg_temp1_m1 -= (h2inv * (((0.03f * (((sh_Ac_c0[j-j0-1][i-i0-1] + sh_Ac_c0[j-j0-1][i-i0+1]) + sh_Ac_c0[j-j0+1][i-i0-1]) + sh_Ac_c0[j-j0+1][i-i0+1])) + (0.1f * (((sh_Ac_c0[j-j0-1][i-i0] + sh_Ac_c0[j-j0][i-i0-1]) + sh_Ac_c0[j-j0][i-i0+1]) + sh_Ac_c0[j-j0+1][i-i0]))) + (0.46f * sh_Ac_c0[j-j0][i-i0])));
			sh_out1_m1[j-j0][i-i0] = ((reg_Ac_m1 + (a * (reg_Ac_m1 - reg_Ap_m1))) + ((b * reg_Dinv_m1) * (reg_RHS_m1 - reg_temp1_m1)));
			out1[max(k-1,0)*M*N+j*N+i] = sh_out1_m1[j-j0][i-i0];
		}
		__syncthreads ();
		if (j >= max (j0+2, 1) & j <= min (j0+blockdim_j-3, M-2) & i >= max (i0+2, 1) & i <= min (i0+blockdim_i-3, N-2)) {
			reg_temp2_c0 -= (h2inv * (((0.03f * (((sh_out1_m1[j-j0-1][i-i0-1] + sh_out1_m1[j-j0-1][i-i0+1]) + sh_out1_m1[j-j0+1][i-i0-1]) + sh_out1_m1[j-j0+1][i-i0+1])) + (0.1f * (((sh_out1_m1[j-j0-1][i-i0] + sh_out1_m1[j-j0][i-i0-1]) + sh_out1_m1[j-j0][i-i0+1]) + sh_out1_m1[j-j0+1][i-i0]))) + (0.46f * sh_out1_m1[j-j0][i-i0])));
			reg_temp2_m1 += (sh_out1_m1[j-j0][i-i0] - (h2inv * (((0.1f * (((sh_out1_m1[j-j0-1][i-i0-1] + sh_out1_m1[j-j0-1][i-i0+1]) + sh_out1_m1[j-j0+1][i-i0-1]) + sh_out1_m1[j-j0+1][i-i0+1])) + (0.46f * (((sh_out1_m1[j-j0-1][i-i0] + sh_out1_m1[j-j0][i-i0-1]) + sh_out1_m1[j-j0][i-i0+1]) + sh_out1_m1[j-j0+1][i-i0]))) - (4.26f * sh_out1_m1[j-j0][i-i0]))));
			reg_temp2_m2 -= (h2inv * (((0.03f * (((sh_out1_m1[j-j0-1][i-i0-1] + sh_out1_m1[j-j0-1][i-i0+1]) + sh_out1_m1[j-j0+1][i-i0-1]) + sh_out1_m1[j-j0+1][i-i0+1])) + (0.1f * (((sh_out1_m1[j-j0-1][i-i0] + sh_out1_m1[j-j0][i-i0-1]) + sh_out1_m1[j-j0][i-i0+1]) + sh_out1_m1[j-j0+1][i-i0]))) + (0.46f * sh_out1_m1[j-j0][i-i0])));
			out[max(k-2,0)*M*N + j*N + i] = ((reg_out1_m2 + (c * (reg_out1_m2 - reg_Ac_m2))) + ((d * reg_Dinv_m2) * (reg_RHS_m2 - reg_temp2_m2)));
		}
		__syncthreads ();
		//Value rotation

		reg_Ac_m2 = reg_Ac_m1;
		reg_Ac_m1 = sh_Ac_c0[j-j0][i-i0];
		sh_Ac_c0[j-j0][i-i0] = 0;
		reg_Dinv_m2 = reg_Dinv_m1;
		reg_Dinv_m1 = 0;
		reg_RHS_m2 = reg_RHS_m1;
		reg_RHS_m1 = 0;
		reg_out1_m2 = sh_out1_m1[j-j0][i-i0];
		sh_out1_m1[j-j0][i-i0] = 0;
		reg_temp1_m1 = reg_temp1_c0;
		reg_temp1_c0 = reg_temp1_p1;
		reg_temp1_p1 = 0;
		reg_temp2_m2 = reg_temp2_m1;
		reg_temp2_m1 = reg_temp2_c0;
		reg_temp2_c0 = 0;
		__syncthreads ();
	}
}

extern "C" void host_code (float *h_Ac, float *h_Ap, float *h_Dinv, float *h_RHS, float *h_out, float *c1, float *c2, float h2inv, int L, int M, int N) {
	float *Ac;
	hipMalloc (&Ac, sizeof(float)*L*M*N);
	check_error ("Failed to allocate device memory for Ac\n");
	hipMemcpy (Ac, h_Ac, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
	float *Ap;
	hipMalloc (&Ap, sizeof(float)*L*M*N);
	check_error ("Failed to allocate device memory for Ap\n");
	hipMemcpy (Ap, h_Ap, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
	float *Dinv;
	hipMalloc (&Dinv, sizeof(float)*L*M*N);
	check_error ("Failed to allocate device memory for Dinv\n");
	hipMemcpy (Dinv, h_Dinv, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
	float *RHS;
	hipMalloc (&RHS, sizeof(float)*L*M*N);
	check_error ("Failed to allocate device memory for RHS\n");
	hipMemcpy (RHS, h_RHS, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
	float *out;
	hipMalloc (&out, sizeof(float)*L*M*N);
	check_error ("Failed to allocate device memory for out\n");
	float *out1;
	hipMalloc (&out1, sizeof(float)*L*M*N);
	check_error ("Failed to allocate device memory for out1\n");
	float *out2;
	hipMalloc (&out2, sizeof(float)*L*M*N);
	check_error ("Failed to allocate device memory for out2\n");
	float *out3;
	hipMalloc (&out3, sizeof(float)*L*M*N);
	check_error ("Failed to allocate device memory for out3\n");
	dim3 blockconfig_1 (32, 32, 1);
	dim3 gridconfig_1 (ceil(N, blockconfig_1.x-4), ceil(M, blockconfig_1.y-4), 1);
	dim3 blockconfig_2 (32, 32, 1);
	dim3 gridconfig_2 (ceil(N, blockconfig_2.x-4), ceil(M, blockconfig_2.y-4), 1);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<500; x++) {
		cheby <<<gridconfig_1, blockconfig_1>>> (h2inv, c1[0], c2[0], c1[1], c2[1], RHS, Ap, Dinv, Ac, out2, L, M, N, out1);
		cheby <<<gridconfig_2, blockconfig_2>>> (h2inv, c1[2], c2[2], c1[3], c2[3], RHS, out1, Dinv, out2, out, L, M, N, out3);
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();
	hipMemcpy (h_out, out, sizeof(float)*L*M*N, hipMemcpyDeviceToHost);
}

/* --------------------------------------------------------------------
        OPTIMIZED CODE MAKING USE OF REGISTERS + SHARED MEMORY
----------------------------------------------------------------------*/

#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
  hipError_t error = hipGetLastError ();
  if (error != hipSuccess) {
    printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
    exit(-1);
  }
}

__global__ void heat (float * __restrict__ in, float * __restrict__ out1, int L, int M, int N) {
  //Determing the block's indices
  int blockdim_i= (int)(blockDim.x);
  int i0 = (int)(blockIdx.x)*(blockdim_i-4);
  int i = max (i0, 0) + (int)(threadIdx.x);
  int blockdim_j= (int)(blockDim.y);
  int j0 = (int)(blockIdx.y)*(blockdim_j-4);
  int j = max (j0, 0) + (int)(threadIdx.y);

  //Declarations
  float reg_in_m1=0, __shared__ sh_in_c0[16][32], reg_in_p1=0;
  float reg_out_m2=0, __shared__ sh_out_m1[16][32], reg_out_c0=0;

  //Value Initialization
  if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
    reg_in_m1 = in[0 + j*N + i];
    sh_in_c0[j-j0][i-i0] = in[1*M*N + j*N + i];
  }

  //Rest of the computation
  for (int k=1; k<=L-2; ++k) {
    //Fetch new plane
    if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
        reg_in_p1 = in[(k+1)*M*N + j*N + i];
    }
    __syncthreads ();
    if (j >= max (j0+1, 1) & j <= min (j0+blockdim_j-2, M-2) & i >= max (i0+1, 1) & i <= min (i0+blockdim_i-2, N-2)) {
        reg_out_c0 = ((((0.125f * ((reg_in_p1 - (2.0f * sh_in_c0[j-j0][i-i0])) + reg_in_m1)) + (0.125f * ((sh_in_c0[j-j0+1][i-i0] - (2.0f * sh_in_c0[j-j0][i-i0])) + sh_in_c0[j-j0-1][i-i0]))) + (0.125f * ((sh_in_c0[j-j0][i-i0+1] - (2.0f * sh_in_c0[j-j0][i-i0])) + sh_in_c0[j-j0][i-i0-1]))) + sh_in_c0[j-j0][i-i0]);
    }
    if (j >= max (j0+2, 1) & j <= min (j0+blockdim_j-3, M-2) & i >= max (i0+2, 1) & i <= min (i0+blockdim_i-3, N-2)) {
        out1[max(k-1,0)*M*N + j*N + i] = ((((0.125f * ((reg_out_c0 - (2.0f * sh_out_m1[j-j0][i-i0])) + reg_out_m2)) + (0.125f * ((sh_out_m1[j-j0+1][i-i0] - (2.0f * sh_out_m1[j-j0][i-i0])) + sh_out_m1[j-j0-1][i-i0]))) + (0.125f * ((sh_out_m1[j-j0][i-i0+1] - (2.0f * sh_out_m1[j-j0][i-i0])) + sh_out_m1[j-j0][i-i0-1]))) + sh_out_m1[j-j0][i-i0]);
    }
    __syncthreads ();
    //Value rotation
    if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
        reg_in_m1 = sh_in_c0[j-j0][i-i0];
        sh_in_c0[j-j0][i-i0] = reg_in_p1;
        reg_out_m2 = sh_out_m1[j-j0][i-i0];
        sh_out_m1[j-j0][i-i0] = reg_out_c0;
    }
    __syncthreads ();
  }
}

extern "C" void host_code (float *h_in, float *h_out2, int L, int M, int N) {
  float *in;
  hipMalloc (&in, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for in\n");
  hipMemcpy (in, h_in, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *out1;
  hipMalloc (&out1, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for out1\n");
  float *out2;
  hipMalloc (&out2, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for out2\n");

  dim3 blockconfig_1 (32, 16, 1);
  dim3 gridconfig_1 (ceil(N, blockconfig_1.x-4), ceil(M, blockconfig_1.y-4), 1);
  heat <<<gridconfig_1, blockconfig_1>>> (in, out1, L, M, N);

  dim3 blockconfig_2 (32, 16, 1);
  dim3 gridconfig_2 (ceil(N, blockconfig_2.x-4), ceil(M, blockconfig_2.y-4), 1);
  heat <<<gridconfig_2, blockconfig_2>>> (out1, out2, L, M, N);

  hipMemcpy (h_out2, out2, sizeof(float)*L*M*N, hipMemcpyDeviceToHost);
}

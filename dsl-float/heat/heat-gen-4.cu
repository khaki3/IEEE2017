#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
  hipError_t error = hipGetLastError ();
  if (error != hipSuccess) {
    printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
    exit(-1);
  }
}

__global__ void heat (float * __restrict__ in, float * __restrict__ out1, float * __restrict__ out2, float * __restrict__ out3, float * __restrict__ out, int L, int M, int N) {
  //Determining the block's indices
  int blockdim_i= (int)(blockDim.x);
  int i0 = (int)(blockIdx.x)*(blockdim_i-8);
  int i = i0 + (int)(threadIdx.x);
  int blockdim_j= (int)(blockDim.y);
  int j0 = (int)(blockIdx.y)*(blockdim_j-8);
  int j = j0 + (int)(threadIdx.y);

  //Declarations
  float reg_in_m1=0, __shared__ sh_in_c0[32][32], reg_in_p1=0;
  float reg_out1_m2=0, __shared__ sh_out1_m1[32][32], reg_out1_c0=0;
  float reg_out2_m3=0, __shared__ sh_out2_m2[32][32], reg_out2_m1=0;
  float reg_out3_m4=0, __shared__ sh_out3_m3[32][32], reg_out3_m2=0;

  //Value Initialization
  if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
    reg_in_m1 = in[0 + j*N + i];
    sh_in_c0[j-j0][i-i0] = in[1*M*N + j*N + i];
  }

  //Rest of the computation
  #pragma unroll 2
  for (int k=1; k<=L-2; ++k) {
    //Fetch new plane
    if (j <= min (j0+blockdim_j-1, M-1) & i <= min (i0+blockdim_i-1, N-1)) {
        reg_in_p1 = in[(k+1)*M*N + j*N + i];
    }
    __syncthreads ();
    if (j >= j0+1 & j <= min (j0+blockdim_j-2, M-2) & i >= i0+1 & i <= min (i0+blockdim_i-2, N-2)) {
        reg_out1_c0 = ((((0.125f * ((reg_in_p1 - (2.0f * sh_in_c0[j-j0][i-i0])) + reg_in_m1)) + (0.125f * ((sh_in_c0[j-j0+1][i-i0] - (2.0f * sh_in_c0[j-j0][i-i0])) + sh_in_c0[j-j0-1][i-i0]))) + (0.125f * ((sh_in_c0[j-j0][i-i0+1] - (2.0f * sh_in_c0[j-j0][i-i0])) + sh_in_c0[j-j0][i-i0-1]))) + sh_in_c0[j-j0][i-i0]);
    }
    __syncthreads ();
    if (j >= j0+2 & j <= min (j0+blockdim_j-3, M-2) & i >= i0+2 & i <= min (i0+blockdim_i-3, N-2)) {
        reg_out2_m1 = ((((0.125f * ((reg_out1_c0 - (2.0f * sh_out1_m1[j-j0][i-i0])) + reg_out1_m2)) + (0.125f * ((sh_out1_m1[j-j0+1][i-i0] - (2.0f * sh_out1_m1[j-j0][i-i0])) + sh_out1_m1[j-j0-1][i-i0]))) + (0.125f * ((sh_out1_m1[j-j0][i-i0+1] - (2.0f * sh_out1_m1[j-j0][i-i0])) + sh_out1_m1[j-j0][i-i0-1]))) + sh_out1_m1[j-j0][i-i0]);
    }
    __syncthreads ();
    if (j >= j0+3 & j <= min (j0+blockdim_j-4, M-2) & i >= i0+3 & i <= min (i0+blockdim_i-4, N-2)) {
        reg_out3_m2 = ((((0.125f * ((reg_out2_m1 - (2.0f * sh_out2_m2[j-j0][i-i0])) + reg_out2_m3)) + (0.125f * ((sh_out2_m2[j-j0+1][i-i0] - (2.0f * sh_out2_m2[j-j0][i-i0])) + sh_out2_m2[j-j0-1][i-i0]))) + (0.125f * ((sh_out2_m2[j-j0][i-i0+1] - (2.0f * sh_out2_m2[j-j0][i-i0])) + sh_out2_m2[j-j0][i-i0-1]))) + sh_out2_m2[j-j0][i-i0]);
    }
    __syncthreads ();
    if (j >= j0+4 & j <= min (j0+blockdim_j-5, M-2) & i >= i0+4 & i <= min (i0+blockdim_i-5, N-2)) {
        out[max(k-3,0)*M*N + j*N + i] = ((((0.125f * ((reg_out3_m2 - (2.0f * sh_out3_m3[j-j0][i-i0])) + reg_out3_m4)) + (0.125f * ((sh_out3_m3[j-j0+1][i-i0] - (2.0f * sh_out3_m3[j-j0][i-i0])) + sh_out3_m3[j-j0-1][i-i0]))) + (0.125f * ((sh_out3_m3[j-j0][i-i0+1] - (2.0f * sh_out3_m3[j-j0][i-i0])) + sh_out3_m3[j-j0][i-i0-1]))) + sh_out3_m3[j-j0][i-i0]);
    }
    __syncthreads ();
    //Value rotation
    reg_in_m1 = sh_in_c0[j-j0][i-i0];
    sh_in_c0[j-j0][i-i0] = reg_in_p1;
    reg_out1_m2 = sh_out1_m1[j-j0][i-i0];
    sh_out1_m1[j-j0][i-i0] = reg_out1_c0;
    reg_out2_m3 = sh_out2_m2[j-j0][i-i0];
    sh_out2_m2[j-j0][i-i0] = reg_out2_m1;
    reg_out3_m4 = sh_out3_m3[j-j0][i-i0];
    sh_out3_m3[j-j0][i-i0] = reg_out3_m2;
  }
}

extern "C" void host_code (float *h_in, float *h_out, int L, int M, int N) {
  float *in;
  hipMalloc (&in, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for in\n");
  hipMemcpy (in, h_in, sizeof(float)*L*M*N, hipMemcpyHostToDevice);
  float *out;
  hipMalloc (&out, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for out\n");
  float *out1;
  hipMalloc (&out1, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for out1\n");
  float *out2;
  hipMalloc (&out2, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for out2\n");
  float *out3;
  hipMalloc (&out3, sizeof(float)*L*M*N);
  check_error ("Failed to allocate device memory for out3\n");

  dim3 blockconfig_1 (32, 32, 1);
  dim3 gridconfig_1 (ceil(N, blockconfig_1.x-8), ceil(M, blockconfig_1.y-8), 1);
  heat <<<gridconfig_1, blockconfig_1>>> (in, out1, out2, out3, out, L, M, N);

  hipMemcpy (h_out, out, sizeof(float)*L*M*N, hipMemcpyDeviceToHost);

  //Free allocated memory
  hipFree (in);
  hipFree (out);
  hipFree (out1);
  hipFree (out2);
  hipFree (out3);
}

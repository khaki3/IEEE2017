#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * input, int N, int M, float * __var_4__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_0__ <= (M-2)){
    int __iter_1__;
    __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_1__ <= (N-2)){
      float __temp_0__;
      __temp_0__ = (7 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(-1))]);
      float __temp_1__;
      __temp_1__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(-1))]);
      float __temp_2__;
      __temp_2__ = (__temp_0__ + __temp_1__);
      float __temp_3__;
      __temp_3__ = (9 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(-1))]);
      float __temp_4__;
      __temp_4__ = (__temp_2__ + __temp_3__);
      float __temp_5__;
      __temp_5__ = (12 * input[__iter_0__+(-1)+(M-0)*(__iter_1__)]);
      float __temp_6__;
      __temp_6__ = (__temp_4__ + __temp_5__);
      float __temp_7__;
      __temp_7__ = (15 * input[__iter_0__+(M-0)*(__iter_1__)]);
      float __temp_8__;
      __temp_8__ = (__temp_6__ + __temp_7__);
      float __temp_9__;
      __temp_9__ = (12 * input[__iter_0__+(1)+(M-0)*(__iter_1__)]);
      float __temp_10__;
      __temp_10__ = (__temp_8__ + __temp_9__);
      float __temp_11__;
      __temp_11__ = (9 * input[__iter_0__+(-1)+(M-0)*(__iter_1__+(1))]);
      float __temp_12__;
      __temp_12__ = (__temp_10__ + __temp_11__);
      float __temp_13__;
      __temp_13__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(1))]);
      float __temp_14__;
      __temp_14__ = (__temp_12__ + __temp_13__);
      float __temp_15__;
      __temp_15__ = (7 * input[__iter_0__+(1)+(M-0)*(__iter_1__+(1))]);
      float __temp_16__;
      __temp_16__ = (__temp_14__ + __temp_15__);
      float __temp_17__;
      __temp_17__ = (__temp_16__ / 118);
      __var_4__[__iter_0__+(M-0)*(__iter_1__)] = __temp_17__;
    }
  }
}
__global__ void __kernel___forma_kernel__1__(float * __var_4__, int N, int M, float * __var_3__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_2__ <= (M-2)){
    int __iter_3__;
    __iter_3__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_3__ <= (N-2)){
      float __temp_18__;
      __temp_18__ = (7 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(-1))]);
      float __temp_19__;
      __temp_19__ = (5 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(-1))]);
      float __temp_20__;
      __temp_20__ = (__temp_18__ + __temp_19__);
      float __temp_21__;
      __temp_21__ = (9 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(-1))]);
      float __temp_22__;
      __temp_22__ = (__temp_20__ + __temp_21__);
      float __temp_23__;
      __temp_23__ = (12 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__)]);
      float __temp_24__;
      __temp_24__ = (__temp_22__ + __temp_23__);
      float __temp_25__;
      __temp_25__ = (15 * __var_4__[__iter_2__+(M-0)*(__iter_3__)]);
      float __temp_26__;
      __temp_26__ = (__temp_24__ + __temp_25__);
      float __temp_27__;
      __temp_27__ = (12 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__)]);
      float __temp_28__;
      __temp_28__ = (__temp_26__ + __temp_27__);
      float __temp_29__;
      __temp_29__ = (9 * __var_4__[__iter_2__+(-1)+(M-0)*(__iter_3__+(1))]);
      float __temp_30__;
      __temp_30__ = (__temp_28__ + __temp_29__);
      float __temp_31__;
      __temp_31__ = (5 * __var_4__[__iter_2__+(M-0)*(__iter_3__+(1))]);
      float __temp_32__;
      __temp_32__ = (__temp_30__ + __temp_31__);
      float __temp_33__;
      __temp_33__ = (7 * __var_4__[__iter_2__+(1)+(M-0)*(__iter_3__+(1))]);
      float __temp_34__;
      __temp_34__ = (__temp_32__ + __temp_33__);
      float __temp_35__;
      __temp_35__ = (__temp_34__ / 118);
      __var_3__[__iter_2__+(M-0)*(__iter_3__)] = __temp_35__;
    }
  }
}
__global__ void __kernel___forma_kernel__2__(float * __var_3__, int N, int M, float * __var_2__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_4__;
  __iter_4__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_4__ <= (M-2)){
    int __iter_5__;
    __iter_5__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_5__ <= (N-2)){
      float __temp_36__;
      __temp_36__ = (7 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(-1))]);
      float __temp_37__;
      __temp_37__ = (5 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(-1))]);
      float __temp_38__;
      __temp_38__ = (__temp_36__ + __temp_37__);
      float __temp_39__;
      __temp_39__ = (9 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(-1))]);
      float __temp_40__;
      __temp_40__ = (__temp_38__ + __temp_39__);
      float __temp_41__;
      __temp_41__ = (12 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__)]);
      float __temp_42__;
      __temp_42__ = (__temp_40__ + __temp_41__);
      float __temp_43__;
      __temp_43__ = (15 * __var_3__[__iter_4__+(M-0)*(__iter_5__)]);
      float __temp_44__;
      __temp_44__ = (__temp_42__ + __temp_43__);
      float __temp_45__;
      __temp_45__ = (12 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__)]);
      float __temp_46__;
      __temp_46__ = (__temp_44__ + __temp_45__);
      float __temp_47__;
      __temp_47__ = (9 * __var_3__[__iter_4__+(-1)+(M-0)*(__iter_5__+(1))]);
      float __temp_48__;
      __temp_48__ = (__temp_46__ + __temp_47__);
      float __temp_49__;
      __temp_49__ = (5 * __var_3__[__iter_4__+(M-0)*(__iter_5__+(1))]);
      float __temp_50__;
      __temp_50__ = (__temp_48__ + __temp_49__);
      float __temp_51__;
      __temp_51__ = (7 * __var_3__[__iter_4__+(1)+(M-0)*(__iter_5__+(1))]);
      float __temp_52__;
      __temp_52__ = (__temp_50__ + __temp_51__);
      float __temp_53__;
      __temp_53__ = (__temp_52__ / 118);
      __var_2__[__iter_4__+(M-0)*(__iter_5__)] = __temp_53__;
    }
  }
}
__global__ void __kernel___forma_kernel__3__(float * __var_2__, int N, int M, float * __var_1__){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_6__;
  __iter_6__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_6__ <= (M-2)){
    int __iter_7__;
    __iter_7__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_7__ <= (N-2)){
      float __temp_54__;
      __temp_54__ = (7 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(-1))]);
      float __temp_55__;
      __temp_55__ = (5 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(-1))]);
      float __temp_56__;
      __temp_56__ = (__temp_54__ + __temp_55__);
      float __temp_57__;
      __temp_57__ = (9 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(-1))]);
      float __temp_58__;
      __temp_58__ = (__temp_56__ + __temp_57__);
      float __temp_59__;
      __temp_59__ = (12 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__)]);
      float __temp_60__;
      __temp_60__ = (__temp_58__ + __temp_59__);
      float __temp_61__;
      __temp_61__ = (15 * __var_2__[__iter_6__+(M-0)*(__iter_7__)]);
      float __temp_62__;
      __temp_62__ = (__temp_60__ + __temp_61__);
      float __temp_63__;
      __temp_63__ = (12 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__)]);
      float __temp_64__;
      __temp_64__ = (__temp_62__ + __temp_63__);
      float __temp_65__;
      __temp_65__ = (9 * __var_2__[__iter_6__+(-1)+(M-0)*(__iter_7__+(1))]);
      float __temp_66__;
      __temp_66__ = (__temp_64__ + __temp_65__);
      float __temp_67__;
      __temp_67__ = (5 * __var_2__[__iter_6__+(M-0)*(__iter_7__+(1))]);
      float __temp_68__;
      __temp_68__ = (__temp_66__ + __temp_67__);
      float __temp_69__;
      __temp_69__ = (7 * __var_2__[__iter_6__+(1)+(M-0)*(__iter_7__+(1))]);
      float __temp_70__;
      __temp_70__ = (__temp_68__ + __temp_69__);
      float __temp_71__;
      __temp_71__ = (__temp_70__ / 118);
      __var_1__[__iter_6__+(M-0)*(__iter_7__)] = __temp_71__;
    }
  }
}
/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
  float * __var_3__;
  hipMalloc(&__var_3__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_3__\n");
  float * __var_4__;
  hipMalloc(&__var_4__,sizeof(float)*((N-0)*(M-0)));
  Check_CUDA_Error("Allocation Error!! : __var_4__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 32;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_4__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_4__, N, M, __var_3__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_3__, N, M, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, N, M, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");


  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
  hipFree(__var_3__);
  hipFree(__var_4__);
}
/*Host Free End*/

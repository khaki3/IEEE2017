#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float * __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*FORMA_BLOCKDIM_X;
  float * __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*FORMA_BLOCKDIM_X;

  float t2=0.0f, t3=0.0f, out = 0.0f;
  float b2=0.0f, b3=0.0f;
  int __iter_0__ = (int)(blockIdx.x)*((int)FORMA_BLOCKDIM_X-4);

  // Initialize the values
  int __iter_3__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
    __tilevar_1__[__iter_3__-__iter_0__] = 0.0f;
  }
  // Rest of the computation
  for (int __iter_1__ = 0; __iter_1__ <= N-1; __iter_1__++) {
    if (__iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(M-1))) {
      __tilevar_0__[__iter_3__-__iter_0__] = input[__iter_3__+M*(__iter_1__)];
    }
    __syncthreads ();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+1),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(M-2))) {
	// Bottom
        float __temp_2__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        float __temp_5__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        float __temp_9__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        float __temp_10__ = (__temp_6__ + 9 * __temp_9__) / 118;
	t2 += __temp_10__;
	// Mid
        float __temp_13__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        float __temp_17__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        float __temp_18__ = (12 * __temp_13__ + 15 * __temp_17__);
        float __temp_21__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        float __temp_22__ = (__temp_18__ + 12 * __temp_21__) / 118;
	b2 += __temp_22__;
	// Top
        float __temp_25__ = (__tilevar_0__[__iter_3__-1-__iter_0__]);
        float __temp_29__ = (__tilevar_0__[__iter_3__-__iter_0__]);
        float __temp_30__ = (9 * __temp_25__ + 5 * __temp_29__);
        float __temp_33__ = (__tilevar_0__[__iter_3__+1-__iter_0__]);
        float __temp_34__ = (__temp_30__ + 7 * __temp_33__) / 118;
        __tilevar_1__[__iter_3__-__iter_0__] += __temp_34__;
    }
    __syncthreads();
    if(__iter_3__ >= FORMA_MAX((__iter_0__+2),1) & __iter_3__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(M-2))) {
	// Bottom
        float __temp_2__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        float __temp_5__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        float __temp_6__ = (7 * __temp_2__ + 5 * __temp_5__);
        float __temp_9__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        float __temp_10__ = (__temp_6__ + 9 * __temp_9__) / 118;
	t3 += __temp_10__;
	// Mid
        float __temp_13__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        float __temp_17__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        float __temp_18__ = (12 * __temp_13__ + 15 * __temp_17__);
        float __temp_21__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        float __temp_22__ = (__temp_18__ + 12 * __temp_21__) / 118;
	b3 += __temp_22__;
	// Top
        float __temp_25__ = (__tilevar_1__[__iter_3__-1-__iter_0__]);
        float __temp_29__ = (__tilevar_1__[__iter_3__-__iter_0__]);
        float __temp_30__ = (9 * __temp_25__ + 5 * __temp_29__);
        float __temp_33__ = (__tilevar_1__[__iter_3__+1-__iter_0__]);
        float __temp_34__ = (__temp_30__ + 7 * __temp_33__) / 118;
	out += __temp_34__;
        __var_1__[__iter_3__+M*FORMA_MAX(__iter_1__-2,0)] = out;
    }
    __syncthreads ();
    // Now rotate
    __tilevar_1__[__iter_3__-__iter_0__] = b2;
    b2 = t2;
    t2 = 0.0f;
    out= b3;
    b3 = t3;
    t3 = 0.0f;
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(2*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi(float * h_input, int N, int M, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((N)*(M)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((N)*(M)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 64;
  int __block_1___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-4);
  int __grid_1___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_2__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, N, M, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __var_1__);

  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N)*(M)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

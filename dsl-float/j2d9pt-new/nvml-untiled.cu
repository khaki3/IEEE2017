#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"
#include <nvml.h>
#include <assert.h>


#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int N, int M, float * __restrict__ __var_4__){
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_0__;
	__iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 2;
	if(__iter_0__ <= (M-3)){
		int __iter_1__;
		__iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 2;
		if(__iter_1__ <= (N-3)){
			float __temp_0__;
			__temp_0__ = (7 * input[__iter_0__+(M-0)*(__iter_1__+(-2))]);
			float __temp_1__;
			__temp_1__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(-1))]);
			float __temp_2__;
			__temp_2__ = (__temp_0__ + __temp_1__);
			float __temp_3__;
			__temp_3__ = (9 * input[__iter_0__+(-2)+(M-0)*(__iter_1__)]);
			float __temp_4__;
			__temp_4__ = (__temp_2__ + __temp_3__);
			float __temp_5__;
			__temp_5__ = (12 * input[__iter_0__+(-1)+(M-0)*(__iter_1__)]);
			float __temp_6__;
			__temp_6__ = (__temp_4__ + __temp_5__);
			float __temp_7__;
			__temp_7__ = (15 * input[__iter_0__+(M-0)*(__iter_1__)]);
			float __temp_8__;
			__temp_8__ = (__temp_6__ + __temp_7__);
			float __temp_9__;
			__temp_9__ = (12 * input[__iter_0__+(1)+(M-0)*(__iter_1__)]);
			float __temp_10__;
			__temp_10__ = (__temp_8__ + __temp_9__);
			float __temp_11__;
			__temp_11__ = (9 * input[__iter_0__+(2)+(M-0)*(__iter_1__)]);
			float __temp_12__;
			__temp_12__ = (__temp_10__ + __temp_11__);
			float __temp_13__;
			__temp_13__ = (5 * input[__iter_0__+(M-0)*(__iter_1__+(1))]);
			float __temp_14__;
			__temp_14__ = (__temp_12__ + __temp_13__);
			float __temp_15__;
			__temp_15__ = (7 * input[__iter_0__+(M-0)*(__iter_1__+(2))]);
			float __temp_16__;
			__temp_16__ = (__temp_14__ + __temp_15__);
			float __temp_17__;
			__temp_17__ = (__temp_16__ / 118);
			__var_4__[__iter_0__+(M-0)*(__iter_1__)] = __temp_17__;
		}
	}
}

/*Device code End */
/* Host Code Begin */
extern "C" void jacobi (float * h_input, int N, int M, float * __var_0__){

	/* Host allocation Begin */
	float * input;
	hipMalloc(&input,sizeof(float)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : input\n");
	hipPointerAttribute_t ptrAttrib_h_input;
	hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
	if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
		if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
			memcpy_kind_h_input = hipMemcpyDeviceToDevice;
	hipGetLastError();
	if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
		hipMemcpy(input,h_input,sizeof(float)*((N-0)*(M-0)), memcpy_kind_h_input);
	}
	float * __var_1__;
	hipMalloc(&__var_1__,sizeof(float)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : __var_1__\n");
	float * __var_2__;
	hipMalloc(&__var_2__,sizeof(float)*((N-0)*(M-0)));
	Check_CUDA_Error("Allocation Error!! : __var_2__\n");
	/*Host Allocation End */
	/* Kernel Launch Begin */
#ifdef _TIMER_
	hipEvent_t _forma_timer_start_,_forma_timer_stop_;
	hipEventCreate(&_forma_timer_start_);
	hipEventCreate(&_forma_timer_stop_);
	hipEventRecord(_forma_timer_start_,0);
#endif
	int __size_0___kernel___forma_kernel__0__ = ((M-3) - 2 ) + 1;
	int __size_1___kernel___forma_kernel__0__ = ((N-3) - 2 ) + 1;
	int __max_occupancy_blocksize___kernel___forma_kernel__0__;
	int _max_occupancy_gridsize___kernel___forma_kernel__0__;
	hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
	int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
	__max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
	int __block_0___kernel___forma_kernel__0__ = 32;
	int __block_1___kernel___forma_kernel__0__ = 32;
	dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
	int __SMemSize___kernel___forma_kernel__0__ = 0;
	int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
	int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
	dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<1000; x++) {
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, N, M, __var_2__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, N, M, __var_1__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_1__, N, M, __var_2__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, N, M, __var_1__);
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();



	Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
	hipPointerAttribute_t ptrAttrib___var_0__;
	hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
	if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
		if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
			memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
	hipGetLastError();
	hipMemcpy(__var_0__,__var_1__, sizeof(float)*((N-0)*(M-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
	hipEventRecord(_forma_timer_stop_,0);
	hipEventSynchronize(_forma_timer_stop_);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
	printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
	hipEventDestroy(_forma_timer_start_);
	hipEventDestroy(_forma_timer_stop_);
#endif
	/*Kernel Launch End */
	/* Host Free Begin */
	hipFree(input);
	hipFree(__var_1__);
	hipFree(__var_2__);
}
/*Host Free End*/

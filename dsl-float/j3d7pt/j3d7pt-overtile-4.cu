#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (4-(-4));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-8))*(FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-6))*(FORMA_BLOCKDIM_Y-(-6))*(FORMA_BLOCKDIM_X-(-6))));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-2))*(FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 1;
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 1;
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + 1;
  int __iter_3__;
  __iter_3__ = (FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)+(-1)),1)+(-1)) + (int)(threadIdx.z) ; 
  if( __iter_3__ <= (FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2))+1),(L-2))+1),(L-2))+1),(L-2))+1) ){
    int __iter_4__;
    __iter_4__ = (FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)+(-1)) + (int)(threadIdx.y) ; 
    if( __iter_4__ <= (FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2))+1),(M-2))+1),(M-2))+1),(M-2))+1) ){
      int __iter_5__;
      __iter_5__ = (FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)+(-1)) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= (FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2))+1),(N-2))+1),(N-2))+1),(N-2))+1) ){
        int __temp_0__;
        __temp_0__ = __iter_3__;
        int __temp_1__;
        __temp_1__ = __iter_4__;
        int __temp_2__;
        __temp_2__ = __iter_5__;
        __tilevar_3__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_3__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))))] = input[__temp_2__+(N-0)*(__temp_1__+(M-0)*(__temp_0__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__;
  __iter_6__ = FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)+(-1)),1) + (int)(threadIdx.z) ; 
  if( __iter_6__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2))+1),(L-2))+1),(L-2))+1),(L-2)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1) + (int)(threadIdx.y) ; 
    if( __iter_7__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2))+1),(M-2))+1),(M-2))+1),(M-2)) ){
      int __iter_8__;
      __iter_8__ = FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2))+1),(N-2))+1),(N-2))+1),(N-2)) ){
        int __temp_3__;
        __temp_3__ = __iter_6__;
        int __temp_4__;
        __temp_4__ = __iter_7__;
        int __temp_5__;
        __temp_5__ = __iter_8__+(1);
        float __temp_6__;
        __temp_6__ = (0.161000f * __tilevar_3__[__temp_5__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_4__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-8))*(__temp_3__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))))]);
        int __temp_7__;
        __temp_7__ = __iter_6__;
        int __temp_8__;
        __temp_8__ = __iter_7__;
        int __temp_9__;
        __temp_9__ = __iter_8__+(-1);
        float __temp_10__;
        __temp_10__ = (0.162000f * __tilevar_3__[__temp_9__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_8__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-8))*(__temp_7__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_11__;
        __temp_11__ = (__temp_6__ + __temp_10__);
        int __temp_12__;
        __temp_12__ = __iter_6__;
        int __temp_13__;
        __temp_13__ = __iter_7__+(1);
        int __temp_14__;
        __temp_14__ = __iter_8__;
        float __temp_15__;
        __temp_15__ = (0.163000f * __tilevar_3__[__temp_14__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_13__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-8))*(__temp_12__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_16__;
        __temp_16__ = (__temp_11__ + __temp_15__);
        int __temp_17__;
        __temp_17__ = __iter_6__;
        int __temp_18__;
        __temp_18__ = __iter_7__+(-1);
        int __temp_19__;
        __temp_19__ = __iter_8__;
        float __temp_20__;
        __temp_20__ = (0.164000f * __tilevar_3__[__temp_19__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_18__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-8))*(__temp_17__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_21__;
        __temp_21__ = (__temp_16__ + __temp_20__);
        int __temp_22__;
        __temp_22__ = __iter_6__+(1);
        int __temp_23__;
        __temp_23__ = __iter_7__;
        int __temp_24__;
        __temp_24__ = __iter_8__;
        float __temp_25__;
        __temp_25__ = (0.165000f * __tilevar_3__[__temp_24__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_23__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-8))*(__temp_22__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_26__;
        __temp_26__ = (__temp_21__ + __temp_25__);
        int __temp_27__;
        __temp_27__ = __iter_6__+(-1);
        int __temp_28__;
        __temp_28__ = __iter_7__;
        int __temp_29__;
        __temp_29__ = __iter_8__;
        float __temp_30__;
        __temp_30__ = (0.166000f * __tilevar_3__[__temp_29__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_28__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-8))*(__temp_27__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_31__;
        __temp_31__ = (__temp_26__ + __temp_30__);
        int __temp_32__;
        __temp_32__ = __iter_6__;
        int __temp_33__;
        __temp_33__ = __iter_7__;
        int __temp_34__;
        __temp_34__ = __iter_8__;
        float __temp_35__;
        __temp_35__ = (0.167000f * __tilevar_3__[__temp_34__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-8))*(__temp_33__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-8))*(__temp_32__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_36__;
        __temp_36__ = (__temp_31__ - __temp_35__);
        __tilevar_2__[__iter_8__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__iter_7__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-6))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)-1))))] = __temp_36__;
      }
    }
  }
  __syncthreads();
  int __iter_9__;
  __iter_9__ = FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1) + (int)(threadIdx.z) ; 
  if( __iter_9__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2))+1),(L-2))+1),(L-2)) ){
    int __iter_10__;
    __iter_10__ = FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1) + (int)(threadIdx.y) ; 
    if( __iter_10__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2))+1),(M-2))+1),(M-2)) ){
      int __iter_11__;
      __iter_11__ = FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN((FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2))+1),(N-2))+1),(N-2)) ){
        int __temp_37__;
        __temp_37__ = __iter_9__;
        int __temp_38__;
        __temp_38__ = __iter_10__;
        int __temp_39__;
        __temp_39__ = __iter_11__+(1);
        float __temp_40__;
        __temp_40__ = (0.161000f * __tilevar_2__[__temp_39__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_38__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-6))*(__temp_37__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)-1))))]);
        int __temp_41__;
        __temp_41__ = __iter_9__;
        int __temp_42__;
        __temp_42__ = __iter_10__;
        int __temp_43__;
        __temp_43__ = __iter_11__+(-1);
        float __temp_44__;
        __temp_44__ = (0.162000f * __tilevar_2__[__temp_43__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_42__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-6))*(__temp_41__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_45__;
        __temp_45__ = (__temp_40__ + __temp_44__);
        int __temp_46__;
        __temp_46__ = __iter_9__;
        int __temp_47__;
        __temp_47__ = __iter_10__+(1);
        int __temp_48__;
        __temp_48__ = __iter_11__;
        float __temp_49__;
        __temp_49__ = (0.163000f * __tilevar_2__[__temp_48__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_47__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-6))*(__temp_46__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_50__;
        __temp_50__ = (__temp_45__ + __temp_49__);
        int __temp_51__;
        __temp_51__ = __iter_9__;
        int __temp_52__;
        __temp_52__ = __iter_10__+(-1);
        int __temp_53__;
        __temp_53__ = __iter_11__;
        float __temp_54__;
        __temp_54__ = (0.164000f * __tilevar_2__[__temp_53__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_52__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-6))*(__temp_51__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_55__;
        __temp_55__ = (__temp_50__ + __temp_54__);
        int __temp_56__;
        __temp_56__ = __iter_9__+(1);
        int __temp_57__;
        __temp_57__ = __iter_10__;
        int __temp_58__;
        __temp_58__ = __iter_11__;
        float __temp_59__;
        __temp_59__ = (0.165000f * __tilevar_2__[__temp_58__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_57__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-6))*(__temp_56__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_60__;
        __temp_60__ = (__temp_55__ + __temp_59__);
        int __temp_61__;
        __temp_61__ = __iter_9__+(-1);
        int __temp_62__;
        __temp_62__ = __iter_10__;
        int __temp_63__;
        __temp_63__ = __iter_11__;
        float __temp_64__;
        __temp_64__ = (0.166000f * __tilevar_2__[__temp_63__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_62__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-6))*(__temp_61__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_65__;
        __temp_65__ = (__temp_60__ + __temp_64__);
        int __temp_66__;
        __temp_66__ = __iter_9__;
        int __temp_67__;
        __temp_67__ = __iter_10__;
        int __temp_68__;
        __temp_68__ = __iter_11__;
        float __temp_69__;
        __temp_69__ = (0.167000f * __tilevar_2__[__temp_68__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-6))*(__temp_67__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-6))*(__temp_66__+(0-(FORMA_MAX((FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)+(-1)),1)-1))))]);
        float __temp_70__;
        __temp_70__ = (__temp_65__ - __temp_69__);
        __tilevar_1__[__iter_11__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)-1))))] = __temp_70__;
      }
    }
  }
  __syncthreads();
  int __iter_12__;
  __iter_12__ = FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1) + (int)(threadIdx.z) ; 
  if( __iter_12__ <= FORMA_MIN((FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2))+1),(L-2)) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1) + (int)(threadIdx.y) ; 
    if( __iter_13__ <= FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2))+1),(M-2)) ){
      int __iter_14__;
      __iter_14__ = FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1) + (int)(threadIdx.x) ; 
      if( __iter_14__ <= FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2))+1),(N-2)) ){
        float __temp_71__;
        __temp_71__ = (0.161000f * __tilevar_1__[__iter_14__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)-1))))]);
        float __temp_72__;
        __temp_72__ = (0.162000f * __tilevar_1__[__iter_14__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)-1))))]);
        float __temp_73__;
        __temp_73__ = (__temp_71__ + __temp_72__);
        float __temp_74__;
        __temp_74__ = (0.163000f * __tilevar_1__[__iter_14__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_13__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)-1))))]);
        float __temp_75__;
        __temp_75__ = (__temp_73__ + __temp_74__);
        float __temp_76__;
        __temp_76__ = (0.164000f * __tilevar_1__[__iter_14__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_13__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)-1))))]);
        float __temp_77__;
        __temp_77__ = (__temp_75__ + __temp_76__);
        float __temp_78__;
        __temp_78__ = (0.165000f * __tilevar_1__[__iter_14__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_12__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)-1))))]);
        float __temp_79__;
        __temp_79__ = (__temp_77__ + __temp_78__);
        float __temp_80__;
        __temp_80__ = (0.166000f * __tilevar_1__[__iter_14__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_12__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)-1))))]);
        float __temp_81__;
        __temp_81__ = (__temp_79__ + __temp_80__);
        float __temp_82__;
        __temp_82__ = (0.167000f * __tilevar_1__[__iter_14__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_13__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_12__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,1)+(-1)),1)-1))))]);
        float __temp_83__;
        __temp_83__ = (__temp_81__ - __temp_82__);
        __tilevar_0__[__iter_14__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_13__+(0-(FORMA_MAX(__iter_1__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_12__+(0-(FORMA_MAX(__iter_2__,1)-1))))] = __temp_83__;
      }
    }
  }
  __syncthreads();
  int __iter_15__;
  __iter_15__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z) ; 
  if( __iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2)) ){
    int __iter_16__;
    __iter_16__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
    if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) ){
      int __iter_17__;
      __iter_17__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        float __temp_84__;
        __temp_84__ = (0.161000f * __tilevar_0__[__iter_17__+(1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_16__+(0-(FORMA_MAX(__iter_1__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_15__+(0-(FORMA_MAX(__iter_2__,1)-1))))]);
        float __temp_85__;
        __temp_85__ = (0.162000f * __tilevar_0__[__iter_17__+(-1)+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_16__+(0-(FORMA_MAX(__iter_1__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_15__+(0-(FORMA_MAX(__iter_2__,1)-1))))]);
        float __temp_86__;
        __temp_86__ = (__temp_84__ + __temp_85__);
        float __temp_87__;
        __temp_87__ = (0.163000f * __tilevar_0__[__iter_17__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_16__+(1)+(0-(FORMA_MAX(__iter_1__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_15__+(0-(FORMA_MAX(__iter_2__,1)-1))))]);
        float __temp_88__;
        __temp_88__ = (__temp_86__ + __temp_87__);
        float __temp_89__;
        __temp_89__ = (0.164000f * __tilevar_0__[__iter_17__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_16__+(-1)+(0-(FORMA_MAX(__iter_1__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_15__+(0-(FORMA_MAX(__iter_2__,1)-1))))]);
        float __temp_90__;
        __temp_90__ = (__temp_88__ + __temp_89__);
        float __temp_91__;
        __temp_91__ = (0.165000f * __tilevar_0__[__iter_17__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_16__+(0-(FORMA_MAX(__iter_1__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_15__+(1)+(0-(FORMA_MAX(__iter_2__,1)-1))))]);
        float __temp_92__;
        __temp_92__ = (__temp_90__ + __temp_91__);
        float __temp_93__;
        __temp_93__ = (0.166000f * __tilevar_0__[__iter_17__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_16__+(0-(FORMA_MAX(__iter_1__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_15__+(-1)+(0-(FORMA_MAX(__iter_2__,1)-1))))]);
        float __temp_94__;
        __temp_94__ = (__temp_92__ + __temp_93__);
        float __temp_95__;
        __temp_95__ = (0.167000f * __tilevar_0__[__iter_17__+(0-(FORMA_MAX(__iter_0__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_16__+(0-(FORMA_MAX(__iter_1__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_15__+(0-(FORMA_MAX(__iter_2__,1)-1))))]);
        float __temp_96__;
        __temp_96__ = (__temp_94__ - __temp_95__);
        __var_1__[__iter_17__+(N-0)*(__iter_16__+(M-0)*(__iter_15__))] = __temp_96__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (4-(-4));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-8))*(FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-6))*(FORMA_BLOCKDIM_Y-(-6))*(FORMA_BLOCKDIM_X-(-6))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-2))*(FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d7pt (float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel___forma_kernel__0__ = ((L-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___2 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_2___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___2,__size_2___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_2),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_2___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.z/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.z /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,9))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  __block_0___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.x-(4-(-4));
  __block_1___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.y-(4-(-4));
  __block_2___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.z-(4-(-4));
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
}
/*Host Free End*/

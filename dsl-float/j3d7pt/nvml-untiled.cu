#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"
#include <nvml.h>
#include <assert.h>

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
	template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
	int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
	input[loc] = value;
}


	template<typename T>
void initialize_array(T* d_input, int size, T value)
{
	dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
	dim3 init_block(FORMA_MAX_BLOCKDIM_0);
	__kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __var_4__){
	int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_0__;
	__iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
	if(__iter_0__ <= (N-2)){
		int __iter_1__;
		__iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
		if(__iter_1__ <= (M-2)){
			int __iter_2__;
			__iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
			if(__iter_2__ <= (L-2)){
				float __temp_0__;
				__temp_0__ = (0.161000f * input[__iter_0__+(1)+N*(__iter_1__+M*(__iter_2__))]);
				float __temp_1__;
				__temp_1__ = (0.162000f * input[__iter_0__+(-1)+N*(__iter_1__+M*(__iter_2__))]);
				float __temp_2__;
				__temp_2__ = (__temp_0__ + __temp_1__);
				float __temp_3__;
				__temp_3__ = (0.163000f * input[__iter_0__+N*(__iter_1__+(1)+M*(__iter_2__))]);
				float __temp_4__;
				__temp_4__ = (__temp_2__ + __temp_3__);
				float __temp_5__;
				__temp_5__ = (0.164000f * input[__iter_0__+N*(__iter_1__+(-1)+M*(__iter_2__))]);
				float __temp_6__;
				__temp_6__ = (__temp_4__ + __temp_5__);
				float __temp_7__;
				__temp_7__ = (0.165000f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__+(1)))]);
				float __temp_8__;
				__temp_8__ = (__temp_6__ + __temp_7__);
				float __temp_9__;
				__temp_9__ = (0.166000f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__+(-1)))]);
				float __temp_10__;
				__temp_10__ = (__temp_8__ + __temp_9__);
				float __temp_11__;
				__temp_11__ = (1.670000f * input[__iter_0__+N*(__iter_1__+M*(__iter_2__))]);
				float __temp_12__;
				__temp_12__ = (__temp_10__ - __temp_11__);
				__var_4__[__iter_0__+N*(__iter_1__+M*(__iter_2__))] = __temp_12__;
			}
		}
	}
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d7pt(float * h_input, int L, int M, int N, float * __var_0__){

	/* Host allocation Begin */
	float * input;
	hipMalloc(&input,sizeof(float)*(L*M*N));
	Check_CUDA_Error("Allocation Error!! : input\n");
	hipPointerAttribute_t ptrAttrib_h_input;
	hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
	if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
		if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
			memcpy_kind_h_input = hipMemcpyDeviceToDevice;
	hipGetLastError();
	if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
		hipMemcpy(input,h_input,sizeof(float)*(L*M*N), memcpy_kind_h_input);
	}
	float * __var_1__;
	hipMalloc(&__var_1__,sizeof(float)*(L*M*N));
	Check_CUDA_Error("Allocation Error!! : __var_1__\n");
	float * __var_2__;
	hipMalloc(&__var_2__,sizeof(float)*(L*M*N));
	Check_CUDA_Error("Allocation Error!! : __var_2__\n");
	/*Host Allocation End */
	/* Kernel Launch Begin */
#ifdef _TIMER_
	hipEvent_t _forma_timer_start_,_forma_timer_stop_;
	hipEventCreate(&_forma_timer_start_);
	hipEventCreate(&_forma_timer_stop_);
	hipEventRecord(_forma_timer_start_,0);
#endif
	int __size_0___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
	int __size_1___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
	int __size_2___kernel___forma_kernel__0__ = ((L-2) - 1 ) + 1;
	int __block_0___kernel___forma_kernel__0__ = 16;
	int __block_1___kernel___forma_kernel__0__ = 4;
	int __block_2___kernel___forma_kernel__0__ = 4;
	dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
	int __SMemSize___kernel___forma_kernel__0__ = 0;
	int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
	int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
	int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
	dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<500; x++) {
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __var_2__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, L, M, N, __var_1__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_1__, L, M, N, __var_2__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, L, M, N, __var_1__);
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();

	hipPointerAttribute_t ptrAttrib___var_0__;
	hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
	if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
		if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
			memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
	hipGetLastError();
	hipMemcpy(__var_0__,__var_1__, sizeof(float)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
	hipEventRecord(_forma_timer_stop_,0);
	hipEventSynchronize(_forma_timer_stop_);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
	printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
	hipEventDestroy(_forma_timer_start_);
	hipEventDestroy(_forma_timer_stop_);
#endif
	/*Kernel Launch End */
	/* Host Free Begin */
	hipFree(input);
	hipFree(__var_1__);
	hipFree(__var_2__);
}
/*Host Free End*/

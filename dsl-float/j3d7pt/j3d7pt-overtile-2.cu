#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */

__global__ void __kernel___forma_kernel__1__(float * __restrict__ __var_2__, int L, int M, int N, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (2-(-2));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-2))*(FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  int __iter_12__;
  __iter_12__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 1;
  int __iter_13__;
  __iter_13__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 1;
  int __iter_14__;
  __iter_14__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + 1;
  int __iter_15__;
  __iter_15__ = (FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)+(-1)) + (int)(threadIdx.z) ; 
  if( __iter_15__ <= (FORMA_MIN((FORMA_MIN(((__iter_14__+FORMA_BLOCKDIM_Z)-1),(L-2))+1),(L-2))+1) ){
    int __iter_16__;
    __iter_16__ = (FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)+(-1)) + (int)(threadIdx.y) ; 
    if( __iter_16__ <= (FORMA_MIN((FORMA_MIN(((__iter_13__+FORMA_BLOCKDIM_Y)-1),(M-2))+1),(M-2))+1) ){
      int __iter_17__;
      __iter_17__ = (FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)+(-1)) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= (FORMA_MIN((FORMA_MIN(((__iter_12__+FORMA_BLOCKDIM_X)-1),(N-2))+1),(N-2))+1) ){
        int __temp_29__;
        __temp_29__ = __iter_15__;
        int __temp_30__;
        __temp_30__ = __iter_16__;
        int __temp_31__;
        __temp_31__ = __iter_17__;
        __tilevar_3__[__iter_17__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_16__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_15__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)-1))))] = __var_2__[__temp_31__+(N-0)*(__temp_30__+(M-0)*(__temp_29__))];
      }
    }
  }
  __syncthreads();
  int __iter_18__;
  __iter_18__ = FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1) + (int)(threadIdx.z) ; 
  if( __iter_18__ <= FORMA_MIN((FORMA_MIN(((__iter_14__+FORMA_BLOCKDIM_Z)-1),(L-2))+1),(L-2)) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1) + (int)(threadIdx.y) ; 
    if( __iter_19__ <= FORMA_MIN((FORMA_MIN(((__iter_13__+FORMA_BLOCKDIM_Y)-1),(M-2))+1),(M-2)) ){
      int __iter_20__;
      __iter_20__ = FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN((FORMA_MIN(((__iter_12__+FORMA_BLOCKDIM_X)-1),(N-2))+1),(N-2)) ){
        float __temp_32__;
        __temp_32__ = (0.161000f * __tilevar_3__[__iter_20__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)-1))))]);
        float __temp_33__;
        __temp_33__ = (0.162000f * __tilevar_3__[__iter_20__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)-1))))]);
        float __temp_34__;
        __temp_34__ = (__temp_32__ + __temp_33__);
        float __temp_35__;
        __temp_35__ = (0.163000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_19__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)-1))))]);
        float __temp_36__;
        __temp_36__ = (__temp_34__ + __temp_35__);
        float __temp_37__;
        __temp_37__ = (0.164000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_19__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)-1))))]);
        float __temp_38__;
        __temp_38__ = (__temp_36__ + __temp_37__);
        float __temp_39__;
        __temp_39__ = (0.165000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_18__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)-1))))]);
        float __temp_40__;
        __temp_40__ = (__temp_38__ + __temp_39__);
        float __temp_41__;
        __temp_41__ = (0.166000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_18__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)-1))))]);
        float __temp_42__;
        __temp_42__ = (__temp_40__ + __temp_41__);
        float __temp_43__;
        __temp_43__ = (1.670000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_X-(-4))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,1)+(-1)),1)-1))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,1)+(-1)),1)-1))))]);
        float __temp_44__;
        __temp_44__ = (__temp_42__ - __temp_43__);
        __tilevar_2__[__iter_20__+(0-(FORMA_MAX(__iter_12__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_19__+(0-(FORMA_MAX(__iter_13__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_18__+(0-(FORMA_MAX(__iter_14__,1)-1))))] = __temp_44__;
      }
    }
  }
  __syncthreads();
  int __iter_21__;
  __iter_21__ = FORMA_MAX(__iter_14__,1) + (int)(threadIdx.z) ; 
  if( __iter_21__ <= FORMA_MIN(((__iter_14__+FORMA_BLOCKDIM_Z)-1),(L-2)) ){
    int __iter_22__;
    __iter_22__ = FORMA_MAX(__iter_13__,1) + (int)(threadIdx.y) ; 
    if( __iter_22__ <= FORMA_MIN(((__iter_13__+FORMA_BLOCKDIM_Y)-1),(M-2)) ){
      int __iter_23__;
      __iter_23__ = FORMA_MAX(__iter_12__,1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_12__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        float __temp_45__;
        __temp_45__ = (0.161000f * __tilevar_2__[__iter_23__+(1)+(0-(FORMA_MAX(__iter_12__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,1)-1))))]);
        float __temp_46__;
        __temp_46__ = (0.162000f * __tilevar_2__[__iter_23__+(-1)+(0-(FORMA_MAX(__iter_12__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,1)-1))))]);
        float __temp_47__;
        __temp_47__ = (__temp_45__ + __temp_46__);
        float __temp_48__;
        __temp_48__ = (0.163000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_22__+(1)+(0-(FORMA_MAX(__iter_13__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,1)-1))))]);
        float __temp_49__;
        __temp_49__ = (__temp_47__ + __temp_48__);
        float __temp_50__;
        __temp_50__ = (0.164000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_22__+(-1)+(0-(FORMA_MAX(__iter_13__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,1)-1))))]);
        float __temp_51__;
        __temp_51__ = (__temp_49__ + __temp_50__);
        float __temp_52__;
        __temp_52__ = (0.165000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_21__+(1)+(0-(FORMA_MAX(__iter_14__,1)-1))))]);
        float __temp_53__;
        __temp_53__ = (__temp_51__ + __temp_52__);
        float __temp_54__;
        __temp_54__ = (0.166000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_21__+(-1)+(0-(FORMA_MAX(__iter_14__,1)-1))))]);
        float __temp_55__;
        __temp_55__ = (__temp_53__ + __temp_54__);
        float __temp_56__;
        __temp_56__ = (1.670000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,1)-1))+(FORMA_BLOCKDIM_X-(-2))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,1)-1))+(FORMA_BLOCKDIM_Y-(-2))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,1)-1))))]);
        float __temp_57__;
        __temp_57__ = (__temp_55__ - __temp_56__);
        __var_1__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__))] = __temp_57__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__1__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (2-(-2));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (2-(-2));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (2-(-2));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-2))*(FORMA_BLOCKDIM_Y-(-2))*(FORMA_BLOCKDIM_X-(-2))));
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d7pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel___forma_kernel__0__ = ((L-2) - 1 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__1__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = 16;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),5);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  int __block_1___kernel___forma_kernel__0__ = 8;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),5);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___2 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_2___kernel___forma_kernel__0__ = 8;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___2,__size_2___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_2),5);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_2___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.z/2 > 5)
      __blockConfig___kernel___forma_kernel__0__.z /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 5)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,5))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__);
  }
  __block_0___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.x-(2-(-2));
  __block_1___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.y-(2-(-2));
  __block_2___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.z-(2-(-2));
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__1__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__1__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __var_2__);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__1__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, L, M, N, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

#define GAPX (22)
#define GAPY (22)
#define EXTENT (5)

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
/* X, Y, Z */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_4__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_5__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);

  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)-8);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY);
  float t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f;
  float b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;

  // Initialize the values
  int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
  int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
  if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1))) {
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
      __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = input[__iter_5__+N*(__iter_4__+M*(0))];
      t2 = input[__iter_5__+N*(__iter_4__+M*(1))];  
    }
  }
  // Rest of the computation
  for (int __iter_2__ = 1; __iter_2__ < L-1; __iter_2__++) {
    if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1))) {
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        b2 = __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
        __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t2; 
        t2 = input[__iter_5__+N*(__iter_4__+M*(__iter_2__+1))]; 
      }
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      if(__iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        float __temp_a3__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a7__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a8__ = (0.161f * __temp_a3__ + 0.162f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_a13__ = (__temp_a8__ + 0.163f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_a18__ = (__temp_a13__ + 0.164f * __temp_a17__);
        float __temp_a23__ = (__temp_a18__ + 0.165f * t2);
        float __temp_a28__ = (__temp_a23__ + 0.166f * b2);
        float __temp_a32__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a33__ = (__temp_a28__ - 1.670f * __temp_a32__);
	b3 = __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]; 
        __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t3;
        t3 = __temp_a33__;
      }
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__+1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))) & (__iter_4__ < (FORMA_MAX((__iter_1__+1),1)+2) | __iter_4__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2))) {
      __copy_arr_0__[__iter_5__+N*(__iter_4__+M*(__iter_2__))] = t3;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      if(__iter_5__ >= FORMA_MAX((__iter_0__+2),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        float __temp_a50__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a54__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a55__ = (0.161f * __temp_a50__ + 0.162f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_a60__ = (__temp_a55__ + 0.163f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_a65__ = (__temp_a60__ + 0.164f * __temp_a64__);
        float __temp_a70__ = (__temp_a65__ + 0.165f * t3);
        float __temp_a75__ = (__temp_a70__ + 0.166f * b3);
        float __temp_a79__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a80__ = (__temp_a75__ - 1.670f * __temp_a79__);
        b4 = __tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
        __tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t4;
        t4 = __temp_a80__;
      }
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__+2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__+2),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))) & (__iter_4__ < (FORMA_MAX((__iter_1__+2),1)+2) | __iter_4__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))-2))) {
      __copy_arr_1__[__iter_5__+N*(__iter_4__+M*(__iter_2__))] = t4;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+3),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      if( __iter_5__ >= FORMA_MAX((__iter_0__+3),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        float __temp_a94__ = (__tilevar_4__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a95__ = (__tilevar_4__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a96__ = (0.161f * __temp_a94__ + 0.162f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_a98__ = (__temp_a96__ + 0.163f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_a100__ = (__temp_a98__ + 0.164f * __temp_a99__);
        float __temp_a102__ = (__temp_a100__ + 0.165f * t4);
        float __temp_a104__ = (__temp_a102__ + 0.166f * b4);
        float __temp_a105__ = (__tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a106__ = (__temp_a104__ - 1.670f * __temp_a105__);
        b5 = __tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
        __tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t5;
        t5 = __temp_a106__;
      }
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__+3),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__+3),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))) & (__iter_4__ < (FORMA_MAX((__iter_1__+3),1)+2) | __iter_4__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))-2))) {
      __copy_arr_2__[__iter_5__+N*(__iter_4__+M*(__iter_2__))] = t5;
    }
    __syncthreads (); 
    if(__iter_4__ >= FORMA_MAX((__iter_1__+4),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2))) {
      if(__iter_5__ >= FORMA_MAX((__iter_0__+4),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
        float __temp_a120__ = (__tilevar_5__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a121__ = (__tilevar_5__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a122__ = (0.161f * __temp_a120__ + 0.162f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_a124__ = (__temp_a122__ + 0.163f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_a126__ = (__temp_a124__ + 0.164f * __temp_a125__);
        float __temp_a128__ = (__temp_a126__ + 0.165f * t5);
        float __temp_a130__ = (__temp_a128__ + 0.166f * b5);
        float __temp_a131__ = (__tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_a132__ = (__temp_a130__ - 1.670f * __temp_a131__);
        __var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-3,0))] = __temp_a132__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(4*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/* X, Y+GAP, Z */
__global__ void __kernel___forma_kernel__2__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_4__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_5__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);

  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY) + (int)(FORMA_BLOCKDIM_Y);
  float t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f;
  float b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;

  int __iter_4__ = FORMA_MAX(__iter_1__-EXTENT,0) + (int)(threadIdx.y) ;
  int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
  if(__iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1))) {
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
       __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = input[__iter_5__+N*(__iter_4__+M*(0))];
       t2 = input[__iter_5__+N*(__iter_4__+M*(1))]; 
    }
  }
  // Rest of the computation
  for (int __iter_2__ = 1; __iter_2__ < L-1; __iter_2__++) {
    if(__iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1))) {
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
	b2 = __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))];
	__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = t2;
	t2 = input[__iter_5__+N*(__iter_4__+M*(__iter_2__+1))];
      }
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__-1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2))) {
      if(__iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        float __temp_a3__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a7__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a8__ = (0.161f * __temp_a3__ + 0.162f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1+(EXTENT-__iter_1__))]);
        float __temp_a13__ = (__temp_a8__ + 0.163f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1+(EXTENT-__iter_1__))]);
        float __temp_a18__ = (__temp_a13__ + 0.164f * __temp_a17__);
        float __temp_a23__ = (__temp_a18__ + 0.165f * t2);
        float __temp_a28__ = (__temp_a23__ + 0.166f * b2);
        float __temp_a32__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a33__ = (__temp_a28__ - 1.670f * __temp_a32__);
	b3 = __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))];
	__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = t3;
	t3 = __temp_a33__;
      }
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__-1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))) & (__iter_5__ < (FORMA_MAX((__iter_0__+1),1)+2) | __iter_5__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))-2))) {
      __copy_arr_0__[__iter_5__+N*(__iter_4__+M*(__iter_2__))] = t3;
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__-3),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__+1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))) & (__iter_4__ < FORMA_MAX((__iter_1__-1),1) | __iter_4__ > FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)))) {
      b3 = __copy_arr_0__[__iter_5__+N*(__iter_4__+M*(__iter_2__-2))];
      __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = __copy_arr_0__[__iter_5__+N*(__iter_4__+M*(__iter_2__-1))];
      t3 = __copy_arr_0__[__iter_5__+N*(__iter_4__+M*(__iter_2__))];
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__-2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) {
      if(__iter_5__ >= FORMA_MAX((__iter_0__+2),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        float __temp_a50__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a54__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a55__ = (0.161f * __temp_a50__ + 0.162f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1+(EXTENT-__iter_1__))]);
        float __temp_a60__ = (__temp_a55__ + 0.163f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1+(EXTENT-__iter_1__))]);
        float __temp_a65__ = (__temp_a60__ + 0.164f * __temp_a64__);
        float __temp_a70__ = (__temp_a65__ + 0.165f * t3);
        float __temp_a75__ = (__temp_a70__ + 0.166f * b3);
        float __temp_a79__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a80__ = (__temp_a75__ - 1.670f * __temp_a79__);
	b4 = __tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))];
        __tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = t4;
	t4 = __temp_a80__;
      }
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__-2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__+2),1) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))) & (__iter_5__ < (FORMA_MAX((__iter_0__+2),1)+2) | __iter_5__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))-2))) {
      __copy_arr_1__[__iter_5__+N*(__iter_4__+M*(__iter_2__))] = t4;
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__-4),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__+2),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))) & (__iter_4__ < FORMA_MAX((__iter_1__-2),1) | __iter_4__ > FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2)))) {
      b4 = __copy_arr_1__[__iter_5__+N*(__iter_4__+M*(__iter_2__-2))];
      __tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = __copy_arr_1__[__iter_5__+N*(__iter_4__+M*(__iter_2__-1))];
      t4 = __copy_arr_1__[__iter_5__+N*(__iter_4__+M*(__iter_2__))];
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__-3),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) {
      if(__iter_5__ >= FORMA_MAX((__iter_0__+3),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        float __temp_a94__ = (__tilevar_4__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a95__ = (__tilevar_4__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a96__ = (0.161f * __temp_a94__ + 0.162f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1+(EXTENT-__iter_1__))]);
        float __temp_a98__ = (__temp_a96__ + 0.163f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1+(EXTENT-__iter_1__))]);
        float __temp_a100__ = (__temp_a98__ + 0.164f * __temp_a99__);
        float __temp_a102__ = (__temp_a100__ + 0.165f * t4);
        float __temp_a104__ = (__temp_a102__ + 0.166f * b4);
        float __temp_a105__ = (__tilevar_4__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a106__ = (__temp_a104__ - 1.670f * __temp_a105__);
	b5 = __tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))];
        __tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = t5;
	t5 = __temp_a106__;
      }
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__-3),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__+3),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))) & (__iter_5__ < (FORMA_MAX((__iter_0__+3),1)+2) | __iter_5__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))-2))) {
      __copy_arr_2__[__iter_5__+N*(__iter_4__+M*(__iter_2__))] = t5;
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__-5),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__+3),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))) & (__iter_4__ < FORMA_MAX((__iter_1__-3),1) | __iter_4__ > FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)))) {
     b5 = __copy_arr_2__[__iter_5__+N*(__iter_4__+M*(__iter_2__-2))];
      __tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = __copy_arr_2__[__iter_5__+N*(__iter_4__+M*(__iter_2__-1))];
     t5 = __copy_arr_2__[__iter_5__+N*(__iter_4__+M*(__iter_2__))];
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__-4),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2))) {
      if( __iter_5__ >= FORMA_MAX((__iter_0__+4),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
        float __temp_a120__ = (__tilevar_5__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a121__ = (__tilevar_5__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a122__ = (0.161f * __temp_a120__ + 0.162f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1+(EXTENT-__iter_1__))]);
        float __temp_a124__ = (__temp_a122__ + 0.163f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1+(EXTENT-__iter_1__))]);
        float __temp_a126__ = (__temp_a124__ + 0.164f * __temp_a125__);
        float __temp_a128__ = (__temp_a126__ + 0.165f * t5);
        float __temp_a130__ = (__temp_a128__ + 0.166f * b5);
        float __temp_a131__ = (__tilevar_5__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a132__ = (__temp_a130__ - 1.670f * __temp_a131__);
        __var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-3,0))] = __temp_a132__;
      }
    }
  }
}

/* X+GAP, Y+GAP, Z */
__global__ void __kernel___forma_kernel__3__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_4__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_5__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);

  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX) + (int)(FORMA_BLOCKDIM_X);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY) + (int)(FORMA_BLOCKDIM_Y);
  float t2=0.0f, t3=0.0f, t4=0.0f, t5=0.0f;
  float b2=0.0f, b3=0.0f, b4=0.0f, b5=0.0f;

  int __iter_4__ = FORMA_MAX(__iter_1__-EXTENT,0) + (int)(threadIdx.y) ;
  int __iter_5__ = FORMA_MAX(__iter_0__-EXTENT,0) + (int)(threadIdx.x) ;
  if(__iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1))) {
    if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-1)) ){
      __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = input[__iter_5__+N*(__iter_4__+M*(0))];
      t2 = input[__iter_5__+N*(__iter_4__+M*(1))];
    }
  }
  for (int __iter_2__ = 1; __iter_2__ < L-1; __iter_2__++) {
    if(__iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1))) {
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-1)) ){
        b2 = __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))];
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = t2;
        t2 = input[__iter_5__+N*(__iter_4__+M*(__iter_2__+1))];
      }
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__-1),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2))) {
      if(__iter_5__ >= FORMA_MAX((__iter_0__-1),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
        float __temp_a3__ = (__tilevar_2__[__iter_5__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a7__ = (__tilevar_2__[__iter_5__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a8__ = (0.161f * __temp_a3__ + 0.162f * __temp_a7__);
        float __temp_a12__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+1+(EXTENT-__iter_1__))]);
        float __temp_a13__ = (__temp_a8__ + 0.163f * __temp_a12__);
        float __temp_a17__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__-1+(EXTENT-__iter_1__))]);
        float __temp_a18__ = (__temp_a13__ + 0.164f * __temp_a17__);
        float __temp_a23__ = (__temp_a18__ + 0.165f * t2);
        float __temp_a28__ = (__temp_a23__ + 0.166f * b2);
        float __temp_a32__ = (__tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a33__ = (__temp_a28__ - 1.670f * __temp_a32__);
	b3 = __tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))];
        __tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = t3;
	t3 = __temp_a33__;
      }
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__-3),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__-3),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2))) & (__iter_4__ < FORMA_MAX((__iter_1__-1),1) | __iter_4__ > FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)) | __iter_5__ < FORMA_MAX((__iter_0__-1),1) | __iter_5__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)))) {
     b3 = __copy_arr_0__[__iter_5__+N*(__iter_4__+M*(__iter_2__-2))];
     __tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = __copy_arr_0__[__iter_5__+N*(__iter_4__+M*(__iter_2__-1))];
     t3 = __copy_arr_0__[__iter_5__+N*(__iter_4__+M*(__iter_2__))];
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__-2),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) {
      if(__iter_5__ >= FORMA_MAX((__iter_0__-2),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
        float __temp_a50__ = (__tilevar_3__[__iter_5__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a54__ = (__tilevar_3__[__iter_5__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a55__ = (0.161f * __temp_a50__ + 0.162f * __temp_a54__);
        float __temp_a59__ = (__tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+1+(EXTENT-__iter_1__))]);
        float __temp_a60__ = (__temp_a55__ + 0.163f * __temp_a59__);
        float __temp_a64__ = (__tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__-1+(EXTENT-__iter_1__))]);
        float __temp_a65__ = (__temp_a60__ + 0.164f * __temp_a64__);
        float __temp_a70__ = (__temp_a65__ + 0.165f * t3);
        float __temp_a75__ = (__temp_a70__ + 0.166f * b3);
        float __temp_a79__ = (__tilevar_3__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a80__ = (__temp_a75__ - 1.670f * __temp_a79__);
	b4 = __tilevar_4__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))];
        __tilevar_4__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = t4;
	t4 = __temp_a80__;
      }
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__-4),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__-4),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2))) & (__iter_4__ < (FORMA_MAX((__iter_1__-2),1)) | __iter_4__ > (FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) | __iter_5__ < FORMA_MAX((__iter_0__-2),1) | __iter_5__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)))) {
      b4 = __copy_arr_1__[__iter_5__+N*(__iter_4__+M*(__iter_2__-2))];
      __tilevar_4__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = __copy_arr_1__[__iter_5__+N*(__iter_4__+M*(__iter_2__-1))];
      t4 = __copy_arr_1__[__iter_5__+N*(__iter_4__+M*(__iter_2__))];
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__-3),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) {
      if( __iter_5__ >= FORMA_MAX((__iter_0__-3),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        float __temp_a94__ = (__tilevar_4__[__iter_5__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a95__ = (__tilevar_4__[__iter_5__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a96__ = (0.161f * __temp_a94__ + 0.162f * __temp_a95__);
        float __temp_a97__ = (__tilevar_4__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+1+(EXTENT-__iter_1__))]);
        float __temp_a98__ = (__temp_a96__ + 0.163f * __temp_a97__);
        float __temp_a99__ = (__tilevar_4__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__-1+(EXTENT-__iter_1__))]);
        float __temp_a100__ = (__temp_a98__ + 0.164f * __temp_a99__);
        float __temp_a102__ = (__temp_a100__ + 0.165f * t4);
        float __temp_a104__ = (__temp_a102__ + 0.166f * b4);
        float __temp_a105__ = (__tilevar_4__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a106__ = (__temp_a104__ - 1.670f * __temp_a105__);
	b5 = __tilevar_5__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))];
        __tilevar_5__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = t5;
	t5 = __temp_a106__;
      }
    }
    if ((__iter_4__ >= FORMA_MAX((__iter_1__-5),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(M-2))) & (__iter_5__ >= FORMA_MAX((__iter_0__-5),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(N-2))) & (__iter_4__ < (FORMA_MAX((__iter_1__-3),1)) | __iter_4__ > (FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) | __iter_5__ < FORMA_MAX((__iter_0__-3),1) | __iter_5__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)))) {
      b5 = __copy_arr_2__[__iter_5__+N*(__iter_4__+M*(__iter_2__-2))];
      __tilevar_5__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))] = __copy_arr_2__[__iter_5__+N*(__iter_4__+M*(__iter_2__-1))];
      t5 = __copy_arr_2__[__iter_5__+N*(__iter_4__+M*(__iter_2__))];
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__-4),1) & __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2))) {
      if(__iter_5__ >= FORMA_MAX((__iter_0__-4),1) & __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
        float __temp_a120__ = (__tilevar_5__[__iter_5__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a121__ = (__tilevar_5__[__iter_5__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a122__ = (0.161f * __temp_a120__ + 0.162f * __temp_a121__);
        float __temp_a123__ = (__tilevar_5__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+1+(EXTENT-__iter_1__))]);
        float __temp_a124__ = (__temp_a122__ + 0.163f * __temp_a123__);
        float __temp_a125__ = (__tilevar_5__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__-1+(EXTENT-__iter_1__))]);
        float __temp_a126__ = (__temp_a124__ + 0.164f * __temp_a125__);
        float __temp_a128__ = (__temp_a126__ + 0.165f * t5);
        float __temp_a130__ = (__temp_a128__ + 0.166f * b5);
        float __temp_a131__ = (__tilevar_5__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__))]);
        float __temp_a132__ = (__temp_a130__ - 1.670f * __temp_a131__);
        __var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-3,0))] = __temp_a132__;
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d7pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*(L*M*N), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = N;
  int __size_1___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 32;
  int __block_2___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0t__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-8);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y+GAPY);
  int __grid_2___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0t__(__grid_0___kernel___forma_kernel__0t__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 unrollConfig (__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0t__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");

  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

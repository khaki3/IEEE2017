#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )
#define mod(x,y) ( (x) & (y-1))

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

#define GAPX (22)
#define GAPY (2)
#define EXTENT (5)

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
/* X, Y, Z */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z+4)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z+4)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;

  int rowz = FORMA_BLOCKDIM_Z+4;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY);

  for (int __iter_2__ = 0; __iter_2__ <= L-1; __iter_2__ += FORMA_BLOCKDIM_Z) {
    int __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
    if(__iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1))) {
      int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
      for(; __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)); __iter_4__+=(int)(blockDim.y)) {
        int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
        if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
          __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_3__,rowz))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
        }
      }
    }
    __syncthreads();
    int __iter_6__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))) {
      int __iter_7__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ;
      for(; __iter_7__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)); __iter_7__+=(int)(blockDim.y)) {
        int __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
        if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
          float __temp_a3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a8__ = (0.161f * __temp_a3__ + 0.162f * __temp_a7__);
          float __temp_a12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a13__ = (__temp_a8__ + 0.163f * __temp_a12__);
          float __temp_a17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a18__ = (__temp_a13__ + 0.164f * __temp_a17__);
          float __temp_a22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__+1,rowz))]);
          float __temp_a23__ = (__temp_a18__ + 0.165f * __temp_a22__);
          float __temp_a27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__-1,rowz))]);
          float __temp_a28__ = (__temp_a23__ + 0.166f * __temp_a27__);
          float __temp_a32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a33__ = (__temp_a28__ - 1.670f * __temp_a32__);
          __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))] = __temp_a33__;
        }
      }
    }
    int __iter_9__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
    if(__iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))) {
      int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
      for(; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
        int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
        if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
          if (__iter_10__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_10__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2) || __iter_11__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_11__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))-2)) {
            __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))] = __tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_10__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_9__,rowz))];
          }
        }
      }
    }
    __syncthreads ();
    int __iter_15__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
    if(__iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))) {
      int __iter_16__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ;
      for(; __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)); __iter_16__+=(int)(blockDim.y)) {
        int __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
        if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
          float __temp_a50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a55__ = (0.161f * __temp_a50__ + 0.162f * __temp_a54__);
          float __temp_a59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a60__ = (__temp_a55__ + 0.163f * __temp_a59__);
          float __temp_a64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a65__ = (__temp_a60__ + 0.164f * __temp_a64__);
          float __temp_a69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__+1,rowz))]);
          float __temp_a70__ = (__temp_a65__ + 0.165f * __temp_a69__);
          float __temp_a74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__-1,rowz))]);
          float __temp_a75__ = (__temp_a70__ + 0.166f * __temp_a74__);
          float __temp_a79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a80__ = (__temp_a75__ - 1.670f * __temp_a79__);
          __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))] = __temp_a80__;
        }
      }
    }
    int __iter_18__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))) {
      int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
      for(; __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
        int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
        if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
          if (__iter_19__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))-2) || __iter_20__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_20__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))-2)) {
            __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))] = __tilevar_4__[__iter_20__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_19__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_18__,rowz))];
          }
        }
      }
    }
    __syncthreads ();
    int __iter_24__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
    if(__iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))) {
      int __iter_25__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ;
      for(; __iter_25__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)); __iter_25__+=(int)(blockDim.y)) {
        int __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
        if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
          float __temp_a94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a96__ = (0.161f * __temp_a94__ + 0.162f * __temp_a95__);
          float __temp_a97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a98__ = (__temp_a96__ + 0.163f * __temp_a97__);
          float __temp_a99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a100__ = (__temp_a98__ + 0.164f * __temp_a99__);
          float __temp_a101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__+1,rowz))]);
          float __temp_a102__ = (__temp_a100__ + 0.165f * __temp_a101__);
          float __temp_a103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__-1,rowz))]);
          float __temp_a104__ = (__temp_a102__ + 0.166f * __temp_a103__);
          float __temp_a105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a106__ = (__temp_a104__ - 1.670f * __temp_a105__);
          __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))] = __temp_a106__;
        }
      }
    }
    int __iter_27__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
    if(__iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))) {
      int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
      for(; __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
        int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
        if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
          if (__iter_28__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_28__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))-2) || __iter_29__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_29__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))-2)) {
            __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))] = __tilevar_5__[__iter_29__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_28__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_27__,rowz))];
          }
        }
      }
    }
    __syncthreads (); 
    int __iter_33__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
    if(__iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2))) {
      int __iter_34__ = FORMA_MAX((__iter_1__+4),1) + (int)(threadIdx.y) ;
      for (; __iter_34__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2)); __iter_34__+=(int)(blockDim.y)) {
        int __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
        if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
          float __temp_a120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a122__ = (0.161f * __temp_a120__ + 0.162f * __temp_a121__);
          float __temp_a123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a124__ = (__temp_a122__ + 0.163f * __temp_a123__);
          float __temp_a125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a126__ = (__temp_a124__ + 0.164f * __temp_a125__);
          float __temp_a127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__+1,rowz))]);
          float __temp_a128__ = (__temp_a126__ + 0.165f * __temp_a127__);
          float __temp_a129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__-1,rowz))]);
          float __temp_a130__ = (__temp_a128__ + 0.166f * __temp_a129__);
          float __temp_a131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a132__ = (__temp_a130__ - 1.670f * __temp_a131__);
          __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
        }
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(2*(FORMA_BLOCKDIM_Z+4)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/* X+GAP, Y, Z */
__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z+4)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z+4)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;

  int rowz = FORMA_BLOCKDIM_Z+4;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX) + (int)(FORMA_BLOCKDIM_X);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY);

  for (int __iter_2__ = 0; __iter_2__ <= L-1; __iter_2__ += FORMA_BLOCKDIM_Z) {
    int __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
    if(__iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1))) {
      int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
       for(; __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)); __iter_4__+=(int)(blockDim.y)) {
        int __iter_5__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
        if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-1)) ){
          __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_3__,rowz))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
        }
      }
    }
    __syncthreads();
    int __iter_6__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))) {
      int __iter_7__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ;
      for(; __iter_7__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)); __iter_7__+=(int)(blockDim.y)) {
        int __iter_8__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
        if( __iter_8__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
          float __temp_a3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a8__ = (0.161f * __temp_a3__ + 0.162f * __temp_a7__);
          float __temp_a12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a13__ = (__temp_a8__ + 0.163f * __temp_a12__);
          float __temp_a17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a18__ = (__temp_a13__ + 0.164f * __temp_a17__);
          float __temp_a22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__+1,rowz))]);
          float __temp_a23__ = (__temp_a18__ + 0.165f * __temp_a22__);
          float __temp_a27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__-1,rowz))]);
          float __temp_a28__ = (__temp_a23__ + 0.166f * __temp_a27__);
          float __temp_a32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod( __iter_6__,rowz))]);
          float __temp_a33__ = (__temp_a28__ - 1.670f * __temp_a32__);
          __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))] = __temp_a33__;
        }
      }
    }
    int __iter_9__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
    if(__iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))) {
      int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
      for(; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
        int __iter_11__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
        if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
          if (__iter_10__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_10__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2)) {
            __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))] = __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_10__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_9__,rowz))];
          }
        }
      }
    }
    __iter_9__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ;
    if(__iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))) {
      int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ;
      for(; __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
        int __iter_11__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ;
        if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
          if (__iter_11__ < FORMA_MAX((__iter_0__-1),1) || __iter_11__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2))) {
           __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_10__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_9__,rowz))] = __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))];
          }
        }
      }
    }
    __syncthreads();
    int __iter_15__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
    if(__iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))) {
      int __iter_16__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ;
      for (;__iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)); __iter_16__+=(int)(blockDim.y)) {
        int __iter_17__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
        if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
          float __temp_a50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a55__ = (0.161f * __temp_a50__ + 0.162f * __temp_a54__);
          float __temp_a59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a60__ = (__temp_a55__ + 0.163f * __temp_a59__);
          float __temp_a64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a65__ = (__temp_a60__ + 0.164f * __temp_a64__);
          float __temp_a69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__+1,rowz))]);
          float __temp_a70__ = (__temp_a65__ + 0.165f * __temp_a69__);
          float __temp_a74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__-1,rowz))]);
          float __temp_a75__ = (__temp_a70__ + 0.166f * __temp_a74__);
          float __temp_a79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a80__ = (__temp_a75__ - 1.670f * __temp_a79__);
          __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))] = __temp_a80__;
        }
      }
    }
    int __iter_18__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))) {
      int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
      for(; __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
        int __iter_20__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
        if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
          if (__iter_19__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))-2)) {
            __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))] = __tilevar_4__[__iter_20__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_19__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_18__,rowz))];
          }
        }
      }
    }
    __iter_18__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))) {
      int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
      for(; __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
        int __iter_20__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
        if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
          if (__iter_20__ < FORMA_MAX((__iter_0__-2),1) || __iter_20__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2))) {
            __tilevar_4__[__iter_20__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_19__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_18__,rowz))] = __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))];
          }
        }
      }
    }
    __syncthreads();
    int __iter_24__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
    if(__iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))) {
      int __iter_25__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ;
      for(; __iter_25__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)); __iter_25__+=(int)(blockDim.y)) {
        int __iter_26__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
        if( __iter_26__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
          float __temp_a94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a96__ = (0.161f * __temp_a94__ + 0.162f * __temp_a95__);
          float __temp_a97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a98__ = (__temp_a96__ + 0.163f * __temp_a97__);
          float __temp_a99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a100__ = (__temp_a98__ + 0.164f * __temp_a99__);
          float __temp_a101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__+1,rowz))]);
          float __temp_a102__ = (__temp_a100__ + 0.165f * __temp_a101__);
          float __temp_a103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__-1,rowz))]);
          float __temp_a104__ = (__temp_a102__ + 0.166f * __temp_a103__);
          float __temp_a105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a106__ = (__temp_a104__ - 1.670f * __temp_a105__);
          __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))] = __temp_a106__;
        }
      }
    }
    int __iter_27__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
    if(__iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))) {
      int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
      for(; __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
        int __iter_29__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
        if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
          if (__iter_28__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_28__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))-2)) {
            __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))] = __tilevar_5__[__iter_29__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_28__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_27__,rowz))];
          }
        }
      }
    }
    __iter_27__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
    if(__iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))) {
      int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
      for(; __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
        int __iter_29__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
        if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(N-2)) ){
          if (__iter_29__ < FORMA_MAX((__iter_0__-3),1) || __iter_29__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2))) {
            __tilevar_5__[__iter_29__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_28__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_27__,rowz))] = __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))];
          }
        }
      }
    }
    __syncthreads();
    int __iter_33__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
    if(__iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2))) {
      int __iter_34__ = FORMA_MAX((__iter_1__+4),1) + (int)(threadIdx.y) ;
      for(; __iter_34__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2)); __iter_34__+=(int)(blockDim.y)) {
        int __iter_35__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
        if( __iter_35__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
          float __temp_a120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a122__ = (0.161f * __temp_a120__ + 0.162f * __temp_a121__);
          float __temp_a123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a124__ = (__temp_a122__ + 0.163f * __temp_a123__);
          float __temp_a125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a126__ = (__temp_a124__ + 0.164f * __temp_a125__);
          float __temp_a127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__+1,rowz))]);
          float __temp_a128__ = (__temp_a126__ + 0.165f * __temp_a127__);
          float __temp_a129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__-1,rowz))]);
          float __temp_a130__ = (__temp_a128__ + 0.166f * __temp_a129__);
          float __temp_a131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-__iter_1__+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a132__ = (__temp_a130__ - 1.670f * __temp_a131__);
          __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
        }
      }
    }
  }
}

/* X, Y+GAP, Z */
__global__ void __kernel___forma_kernel__2__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z+4)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z+4)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;

  int rowz = FORMA_BLOCKDIM_Z+4;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY) + (int)(FORMA_BLOCKDIM_Y);

  for (int __iter_2__ = 0; __iter_2__ <= L-1; __iter_2__ += FORMA_BLOCKDIM_Z) {
    int __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
    if(__iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1))) {
      int __iter_4__ = FORMA_MAX(__iter_1__-2,0) + (int)(threadIdx.y) ;
      for(; __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1)); __iter_4__+=(int)(blockDim.y)) {
        int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
        if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
          __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_3__,rowz))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
        }
      }
    }
    __syncthreads();
    int __iter_6__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))) {
      int __iter_7__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ;
      for(; __iter_7__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)); __iter_7__+=(int)(blockDim.y)) {
        int __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
        if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
          float __temp_a3__ = (__tilevar_2__[__iter_8__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a7__ = (__tilevar_2__[__iter_8__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a8__ = (0.161f * __temp_a3__ + 0.162f * __temp_a7__);
          float __temp_a12__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a13__ = (__temp_a8__ + 0.163f * __temp_a12__);
          float __temp_a17__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a18__ = (__temp_a13__ + 0.164f * __temp_a17__);
          float __temp_a22__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__+1,rowz))]);
          float __temp_a23__ = (__temp_a18__ + 0.165f * __temp_a22__);
          float __temp_a27__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__-1,rowz))]);
          float __temp_a28__ = (__temp_a23__ + 0.166f * __temp_a27__);
          float __temp_a32__ = (__tilevar_2__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a33__ = (__temp_a28__ - 1.670f * __temp_a32__);
          __tilevar_3__[__iter_8__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))] = __temp_a33__;
        }
      }
    }
    int __iter_9__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
    if(__iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))) {
      int __iter_10__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
      for(; __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
        int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
        if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
          if (__iter_11__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_11__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))-2)) {
            __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))] = __tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_10__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_9__,rowz))];
          }
        }
      }
    }
    __iter_9__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
    if(__iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))) {
      int __iter_10__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
      for(; __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
        int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
        if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
          if (__iter_10__ < FORMA_MAX((__iter_1__-1),1) || __iter_10__ > FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2))) {
            __tilevar_3__[__iter_11__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_10__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_9__,rowz))] = __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))];
          }
        }
      }
    }
    __syncthreads();
    int __iter_15__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
    if(__iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))) {
      int __iter_16__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ;
      for(; __iter_16__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2)); __iter_16__+=(int)(blockDim.y)) {
        int __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
        if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
          float __temp_a50__ = (__tilevar_3__[__iter_17__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a54__ = (__tilevar_3__[__iter_17__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a55__ = (0.161f * __temp_a50__ + 0.162f * __temp_a54__);
          float __temp_a59__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a60__ = (__temp_a55__ + 0.163f * __temp_a59__);
          float __temp_a64__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a65__ = (__temp_a60__ + 0.164f * __temp_a64__);
          float __temp_a69__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__+1,rowz))]);
          float __temp_a70__ = (__temp_a65__ + 0.165f * __temp_a69__);
          float __temp_a74__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__-1,rowz))]);
          float __temp_a75__ = (__temp_a70__ + 0.166f * __temp_a74__);
          float __temp_a79__ = (__tilevar_3__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a80__ = (__temp_a75__ - 1.670f * __temp_a79__);
          __tilevar_4__[__iter_17__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))] = __temp_a80__;
        }
      }
    }
    int __iter_18__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))) {
      int __iter_19__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ; 
      for(; __iter_19__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
        int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
        if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
          if (__iter_20__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_20__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))-2)) {
            __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))] = __tilevar_4__[__iter_20__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_19__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_18__,rowz))];
          }
        }
      }
    }
    __iter_18__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))) {
      int __iter_19__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
      for(; __iter_19__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
        int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
        if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
          if (__iter_19__ < FORMA_MAX((__iter_1__-2),1) || __iter_19__ > FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) {
            __tilevar_4__[__iter_20__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_19__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_18__,rowz))] = __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))];
          }
        }
      }
    }
    __syncthreads();
    int __iter_24__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
    if(__iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))) {
      int __iter_25__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ;
      for (; __iter_25__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)); __iter_25__+=(int)(blockDim.y)) {
        int __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
        if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
          float __temp_a94__ = (__tilevar_4__[__iter_26__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a95__ = (__tilevar_4__[__iter_26__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a96__ = (0.161f * __temp_a94__ + 0.162f * __temp_a95__);
          float __temp_a97__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a98__ = (__temp_a96__ + 0.163f * __temp_a97__);
          float __temp_a99__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a100__ = (__temp_a98__ + 0.164f * __temp_a99__);
          float __temp_a101__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__+1,rowz))]);
          float __temp_a102__ = (__temp_a100__ + 0.165f * __temp_a101__);
          float __temp_a103__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__-1,rowz))]);
          float __temp_a104__ = (__temp_a102__ + 0.166f * __temp_a103__);
          float __temp_a105__ = (__tilevar_4__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a106__ = (__temp_a104__ - 1.670f * __temp_a105__);
          __tilevar_5__[__iter_26__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))] = __temp_a106__;
        }
      }
    }
    int __iter_27__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
    if(__iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))) {
      int __iter_28__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
      for(; __iter_28__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
        int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
        if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
          if (__iter_29__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_29__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))-2)) {
            __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))] = __tilevar_5__[__iter_29__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_28__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_27__,rowz))];
          }
        }
      }
    }
    __iter_27__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
    if(__iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))) {
      int __iter_28__ = FORMA_MAX((__iter_1__-5),1) + (int)(threadIdx.y) ; 
      for(; __iter_28__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
        int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
        if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
          if (__iter_28__ < FORMA_MAX((__iter_1__-3),1) || __iter_28__ > FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) {
            __tilevar_5__[__iter_29__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_28__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_27__,rowz))] = __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))];
          }
        }
      }
    }
    __syncthreads();
    int __iter_33__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
    if(__iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2))) {
      int __iter_34__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ;
      for(; __iter_34__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)); __iter_34__+=(int)(blockDim.y)) {
        int __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
        if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
          float __temp_a120__ = (__tilevar_5__[__iter_35__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a121__ = (__tilevar_5__[__iter_35__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a122__ = (0.161f * __temp_a120__ + 0.162f * __temp_a121__);
          float __temp_a123__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a124__ = (__temp_a122__ + 0.163f * __temp_a123__);
          float __temp_a125__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a126__ = (__temp_a124__ + 0.164f * __temp_a125__);
          float __temp_a127__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__+1,rowz))]);
          float __temp_a128__ = (__temp_a126__ + 0.165f * __temp_a127__);
          float __temp_a129__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__-1,rowz))]);
          float __temp_a130__ = (__temp_a128__ + 0.166f * __temp_a129__);
          float __temp_a131__ = (__tilevar_5__[__iter_35__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a132__ = (__temp_a130__ - 1.670f * __temp_a131__);
          __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
        }
      }
    }
  }
}

/* X+GAP, Y+GAP, Z */
__global__ void __kernel___forma_kernel__3__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z+4)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z+4)*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;

  int rowz = FORMA_BLOCKDIM_Z+4;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX) + (int)(FORMA_BLOCKDIM_X);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)+GAPY) + (int)(FORMA_BLOCKDIM_Y);

  for (int __iter_2__ = 0; __iter_2__ <= L-1; __iter_2__ += FORMA_BLOCKDIM_Z) {
    int __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
    if(__iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1))) {
      int __iter_4__ = FORMA_MAX(__iter_1__-2,0) + (int)(threadIdx.y) ;
      for(; __iter_4__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-1)); __iter_4__+=(int)(blockDim.y)) {
        int __iter_5__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
        if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-1)) ){
          __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_4__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_3__,rowz))] = input[__iter_5__+N*(__iter_4__+M*(__iter_3__))];
        }
      }
    }
    __syncthreads();
    int __iter_6__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ; 
    if(__iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))) {
      int __iter_7__ = FORMA_MAX((__iter_1__-1),1) + (int)(threadIdx.y) ; 
      for(; __iter_7__ <= FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)); __iter_7__+=(int)(blockDim.y)) {
        int __iter_8__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
        if( __iter_8__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
          float __temp_a3__ = (__tilevar_2__[__iter_8__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a7__ = (__tilevar_2__[__iter_8__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a8__ = (0.161f * __temp_a3__ + 0.162f * __temp_a7__);
          float __temp_a12__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a13__ = (__temp_a8__ + 0.163f * __temp_a12__);
          float __temp_a17__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a18__ = (__temp_a13__ + 0.164f * __temp_a17__);
          float __temp_a22__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__+1,rowz))]);
          float __temp_a23__ = (__temp_a18__ + 0.165f * __temp_a22__);
          float __temp_a27__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__-1,rowz))]);
          float __temp_a28__ = (__temp_a23__ + 0.166f * __temp_a27__);
          float __temp_a32__ = (__tilevar_2__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))]);
          float __temp_a33__ = (__temp_a28__ - 1.670f * __temp_a32__);
          __tilevar_3__[__iter_8__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_7__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_6__,rowz))] = __temp_a33__;
        }
      }
    }
    int __iter_9__ = FORMA_MAX((__iter_2__-1),1) + (int)(threadIdx.z) ;
    if(__iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))) {
      int __iter_10__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ;
      for(; __iter_10__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)) ; __iter_10__ += (int)(blockDim.y)){
        int __iter_11__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ;
        if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
          if (__iter_10__ < FORMA_MAX((__iter_1__-1),1) || __iter_10__ > FORMA_MIN(((__iter_1__+GAPY+1)-1),(M-2)) || __iter_11__ < FORMA_MAX((__iter_0__-1),1) || __iter_11__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2))) {
           __tilevar_3__[__iter_11__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_10__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_9__,rowz))] = __copy_arr_0__[__iter_11__+N*(__iter_10__+M*(__iter_9__))];
          }
        }
      }
    }
   __syncthreads();
    int __iter_15__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
    if(__iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))) {
      int __iter_16__ = FORMA_MAX((__iter_1__-2),1) + (int)(threadIdx.y) ;
      for(; __iter_16__ <= FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2)); __iter_16__+=(int)(blockDim.y)) {
        int __iter_17__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
        if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
          float __temp_a50__ = (__tilevar_3__[__iter_17__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a54__ = (__tilevar_3__[__iter_17__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a55__ = (0.161f * __temp_a50__ + 0.162f * __temp_a54__);
          float __temp_a59__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a60__ = (__temp_a55__ + 0.163f * __temp_a59__);
          float __temp_a64__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a65__ = (__temp_a60__ + 0.164f * __temp_a64__);
          float __temp_a69__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__+1,rowz))]);
          float __temp_a70__ = (__temp_a65__ + 0.165f * __temp_a69__);
          float __temp_a74__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__-1,rowz))]);
          float __temp_a75__ = (__temp_a70__ + 0.166f * __temp_a74__);
          float __temp_a79__ = (__tilevar_3__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))]);
          float __temp_a80__ = (__temp_a75__ - 1.670f * __temp_a79__);
          __tilevar_4__[__iter_17__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_16__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_15__,rowz))] = __temp_a80__;
        }
      }
    }
    int __iter_18__ = FORMA_MAX((__iter_2__-2),1) + (int)(threadIdx.z) ; 
    if(__iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))) {
      int __iter_19__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
      for(; __iter_19__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)) ; __iter_19__ += (int)(blockDim.y) ){
        int __iter_20__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
        if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
          if (__iter_19__ < (FORMA_MAX((__iter_1__-2),1)) || __iter_19__ > (FORMA_MIN(((__iter_1__+GAPY+2)-1),(M-2))) || __iter_20__ < FORMA_MAX((__iter_0__-2),1) || __iter_20__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2))) {
            __tilevar_4__[__iter_20__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_19__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_18__,rowz))] = __copy_arr_1__[__iter_20__+N*(__iter_19__+M*(__iter_18__))];
          }
        }
      }
    }
    __syncthreads();
    int __iter_24__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
    if(__iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))) {
      int __iter_25__ = FORMA_MAX((__iter_1__-3),1) + (int)(threadIdx.y) ; 
      for(; __iter_25__ <= FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2)); __iter_25__+=(int)(blockDim.y)) {
        int __iter_26__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
        if( __iter_26__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
          float __temp_a94__ = (__tilevar_4__[__iter_26__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a95__ = (__tilevar_4__[__iter_26__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a96__ = (0.161f * __temp_a94__ + 0.162f * __temp_a95__);
          float __temp_a97__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a98__ = (__temp_a96__ + 0.163f * __temp_a97__);
          float __temp_a99__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a100__ = (__temp_a98__ + 0.164f * __temp_a99__);
          float __temp_a101__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__+1,rowz))]);
          float __temp_a102__ = (__temp_a100__ + 0.165f * __temp_a101__);
          float __temp_a103__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__-1,rowz))]);
          float __temp_a104__ = (__temp_a102__ + 0.166f * __temp_a103__);
          float __temp_a105__ = (__tilevar_4__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))]);
          float __temp_a106__ = (__temp_a104__ - 1.670f * __temp_a105__);
          __tilevar_5__[__iter_26__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_25__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_24__,rowz))] = __temp_a106__;
        }
      }
    }
    int __iter_27__ = FORMA_MAX((__iter_2__-3),1) + (int)(threadIdx.z) ; 
    if(__iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))) {
      int __iter_28__ = FORMA_MAX((__iter_1__-5),1) + (int)(threadIdx.y) ; 
      for(; __iter_28__ <= FORMA_MIN(((__iter_1__+GAPY+5)-1),(M-2)) ; __iter_28__ += (int)(blockDim.y)){
        int __iter_29__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
        if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(N-2)) ){
          if (__iter_28__ < (FORMA_MAX((__iter_1__-3),1)) || __iter_28__ > (FORMA_MIN(((__iter_1__+GAPY+3)-1),(M-2))) || __iter_29__ < FORMA_MAX((__iter_0__-3),1) || __iter_29__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2))) {
            __tilevar_5__[__iter_29__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_28__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_27__,rowz))] = __copy_arr_2__[__iter_29__+N*(__iter_28__+M*(__iter_27__))];
          }
        }
      }
    }
    __syncthreads();
    int __iter_33__ = FORMA_MAX((__iter_2__-4),1) + (int)(threadIdx.z) ; 
    if(__iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2))) {
      int __iter_34__ = FORMA_MAX((__iter_1__-4),1) + (int)(threadIdx.y) ; 
      for(; __iter_34__ <= FORMA_MIN(((__iter_1__+GAPY+4)-1),(M-2)); __iter_34__+=(int)(blockDim.y)) {
        int __iter_35__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
        if( __iter_35__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
          float __temp_a120__ = (__tilevar_5__[__iter_35__+1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a121__ = (__tilevar_5__[__iter_35__-1+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a122__ = (0.161f * __temp_a120__ + 0.162f * __temp_a121__);
          float __temp_a123__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a124__ = (__temp_a122__ + 0.163f * __temp_a123__);
          float __temp_a125__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__-1+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a126__ = (__temp_a124__ + 0.164f * __temp_a125__);
          float __temp_a127__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__+1,rowz))]);
          float __temp_a128__ = (__temp_a126__ + 0.165f * __temp_a127__);
          float __temp_a129__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__-1,rowz))]);
          float __temp_a130__ = (__temp_a128__ + 0.166f * __temp_a129__);
          float __temp_a131__ = (__tilevar_5__[__iter_35__+(EXTENT-__iter_0__)+FORMA_BLOCKDIM_X*(__iter_34__+(EXTENT-__iter_1__)+FORMA_BLOCKDIM_Y*mod(__iter_33__,rowz))]);
          float __temp_a132__ = (__temp_a130__ - 1.670f * __temp_a131__);
          __var_1__[__iter_35__+N*(__iter_34__+M*(__iter_33__))] = __temp_a132__;
        }
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d7pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*(L*M*N), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = N;
  int __size_1___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 12;
  int __block_2___kernel___forma_kernel__0__ = 4;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y+GAPY);
  int __grid_2___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 unrollConfig (__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/3, __blockConfig___kernel___forma_kernel__0__.z);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");

  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
#define GAPX (22)
#define EXTENT (10)

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y));
  int __iter_2__ = (int)(blockIdx.z)*((int)(FORMA_BLOCKDIM_Z));

  int __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ;
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1)) ; __iter_3__+=(int)(blockDim.z) ){
    int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
    if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1))) {
      int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ;
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        __tilevar_2__[__iter_5__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-__iter_1__)+(FORMA_BLOCKDIM_Y-0)*(__iter_3__+(0-__iter_2__)))] = input[__iter_5__+(N-0)*(__iter_4__+(M-0)*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ;
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_6__+=(int)(blockDim.z) ){
    int __iter_7__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ;
    if(__iter_7__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      int __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ;
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        float __temp_3__ = (__tilevar_2__[__iter_8__+(1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+0))))]);
        float __temp_7__ = (__tilevar_2__[__iter_8__+(-1)+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+0))))]);
        float __temp_8__ = (0.161000f * __temp_3__ + 0.162000f * __temp_7__);
        float __temp_12__ = (__tilevar_2__[__iter_8__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(1)+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+0))))]);
        float __temp_13__ = (__temp_8__ + 0.163000f * __temp_12__);
        float __temp_17__ = (__tilevar_2__[__iter_8__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(-1)+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+0))))]);
        float __temp_18__ = (__temp_13__ + 0.164000f * __temp_17__);
        float __temp_22__ = (__tilevar_2__[__iter_8__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(1)+(0-(__iter_2__+0))))]);
        float __temp_23__ = (__temp_18__ + 0.165000f * __temp_22__);
        float __temp_27__ = (__tilevar_2__[__iter_8__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(-1)+(0-(__iter_2__+0))))]);
        float __temp_28__ = (__temp_23__ + 0.166000f * __temp_27__);
        float __temp_32__ = (__tilevar_2__[__iter_8__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*( __iter_6__+(0-(__iter_2__+0))))]);
        float __temp_33__ = (__temp_28__ - 1.670000f * __temp_32__);
        __tilevar_3__[__iter_8__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+1))))] = __temp_33__;
      }
    }
  }
  __syncthreads ();
  int __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ;
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ;
    if(__iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      int __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ;
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__+1),1)+2) || __iter_9__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))-2) || __iter_10__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_10__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2) || __iter_11__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_11__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))-2)) {
          __copy_arr_0__[__iter_11__+(N-0)*(__iter_10__+(M-0)*(__iter_9__))] = __tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_9__+(0-(__iter_2__+1))))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_15__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ;
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ;
    if(__iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      int __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ;
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        float __temp_50__ = (__tilevar_3__[__iter_17__+(1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+1))))]);
        float __temp_54__ = (__tilevar_3__[__iter_17__+(-1)+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+1))))]);
        float __temp_55__ = (0.161000f * __temp_50__ + 0.162000f * __temp_54__);
        float __temp_59__ = (__tilevar_3__[__iter_17__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+1))))]);
        float __temp_60__ = (__temp_55__ + 0.163000f * __temp_59__);
        float __temp_64__ = (__tilevar_3__[__iter_17__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+1))))]);
        float __temp_65__ = (__temp_60__ + 0.164000f * __temp_64__);
        float __temp_69__ = (__tilevar_3__[__iter_17__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(1)+(0-(__iter_2__+1))))]);
        float __temp_70__ = (__temp_65__ + 0.165000f * __temp_69__);
        float __temp_74__ = (__tilevar_3__[__iter_17__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(-1)+(0-(__iter_2__+1))))]);
        float __temp_75__ = (__temp_70__ + 0.166000f * __temp_74__);
        float __temp_79__ = (__tilevar_3__[__iter_17__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+1))))]);
        float __temp_80__ = (__temp_75__ - 1.670000f * __temp_79__);
        __tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+2))))] = __temp_80__;
      }
    }
  }
  __syncthreads ();
  int __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ;
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ;
    if(__iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      int __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ;
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__+2),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))-2) || __iter_19__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))-2) || __iter_20__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_20__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))-2)) {
          __copy_arr_1__[__iter_20__+(N-0)*(__iter_19__+(M-0)*(__iter_18__))] = __tilevar_4__[__iter_20__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_19__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_18__+(0-(__iter_2__+2))))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ;
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ;
    if(__iter_25__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      int __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ;
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        float __temp_94__ = (__tilevar_4__[__iter_26__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+2))))]);
        float __temp_95__ = (__tilevar_4__[__iter_26__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+2))))]);
        float __temp_96__ = (0.161000f * __temp_94__ + 0.162000f * __temp_95__);
        float __temp_97__ = (__tilevar_4__[__iter_26__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(1)+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+2))))]);
        float __temp_98__ = (__temp_96__ + 0.163000f * __temp_97__);
        float __temp_99__ = (__tilevar_4__[__iter_26__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(-1)+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+2))))]);
        float __temp_100__ = (__temp_98__ + 0.164000f * __temp_99__);
        float __temp_101__ = (__tilevar_4__[__iter_26__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(1)+(0-(__iter_2__+2))))]);
        float __temp_102__ = (__temp_100__ + 0.165000f * __temp_101__);
        float __temp_103__ = (__tilevar_4__[__iter_26__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(-1)+(0-(__iter_2__+2))))]);
        float __temp_104__ = (__temp_102__ + 0.166000f * __temp_103__);
        float __temp_105__ = (__tilevar_4__[__iter_26__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+2))))]);
        float __temp_106__ = (__temp_104__ - 1.670000f * __temp_105__);
        __tilevar_5__[__iter_26__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+3))))] = __temp_106__;
      }
    }
  }
  __syncthreads ();
  int __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ;
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ;
    if(__iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      int __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ;
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__+3),1)+2) || __iter_27__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))-2) || __iter_28__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_28__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))-2) || __iter_29__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_29__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))-2)) {
          __copy_arr_2__[__iter_29__+(N-0)*(__iter_28__+(M-0)*(__iter_27__))] = __tilevar_5__[__iter_29__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_28__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_27__+(0-(__iter_2__+3))))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__ = FORMA_MAX((__iter_2__+4),1) + (int)(threadIdx.z) ;
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__ = FORMA_MAX((__iter_1__+4),1) + (int)(threadIdx.y) ;
    if(__iter_34__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2))) {
      int __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ;
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
        float __temp_120__ = (__tilevar_5__[__iter_35__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+3))))]);
        float __temp_121__ = (__tilevar_5__[__iter_35__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+3))))]);
        float __temp_122__ = (0.161000f * __temp_120__ + 0.162000f * __temp_121__);
        float __temp_123__ = (__tilevar_5__[__iter_35__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(1)+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+3))))]);
        float __temp_124__ = (__temp_122__ + 0.163000f * __temp_123__);
        float __temp_125__ = (__tilevar_5__[__iter_35__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(-1)+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+3))))]);
        float __temp_126__ = (__temp_124__ + 0.164000f * __temp_125__);
        float __temp_127__ = (__tilevar_5__[__iter_35__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(1)+(0-(__iter_2__+3))))]);
        float __temp_128__ = (__temp_126__ + 0.165000f * __temp_127__);
        float __temp_129__ = (__tilevar_5__[__iter_35__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(-1)+(0-(__iter_2__+3))))]);
        float __temp_130__ = (__temp_128__ + 0.166000f * __temp_129__);
        float __temp_131__ = (__tilevar_5__[__iter_35__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+3))))]);
        float __temp_132__ = (__temp_130__ - 1.670000f * __temp_131__);
        __var_1__[__iter_35__+(N-0)*(__iter_34__+(M-0)*(__iter_33__))] = __temp_132__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)+GAPX) + (int)(FORMA_BLOCKDIM_X);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y));
  int __iter_2__ = (int)(blockIdx.z)*((int)(FORMA_BLOCKDIM_Z));

  int __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1)) ; __iter_3__+=(int)(blockDim.z) ){
    int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
    if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1))) {
      int __iter_5__ = FORMA_MAX(__iter_0__-2,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-1)) ){
        __tilevar_2__[__iter_5__+(EXTENT-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-__iter_1__)+(FORMA_BLOCKDIM_Y-0)*(__iter_3__+(0-__iter_2__)))] = input[__iter_5__+(N-0)*(__iter_4__+(M-0)*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_6__+=(int)(blockDim.z) ){
    int __iter_7__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    if(__iter_7__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      int __iter_8__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
        float __temp_3__ = (__tilevar_2__[__iter_8__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+0))))]);
        float __temp_7__ = (__tilevar_2__[__iter_8__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+0))))]);
        float __temp_8__ = (0.161000f * __temp_3__ + 0.162000f * __temp_7__);
        float __temp_12__ = (__tilevar_2__[__iter_8__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(1)+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+0))))]);
        float __temp_13__ = (__temp_8__ + 0.163000f * __temp_12__);
        float __temp_17__ = (__tilevar_2__[__iter_8__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(-1)+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+0))))]);
        float __temp_18__ = (__temp_13__ + 0.164000f * __temp_17__);
        float __temp_22__ = (__tilevar_2__[__iter_8__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(1)+(0-(__iter_2__+0))))]);
        float __temp_23__ = (__temp_18__ + 0.165000f * __temp_22__);
        float __temp_27__ = (__tilevar_2__[__iter_8__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(-1)+(0-(__iter_2__+0))))]);
        float __temp_28__ = (__temp_23__ + 0.166000f * __temp_27__);
        float __temp_32__ = (__tilevar_2__[__iter_8__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*( __iter_6__+(0-(__iter_2__+0))))]);
        float __temp_33__ = (__temp_28__ - 1.670000f * __temp_32__);
        __tilevar_3__[__iter_8__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+0))))] = __temp_33__;
      }
    }
  }
  __syncthreads();
  int __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    if(__iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      int __iter_11__ = FORMA_MAX((__iter_0__-1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__+1),1)+2) || __iter_9__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))-2) || __iter_10__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_10__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2)) {
          __copy_arr_0__[__iter_11__+(N-0)*(__iter_10__+(M-0)*(__iter_9__))] = __tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_9__+(0-(__iter_2__+0))))];
        }
      }
    }
  }
  __syncthreads();
  __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ;
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ;
    if(__iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
      int __iter_11__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ;
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        if (__iter_11__ < FORMA_MAX((__iter_0__-1),1) || __iter_11__ > FORMA_MIN(((__iter_0__+GAPX+1)-1),(N-2))) {
         __tilevar_3__[__iter_11__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_9__+(0-(__iter_2__+0))))] = __copy_arr_0__[__iter_11__+(N-0)*(__iter_10__+(M-0)*(__iter_9__))];
        }
      }
    }
  }
 __syncthreads();
  int __iter_15__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    if(__iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      int __iter_17__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
        float __temp_50__ = (__tilevar_3__[__iter_17__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+0))))]);
        float __temp_54__ = (__tilevar_3__[__iter_17__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+0))))]);
        float __temp_55__ = (0.161000f * __temp_50__ + 0.162000f * __temp_54__);
        float __temp_59__ = (__tilevar_3__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(1)+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+0))))]);
        float __temp_60__ = (__temp_55__ + 0.163000f * __temp_59__);
        float __temp_64__ = (__tilevar_3__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(-1)+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+0))))]);
        float __temp_65__ = (__temp_60__ + 0.164000f * __temp_64__);
        float __temp_69__ = (__tilevar_3__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(1)+(0-(__iter_2__+0))))]);
        float __temp_70__ = (__temp_65__ + 0.165000f * __temp_69__);
        float __temp_74__ = (__tilevar_3__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(-1)+(0-(__iter_2__+0))))]);
        float __temp_75__ = (__temp_70__ + 0.166000f * __temp_74__);
        float __temp_79__ = (__tilevar_3__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+0))))]);
        float __temp_80__ = (__temp_75__ - 1.670000f * __temp_79__);
        __tilevar_4__[__iter_17__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+0))))] = __temp_80__;
      }
    }
  }
  __syncthreads();
  int __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    if(__iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      int __iter_20__ = FORMA_MAX((__iter_0__-2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__+2),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))-2) || __iter_19__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))-2)) {
          __copy_arr_1__[__iter_20__+(N-0)*(__iter_19__+(M-0)*(__iter_18__))] = __tilevar_4__[__iter_20__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_19__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_18__+(0-(__iter_2__+0))))];
        }
      }
    }
  }
  __syncthreads();
  __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    if(__iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
      int __iter_20__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
        if (__iter_20__ < FORMA_MAX((__iter_0__-2),1) || __iter_20__ > FORMA_MIN(((__iter_0__+GAPX+2)-1),(N-2))) {
          __tilevar_4__[__iter_20__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_19__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_18__+(0-(__iter_2__+0))))] = __copy_arr_1__[__iter_20__+(N-0)*(__iter_19__+(M-0)*(__iter_18__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    if(__iter_25__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      int __iter_26__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        float __temp_94__ = (__tilevar_4__[__iter_26__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+0))))]);
        float __temp_95__ = (__tilevar_4__[__iter_26__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+0))))]);
        float __temp_96__ = (0.161000f * __temp_94__ + 0.162000f * __temp_95__);
        float __temp_97__ = (__tilevar_4__[__iter_26__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(1)+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+0))))]);
        float __temp_98__ = (__temp_96__ + 0.163000f * __temp_97__);
        float __temp_99__ = (__tilevar_4__[__iter_26__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(-1)+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+0))))]);
        float __temp_100__ = (__temp_98__ + 0.164000f * __temp_99__);
        float __temp_101__ = (__tilevar_4__[__iter_26__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(1)+(0-(__iter_2__+0))))]);
        float __temp_102__ = (__temp_100__ + 0.165000f * __temp_101__);
        float __temp_103__ = (__tilevar_4__[__iter_26__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(-1)+(0-(__iter_2__+0))))]);
        float __temp_104__ = (__temp_102__ + 0.166000f * __temp_103__);
        float __temp_105__ = (__tilevar_4__[__iter_26__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+0))))]);
        float __temp_106__ = (__temp_104__ - 1.670000f * __temp_105__);
        __tilevar_5__[__iter_26__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+0))))] = __temp_106__;
      }
    }
  }
  __syncthreads();
  int __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    if(__iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      int __iter_29__ = FORMA_MAX((__iter_0__-3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__+3),1)+2) || __iter_27__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))-2) || __iter_28__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_28__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))-2)) {
          __copy_arr_2__[__iter_29__+(N-0)*(__iter_28__+(M-0)*(__iter_27__))] = __tilevar_5__[__iter_29__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_28__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_27__+(0-(__iter_2__+0))))];
        }
      }
    }
  }
  __syncthreads();
  __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    if(__iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
      int __iter_29__ = FORMA_MAX((__iter_0__-5),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+GAPX+5)-1),(N-2)) ){
        if (__iter_29__ < FORMA_MAX((__iter_0__-3),1) || __iter_29__ > FORMA_MIN(((__iter_0__+GAPX+3)-1),(N-2))) {
          __tilevar_5__[__iter_29__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_28__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_27__+(0-(__iter_2__+0))))] = __copy_arr_2__[__iter_29__+(N-0)*(__iter_28__+(M-0)*(__iter_27__))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__ = FORMA_MAX((__iter_2__+4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__ = FORMA_MAX((__iter_1__+4),1) + (int)(threadIdx.y) ; 
    if(__iter_34__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2))) {
      int __iter_35__ = FORMA_MAX((__iter_0__-4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+GAPX+4)-1),(N-2)) ){
        float __temp_120__ = (__tilevar_5__[__iter_35__+(1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+0))))]);
        float __temp_121__ = (__tilevar_5__[__iter_35__+(-1)+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+0))))]);
        float __temp_122__ = (0.161000f * __temp_120__ + 0.162000f * __temp_121__);
        float __temp_123__ = (__tilevar_5__[__iter_35__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(1)+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+0))))]);
        float __temp_124__ = (__temp_122__ + 0.163000f * __temp_123__);
        float __temp_125__ = (__tilevar_5__[__iter_35__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(-1)+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+0))))]);
        float __temp_126__ = (__temp_124__ + 0.164000f * __temp_125__);
        float __temp_127__ = (__tilevar_5__[__iter_35__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(1)+(0-(__iter_2__+0))))]);
        float __temp_128__ = (__temp_126__ + 0.165000f * __temp_127__);
        float __temp_129__ = (__tilevar_5__[__iter_35__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(-1)+(0-(__iter_2__+0))))]);
        float __temp_130__ = (__temp_128__ + 0.166000f * __temp_129__);
        float __temp_131__ = (__tilevar_5__[__iter_35__+(EXTENT-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+0))))]);
        float __temp_132__ = (__temp_130__ - 1.670000f * __temp_131__);
        __var_1__[__iter_35__+(N-0)*(__iter_34__+(M-0)*(__iter_33__))] = __temp_132__;
      }
    }
  }
}

__global__ void __kernel___forma_kernel__2__(float * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __copy_arr_0__){
  int __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_12__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z) ; 
  for( ; __iter_12__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2)) ; __iter_12__ += (int)(blockDim.z) ){
    int __iter_13__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
    for( ; __iter_13__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) ; __iter_13__ += (int)(blockDim.y) ){
      int __iter_14__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_14__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        if (__iter_12__ < FORMA_MAX((__iter_2__+1),1) || __iter_12__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) || __iter_13__ < FORMA_MAX((__iter_1__+1),1) || __iter_13__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))) {
          float __temp_34__ = (0.161000f * input[__iter_14__+(1)+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_35__ = (0.162000f * input[__iter_14__+(-1)+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_36__ = (__temp_34__ + __temp_35__);
          float __temp_37__ = (0.163000f * input[__iter_14__+(N-0)*(__iter_13__+(1)+(M-0)*(__iter_12__))]);
          float __temp_38__ = (__temp_36__ + __temp_37__);
          float __temp_39__ = (0.164000f * input[__iter_14__+(N-0)*(__iter_13__+(-1)+(M-0)*(__iter_12__))]);
          float __temp_40__ = (__temp_38__ + __temp_39__);
          float __temp_41__ = (0.165000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__+(1)))]);
          float __temp_42__ = (__temp_40__ + __temp_41__);
          float __temp_43__ = (0.166000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__+(-1)))]);
          float __temp_44__ = (__temp_42__ + __temp_43__);
          float __temp_45__ = (1.670000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_46__ = (__temp_44__ - __temp_45__);
          __copy_arr_0__[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__))] = __temp_46__;
        }
      }
    }
  }
}

__global__ void __kernel___forma_kernel__3__(float * __restrict__ __copy_arr_0__, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __copy_arr_1__){
  int __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_21__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z) ; 
  for( ; __iter_21__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2)) ; __iter_21__ += (int)(blockDim.z) ){
    int __iter_22__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
    for( ; __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) ; __iter_22__ += (int)(blockDim.y) ){
      int __iter_23__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        if (__iter_21__ < FORMA_MAX((__iter_2__+2),1) || __iter_21__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) || __iter_22__ < FORMA_MAX((__iter_1__+2),1) || __iter_22__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))) {
          float __temp_81__ = (0.161000f * __copy_arr_0__[__iter_23__+(1)+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_82__ = (0.162000f * __copy_arr_0__[__iter_23__+(-1)+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_83__ = (__temp_81__ + __temp_82__);
          float __temp_84__ = (0.163000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(1)+(M-0)*(__iter_21__))]);
          float __temp_85__ = (__temp_83__ + __temp_84__);
          float __temp_86__ = (0.164000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(-1)+(M-0)*(__iter_21__))]);
          float __temp_87__ = (__temp_85__ + __temp_86__);
          float __temp_88__ = (0.165000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__+(1)))]);
          float __temp_89__ = (__temp_87__ + __temp_88__);
          float __temp_90__ = (0.166000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__+(-1)))]);
          float __temp_91__ = (__temp_89__ + __temp_90__);
          float __temp_92__ = (1.670000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_93__ = (__temp_91__ - __temp_92__);
          __copy_arr_1__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__))] = __temp_93__;
        }
      }
    }
  }
}

__global__ void __kernel___forma_kernel__4__(float * __restrict__ __copy_arr_1__, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __copy_arr_2__){
  int __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_30__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z) ; 
  for( ; __iter_30__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2)) ; __iter_30__ += (int)(blockDim.z) ){
    int __iter_31__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
    for( ; __iter_31__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) ; __iter_31__ += (int)(blockDim.y) ){
      int __iter_32__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_32__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        if (__iter_30__ < FORMA_MAX((__iter_2__+3),1) || __iter_30__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) || __iter_31__ < FORMA_MAX((__iter_1__+3),1) || __iter_31__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))) {
          float __temp_107__ = (0.161000f * __copy_arr_1__[__iter_32__+(1)+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_108__ = (0.162000f * __copy_arr_1__[__iter_32__+(-1)+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_109__ = (__temp_107__ + __temp_108__);
          float __temp_110__ = (0.163000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(1)+(M-0)*(__iter_30__))]);
          float __temp_111__ = (__temp_109__ + __temp_110__);
          float __temp_112__ = (0.164000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(-1)+(M-0)*(__iter_30__))]);
          float __temp_113__ = (__temp_111__ + __temp_112__);
          float __temp_114__ = (0.165000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__+(1)))]);
          float __temp_115__ = (__temp_113__ + __temp_114__);
          float __temp_116__ = (0.166000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__+(-1)))]);
          float __temp_117__ = (__temp_115__ + __temp_116__);
          float __temp_118__ = (1.670000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_119__ = (__temp_117__ - __temp_118__);
          __copy_arr_2__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__))] = __temp_119__;
        }
      }
    }
  }
}

__global__ void __kernel___forma_kernel__5__(float * __restrict__ __copy_arr_2__, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_36__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z) ; 
  for( ; __iter_36__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2)) ; __iter_36__ += (int)(blockDim.z) ){
    int __iter_37__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
    for( ; __iter_37__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) ; __iter_37__ += (int)(blockDim.y) ){
      int __iter_38__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_38__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        if (__iter_36__ < FORMA_MAX((__iter_2__+4),1) || __iter_36__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2)) || __iter_37__ < FORMA_MAX((__iter_1__+4),1) || __iter_37__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2))) {
          float __temp_133__ = (0.161000f * __copy_arr_2__[__iter_38__+(1)+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_134__ = (0.162000f * __copy_arr_2__[__iter_38__+(-1)+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_135__ = (__temp_133__ + __temp_134__);
          float __temp_136__ = (0.163000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(1)+(M-0)*(__iter_36__))]);
          float __temp_137__ = (__temp_135__ + __temp_136__);
          float __temp_138__ = (0.164000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(-1)+(M-0)*(__iter_36__))]);
          float __temp_139__ = (__temp_137__ + __temp_138__);
          float __temp_140__ = (0.165000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__+(1)))]);
          float __temp_141__ = (__temp_139__ + __temp_140__);
          float __temp_142__ = (0.166000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__+(-1)))]);
          float __temp_143__ = (__temp_141__ + __temp_142__);
          float __temp_144__ = (1.670000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_145__ = (__temp_143__ - __temp_144__);
          __var_1__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__))] = __temp_145__;
        }
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d7pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_2___kernel___forma_kernel__0__ = ((L-1) - 0 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 16;
  int __block_2___kernel___forma_kernel__0__ = 12;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x+GAPX);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y);
  int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.z);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 unrollConfig (32,16,1);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  __gridConfig___kernel___forma_kernel__0__.x = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x); 
  dim3 __blockConfig___kernel___forma_kernel__2__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__2__>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __copy_arr_0__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");

  dim3 __blockConfig___kernel___forma_kernel__3__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__3__>>> (__copy_arr_0__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __copy_arr_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  dim3 __blockConfig___kernel___forma_kernel__4__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__4__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__4__>>> (__copy_arr_1__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __copy_arr_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__4__\n");

  dim3 __blockConfig___kernel___forma_kernel__5__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__5__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__5__>>> (__copy_arr_2__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__5__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __copy_arr_0__, float * __restrict__ __copy_arr_1__, float * __restrict__ __copy_arr_2__, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  float * __tilevar_2__ = __tilevar_0__;
  float * __tilevar_3__ = __tilevar_1__;
  float * __tilevar_4__ = __tilevar_0__;
  float * __tilevar_5__ = __tilevar_1__;
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_3__;
  __iter_3__ = FORMA_MAX(__iter_2__,0) + (int)(threadIdx.z) ; 
  for( ; __iter_3__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-1)) ; __iter_3__ += (int)(blockDim.z) ){
    int __iter_4__;
    __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ; 
    if( __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) ){
      int __iter_5__;
      __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1)) ){
        __tilevar_2__[__iter_5__+(0-__iter_0__)+(FORMA_BLOCKDIM_X-0)*(__iter_4__+(0-__iter_1__)+(FORMA_BLOCKDIM_Y-0)*(__iter_3__+(0-__iter_2__)))] = input[__iter_5__+(N-0)*(__iter_4__+(M-0)*(__iter_3__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__;
  __iter_6__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_6__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_6__ += (int)(blockDim.z) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    if( __iter_7__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ){
      int __iter_8__;
      __iter_8__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        int __temp_0__;
        __temp_0__ = __iter_6__;
        int __temp_1__;
        __temp_1__ = __iter_7__;
        int __temp_2__;
        __temp_2__ = __iter_8__+(1);
        float __temp_3__;
        __temp_3__ = (0.161000f * __tilevar_2__[__temp_2__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_1__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_0__+(0-(__iter_2__+0))))]);
        int __temp_4__;
        __temp_4__ = __iter_6__;
        int __temp_5__;
        __temp_5__ = __iter_7__;
        int __temp_6__;
        __temp_6__ = __iter_8__+(-1);
        float __temp_7__;
        __temp_7__ = (0.162000f * __tilevar_2__[__temp_6__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_5__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_4__+(0-(__iter_2__+0))))]);
        float __temp_8__;
        __temp_8__ = (__temp_3__ + __temp_7__);
        int __temp_9__;
        __temp_9__ = __iter_6__;
        int __temp_10__;
        __temp_10__ = __iter_7__+(1);
        int __temp_11__;
        __temp_11__ = __iter_8__;
        float __temp_12__;
        __temp_12__ = (0.163000f * __tilevar_2__[__temp_11__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_10__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_9__+(0-(__iter_2__+0))))]);
        float __temp_13__;
        __temp_13__ = (__temp_8__ + __temp_12__);
        int __temp_14__;
        __temp_14__ = __iter_6__;
        int __temp_15__;
        __temp_15__ = __iter_7__+(-1);
        int __temp_16__;
        __temp_16__ = __iter_8__;
        float __temp_17__;
        __temp_17__ = (0.164000f * __tilevar_2__[__temp_16__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_15__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_14__+(0-(__iter_2__+0))))]);
        float __temp_18__;
        __temp_18__ = (__temp_13__ + __temp_17__);
        int __temp_19__;
        __temp_19__ = __iter_6__+(1);
        int __temp_20__;
        __temp_20__ = __iter_7__;
        int __temp_21__;
        __temp_21__ = __iter_8__;
        float __temp_22__;
        __temp_22__ = (0.165000f * __tilevar_2__[__temp_21__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_20__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_19__+(0-(__iter_2__+0))))]);
        float __temp_23__;
        __temp_23__ = (__temp_18__ + __temp_22__);
        int __temp_24__;
        __temp_24__ = __iter_6__+(-1);
        int __temp_25__;
        __temp_25__ = __iter_7__;
        int __temp_26__;
        __temp_26__ = __iter_8__;
        float __temp_27__;
        __temp_27__ = (0.166000f * __tilevar_2__[__temp_26__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_25__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_24__+(0-(__iter_2__+0))))]);
        float __temp_28__;
        __temp_28__ = (__temp_23__ + __temp_27__);
        int __temp_29__;
        __temp_29__ = __iter_6__;
        int __temp_30__;
        __temp_30__ = __iter_7__;
        int __temp_31__;
        __temp_31__ = __iter_8__;
        float __temp_32__;
        __temp_32__ = (1.670000f * __tilevar_2__[__temp_31__+(0-(__iter_0__+0))+(FORMA_BLOCKDIM_X-0)*(__temp_30__+(0-(__iter_1__+0))+(FORMA_BLOCKDIM_Y-0)*(__temp_29__+(0-(__iter_2__+0))))]);
        float __temp_33__;
        __temp_33__ = (__temp_28__ - __temp_32__);
        __tilevar_3__[__iter_8__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_7__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_6__+(0-(__iter_2__+1))))] = __temp_33__;
      }
    }
  }
  int __iter_9__;
  __iter_9__ = FORMA_MAX((__iter_2__+1),1) + (int)(threadIdx.z) ; 
  for( ; __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) ; __iter_9__ += (int)(blockDim.z) ){
    int __iter_10__;
    __iter_10__ = FORMA_MAX((__iter_1__+1),1) + (int)(threadIdx.y) ; 
    if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) ){
      int __iter_11__;
      __iter_11__ = FORMA_MAX((__iter_0__+1),1) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2)) ){
        if (__iter_9__ < (FORMA_MAX((__iter_2__+1),1)+2) || __iter_9__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2))-2) || __iter_10__ < (FORMA_MAX((__iter_1__+1),1)+2) || __iter_10__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2))-2) || __iter_11__ < (FORMA_MAX((__iter_0__+1),1)+2) || __iter_11__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))-2)) {
          __copy_arr_0__[__iter_11__+(N-0)*(__iter_10__+(M-0)*(__iter_9__))] = __tilevar_3__[__iter_11__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__iter_10__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__iter_9__+(0-(__iter_2__+1))))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_15__;
  __iter_15__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_15__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_15__ += (int)(blockDim.z) ){
    int __iter_16__;
    __iter_16__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    if( __iter_16__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ){
      int __iter_17__;
      __iter_17__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        int __temp_47__;
        __temp_47__ = __iter_15__;
        int __temp_48__;
        __temp_48__ = __iter_16__;
        int __temp_49__;
        __temp_49__ = __iter_17__+(1);
        float __temp_50__;
        __temp_50__ = (0.161000f * __tilevar_3__[__temp_49__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_48__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__temp_47__+(0-(__iter_2__+1))))]);
        int __temp_51__;
        __temp_51__ = __iter_15__;
        int __temp_52__;
        __temp_52__ = __iter_16__;
        int __temp_53__;
        __temp_53__ = __iter_17__+(-1);
        float __temp_54__;
        __temp_54__ = (0.162000f * __tilevar_3__[__temp_53__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_52__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__temp_51__+(0-(__iter_2__+1))))]);
        float __temp_55__;
        __temp_55__ = (__temp_50__ + __temp_54__);
        int __temp_56__;
        __temp_56__ = __iter_15__;
        int __temp_57__;
        __temp_57__ = __iter_16__+(1);
        int __temp_58__;
        __temp_58__ = __iter_17__;
        float __temp_59__;
        __temp_59__ = (0.163000f * __tilevar_3__[__temp_58__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_57__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__temp_56__+(0-(__iter_2__+1))))]);
        float __temp_60__;
        __temp_60__ = (__temp_55__ + __temp_59__);
        int __temp_61__;
        __temp_61__ = __iter_15__;
        int __temp_62__;
        __temp_62__ = __iter_16__+(-1);
        int __temp_63__;
        __temp_63__ = __iter_17__;
        float __temp_64__;
        __temp_64__ = (0.164000f * __tilevar_3__[__temp_63__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_62__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__temp_61__+(0-(__iter_2__+1))))]);
        float __temp_65__;
        __temp_65__ = (__temp_60__ + __temp_64__);
        int __temp_66__;
        __temp_66__ = __iter_15__+(1);
        int __temp_67__;
        __temp_67__ = __iter_16__;
        int __temp_68__;
        __temp_68__ = __iter_17__;
        float __temp_69__;
        __temp_69__ = (0.165000f * __tilevar_3__[__temp_68__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_67__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__temp_66__+(0-(__iter_2__+1))))]);
        float __temp_70__;
        __temp_70__ = (__temp_65__ + __temp_69__);
        int __temp_71__;
        __temp_71__ = __iter_15__+(-1);
        int __temp_72__;
        __temp_72__ = __iter_16__;
        int __temp_73__;
        __temp_73__ = __iter_17__;
        float __temp_74__;
        __temp_74__ = (0.166000f * __tilevar_3__[__temp_73__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_72__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__temp_71__+(0-(__iter_2__+1))))]);
        float __temp_75__;
        __temp_75__ = (__temp_70__ + __temp_74__);
        int __temp_76__;
        __temp_76__ = __iter_15__;
        int __temp_77__;
        __temp_77__ = __iter_16__;
        int __temp_78__;
        __temp_78__ = __iter_17__;
        float __temp_79__;
        __temp_79__ = (1.670000f * __tilevar_3__[__temp_78__+(0-(__iter_0__+1))+(FORMA_BLOCKDIM_X-0)*(__temp_77__+(0-(__iter_1__+1))+(FORMA_BLOCKDIM_Y-0)*(__temp_76__+(0-(__iter_2__+1))))]);
        float __temp_80__;
        __temp_80__ = (__temp_75__ - __temp_79__);
        __tilevar_4__[__iter_17__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_16__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_15__+(0-(__iter_2__+2))))] = __temp_80__;
      }
    }
  }
  int __iter_18__;
  __iter_18__ = FORMA_MAX((__iter_2__+2),1) + (int)(threadIdx.z) ; 
  for( ; __iter_18__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) ; __iter_18__ += (int)(blockDim.z) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((__iter_1__+2),1) + (int)(threadIdx.y) ; 
    if( __iter_19__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) ){
      int __iter_20__;
      __iter_20__ = FORMA_MAX((__iter_0__+2),1) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2)) ){
        if (__iter_18__ < (FORMA_MAX((__iter_2__+2),1)+2) || __iter_18__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2))-2) || __iter_19__ < (FORMA_MAX((__iter_1__+2),1)+2) || __iter_19__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2))-2) || __iter_20__ < (FORMA_MAX((__iter_0__+2),1)+2) || __iter_20__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))-2)) {
          __copy_arr_1__[__iter_20__+(N-0)*(__iter_19__+(M-0)*(__iter_18__))] = __tilevar_4__[__iter_20__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_19__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_18__+(0-(__iter_2__+2))))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_24__;
  __iter_24__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_24__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_24__ += (int)(blockDim.z) ){
    int __iter_25__;
    __iter_25__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    if( __iter_25__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ){
      int __iter_26__;
      __iter_26__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_26__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        float __temp_94__;
        __temp_94__ = (0.161000f * __tilevar_4__[__iter_26__+(1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+2))))]);
        float __temp_95__;
        __temp_95__ = (0.162000f * __tilevar_4__[__iter_26__+(-1)+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+2))))]);
        float __temp_96__;
        __temp_96__ = (__temp_94__ + __temp_95__);
        float __temp_97__;
        __temp_97__ = (0.163000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(1)+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+2))))]);
        float __temp_98__;
        __temp_98__ = (__temp_96__ + __temp_97__);
        float __temp_99__;
        __temp_99__ = (0.164000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(-1)+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+2))))]);
        float __temp_100__;
        __temp_100__ = (__temp_98__ + __temp_99__);
        float __temp_101__;
        __temp_101__ = (0.165000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(1)+(0-(__iter_2__+2))))]);
        float __temp_102__;
        __temp_102__ = (__temp_100__ + __temp_101__);
        float __temp_103__;
        __temp_103__ = (0.166000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(-1)+(0-(__iter_2__+2))))]);
        float __temp_104__;
        __temp_104__ = (__temp_102__ + __temp_103__);
        float __temp_105__;
        __temp_105__ = (1.670000f * __tilevar_4__[__iter_26__+(0-(__iter_0__+2))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+2))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+2))))]);
        float __temp_106__;
        __temp_106__ = (__temp_104__ - __temp_105__);
        __tilevar_5__[__iter_26__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_25__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_24__+(0-(__iter_2__+3))))] = __temp_106__;
      }
    }
  }
  int __iter_27__;
  __iter_27__ = FORMA_MAX((__iter_2__+3),1) + (int)(threadIdx.z) ; 
  for( ; __iter_27__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) ; __iter_27__ += (int)(blockDim.z) ){
    int __iter_28__;
    __iter_28__ = FORMA_MAX((__iter_1__+3),1) + (int)(threadIdx.y) ; 
    if( __iter_28__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) ){
      int __iter_29__;
      __iter_29__ = FORMA_MAX((__iter_0__+3),1) + (int)(threadIdx.x) ; 
      if( __iter_29__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2)) ){
        if (__iter_27__ < (FORMA_MAX((__iter_2__+3),1)+2) || __iter_27__ > (FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2))-2) || __iter_28__ < (FORMA_MAX((__iter_1__+3),1)+2) || __iter_28__ > (FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2))-2) || __iter_29__ < (FORMA_MAX((__iter_0__+3),1)+2) || __iter_29__ > (FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))-2)) {
          __copy_arr_2__[__iter_29__+(N-0)*(__iter_28__+(M-0)*(__iter_27__))] = __tilevar_5__[__iter_29__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_28__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_27__+(0-(__iter_2__+3))))];
        }
      }
    }
  }
  __syncthreads();
  int __iter_33__;
  __iter_33__ = FORMA_MAX((__iter_2__+4),1) + (int)(threadIdx.z) ; 
  for( ; __iter_33__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2)) ; __iter_33__ += (int)(blockDim.z) ){
    int __iter_34__;
    __iter_34__ = FORMA_MAX((__iter_1__+4),1) + (int)(threadIdx.y) ; 
    if( __iter_34__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2)) ){
      int __iter_35__;
      __iter_35__ = FORMA_MAX((__iter_0__+4),1) + (int)(threadIdx.x) ; 
      if( __iter_35__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2)) ){
        float __temp_120__;
        __temp_120__ = (0.161000f * __tilevar_5__[__iter_35__+(1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+3))))]);
        float __temp_121__;
        __temp_121__ = (0.162000f * __tilevar_5__[__iter_35__+(-1)+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+3))))]);
        float __temp_122__;
        __temp_122__ = (__temp_120__ + __temp_121__);
        float __temp_123__;
        __temp_123__ = (0.163000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(1)+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+3))))]);
        float __temp_124__;
        __temp_124__ = (__temp_122__ + __temp_123__);
        float __temp_125__;
        __temp_125__ = (0.164000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(-1)+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+3))))]);
        float __temp_126__;
        __temp_126__ = (__temp_124__ + __temp_125__);
        float __temp_127__;
        __temp_127__ = (0.165000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(1)+(0-(__iter_2__+3))))]);
        float __temp_128__;
        __temp_128__ = (__temp_126__ + __temp_127__);
        float __temp_129__;
        __temp_129__ = (0.166000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(-1)+(0-(__iter_2__+3))))]);
        float __temp_130__;
        __temp_130__ = (__temp_128__ + __temp_129__);
        float __temp_131__;
        __temp_131__ = (1.670000f * __tilevar_5__[__iter_35__+(0-(__iter_0__+3))+(FORMA_BLOCKDIM_X-0)*(__iter_34__+(0-(__iter_1__+3))+(FORMA_BLOCKDIM_Y-0)*(__iter_33__+(0-(__iter_2__+3))))]);
        float __temp_132__;
        __temp_132__ = (__temp_130__ - __temp_131__);
        __var_1__[__iter_35__+(N-0)*(__iter_34__+(M-0)*(__iter_33__))] = __temp_132__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-0)*(FORMA_BLOCKDIM_Y-0)*(FORMA_BLOCKDIM_X-0)));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __copy_arr_0__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_12__;
  __iter_12__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z) ; 
  for( ; __iter_12__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2)) ; __iter_12__ += (int)(blockDim.z) ){
    int __iter_13__;
    __iter_13__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
    for( ; __iter_13__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) ; __iter_13__ += (int)(blockDim.y) ){
      int __iter_14__;
      __iter_14__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_14__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        if (__iter_12__ < FORMA_MAX((__iter_2__+1),1) || __iter_12__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-2),(L-2)) || __iter_13__ < FORMA_MAX((__iter_1__+1),1) || __iter_13__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-2),(M-2)) || __iter_14__ < FORMA_MAX((__iter_0__+1),1) || __iter_14__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-2),(N-2))) {
          float __temp_34__;
          __temp_34__ = (0.161000f * input[__iter_14__+(1)+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_35__;
          __temp_35__ = (0.162000f * input[__iter_14__+(-1)+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_36__;
          __temp_36__ = (__temp_34__ + __temp_35__);
          float __temp_37__;
          __temp_37__ = (0.163000f * input[__iter_14__+(N-0)*(__iter_13__+(1)+(M-0)*(__iter_12__))]);
          float __temp_38__;
          __temp_38__ = (__temp_36__ + __temp_37__);
          float __temp_39__;
          __temp_39__ = (0.164000f * input[__iter_14__+(N-0)*(__iter_13__+(-1)+(M-0)*(__iter_12__))]);
          float __temp_40__;
          __temp_40__ = (__temp_38__ + __temp_39__);
          float __temp_41__;
          __temp_41__ = (0.165000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__+(1)))]);
          float __temp_42__;
          __temp_42__ = (__temp_40__ + __temp_41__);
          float __temp_43__;
          __temp_43__ = (0.166000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__+(-1)))]);
          float __temp_44__;
          __temp_44__ = (__temp_42__ + __temp_43__);
          float __temp_45__;
          __temp_45__ = (1.670000f * input[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__))]);
          float __temp_46__;
          __temp_46__ = (__temp_44__ - __temp_45__);
          __copy_arr_0__[__iter_14__+(N-0)*(__iter_13__+(M-0)*(__iter_12__))] = __temp_46__;
        }
      }
    }
  }
}

__global__ void __kernel___forma_kernel__2__(float * __restrict__ __copy_arr_0__, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __copy_arr_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_21__;
  __iter_21__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z) ; 
  for( ; __iter_21__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2)) ; __iter_21__ += (int)(blockDim.z) ){
    int __iter_22__;
    __iter_22__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
    for( ; __iter_22__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) ; __iter_22__ += (int)(blockDim.y) ){
      int __iter_23__;
      __iter_23__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        if (__iter_21__ < FORMA_MAX((__iter_2__+2),1) || __iter_21__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-3),(L-2)) || __iter_22__ < FORMA_MAX((__iter_1__+2),1) || __iter_22__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-2)) || __iter_23__ < FORMA_MAX((__iter_0__+2),1) || __iter_23__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-2))) {
          float __temp_81__;
          __temp_81__ = (0.161000f * __copy_arr_0__[__iter_23__+(1)+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_82__;
          __temp_82__ = (0.162000f * __copy_arr_0__[__iter_23__+(-1)+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_83__;
          __temp_83__ = (__temp_81__ + __temp_82__);
          float __temp_84__;
          __temp_84__ = (0.163000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(1)+(M-0)*(__iter_21__))]);
          float __temp_85__;
          __temp_85__ = (__temp_83__ + __temp_84__);
          float __temp_86__;
          __temp_86__ = (0.164000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(-1)+(M-0)*(__iter_21__))]);
          float __temp_87__;
          __temp_87__ = (__temp_85__ + __temp_86__);
          float __temp_88__;
          __temp_88__ = (0.165000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__+(1)))]);
          float __temp_89__;
          __temp_89__ = (__temp_87__ + __temp_88__);
          float __temp_90__;
          __temp_90__ = (0.166000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__+(-1)))]);
          float __temp_91__;
          __temp_91__ = (__temp_89__ + __temp_90__);
          float __temp_92__;
          __temp_92__ = (1.670000f * __copy_arr_0__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__))]);
          float __temp_93__;
          __temp_93__ = (__temp_91__ - __temp_92__);
          __copy_arr_1__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__))] = __temp_93__;
        }
      }
    }
  }
}

__global__ void __kernel___forma_kernel__3__(float * __restrict__ __copy_arr_1__, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __copy_arr_2__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_30__;
  __iter_30__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z) ; 
  for( ; __iter_30__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2)) ; __iter_30__ += (int)(blockDim.z) ){
    int __iter_31__;
    __iter_31__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
    for( ; __iter_31__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) ; __iter_31__ += (int)(blockDim.y) ){
      int __iter_32__;
      __iter_32__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_32__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        if (__iter_30__ < FORMA_MAX((__iter_2__+3),1) || __iter_30__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-4),(L-2)) || __iter_31__ < FORMA_MAX((__iter_1__+3),1) || __iter_31__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-4),(M-2)) || __iter_32__ < FORMA_MAX((__iter_0__+3),1) || __iter_32__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-4),(N-2))) {
          float __temp_107__;
          __temp_107__ = (0.161000f * __copy_arr_1__[__iter_32__+(1)+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_108__;
          __temp_108__ = (0.162000f * __copy_arr_1__[__iter_32__+(-1)+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_109__;
          __temp_109__ = (__temp_107__ + __temp_108__);
          float __temp_110__;
          __temp_110__ = (0.163000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(1)+(M-0)*(__iter_30__))]);
          float __temp_111__;
          __temp_111__ = (__temp_109__ + __temp_110__);
          float __temp_112__;
          __temp_112__ = (0.164000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(-1)+(M-0)*(__iter_30__))]);
          float __temp_113__;
          __temp_113__ = (__temp_111__ + __temp_112__);
          float __temp_114__;
          __temp_114__ = (0.165000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__+(1)))]);
          float __temp_115__;
          __temp_115__ = (__temp_113__ + __temp_114__);
          float __temp_116__;
          __temp_116__ = (0.166000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__+(-1)))]);
          float __temp_117__;
          __temp_117__ = (__temp_115__ + __temp_116__);
          float __temp_118__;
          __temp_118__ = (1.670000f * __copy_arr_1__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__))]);
          float __temp_119__;
          __temp_119__ = (__temp_117__ - __temp_118__);
          __copy_arr_2__[__iter_32__+(N-0)*(__iter_31__+(M-0)*(__iter_30__))] = __temp_119__;
        }
      }
    }
  }
}

__global__ void __kernel___forma_kernel__4__(float * __restrict__ __copy_arr_2__, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X);
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y);
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z);
  int __iter_36__;
  __iter_36__ = FORMA_MAX(__iter_2__,1) + (int)(threadIdx.z) ; 
  for( ; __iter_36__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-2)) ; __iter_36__ += (int)(blockDim.z) ){
    int __iter_37__;
    __iter_37__ = FORMA_MAX(__iter_1__,1) + (int)(threadIdx.y) ; 
    for( ; __iter_37__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-2)) ; __iter_37__ += (int)(blockDim.y) ){
      int __iter_38__;
      __iter_38__ = FORMA_MAX(__iter_0__,1) + (int)(threadIdx.x) ; 
      if( __iter_38__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-2)) ){
        if (__iter_36__ < FORMA_MAX((__iter_2__+4),1) || __iter_36__ > FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-5),(L-2)) || __iter_37__ < FORMA_MAX((__iter_1__+4),1) || __iter_37__ > FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-2)) || __iter_38__ < FORMA_MAX((__iter_0__+4),1) || __iter_38__ > FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-2))) {
          float __temp_133__;
          __temp_133__ = (0.161000f * __copy_arr_2__[__iter_38__+(1)+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_134__;
          __temp_134__ = (0.162000f * __copy_arr_2__[__iter_38__+(-1)+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_135__;
          __temp_135__ = (__temp_133__ + __temp_134__);
          float __temp_136__;
          __temp_136__ = (0.163000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(1)+(M-0)*(__iter_36__))]);
          float __temp_137__;
          __temp_137__ = (__temp_135__ + __temp_136__);
          float __temp_138__;
          __temp_138__ = (0.164000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(-1)+(M-0)*(__iter_36__))]);
          float __temp_139__;
          __temp_139__ = (__temp_137__ + __temp_138__);
          float __temp_140__;
          __temp_140__ = (0.165000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__+(1)))]);
          float __temp_141__;
          __temp_141__ = (__temp_139__ + __temp_140__);
          float __temp_142__;
          __temp_142__ = (0.166000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__+(-1)))]);
          float __temp_143__;
          __temp_143__ = (__temp_141__ + __temp_142__);
          float __temp_144__;
          __temp_144__ = (1.670000f * __copy_arr_2__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__))]);
          float __temp_145__;
          __temp_145__ = (__temp_143__ - __temp_144__);
          __var_1__[__iter_38__+(N-0)*(__iter_37__+(M-0)*(__iter_36__))] = __temp_145__;
        }
      }
    }
  }
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d7pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __copy_arr_0__;
  hipMalloc(&__copy_arr_0__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_0__\n");
  float * __copy_arr_1__;
  hipMalloc(&__copy_arr_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_1__\n");
  float * __copy_arr_2__;
  hipMalloc(&__copy_arr_2__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __copy_arr_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((N-1) - 0 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((M-1) - 0 ) + 1;
  int __size_2___kernel___forma_kernel__0__ = ((L-1) - 0 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  int __block_1___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___2 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_2___kernel___forma_kernel__0__ = FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___2,__size_2___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_2),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_2___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.z/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.z /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,9))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y);
  int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.z);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 __blockConfig_st__kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, 2);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig_st__kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __copy_arr_0__, __copy_arr_1__, __copy_arr_2__, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  dim3 __blockConfig___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__1__>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __copy_arr_0__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");

  dim3 __blockConfig___kernel___forma_kernel__2__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__2__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__2__>>> (__copy_arr_0__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __copy_arr_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__2__\n");

  dim3 __blockConfig___kernel___forma_kernel__3__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__3__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__3__>>> (__copy_arr_1__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __copy_arr_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__3__\n");

  dim3 __blockConfig___kernel___forma_kernel__4__(__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y/2, 1);
  __kernel___forma_kernel__4__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__4__>>> (__copy_arr_2__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__4__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__copy_arr_0__);
  hipFree(__copy_arr_1__);
  hipFree(__copy_arr_2__);
}
/*Host Free End*/

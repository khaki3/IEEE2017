#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"
#include <nvml.h>
#include <assert.h>

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
	template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
	int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
	input[loc] = value;
}


	template<typename T>
void initialize_array(T* d_input, int size, T value)
{
	dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
	dim3 init_block(FORMA_MAX_BLOCKDIM_0);
	__kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __var_4__){
	int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
	int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
	int FORMA_BLOCKDIM_X = (int)(blockDim.x);
	int __iter_0__;
	__iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 2;
	if(__iter_0__ <= (N-3)){
		int __iter_1__;
		__iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 2;
		if(__iter_1__ <= (M-3)){
			int __iter_2__;
			__iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 2;
			if(__iter_2__ <= (L-3)){
				float __temp_0__;
				__temp_0__ = (0.083000f * input[__iter_0__+(2)+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
				float __temp_1__;
				__temp_1__ = (0.083000f * input[__iter_0__+(1)+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
				float __temp_2__;
				__temp_2__ = (__temp_0__ + __temp_1__);
				float __temp_3__;
				__temp_3__ = (0.083000f * input[__iter_0__+(-1)+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
				float __temp_4__;
				__temp_4__ = (__temp_2__ + __temp_3__);
				float __temp_5__;
				__temp_5__ = (0.083000f * input[__iter_0__+(-2)+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
				float __temp_6__;
				__temp_6__ = (__temp_4__ + __temp_5__);
				float __temp_7__;
				__temp_7__ = (0.083000f * input[__iter_0__+(N-0)*(__iter_1__+(2)+(M-0)*(__iter_2__))]);
				float __temp_8__;
				__temp_8__ = (__temp_6__ + __temp_7__);
				float __temp_9__;
				__temp_9__ = (0.083000f * input[__iter_0__+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__))]);
				float __temp_10__;
				__temp_10__ = (__temp_8__ + __temp_9__);
				float __temp_11__;
				__temp_11__ = (0.083000f * input[__iter_0__+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__))]);
				float __temp_12__;
				__temp_12__ = (__temp_10__ + __temp_11__);
				float __temp_13__;
				__temp_13__ = (0.083000f * input[__iter_0__+(N-0)*(__iter_1__+(-2)+(M-0)*(__iter_2__))]);
				float __temp_14__;
				__temp_14__ = (__temp_12__ + __temp_13__);
				float __temp_15__;
				__temp_15__ = (0.083000f * input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__+(2)))]);
				float __temp_16__;
				__temp_16__ = (__temp_14__ + __temp_15__);
				float __temp_17__;
				__temp_17__ = (0.083000f * input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__+(1)))]);
				float __temp_18__;
				__temp_18__ = (__temp_16__ + __temp_17__);
				float __temp_19__;
				__temp_19__ = (0.083000f * input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__+(-1)))]);
				float __temp_20__;
				__temp_20__ = (__temp_18__ + __temp_19__);
				float __temp_21__;
				__temp_21__ = (0.083000f * input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__+(-2)))]);
				float __temp_22__;
				__temp_22__ = (__temp_20__ + __temp_21__);
				float __temp_23__;
				__temp_23__ = (0.996000f * input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
				float __temp_24__;
				__temp_24__ = (__temp_22__ - __temp_23__);
				__var_4__[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))] = __temp_24__;
			}
		}
	}
}

/* Host Code Begin */
extern "C" void j3d13pt(float * h_input, int L, int M, int N, float * __var_0__){

	/* Host allocation Begin */
	float * input;
	hipMalloc(&input,sizeof(float)*((L-0)*(M-0)*(N-0)));
	Check_CUDA_Error("Allocation Error!! : input\n");
	hipPointerAttribute_t ptrAttrib_h_input;
	hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
	if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
		if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
			memcpy_kind_h_input = hipMemcpyDeviceToDevice;
	hipGetLastError();
	if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
		hipMemcpy(input,h_input,sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
	}
	float * __var_1__;
	hipMalloc(&__var_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
	Check_CUDA_Error("Allocation Error!! : __var_1__\n");
	float * __var_2__;
	hipMalloc(&__var_2__,sizeof(float)*((L-0)*(M-0)*(N-0)));
	Check_CUDA_Error("Allocation Error!! : __var_2__\n");
	float * __var_3__;
	hipMalloc(&__var_3__,sizeof(float)*((L-0)*(M-0)*(N-0)));
	Check_CUDA_Error("Allocation Error!! : __var_3__\n");
	float * __var_4__;
	hipMalloc(&__var_4__,sizeof(float)*((L-0)*(M-0)*(N-0)));
	Check_CUDA_Error("Allocation Error!! : __var_4__\n");
	/*Host Allocation End */
	/* Kernel Launch Begin */
#ifdef _TIMER_
	hipEvent_t _forma_timer_start_,_forma_timer_stop_;
	hipEventCreate(&_forma_timer_start_);
	hipEventCreate(&_forma_timer_stop_);
	hipEventRecord(_forma_timer_start_,0);
#endif
	int __size_0___kernel___forma_kernel__0__ = ((N-3) - 2 ) + 1;
	int __size_1___kernel___forma_kernel__0__ = ((M-3) - 2 ) + 1;
	int __size_2___kernel___forma_kernel__0__ = ((L-3) - 2 ) + 1;
	int __block_0___kernel___forma_kernel__0__ = 16;
	int __block_1___kernel___forma_kernel__0__ = 4;
	int __block_2___kernel___forma_kernel__0__ = 4;
	dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
	int __SMemSize___kernel___forma_kernel__0__ = 0;
	int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
	int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
	int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
	dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);

	unsigned int power1, power2;
	nvmlReturn_t result;
	nvmlDevice_t device;
	nvmlEnableState_t mode;
	result=nvmlInit();
	result = nvmlDeviceGetHandleByIndex(0, &device);
	assert(NVML_SUCCESS == result);
	result=nvmlDeviceGetPowerManagementMode(device, &mode);
	printf("enabled = %d\n", mode);
	result=nvmlDeviceGetPowerUsage(device,&power1);
	assert(NVML_SUCCESS == result);
	hipDeviceSynchronize();

	for (int x=0; x<500; x++) {

		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __var_4__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_4__, L, M, N, __var_3__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_3__, L, M, N, __var_2__);
		__kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, L, M, N, __var_1__);
		Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
	}

	hipDeviceSynchronize();
	result=nvmlDeviceGetPowerUsage(device,&power2);
	assert(NVML_SUCCESS == result);
	power2 -= power1;
	printf("%u\n", power2);
	nvmlShutdown();
	hipPointerAttribute_t ptrAttrib___var_0__;
	hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
	if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
		if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
			memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
	hipGetLastError();
	hipMemcpy(__var_0__,__var_1__, sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
	hipEventRecord(_forma_timer_stop_,0);
	hipEventSynchronize(_forma_timer_stop_);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
	printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
	hipEventDestroy(_forma_timer_start_);
	hipEventDestroy(_forma_timer_stop_);
#endif
	/*Kernel Launch End */
	/* Host Free Begin */
	hipFree(input);
	hipFree(__var_1__);
	hipFree(__var_2__);
	hipFree(__var_3__);
	hipFree(__var_4__);
}
/*Host Free End*/

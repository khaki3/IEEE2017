#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif

template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
/* X, Y, Z */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, int FORMA_BLOCKDIM_X, int FORMA_BLOCKDIM_Y, int FORMA_BLOCKDIM_Z, float * __restrict__ __var_1__){
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);

  int __iter_0__ = (int)(blockIdx.x)*((int)(FORMA_BLOCKDIM_X)-8);
  int __iter_1__ = (int)(blockIdx.y)*((int)(FORMA_BLOCKDIM_Y)-8);
  float t2_0=0.0f, t3_0=0.0f, t2_1=0.0f, t3_1=0.0f;
  float b2_0=0.0f, b3_0=0.0f, b2_1=0.0f, b3_1=0.0f;

  // Initialize the values
  int __iter_4__ = FORMA_MAX(__iter_1__,0) + (int)(threadIdx.y) ;
  int __iter_5__ = FORMA_MAX(__iter_0__,0) + (int)(threadIdx.x) ; 
  if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1))) {
      b2_0 = input[__iter_5__+N*(__iter_4__+M*(0))];
      __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = input[__iter_5__+N*(__iter_4__+M*(1))];
      t2_0 = input[__iter_5__+N*(__iter_4__+M*(2))];  
      t2_1 = input[__iter_5__+N*(__iter_4__+M*(3))];
  }
  // Rest of the computation
  for (int __iter_2__ = 2; __iter_2__ < L-2; __iter_2__++) {
    if(__iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-1)) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-1))){
      b2_1 = b2_0;
      b2_0  = __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
      __tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t2_0;
      t2_0 = t2_1; 
      t2_1 = input[__iter_5__+N*(__iter_4__+M*(__iter_2__+2))]; 
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+2),2) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-3),(M-3)) & __iter_5__ >= FORMA_MAX((__iter_0__+2),2) & __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-3),(N-3)) ){
        float __temp_3__ = (__tilevar_2__[__iter_5__+2-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_7__ = (__tilevar_2__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_8__ = (0.083000f * __temp_3__ + 0.083000f * __temp_7__);
        float __temp_12__ = (__tilevar_2__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_13__ = (__temp_8__ + 0.083000f * __temp_12__);
        float __temp_17__ = (__tilevar_2__[__iter_5__-2-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_18__ = (__temp_13__ + 0.083000f * __temp_17__);
        float __temp_22__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+2-__iter_1__)]);
        float __temp_23__ = (__temp_18__ + 0.083000f * __temp_22__);
        float __temp_27__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_28__ = (__temp_23__ + 0.083000f * __temp_27__);
        float __temp_32__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_33__ = (__temp_28__ + 0.083000f * __temp_32__);
        float __temp_37__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-2-__iter_1__)]);
        float __temp_38__ = (__temp_33__ + 0.083000f * __temp_37__);
        float __temp_43__ = (__temp_38__ + 0.083000f * t2_1);
        float __temp_48__ = (__temp_43__ + 0.083000f * t2_0);
        float __temp_53__ = (__temp_48__ + 0.083000f * b2_0);
        float __temp_58__ = (__temp_53__ + 0.083000f * b2_1);
        float __temp_62__ = (__tilevar_2__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_63__ = (__temp_58__ - 0.996000f * __temp_62__);
	b3_1 = b3_0;
        b3_0 = __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)];
        __tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)] = t3_0;
        t3_0 = t3_1;
        t3_1 = __temp_63__;
    }
    __syncthreads ();
    if(__iter_4__ >= FORMA_MAX((__iter_1__+4),2) & __iter_4__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-5),(M-3)) & __iter_5__ >= FORMA_MAX((__iter_0__+4),2) &  __iter_5__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-5),(N-3)) ){
        float __temp_3__ = (__tilevar_3__[__iter_5__+2-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_7__ = (__tilevar_3__[__iter_5__+1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_8__ = (0.083000f * __temp_3__ + 0.083000f * __temp_7__);
        float __temp_12__ = (__tilevar_3__[__iter_5__-1-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_13__ = (__temp_8__ + 0.083000f * __temp_12__);
        float __temp_17__ = (__tilevar_3__[__iter_5__-2-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_18__ = (__temp_13__ + 0.083000f * __temp_17__);
        float __temp_22__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+2-__iter_1__)]);
        float __temp_23__ = (__temp_18__ + 0.083000f * __temp_22__);
        float __temp_27__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__+1-__iter_1__)]);
        float __temp_28__ = (__temp_23__ + 0.083000f * __temp_27__);
        float __temp_32__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-1-__iter_1__)]);
        float __temp_33__ = (__temp_28__ + 0.083000f * __temp_32__);
        float __temp_37__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-2-__iter_1__)]);
        float __temp_38__ = (__temp_33__ + 0.083000f * __temp_37__);
        float __temp_43__ = (__temp_38__ + 0.083000f * t3_1);
        float __temp_48__ = (__temp_43__ + 0.083000f * t3_0);
        float __temp_53__ = (__temp_48__ + 0.083000f * b3_0);
        float __temp_58__ = (__temp_53__ + 0.083000f * b3_1);
        float __temp_62__ = (__tilevar_3__[__iter_5__-__iter_0__+FORMA_BLOCKDIM_X*(__iter_4__-__iter_1__)]);
        float __temp_63__ = (__temp_58__ - 0.996000f * __temp_62__);
        __var_1__[__iter_5__+N*(__iter_4__+M*FORMA_MAX(__iter_2__-2,0))] = __temp_63__;
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int SMemSize = 0;
  SMemSize += sizeof(float)*(2*FORMA_BLOCKDIM_Y*FORMA_BLOCKDIM_X);
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d13pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*(L*M*N), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*(L*M*N));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = N;
  int __size_1___kernel___forma_kernel__0__ = M;
  int __block_0___kernel___forma_kernel__0__ = 32;
  int __block_1___kernel___forma_kernel__0__ = 32;
  int __block_2___kernel___forma_kernel__0__ = 1;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.x-8);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__blockConfig___kernel___forma_kernel__0__.y-8);
  int __grid_2___kernel___forma_kernel__0__ = 1;
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  dim3 unrollConfig (__blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, unrollConfig, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, L, M, N, __blockConfig___kernel___forma_kernel__0__.x, __blockConfig___kernel___forma_kernel__0__.y, __blockConfig___kernel___forma_kernel__0__.z, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");

  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*(L*M*N), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

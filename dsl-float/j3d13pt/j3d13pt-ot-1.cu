#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/*Shared Memory Variable */
extern __shared__ char __FORMA_SHARED_MEM__[];
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __var_2__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (4-(-4));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_1__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-8))*(FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  float* __tilevar_0__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 2;
  int __iter_1__;
  __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 2;
  int __iter_2__;
  __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + 2;
  int __iter_3__;
  __iter_3__ = (FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)+(-2)) + (int)(threadIdx.z) ; 
  if( __iter_3__ <= (FORMA_MIN((FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-3))+2),(L-3))+2) ){
    int __iter_4__;
    __iter_4__ = (FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)+(-2)) + (int)(threadIdx.y) ; 
    if( __iter_4__ <= (FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-3))+2),(M-3))+2) ){
      int __iter_5__;
      __iter_5__ = (FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)+(-2)) + (int)(threadIdx.x) ; 
      if( __iter_5__ <= (FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-3))+2),(N-3))+2) ){
        int __temp_0__;
        __temp_0__ = __iter_3__;
        int __temp_1__;
        __temp_1__ = __iter_4__;
        int __temp_2__;
        __temp_2__ = __iter_5__;
        __tilevar_1__[__iter_5__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_4__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_3__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))] = input[__temp_2__+(N-0)*(__temp_1__+(M-0)*(__temp_0__))];
      }
    }
  }
  __syncthreads();
  int __iter_6__;
  __iter_6__ = FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2) + (int)(threadIdx.z) ; 
  if( __iter_6__ <= FORMA_MIN((FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-3))+2),(L-3)) ){
    int __iter_7__;
    __iter_7__ = FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2) + (int)(threadIdx.y) ; 
    if( __iter_7__ <= FORMA_MIN((FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-3))+2),(M-3)) ){
      int __iter_8__;
      __iter_8__ = FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2) + (int)(threadIdx.x) ; 
      if( __iter_8__ <= FORMA_MIN((FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-3))+2),(N-3)) ){
        float __temp_3__;
        __temp_3__ = (0.083000f * __tilevar_1__[__iter_8__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_4__;
        __temp_4__ = (0.083000f * __tilevar_1__[__iter_8__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_5__;
        __temp_5__ = (__temp_3__ + __temp_4__);
        float __temp_6__;
        __temp_6__ = (0.083000f * __tilevar_1__[__iter_8__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_7__;
        __temp_7__ = (__temp_5__ + __temp_6__);
        float __temp_8__;
        __temp_8__ = (0.083000f * __tilevar_1__[__iter_8__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_9__;
        __temp_9__ = (__temp_7__ + __temp_8__);
        float __temp_10__;
        __temp_10__ = (0.083000f * __tilevar_1__[__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_11__;
        __temp_11__ = (__temp_9__ + __temp_10__);
        float __temp_12__;
        __temp_12__ = (0.083000f * __tilevar_1__[__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_13__;
        __temp_13__ = (__temp_11__ + __temp_12__);
        float __temp_14__;
        __temp_14__ = (0.083000f * __tilevar_1__[__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_15__;
        __temp_15__ = (__temp_13__ + __temp_14__);
        float __temp_16__;
        __temp_16__ = (0.083000f * __tilevar_1__[__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_17__;
        __temp_17__ = (__temp_15__ + __temp_16__);
        float __temp_18__;
        __temp_18__ = (0.083000f * __tilevar_1__[__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_19__;
        __temp_19__ = (__temp_17__ + __temp_18__);
        float __temp_20__;
        __temp_20__ = (0.083000f * __tilevar_1__[__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_21__;
        __temp_21__ = (__temp_19__ + __temp_20__);
        float __temp_22__;
        __temp_22__ = (0.083000f * __tilevar_1__[__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_23__;
        __temp_23__ = (__temp_21__ + __temp_22__);
        float __temp_24__;
        __temp_24__ = (0.083000f * __tilevar_1__[__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_25__;
        __temp_25__ = (__temp_23__ + __temp_24__);
        float __temp_26__;
        __temp_26__ = (0.996000f * __tilevar_1__[__iter_8__+(0-(FORMA_MAX((FORMA_MAX(__iter_0__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_7__+(0-(FORMA_MAX((FORMA_MAX(__iter_1__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_6__+(0-(FORMA_MAX((FORMA_MAX(__iter_2__,2)+(-2)),2)-2))))]);
        float __temp_27__;
        __temp_27__ = (__temp_25__ - __temp_26__);
        __tilevar_0__[__iter_8__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_7__+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_6__+(0-(FORMA_MAX(__iter_2__,2)-2))))] = __temp_27__;
      }
    }
  }
  __syncthreads();
  int __iter_9__;
  __iter_9__ = FORMA_MAX(__iter_2__,2) + (int)(threadIdx.z) ; 
  if( __iter_9__ <= FORMA_MIN(((__iter_2__+FORMA_BLOCKDIM_Z)-1),(L-3)) ){
    int __iter_10__;
    __iter_10__ = FORMA_MAX(__iter_1__,2) + (int)(threadIdx.y) ; 
    if( __iter_10__ <= FORMA_MIN(((__iter_1__+FORMA_BLOCKDIM_Y)-1),(M-3)) ){
      int __iter_11__;
      __iter_11__ = FORMA_MAX(__iter_0__,2) + (int)(threadIdx.x) ; 
      if( __iter_11__ <= FORMA_MIN(((__iter_0__+FORMA_BLOCKDIM_X)-1),(N-3)) ){
        float __temp_28__;
        __temp_28__ = (0.083000f * __tilevar_0__[__iter_11__+(2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_29__;
        __temp_29__ = (0.083000f * __tilevar_0__[__iter_11__+(1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_30__;
        __temp_30__ = (__temp_28__ + __temp_29__);
        float __temp_31__;
        __temp_31__ = (0.083000f * __tilevar_0__[__iter_11__+(-1)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_32__;
        __temp_32__ = (__temp_30__ + __temp_31__);
        float __temp_33__;
        __temp_33__ = (0.083000f * __tilevar_0__[__iter_11__+(-2)+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_34__;
        __temp_34__ = (__temp_32__ + __temp_33__);
        float __temp_35__;
        __temp_35__ = (0.083000f * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(2)+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_36__;
        __temp_36__ = (__temp_34__ + __temp_35__);
        float __temp_37__;
        __temp_37__ = (0.083000f * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(1)+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_38__;
        __temp_38__ = (__temp_36__ + __temp_37__);
        float __temp_39__;
        __temp_39__ = (0.083000f * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(-1)+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_40__;
        __temp_40__ = (__temp_38__ + __temp_39__);
        float __temp_41__;
        __temp_41__ = (0.083000f * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(-2)+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_42__;
        __temp_42__ = (__temp_40__ + __temp_41__);
        float __temp_43__;
        __temp_43__ = (0.083000f * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(2)+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_44__;
        __temp_44__ = (__temp_42__ + __temp_43__);
        float __temp_45__;
        __temp_45__ = (0.083000f * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(1)+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_46__;
        __temp_46__ = (__temp_44__ + __temp_45__);
        float __temp_47__;
        __temp_47__ = (0.083000f * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(-1)+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_48__;
        __temp_48__ = (__temp_46__ + __temp_47__);
        float __temp_49__;
        __temp_49__ = (0.083000f * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(-2)+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_50__;
        __temp_50__ = (__temp_48__ + __temp_49__);
        float __temp_51__;
        __temp_51__ = (0.996000f * __tilevar_0__[__iter_11__+(0-(FORMA_MAX(__iter_0__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_10__+(0-(FORMA_MAX(__iter_1__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_9__+(0-(FORMA_MAX(__iter_2__,2)-2))))]);
        float __temp_52__;
        __temp_52__ = (__temp_50__ - __temp_51__);
        __var_2__[__iter_11__+(N-0)*(__iter_10__+(M-0)*(__iter_9__))] = __temp_52__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__0__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (4-(-4));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-8))*(FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  return SMemSize;
}

__global__ void __kernel___forma_kernel__1__(float * __restrict__ __var_2__, int L, int M, int N, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (4-(-4));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int __FORMA_SHARED_MEM_OFFSET__ = 0;
  float* __tilevar_3__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-8))*(FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  float* __tilevar_2__ = (float*)(__FORMA_SHARED_MEM__+__FORMA_SHARED_MEM_OFFSET__);
  __FORMA_SHARED_MEM_OFFSET__ += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  int __iter_12__;
  __iter_12__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + 2;
  int __iter_13__;
  __iter_13__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + 2;
  int __iter_14__;
  __iter_14__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + 2;
  int __iter_15__;
  __iter_15__ = (FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)+(-2)) + (int)(threadIdx.z) ; 
  if( __iter_15__ <= (FORMA_MIN((FORMA_MIN(((__iter_14__+FORMA_BLOCKDIM_Z)-1),(L-3))+2),(L-3))+2) ){
    int __iter_16__;
    __iter_16__ = (FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)+(-2)) + (int)(threadIdx.y) ; 
    if( __iter_16__ <= (FORMA_MIN((FORMA_MIN(((__iter_13__+FORMA_BLOCKDIM_Y)-1),(M-3))+2),(M-3))+2) ){
      int __iter_17__;
      __iter_17__ = (FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)+(-2)) + (int)(threadIdx.x) ; 
      if( __iter_17__ <= (FORMA_MIN((FORMA_MIN(((__iter_12__+FORMA_BLOCKDIM_X)-1),(N-3))+2),(N-3))+2) ){
        int __temp_53__;
        __temp_53__ = __iter_15__;
        int __temp_54__;
        __temp_54__ = __iter_16__;
        int __temp_55__;
        __temp_55__ = __iter_17__;
        __tilevar_3__[__iter_17__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_16__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_15__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))] = __var_2__[__temp_55__+(N-0)*(__temp_54__+(M-0)*(__temp_53__))];
      }
    }
  }
  __syncthreads();
  int __iter_18__;
  __iter_18__ = FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2) + (int)(threadIdx.z) ; 
  if( __iter_18__ <= FORMA_MIN((FORMA_MIN(((__iter_14__+FORMA_BLOCKDIM_Z)-1),(L-3))+2),(L-3)) ){
    int __iter_19__;
    __iter_19__ = FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2) + (int)(threadIdx.y) ; 
    if( __iter_19__ <= FORMA_MIN((FORMA_MIN(((__iter_13__+FORMA_BLOCKDIM_Y)-1),(M-3))+2),(M-3)) ){
      int __iter_20__;
      __iter_20__ = FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2) + (int)(threadIdx.x) ; 
      if( __iter_20__ <= FORMA_MIN((FORMA_MIN(((__iter_12__+FORMA_BLOCKDIM_X)-1),(N-3))+2),(N-3)) ){
        float __temp_56__;
        __temp_56__ = (0.083000f * __tilevar_3__[__iter_20__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_57__;
        __temp_57__ = (0.083000f * __tilevar_3__[__iter_20__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_58__;
        __temp_58__ = (__temp_56__ + __temp_57__);
        float __temp_59__;
        __temp_59__ = (0.083000f * __tilevar_3__[__iter_20__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_60__;
        __temp_60__ = (__temp_58__ + __temp_59__);
        float __temp_61__;
        __temp_61__ = (0.083000f * __tilevar_3__[__iter_20__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_62__;
        __temp_62__ = (__temp_60__ + __temp_61__);
        float __temp_63__;
        __temp_63__ = (0.083000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_64__;
        __temp_64__ = (__temp_62__ + __temp_63__);
        float __temp_65__;
        __temp_65__ = (0.083000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_66__;
        __temp_66__ = (__temp_64__ + __temp_65__);
        float __temp_67__;
        __temp_67__ = (0.083000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_68__;
        __temp_68__ = (__temp_66__ + __temp_67__);
        float __temp_69__;
        __temp_69__ = (0.083000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_70__;
        __temp_70__ = (__temp_68__ + __temp_69__);
        float __temp_71__;
        __temp_71__ = (0.083000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(2)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_72__;
        __temp_72__ = (__temp_70__ + __temp_71__);
        float __temp_73__;
        __temp_73__ = (0.083000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_74__;
        __temp_74__ = (__temp_72__ + __temp_73__);
        float __temp_75__;
        __temp_75__ = (0.083000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(-1)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_76__;
        __temp_76__ = (__temp_74__ + __temp_75__);
        float __temp_77__;
        __temp_77__ = (0.083000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(-2)+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_78__;
        __temp_78__ = (__temp_76__ + __temp_77__);
        float __temp_79__;
        __temp_79__ = (0.996000f * __tilevar_3__[__iter_20__+(0-(FORMA_MAX((FORMA_MAX(__iter_12__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_X-(-8))*(__iter_19__+(0-(FORMA_MAX((FORMA_MAX(__iter_13__,2)+(-2)),2)-2))+(FORMA_BLOCKDIM_Y-(-8))*(__iter_18__+(0-(FORMA_MAX((FORMA_MAX(__iter_14__,2)+(-2)),2)-2))))]);
        float __temp_80__;
        __temp_80__ = (__temp_78__ - __temp_79__);
        __tilevar_2__[__iter_20__+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_19__+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_18__+(0-(FORMA_MAX(__iter_14__,2)-2))))] = __temp_80__;
      }
    }
  }
  __syncthreads();
  int __iter_21__;
  __iter_21__ = FORMA_MAX(__iter_14__,2) + (int)(threadIdx.z) ; 
  if( __iter_21__ <= FORMA_MIN(((__iter_14__+FORMA_BLOCKDIM_Z)-1),(L-3)) ){
    int __iter_22__;
    __iter_22__ = FORMA_MAX(__iter_13__,2) + (int)(threadIdx.y) ; 
    if( __iter_22__ <= FORMA_MIN(((__iter_13__+FORMA_BLOCKDIM_Y)-1),(M-3)) ){
      int __iter_23__;
      __iter_23__ = FORMA_MAX(__iter_12__,2) + (int)(threadIdx.x) ; 
      if( __iter_23__ <= FORMA_MIN(((__iter_12__+FORMA_BLOCKDIM_X)-1),(N-3)) ){
        float __temp_81__;
        __temp_81__ = (0.083000f * __tilevar_2__[__iter_23__+(2)+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_82__;
        __temp_82__ = (0.083000f * __tilevar_2__[__iter_23__+(1)+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_83__;
        __temp_83__ = (__temp_81__ + __temp_82__);
        float __temp_84__;
        __temp_84__ = (0.083000f * __tilevar_2__[__iter_23__+(-1)+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_85__;
        __temp_85__ = (__temp_83__ + __temp_84__);
        float __temp_86__;
        __temp_86__ = (0.083000f * __tilevar_2__[__iter_23__+(-2)+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_87__;
        __temp_87__ = (__temp_85__ + __temp_86__);
        float __temp_88__;
        __temp_88__ = (0.083000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(2)+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_89__;
        __temp_89__ = (__temp_87__ + __temp_88__);
        float __temp_90__;
        __temp_90__ = (0.083000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(1)+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_91__;
        __temp_91__ = (__temp_89__ + __temp_90__);
        float __temp_92__;
        __temp_92__ = (0.083000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(-1)+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_93__;
        __temp_93__ = (__temp_91__ + __temp_92__);
        float __temp_94__;
        __temp_94__ = (0.083000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(-2)+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_95__;
        __temp_95__ = (__temp_93__ + __temp_94__);
        float __temp_96__;
        __temp_96__ = (0.083000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(2)+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_97__;
        __temp_97__ = (__temp_95__ + __temp_96__);
        float __temp_98__;
        __temp_98__ = (0.083000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(1)+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_99__;
        __temp_99__ = (__temp_97__ + __temp_98__);
        float __temp_100__;
        __temp_100__ = (0.083000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(-1)+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_101__;
        __temp_101__ = (__temp_99__ + __temp_100__);
        float __temp_102__;
        __temp_102__ = (0.083000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(-2)+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_103__;
        __temp_103__ = (__temp_101__ + __temp_102__);
        float __temp_104__;
        __temp_104__ = (0.996000f * __tilevar_2__[__iter_23__+(0-(FORMA_MAX(__iter_12__,2)-2))+(FORMA_BLOCKDIM_X-(-4))*(__iter_22__+(0-(FORMA_MAX(__iter_13__,2)-2))+(FORMA_BLOCKDIM_Y-(-4))*(__iter_21__+(0-(FORMA_MAX(__iter_14__,2)-2))))]);
        float __temp_105__;
        __temp_105__ = (__temp_103__ - __temp_104__);
        __var_1__[__iter_23__+(N-0)*(__iter_22__+(M-0)*(__iter_21__))] = __temp_105__;
      }
    }
  }
}

int __blockSizeToSMemSize___kernel___forma_kernel__1__(dim3 blockDim){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z)- (4-(-4));
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y)- (4-(-4));
  int FORMA_BLOCKDIM_X = (int)(blockDim.x)- (4-(-4));
  int SMemSize = 0;
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-8))*(FORMA_BLOCKDIM_Y-(-8))*(FORMA_BLOCKDIM_X-(-8))));
  SMemSize += sizeof(float)*(((FORMA_BLOCKDIM_Z-(-4))*(FORMA_BLOCKDIM_Y-(-4))*(FORMA_BLOCKDIM_X-(-4))));
  return SMemSize;
}

/*Device code End */
/* Host Code Begin */
extern "C" void j3d13pt (float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
  int __FORMA_MAX_SHARED_MEM__;
  hipDeviceGetAttribute(&__FORMA_MAX_SHARED_MEM__,hipDeviceAttributeMaxSharedMemoryPerBlock,0);
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((N-3) - 2 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((M-3) - 2 ) + 1;
  int __size_2___kernel___forma_kernel__0__ = ((L-3) - 2 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int _max_occupancy_gridsize___kernel___forma_kernel__0__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__0__,&__max_occupancy_blocksize___kernel___forma_kernel__0__,(const void*)__kernel___forma_kernel__0__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__0___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel___forma_kernel__0___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__0___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__0__ = 12;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___0,FORMA_MAX((__size_0___kernel___forma_kernel__0__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_0___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___1 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__0__, (double)(1.0/(double)2));
  int __block_1___kernel___forma_kernel__0__ = 9;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___1,__size_1___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_1),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_1___kernel___forma_kernel__0__;
  int __max_occupancy_blocksize___kernel___forma_kernel__0___2 = __max_occupancy_blocksize___kernel___forma_kernel__0__;
  int __block_2___kernel___forma_kernel__0__ = 9;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__0___2,__size_2___kernel___forma_kernel__0__),FORMA_MAX_BLOCKDIM_2),9);
  __max_occupancy_blocksize___kernel___forma_kernel__0__ /= __block_2___kernel___forma_kernel__0__;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  while( __SMemSize___kernel___forma_kernel__0__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__0__.z/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.z /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.y/2 > 9)
      __blockConfig___kernel___forma_kernel__0__.y /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
    if( __SMemSize___kernel___forma_kernel__0__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__0__.x/2 > FORMA_MIN(32,9))
      __blockConfig___kernel___forma_kernel__0__.x /= 2;
    __SMemSize___kernel___forma_kernel__0__ = __blockSizeToSMemSize___kernel___forma_kernel__0__(__blockConfig___kernel___forma_kernel__0__);
  }
  __block_0___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.x-(4-(-4));
  __block_1___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.y-(4-(-4));
  __block_2___kernel___forma_kernel__0__ = __blockConfig___kernel___forma_kernel__0__.z-(4-(-4));
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __var_2__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");
  int __size_0___kernel___forma_kernel__1__ = ((N-3) - 2 ) + 1;
  int __size_1___kernel___forma_kernel__1__ = ((M-3) - 2 ) + 1;
  int __size_2___kernel___forma_kernel__1__ = ((L-3) - 2 ) + 1;
  int __max_occupancy_blocksize___kernel___forma_kernel__1__;
  int _max_occupancy_gridsize___kernel___forma_kernel__1__;
  hipOccupancyMaxPotentialBlockSize(&_max_occupancy_gridsize___kernel___forma_kernel__1__,&__max_occupancy_blocksize___kernel___forma_kernel__1__,(const void*)__kernel___forma_kernel__1__,0,0);
  int __max_occupancy_blocksize___kernel___forma_kernel__1___0 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__1__, (double)(1.0/(double)3));
  __max_occupancy_blocksize___kernel___forma_kernel__1___0 = FORMA_MAX(__max_occupancy_blocksize___kernel___forma_kernel__1___0/32, 1)*32;
  int __block_0___kernel___forma_kernel__1__ = 12;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__1___0,FORMA_MAX((__size_0___kernel___forma_kernel__1__)/32,1)*32),FORMA_MAX_BLOCKDIM_0),9);
  __max_occupancy_blocksize___kernel___forma_kernel__1__ /= __block_0___kernel___forma_kernel__1__;
  int __max_occupancy_blocksize___kernel___forma_kernel__1___1 = pow((double)__max_occupancy_blocksize___kernel___forma_kernel__1__, (double)(1.0/(double)2));
  int __block_1___kernel___forma_kernel__1__ = 9;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__1___1,__size_1___kernel___forma_kernel__1__),FORMA_MAX_BLOCKDIM_1),9);
  __max_occupancy_blocksize___kernel___forma_kernel__1__ /= __block_1___kernel___forma_kernel__1__;
  int __max_occupancy_blocksize___kernel___forma_kernel__1___2 = __max_occupancy_blocksize___kernel___forma_kernel__1__;
  int __block_2___kernel___forma_kernel__1__ = 9;//FORMA_MAX(FORMA_MIN(FORMA_MIN(__max_occupancy_blocksize___kernel___forma_kernel__1___2,__size_2___kernel___forma_kernel__1__),FORMA_MAX_BLOCKDIM_2),9);
  __max_occupancy_blocksize___kernel___forma_kernel__1__ /= __block_2___kernel___forma_kernel__1__;
  dim3 __blockConfig___kernel___forma_kernel__1__(__block_0___kernel___forma_kernel__1__,__block_1___kernel___forma_kernel__1__,__block_2___kernel___forma_kernel__1__);
  int __SMemSize___kernel___forma_kernel__1__ = 0;
  __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
  while( __SMemSize___kernel___forma_kernel__1__ > __FORMA_MAX_SHARED_MEM__){
    if( __blockConfig___kernel___forma_kernel__1__.z/2 > 9)
      __blockConfig___kernel___forma_kernel__1__.z /= 2;
    __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
    if( __SMemSize___kernel___forma_kernel__1__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__1__.y/2 > 9)
      __blockConfig___kernel___forma_kernel__1__.y /= 2;
    __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
    if( __SMemSize___kernel___forma_kernel__1__ <= __FORMA_MAX_SHARED_MEM__)
      break;
    if( __blockConfig___kernel___forma_kernel__1__.x/2 > FORMA_MIN(32,9))
      __blockConfig___kernel___forma_kernel__1__.x /= 2;
    __SMemSize___kernel___forma_kernel__1__ = __blockSizeToSMemSize___kernel___forma_kernel__1__(__blockConfig___kernel___forma_kernel__1__);
  }
  __block_0___kernel___forma_kernel__1__ = __blockConfig___kernel___forma_kernel__1__.x-(4-(-4));
  __block_1___kernel___forma_kernel__1__ = __blockConfig___kernel___forma_kernel__1__.y-(4-(-4));
  __block_2___kernel___forma_kernel__1__ = __blockConfig___kernel___forma_kernel__1__.z-(4-(-4));
  int __grid_0___kernel___forma_kernel__1__ = FORMA_CEIL(__size_0___kernel___forma_kernel__1__,__block_0___kernel___forma_kernel__1__);
  int __grid_1___kernel___forma_kernel__1__ = FORMA_CEIL(__size_1___kernel___forma_kernel__1__,__block_1___kernel___forma_kernel__1__);
  int __grid_2___kernel___forma_kernel__1__ = FORMA_CEIL(__size_2___kernel___forma_kernel__1__,__block_2___kernel___forma_kernel__1__);
  dim3 __gridConfig___kernel___forma_kernel__1__(__grid_0___kernel___forma_kernel__1__,__grid_1___kernel___forma_kernel__1__,__grid_2___kernel___forma_kernel__1__);
  __kernel___forma_kernel__1__<<<__gridConfig___kernel___forma_kernel__1__, __blockConfig___kernel___forma_kernel__1__, __SMemSize___kernel___forma_kernel__1__>>> (__var_2__, L, M, N, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__1__\n");
  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
}
/*Host Free End*/

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include "stdio.h"

#define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
#define max(a,b) FORMA_MAX(a,b)
#define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
#define min(a,b) FORMA_MIN(a,b)
#define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )

#ifndef FORMA_MAX_BLOCKDIM_0
#define FORMA_MAX_BLOCKDIM_0 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_1
#define FORMA_MAX_BLOCKDIM_1 1024
#endif
#ifndef FORMA_MAX_BLOCKDIM_2
#define FORMA_MAX_BLOCKDIM_2 1024
#endif
template<typename T>
__global__ void  __kernel_init__(T* input, T value)
{
  int loc = (int)(blockIdx.x)*(int)(blockDim.x)+(int)(threadIdx.x);
  input[loc] = value;
}


template<typename T>
void initialize_array(T* d_input, int size, T value)
{
  dim3 init_grid(FORMA_CEIL(size,FORMA_MAX_BLOCKDIM_0));
  dim3 init_block(FORMA_MAX_BLOCKDIM_0);
  __kernel_init__<<<init_grid,init_block>>>(d_input,value);
}


void Check_CUDA_Error(const char* message);
/*Texture references */
/* Device code Begin */
__global__ void __kernel___forma_kernel__0__(float * __restrict__ input, int L, int M, int N, float * __restrict__ __var_4__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_0__;
  __iter_0__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_0__ <= (N-2)){
    int __iter_1__;
    __iter_1__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_1__ <= (M-2)){
      int __iter_2__;
      __iter_2__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_2__ <= (L-2)){
        float __temp_0__;
        __temp_0__ = (input[__iter_0__+(-1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__+(-1)))] + input[__iter_0__+(1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__+(-1)))]);
        float __temp_1__;
        __temp_1__ = (__temp_0__ + input[__iter_0__+(-1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__+(-1)))]);
        float __temp_2__;
        __temp_2__ = (__temp_1__ + input[__iter_0__+(1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__+(-1)))]);
        float __temp_3__;
        __temp_3__ = (0.500000f * __temp_2__);
        float __temp_4__;
        __temp_4__ = (0.510000f * input[__iter_0__+(-1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__))]);
        float __temp_5__;
        __temp_5__ = (__temp_3__ + __temp_4__);
        float __temp_6__;
        __temp_6__ = (0.710000f * input[__iter_0__+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__))]);
        float __temp_7__;
        __temp_7__ = (__temp_5__ + __temp_6__);
        float __temp_8__;
        __temp_8__ = (0.910000f * input[__iter_0__+(1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__))]);
        float __temp_9__;
        __temp_9__ = (__temp_7__ + __temp_8__);
        float __temp_10__;
        __temp_10__ = (1.210000f * input[__iter_0__+(-1)+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
        float __temp_11__;
        __temp_11__ = (__temp_9__ + __temp_10__);
        float __temp_12__;
        __temp_12__ = (1.510000f * input[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
        float __temp_13__;
        __temp_13__ = (__temp_11__ + __temp_12__);
        float __temp_14__;
        __temp_14__ = (1.210000f * input[__iter_0__+(1)+(N-0)*(__iter_1__+(M-0)*(__iter_2__))]);
        float __temp_15__;
        __temp_15__ = (__temp_13__ + __temp_14__);
        float __temp_16__;
        __temp_16__ = (0.910000f * input[__iter_0__+(-1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__))]);
        float __temp_17__;
        __temp_17__ = (__temp_15__ + __temp_16__);
        float __temp_18__;
        __temp_18__ = (0.710000f * input[__iter_0__+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__))]);
        float __temp_19__;
        __temp_19__ = (__temp_17__ + __temp_18__);
        float __temp_20__;
        __temp_20__ = (0.510000f * input[__iter_0__+(1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__))]);
        float __temp_21__;
        __temp_21__ = (__temp_19__ + __temp_20__);
        float __temp_22__;
        __temp_22__ = (input[__iter_0__+(-1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__+(1)))] + input[__iter_0__+(1)+(N-0)*(__iter_1__+(-1)+(M-0)*(__iter_2__+(1)))]);
        float __temp_23__;
        __temp_23__ = (__temp_22__ + input[__iter_0__+(-1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__+(1)))]);
        float __temp_24__;
        __temp_24__ = (__temp_23__ + input[__iter_0__+(1)+(N-0)*(__iter_1__+(1)+(M-0)*(__iter_2__+(1)))]);
        float __temp_25__;
        __temp_25__ = (0.520000f * __temp_24__);
        float __temp_26__;
        __temp_26__ = (__temp_21__ + __temp_25__);
        float __temp_27__;
        __temp_27__ = (__temp_26__ / 159);
        __var_4__[__iter_0__+(N-0)*(__iter_1__+(M-0)*(__iter_2__))] = __temp_27__;
      }
    }
  }
}
__global__ void __kernel___forma_kernel__1__(float * __restrict__ __var_4__, int L, int M, int N, float * __restrict__ __var_3__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_3__;
  __iter_3__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_3__ <= (N-2)){
    int __iter_4__;
    __iter_4__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_4__ <= (M-2)){
      int __iter_5__;
      __iter_5__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_5__ <= (L-2)){
        float __temp_28__;
        __temp_28__ = (__var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__+(-1)))] + __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__+(-1)))]);
        float __temp_29__;
        __temp_29__ = (__temp_28__ + __var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__+(-1)))]);
        float __temp_30__;
        __temp_30__ = (__temp_29__ + __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__+(-1)))]);
        float __temp_31__;
        __temp_31__ = (0.500000f * __temp_30__);
        float __temp_32__;
        __temp_32__ = (0.510000f * __var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__))]);
        float __temp_33__;
        __temp_33__ = (__temp_31__ + __temp_32__);
        float __temp_34__;
        __temp_34__ = (0.710000f * __var_4__[__iter_3__+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__))]);
        float __temp_35__;
        __temp_35__ = (__temp_33__ + __temp_34__);
        float __temp_36__;
        __temp_36__ = (0.910000f * __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__))]);
        float __temp_37__;
        __temp_37__ = (__temp_35__ + __temp_36__);
        float __temp_38__;
        __temp_38__ = (1.210000f * __var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(M-0)*(__iter_5__))]);
        float __temp_39__;
        __temp_39__ = (__temp_37__ + __temp_38__);
        float __temp_40__;
        __temp_40__ = (1.510000f * __var_4__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))]);
        float __temp_41__;
        __temp_41__ = (__temp_39__ + __temp_40__);
        float __temp_42__;
        __temp_42__ = (1.210000f * __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(M-0)*(__iter_5__))]);
        float __temp_43__;
        __temp_43__ = (__temp_41__ + __temp_42__);
        float __temp_44__;
        __temp_44__ = (0.910000f * __var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__))]);
        float __temp_45__;
        __temp_45__ = (__temp_43__ + __temp_44__);
        float __temp_46__;
        __temp_46__ = (0.710000f * __var_4__[__iter_3__+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__))]);
        float __temp_47__;
        __temp_47__ = (__temp_45__ + __temp_46__);
        float __temp_48__;
        __temp_48__ = (0.510000f * __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__))]);
        float __temp_49__;
        __temp_49__ = (__temp_47__ + __temp_48__);
        float __temp_50__;
        __temp_50__ = (__var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__+(1)))] + __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(-1)+(M-0)*(__iter_5__+(1)))]);
        float __temp_51__;
        __temp_51__ = (__temp_50__ + __var_4__[__iter_3__+(-1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__+(1)))]);
        float __temp_52__;
        __temp_52__ = (__temp_51__ + __var_4__[__iter_3__+(1)+(N-0)*(__iter_4__+(1)+(M-0)*(__iter_5__+(1)))]);
        float __temp_53__;
        __temp_53__ = (0.520000f * __temp_52__);
        float __temp_54__;
        __temp_54__ = (__temp_49__ + __temp_53__);
        float __temp_55__;
        __temp_55__ = (__temp_54__ / 159);
        __var_3__[__iter_3__+(N-0)*(__iter_4__+(M-0)*(__iter_5__))] = __temp_55__;
      }
    }
  }
}
__global__ void __kernel___forma_kernel__2__(float * __restrict__ __var_3__, int L, int M, int N, float * __restrict__ __var_2__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_6__;
  __iter_6__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_6__ <= (N-2)){
    int __iter_7__;
    __iter_7__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_7__ <= (M-2)){
      int __iter_8__;
      __iter_8__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_8__ <= (L-2)){
        float __temp_56__;
        __temp_56__ = (__var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__+(-1)))] + __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__+(-1)))]);
        float __temp_57__;
        __temp_57__ = (__temp_56__ + __var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__+(-1)))]);
        float __temp_58__;
        __temp_58__ = (__temp_57__ + __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__+(-1)))]);
        float __temp_59__;
        __temp_59__ = (0.500000f * __temp_58__);
        float __temp_60__;
        __temp_60__ = (0.510000f * __var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__))]);
        float __temp_61__;
        __temp_61__ = (__temp_59__ + __temp_60__);
        float __temp_62__;
        __temp_62__ = (0.710000f * __var_3__[__iter_6__+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__))]);
        float __temp_63__;
        __temp_63__ = (__temp_61__ + __temp_62__);
        float __temp_64__;
        __temp_64__ = (0.910000f * __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__))]);
        float __temp_65__;
        __temp_65__ = (__temp_63__ + __temp_64__);
        float __temp_66__;
        __temp_66__ = (1.210000f * __var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(M-0)*(__iter_8__))]);
        float __temp_67__;
        __temp_67__ = (__temp_65__ + __temp_66__);
        float __temp_68__;
        __temp_68__ = (1.510000f * __var_3__[__iter_6__+(N-0)*(__iter_7__+(M-0)*(__iter_8__))]);
        float __temp_69__;
        __temp_69__ = (__temp_67__ + __temp_68__);
        float __temp_70__;
        __temp_70__ = (1.210000f * __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(M-0)*(__iter_8__))]);
        float __temp_71__;
        __temp_71__ = (__temp_69__ + __temp_70__);
        float __temp_72__;
        __temp_72__ = (0.910000f * __var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__))]);
        float __temp_73__;
        __temp_73__ = (__temp_71__ + __temp_72__);
        float __temp_74__;
        __temp_74__ = (0.710000f * __var_3__[__iter_6__+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__))]);
        float __temp_75__;
        __temp_75__ = (__temp_73__ + __temp_74__);
        float __temp_76__;
        __temp_76__ = (0.510000f * __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__))]);
        float __temp_77__;
        __temp_77__ = (__temp_75__ + __temp_76__);
        float __temp_78__;
        __temp_78__ = (__var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__+(1)))] + __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(-1)+(M-0)*(__iter_8__+(1)))]);
        float __temp_79__;
        __temp_79__ = (__temp_78__ + __var_3__[__iter_6__+(-1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__+(1)))]);
        float __temp_80__;
        __temp_80__ = (__temp_79__ + __var_3__[__iter_6__+(1)+(N-0)*(__iter_7__+(1)+(M-0)*(__iter_8__+(1)))]);
        float __temp_81__;
        __temp_81__ = (0.520000f * __temp_80__);
        float __temp_82__;
        __temp_82__ = (__temp_77__ + __temp_81__);
        float __temp_83__;
        __temp_83__ = (__temp_82__ / 159);
        __var_2__[__iter_6__+(N-0)*(__iter_7__+(M-0)*(__iter_8__))] = __temp_83__;
      }
    }
  }
}
__global__ void __kernel___forma_kernel__3__(float * __restrict__ __var_2__, int L, int M, int N, float * __restrict__ __var_1__){
  int FORMA_BLOCKDIM_Z = (int)(blockDim.z);
  int FORMA_BLOCKDIM_Y = (int)(blockDim.y);
  int FORMA_BLOCKDIM_X = (int)(blockDim.x);
  int __iter_9__;
  __iter_9__ = (int)(blockIdx.x)*(int)(FORMA_BLOCKDIM_X) + (int)(threadIdx.x) + 1;
  if(__iter_9__ <= (N-2)){
    int __iter_10__;
    __iter_10__ = (int)(blockIdx.y)*(int)(FORMA_BLOCKDIM_Y) + (int)(threadIdx.y) + 1;
    if(__iter_10__ <= (M-2)){
      int __iter_11__;
      __iter_11__ = (int)(blockIdx.z)*(int)(FORMA_BLOCKDIM_Z) + (int)(threadIdx.z) + 1;
      if(__iter_11__ <= (L-2)){
        float __temp_84__;
        __temp_84__ = (__var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__+(-1)))] + __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__+(-1)))]);
        float __temp_85__;
        __temp_85__ = (__temp_84__ + __var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__+(-1)))]);
        float __temp_86__;
        __temp_86__ = (__temp_85__ + __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__+(-1)))]);
        float __temp_87__;
        __temp_87__ = (0.500000f * __temp_86__);
        float __temp_88__;
        __temp_88__ = (0.510000f * __var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__))]);
        float __temp_89__;
        __temp_89__ = (__temp_87__ + __temp_88__);
        float __temp_90__;
        __temp_90__ = (0.710000f * __var_2__[__iter_9__+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__))]);
        float __temp_91__;
        __temp_91__ = (__temp_89__ + __temp_90__);
        float __temp_92__;
        __temp_92__ = (0.910000f * __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__))]);
        float __temp_93__;
        __temp_93__ = (__temp_91__ + __temp_92__);
        float __temp_94__;
        __temp_94__ = (1.210000f * __var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(M-0)*(__iter_11__))]);
        float __temp_95__;
        __temp_95__ = (__temp_93__ + __temp_94__);
        float __temp_96__;
        __temp_96__ = (1.510000f * __var_2__[__iter_9__+(N-0)*(__iter_10__+(M-0)*(__iter_11__))]);
        float __temp_97__;
        __temp_97__ = (__temp_95__ + __temp_96__);
        float __temp_98__;
        __temp_98__ = (1.210000f * __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(M-0)*(__iter_11__))]);
        float __temp_99__;
        __temp_99__ = (__temp_97__ + __temp_98__);
        float __temp_100__;
        __temp_100__ = (0.910000f * __var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__))]);
        float __temp_101__;
        __temp_101__ = (__temp_99__ + __temp_100__);
        float __temp_102__;
        __temp_102__ = (0.710000f * __var_2__[__iter_9__+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__))]);
        float __temp_103__;
        __temp_103__ = (__temp_101__ + __temp_102__);
        float __temp_104__;
        __temp_104__ = (0.510000f * __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__))]);
        float __temp_105__;
        __temp_105__ = (__temp_103__ + __temp_104__);
        float __temp_106__;
        __temp_106__ = (__var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__+(1)))] + __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(-1)+(M-0)*(__iter_11__+(1)))]);
        float __temp_107__;
        __temp_107__ = (__temp_106__ + __var_2__[__iter_9__+(-1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__+(1)))]);
        float __temp_108__;
        __temp_108__ = (__temp_107__ + __var_2__[__iter_9__+(1)+(N-0)*(__iter_10__+(1)+(M-0)*(__iter_11__+(1)))]);
        float __temp_109__;
        __temp_109__ = (0.520000f * __temp_108__);
        float __temp_110__;
        __temp_110__ = (__temp_105__ + __temp_109__);
        float __temp_111__;
        __temp_111__ = (__temp_110__ / 159);
        __var_1__[__iter_9__+(N-0)*(__iter_10__+(M-0)*(__iter_11__))] = __temp_111__;
      }
    }
  }
}
/*Device code End */
/* Host Code Begin */
extern "C" void j3d17pt(float * h_input, int L, int M, int N, float * __var_0__){

/* Host allocation Begin */
  float * input;
  hipMalloc(&input,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipPointerAttribute_t ptrAttrib_h_input;
  hipMemcpyKind memcpy_kind_h_input = hipMemcpyHostToDevice;
  if (hipPointerGetAttributes(&ptrAttrib_h_input, h_input) == hipSuccess)
    if (ptrAttrib_h_input.memoryType == hipMemoryTypeDevice)
      memcpy_kind_h_input = hipMemcpyDeviceToDevice;
  hipGetLastError();
  if( memcpy_kind_h_input != hipMemcpyDeviceToDevice ){
    hipMemcpy(input,h_input,sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind_h_input);
  }
  float * __var_1__;
  hipMalloc(&__var_1__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  float * __var_2__;
  hipMalloc(&__var_2__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");
  float * __var_3__;
  hipMalloc(&__var_3__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_3__\n");
  float * __var_4__;
  hipMalloc(&__var_4__,sizeof(float)*((L-0)*(M-0)*(N-0)));
  Check_CUDA_Error("Allocation Error!! : __var_4__\n");
/*Host Allocation End */
/* Kernel Launch Begin */
#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
  int __size_0___kernel___forma_kernel__0__ = ((N-2) - 1 ) + 1;
  int __size_1___kernel___forma_kernel__0__ = ((M-2) - 1 ) + 1;
  int __size_2___kernel___forma_kernel__0__ = ((L-2) - 1 ) + 1;
  int __block_0___kernel___forma_kernel__0__ = 16;
  int __block_1___kernel___forma_kernel__0__ = 4;
  int __block_2___kernel___forma_kernel__0__ = 4;
  dim3 __blockConfig___kernel___forma_kernel__0__(__block_0___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  int __SMemSize___kernel___forma_kernel__0__ = 0;
  int __grid_0___kernel___forma_kernel__0__ = FORMA_CEIL(__size_0___kernel___forma_kernel__0__,__block_0___kernel___forma_kernel__0__);
  int __grid_1___kernel___forma_kernel__0__ = FORMA_CEIL(__size_1___kernel___forma_kernel__0__,__block_1___kernel___forma_kernel__0__);
  int __grid_2___kernel___forma_kernel__0__ = FORMA_CEIL(__size_2___kernel___forma_kernel__0__,__block_2___kernel___forma_kernel__0__);
  dim3 __gridConfig___kernel___forma_kernel__0__(__grid_0___kernel___forma_kernel__0__,__grid_1___kernel___forma_kernel__0__,__grid_2___kernel___forma_kernel__0__);

  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (input, L, M, N, __var_4__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_4__, L, M, N, __var_3__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_3__, L, M, N, __var_2__);
  __kernel___forma_kernel__0__<<<__gridConfig___kernel___forma_kernel__0__, __blockConfig___kernel___forma_kernel__0__, __SMemSize___kernel___forma_kernel__0__>>> (__var_2__, L, M, N, __var_1__);
  Check_CUDA_Error("Kernel Launch Error!! : __kernel___forma_kernel__0__\n");


  hipPointerAttribute_t ptrAttrib___var_0__;
  hipMemcpyKind memcpy_kind___var_0__ = hipMemcpyDeviceToHost;
  if (hipPointerGetAttributes(&ptrAttrib___var_0__, __var_0__) == hipSuccess)
    if (ptrAttrib___var_0__.memoryType == hipMemoryTypeDevice)
      memcpy_kind___var_0__ = hipMemcpyDeviceToDevice;
  hipGetLastError();
  hipMemcpy(__var_0__,__var_1__, sizeof(float)*((L-0)*(M-0)*(N-0)), memcpy_kind___var_0__);
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
  hipFree(__var_3__);
  hipFree(__var_4__);
}
/*Host Free End*/
